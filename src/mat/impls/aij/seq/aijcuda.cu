


#include <../src/mat/impls/aij/seq/aij.h>          /*I "petscmat.h" I*/
PETSC_CUDA_EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqAIJ"
PetscErrorCode MatMult_SeqAIJ(Mat A,Vec xx,Vec yy){
  PetscFunctionBegin;
  hipDeviceSynchronize();
  Mat_SeqAIJ         *a = (Mat_SeqAIJ*)A->data;
  PetscErrorCode     ierr;
  //PetscInt           matsize=A->rmap->n;
  PetscInt           i,n,m,nnz,*rowoffsets,*cindices;
  PetscScalar        *aa;
  hipsparseHandle_t   handle=0;
  hipsparseMatDescr_t descrip=0;
  hipError_t        cs;
  hipsparseStatus_t   csrs=HIPSPARSE_STATUS_SUCCESS;
  
  /* set up cusparse library handle and environment */
  if(hipsparseCreate(&handle)!=HIPSPARSE_STATUS_SUCCESS){
     printf("cusparse handle creation error.\nExiting...\n");
     PetscFunctionReturn(PETSC_ERR_LIB);
  }

  if(hipsparseCreateMatDescr(&descrip)!=HIPSPARSE_STATUS_SUCCESS){
     printf("cusparse matrix descriptor creation error.\nExiting...\n");
     PetscFunctionReturn(PETSC_ERR_LIB);
  }
  hipsparseSetMatType(descrip,HIPSPARSE_MATRIX_TYPE_GENERAL);/* default anyways... */
  hipsparseSetMatIndexBase(descrip,HIPSPARSE_INDEX_BASE_ZERO);

  aa  = a->a;                     /* nonzero elements */
  rowoffsets  = a->i;             /* pointer to beginning of each row */
  cindices = a->j;                /* column indices */
  nnz = a->nz;                    /* nonzeros */

  /* declare and allocate device csr memory and dense vectors x, y */
  ierr = MatGetLocalSize(A,&m,&n);CHKERRQ(ierr);
  int* dev_csrRowOffsets;
  int* dev_csrIndices;
  double* dev_dataA;

  /* Allocate CSR device memory */

  cs=hipMalloc((void**)&dev_csrRowOffsets,(m+1)*sizeof(int));
  if(cs!=hipSuccess)printf("Error1: %s\n",hipGetErrorString(cs));

  cs=hipMalloc((void**)&dev_csrIndices,nnz*sizeof(int));
  if(cs!=hipSuccess)printf("Error2: %s\n",hipGetErrorString(cs));

  cs=hipMalloc((void**)&dev_dataA,nnz*sizeof(double));
  if(cs!=hipSuccess)printf("Error3: %s\n",hipGetErrorString(cs));

  /* Send off data to device */

  cs=hipMemcpy(dev_csrRowOffsets,rowoffsets,(m+1)*sizeof(int),hipMemcpyHostToDevice);
  if(cs!=hipSuccess)printf("Error4: %s\n",hipGetErrorString(cs));

  cs=hipMemcpy(dev_csrIndices,cindices,nnz*sizeof(int),hipMemcpyHostToDevice);
  if(cs!=hipSuccess)printf("Error5: %s\n",hipGetErrorString(cs));

  cs=hipMemcpy(dev_dataA,aa,nnz*sizeof(double),hipMemcpyHostToDevice);
  if(cs!=hipSuccess)printf("Error6: %s\n",hipGetErrorString(cs));

  Vec_SeqGPU *xd=(Vec_SeqGPU*)xx->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)yy->data;

  /*if(yd->syncState == VEC_GPU || yd->syncState == VEC_SYNCHED){
    cs=hipMemcpy(yd->cpuptr,yd->devptr,yy->map->n*sizeof(double),hipMemcpyDeviceToHost);
    if(cs!=hipSuccess)printf("Error7: %s\n",hipGetErrorString(cs));
    }else */
  if(yd->syncState == VEC_CPU){
    cs=hipMemcpy(yd->devptr,yd->cpuptr,yy->map->n*sizeof(double),hipMemcpyHostToDevice);
    if(cs!=hipSuccess)printf("Error8: %s\n",hipGetErrorString(cs));
  }
  //for(i=0;i<yy->map->n;i++){
  //   if(yd->cpuptr[i]!=0.)printf("preMM Y[%d]: %e\n",i,yd->cpuptr[i]);
  //}
  //ierr = VecCheck_SeqGPU(yy);CHKERRQ(ierr);

  /*if(xd->syncState == VEC_GPU || xd->syncState == VEC_SYNCHED){
    cs=hipMemcpy(xd->cpuptr,xd->devptr,xx->map->n*sizeof(double),hipMemcpyDeviceToHost);
    if(cs!=hipSuccess)printf("Error9: %s\n",hipGetErrorString(cs));
    }else*/
   if(xd->syncState == VEC_CPU){
    cs=hipMemcpy(xd->devptr,xd->cpuptr,xx->map->n*sizeof(double),hipMemcpyHostToDevice);
    if(cs!=hipSuccess)printf("Error10: %s\n",hipGetErrorString(cs));
  }
  // for(i=0;i<xx->map->n;i++){
  //    if(xd->cpuptr[i]!=0.)printf("preMM X[%d]: %e\n",i,xd->cpuptr[i]);
  //}
  hipDeviceSynchronize();
  csrs=hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,
                      m,n,1.0,descrip,
		      dev_dataA,dev_csrRowOffsets,dev_csrIndices,
		      xd->devptr,0.,yd->devptr);

  //printf("Error code %d returned from hipsparseDcsrmv call: ",csrs);
  hipDeviceSynchronize();
  if(csrs!=HIPSPARSE_STATUS_SUCCESS) {
    printf("SpMV cuspare lib failed.\n");
    PetscFunctionReturn(PETSC_ERR_LIB);
  }
  yd->syncState = VEC_GPU;

  if(dev_csrRowOffsets)hipFree(dev_csrRowOffsets);
  if(dev_csrIndices)hipFree(dev_csrIndices);
  if(dev_dataA)hipFree(dev_dataA);
  //hipDeviceSynchronize();

  //cs=hipMemcpy(yd->cpuptr,yd->devptr,yy->map->n*sizeof(double),hipMemcpyDeviceToHost);
  //if(cs!=hipSuccess)printf("Error11: %s\n",hipGetErrorString(cs));

  //for(i=0;i<yy->map->n;i++){
  //   if(yd->cpuptr[i]!=0.)printf("postMM Y[%d]: %e\n",i,yd->cpuptr[i]);
  //}

  PetscFunctionReturn(0);
}
PETSC_CUDA_EXTERN_C_END


