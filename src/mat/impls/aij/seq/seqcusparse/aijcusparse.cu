/*
    Defines the basic matrix operations for the AIJ (compressed row)
  matrix storage format.
*/

#include "petscconf.h"
PETSC_CUDA_EXTERN_C_BEGIN
#include "../src/mat/impls/aij/seq/aij.h"          /*I "petscmat.h" I*/
//#include "petscbt.h"
#include "../src/vec/vec/impls/dvecimpl.h"
#include "petsc-private/vecimpl.h"
PETSC_CUDA_EXTERN_C_END
#undef VecType
#include "cusparsematimpl.h"
const char * const MatCUSPARSEStorageFormats[] = {"CSR","ELL","HYB","MatCUSPARSEStorageFormat","MAT_CUSPARSE_",0};

/* this is such a hack ... but I don't know of another way to pass this variable
   from one GPU_Matrix_Ifc class to another. This is necessary for the parallel
   SpMV. Essentially, I need to use the same stream variable in two different
   data structures. I do this by creating a single instance of that stream
   and reuse it. */
hipStream_t theBodyStream=0;

PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat,Mat,IS,IS,const MatFactorInfo*);
PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat,Mat,const MatFactorInfo *);
PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat,Vec,Vec);
PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat,Vec,Vec);
PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(Mat);
PetscErrorCode MatSeqAIJCUSPARSEAnalysisAndCopyToGPU(Mat);
PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat,Vec,Vec);
PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);
PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat,Vec,Vec);
PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat,Vec,Vec,Vec);

#undef __FUNCT__  
#define __FUNCT__ "MatFactorGetSolverPackage_seqaij_cusparse"
PetscErrorCode MatFactorGetSolverPackage_seqaij_cusparse(Mat A,const MatSolverPackage *type)
{
  PetscFunctionBegin;
  *type = MATSOLVERCUSPARSE;
  PetscFunctionReturn(0);
}

EXTERN_C_BEGIN
extern PetscErrorCode MatGetFactor_seqaij_petsc(Mat,MatFactorType,Mat*);
EXTERN_C_END
/*
  MATSOLVERCUSPARSE = "cusparse" - A matrix type providing triangular solvers (ILU) for seq matrices 
  on the GPU of type, seqaijcusparse, aijcusparse, or seqaijcusp, aijcusp

   Level: beginner
*/

EXTERN_C_BEGIN
#undef __FUNCT__  
#define __FUNCT__ "MatGetFactor_seqaij_cusparse"
PetscErrorCode MatGetFactor_seqaij_cusparse(Mat A,MatFactorType ftype,Mat *B)
{
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = MatGetFactor_seqaij_petsc(A,ftype,B);CHKERRQ(ierr);
  if (ftype == MAT_FACTOR_LU || ftype == MAT_FACTOR_ILU || ftype == MAT_FACTOR_ILUDT){
    ierr = MatSetType(*B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
    ierr = MatSetFromOptions_SeqAIJCUSPARSE(*B);CHKERRQ(ierr);
    ierr = PetscObjectComposeFunctionDynamic((PetscObject)(*B),"MatFactorGetSolverPackage_C","MatFactorGetSolverPackage_seqaij_cusparse",MatFactorGetSolverPackage_seqaij_cusparse);CHKERRQ(ierr);
    (*B)->ops->ilufactorsymbolic = MatILUFactorSymbolic_SeqAIJCUSPARSE;
    (*B)->ops->lufactorsymbolic  = MatLUFactorSymbolic_SeqAIJCUSPARSE;
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Factor type not supported for CUSPARSE Matrix Types");
  (*B)->factortype = ftype;
  PetscFunctionReturn(0);
}
EXTERN_C_END

EXTERN_C_BEGIN
#undef __FUNCT__  
#define __FUNCT__ "MatCUSPARSESetFormat_SeqAIJCUSPARSE"
PetscErrorCode MatCUSPARSESetFormat_SeqAIJCUSPARSE(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  Mat_SeqAIJCUSPARSE *cusparseMat  = (Mat_SeqAIJCUSPARSE*)A->spptr; 
  PetscFunctionBegin;  
  switch (op) {
  case MAT_CUSPARSE_MULT:
    cusparseMat->format = format;
    break;
  case MAT_CUSPARSE_SOLVE:
    cusparseMatSolveStorageFormat = format;
    break;
  case MAT_CUSPARSE_ALL:
    cusparseMat->format = format;
    cusparseMatSolveStorageFormat = format;
    break;
  default:
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"unsupported operation %d for MatCUSPARSEFormatOperation. MAT_CUSPARSE_MULT, MAT_CUSPARSE_SOLVE, and MAT_CUSPARSE_ALL are currently supported.",op);
  }
  PetscFunctionReturn(0);  
}
EXTERN_C_END


/*@
   MatCUSPARSESetFormat - Sets the storage format of CUSPARSE matrices for a particular
   operation. Only the MatMult operation can use different GPU storage formats
   for AIJCUSPARSE matrices. This requires the txpetscgpu package. Use --download-txpetscgpu 
   to build/install PETSc to use this package.

   Not Collective

   Input Parameters:
+  A : Matrix of type SEQAIJCUSPARSE
.  op : MatCUSPARSEFormatOperation. SEQAIJCUSPARSE matrices support MAT_CUSPARSE_MULT, MAT_CUSPARSE_SOLVE, and MAT_CUSPARSE_ALL. MPIAIJCUSPARSE matrices support MAT_CUSPARSE_MULT_DIAG, MAT_CUSPARSE_MULT_OFFDIAG, and MAT_CUSPARSE_ALL.
-  format : MatCUSPARSEStorageFormat (one of MAT_CUSPARSE_CSR, MAT_CUSPARSE_ELL, MAT_CUSPARSE_HYB)

   Output Parameter:

   Level: intermediate

.seealso: MatCUSPARSEStorageFormat, MatCUSPARSEARSEFormatOperation
@*/
#undef __FUNCT__
#define __FUNCT__ "MatCUSPARSESetFormat"
PetscErrorCode MatCUSPARSESetFormat(Mat A,MatCUSPARSEFormatOperation op,MatCUSPARSEStorageFormat format)
{
  PetscErrorCode ierr;
  PetscFunctionBegin;
  PetscValidHeaderSpecific(A, MAT_CLASSID,1);
  ierr = PetscTryMethod(A, "MatCUSPARSESetFormat_C",(Mat,MatCUSPARSEFormatOperation,MatCUSPARSEStorageFormat),(A,op,format));CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "MatSetFromOptions_SeqAIJCUSPARSE"
PetscErrorCode MatSetFromOptions_SeqAIJCUSPARSE(Mat A)
{
  PetscErrorCode     ierr;
  MatCUSPARSEStorageFormat format;
  PetscBool      flg;
  PetscFunctionBegin;
  ierr = PetscOptionsHead("SeqAIJCUSPARSE options");CHKERRQ(ierr);
  ierr = PetscObjectOptionsBegin((PetscObject)A);
  if (A->factortype==MAT_FACTOR_NONE) {
    ierr = PetscOptionsEnum("-mat_cusparse_mult_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV",
			    "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)MAT_CUSPARSE_CSR,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_MULT,format);CHKERRQ(ierr);
    }
  }
  else { 
    ierr = PetscOptionsEnum("-mat_cusparse_solve_storage_format","sets storage format of (seq)aijcusparse gpu matrices for TriSolve",
			    "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)MAT_CUSPARSE_CSR,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
    if (flg) {
      ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_SOLVE,format);CHKERRQ(ierr);
    }
  }
  ierr = PetscOptionsEnum("-mat_cusparse_storage_format","sets storage format of (seq)aijcusparse gpu matrices for SpMV and TriSolve",
                          "MatCUSPARSESetFormat",MatCUSPARSEStorageFormats,(PetscEnum)MAT_CUSPARSE_CSR,(PetscEnum*)&format,&flg);CHKERRQ(ierr);
  if (flg) {
    ierr = MatCUSPARSESetFormat(A,MAT_CUSPARSE_ALL,format);CHKERRQ(ierr);
  }
  ierr = PetscOptionsEnd();CHKERRQ(ierr);
  PetscFunctionReturn(0);

}

#undef __FUNCT__  
#define __FUNCT__ "MatILUFactorSymbolic_SeqAIJCUSPARSE"
PetscErrorCode MatILUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = MatILUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info);CHKERRQ(ierr);
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "MatLUFactorSymbolic_SeqAIJCUSPARSE"
PetscErrorCode MatLUFactorSymbolic_SeqAIJCUSPARSE(Mat B,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  ierr = MatLUFactorSymbolic_SeqAIJ(B,A,isrow,iscol,info);CHKERRQ(ierr);
  B->ops->lufactornumeric = MatLUFactorNumeric_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "MatSeqAIJCUSPARSEBuildLowerTriMatrix"
PetscErrorCode MatSeqAIJCUSPARSEBuildLowerTriMatrix(Mat A)
{
  Mat_SeqAIJ        *a = (Mat_SeqAIJ*)A->data;
  PetscInt          n = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors  = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc* cusparseMat  = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
  hipsparseStatus_t stat;
  const PetscInt    *ai = a->i,*aj = a->j,*vi;
  const MatScalar   *aa = a->a,*v;
  PetscErrorCode     ierr;
  PetscInt *AiLo, *AjLo;
  PetscScalar *AALo;
  PetscInt i,nz, nzLower, offset, rowOffset;
  
  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU){	
    try {	
      /* first figure out the number of nonzeros in the lower triangular matrix including 1's on the diagonal. */
      nzLower=n+ai[n]-ai[1];
      
      /* Allocate Space for the lower triangular matrix */	
      ierr = hipHostMalloc((void **) &AiLo, (n+1)*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void **) &AjLo, nzLower*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void **) &AALo, nzLower*sizeof(PetscScalar));CHKERRCUSP(ierr);
      
      /* Fill the lower triangular matrix */
      AiLo[0]=(PetscInt) 0;
      AiLo[n]=nzLower;
      AjLo[0]=(PetscInt) 0;
      AALo[0]=(MatScalar) 1.0;
      v    = aa;
      vi   = aj;
      offset=1;
      rowOffset=1;
      for (i=1; i<n; i++) {
	nz  = ai[i+1] - ai[i];
	/* additional 1 for the term on the diagonal */
	AiLo[i]=rowOffset;
	rowOffset+=nz+1;

	ierr = PetscMemcpy(&(AjLo[offset]), vi, nz*sizeof(PetscInt));CHKERRQ(ierr);
	ierr = PetscMemcpy(&(AALo[offset]), v, nz*sizeof(PetscScalar));CHKERRQ(ierr);
	
	offset+=nz;
	AjLo[offset]=(PetscInt) i;
	AALo[offset]=(MatScalar) 1.0;
	offset+=1;
	
	v  += nz;
	vi += nz;
      }    
      cusparseMat = GPU_Matrix_Factory::getNew(MatCUSPARSEStorageFormats[cusparseTriFactors->format]);
      stat = cusparseMat->initializeCusparse(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_TRIANGULAR, HIPSPARSE_FILL_MODE_LOWER);CHKERRCUSP(stat);
      ierr = cusparseMat->setMatrix(n, n, nzLower, AiLo, AjLo, AALo);CHKERRCUSP(ierr);
      stat = cusparseMat->solveAnalysis();CHKERRCUSP(stat);
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->loTriFactorPtr = cusparseMat;
      ierr = hipHostFree(AiLo);CHKERRCUSP(ierr);
      ierr = hipHostFree(AjLo);CHKERRCUSP(ierr);
      ierr = hipHostFree(AALo);CHKERRCUSP(ierr);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);	  
}

#undef __FUNCT__  
#define __FUNCT__ "MatSeqAIJCUSPARSEBuildUpperTriMatrix"
PetscErrorCode MatSeqAIJCUSPARSEBuildUpperTriMatrix(Mat A)
{
  Mat_SeqAIJ        *a = (Mat_SeqAIJ*)A->data;
  PetscInt          n = A->rmap->n;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors  = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc* cusparseMat  = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
  hipsparseStatus_t stat;
  const PetscInt    *aj = a->j,*adiag = a->diag,*vi;
  const MatScalar   *aa = a->a,*v;
  PetscInt *AiUp, *AjUp;
  PetscScalar *AAUp;
  PetscInt i,nz, nzUpper, offset;
  PetscErrorCode     ierr;
  
  PetscFunctionBegin;

  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU){	
    try {	
      /* next, figure out the number of nonzeros in the upper triangular matrix. */
      nzUpper = adiag[0]-adiag[n];
      
      /* Allocate Space for the upper triangular matrix */
      ierr = hipHostMalloc((void **) &AiUp, (n+1)*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void **) &AjUp, nzUpper*sizeof(PetscInt));CHKERRCUSP(ierr);
      ierr = hipHostMalloc((void **) &AAUp, nzUpper*sizeof(PetscScalar));CHKERRCUSP(ierr);
      
      /* Fill the upper triangular matrix */
      AiUp[0]=(PetscInt) 0;
      AiUp[n]=nzUpper;
      offset = nzUpper;
      for (i=n-1; i>=0; i--){
	v   = aa + adiag[i+1] + 1;
	vi  = aj + adiag[i+1] + 1;
	
	/* number of elements NOT on the diagonal */
	nz = adiag[i] - adiag[i+1]-1;
	
	/* decrement the offset */
	offset -= (nz+1);
	
	/* first, set the diagonal elements */
	AjUp[offset] = (PetscInt) i;
	AAUp[offset] = 1./v[nz];
	AiUp[i] = AiUp[i+1] - (nz+1);
	
	ierr = PetscMemcpy(&(AjUp[offset+1]), vi, nz*sizeof(PetscInt));CHKERRQ(ierr);
	ierr = PetscMemcpy(&(AAUp[offset+1]), v, nz*sizeof(PetscScalar));CHKERRQ(ierr);
      }      
      cusparseMat = GPU_Matrix_Factory::getNew(MatCUSPARSEStorageFormats[cusparseTriFactors->format]);
      stat = cusparseMat->initializeCusparse(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_TRIANGULAR, HIPSPARSE_FILL_MODE_UPPER);CHKERRCUSP(stat);
      ierr = cusparseMat->setMatrix(n, n, nzUpper, AiUp, AjUp, AAUp);CHKERRCUSP(ierr);
      stat = cusparseMat->solveAnalysis();CHKERRCUSP(stat);
      ((Mat_SeqAIJCUSPARSETriFactors*)A->spptr)->upTriFactorPtr = cusparseMat;
      ierr = hipHostFree(AiUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AjUp);CHKERRCUSP(ierr);
      ierr = hipHostFree(AAUp);CHKERRCUSP(ierr);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    }
  }
  PetscFunctionReturn(0);	  
}

#undef __FUNCT__  
#define __FUNCT__ "MatSeqAIJCUSPARSEAnalysisAndCopyToGPU"
PetscErrorCode MatSeqAIJCUSPARSEAnalysisAndCopyToGPU(Mat A)
{
  PetscErrorCode     ierr;
  Mat_SeqAIJ *a=(Mat_SeqAIJ *)A->data;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors  = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  IS               isrow = a->row,iscol = a->icol;
  PetscBool        row_identity,col_identity;
  const PetscInt   *r,*c;
  PetscInt          n = A->rmap->n;

  PetscFunctionBegin;       
  ierr = MatSeqAIJCUSPARSEBuildLowerTriMatrix(A);CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSEBuildUpperTriMatrix(A);CHKERRQ(ierr);
  cusparseTriFactors->tempvec = new CUSPARRAY;
  cusparseTriFactors->tempvec->resize(n);

  A->valid_GPU_matrix = PETSC_CUSP_BOTH;
  /*lower triangular indices */
  ierr = ISGetIndices(isrow,&r);CHKERRQ(ierr);
  ierr = ISIdentity(isrow,&row_identity);CHKERRQ(ierr);
  if (!row_identity)     
    ierr = cusparseTriFactors->loTriFactorPtr->setOrdIndices(r, n);CHKERRCUSP(ierr);
  ierr = ISRestoreIndices(isrow,&r);CHKERRQ(ierr);

  /*upper triangular indices */
  ierr = ISGetIndices(iscol,&c);CHKERRQ(ierr);
  ierr = ISIdentity(iscol,&col_identity);CHKERRQ(ierr);
  if (!col_identity)
    ierr = cusparseTriFactors->upTriFactorPtr->setOrdIndices(c, n);CHKERRCUSP(ierr);
  ierr = ISRestoreIndices(iscol,&c);CHKERRQ(ierr);
  PetscFunctionReturn(0);	  
}

#undef __FUNCT__  
#define __FUNCT__ "MatLUFactorNumeric_SeqAIJCUSPARSE"
PetscErrorCode MatLUFactorNumeric_SeqAIJCUSPARSE(Mat B,Mat A,const MatFactorInfo *info)
{
  PetscErrorCode   ierr;
  Mat_SeqAIJ       *b=(Mat_SeqAIJ *)B->data;
  IS               isrow = b->row,iscol = b->col;
  PetscBool        row_identity,col_identity;

  PetscFunctionBegin;
  ierr = MatLUFactorNumeric_SeqAIJ(B,A,info);CHKERRQ(ierr);
  /* determine which version of MatSolve needs to be used. */
  ierr = ISIdentity(isrow,&row_identity);CHKERRQ(ierr);
  ierr = ISIdentity(iscol,&col_identity);CHKERRQ(ierr);
  if (row_identity && col_identity) B->ops->solve = MatSolve_SeqAIJCUSPARSE_NaturalOrdering;    
  else                              B->ops->solve = MatSolve_SeqAIJCUSPARSE; 

  /* get the triangular factors */
  ierr = MatSeqAIJCUSPARSEAnalysisAndCopyToGPU(B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}



#undef __FUNCT__  
#define __FUNCT__ "MatSolve_SeqAIJCUSPARSE"
PetscErrorCode MatSolve_SeqAIJCUSPARSE(Mat A,Vec bb,Vec xx)
{
  Mat_SeqAIJ     *a = (Mat_SeqAIJ*)A->data;
  PetscErrorCode ierr;
  CUSPARRAY      *xGPU, *bGPU;
  hipsparseStatus_t stat;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors  = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc *cusparseMatLo  = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
  GPU_Matrix_Ifc *cusparseMatUp  = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
  CUSPARRAY * tempGPU = (CUSPARRAY*) cusparseTriFactors->tempvec;

  PetscFunctionBegin;
  /* Get the GPU pointers */
  ierr = VecCUSPGetArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(bb,&bGPU);CHKERRQ(ierr);

  /* solve with reordering */
  ierr = cusparseMatLo->reorderIn(xGPU, bGPU);CHKERRCUSP(ierr);
  stat = cusparseMatLo->solve(xGPU, tempGPU);CHKERRCUSP(stat);
  stat = cusparseMatUp->solve(tempGPU, xGPU);CHKERRCUSP(stat);
  ierr = cusparseMatUp->reorderOut(xGPU);CHKERRCUSP(ierr);
  	  	  
  ierr = VecCUSPRestoreArrayRead(bb,&bGPU);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(2.0*a->nz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}




#undef __FUNCT__  
#define __FUNCT__ "MatSolve_SeqAIJCUSPARSE_NaturalOrdering"
PetscErrorCode MatSolve_SeqAIJCUSPARSE_NaturalOrdering(Mat A,Vec bb,Vec xx)
{
  Mat_SeqAIJ        *a = (Mat_SeqAIJ*)A->data;
  PetscErrorCode    ierr;
  CUSPARRAY         *xGPU, *bGPU;
  hipsparseStatus_t stat;
  Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors  = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
  GPU_Matrix_Ifc *cusparseMatLo  = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
  GPU_Matrix_Ifc *cusparseMatUp  = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
  CUSPARRAY * tempGPU = (CUSPARRAY*) cusparseTriFactors->tempvec;

  PetscFunctionBegin;
  /* Get the GPU pointers */
  ierr = VecCUSPGetArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(bb,&bGPU);CHKERRQ(ierr);

  /* solve */
  stat = cusparseMatLo->solve(bGPU, tempGPU);CHKERRCUSP(stat);
  stat = cusparseMatUp->solve(tempGPU, xGPU);CHKERRCUSP(stat);

  ierr = VecCUSPRestoreArrayRead(bb,&bGPU);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(xx,&xGPU);CHKERRQ(ierr);
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(2.0*a->nz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqAIJCUSPARSECopyToGPU"
PetscErrorCode MatSeqAIJCUSPARSECopyToGPU(Mat A)
{

  Mat_SeqAIJCUSPARSE *cusparseMat  = (Mat_SeqAIJCUSPARSE*)A->spptr;
  Mat_SeqAIJ      *a          = (Mat_SeqAIJ*)A->data;
  PetscInt        m           = A->rmap->n,*ii,*ridx;
  PetscErrorCode  ierr;


  PetscFunctionBegin;
  if (A->valid_GPU_matrix == PETSC_CUSP_UNALLOCATED || A->valid_GPU_matrix == PETSC_CUSP_CPU){    
    ierr = PetscLogEventBegin(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
    /*
      It may be possible to reuse nonzero structure with new matrix values but 
      for simplicity and insured correctness we delete and build a new matrix on
      the GPU. Likely a very small performance hit.
    */
    if (cusparseMat->mat){
      try {
	delete cusparseMat->mat;
	if (cusparseMat->tempvec)
	  delete cusparseMat->tempvec;
	
      } catch(char* ex) {
	SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
      } 
    }
    try {
      cusparseMat->nonzerorow=0;
      for (int j = 0; j<m; j++)
	cusparseMat->nonzerorow += ((a->i[j+1]-a->i[j])>0);

      if (a->compressedrow.use) {	
	m    = a->compressedrow.nrows;
	ii   = a->compressedrow.i;
	ridx = a->compressedrow.rindex;
      } else {
	/* Forcing compressed row on the GPU ... only relevant for CSR storage */
	int k=0;
	ierr = PetscMalloc((cusparseMat->nonzerorow+1)*sizeof(PetscInt), &ii);CHKERRQ(ierr);
	ierr = PetscMalloc((cusparseMat->nonzerorow)*sizeof(PetscInt), &ridx);CHKERRQ(ierr);
	ii[0]=0;
	for (int j = 0; j<m; j++) {
	  if ((a->i[j+1]-a->i[j])>0) {
	    ii[k] = a->i[j];
	    ridx[k]= j;
	    k++;
	  }
	}
	ii[cusparseMat->nonzerorow] = a->nz;
	m = cusparseMat->nonzerorow;
      }

      /* Build our matrix ... first determine the GPU storage type */
      cusparseMat->mat = GPU_Matrix_Factory::getNew(MatCUSPARSEStorageFormats[cusparseMat->format]);

      /* Create the streams and events (if desired).  */
      PetscMPIInt    size;
      ierr = MPI_Comm_size(PETSC_COMM_WORLD,&size);CHKERRQ(ierr);
      ierr = cusparseMat->mat->buildStreamsAndEvents(size, &theBodyStream);CHKERRCUSP(ierr);	

      /* FILL MODE UPPER is irrelevant */
      hipsparseStatus_t stat = cusparseMat->mat->initializeCusparse(MAT_cusparseHandle, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_FILL_MODE_UPPER);CHKERRCUSP(stat);
      
      /* lastly, build the matrix */
      ierr = cusparseMat->mat->setMatrix(m, A->cmap->n, a->nz, ii, a->j, a->a);CHKERRCUSP(ierr);
      cusparseMat->mat->setCPRowIndices(ridx, m);
      if (!a->compressedrow.use) {	
	ierr = PetscFree(ii);CHKERRQ(ierr);
	ierr = PetscFree(ridx);CHKERRQ(ierr);
      }
      cusparseMat->tempvec = new CUSPARRAY;
      cusparseMat->tempvec->resize(m);
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    } 
    ierr = WaitForGPU();CHKERRCUSP(ierr);
    A->valid_GPU_matrix = PETSC_CUSP_BOTH;
    ierr = PetscLogEventEnd(MAT_CUSPARSECopyToGPU,A,0,0,0);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatGetVecs_SeqAIJCUSPARSE"
PetscErrorCode MatGetVecs_SeqAIJCUSPARSE(Mat mat, Vec *right, Vec *left)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;

  if (right) {
    ierr = VecCreate(((PetscObject)mat)->comm,right);CHKERRQ(ierr);
    ierr = VecSetSizes(*right,mat->cmap->n,PETSC_DETERMINE);CHKERRQ(ierr);
    ierr = VecSetBlockSize(*right,mat->rmap->bs);CHKERRQ(ierr);
    ierr = VecSetType(*right,VECSEQCUSP);CHKERRQ(ierr);
    ierr = PetscLayoutReference(mat->cmap,&(*right)->map);CHKERRQ(ierr);
  }
  if (left) {
    ierr = VecCreate(((PetscObject)mat)->comm,left);CHKERRQ(ierr);
    ierr = VecSetSizes(*left,mat->rmap->n,PETSC_DETERMINE);CHKERRQ(ierr);
    ierr = VecSetBlockSize(*left,mat->rmap->bs);CHKERRQ(ierr);
    ierr = VecSetType(*left,VECSEQCUSP);CHKERRQ(ierr);
    ierr = PetscLayoutReference(mat->rmap,&(*left)->map);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "MatMult_SeqAIJCUSPARSE"
PetscErrorCode MatMult_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  Mat_SeqAIJ     *a = (Mat_SeqAIJ*)A->data;
  PetscErrorCode ierr;
  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE *)A->spptr;
  CUSPARRAY      *xarray,*yarray;

  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayWrite(yy,&yarray);CHKERRQ(ierr);
  try {
    ierr = cusparseMat->mat->multiply(xarray, yarray);CHKERRCUSP(ierr);
  } catch (char* ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(yy,&yarray);CHKERRQ(ierr);
  if (!cusparseMat->mat->hasNonZeroStream()) {
    ierr = WaitForGPU();CHKERRCUSP(ierr);
  }
  ierr = PetscLogFlops(2.0*a->nz - cusparseMat->nonzerorow);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "MatMultTranspose_SeqAIJCUSPARSE"
PetscErrorCode MatMultTranspose_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy)
{
  Mat_SeqAIJ     *a = (Mat_SeqAIJ*)A->data;
  PetscErrorCode ierr;
  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE *)A->spptr;
  CUSPARRAY      *xarray,*yarray;

  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPGetArrayWrite(yy,&yarray);CHKERRQ(ierr);
  try {
#if !defined(PETSC_USE_COMPLEX)
    ierr = cusparseMat->mat->multiply(xarray, yarray, TRANSPOSE);CHKERRCUSP(ierr);
#else
    ierr = cusparseMat->mat->multiply(xarray, yarray, HERMITIAN);CHKERRCUSP(ierr);
#endif
  } catch (char* ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(yy,&yarray);CHKERRQ(ierr);
  if (!cusparseMat->mat->hasNonZeroStream()) {
    ierr = WaitForGPU();CHKERRCUSP(ierr);
  }
  ierr = PetscLogFlops(2.0*a->nz - cusparseMat->nonzerorow);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "MatMultAdd_SeqAIJCUSPARSE"
PetscErrorCode MatMultAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  Mat_SeqAIJ     *a = (Mat_SeqAIJ*)A->data;
  PetscErrorCode ierr;
  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE *)A->spptr;
  CUSPARRAY      *xarray,*yarray,*zarray;
  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  try {      
    ierr = VecCopy_SeqCUSP(yy,zz);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayWrite(zz,&zarray);CHKERRQ(ierr);

    /* multiply add */
    ierr = cusparseMat->mat->multiplyAdd(xarray, zarray);CHKERRCUSP(ierr);

    ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(zz,&zarray);CHKERRQ(ierr);
    
  } catch(char* ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(2.0*a->nz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "MatMultAdd_SeqAIJCUSPARSE"
PetscErrorCode MatMultTransposeAdd_SeqAIJCUSPARSE(Mat A,Vec xx,Vec yy,Vec zz)
{
  Mat_SeqAIJ     *a = (Mat_SeqAIJ*)A->data;
  PetscErrorCode ierr;
  Mat_SeqAIJCUSPARSE *cusparseMat = (Mat_SeqAIJCUSPARSE *)A->spptr;
  CUSPARRAY      *xarray,*yarray,*zarray;
  PetscFunctionBegin;
  /* The line below should not be necessary as it has been moved to MatAssemblyEnd_SeqAIJCUSPARSE
     ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr); */
  try {      
    ierr = VecCopy_SeqCUSP(yy,zz);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayWrite(zz,&zarray);CHKERRQ(ierr);

    /* multiply add with matrix transpose */
#if !defined(PETSC_USE_COMPLEX)
    ierr = cusparseMat->mat->multiplyAdd(xarray, yarray, TRANSPOSE);CHKERRCUSP(ierr);
#else
    ierr = cusparseMat->mat->multiplyAdd(xarray, yarray, HERMITIAN);CHKERRCUSP(ierr);
#endif

    ierr = VecCUSPRestoreArrayRead(xx,&xarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayRead(yy,&yarray);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(zz,&zarray);CHKERRQ(ierr);
    
  } catch(char* ex) {
    SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
  }
  ierr = WaitForGPU();CHKERRCUSP(ierr);
  ierr = PetscLogFlops(2.0*a->nz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "MatAssemblyEnd_SeqAIJCUSPARSE"
PetscErrorCode MatAssemblyEnd_SeqAIJCUSPARSE(Mat A,MatAssemblyType mode)
{
  PetscErrorCode  ierr;  
  PetscFunctionBegin;
  ierr = MatAssemblyEnd_SeqAIJ(A,mode);CHKERRQ(ierr);
  ierr = MatSeqAIJCUSPARSECopyToGPU(A);CHKERRQ(ierr);
  if (mode == MAT_FLUSH_ASSEMBLY) PetscFunctionReturn(0);
  A->ops->mult             = MatMult_SeqAIJCUSPARSE;
  A->ops->multadd          = MatMultAdd_SeqAIJCUSPARSE;
  A->ops->multtranspose    = MatMultTranspose_SeqAIJCUSPARSE;
  A->ops->multtransposeadd = MatMultTransposeAdd_SeqAIJCUSPARSE;
  PetscFunctionReturn(0);
}

/* --------------------------------------------------------------------------------*/
#undef __FUNCT__  
#define __FUNCT__ "MatCreateSeqAIJCUSPARSE"
/*@
   MatCreateSeqAIJCUSPARSE - Creates a sparse matrix in AIJ (compressed row) format
   (the default parallel PETSc format). This matrix will ultimately pushed down
   to NVidia GPUs and use the CUSPARSE library for calculations. For good matrix 
   assembly performance the user should preallocate the matrix storage by setting 
   the parameter nz (or the array nnz).  By setting these parameters accurately, 
   performance during matrix assembly can be increased by more than a factor of 50.

   Collective on MPI_Comm

   Input Parameters:
+  comm - MPI communicator, set to PETSC_COMM_SELF
.  m - number of rows
.  n - number of columns
.  nz - number of nonzeros per row (same for all rows)
-  nnz - array containing the number of nonzeros in the various rows 
         (possibly different for each row) or PETSC_NULL

   Output Parameter:
.  A - the matrix 

   It is recommended that one use the MatCreate(), MatSetType() and/or MatSetFromOptions(),
   MatXXXXSetPreallocation() paradgm instead of this routine directly.
   [MatXXXXSetPreallocation() is, for example, MatSeqAIJSetPreallocation]

   Notes:
   If nnz is given then nz is ignored

   The AIJ format (also called the Yale sparse matrix format or
   compressed row storage), is fully compatible with standard Fortran 77
   storage.  That is, the stored row and column indices can begin at
   either one (as in Fortran) or zero.  See the users' manual for details.

   Specify the preallocated storage with either nz or nnz (not both).
   Set nz=PETSC_DEFAULT and nnz=PETSC_NULL for PETSc to control dynamic memory 
   allocation.  For large problems you MUST preallocate memory or you 
   will get TERRIBLE performance, see the users' manual chapter on matrices.

   By default, this format uses inodes (identical nodes) when possible, to 
   improve numerical efficiency of matrix-vector products and solves. We 
   search for consecutive rows with the same nonzero structure, thereby
   reusing matrix information to achieve increased efficiency.

   Level: intermediate

.seealso: MatCreate(), MatCreateAIJ(), MatSetValues(), MatSeqAIJSetColumnIndices(), MatCreateSeqAIJWithArrays(), MatCreateAIJ(), MATSEQAIJCUSPARSE, MATAIJCUSPARSE
@*/
PetscErrorCode  MatCreateSeqAIJCUSPARSE(MPI_Comm comm,PetscInt m,PetscInt n,PetscInt nz,const PetscInt nnz[],Mat *A)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = MatCreate(comm,A);CHKERRQ(ierr);
  ierr = MatSetSizes(*A,m,n,m,n);CHKERRQ(ierr);
  ierr = MatSetType(*A,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  ierr = MatSeqAIJSetPreallocation_SeqAIJ(*A,nz,(PetscInt*)nnz);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqAIJCUSPARSE"
PetscErrorCode MatDestroy_SeqAIJCUSPARSE(Mat A)
{
  PetscErrorCode        ierr;
  Mat_SeqAIJCUSPARSE      *cusparseMat = (Mat_SeqAIJCUSPARSE*)A->spptr;

  PetscFunctionBegin;
  if (A->factortype==MAT_FACTOR_NONE) {
    try {
      if (A->valid_GPU_matrix != PETSC_CUSP_UNALLOCATED){
	delete (GPU_Matrix_Ifc *)(cusparseMat->mat);
      }
      if (cusparseMat->tempvec!=0)
	delete cusparseMat->tempvec;
      delete cusparseMat;
      A->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    } 
  } else {
    /* The triangular factors */
    try {
      Mat_SeqAIJCUSPARSETriFactors *cusparseTriFactors  = (Mat_SeqAIJCUSPARSETriFactors*)A->spptr;
      GPU_Matrix_Ifc *cusparseMatLo  = (GPU_Matrix_Ifc*)cusparseTriFactors->loTriFactorPtr;
      GPU_Matrix_Ifc *cusparseMatUp  = (GPU_Matrix_Ifc*)cusparseTriFactors->upTriFactorPtr;
      delete (GPU_Matrix_Ifc *)cusparseMatLo;
      delete (GPU_Matrix_Ifc *)cusparseMatUp;	  
      delete (CUSPARRAY*) cusparseTriFactors->tempvec;
      delete cusparseTriFactors;
    } catch(char* ex) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_LIB,"CUSPARSE error: %s", ex);
    } 
  }
  if (MAT_cusparseHandle) {
    hipsparseStatus_t stat;
    stat = hipsparseDestroy(MAT_cusparseHandle);CHKERRCUSP(stat);  
    MAT_cusparseHandle=0;
  }
  /*this next line is because MatDestroy tries to PetscFree spptr if it is not zero, and PetscFree only works if the memory was allocated with PetscNew or PetscMalloc, which don't call the constructor */
  A->spptr = 0;

  ierr = MatDestroy_SeqAIJ(A);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

EXTERN_C_BEGIN
#undef __FUNCT__  
#define __FUNCT__ "MatCreate_SeqAIJCUSPARSE"
PetscErrorCode  MatCreate_SeqAIJCUSPARSE(Mat B)
{
  PetscErrorCode ierr;
    
  PetscFunctionBegin;
  ierr            = MatCreate_SeqAIJ(B);CHKERRQ(ierr);
  if (B->factortype==MAT_FACTOR_NONE) { 
    /* you cannot check the inode.use flag here since the matrix was just created.
       now build a GPU matrix data structure */
    B->spptr        = new Mat_SeqAIJCUSPARSE;
    ((Mat_SeqAIJCUSPARSE *)B->spptr)->mat = 0;
    ((Mat_SeqAIJCUSPARSE *)B->spptr)->tempvec = 0;
    ((Mat_SeqAIJCUSPARSE *)B->spptr)->format = MAT_CUSPARSE_CSR;
  } else {
    /* NEXT, set the pointers to the triangular factors */
    B->spptr        = new Mat_SeqAIJCUSPARSETriFactors;
    ((Mat_SeqAIJCUSPARSETriFactors *)B->spptr)->loTriFactorPtr = 0;
    ((Mat_SeqAIJCUSPARSETriFactors *)B->spptr)->upTriFactorPtr = 0;
    ((Mat_SeqAIJCUSPARSETriFactors *)B->spptr)->tempvec = 0;
    ((Mat_SeqAIJCUSPARSETriFactors *)B->spptr)->format = cusparseMatSolveStorageFormat;
  }
  /* Create a single instance of the MAT_cusparseHandle for any matrix (matMult, TriSolve, ...) */
  if (!MAT_cusparseHandle) {
    hipsparseStatus_t stat;  
    stat = hipsparseCreate(&MAT_cusparseHandle);CHKERRCUSP(stat);  
  }
  /* Here we overload MatGetFactor_petsc_C which enables -mat_type aijcusparse to use the 
     default cusparse tri solve. Note the difference with the implementation in 
     MatCreate_SeqAIJCUSP in ../seqcusp/aijcusp.cu */
  ierr = PetscObjectComposeFunctionDynamic((PetscObject)B,"MatGetFactor_petsc_C","MatGetFactor_seqaij_cusparse",MatGetFactor_seqaij_cusparse);CHKERRQ(ierr);
  B->ops->assemblyend      = MatAssemblyEnd_SeqAIJCUSPARSE;
  B->ops->destroy          = MatDestroy_SeqAIJCUSPARSE;
  B->ops->getvecs          = MatGetVecs_SeqAIJCUSPARSE;
  B->ops->setfromoptions   = MatSetFromOptions_SeqAIJCUSPARSE;
  B->ops->mult             = MatMult_SeqAIJCUSPARSE;
  B->ops->multadd          = MatMultAdd_SeqAIJCUSPARSE;
  B->ops->multtranspose    = MatMultTranspose_SeqAIJCUSPARSE;
  B->ops->multtransposeadd = MatMultTransposeAdd_SeqAIJCUSPARSE;
  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSEQAIJCUSPARSE);CHKERRQ(ierr);
  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
  ierr = PetscObjectComposeFunctionDynamic((PetscObject)B, "MatCUSPARSESetFormat_C", "MatCUSPARSESetFormat_SeqAIJCUSPARSE", MatCUSPARSESetFormat_SeqAIJCUSPARSE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
EXTERN_C_END

/*M
   MATSEQAIJCUSPARSE - MATAIJCUSPARSE = "(seq)aijcusparse" - A matrix type to be used for sparse matrices.

   A matrix type type whose data resides on Nvidia GPUs. These matrices can be in either 
   CSR, ELL, or Hybrid format. All matrix calculations are performed on Nvidia GPUs using 
   the CUSPARSE library. This type is only available when using the 'txpetscgpu' package.
   Use --download-txpetscgpu to build/install PETSc to use different CUSPARSE library and
   the different GPU storage formats.

   Options Database Keys:
+  -mat_type aijcusparse - sets the matrix type to "seqaijcusparse" during a call to MatSetFromOptions()
.  -mat_cusparse_storage_format csr (ell (ellpack) or hyb (hybrid)) sets the storage format of matrices (for MatMult and factors in MatSolve) during a call to MatSetFromOptions(). Only available with 'txpetscgpu' package.
.  -mat_cusparse_mult_storage_format csr (ell (ellpack) or hyb (hybrid)) sets the storage format of matrices (for MatMult) during a call to MatSetFromOptions(). Only available with 'txpetscgpu' package.
-  -mat_cusparse_solve_storage_format csr (ell (ellpack) or hyb (hybrid)) sets the storage format matrices (for factors in MatSolve) during a call to MatSetFromOptions(). Only available with 'txpetscgpu' package.

  Level: beginner

.seealso: MatCreateMPIAIJ,MATSEQAIJ,MATMPIAIJ, MATMPIAIJCUSPARSE, MATSEQAIJCUSPARSE, MatCUSPARSESetFormat(), MatCUSPARSEStorageFormat, MatCUSPARSEFormatOperation
M*/
