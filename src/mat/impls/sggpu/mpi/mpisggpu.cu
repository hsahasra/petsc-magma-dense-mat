#include "hip/hip_runtime.h"
#include <../src/mat/impls/sggpu/mpi/mpisggpu.h>

// Matrix function table
static struct _MatOps MatOps_Values = {
/*0*/ MatSetValues_MPISGGPU,MatGetRow_MPISGGPU,MatRestoreRow_MPISGGPU,MatMult_MPISGGPU,0,
/*5*/0,0,0,0,0,
/*10*/0,0,0,0,0,
/*15*/0,0,MatGetDiagonal_MPISGGPU,MatDiagonalScale_MPISGGPU,0,
/*20*/MatAssemblyBegin_MPISGGPU,MatAssemblyEnd_MPISGGPU,0,MatZeroEntries_MPISGGPU,0,
/*25*/0,0,0,0,MatSetUp_MPISGGPU,
/*30*/0,0,0,0,0,
/*35*/0,0,0,0,0,
/*40*/0,0,0,0,0,
/*45*/0,0,0,0,0,
/*50*/0,0,MatGetColumnIJ_MPISGGPU,0,MatFDColoringCreate_MPISGGPU,
/*55*/0,0,0,MatSetValuesBlocked_MPISGGPU,0,
/*60*/MatDestroy_MPISGGPU,MatView_MPISGGPU,0,0,0,
/*65*/0,0,MatSetValues_MPISGGPU,0,MatGetRowMaxAbs_MPISGGPU,
/*70*/0,0,0,0,0,
/*75*/MatFDColoringApply_MPISGGPU,0,0,0,0,
/*80*/0,0,0,0,0,
/*85*/0,0,MatSetValuesBlocked_MPISGGPU,0,0,
/*90*/0,0,0,0,0,
/*95*/0,0,0,0,0,
/*100*/0,0,0,0,0,
/*105*/0,0,0,0,0,
/*110*/0,0,0,0,0,
/*115*/MatCreate_MPISGGPU,0,0,0,0,
/*120*/0,0,0,0,0,
/*125*/0,0,0,0,0,
/*130*/0,0,0,0,0,
/*135*/0,0,0,0,0,
/*140*/0,0,
/*142*/MatSetGrid_MPISGGPU
};



EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_MPISGGPU"
PetscErrorCode MatCreate_MPISGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;
  PetscMPIInt size;

  PetscFunctionBegin;
  SGTrace;

//  ierr = MPI_Comm_size(((PetscObject)A)->comm, &size); CHKERRQ(ierr);
//  if (size > 1)
//    SETERRQ(PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "Comm must be size 1");

  // Create internal matrix structure
  ierr = PetscMalloc(sizeof(Mat_SeqSGGPU), &mat); CHKERRQ(ierr);
  memset(mat, 0, sizeof(Mat_SeqSGGPU));
  mat->diag_starts = new std::map<int, int>();
  mat->diagonals = new std::vector<int>();

  checkCudaError(hipStreamCreate(&mat->stream));

  // Fill out PETSc matrix structure
  A->data = mat;
  memcpy(A->ops, &MatOps_Values, sizeof(struct _MatOps));
  A->same_nonzero= PETSC_FALSE;
  A->spptr = 0;

  // Set object type
  ierr = PetscObjectChangeTypeName((PetscObject)A, MATMPISGGPU); CHKERRQ(ierr);

  ierr = PetscObjectComposeFunctionDynamic((PetscObject)A,
        "MatMPISGGPUSetPreallocation_C","MatMPISGGPUSetPreallocation_MPIDIA",
        MatMPISGGPUSetPreallocation_MPISGGPU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
EXTERN_C_END




#undef __FUNCT__
#define __FUNCT__ "MatDestroy_MPISGGPU"
PetscErrorCode MatDestroy_MPISGGPU(Mat A) {
  MatDestroy_SeqSGGPU(A);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetGrid_MPISGGPU"
PetscErrorCode MatSetGrid_MPISGGPU(Mat B, PetscInt m, PetscInt n, PetscInt p) 
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)B->data;

  PetscFunctionBegin;
  SGTrace;

  mat->m = m;
  mat->n = n;
  mat->p = p;

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatMult_MPISGGPU"
PetscErrorCode MatMult_MPISGGPU(Mat mat, Vec x, Vec y) {

  MatView_MPISGGPU(mat,PETSC_VIEWER_STDOUT_WORLD);	

  MatMult_SeqSGGPU(mat, x, y); 

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValuesBlocked_MPISGGPU"
PetscErrorCode MatSetValuesBlocked_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) {
  MatSetValuesBlocked_SeqSGGPU(A, nrow, irow, ncol, icol, y, is);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValues_MPISGGPU"
PetscErrorCode MatSetValues_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) {

  int i, j;
  PetscErrorCode ierr;
  PetscBool resizegpu = PETSC_FALSE;
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;

  PetscInt row, col;	
  PetscInt rank;
  PetscInt numprocs; 	

  PetscInt buf[8192],*bufr=0,*bufc=0,*irowm,*icolm;

  PetscFunctionBegin;
  SGTrace;
  
  PetscInt rstart = A->rmap->rstart, rend = A->rmap->rend;
  PetscInt cstart = A->cmap->rstart, cend = A->cmap->rend;  
 
    if ((nrow+ncol) <= (PetscInt)(sizeof(buf)/sizeof(PetscInt))) 
	{
	      irowm = buf; icolm = buf+nrow;
        } 
    else 
	{
	      ierr = PetscMalloc2(nrow,PetscInt,&bufr,ncol,PetscInt,&bufc);CHKERRQ(ierr);
	      irowm = bufr; icolm = bufc;
	}
   
   MPI_Comm_rank(PETSC_COMM_WORLD, &rank);
   MPI_Comm_size(PETSC_COMM_WORLD, &numprocs);
   
   ierr = ISLocalToGlobalMappingApply(A->rmap->mapping,nrow,irow,irowm);CHKERRQ(ierr);
   ierr = ISLocalToGlobalMappingApply(A->cmap->mapping,ncol,icol,icolm);CHKERRQ(ierr);


  // Handle each element
  for (i = 0; i < nrow; i++) {
	if (irowm[i] < 0) continue;
	        row = irowm[i]; 

    for (j = 0; j < ncol; j++) {
  	
	if(irowm[i] >= rstart && irowm[i] < rend) 
	{
		if (icolm[j] >= cstart && icolm[j] < cend) 
		{
	        	col = icolm[j]; //-cstart;
		} 
		else if (icolm[j] < 0) 
			continue;
		else
			col = icolm[j];


      	      // Compute the diagonal and offset into the diagonal storage
	      // for the element
	      //int row = irow[i];
	      //int col = icol[j];
      
	      int diff = col - row;
	      int left = row % mat->dof;
	      int diag = int(floor((double)(diff + left) / mat->dof));
	      int col_offset = col % mat->dof;
	      int num_elems = mat->m * mat->n * mat->p * mat->dof;
	      int offset = col_offset * num_elems + row;

#if _TRACE
      printf("- row: %d  col: %d  val: %lf  diag: %d  offset: %d\n", row, col, y[i*ncol+j], diag, offset);
#endif

      std::map<int, int> &diag_starts = *(mat->diag_starts);
      std::map<int, int>::iterator I = diag_starts.find(diag);
      int diag_offset = 0;
      if (I == diag_starts.end()) {
        printf("WARNING: malloc() in MatSetValues\n");
        resizegpu = PETSC_TRUE;
        // The diagonal does not yet exist, so add a new diagonal
        int num_diags = diag_starts.size() + 1;
        int size = num_diags * mat->m * mat->n * mat->p * mat->dof * mat->dof;
        PetscScalar *newData;
        ierr = PetscMalloc(size * sizeof(PetscScalar), &newData); CHKERRQ(ierr);
        memset(newData, 0, size * sizeof(PetscScalar));
        size -= mat->m * mat->n * mat->p * mat->dof * mat->dof;
        if (num_diags > 1) {
          // This is not the first diagonal, so copy
#if _TRACE
          printf("- Memcpy of %d elements\n", size);
#endif
          memcpy(newData, mat->hostData, size * sizeof(PetscScalar));
        }
        PetscFree(mat->hostData);
        mat->hostData = newData;
        diag_offset = size;
        diag_starts[diag] = diag_offset;
        mat->diagonals->push_back(diag);
      } else {
        // The diagonal already exists, so get the base offset
        diag_offset = I->second;
      }

      diag_offset += offset;

      if (is == INSERT_VALUES)
        mat->hostData[diag_offset] = y[i * ncol + j];
      else
        mat->hostData[diag_offset] += y[i * ncol + j];

      mat->non_zeros++;
	}
    }
  }

  if (resizegpu) {
    int size,mat_size;
    // Create GPU buffer
    if (mat->deviceData) {
      hipFree(mat->deviceData);
    }
    size = mat->diag_starts->size() * mat->m * mat->n * mat->p * mat->dof * mat->dof;
    checkCudaError(hipMalloc(&mat->deviceData, sizeof(PetscScalar) * size));


    mat_size = mat->m * mat->n * mat->p * mat->dof;

    if (mat->deviceX) {
      hipFree(mat->deviceX);
    }
    if (mat->deviceY) {
      hipFree(mat->deviceY);
    }
    if (mat->deviceDiags) {
      hipFree(mat->deviceDiags);
    }
    // We know the expected size of x, y, so go ahead and allocate them now
    checkCudaError(hipMalloc(&mat->deviceX, mat_size * sizeof(PetscScalar)));
    checkCudaError(hipMalloc(&mat->deviceY, mat_size * sizeof(PetscScalar)));

    // We also know how many diagonals we have, and their indices
    checkCudaError(hipMalloc(&mat->deviceDiags, sizeof(int) * mat->diagonals->size()));
  }

  PetscFunctionReturn(0);

}


//#undef __FUNCT__
//#define __FUNCT__ "MatSetStencil_MPISGGPU"
//PetscErrorCode MatSetStencil_MPISGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof) {
//  MatSetStencil_SeqSGGPU(A, dim, dims, starts, dof);
//  PetscFunctionReturn(0);
//}


#undef __FUNCT__
#define __FUNCT__ "MatSetUp_MPISGGPU"
PetscErrorCode MatSetUp_MPISGGPU(Mat mat) {
  MatSetUp_SeqSGGPU(mat);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatZeroEntries_MPISGGPU"
PetscErrorCode MatZeroEntries_MPISGGPU(Mat A) {
  MatZeroEntries_SeqSGGPU(A);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetDiagonal_MPISGGPU"
PetscErrorCode MatGetDiagonal_MPISGGPU(Mat A, Vec v) {
  MatGetDiagonal_SeqSGGPU(A, v);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatDiagonalScale_MPISGGPU"
PetscErrorCode MatDiagonalScale_MPISGGPU(Mat A, Vec ll, Vec rr) {
  MatDiagonalScale_SeqSGGPU(A, ll, rr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRow_MPISGGPU"
PetscErrorCode MatGetRow_MPISGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v) {
  MatGetRow_SeqSGGPU(A, row, nz, idx , v);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatRestoreRow_MPISGGPU"
PetscErrorCode MatRestoreRow_MPISGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v) {
  MatRestoreRow_SeqSGGPU(A, row, nz, idx, v);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRowMaxAbs_MPISGGPU"
PetscErrorCode MatGetRowMaxAbs_MPISGGPU(Mat A, Vec v, PetscInt idx[]) {
  MatGetRowMaxAbs_SeqSGGPU(A, v, idx);
  PetscFunctionReturn(0);
}


void DisplayLocalMatrix(Mat A);

#undef __FUNCT__
#define __FUNCT__ "MatView_MPISGGPU"
PetscErrorCode MatView_MPISGGPU(Mat A, PetscViewer viewer) 
{

  PetscInt rank, i, numprocs;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  MPI_Comm_size(PETSC_COMM_WORLD,&numprocs);	
	
  for(i = 0; i < numprocs;++i)
	{
	 if(i == rank)
	    DisplayLocalMatrix(A);
         MPI_Barrier(PETSC_COMM_WORLD);   
    	}
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyBegin_MPISGGPU"
PetscErrorCode MatAssemblyBegin_MPISGGPU(Mat A, MatAssemblyType type) {
  MatAssemblyBegin_SeqSGGPU(A, type);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_MPISGGPU"
PetscErrorCode MatAssemblyEnd_MPISGGPU(Mat A, MatAssemblyType type) {
  MatAssemblyEnd_SeqSGGPU(A, type);
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "MatFDColoringApply_MPISGGPU"
PetscErrorCode  MatFDColoringApply_MPISGGPU(Mat J,MatFDColoring coloring,Vec x1,MatStructure *flag,void *sctx) {
  MatFDColoringApply_SeqSGGPU(J, coloring, x1, flag, sctx);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatFDColoringCreate_MPISGGPU"
PetscErrorCode MatFDColoringCreate_MPISGGPU(Mat mat,ISColoring iscoloring,MatFDColoring c) {
  MatFDColoringCreate_SeqSGGPU(mat, iscoloring, c);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetColumnIJ_MPISGGPU"
PetscErrorCode MatGetColumnIJ_MPISGGPU(Mat A,PetscInt oshift,PetscBool  symmetric,PetscBool  inodecompressed,PetscInt *nn, const PetscInt *ia[], const PetscInt *ja[],PetscBool  *done) {
  MatGetColumnIJ_SeqSGGPU(A, oshift, symmetric, inodecompressed, nn, ia, ja, done);
  PetscFunctionReturn(0);
}


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMPISGGPUSetPreallocation"
PetscErrorCode MatMPISGGPUSetPreallocation(Mat A,PetscInt stencil_type, PetscInt dof)
{
  PetscErrorCode ierr;
  Mat_SeqSGGPU *mat = (Mat_SeqSGGPU*)A->data;

  PetscFunctionBegin;

  mat->stencil_type = stencil_type;
  mat->dof = dof;
  if(A->preallocated)PetscFunctionReturn(0);
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  
  ierr = PetscTryMethod(A,"MatMPISGGPUSetPreallocation_C",(Mat,PetscInt,const PetscInt []),(A,0,0));CHKERRQ(ierr);
  A->preallocated=PETSC_TRUE;
  PetscFunctionReturn(0);
}
EXTERN_C_END


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMPISGGPUSetPreallocation_MPISGGPU"
extern PetscErrorCode MatMPISGGPUSetPreallocation_MPISGGPU(Mat A,PetscInt nz, const PetscInt nnz[])
{
  PetscErrorCode ierr;
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;

  PetscInt dim,diag_size,size,num_diags,i,vecsize;

  ierr = PetscLayoutSetBlockSize(A->rmap,1);CHKERRQ(ierr);
  ierr = PetscLayoutSetBlockSize(A->cmap,1);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->cmap);CHKERRQ(ierr);

  dim = A->stencil.dim;
  if (mat->dof > 1) {
    dim--;
  }

//  mat->m = mat->n = mat->p = 1;
  mat->dim = dim;
//  if (mat->dim > 0) mat->m = A->stencil.dims[dim-1];
//  if (mat->dim > 1) mat->n = A->stencil.dims[dim-2];
//  if (mat->dim > 2) mat->p = A->stencil.dims[dim-3];

  if (mat->stencil_type == 0) {
    /* star stencil */
    num_diags = 2*mat->dim + 1;
  } else {
    /* box stencil */
    num_diags =  1;
    for (i=0;i<mat->dim;i++) num_diags*=3;
  }

  diag_size = mat->m * mat->n * mat->p * mat->dof * mat->dof;
  size = num_diags * diag_size;

  if (mat->m == 0 || mat->n == 0 || mat->p == 0 || mat->dof == 0) {
    SETERRQ(PETSC_COMM_SELF,0,"MatSetPreallocation_SeqSGGPU called without valid m, n, p, and dof!");
  }


  ierr = PetscMalloc(sizeof(PetscInt)*num_diags,&mat->diag_offsets);
  ierr = PetscMalloc(size * sizeof(PetscScalar), &mat->hostData); CHKERRQ(ierr);
  memset(mat->hostData, 0, size * sizeof(PetscScalar));

  (*mat->diag_starts)[0]  = 0 * diag_size;
  (*mat->diagonals).push_back(0);
  (*mat->diag_starts)[1]  = 1 * diag_size;
  (*mat->diagonals).push_back(1);
  (*mat->diag_starts)[-1] = 2 * diag_size;
  (*mat->diagonals).push_back(-1);
  if (mat->stencil_type == 0) {
    if (mat->dim == 2) {
      (*mat->diag_starts)[mat->m] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->m] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
    } else if (mat->dim == 3) {
      (*mat->diag_starts)[mat->m] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->m] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);

      (*mat->diag_starts)[mat->m*mat->n] = 5 * diag_size;
      (*mat->diagonals).push_back(mat->m*mat->n);
      (*mat->diag_starts)[-mat->m*mat->n] = 6 * diag_size;
      (*mat->diagonals).push_back(-mat->m*mat->n);
    }
  } else {
    if (mat->dim == 2) {
      (*mat->diag_starts)[mat->n-1] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n-1] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
      (*mat->diag_starts)[mat->n] = 5 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n] = 6 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
      (*mat->diag_starts)[mat->n+1] = 7 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n+1] = 8 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
    }
  }
  /*
  printf("Diagonals preallocated:\n");
  for (std::map<int, int>::iterator I = mat->diag_starts->begin(),
         E = mat->diag_starts->end(); I != E; ++I) {
    printf("%4d --> %4d\n",I->first,I->second);
  }
   */
  
  
  // Create GPU buffer
  if (mat->deviceData) {
    hipFree(mat->deviceData);
  }
  checkCudaError(hipMalloc(&mat->deviceData, sizeof(PetscScalar) * size));
  checkCudaError(hipMemset(mat->deviceData,0.0,sizeof(PetscScalar)*size));

  // Copy data to device
  checkCudaError(hipMemcpy(mat->deviceData, mat->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));


  vecsize = mat->m * mat->n * mat->p * mat->dof;

  // We know the expected size of x, y, so go ahead and allocate them now
  checkCudaError(hipMalloc(&mat->deviceX, vecsize * sizeof(PetscScalar)));
  checkCudaError(hipMalloc(&mat->deviceY, vecsize * sizeof(PetscScalar)));

  // We also know how many diagonals we have, and their indices
  checkCudaError(hipMalloc(&mat->deviceDiags, sizeof(int) * mat->diagonals->size()));
  A->preallocated = PETSC_TRUE;
  ierr = MatAssemblyBegin(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyEnd(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  

  PetscFunctionReturn(0);
}
EXTERN_C_END



void DisplayLocalMatrix(Mat A)
{

  Mat_SeqSGGPU *a;
  a  = (Mat_SeqSGGPU*)A->data;
  PetscErrorCode ierr;
  PetscInt nrows,ndiag,dof,i,j,iblock,col,index,offset;
  std::map<int, int> &diag_starts = *(a->diag_starts);
  
  nrows = a->m * a->n * a->p * a->dof;
  ndiag = a->diagonals->size();
  dof = a->dof;

  for (std::map<int, int>::iterator I = diag_starts.begin(),
         E = diag_starts.end(); I != E; ++I) 
	{
	fprintf(stdout,"- Diag %d:%d\n", I->first, I->second);
  	}

	fprintf(stdout,"\n");
	fprintf(stdout,"hostData:\n");

  for (i=0;i<nrows;i++) 
	{
	fprintf(stdout,"row %2.2d:",i); 

	    for (j=0;j<ndiag*dof;j++) 
		{
			fprintf(stdout," %4g ",a->hostData[i+j*nrows]);
		}
	fprintf(stdout,"\n");
  	}
	fprintf(stdout,"\n\n");


  for (iblock=0;iblock<nrows/dof;iblock++)  
	{
	    for (i=iblock*dof;i<(iblock+1)*dof;i++) 
		{
		fprintf(stdout,"row %d:",i);
	
		  for (std::map<int, int>::iterator I = a->diag_starts->begin(),
        	     E = a->diag_starts->end(); I != E; ++I) 
		  {
	        	/* Ignore 0 padding */
		        offset = I->first;
		        if (offset + iblock < 0) 
			{
	        	  continue;
        		}
	        	if (offset + iblock >= (nrows/dof)) 
			{
        	  	break;
        		}
        
	        	for (j=0;j<dof;j++) 
			{
	        	  index = i + I->second + j*nrows; // column-major
	        	  col = offset*dof+(iblock*dof) + j;
	                  fprintf(stdout," (%d, %g) ",col,a->hostData[index]);
        		}
      	 	  }
		fprintf(stdout,"\n");
    		}
  	}

}