#include "hip/hip_runtime.h"
#include <../src/mat/impls/sggpu/mpi/mpisggpu.h>
#include <../src/mat/impls/aij/mpi/mpiaij.h>

#include <petsc-private/daimpl.h>

// Direct access to seqgpu vector type
#include "../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h"

// Interop with CUSP vector
#include "../src/vec/vec/impls/seq/seqcusp/cuspvecimpl.h"

#include "cuPrintf.cu"

#define BLOCKWIDTH_X 128
#define BLOCKWIDTH_Y 1

//===-- CUDA Device Code -------------------------------------------------===//
 
texture<int2, 1> vector_x;
     
static __inline__ __device__ double fetch_doubleMPI(texture<int2, 1> tex, int i)
     {
       int2 v = tex1Dfetch(tex, i);
       return __hiloint2double(v.y, v.x);
     }
     
__global__ void MatMultKernelMPI(PetscScalar * coeff, PetscScalar * y, PetscScalar *x,PetscInt mat_size, PetscInt num_diags, int * diagonals, PetscInt dof, PetscInt vec_size) 
{
     int idx = blockDim.x * blockIdx.x * 1 + threadIdx.x * 1;

     if (idx >= mat_size) 
      	return;
     
int diag_size = mat_size * dof;
     
PetscScalar yval0 = 0.0;
int idx0 = idx;

//#pragma unroll 4
for (int i = 0; i < num_diags; ++i) 
	{
	    int d = diagonals[i];
    
	    int offset0 = diag_size * i + idx0;
	    int block0 = (idx0/dof + d) * dof;
     
	    //#pragma unroll 12
	    for (int j = 0; j < dof; ++j) 
		{
		      // Get coefficient
		      PetscScalar aval0 = coeff[offset0 + mat_size*j];
		      // Get X value
    	              PetscScalar xval0 = 0;

	              if ((block0 >= 0) && (block0 < vec_size))
			//xval0 = x[block0 + j];
			xval0 = fetch_doubleMPI(vector_x, block0 + j);

		      yval0 += aval0 * xval0;
	        }
	}
   
      y[idx0] = yval0;
}
    

//===-- Host Code --------------------------------------------------------===//

// Matrix function table
static struct _MatOps MatOps_Values = {
/*0*/ MatSetValues_MPISGGPU,MatGetRow_MPISGGPU,MatRestoreRow_MPISGGPU,MatMult_MPISGGPU,0,
/*5*/0,0,0,0,0,
/*10*/0,0,0,0,0,
/*15*/MatGetInfo_MPISGGPU,0,MatGetDiagonal_MPISGGPU,MatDiagonalScale_MPISGGPU,0,
/*20*/MatAssemblyBegin_MPISGGPU,MatAssemblyEnd_MPISGGPU,0,MatZeroEntries_MPISGGPU,0,
/*25*/0,0,0,0,MatSetUp_MPISGGPU,
/*30*/0,0,0,0,0,
/*35*/0,0,MatILUFactor_MPISGGPU,0,0,
/*40*/0,0,0,0,0,
/*45*/0,0,0,0,0,
/*50*/0,0,MatGetColumnIJ_MPISGGPU,0,MatFDColoringCreate_MPISGGPU,
/*55*/0,0,0,MatSetValuesBlocked_MPISGGPU,0,
/*60*/MatDestroy_MPISGGPU,MatView_MPISGGPU,0,0,0,
/*65*/0,0,MatSetValuesLocal_MPISGGPU,0,MatGetRowMaxAbs_MPISGGPU,
/*70*/0,0,0,0,0,
/*75*/MatFDColoringApply_MPISGGPU,0,0,0,0,
/*80*/0,0,0,0,0,
/*85*/0,0,MatSetValuesBlocked_MPISGGPU,0,0,
/*90*/0,0,0,0,0,
/*95*/0,0,0,0,0,
/*100*/0,0,0,0,0,
/*105*/0,0,0,0,0,
/*110*/0,0,0,0,0,
/*115*/MatCreate_MPISGGPU,0,0,0,0,
/*120*/0,0,0,0,0,
/*125*/0,0,0,0,0,
/*130*/0,0,0,0,0,
/*135*/0,0,0,0,0,
/*140*/0,0,
/*142*/MatSetGrid_MPISGGPU
};


#undef __FUNCT__
#define __FUNCT__ "MatGetDiagonalBlock_MPISGGPU"
PetscErrorCode  MatGetDiagonalBlock_MPISGGPU(Mat A,Mat *a)
{
  PetscFunctionBegin;
  
  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)(A->data);
  Mat_MPIAIJ *mpi_aij = (Mat_MPIAIJ*)(mat->mpi_aij->data);
  *a = mpi_aij->A;

  PetscFunctionReturn(0);
}



#undef __FUNCT__  
#define __FUNCT__ "MatGetInfo_MPISGGPU"
PetscErrorCode MatGetInfo_MPISGGPU(Mat A,MatInfoType flag,MatInfo *info)
{
  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)(A->data);
  Mat_SeqSGGPU *a = (Mat_SeqSGGPU*)(mat->mat_seq);
  
  PetscFunctionBegin;
  info->block_size     = (double)(a->dof * a->dof);
  info->nz_allocated   = (double)(a->m * a->n * a->p * a->dof * a->dof * a->stpoints);
  
//  PetscInt diag_size = (a->dof)*(A->rmap->n);
//  PetscInt size = num_diags * diag_size;
  
  info->nz_used        = (double)a->non_zeros;
  info->nz_unneeded    = info->nz_allocated - info->nz_used;
  info->assemblies     = (double)A->num_ass;
  info->mallocs        = (double)A->info.mallocs;
  info->memory         = ((PetscObject)A)->mem;
  if (A->factortype) {
    info->fill_ratio_given  = A->info.fill_ratio_given;
    info->fill_ratio_needed = A->info.fill_ratio_needed;
    info->factor_mallocs    = A->info.factor_mallocs;
  } else {
    info->fill_ratio_given  = 0;
    info->fill_ratio_needed = 0;
    info->factor_mallocs    = 0;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatILUFactor_MPISGGPU"
PetscErrorCode MatILUFactor_MPISGGPU(Mat inA, IS row, IS col, const MatFactorInfo *info)
{
  PetscErrorCode ierr;
  Mat_MPISGGPU* mat = (Mat_MPISGGPU*)inA->data;
  Mat_SeqSGGPU* mat_seq = mat->mat_seq;

  ierr = MatILUFactor_SeqSGGPU((Mat)mat_seq,row,col,info); CHKERRQ(ierr);
  PetscFunctionReturn(0);   	
}


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_MPISGGPU"
PetscErrorCode MatCreate_MPISGGPU(Mat A)
{
  Mat_MPISGGPU * mat;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  SGTrace;

  PetscInt rank, numprocs;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  MPI_Comm_size(((PetscObject)A)->comm, &numprocs); CHKERRQ(ierr);

  // Create internal matrix structure
  ierr = PetscMalloc(sizeof(Mat_MPISGGPU), &mat); CHKERRQ(ierr);
  memset(mat, 0, sizeof(Mat_MPISGGPU));
  ierr = PetscMalloc(sizeof(Mat_SeqSGGPU), &mat->mat_seq); CHKERRQ(ierr);
  memset(mat->mat_seq, 0, sizeof(Mat_SeqSGGPU));
  mat->mat_seq->diag_starts = new std::map<int, int>();
  mat->mat_seq->diagonals = new std::vector<int>();
    
  ierr = MatCreate(((PetscObject)A)->comm,&(mat->mpi_aij));CHKERRQ(ierr);
  ierr = MatSetSizes(((Mat)(mat->mpi_aij)),A->rmap->n,A->cmap->n,A->rmap->N,A->cmap->N);
  ierr = MatSetType(((Mat)(mat->mpi_aij)),MATMPIAIJ);
     
  checkCudaError(hipStreamCreate(&mat->mat_seq->stream));
  
  // Fill out PETSc matrix structure
  A->data = mat;
  memcpy(A->ops, &MatOps_Values, sizeof(struct _MatOps));
  A->same_nonzero= PETSC_FALSE;
  A->spptr = 0;

  // Set object type
  ierr = PetscObjectChangeTypeName((PetscObject)A, MATMPISGGPU); CHKERRQ(ierr);

  ierr = PetscObjectComposeFunctionDynamic((PetscObject)A,
        "MatMPISGGPUSetPreallocation_C","MatMPISGGPUSetPreallocation_MPIDIA",
        MatMPISGGPUSetPreallocation_MPISGGPU);CHKERRQ(ierr);

  ierr = PetscObjectComposeFunctionDynamic((PetscObject)A,
        "MatGetDiagonalBlock_C","MatGetDiagonalBlock_MPISGGPU",
        MatGetDiagonalBlock_MPISGGPU);CHKERRQ(ierr);



  PetscFunctionReturn(0);
}
EXTERN_C_END


#undef __FUNCT__
#define __FUNCT__ "MatDestroy_MPISGGPU"
PetscErrorCode MatDestroy_MPISGGPU(Mat A) {

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;  
  Mat_MPIAIJ  *mpi_aij = (Mat_MPIAIJ*)((mat->mpi_aij)->data);
  PetscErrorCode ierr;
  
  PetscFunctionBegin;
  
  if (mat_seq->hostData) {
    ierr = PetscFree(mat_seq->hostData); CHKERRQ(ierr);
  }
  if (mat_seq->deviceData) {
    hipFree(mat_seq->deviceData);
  }
  if (mat_seq->diag_starts) {
    delete mat_seq->diag_starts;
  }
  ierr = PetscFree(mat_seq->diag_offsets); CHKERRQ(ierr);
  if (mat_seq->diagonals) {
    delete mat_seq->diagonals;
  }
  if (mat_seq->deviceX) {
    hipFree(mat_seq->deviceX);
  }
  if (mat_seq->deviceY) {
    hipFree(mat_seq->deviceY);
  }
  if (mat_seq->deviceDiags) {
    hipFree(mat_seq->deviceDiags);
  }
  
  if(mat_seq->ja) { ierr = PetscFree(mat_seq->ja); CHKERRQ(ierr); }
  
  if(mat_seq->ia) { ierr = PetscFree(mat_seq->ia); CHKERRQ(ierr); }
  
  checkCudaError(hipStreamDestroy(mat_seq->stream));

  if(mat_seq)
	  ierr = PetscFree(mat_seq); CHKERRQ(ierr);
    
  ierr = VecDestroy(&mpi_aij->lvec); CHKERRQ(ierr);
  ierr = VecDestroy(&mat->lvec); CHKERRQ(ierr);
  ierr = VecScatterDestroy(&mpi_aij->Mvctx); CHKERRQ(ierr);
  ierr = PetscFree(A->data); CHKERRQ(ierr);
  
  ierr = PetscObjectChangeTypeName((PetscObject)A, 0); CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetGrid_MPISGGPU"
PetscErrorCode MatSetGrid_MPISGGPU(Mat A, PetscInt m, PetscInt n, PetscInt p) 
{
  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;

  PetscFunctionBegin;
  SGTrace;

  mat_seq->m = m;
  mat_seq->n = n;
  mat_seq->p = p;

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatMult_MPISGGPU"
PetscErrorCode MatMult_MPISGGPU(Mat A, Vec x, Vec y) {

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;
  Mat_MPIAIJ *mpi_aij = (Mat_MPIAIJ*)((mat->mpi_aij)->data);

  PetscBool xisseqcusp,xismpicusp,xisseqgpu,xismpigpu,xiscusp,xisgpu;
  PetscBool yisseqcusp,yismpicusp,yisseqgpu,yismpigpu,yiscusp,yisgpu;
  PetscErrorCode ierr;
  PetscInt mat_size, vec_size, numprocs,rank;
  CUSPARRAY *xgpu,*ygpu;
  PetscScalar *devX,*devY;
  PetscFunctionBegin;

  SGTrace;
  ierr = MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  ierr = MPI_Comm_size(PETSC_COMM_WORLD,&numprocs);


  // Initialize y to zero
  ierr = VecSet(y, 0.0); CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQCUSP,&xisseqcusp);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)x,VECMPICUSP,&xismpicusp);CHKERRQ(ierr);
  xiscusp = (xisseqcusp || xismpicusp) ? PETSC_TRUE : PETSC_FALSE;
  ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQGPU,&xisseqgpu);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQGPU,&xismpigpu);CHKERRQ(ierr);
  xisgpu = (xisseqgpu || xismpigpu) ? PETSC_TRUE : PETSC_FALSE;

  if (xisgpu) {
    dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);

dim3 grid((int)ceil((float)(A->rmap->n)/(float)BLOCKWIDTH_X / 1.0), 1);

    int shared_size = 0;
    Vec_SeqGPU *vx = (Vec_SeqGPU*) x->data;
    Vec_SeqGPU *vy = (Vec_SeqGPU*) y->data;
    /* Make sure y is also VECSEQGPU */
    ierr = PetscObjectTypeCompare((PetscObject)y,VECSEQGPU,&yismpigpu);CHKERRQ(ierr);
    if (!yismpigpu) {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Both x and y must be same type");
    }

    /* synch up x */
    if (vx->syncState==VEC_CPU) {
      ierr = VecCopyOverH2D(x,vx->cpuptr);CHKERRQ(ierr);
      vx->syncState=VEC_SYNCHED;
    }
    /* Get device pointer for X */
    devX = vx->devptr;
    devY = vy->devptr;
 
   /* Bind X to device texture */
//    mat_size = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/mpi_aij->size;

    mat_size = A->rmap->n;

    vec_size = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof);

    checkCudaError(hipBindTexture(0, vector_x, devX, vec_size * sizeof(PetscScalar)));

    MatMultKernelMPI<<<grid, block, shared_size, mat_seq->stream>>>(mat_seq->deviceData, devY, devX, mat_size, mat_seq->diagonals->size(), mat_seq->deviceDiags, mat_seq->dof, vec_size);

    hipUnbindTexture(vector_x);
    hipDeviceSynchronize();
  } 
  else if (xiscusp) 
  {
	dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
	dim3 grid((int)ceil((float)(A->rmap->N)/(float)BLOCKWIDTH_X / 1.0), 1);

    int shared_size = 0;
    /* Make sure y is also VECCUSP */
    ierr = PetscObjectTypeCompare((PetscObject)y,VECSEQCUSP,&yisseqcusp);CHKERRQ(ierr);
    ierr = PetscObjectTypeCompare((PetscObject)y,VECMPICUSP,&yismpicusp);CHKERRQ(ierr);
    yiscusp = ((xisseqcusp && yisseqcusp) || (xismpicusp && yismpicusp)) ? 
					PETSC_TRUE : PETSC_FALSE;
    if (!yiscusp) 
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Both x and y must be same type");
            
    if(!mat->da)    
    {
    ierr = PetscObjectQuery((PetscObject)x,"DM",(PetscObject*)&(mat->da));CHKERRQ(ierr);
    if (!(mat->da)) SETERRQ(((PetscObject)x)->comm,PETSC_ERR_ARG_WRONG,"Vector not generated from a DMDA");
    }

    DMDACreateNaturalVector(mat->da,&mat->vnatural); 
    ierr = DMDAGlobalToNaturalBegin(mat->da,x,INSERT_VALUES,mat->vnatural);CHKERRQ(ierr);
    ierr = DMDAGlobalToNaturalEnd(mat->da,x,INSERT_VALUES,mat->vnatural);CHKERRQ(ierr);

    ierr = VecScatterBegin(mat->Mvctx,mat->vnatural,mat->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
    ierr = VecScatterEnd(mat->Mvctx,mat->vnatural,mat->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
    ierr = VecDestroy(&mat->vnatural);CHKERRQ(ierr);
 	
//  mat_size = ((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/numprocs);   
    mat_size = A->rmap->n;

    //The entire vector is required because of row-wise distribution
    vec_size = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof);
 
    Vec result;
    VecDuplicate(y,&result);

    ierr = VecCUSPGetArrayWrite(result, &ygpu);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(mat->lvec, &xgpu);CHKERRQ(ierr);
    devY = thrust::raw_pointer_cast(&(*ygpu)[0]);
    devX = thrust::raw_pointer_cast(&(*xgpu)[0]);
    
    /* Bind X to device texture */
    checkCudaError(hipBindTexture(0, vector_x, devX, vec_size * sizeof(PetscScalar)));
  
#if _TRACE
    printf("Host diagonals:\n");
    for (int i = 0; i < mat_seq->diagonals->size(); ++i) 
	      printf("- %d\n", (*mat_seq->diagonals)[i]);
#endif

    /* Invoke */

#if _TIME
    double start, end;
    start = getclock();
#endif

cudaPrintfInit();
MatMultKernelMPI<<<grid, block, shared_size, mat_seq->stream>>>(mat_seq->deviceData, devY, devX, mat_size, mat_seq->diagonals->size(), mat_seq->deviceDiags, mat_seq->dof,vec_size);
cudaPrintfDisplay(stdout,true);
cudaPrintfEnd();

#if _TIME
    checkCudaError(hipStreamSynchronize(mat_seq->stream));
    end = getclock();
    double elapsed = end - start;
    double gflops = (2.0 * mat_seq->non_zeros / elapsed / 1e9);

    double nos = ((mat_seq->p == 1 ? 2 : 3) * 2 + 1) * (2*mat_seq->dof - 1);
    double nz = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/numprocs;
    double alt_gflops = (2.0 * nos * nz) / ((end - start)*1024*1024*1024);

#if _CSV_OUT
    fprintf(stderr, "%d,%lf,%lf,\n", (mat_seq->m, mat_seq->n, mat_seq->p, mat_seq->dof)/numprocs, elapsed, gflops);
#endif
    printf("SGGPU Kernel Time:           %lf sec\n", elapsed);
    printf("SGGPU Kernel GFlop/s:        %lf\n", gflops);
    printf("SGGPU Kernel GFlop/s (alt):  %lf\n", alt_gflops);
#endif

    /* Cleanup */
    hipUnbindTexture(vector_x);
 
    ierr = VecCUSPRestoreArrayRead(x, &xgpu);CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(result, &ygpu);CHKERRQ(ierr);
    ierr = WaitForGPU();CHKERRCUSP(ierr);

    ierr = VecScatterBegin(((DM_DA*)(mat->da->data))->gton,result,y,INSERT_VALUES,SCATTER_REVERSE);CHKERRQ(ierr);
    ierr = VecScatterEnd(((DM_DA*)(mat->da->data))->gton,result,y,INSERT_VALUES,SCATTER_REVERSE);CHKERRQ(ierr);

    ierr = VecDestroy(&result);

    hipDeviceSynchronize();
  }
  else 
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Vec must be seqgpu or cusp type");

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValuesBlocked_MPISGGPU"
PetscErrorCode MatSetValuesBlocked_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) {
  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatSetValuesBlocked_MPISGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValuesLocal_MPISGGPU"
PetscErrorCode MatSetValuesLocal_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) 
{
   Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;

   MatSetValuesLocal((Mat)(mat->mpi_aij),nrow,irow,ncol,icol,y,is);
   PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValues_MPISGGPU"
PetscErrorCode MatSetValues_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) 
{
   PetscErrorCode ierr;
   Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
   Mat_SeqSGGPU *mat_seq = (Mat_SeqSGGPU*)(mat->mat_seq);
   Mat_MPIAIJ *mpi_aij = (Mat_MPIAIJ*)(mat->mpi_aij->data);
   PetscBool resizegpu = PETSC_FALSE;   
   PetscInt i, j, row, col;
   PetscInt rank, numprocs;   
   PetscInt  rstart = A->rmap->rstart; 

   MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
   MPI_Comm_size(PETSC_COMM_WORLD,&numprocs);

   // Handle each element
   for (i = 0; i < nrow; i++) 
   {
          row = irow[i];
	   for (j = 0; j < ncol; j++) 
           {
         	      col = icol[j];      
 		      int diff = col - row;
		      int left = row % mat_seq->dof;
		      int diag = int(floor((double)(diff + left) / mat_seq->dof));
             
		      diag = rstart + diag;

		      int col_offset = col % mat_seq->dof;
		      //int num_elems = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/mpi_aij->size;
			int num_elems = A->rmap->n; 
		      int offset = col_offset * num_elems + row - rstart;  //(rank*num_elems);

#if _TRACE
    printf("- row: %d  col: %d  val: %lf  diag: %d  offset: %d\n", row, col, y[i*ncol+j], diag, offset);
#endif

      std::map<int, int> &diag_starts = *(mat_seq->diag_starts);
      std::map<int, int>::iterator I = diag_starts.find(diag);
      int diag_offset = 0;

      if (I == diag_starts.end()) 
	{
        printf("WARNING: malloc() in MatSetValues\n");
        resizegpu = PETSC_TRUE;
        // The diagonal does not yet exist, so add a new diagonal
        int num_diags = diag_starts.size() + 1;
        //int size = num_diags * ((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof * mat_seq->dof)/mpi_aij->size);

	int size = num_diags * (mat_seq->dof)*(A->rmap->n);

        PetscScalar *newData;
        ierr = PetscMalloc(size * sizeof(PetscScalar), &newData); CHKERRQ(ierr);
        memset(newData, 0, size * sizeof(PetscScalar));

	//        size -= ((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof * mat_seq->dof)/mpi_aij->size);

	size -= (mat_seq->dof)*(A->rmap->n);

        if (num_diags > 1) 
	{
        // This is not the first diagonal, so copy
	#if _TRACE
          printf("- Memcpy of %d elements\n", size);
	#endif
          memcpy(newData, mat_seq->hostData, size * sizeof(PetscScalar));
        }
        PetscFree(mat_seq->hostData);
        mat_seq->hostData = newData;
        diag_offset = size;
        diag_starts[diag] = diag_offset;
        mat_seq->diagonals->push_back(diag);
       }
      else 
      {
       // The diagonal already exists, so get the base offset
        diag_offset = I->second;
      }


      diag_offset += offset;

      if (is == INSERT_VALUES)
        mat_seq->hostData[diag_offset] = y[i * ncol + j];
      else
        mat_seq->hostData[diag_offset] += y[i * ncol + j];

      mat_seq->non_zeros++;
    }
  }

  if (resizegpu) {
    int size,mat_size;
    // Create GPU buffer
    if (mat_seq->deviceData) {
      hipFree(mat_seq->deviceData);
    }
//    size = mat_seq->diag_starts->size() * ((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof * mat_seq->dof)/numprocs);

    size = mat_seq->diag_starts->size() * (mat_seq->dof)*(A->rmap->n);

    checkCudaError(hipMalloc(&mat_seq->deviceData, sizeof(PetscScalar) * size));

//    mat_size = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/numprocs;

     mat_size = A->rmap->n;

    if (mat_seq->deviceX) {
      hipFree(mat_seq->deviceX);
    }
    if (mat_seq->deviceY) {
      hipFree(mat_seq->deviceY);
    }
    if (mat_seq->deviceDiags) {
      hipFree(mat_seq->deviceDiags);
    }
    // We know the expected size of x, y, so go ahead and allocate them now
    checkCudaError(hipMalloc(&mat_seq->deviceX, mat_size * sizeof(PetscScalar)));
    checkCudaError(hipMalloc(&mat_seq->deviceY, mat_size * sizeof(PetscScalar)));

    // We also know how many diagonals we have, and their indices
    checkCudaError(hipMalloc(&mat_seq->deviceDiags, sizeof(int) * mat_seq->diagonals->size()));
  }

   PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetUp_MPISGGPU"
PetscErrorCode MatSetUp_MPISGGPU(Mat A) {

  PetscFunctionBegin;
  SGTrace;

  PetscPrintf(PETSC_COMM_WORLD,"MatSetUp_MPISGGPU() not implemented\n");
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatZeroEntries_MPISGGPU"
PetscErrorCode MatZeroEntries_MPISGGPU(Mat A) {

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;
  Mat_MPIAIJ *mpi_aij = (Mat_MPIAIJ*)(mat->mpi_aij)->data;  

  PetscInt size;
  PetscFunctionBegin;
  SGTrace;
  
  size = mat_seq->diag_starts->size() * ((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof * mat_seq->dof)/A->rmap->n);
  memset(mat_seq->hostData, 0, size * sizeof(PetscScalar));
  
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetDiagonal_MPISGGPU"
PetscErrorCode MatGetDiagonal_MPISGGPU(Mat A, Vec v) {

  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatGetDiagonal_MPISGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatDiagonalScale_MPISGGPU"
PetscErrorCode MatDiagonalScale_MPISGGPU(Mat A, Vec ll, Vec rr) {

  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatDiagonalScale_MPISGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRow_MPISGGPU"
PetscErrorCode MatGetRow_MPISGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v) {

  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatGetRow_MPISGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatRestoreRow_MPISGGPU"
PetscErrorCode MatRestoreRow_MPISGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v) {

  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatRestoreRow_MPISGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRowMaxAbs_MPISGGPU"
PetscErrorCode MatGetRowMaxAbs_MPISGGPU(Mat A, Vec v, PetscInt idx[]) {

  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatGetRowMaxAbs_MPISGGPU not implemented");
}


void DisplayLocalMatrix(Mat_SeqSGGPU *mat_seq);

#undef __FUNCT__
#define __FUNCT__ "MatView_MPISGGPU"
PetscErrorCode MatView_MPISGGPU(Mat A, PetscViewer viewer) 
{
  PetscInt i;
  Mat_MPISGGPU* mat = (Mat_MPISGGPU*)A->data;
  Mat_MPIAIJ* mpi_aij = (Mat_MPIAIJ*)((mat->mpi_aij)->data);  

  for(i = 0; i < mpi_aij->size;++i)
	{
	 if(i == mpi_aij->rank)
	    DisplayLocalMatrix(mat->mat_seq);
         MPI_Barrier(PETSC_COMM_WORLD);   
    	}
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyBegin_MPISGGPU"
PetscErrorCode MatAssemblyBegin_MPISGGPU(Mat A, MatAssemblyType type) {

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;

  PetscErrorCode ierr;
  PetscFunctionBegin;
  SGTrace;
  
  ierr = MatAssemblyBegin(mat->mpi_aij,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_MPISGGPU"
PetscErrorCode MatAssemblyEnd_MPISGGPU(Mat A, MatAssemblyType type) {

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;
  Mat_MPIAIJ *mpi_aij = (Mat_MPIAIJ*)((mat->mpi_aij)->data); 
  
  PetscInt size, rank;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  PetscErrorCode ierr;

  PetscFunctionBegin;

  ierr = MatAssemblyEnd(mat->mpi_aij,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);

#if _TRACE
  printf("[SeqSGGPU] MatAssemblyEnd_SeqSGGPU\n");

  for (std::map<int, int>::iterator I = mat_seq->diag_starts->begin(),
       E = mat_seq->diag_starts->end(); I != E; ++I) {
    printf("- Diag %d:\n", I->first);
    for (int i = 0; i < mat_seq->dof; ++i) {
      for (int j = 0; j < (mat_seq->dof * mat_seq->m * mat_seq->n * mat_seq->p)/numprocs; ++j) {
        int offset = i * ((mat_seq->dof * mat_seq->m * mat_seq->n * mat_seq->p)/numprocs) + j;
        printf(" %lf ", mat_seq->hostData[offset + I->second]);
      }
      printf("\n");
    }
  }
#endif

//size = (mat_seq->diag_starts->size()*mat_seq->m*mat_seq->n*mat_seq->p*mat_seq->dof*mat_seq->dof)/mpi_aij->size;

size = mat_seq->diag_starts->size()*(mat_seq->dof)*A->rmap->n;

checkCudaError(hipMemcpyAsync(mat_seq->deviceDiags, &(*mat_seq->diagonals)[0], sizeof(int) * mat_seq->diagonals->size(), hipMemcpyHostToDevice, mat_seq->stream));

checkCudaError(hipMemcpy(mat_seq->deviceData, mat_seq->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));

hipDeviceSynchronize();

	Mat Anatural;
	if(mat->preallocated)
	{
ierr = MatGetSubMatrix(((Mat)(mat->mpi_aij)),mat->is,mat->is,MAT_INITIAL_MATRIX,&Anatural);CHKERRQ(ierr);

ierr = MatSetSGGPUMatrix(A,Anatural);CHKERRQ(ierr); 	
	}

 PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetSGGPUMatrix"
PetscErrorCode  MatSetSGGPUMatrix(Mat A,Mat Anatural) 
{
  int i;
  PetscErrorCode ierr;
  Mat_MPIAIJ *An = (Mat_MPIAIJ*)Anatural->data;
  PetscInt row, m, *ai, *aj, *cols, *ct;	

  PetscFunctionBegin;
  SGTrace;

    /* copy over the A part */
    Mat_SeqAIJ* Aloc = (Mat_SeqAIJ*)(An->A->data);
    m = A->rmap->n; ai = Aloc->i; aj = Aloc->j; 
    MatScalar *a = Aloc->a;
    row = A->rmap->rstart;
    for (i=0; i<ai[m]; i++) {aj[i] += A->cmap->rstart ;}
    for (i=0; i<m; i++) {
      ierr = MatSetValues_MPISGGPU(A,1,&row,ai[i+1]-ai[i],aj,a,INSERT_VALUES);CHKERRQ(ierr);
      row++; a += ai[i+1]-ai[i]; aj += ai[i+1]-ai[i];
    }
    aj = Aloc->j;
    for (i=0; i<ai[m]; i++) {aj[i] -= A->cmap->rstart;}

    /* copy over the B part */
    Aloc = (Mat_SeqAIJ*)An->B->data;
    m    = A->rmap->n;  ai = Aloc->i; aj = Aloc->j; a = Aloc->a;
    row  = A->rmap->rstart;
    ierr = PetscMalloc((ai[m]+1)*sizeof(PetscInt),&cols);CHKERRQ(ierr);
    ct   = cols;
    for (i=0; i<ai[m]; i++) {cols[i] = An->garray[aj[i]];}
    for (i=0; i<m; i++) {
      ierr = MatSetValues_MPISGGPU(A,1,&row,ai[i+1]-ai[i],cols,a,INSERT_VALUES);CHKERRQ(ierr);
      row++; a += ai[i+1]-ai[i]; cols += ai[i+1]-ai[i];
    }
    ierr = PetscFree(ct);CHKERRQ(ierr);

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatFDColoringApply_MPISGGPU"
PetscErrorCode  MatFDColoringApply_MPISGGPU(Mat A,MatFDColoring coloring,Vec x1,MatStructure *flag,void *sctx) {

  MatFDColoringApply_SeqSGGPU(A, coloring, x1, flag, sctx);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatFDColoringCreate_MPISGGPU"
PetscErrorCode MatFDColoringCreate_MPISGGPU(Mat A,ISColoring iscoloring,MatFDColoring c) {
  PetscErrorCode ierr;
  PetscInt       i,n,nrows,N,j,k,m,ncols,col;
  const PetscInt *is,*ci,*cj,*rows;
  PetscInt       nis = iscoloring->n,*rowhit,*columnsforrow,l,bs = 1;
  IS             *isa;
  PetscBool      done,flg = PETSC_FALSE;

  PetscFunctionBegin;

  ierr = ISColoringGetIS(iscoloring,PETSC_IGNORE,&isa);CHKERRQ(ierr);
  /* this is ugly way to get blocksize but cannot call MatGetBlockSize() because AIJ can have bs > 1 */

  N          = A->cmap->N/bs;
  c->M       = A->rmap->N/bs;  /* set total rows, columns and local rows */
  c->N       = A->cmap->N/bs;
  c->m       = A->rmap->N/bs;
  c->rstart  = 0;

  c->ncolors = nis;
  ierr       = PetscMalloc(nis*sizeof(PetscInt),&c->ncolumns);CHKERRQ(ierr);
  ierr       = PetscMalloc(nis*sizeof(PetscInt*),&c->columns);CHKERRQ(ierr); 
  ierr       = PetscMalloc(nis*sizeof(PetscInt),&c->nrows);CHKERRQ(ierr);
  ierr       = PetscMalloc(nis*sizeof(PetscInt*),&c->rows);CHKERRQ(ierr);
  ierr       = PetscMalloc(nis*sizeof(PetscInt*),&c->columnsforrow);CHKERRQ(ierr);

  ierr = MatGetColumnIJ(A,0,PETSC_FALSE,PETSC_FALSE,&ncols,&ci,&cj,&done);CHKERRQ(ierr);
  if (!done) SETERRQ1(((PetscObject)A)->comm,PETSC_ERR_SUP,"MatGetColumnIJ() not supported for matrix type %s",((PetscObject)A)->type_name);

  /*
     Temporary option to allow for debugging/testing
  */
  ierr = PetscOptionsGetBool(PETSC_NULL,"-matfdcoloring_slow",&flg,PETSC_NULL);CHKERRQ(ierr);

  ierr = PetscMalloc((N+1)*sizeof(PetscInt),&rowhit);CHKERRQ(ierr);
  ierr = PetscMalloc((N+1)*sizeof(PetscInt),&columnsforrow);CHKERRQ(ierr);

  for (i=0; i<nis; i++) {
    ierr = ISGetLocalSize(isa[i],&n);CHKERRQ(ierr);
    ierr = ISGetIndices(isa[i],&is);CHKERRQ(ierr);
    c->ncolumns[i] = n;
    if (n) {
      ierr = PetscMalloc(n*sizeof(PetscInt),&c->columns[i]);CHKERRQ(ierr);
      ierr = PetscMemcpy(c->columns[i],is,n*sizeof(PetscInt));CHKERRQ(ierr);
    } else {
      c->columns[i]  = 0;
    }

    if (!flg) { /* ------------------------------------------------------------------------------*/
      /* fast, crude version requires O(N*N) work */
      ierr = PetscMemzero(rowhit,N*sizeof(PetscInt));CHKERRQ(ierr);
      /* loop over columns*/
      for (j=0; j<n; j++) {
        col  = is[j];
        rows = cj + ci[col]; 
        m    = ci[col+1] - ci[col];
        /* loop over columns marking them in rowhit */
        for (k=0; k<m; k++) {
          rowhit[*rows++] = col + 1;
        }
      }
      /* count the number of hits */
      nrows = 0;
      for (j=0; j<N; j++) {
        if (rowhit[j]) nrows++;
      }
      c->nrows[i] = nrows;
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->rows[i]);CHKERRQ(ierr);
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->columnsforrow[i]);CHKERRQ(ierr);
      nrows       = 0;
      for (j=0; j<N; j++) {
        if (rowhit[j]) {
          c->rows[i][nrows]          = j;
          c->columnsforrow[i][nrows] = rowhit[j] - 1;
          nrows++;
        }
      }
    } else {  /*-------------------------------------------------------------------------------*/
      /* slow version, using rowhit as a linked list */
      PetscInt currentcol,fm,mfm;
      rowhit[N] = N;
      nrows     = 0;
      /* loop over columns */
      for (j=0; j<n; j++) {
        col   = is[j];
        rows  = cj + ci[col]; 
        m     = ci[col+1] - ci[col];
        /* loop over columns marking them in rowhit */
        fm    = N; /* fm points to first entry in linked list */
        for (k=0; k<m; k++) {
          currentcol = *rows++;
	  /* is it already in the list? */
          do {
            mfm  = fm;
            fm   = rowhit[fm];
          } while (fm < currentcol);
          /* not in list so add it */
          if (fm != currentcol) {
            nrows++;
            columnsforrow[currentcol] = col;
            /* next three lines insert new entry into linked list */
            rowhit[mfm]               = currentcol;
            rowhit[currentcol]        = fm;
            fm                        = currentcol; 
            /* fm points to present position in list since we know the columns are sorted */
          } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Detected invalid coloring");
        }
      }
      c->nrows[i] = nrows;
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->rows[i]);CHKERRQ(ierr);
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->columnsforrow[i]);CHKERRQ(ierr);
      /* now store the linked list of rows into c->rows[i] */
      nrows       = 0;
      fm          = rowhit[N];
      do {
        c->rows[i][nrows]            = fm;
        c->columnsforrow[i][nrows++] = columnsforrow[fm];
        fm                           = rowhit[fm];
      } while (fm < N);
    } /* ---------------------------------------------------------------------------------------*/
    ierr = ISRestoreIndices(isa[i],&is);CHKERRQ(ierr);  
  }
  ierr = MatRestoreColumnIJ(A,0,PETSC_FALSE,PETSC_FALSE,&ncols,&ci,&cj,&done);CHKERRQ(ierr);

  ierr = PetscFree(rowhit);CHKERRQ(ierr);
  ierr = PetscFree(columnsforrow);CHKERRQ(ierr);

  /* Optimize by adding the vscale, and scaleforrow[][] fields */
  /*
       see the version for MPIAIJ
  */
  ierr = VecCreateGhost(((PetscObject)A)->comm,A->rmap->n,PETSC_DETERMINE,0,PETSC_NULL,&c->vscale);CHKERRQ(ierr);
  ierr = PetscMalloc(c->ncolors*sizeof(PetscInt*),&c->vscaleforrow);CHKERRQ(ierr);
  for (k=0; k<c->ncolors; k++) { 
    ierr = PetscMalloc((c->nrows[k]+1)*sizeof(PetscInt),&c->vscaleforrow[k]);CHKERRQ(ierr);
    for (l=0; l<c->nrows[k]; l++) {
      col = c->columnsforrow[k][l];
      c->vscaleforrow[k][l] = col;
    }
  }
  ierr = ISColoringRestoreIS(iscoloring,&isa);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetColumnIJ_MPISGGPU"
PetscErrorCode MatGetColumnIJ_MPISGGPU(Mat A,PetscInt oshift,PetscBool  symmetric,PetscBool  inodecompressed,PetscInt *nn, const PetscInt *ia[], const PetscInt *ja[],PetscBool  *done) {

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *a = mat->mat_seq;  

  PetscErrorCode ierr;
  PetscInt       n = A->cmap->n;
  PetscInt       ndiag = a->diagonals->size();
  PetscInt       nrows = a->m*a->n*a->p*a->dof;
  PetscInt       nz=a->dof*ndiag*nrows;
  PetscInt       iblock,i,j,col,index,colblock,offset;

  PetscFunctionBegin;  

  *nn = nrows;

  if (!ia) PetscFunctionReturn(0);
  if (a->ja) {
    ierr = PetscFree(a->ja); CHKERRQ(ierr);
  }
  if (a->ia) {
    ierr = PetscFree(a->ia); CHKERRQ(ierr);
  }
  ierr = PetscMalloc((n+1)*sizeof(PetscInt),&a->ia);CHKERRQ(ierr);
  ierr = PetscMalloc((nz+1)*sizeof(PetscInt),&a->ja);CHKERRQ(ierr);

  /* Assuming symmetric nonzero structure */
  index=0;
  for (iblock=0;iblock<nrows/a->dof;iblock++) {
    for (i=iblock*a->dof;i<(iblock+1)*a->dof;i++) {
      a->ia[i] = index;
      for (std::map<int, int>::iterator I = a->diag_starts->begin(),
             E = a->diag_starts->end(); I != E; ++I) {
        offset = I->first;
        colblock = offset + iblock;
        /* Ignore 0 padding */
        if (colblock < 0) {
          continue;
        }
        if (colblock >= (nrows/a->dof)) {
          break;
        }
        /* skip some blocks for nonperiodic da */
        if (a->stencil_type==0 &&  a->dim==2 && 
            ((colblock - iblock == 1 && !(colblock % a->n)) ||
             (iblock - colblock == 1 && !(iblock % a->n)))) {
          continue;

        }
      
        for (j=0;j<a->dof;j++) {
          col = (colblock*a->dof)  + j;
          a->ja[index++] = col;
	}

      }
    }
  }
  a->ia[nrows] = index;
  *ia = a->ia;
  *ja = a->ja;

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatMPISGGPUSetPreallocation"
PetscErrorCode MatMPISGGPUSetPreallocation(Mat A,PetscInt stencil_type, PetscInt dof,PetscInt* dnz,PetscInt* onz)
{
  PetscErrorCode ierr;
  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;  

  PetscFunctionBegin;

  mat_seq->stencil_type = stencil_type;
  mat_seq->dof = dof;
  if(A->preallocated)PetscFunctionReturn(0);
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  
  ierr = PetscTryMethod(A,"MatMPISGGPUSetPreallocation_C",(Mat,PetscInt,const PetscInt [],PetscInt* dnz,PetscInt* onz),(A,0,0,dnz,onz));CHKERRQ(ierr);
  A->preallocated=PETSC_TRUE;
  PetscFunctionReturn(0);
}


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMPISGGPUSetPreallocation_MPISGGPU"
extern PetscErrorCode MatMPISGGPUSetPreallocation_MPISGGPU(Mat A,PetscInt nz, const PetscInt nnz[],PetscInt* dnz,PetscInt* onz)
{
  PetscErrorCode ierr;
  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;
  Mat_MPIAIJ *mpi_aij = (Mat_MPIAIJ*)((mat->mpi_aij)->data);

  PetscInt dim,diag_size,size,num_diags,i,vecsize;
  
  PetscInt rank;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);

//  ierr = PetscLayoutSetBlockSize(A->rmap,1);CHKERRQ(ierr);
//  ierr = PetscLayoutSetBlockSize(A->cmap,1);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->cmap);CHKERRQ(ierr);

  dim = A->stencil.dim;
  if (mat_seq->dof > 1) {
    dim--;
  }

  PetscInt rstart = A->rmap->rstart;
  mat_seq->dim = dim;

  if (mat_seq->stencil_type == 0) {
    /* star stencil */
    num_diags = 2*mat_seq->dim + 1;
  } else {
    /* box stencil */
    num_diags =  1;
    for (i=0;i<mat_seq->dim;i++) num_diags*=3;
  }

//diag_size = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof * mat_seq->dof)/mpi_aij->size;

diag_size = (mat_seq->dof)*(A->rmap->n);
 
  size = num_diags * diag_size;

  if (mat_seq->m == 0 || mat_seq->n == 0 || mat_seq->p == 0 || mat_seq->dof == 0) 
  {
    SETERRQ(PETSC_COMM_SELF,0,"MatSetPreallocation_SeqSGGPU called without valid m, n, p, and dof!");
  }

  ierr = PetscMalloc(sizeof(PetscInt)*num_diags,&mat_seq->diag_offsets);
  ierr = PetscMalloc(size * sizeof(PetscScalar), &mat_seq->hostData); CHKERRQ(ierr);
  memset(mat_seq->hostData, 0, size * sizeof(PetscScalar));
  
  (*mat_seq->diag_starts)[rstart + 0]  = 0 * diag_size;
  (*mat_seq->diagonals).push_back(rstart + 0);
  (*mat_seq->diag_starts)[rstart + 1]  = 1 * diag_size;
  (*mat_seq->diagonals).push_back(rstart + 1);
  (*mat_seq->diag_starts)[rstart - 1] = 2 * diag_size;
  (*mat_seq->diagonals).push_back(rstart - 1);
  
  if (mat_seq->stencil_type == 0) {
    if (mat_seq->dim == 2) 
    {
      (*mat_seq->diag_starts)[rstart + mat_seq->m] = 3 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m);
      (*mat_seq->diag_starts)[rstart - mat_seq->m] = 4 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m);
    } 
    else if (mat_seq->dim == 3) 
    {
      (*mat_seq->diag_starts)[rstart + mat_seq->m] = 3 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m);
      (*mat_seq->diag_starts)[rstart - mat_seq->m] = 4 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m);

      (*mat_seq->diag_starts)[rstart + mat_seq->m*mat_seq->n] = 5 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m*mat_seq->n);
      (*mat_seq->diag_starts)[rstart - mat_seq->m*mat_seq->n] = 6 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m*mat_seq->n);
    }
  } else {
    if (mat_seq->dim == 2) {
      (*mat_seq->diag_starts)[rstart + mat_seq->n-1] = 3 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m);
      (*mat_seq->diag_starts)[rstart - mat_seq->n-1] = 4 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m);
      (*mat_seq->diag_starts)[rstart + mat_seq->n] = 5 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m);
      (*mat_seq->diag_starts)[rstart - mat_seq->n] = 6 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m);
      (*mat_seq->diag_starts)[rstart + mat_seq->n+1] = 7 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m);
      (*mat_seq->diag_starts)[rstart - mat_seq->n+1] = 8 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m);
    }
  }
    
/*
  printf("Diagonals preallocated:\n");
  for (std::map<int, int>::iterator I = mat_seq->diag_starts->begin(),
         E = mat_seq->diag_starts->end(); I != E; ++I) 
	{
    printf("Rank:%d\t%4d --> %4d\n",rank,I->first,I->second);
  	}
*/

  
 
  // Create GPU buffer
  if (mat_seq->deviceData) 
  {
    hipFree(mat_seq->deviceData);
  }

  checkCudaError(hipMalloc(&mat_seq->deviceData, sizeof(PetscScalar)*size));
  checkCudaError(hipMemset(mat_seq->deviceData,0,sizeof(PetscScalar)*size));

  // Copy data to device
  checkCudaError(hipMemcpy(mat_seq->deviceData, mat_seq->hostData, sizeof(PetscScalar)*size, hipMemcpyHostToDevice));

//  vecsize = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/mpi_aij->size;

  vecsize = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof);

//   vecsize = A->rmap->n;

  // We know the expected size of x, y, so go ahead and allocate them now
  checkCudaError(hipMalloc(&mat_seq->deviceX, vecsize * sizeof(PetscScalar)));
  checkCudaError(hipMalloc(&mat_seq->deviceY, A->rmap->n * sizeof(PetscScalar)));
  
  ierr = PetscLayoutDuplicate(A->rmap,&((mat->mpi_aij)->rmap));
  ierr = PetscLayoutDuplicate(A->cmap,&((mat->mpi_aij)->cmap));
     
  MatSeqAIJSetPreallocation(((Mat)(mat->mpi_aij)),0,dnz);
  MatMPIAIJSetPreallocation(((Mat)(mat->mpi_aij)),0,dnz,0,onz);
 
  // We also know how many diagonals we have, and their indices
  checkCudaError(hipMalloc(&mat_seq->deviceDiags, sizeof(int) * mat_seq->diagonals->size()));
  
  mat->preallocated = PETSC_FALSE; 

  MatSetUpMultiply_MPISGGPU(A);

PetscFunctionReturn(0);
}
EXTERN_C_END


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MPISGGPUGetMapping"
extern PetscErrorCode MPISGGPUGetMapping(Mat A)
{
  AO ao;
  PetscInt rstart1, rend1,*petsc;
  DM da;
  MPI_Comm comm;
  PetscErrorCode ierr;   
  PetscInt i;

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  ierr = PetscObjectGetComm((PetscObject)A,&comm);CHKERRQ(ierr);  
  ierr = PetscObjectQuery((PetscObject)A,"DM",(PetscObject*)&da);CHKERRQ(ierr);
  if (!da) SETERRQ(((PetscObject)A)->comm,PETSC_ERR_ARG_WRONG,"Matrix not generated from a DMDA");

  ierr = DMDAGetAO(da,&ao);CHKERRQ(ierr);
  ierr = MatGetOwnershipRange(A,&rstart1,&rend1);CHKERRQ(ierr);
  ierr = PetscMalloc((rend1-rstart1)*sizeof(PetscInt),&petsc);CHKERRQ(ierr);
  for (i=rstart1; i<rend1; i++) petsc[i-rstart1] = i;
  ierr = AOApplicationToPetsc(ao,rend1-rstart1,petsc);CHKERRQ(ierr);
  ierr = ISCreateGeneral(comm,rend1-rstart1,petsc,PETSC_OWN_POINTER,&(mat->is));CHKERRQ(ierr);
  
  mat->preallocated = PETSC_TRUE;

  PetscFunctionReturn(0);
}
EXTERN_C_END


void DisplayLocalMatrix(Mat_SeqSGGPU *mat_seq)
{

  PetscInt nrows,ndiag,dof,i,j,iblock,col,index,offset;
  std::map<int, int> &diag_starts = *(mat_seq->diag_starts);

  PetscInt numprocs, rank;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  MPI_Comm_size(PETSC_COMM_WORLD,&numprocs);

  nrows = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/numprocs;

  //nrows = A->rmap->n;
  
  ndiag = mat_seq->diagonals->size();
  dof = mat_seq->dof;

  for (std::map<int, int>::iterator I = diag_starts.begin(),
         E = diag_starts.end(); I != E; ++I) 
	{
		fprintf(stdout,"- Diag %d:%d\n", I->first, I->second);
  	}

	fprintf(stdout,"\n");
	fprintf(stdout,"hostData:\n");

  for (i=0;i<nrows;i++) 
	{
	fprintf(stdout,"row %2.2d:",rank*nrows + i); 

	    for (j=0;j<ndiag*dof;j++) 
		{
			fprintf(stdout," %4g ",mat_seq->hostData[i+j*nrows]);
		}
	fprintf(stdout,"\n");
  	}
	fprintf(stdout,"\n\n");


  for (iblock=0;iblock<(nrows/dof);iblock++)  
	{
	    for (i=iblock*dof;i<(iblock+1)*dof;i++) 
		{
		fprintf(stdout,"row %d:",rank*nrows + i);
	
		  for (std::map<int, int>::iterator I = mat_seq->diag_starts->begin(),
        	     E = mat_seq->diag_starts->end(); I != E; ++I) 
		  {
	        	/* Ignore 0 padding */
		        offset = I->first;

		        if (offset + iblock + (rank*nrows) < 0) 
			{
	        	  continue;
        		}

	        	if (offset + iblock + (rank*nrows) >= ((nrows*numprocs)/dof)) 
			{
        	  	break;
        		}
	        
	        	for (j=0;j<dof;j++) 
			{
	        	  index = i + I->second + j*nrows; // column-major
	        	  col = offset*dof+((iblock+(rank*nrows))*dof) + j;
	                  fprintf(stdout," (%d, %g) ",col,mat_seq->hostData[index]);
        		}
      	 	  }
		fprintf(stdout,"\n");
    		}
  	}

}
