#include "hip/hip_runtime.h"
#include <../src/mat/impls/sggpu/mpi/mpisggpu.h>


// Direct access to seqgpu vector type
#include "../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h"

// Interop with CUSP vector
#include "../src/vec/vec/impls/seq/seqcusp/cuspvecimpl.h"

#include "cuPrintf.cu"

#define BLOCKWIDTH_X 128
#define BLOCKWIDTH_Y 1




//===-- CUDA Device Code -------------------------------------------------===//
 
texture<int2, 1> vector_x;
     
static __inline__ __device__ double fetch_doubleMPI(texture<int2, 1> tex, int i)
     {
       int2 v = tex1Dfetch(tex, i);
       return __hiloint2double(v.y, v.x);
     }
     
__global__ void MatMultKernelMPI(PetscScalar * coeff, PetscScalar * y, PetscScalar *x,PetscInt mat_size, PetscInt num_diags, int * diagonals, PetscInt dof) {
       
int idx = blockDim.x * blockIdx.x * 1 + threadIdx.x * 1;

     if (idx >= mat_size)
      return;
     
int diag_size = mat_size * dof;
     
PetscScalar yval0 = 0.0;
int idx0 = idx;


//#pragma unroll 4
for (int i = 0; i < num_diags; ++i) 
{
    int d = diagonals[i];
    
    int offset0 = diag_size * i + idx0;
    int block0 = (idx0/dof + d) * dof;
     
    //#pragma unroll 12
    for (int j = 0; j < dof; ++j) 
	{
	      // Get coefficient
	      PetscScalar aval0 = coeff[offset0 + mat_size*j];
	      // Get X value
	      ///PetscScalar xval0 = fetch_doubleMPI(vector_x, block0 + j);
	      PetscScalar xval0 = x[block0 + j];

	      yval0 += aval0 * xval0;

	      //21, 22, 25, 26, 37, 38, 41, 42 are the only non zero entries in the 64x1 vector for ex14 when grid size is 4x4x4 	
	      //The following if statement is for testing and debugging and can be removed. 
	      //if ((idx0 == 21) || (idx0 == 22) || (idx0 == 25) || (idx0 == 26) ||  (idx0 == 37) ||  (idx0 == 38) ||  (idx0 == 41) ||  (idx0 == 42))
	      //  cuPrintf("d:%d \t offset:%d \t block0:%d \t aval0:%lf \t xval0:%lf \t yval0 :%lf\n",d,offset0,block0,aval0,xval0, yval0);
        }
}
   
      y[idx0] = yval0;

//      cuPrintf("y[%d]:%g\n",idx0,y[idx0]);

      }
    
//===-- Host Code --------------------------------------------------------===//





// Matrix function table
static struct _MatOps MatOps_Values = {
/*0*/ MatSetValues_MPISGGPU,MatGetRow_MPISGGPU,MatRestoreRow_MPISGGPU,MatMult_MPISGGPU,0,
/*5*/0,0,0,0,0,
/*10*/0,0,0,0,0,
/*15*/0,0,MatGetDiagonal_MPISGGPU,MatDiagonalScale_MPISGGPU,0,
/*20*/MatAssemblyBegin_MPISGGPU,MatAssemblyEnd_MPISGGPU,0,MatZeroEntries_MPISGGPU,0,
/*25*/0,0,0,0,MatSetUp_MPISGGPU,
/*30*/0,0,0,0,0,
/*35*/0,0,0,0,0,
/*40*/0,0,0,0,0,
/*45*/0,0,0,0,0,
/*50*/0,0,MatGetColumnIJ_MPISGGPU,0,MatFDColoringCreate_MPISGGPU,
/*55*/0,0,0,MatSetValuesBlocked_MPISGGPU,0,
/*60*/MatDestroy_MPISGGPU,MatView_MPISGGPU,0,0,0,
/*65*/0,0,MatSetValues_MPISGGPU,0,MatGetRowMaxAbs_MPISGGPU,
/*70*/0,0,0,0,0,
/*75*/MatFDColoringApply_MPISGGPU,0,0,0,0,
/*80*/0,0,0,0,0,
/*85*/0,0,MatSetValuesBlocked_MPISGGPU,0,0,
/*90*/0,0,0,0,0,
/*95*/0,0,0,0,0,
/*100*/0,0,0,0,0,
/*105*/0,0,0,0,0,
/*110*/0,0,0,0,0,
/*115*/MatCreate_MPISGGPU,0,0,0,0,
/*120*/0,0,0,0,0,
/*125*/0,0,0,0,0,
/*130*/0,0,0,0,0,
/*135*/0,0,0,0,0,
/*140*/0,0,
/*142*/MatSetGrid_MPISGGPU
};



EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_MPISGGPU"
PetscErrorCode MatCreate_MPISGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  SGTrace;

  PetscInt rank;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);


  // Create internal matrix structure
  ierr = PetscMalloc(sizeof(Mat_SeqSGGPU), &mat); CHKERRQ(ierr);
  memset(mat, 0, sizeof(Mat_SeqSGGPU));
  mat->diag_starts = new std::map<int, int>();
  mat->diagonals = new std::vector<int>();

  checkCudaError(hipStreamCreate(&mat->stream));

  // Fill out PETSc matrix structure
  A->data = mat;
  memcpy(A->ops, &MatOps_Values, sizeof(struct _MatOps));
  A->same_nonzero= PETSC_FALSE;
  A->spptr = 0;

  // Set object type
  ierr = PetscObjectChangeTypeName((PetscObject)A, MATMPISGGPU); CHKERRQ(ierr);

  ierr = PetscObjectComposeFunctionDynamic((PetscObject)A,
        "MatMPISGGPUSetPreallocation_C","MatMPISGGPUSetPreallocation_MPIDIA",
        MatMPISGGPUSetPreallocation_MPISGGPU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
EXTERN_C_END




#undef __FUNCT__
#define __FUNCT__ "MatDestroy_MPISGGPU"
PetscErrorCode MatDestroy_MPISGGPU(Mat A) {

  PetscInt rank;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);

  MatDestroy_SeqSGGPU(A);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetGrid_MPISGGPU"
PetscErrorCode MatSetGrid_MPISGGPU(Mat B, PetscInt m, PetscInt n, PetscInt p) 
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)B->data;

  PetscFunctionBegin;
  SGTrace;

  PetscInt rank;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);

  mat->m = m;
  mat->n = n;
  mat->p = p;

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatMult_MPISGGPU"
PetscErrorCode MatMult_MPISGGPU(Mat A, Vec x, Vec y) {

  Mat_SeqSGGPU *mat = (Mat_SeqSGGPU*)A->data;

//  MatView_MPISGGPU(A,PETSC_VIEWER_STDOUT_WORLD);	

  PetscInt rank;
  PetscInt numprocs;	
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  MPI_Comm_size(PETSC_COMM_WORLD,&numprocs);	

  PetscBool isseqcusp,isseqgpu,ismpicusp,iscusp;
  PetscErrorCode ierr;
  PetscInt mat_size;
  CUSPARRAY *xgpu,*ygpu;
  PetscScalar *devX,*devY;

  PetscFunctionBegin;
  SGTrace;

  // Initialize y to zero
  ierr = VecSet(y, 0.0); CHKERRQ(ierr);

  ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQCUSP,&isseqcusp);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)x,VECMPICUSP,&ismpicusp);CHKERRQ(ierr);
  iscusp = (isseqcusp || ismpicusp) ? PETSC_TRUE : PETSC_FALSE;
  ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQGPU,&isseqgpu);CHKERRQ(ierr);
  if (isseqgpu) {
    dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
    dim3 grid((int)ceil((float)((mat->m * mat->n * mat->p * mat->dof)/numprocs)/(float)BLOCKWIDTH_X / 1.0), 1);

    int shared_size = 0;
    Vec_SeqGPU *vx = (Vec_SeqGPU*) x->data;
    Vec_SeqGPU *vy = (Vec_SeqGPU*) y->data;
    /* Make sure y is also VECSEQGPU */
    ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQGPU,&isseqgpu);CHKERRQ(ierr);
    if (!isseqgpu) {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Both x and y must be same type");
    }
    /* synch up x */
    if (vx->syncState==VEC_CPU) {
      ierr = VecCopyOverH2D(x,vx->cpuptr);CHKERRQ(ierr);
      vx->syncState=VEC_SYNCHED;
    }
    /* Get device pointer for X */
    devX = vx->devptr;
    devY = vy->devptr;
    /* Bind X to device texture */
    mat_size = (mat->m * mat->n * mat->p * mat->dof)/numprocs;

    checkCudaError(hipBindTexture(0, vector_x, devX, mat_size * sizeof(PetscScalar)));

    MatMultKernelMPI<<<grid, block, shared_size, mat->stream>>>(mat->deviceData, devY, devX, mat_size, mat->diagonals->size(), mat->deviceDiags, mat->dof);

    hipUnbindTexture(vector_x);
    hipDeviceSynchronize();


  } else if (iscusp) {
    dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
    dim3 grid((int)ceil((float)((mat->m * mat->n * mat->p * mat->dof)/numprocs)/(float)BLOCKWIDTH_X / 1.0), 1);

    int shared_size = 0;
    /* Make sure y is also VECCUSP */
    ierr = PetscObjectTypeCompare((PetscObject)x,VECCUSP,&isseqgpu);CHKERRQ(ierr);
    if (!iscusp) 
    {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Both x and y must be same type");
    }
	
    mat_size = ((mat->m * mat->n * mat->p * mat->dof)/numprocs);

   // Vec xx;
   // ierr = VecCreateSeq(PETSC_COMM_SELF,(mat->m*mat->n*mat->p*mat->dof),&xx);
   // VecCopy(x,xx);	 
   // VecView(x,PETSC_VIEWER_STDOUT_WORLD);
   // VecView(xx,PETSC_VIEWER_STDOUT_WORLD);

    ierr = VecCUSPGetArrayWrite(y, &ygpu); CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(x, &xgpu); CHKERRQ(ierr);
    devY = thrust::raw_pointer_cast(&(*ygpu)[0]);
    devX = thrust::raw_pointer_cast(&(*xgpu)[0]);

    /* Bind X to device texture */
    checkCudaError(hipBindTexture(0, vector_x, devX, mat_size * sizeof(PetscScalar)));

#if _TRACE
    printf("Host diagonals:\n");
    for (int i = 0; i < mat->diagonals->size(); ++i) {
      printf("- %d\n", (*mat->diagonals)[i]);
    }
#endif

    /* Invoke */

#if _TIME
    double start, end;
    start = getclock();
#endif

cudaPrintfInit();
MatMultKernelMPI<<<grid, block, shared_size, mat->stream>>>(mat->deviceData, devY, devX, mat_size, mat->diagonals->size(), mat->deviceDiags, mat->dof);
cudaPrintfDisplay(stdout,true);
cudaPrintfEnd();

#if _TIME
    checkCudaError(hipStreamSynchronize(mat->stream));
    end = getclock();
    double elapsed = end - start;
    double gflops = (2.0 * mat->non_zeros / elapsed / 1e9);

    double nos = ((mat->p == 1 ? 2 : 3) * 2 + 1) * (2*mat->dof - 1);
    double nz = (mat->m * mat->n * mat->p * mat->dof)/numprocs;
    double alt_gflops = (2.0 * nos * nz) / ((end - start)*1024*1024*1024);

#if _CSV_OUT
    fprintf(stderr, "%d,%d,%d,%d,%lf,%lf,\n", (mat->m, mat->n, mat->p, mat->dof)/numprocs, elapsed, gflops);
#endif
    printf("SGGPU Kernel Time:           %lf sec\n", elapsed);
    printf("SGGPU Kernel GFlop/s:        %lf\n", gflops);
    printf("SGGPU Kernel GFlop/s (alt):  %lf\n", alt_gflops);
#endif

    /* Cleanup */
	  hipUnbindTexture(vector_x);

    ierr = VecCUSPRestoreArrayRead(x, &xgpu); CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(y, &ygpu); CHKERRQ(ierr);
    ierr = WaitForGPU() ; CHKERRCUSP(ierr);
    hipDeviceSynchronize();
  } else {
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Vec must be seqgpu or cusp type");
  }

//	VecView(x,PETSC_VIEWER_STDOUT_WORLD);
//	VecView(y,PETSC_VIEWER_STDOUT_WORLD);

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValuesBlocked_MPISGGPU"
PetscErrorCode MatSetValuesBlocked_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) {
  MatSetValuesBlocked_SeqSGGPU(A, nrow, irow, ncol, icol, y, is);

  PetscInt rank;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValues_MPISGGPU"
PetscErrorCode MatSetValues_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) {

  int i, j;
  PetscErrorCode ierr;
  PetscBool resizegpu = PETSC_FALSE;
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;

  PetscInt row, col;	
  PetscInt rank;
  PetscInt numprocs; 	

  PetscInt buf[8192],*bufr=0,*bufc=0,*irowm,*icolm;

  PetscFunctionBegin;
  SGTrace;

  PetscInt rstart = A->rmap->rstart, rend = A->rmap->rend;
  PetscInt cstart = A->cmap->rstart, cend = A->cmap->rend;  
 
    if ((nrow+ncol) <= (PetscInt)(sizeof(buf)/sizeof(PetscInt))) 
	{
	      irowm = buf; icolm = buf+nrow;
        } 
    else 
	{
	      ierr = PetscMalloc2(nrow,PetscInt,&bufr,ncol,PetscInt,&bufc);CHKERRQ(ierr);
	      irowm = bufr; icolm = bufc;
	}
   
   MPI_Comm_rank(PETSC_COMM_WORLD, &rank);
   MPI_Comm_size(PETSC_COMM_WORLD, &numprocs);

   ierr = ISLocalToGlobalMappingApply(A->rmap->mapping,nrow,irow,irowm);CHKERRQ(ierr);
   ierr = ISLocalToGlobalMappingApply(A->cmap->mapping,ncol,icol,icolm);CHKERRQ(ierr);

   // Handle each element
   for (i = 0; i < nrow; i++) {
	if (irowm[i] < 0) continue;
	        row = irowm[i]; 

   for (j = 0; j < ncol; j++) {
  	
	if(irowm[i] >= rstart && irowm[i] < rend) 
	{
		if (icolm[j] >= cstart && icolm[j] < cend) 
		{
	        	col = icolm[j]; //-cstart;
		} 
		else if (icolm[j] < 0) 
			continue;
		else
			col = icolm[j];


      	      // Compute the diagonal and offset into the diagonal storage
	      // for the element
	      //int row = irow[i];
	      //int col = icol[j];
      
	      int diff = col - row;
	      int left = row % mat->dof;
	      int diag = int(floor((double)(diff + left) / mat->dof));
	      int col_offset = col % mat->dof;
	      int num_elems = (mat->m * mat->n * mat->p * mat->dof)/numprocs;
	      int offset = col_offset * num_elems + row - (rank*num_elems);

#if _TRACE
      printf("- row: %d  col: %d  val: %lf  diag: %d  offset: %d\n", row, col, y[i*ncol+j], diag, offset);
#endif

      std::map<int, int> &diag_starts = *(mat->diag_starts);
      std::map<int, int>::iterator I = diag_starts.find(diag);
      int diag_offset = 0;
      if (I == diag_starts.end()) {
        printf("WARNING: malloc() in MatSetValues\n");
        resizegpu = PETSC_TRUE;
        // The diagonal does not yet exist, so add a new diagonal
        int num_diags = diag_starts.size() + 1;
        int size = num_diags * ((mat->m * mat->n * mat->p * mat->dof * mat->dof)/numprocs);
        PetscScalar *newData;
        ierr = PetscMalloc(size * sizeof(PetscScalar), &newData); CHKERRQ(ierr);
        memset(newData, 0, size * sizeof(PetscScalar));
        size -= ((mat->m * mat->n * mat->p * mat->dof * mat->dof)/numprocs);
        if (num_diags > 1) {
          // This is not the first diagonal, so copy
#if _TRACE
          printf("- Memcpy of %d elements\n", size);
#endif
          memcpy(newData, mat->hostData, size * sizeof(PetscScalar));
        }
        PetscFree(mat->hostData);
        mat->hostData = newData;
        diag_offset = size;
        diag_starts[diag] = diag_offset;
        mat->diagonals->push_back(diag);
      }
	
	else 
      {
        // The diagonal already exists, so get the base offset
        diag_offset = I->second;
      }

      diag_offset += offset;

      if (is == INSERT_VALUES)
        mat->hostData[diag_offset] = y[i * ncol + j];
      else
        mat->hostData[diag_offset] += y[i * ncol + j];

      mat->non_zeros++;
	}
    }
  }

  if (resizegpu) {
    int size,mat_size;
    // Create GPU buffer
    if (mat->deviceData) {
      hipFree(mat->deviceData);
    }
    size = mat->diag_starts->size() * ((mat->m * mat->n * mat->p * mat->dof * mat->dof)/numprocs);
    checkCudaError(hipMalloc(&mat->deviceData, sizeof(PetscScalar) * size));

    mat_size = (mat->m * mat->n * mat->p * mat->dof)/numprocs;

    if (mat->deviceX) {
      hipFree(mat->deviceX);
    }
    if (mat->deviceY) {
      hipFree(mat->deviceY);
    }
    if (mat->deviceDiags) {
      hipFree(mat->deviceDiags);
    }
    // We know the expected size of x, y, so go ahead and allocate them now
    checkCudaError(hipMalloc(&mat->deviceX, mat_size * sizeof(PetscScalar)));
    checkCudaError(hipMalloc(&mat->deviceY, mat_size * sizeof(PetscScalar)));

    // We also know how many diagonals we have, and their indices
    checkCudaError(hipMalloc(&mat->deviceDiags, sizeof(int) * mat->diagonals->size()));
  }

  PetscFunctionReturn(0);

}


//#undef __FUNCT__
//#define __FUNCT__ "MatSetStencil_MPISGGPU"
//PetscErrorCode MatSetStencil_MPISGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof) {
//  MatSetStencil_SeqSGGPU(A, dim, dims, starts, dof);
//  PetscFunctionReturn(0);
//}


#undef __FUNCT__
#define __FUNCT__ "MatSetUp_MPISGGPU"
PetscErrorCode MatSetUp_MPISGGPU(Mat mat) {

	PetscInt rank;
        MPI_Comm_rank(PETSC_COMM_WORLD,&rank);

  MatSetUp_SeqSGGPU(mat);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatZeroEntries_MPISGGPU"
PetscErrorCode MatZeroEntries_MPISGGPU(Mat A) {

  Mat_SeqSGGPU *mat = (Mat_SeqSGGPU*)A->data;
  PetscInt size;
  PetscInt rank, numprocs;
  PetscFunctionBegin;
  SGTrace;
  
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  MPI_Comm_size(PETSC_COMM_WORLD,&numprocs);

  size = mat->diag_starts->size() * ((mat->m * mat->n * mat->p * mat->dof * mat->dof)/numprocs);
  memset(mat->hostData, 0, size * sizeof(PetscScalar));
  
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "MatGetDiagonal_MPISGGPU"
PetscErrorCode MatGetDiagonal_MPISGGPU(Mat A, Vec v) {

	PetscInt rank;
        MPI_Comm_rank(PETSC_COMM_WORLD,&rank);

  MatGetDiagonal_SeqSGGPU(A, v);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatDiagonalScale_MPISGGPU"
PetscErrorCode MatDiagonalScale_MPISGGPU(Mat A, Vec ll, Vec rr) {

	PetscInt rank;
        MPI_Comm_rank(PETSC_COMM_WORLD,&rank);

  MatDiagonalScale_SeqSGGPU(A, ll, rr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRow_MPISGGPU"
PetscErrorCode MatGetRow_MPISGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v) {

	PetscInt rank;
        MPI_Comm_rank(PETSC_COMM_WORLD,&rank);

  MatGetRow_SeqSGGPU(A, row, nz, idx , v);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatRestoreRow_MPISGGPU"
PetscErrorCode MatRestoreRow_MPISGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v) {
  MatRestoreRow_SeqSGGPU(A, row, nz, idx, v);

	PetscInt rank;
        MPI_Comm_rank(PETSC_COMM_WORLD,&rank);

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRowMaxAbs_MPISGGPU"
PetscErrorCode MatGetRowMaxAbs_MPISGGPU(Mat A, Vec v, PetscInt idx[]) {

	PetscInt rank;
        MPI_Comm_rank(PETSC_COMM_WORLD,&rank);

  MatGetRowMaxAbs_SeqSGGPU(A, v, idx);
  PetscFunctionReturn(0);
}


void DisplayLocalMatrix(Mat A);

#undef __FUNCT__
#define __FUNCT__ "MatView_MPISGGPU"
PetscErrorCode MatView_MPISGGPU(Mat A, PetscViewer viewer) 
{

  PetscInt rank, i, numprocs;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  MPI_Comm_size(PETSC_COMM_WORLD,&numprocs);	
	
  for(i = 0; i < numprocs;++i)
	{
	 if(i == rank)
	    DisplayLocalMatrix(A);
         MPI_Barrier(PETSC_COMM_WORLD);   
    	}
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyBegin_MPISGGPU"
PetscErrorCode MatAssemblyBegin_MPISGGPU(Mat A, MatAssemblyType type) {
  MatAssemblyBegin_SeqSGGPU(A, type);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_MPISGGPU"
PetscErrorCode MatAssemblyEnd_MPISGGPU(Mat A, MatAssemblyType type) {
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;
  PetscInt size;
  PetscInt numprocs;
  PetscFunctionBegin;

  MPI_Comm_size(PETSC_COMM_WORLD,&numprocs);	

#if _TRACE
  printf("[SeqSGGPU] MatAssemblyEnd_SeqSGGPU\n");

  for (std::map<int, int>::iterator I = mat->diag_starts->begin(),
       E = mat->diag_starts->end(); I != E; ++I) {
    printf("- Diag %d:\n", I->first);
    for (int i = 0; i < mat->dof; ++i) {
      for (int j = 0; j < (mat->dof * mat->m * mat->n * mat->p)/numprocs; ++j) {
        int offset = i * ((mat->dof * mat->m * mat->n * mat->p)/numprocs) + j;
        printf(" %lf ", mat->hostData[offset + I->second]);
      }
      printf("\n");
    }
  }
#endif

  size = (mat->diag_starts->size()*mat->m*mat->n*mat->p*mat->dof*mat->dof)/numprocs;

  checkCudaError(hipMemcpyAsync(mat->deviceDiags, &(*mat->diagonals)[0], sizeof(int) * mat->diagonals->size(), hipMemcpyHostToDevice, mat->stream));

  checkCudaError(hipMemcpy(mat->deviceData, mat->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));


  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "MatFDColoringApply_MPISGGPU"
PetscErrorCode  MatFDColoringApply_MPISGGPU(Mat J,MatFDColoring coloring,Vec x1,MatStructure *flag,void *sctx) {
  MatFDColoringApply_SeqSGGPU(J, coloring, x1, flag, sctx);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatFDColoringCreate_MPISGGPU"
PetscErrorCode MatFDColoringCreate_MPISGGPU(Mat mat,ISColoring iscoloring,MatFDColoring c) {
  MatFDColoringCreate_SeqSGGPU(mat, iscoloring, c);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetColumnIJ_MPISGGPU"
PetscErrorCode MatGetColumnIJ_MPISGGPU(Mat A,PetscInt oshift,PetscBool  symmetric,PetscBool  inodecompressed,PetscInt *nn, const PetscInt *ia[], const PetscInt *ja[],PetscBool  *done) {
  MatGetColumnIJ_SeqSGGPU(A, oshift, symmetric, inodecompressed, nn, ia, ja, done);
  PetscFunctionReturn(0);
}


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMPISGGPUSetPreallocation"
PetscErrorCode MatMPISGGPUSetPreallocation(Mat A,PetscInt stencil_type, PetscInt dof)
{
  PetscErrorCode ierr;
  Mat_SeqSGGPU *mat = (Mat_SeqSGGPU*)A->data;

  PetscFunctionBegin;

  mat->stencil_type = stencil_type;
  mat->dof = dof;
  if(A->preallocated)PetscFunctionReturn(0);
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  
  ierr = PetscTryMethod(A,"MatMPISGGPUSetPreallocation_C",(Mat,PetscInt,const PetscInt []),(A,0,0));CHKERRQ(ierr);
  A->preallocated=PETSC_TRUE;
  PetscFunctionReturn(0);
}
EXTERN_C_END


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMPISGGPUSetPreallocation_MPISGGPU"
extern PetscErrorCode MatMPISGGPUSetPreallocation_MPISGGPU(Mat A,PetscInt nz, const PetscInt nnz[])
{
  PetscErrorCode ierr;
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;

  PetscInt dim,diag_size,size,num_diags,i,vecsize;
  PetscInt rank;
  PetscInt numprocs;

  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  MPI_Comm_size(PETSC_COMM_WORLD,&numprocs);

  ierr = PetscLayoutSetBlockSize(A->rmap,1);CHKERRQ(ierr);
  ierr = PetscLayoutSetBlockSize(A->cmap,1);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->cmap);CHKERRQ(ierr);

  dim = A->stencil.dim;
  if (mat->dof > 1) {
    dim--;
  }

//  mat->m = mat->n = mat->p = 1;
  mat->dim = dim;
//  if (mat->dim > 0) mat->m = A->stencil.dims[dim-1];
//  if (mat->dim > 1) mat->n = A->stencil.dims[dim-2];
//  if (mat->dim > 2) mat->p = A->stencil.dims[dim-3];

  if (mat->stencil_type == 0) {
    /* star stencil */
    num_diags = 2*mat->dim + 1;
  } else {
    /* box stencil */
    num_diags =  1;
    for (i=0;i<mat->dim;i++) num_diags*=3;
  }

  diag_size = (mat->m * mat->n * mat->p * mat->dof * mat->dof)/numprocs;
  size = num_diags * diag_size;

  if (mat->m == 0 || mat->n == 0 || mat->p == 0 || mat->dof == 0) {
    SETERRQ(PETSC_COMM_SELF,0,"MatSetPreallocation_SeqSGGPU called without valid m, n, p, and dof!");
  }

 
  ierr = PetscMalloc(sizeof(PetscInt)*num_diags,&mat->diag_offsets);
  ierr = PetscMalloc(size * sizeof(PetscScalar), &mat->hostData); CHKERRQ(ierr);
  memset(mat->hostData, 0, size * sizeof(PetscScalar));

  (*mat->diag_starts)[0]  = 0 * diag_size;
  (*mat->diagonals).push_back(0);
  (*mat->diag_starts)[1]  = 1 * diag_size;
  (*mat->diagonals).push_back(1);
  (*mat->diag_starts)[-1] = 2 * diag_size;
  (*mat->diagonals).push_back(-1);
  if (mat->stencil_type == 0) {
    if (mat->dim == 2) 
    {
      (*mat->diag_starts)[mat->m] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->m] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);

    } 
    else if (mat->dim == 3) 
    {

      (*mat->diag_starts)[mat->m] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->m] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);

      (*mat->diag_starts)[mat->m*mat->n] = 5 * diag_size;
      (*mat->diagonals).push_back(mat->m*mat->n);
      (*mat->diag_starts)[-mat->m*mat->n] = 6 * diag_size;
      (*mat->diagonals).push_back(-mat->m*mat->n);

    }
  } else {
    if (mat->dim == 2) {
      (*mat->diag_starts)[mat->n-1] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n-1] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
      (*mat->diag_starts)[mat->n] = 5 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n] = 6 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
      (*mat->diag_starts)[mat->n+1] = 7 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n+1] = 8 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
    }
  }
  /*
  printf("Diagonals preallocated:\n");
  for (std::map<int, int>::iterator I = mat->diag_starts->begin(),
         E = mat->diag_starts->end(); I != E; ++I) {
    printf("%4d --> %4d\n",I->first,I->second);
  }
   */
  
  
  // Create GPU buffer
  if (mat->deviceData) {
    hipFree(mat->deviceData);
  }
  checkCudaError(hipMalloc(&mat->deviceData, sizeof(PetscScalar) * size));
  checkCudaError(hipMemset(mat->deviceData,0.0,sizeof(PetscScalar)*size));

  // Copy data to device
  checkCudaError(hipMemcpy(mat->deviceData, mat->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));

  vecsize = (mat->m * mat->n * mat->p * mat->dof)/numprocs;

  // We know the expected size of x, y, so go ahead and allocate them now
  checkCudaError(hipMalloc(&mat->deviceX, vecsize * sizeof(PetscScalar)));
  checkCudaError(hipMalloc(&mat->deviceY, vecsize * sizeof(PetscScalar)));

  // We also know how many diagonals we have, and their indices
  checkCudaError(hipMalloc(&mat->deviceDiags, sizeof(int) * mat->diagonals->size()));
  A->preallocated = PETSC_TRUE;
  ierr = MatAssemblyBegin(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyEnd(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  
//  MatSetUpMultiply_MPISGGPU(A);

  PetscFunctionReturn(0);
}
EXTERN_C_END



void DisplayLocalMatrix(Mat A)
{

  Mat_SeqSGGPU *a;
  a  = (Mat_SeqSGGPU*)A->data;
  PetscErrorCode ierr;
  PetscInt nrows,ndiag,dof,i,j,iblock,col,index,offset;
  std::map<int, int> &diag_starts = *(a->diag_starts);

  PetscInt numprocs, rank;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  MPI_Comm_size(PETSC_COMM_WORLD,&numprocs);

  nrows = (a->m * a->n * a->p * a->dof)/numprocs;
  ndiag = a->diagonals->size();
  dof = a->dof;

  for (std::map<int, int>::iterator I = diag_starts.begin(),
         E = diag_starts.end(); I != E; ++I) 
	{
		fprintf(stdout,"- Diag %d:%d\n", I->first, I->second);
  	}

	fprintf(stdout,"\n");
	fprintf(stdout,"hostData:\n");

  for (i=0;i<nrows;i++) 
	{
	fprintf(stdout,"row %2.2d:",rank*nrows + i); 

	    for (j=0;j<ndiag*dof;j++) 
		{
			fprintf(stdout," %4g ",a->hostData[i+j*nrows]);
		}
	fprintf(stdout,"\n");
  	}
	fprintf(stdout,"\n\n");


  for (iblock=0;iblock<(nrows/dof);iblock++)  
	{
	    for (i=iblock*dof;i<(iblock+1)*dof;i++) 
		{
		fprintf(stdout,"row %d:",rank*nrows + i);
	
		  for (std::map<int, int>::iterator I = a->diag_starts->begin(),
        	     E = a->diag_starts->end(); I != E; ++I) 
		  {
	        	/* Ignore 0 padding */
		        offset = I->first;

		        if (offset + iblock + (rank*nrows) < 0) 
			{
	        	  continue;
        		}

	        	if (offset + iblock + (rank*nrows) >= ((nrows*numprocs)/dof)) 
			{
        	  	break;
        		}
	        
	        	for (j=0;j<dof;j++) 
			{
	        	  index = i + I->second + j*nrows; // column-major
	        	  col = offset*dof+((iblock+(rank*nrows))*dof) + j;
	                  fprintf(stdout," (%d, %g) ",col,a->hostData[index]);
        		}
      	 	  }
		fprintf(stdout,"\n");
    		}
  	}

}