#include <../src/mat/impls/sggpu/mpi/mpisggpu.h>


// Matrix function table
static struct _MatOps MatOps_Values = {
/*0*/ MatSetValues_MPISGGPU,MatGetRow_MPISGGPU,MatRestoreRow_MPISGGPU,MatMult_MPISGGPU,0,
/*5*/0,0,0,0,0,
/*10*/0,0,0,0,0,
/*15*/0,0,MatGetDiagonal_MPISGGPU,MatDiagonalScale_MPISGGPU,0,
/*20*/MatAssemblyBegin_MPISGGPU,MatAssemblyEnd_MPISGGPU,0,MatZeroEntries_MPISGGPU,0,
/*25*/0,0,0,0,MatSetUp_MPISGGPU,
/*30*/0,0,0,0,0,
/*35*/0,0,0,0,0,
/*40*/0,0,0,0,0,
/*45*/0,0,0,0,0,
/*50*/0,0,MatGetColumnIJ_MPISGGPU,0,MatFDColoringCreate_MPISGGPU,
/*55*/0,0,0,MatSetValuesBlocked_MPISGGPU,0,
/*60*/MatDestroy_MPISGGPU,MatView_MPISGGPU,0,0,0,
/*65*/0,0,MatSetValues_MPISGGPU,0,MatGetRowMaxAbs_MPISGGPU,
/*70*/0,0,0,0,0,
/*75*/MatFDColoringApply_MPISGGPU,0,0,0,0,
/*80*/0,0,0,0,0,
/*85*/0,0,MatSetValuesBlocked_MPISGGPU,0,0,
/*90*/0,0,0,0,0,
/*95*/0,0,0,0,0,
/*100*/0,0,0,0,0,
/*105*/0,0,0,0,0,
/*110*/0,0,0,0,0,
/*115*/MatCreate_MPISGGPU,0,0,0,0,
/*120*/0,0,0,0,0,
/*125*/0,0,0,0,0,
/*130*/0,0,0,0,0,
/*135*/0,0,0,0,0,
/*140*/0,0,
/*142*/MatSetGrid_MPISGGPU
};



EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_MPISGGPU"
PetscErrorCode MatCreate_MPISGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;
  PetscMPIInt size;

  PetscFunctionBegin;
  SGTrace;

	PetscPrintf(PETSC_COMM_WORLD,"MatCreate_MPISGGPU\n");
	

  ierr = MPI_Comm_size(((PetscObject)A)->comm, &size); CHKERRQ(ierr);
  if (size > 1)
    SETERRQ(PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "Comm must be size 1");

  // Create internal matrix structure
  ierr = PetscMalloc(sizeof(Mat_SeqSGGPU), &mat); CHKERRQ(ierr);
  memset(mat, 0, sizeof(Mat_SeqSGGPU));
  mat->diag_starts = new std::map<int, int>();
  mat->diagonals = new std::vector<int>();

  checkCudaError(hipStreamCreate(&mat->stream));

  // Fill out PETSc matrix structure
  A->data = mat;
  memcpy(A->ops, &MatOps_Values, sizeof(struct _MatOps));
  A->same_nonzero= PETSC_FALSE;
  A->spptr = 0;

  // Set object type
  ierr = PetscObjectChangeTypeName((PetscObject)A, MATMPISGGPU); CHKERRQ(ierr);

  ierr = PetscObjectComposeFunctionDynamic((PetscObject)A,
        "MatSeqSGGPUSetPreallocation_C","MatSeqSGGPUSetPreallocation_SeqDIA",
        MatSeqSGGPUSetPreallocation_SeqSGGPU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
EXTERN_C_END




#undef __FUNCT__
#define __FUNCT__ "MatDestroy_MPISGGPU"
PetscErrorCode MatDestroy_MPISGGPU(Mat A) {
  MatDestroy_SeqSGGPU(A);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetGrid_MPISGGPU"
PetscErrorCode MatSetGrid_MPISGGPU(Mat B, PetscInt m, PetscInt n, PetscInt p) {
  MatSetGrid_SeqSGGPU(B, m, n, p);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatMult_MPISGGPU"
PetscErrorCode MatMult_MPISGGPU(Mat mat, Vec x, Vec y) {
  MatMult_SeqSGGPU(mat, x, y); 
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValuesBlocked_MPISGGPU"
PetscErrorCode MatSetValuesBlocked_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) {
  MatSetValuesBlocked_SeqSGGPU(A, nrow, irow, ncol, icol, y, is);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValues_MPISGGPU"
PetscErrorCode MatSetValues_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) {
  MatSetValues_SeqSGGPU(A, nrow, irow, ncol, icol, y, is);
  PetscFunctionReturn(0);
}


//#undef __FUNCT__
//#define __FUNCT__ "MatSetStencil_MPISGGPU"
//PetscErrorCode MatSetStencil_MPISGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof) {
//  MatSetStencil_SeqSGGPU(A, dim, dims, starts, dof);
//  PetscFunctionReturn(0);
//}


#undef __FUNCT__
#define __FUNCT__ "MatSetUp_MPISGGPU"
PetscErrorCode MatSetUp_MPISGGPU(Mat mat) {
  MatSetUp_SeqSGGPU(mat);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatZeroEntries_MPISGGPU"
PetscErrorCode MatZeroEntries_MPISGGPU(Mat A) {
  MatZeroEntries_SeqSGGPU(A);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetDiagonal_MPISGGPU"
PetscErrorCode MatGetDiagonal_MPISGGPU(Mat A, Vec v) {
  MatGetDiagonal_SeqSGGPU(A, v);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatDiagonalScale_MPISGGPU"
PetscErrorCode MatDiagonalScale_MPISGGPU(Mat A, Vec ll, Vec rr) {
  MatDiagonalScale_SeqSGGPU(A, ll, rr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRow_MPISGGPU"
PetscErrorCode MatGetRow_MPISGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v) {
  MatGetRow_SeqSGGPU(A, row, nz, idx , v);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatRestoreRow_MPISGGPU"
PetscErrorCode MatRestoreRow_MPISGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v) {
  MatRestoreRow_SeqSGGPU(A, row, nz, idx, v);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRowMaxAbs_MPISGGPU"
PetscErrorCode MatGetRowMaxAbs_MPISGGPU(Mat A, Vec v, PetscInt idx[]) {
  MatGetRowMaxAbs_SeqSGGPU(A, v, idx);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatView_MPISGGPU"
PetscErrorCode MatView_MPISGGPU(Mat A, PetscViewer viewer) {
  MatView_SeqSGGPU(A, viewer);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyBegin_MPISGGPU"
PetscErrorCode MatAssemblyBegin_MPISGGPU(Mat A, MatAssemblyType type) {
  MatAssemblyBegin_SeqSGGPU(A, type);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_MPISGGPU"
PetscErrorCode MatAssemblyEnd_MPISGGPU(Mat A, MatAssemblyType type) {
  MatAssemblyEnd_SeqSGGPU(A, type);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatView_MPISGGPU_ASCII"
PetscErrorCode MatView_MPISGGPU_ASCII(Mat A, PetscViewer viewer) {
  MatView_SeqSGGPU_ASCII(A, viewer);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatFDColoringApply_MPISGGPU"
PetscErrorCode  MatFDColoringApply_MPISGGPU(Mat J,MatFDColoring coloring,Vec x1,MatStructure *flag,void *sctx) {
  MatFDColoringApply_SeqSGGPU(J, coloring, x1, flag, sctx);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatFDColoringCreate_MPISGGPU"
PetscErrorCode MatFDColoringCreate_MPISGGPU(Mat mat,ISColoring iscoloring,MatFDColoring c) {
  MatFDColoringCreate_SeqSGGPU(mat, iscoloring, c);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetColumnIJ_MPISGGPU"
PetscErrorCode MatGetColumnIJ_MPISGGPU(Mat A,PetscInt oshift,PetscBool  symmetric,PetscBool  inodecompressed,PetscInt *nn, const PetscInt *ia[], const PetscInt *ja[],PetscBool  *done) {
  MatGetColumnIJ_SeqSGGPU(A, oshift, symmetric, inodecompressed, nn, ia, ja, done);
  PetscFunctionReturn(0);
}
