#include <../src/mat/impls/sggpu/mpi/mpisggpu.h>

// Matrix function table
static struct _MatOps MatOps_Values = {
/*0*/ MatSetValues_MPISGGPU,MatGetRow_MPISGGPU,MatRestoreRow_MPISGGPU,MatMult_MPISGGPU,0,
/*5*/0,0,0,0,0,
/*10*/0,0,0,0,0,
/*15*/0,0,MatGetDiagonal_MPISGGPU,MatDiagonalScale_MPISGGPU,0,
/*20*/MatAssemblyBegin_MPISGGPU,MatAssemblyEnd_MPISGGPU,0,MatZeroEntries_MPISGGPU,0,
/*25*/0,0,0,0,MatSetUp_MPISGGPU,
/*30*/0,0,0,0,0,
/*35*/0,0,0,0,0,
/*40*/0,0,0,0,0,
/*45*/0,0,0,0,0,
/*50*/0,0,MatGetColumnIJ_MPISGGPU,0,MatFDColoringCreate_MPISGGPU,
/*55*/0,0,0,MatSetValuesBlocked_MPISGGPU,0,
/*60*/MatDestroy_MPISGGPU,MatView_MPISGGPU,0,0,0,
/*65*/0,0,MatSetValues_MPISGGPU,0,MatGetRowMaxAbs_MPISGGPU,
/*70*/0,0,0,0,0,
/*75*/MatFDColoringApply_MPISGGPU,0,0,0,0,
/*80*/0,0,0,0,0,
/*85*/0,0,MatSetValuesBlocked_MPISGGPU,0,0,
/*90*/0,0,0,0,0,
/*95*/0,0,0,0,0,
/*100*/0,0,0,0,0,
/*105*/0,0,0,0,0,
/*110*/0,0,0,0,0,
/*115*/MatCreate_MPISGGPU,0,0,0,0,
/*120*/0,0,0,0,0,
/*125*/0,0,0,0,0,
/*130*/0,0,0,0,0,
/*135*/0,0,0,0,0,
/*140*/0,0,
/*142*/MatSetGrid_MPISGGPU
};



EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_MPISGGPU"
PetscErrorCode MatCreate_MPISGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;
  PetscMPIInt size;

  PetscFunctionBegin;
  SGTrace;

	PetscPrintf(PETSC_COMM_WORLD,"MatCreate_MPISGGPU\n");
	
//  ierr = MPI_Comm_size(((PetscObject)A)->comm, &size); CHKERRQ(ierr);
//  if (size > 1)
//    SETERRQ(PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "Comm must be size 1");

  // Create internal matrix structure
  ierr = PetscMalloc(sizeof(Mat_SeqSGGPU), &mat); CHKERRQ(ierr);
  memset(mat, 0, sizeof(Mat_SeqSGGPU));
  mat->diag_starts = new std::map<int, int>();
  mat->diagonals = new std::vector<int>();

  checkCudaError(hipStreamCreate(&mat->stream));

  // Fill out PETSc matrix structure
  A->data = mat;
  memcpy(A->ops, &MatOps_Values, sizeof(struct _MatOps));
  A->same_nonzero= PETSC_FALSE;
  A->spptr = 0;

  // Set object type
  ierr = PetscObjectChangeTypeName((PetscObject)A, MATMPISGGPU); CHKERRQ(ierr);

  ierr = PetscObjectComposeFunctionDynamic((PetscObject)A,
        "MatMPISGGPUSetPreallocation_C","MatMPISGGPUSetPreallocation_MPIDIA",
        MatMPISGGPUSetPreallocation_MPISGGPU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
EXTERN_C_END




#undef __FUNCT__
#define __FUNCT__ "MatDestroy_MPISGGPU"
PetscErrorCode MatDestroy_MPISGGPU(Mat A) {
  MatDestroy_SeqSGGPU(A);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetGrid_MPISGGPU"
PetscErrorCode MatSetGrid_MPISGGPU(Mat B, PetscInt m, PetscInt n, PetscInt p) {
  MatSetGrid_SeqSGGPU(B, m, n, p);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatMult_MPISGGPU"
PetscErrorCode MatMult_MPISGGPU(Mat mat, Vec x, Vec y) {

  MatMult_SeqSGGPU(mat, x, y); 

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValuesBlocked_MPISGGPU"
PetscErrorCode MatSetValuesBlocked_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) {
  MatSetValuesBlocked_SeqSGGPU(A, nrow, irow, ncol, icol, y, is);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValues_MPISGGPU"
PetscErrorCode MatSetValues_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) {
  MatSetValues_SeqSGGPU(A, nrow, irow, ncol, icol, y, is);
  PetscFunctionReturn(0);
}


//#undef __FUNCT__
//#define __FUNCT__ "MatSetStencil_MPISGGPU"
//PetscErrorCode MatSetStencil_MPISGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof) {
//  MatSetStencil_SeqSGGPU(A, dim, dims, starts, dof);
//  PetscFunctionReturn(0);
//}


#undef __FUNCT__
#define __FUNCT__ "MatSetUp_MPISGGPU"
PetscErrorCode MatSetUp_MPISGGPU(Mat mat) {
  MatSetUp_SeqSGGPU(mat);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatZeroEntries_MPISGGPU"
PetscErrorCode MatZeroEntries_MPISGGPU(Mat A) {
  MatZeroEntries_SeqSGGPU(A);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetDiagonal_MPISGGPU"
PetscErrorCode MatGetDiagonal_MPISGGPU(Mat A, Vec v) {
  MatGetDiagonal_SeqSGGPU(A, v);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatDiagonalScale_MPISGGPU"
PetscErrorCode MatDiagonalScale_MPISGGPU(Mat A, Vec ll, Vec rr) {
  MatDiagonalScale_SeqSGGPU(A, ll, rr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRow_MPISGGPU"
PetscErrorCode MatGetRow_MPISGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v) {
  MatGetRow_SeqSGGPU(A, row, nz, idx , v);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatRestoreRow_MPISGGPU"
PetscErrorCode MatRestoreRow_MPISGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v) {
  MatRestoreRow_SeqSGGPU(A, row, nz, idx, v);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRowMaxAbs_MPISGGPU"
PetscErrorCode MatGetRowMaxAbs_MPISGGPU(Mat A, Vec v, PetscInt idx[]) {
  MatGetRowMaxAbs_SeqSGGPU(A, v, idx);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatView_MPISGGPU"
PetscErrorCode MatView_MPISGGPU(Mat A, PetscViewer viewer) {
  MatView_SeqSGGPU(A, viewer);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyBegin_MPISGGPU"
PetscErrorCode MatAssemblyBegin_MPISGGPU(Mat A, MatAssemblyType type) {
  MatAssemblyBegin_SeqSGGPU(A, type);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_MPISGGPU"
PetscErrorCode MatAssemblyEnd_MPISGGPU(Mat A, MatAssemblyType type) {
  MatAssemblyEnd_SeqSGGPU(A, type);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatView_MPISGGPU_ASCII"
PetscErrorCode MatView_MPISGGPU_ASCII(Mat A, PetscViewer viewer) {
  MatView_SeqSGGPU_ASCII(A, viewer);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatFDColoringApply_MPISGGPU"
PetscErrorCode  MatFDColoringApply_MPISGGPU(Mat J,MatFDColoring coloring,Vec x1,MatStructure *flag,void *sctx) {
  MatFDColoringApply_SeqSGGPU(J, coloring, x1, flag, sctx);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatFDColoringCreate_MPISGGPU"
PetscErrorCode MatFDColoringCreate_MPISGGPU(Mat mat,ISColoring iscoloring,MatFDColoring c) {
  MatFDColoringCreate_SeqSGGPU(mat, iscoloring, c);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetColumnIJ_MPISGGPU"
PetscErrorCode MatGetColumnIJ_MPISGGPU(Mat A,PetscInt oshift,PetscBool  symmetric,PetscBool  inodecompressed,PetscInt *nn, const PetscInt *ia[], const PetscInt *ja[],PetscBool  *done) {
  MatGetColumnIJ_SeqSGGPU(A, oshift, symmetric, inodecompressed, nn, ia, ja, done);
  PetscFunctionReturn(0);
}


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMPISGGPUSetPreallocation"
PetscErrorCode MatMPISGGPUSetPreallocation(Mat A,PetscInt stencil_type, PetscInt dof)
{
  PetscErrorCode ierr;
  Mat_SeqSGGPU *mat = (Mat_SeqSGGPU*)A->data;

  PetscFunctionBegin;

	PetscPrintf(PETSC_COMM_WORLD,"MatMPISGGPUSetPreallocation\n");

  mat->stencil_type = stencil_type;
  mat->dof = dof;
  if(A->preallocated)PetscFunctionReturn(0);
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  
  ierr = PetscTryMethod(A,"MatMPISGGPUSetPreallocation_C",(Mat,PetscInt,const PetscInt []),(A,0,0));CHKERRQ(ierr);
  A->preallocated=PETSC_TRUE;
  PetscFunctionReturn(0);
}
EXTERN_C_END


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMPISGGPUSetPreallocation_MPISGGPU"
extern PetscErrorCode MatMPISGGPUSetPreallocation_MPISGGPU(Mat A,PetscInt nz, const PetscInt nnz[])
{
  PetscErrorCode ierr;
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;

  PetscInt dim,diag_size,size,num_diags,i,vecsize;

  PetscPrintf(PETSC_COMM_WORLD,"MatMPISGGPUSetPreallocation_MPISGGPU\n");

  ierr = PetscLayoutSetBlockSize(A->rmap,1);CHKERRQ(ierr);
  ierr = PetscLayoutSetBlockSize(A->cmap,1);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->cmap);CHKERRQ(ierr);

  dim = A->stencil.dim;
  if (mat->dof > 1) {
    dim--;
  }

  mat->m = mat->n = mat->p = 1;
  mat->dim = dim;
  if (mat->dim > 0) mat->m = A->stencil.dims[dim-1];
  if (mat->dim > 1) mat->n = A->stencil.dims[dim-2];
  if (mat->dim > 2) mat->p = A->stencil.dims[dim-3];

  if (mat->stencil_type == 0) {
    /* star stencil */
    num_diags = 2*mat->dim + 1;
  } else {
    /* box stencil */
    num_diags =  1;
    for (i=0;i<mat->dim;i++) num_diags*=3;
  }

  diag_size = mat->m * mat->n * mat->p * mat->dof * mat->dof;
  size = num_diags * diag_size;

  fprintf(stdout,"mat->m: %d\tmat->n %d\tmat->p: %d\tmat->dof: %d \n\n",mat->m,mat->n,mat->p,mat->dof); 

  if (mat->m == 0 || mat->n == 0 || mat->p == 0 || mat->dof == 0) {
    SETERRQ(PETSC_COMM_SELF,0,"MatSetPreallocation_SeqSGGPU called without valid m, n, p, and dof!");
  }


  ierr = PetscMalloc(sizeof(PetscInt)*num_diags,&mat->diag_offsets);
  ierr = PetscMalloc(size * sizeof(PetscScalar), &mat->hostData); CHKERRQ(ierr);
  memset(mat->hostData, 0, size * sizeof(PetscScalar));

  (*mat->diag_starts)[0]  = 0 * diag_size;
  (*mat->diagonals).push_back(0);
  (*mat->diag_starts)[1]  = 1 * diag_size;
  (*mat->diagonals).push_back(1);
  (*mat->diag_starts)[-1] = 2 * diag_size;
  (*mat->diagonals).push_back(-1);
  if (mat->stencil_type == 0) {
    if (mat->dim == 2) {
      (*mat->diag_starts)[mat->m] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->m] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
    } else if (mat->dim == 3) {
      (*mat->diag_starts)[mat->m] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->m] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);

      (*mat->diag_starts)[mat->m*mat->n] = 5 * diag_size;
      (*mat->diagonals).push_back(mat->m*mat->n);
      (*mat->diag_starts)[-mat->m*mat->n] = 6 * diag_size;
      (*mat->diagonals).push_back(-mat->m*mat->n);
    }
  } else {
    if (mat->dim == 2) {
      (*mat->diag_starts)[mat->n-1] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n-1] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
      (*mat->diag_starts)[mat->n] = 5 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n] = 6 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
      (*mat->diag_starts)[mat->n+1] = 7 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n+1] = 8 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
    }
  }
  /*
  printf("Diagonals preallocated:\n");
  for (std::map<int, int>::iterator I = mat->diag_starts->begin(),
         E = mat->diag_starts->end(); I != E; ++I) {
    printf("%4d --> %4d\n",I->first,I->second);
  }
   */
  
  
  // Create GPU buffer
  if (mat->deviceData) {
    hipFree(mat->deviceData);
  }
  checkCudaError(hipMalloc(&mat->deviceData, sizeof(PetscScalar) * size));
  checkCudaError(hipMemset(mat->deviceData,0.0,sizeof(PetscScalar)*size));

  // Copy data to device
  checkCudaError(hipMemcpy(mat->deviceData, mat->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));


  vecsize = mat->m * mat->n * mat->p * mat->dof;

  // We know the expected size of x, y, so go ahead and allocate them now
  checkCudaError(hipMalloc(&mat->deviceX, vecsize * sizeof(PetscScalar)));
  checkCudaError(hipMalloc(&mat->deviceY, vecsize * sizeof(PetscScalar)));

  // We also know how many diagonals we have, and their indices
  checkCudaError(hipMalloc(&mat->deviceDiags, sizeof(int) * mat->diagonals->size()));
  A->preallocated = PETSC_TRUE;
  ierr = MatAssemblyBegin(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyEnd(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  

  PetscFunctionReturn(0);
}
EXTERN_C_END
