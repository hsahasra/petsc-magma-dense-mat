#include "hip/hip_runtime.h"
#include <../src/mat/impls/sggpu/mpi/mpisggpu.h>


// Direct access to seqgpu vector type
#include "../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h"

// Interop with CUSP vector
#include "../src/vec/vec/impls/seq/seqcusp/cuspvecimpl.h"

#include "cuPrintf.cu"

#define BLOCKWIDTH_X 128
#define BLOCKWIDTH_Y 1



//===-- CUDA Device Code -------------------------------------------------===//
 
texture<int2, 1> vector_x;
     
static __inline__ __device__ double fetch_doubleMPI(texture<int2, 1> tex, int i)
     {
       int2 v = tex1Dfetch(tex, i);
       return __hiloint2double(v.y, v.x);
     }
     
__global__ void MatMultKernelMPI(PetscScalar * coeff, PetscScalar * y, PetscScalar *x,PetscInt mat_size, PetscInt num_diags, int * diagonals, PetscInt dof, PetscInt vec_size) {
       
int idx = blockDim.x * blockIdx.x * 1 + threadIdx.x * 1;

     if (idx >= mat_size) 
      return;
     
int diag_size = mat_size * dof;
     
PetscScalar yval0 = 0.0;
int idx0 = idx;


//#pragma unroll 4
for (int i = 0; i < num_diags; ++i) 
	{
	    int d = diagonals[i];
    
	    int offset0 = diag_size * i + idx0;
	    int block0 = (idx0/dof + d) * dof;
     
	    //#pragma unroll 12
	    for (int j = 0; j < dof; ++j) 
		{
		      // Get coefficient
		      PetscScalar aval0 = coeff[offset0 + mat_size*j];
		      // Get X value
    	              PetscScalar xval0 = 0;

	              if ((block0 >= 0) && (block0 < vec_size))
			//xval0 = x[block0 + j];
			xval0 = fetch_doubleMPI(vector_x, block0 + j);

		      yval0 += aval0 * xval0;
	
		      //21, 22, 25, 26, 37, 38, 41, 42 are the only non zero entries in the 64x1 vector for ex14 when grid size is 4x4x4 	
		      //The following if statement is for testing and debugging and can be removed. 
		      // if ((idx0 == 21) || (idx0 == 22) || (idx0 == 25) || (idx0 == 26) ||  (idx0 == 37) ||  (idx0 == 38) ||  (idx0 == 41) ||  (idx0 == 42))
		      //     cuPrintf("d:%d \t offset:%d \t block0:%d \t aval0:%lf \t xval0:%lf \t yval0 :%lf\n",d,offset0,block0,aval0,xval0, yval0);
	        }
	}
   
      y[idx0] = yval0;

}
    
//===-- Host Code --------------------------------------------------------===//


// Matrix function table
static struct _MatOps MatOps_Values = {
/*0*/ MatSetValues_MPISGGPU,MatGetRow_MPISGGPU,MatRestoreRow_MPISGGPU,MatMult_MPISGGPU,0,
/*5*/0,0,0,0,0,
/*10*/0,0,0,0,0,
/*15*/0,0,MatGetDiagonal_MPISGGPU,MatDiagonalScale_MPISGGPU,0,
/*20*/MatAssemblyBegin_MPISGGPU,MatAssemblyEnd_MPISGGPU,0,MatZeroEntries_MPISGGPU,0,
/*25*/0,0,0,0,MatSetUp_MPISGGPU,
/*30*/0,0,0,0,0,
/*35*/0,0,0,0,0,
/*40*/0,0,0,0,0,
/*45*/0,0,0,0,0,
/*50*/0,0,MatGetColumnIJ_MPISGGPU,0,MatFDColoringCreate_MPISGGPU,
/*55*/0,0,0,MatSetValuesBlocked_MPISGGPU,0,
/*60*/MatDestroy_MPISGGPU,MatView_MPISGGPU,0,0,0,
/*65*/0,0,MatSetValues_MPISGGPU,0,MatGetRowMaxAbs_MPISGGPU,
/*70*/0,0,0,0,0,
/*75*/MatFDColoringApply_MPISGGPU,0,0,0,0,
/*80*/0,0,0,0,0,
/*85*/0,0,MatSetValuesBlocked_MPISGGPU,0,0,
/*90*/0,0,0,0,0,
/*95*/0,0,0,0,0,
/*100*/0,0,0,0,0,
/*105*/0,0,0,0,0,
/*110*/0,0,0,0,0,
/*115*/MatCreate_MPISGGPU,0,0,0,0,
/*120*/0,0,0,0,0,
/*125*/0,0,0,0,0,
/*130*/0,0,0,0,0,
/*135*/0,0,0,0,0,
/*140*/0,0,
/*142*/MatSetGrid_MPISGGPU
};



EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_MPISGGPU"
PetscErrorCode MatCreate_MPISGGPU(Mat A)
{
  Mat_MPISGGPU * mat;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  SGTrace;

  PetscInt rank, size;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  MPI_Comm_size(((PetscObject)A)->comm, &size); CHKERRQ(ierr);

  // Create internal matrix structure
  ierr = PetscMalloc(sizeof(Mat_MPISGGPU), &mat); CHKERRQ(ierr);
  memset(mat, 0, sizeof(Mat_MPISGGPU));
  ierr = PetscMalloc(sizeof(Mat_SeqSGGPU), &mat->mat_seq); CHKERRQ(ierr);
  memset(mat->mat_seq, 0, sizeof(Mat_SeqSGGPU));
  mat->mat_seq->diag_starts = new std::map<int, int>();
  mat->mat_seq->diagonals = new std::vector<int>();
  mat->rank = rank;   
  mat->size = size;

  checkCudaError(hipStreamCreate(&mat->mat_seq->stream));

  // Fill out PETSc matrix structure
  A->data = mat;
  memcpy(A->ops, &MatOps_Values, sizeof(struct _MatOps));
  A->same_nonzero= PETSC_FALSE;
  A->spptr = 0;

  // Set object type
  ierr = PetscObjectChangeTypeName((PetscObject)A, MATMPISGGPU); CHKERRQ(ierr);

  ierr = PetscObjectComposeFunctionDynamic((PetscObject)A,
        "MatMPISGGPUSetPreallocation_C","MatMPISGGPUSetPreallocation_MPIDIA",
        MatMPISGGPUSetPreallocation_MPISGGPU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
EXTERN_C_END



#undef __FUNCT__
#define __FUNCT__ "MatDestroy_MPISGGPU"
PetscErrorCode MatDestroy_MPISGGPU(Mat A) {

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;  
  PetscErrorCode ierr;

  PetscFunctionBegin;
 
  if (mat_seq->hostData) {
    ierr = PetscFree(mat_seq->hostData); CHKERRQ(ierr);
  }
  if (mat_seq->deviceData) {
    hipFree(mat_seq->deviceData);
  }
  if (mat_seq->diag_starts) {
    delete mat_seq->diag_starts;
  }
  ierr = PetscFree(mat_seq->diag_offsets); CHKERRQ(ierr);
  if (mat_seq->diagonals) {
    delete mat_seq->diagonals;
  }
  if (mat_seq->deviceX) {
    hipFree(mat_seq->deviceX);
  }
  if (mat_seq->deviceY) {
    hipFree(mat_seq->deviceY);
  }
  if (mat_seq->deviceDiags) {
    hipFree(mat_seq->deviceDiags);
  }
  
  if(mat_seq->ja) { ierr = PetscFree(mat_seq->ja); CHKERRQ(ierr); }
  
  if(mat_seq->ia) { ierr = PetscFree(mat_seq->ia); CHKERRQ(ierr); }
  
  checkCudaError(hipStreamDestroy(mat_seq->stream));
  if(mat_seq)
	  ierr = PetscFree(mat_seq); CHKERRQ(ierr);
    
  ierr = VecDestroy(&mat->lvec); CHKERRQ(ierr);
  ierr = VecScatterDestroy(&mat->Mvctx); CHKERRQ(ierr);
  ierr = PetscFree(A->data); CHKERRQ(ierr);
  
  ierr = PetscObjectChangeTypeName((PetscObject)A, 0); CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetGrid_MPISGGPU"
PetscErrorCode MatSetGrid_MPISGGPU(Mat A, PetscInt m, PetscInt n, PetscInt p) 
{
  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;

  PetscFunctionBegin;
  SGTrace;

  mat_seq->m = m;
  mat_seq->n = n;
  mat_seq->p = p;

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatMult_MPISGGPU"
PetscErrorCode MatMult_MPISGGPU(Mat A, Vec x, Vec y) {

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;

//  MatView_MPISGGPU(A,PETSC_VIEWER_STDOUT_WORLD);	

  PetscBool isseqcusp,isseqgpu,ismpicusp,iscusp;
  PetscErrorCode ierr;
  PetscInt mat_size, vec_size;
  CUSPARRAY *xgpu,*ygpu;
  PetscScalar *devX,*devY;

  PetscFunctionBegin;
  SGTrace;

  // Initialize y to zero
  ierr = VecSet(y, 0.0); CHKERRQ(ierr);

  ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQCUSP,&isseqcusp);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)x,VECMPICUSP,&ismpicusp);CHKERRQ(ierr);
  iscusp = (isseqcusp || ismpicusp) ? PETSC_TRUE : PETSC_FALSE;
  ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQGPU,&isseqgpu);CHKERRQ(ierr);

  if (isseqgpu) {
    dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
    dim3 grid((int)ceil((float)((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/mat->size)/(float)BLOCKWIDTH_X / 1.0), 1);

//     ierr = VecScatterBegin(mat->Mvctx,x,mat->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
//    ierr = VecScatterEnd(mat->Mvctx,x,mat->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);

    int shared_size = 0;
    Vec_SeqGPU *vx = (Vec_SeqGPU*) x->data;
    Vec_SeqGPU *vy = (Vec_SeqGPU*) y->data;
    /* Make sure y is also VECSEQGPU */
    ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQGPU,&isseqgpu);CHKERRQ(ierr);
    if (!isseqgpu) {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Both x and y must be same type");
    }


    /* synch up x */
    if (vx->syncState==VEC_CPU) {
      ierr = VecCopyOverH2D(x,vx->cpuptr);CHKERRQ(ierr);
      vx->syncState=VEC_SYNCHED;
    }
    /* Get device pointer for X */
    devX = vx->devptr;
    devY = vy->devptr;
 

   /* Bind X to device texture */
    mat_size = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/mat->size;
    vec_size = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof);

    checkCudaError(hipBindTexture(0, vector_x, devX, vec_size * sizeof(PetscScalar)));

    MatMultKernelMPI<<<grid, block, shared_size, mat_seq->stream>>>(mat_seq->deviceData, devY, devX, mat_size, mat_seq->diagonals->size(), mat_seq->deviceDiags, mat_seq->dof, vec_size);

    hipUnbindTexture(vector_x);
    hipDeviceSynchronize();

  } else if (iscusp) {
    dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
    dim3 grid((int)ceil((float)((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/mat->size)/(float)BLOCKWIDTH_X / 1.0), 1);
 
    int shared_size = 0;
    /* Make sure y is also VECCUSP */
    ierr = PetscObjectTypeCompare((PetscObject)x,VECCUSP,&isseqgpu);CHKERRQ(ierr);
    if (!iscusp) 
    {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Both x and y must be same type");
    }
 
    ierr = VecScatterBegin(mat->Mvctx,x,mat->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
    ierr = VecScatterEnd(mat->Mvctx,x,mat->lvec,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
 
    mat_size = ((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/mat->size);
    vec_size = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof);
 
    ierr = VecCUSPGetArrayWrite(y, &ygpu);CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(mat->lvec, &xgpu);CHKERRQ(ierr);
    devY = thrust::raw_pointer_cast(&(*ygpu)[0]);
    devX = thrust::raw_pointer_cast(&(*xgpu)[0]);
 
    /* Bind X to device texture */
    checkCudaError(hipBindTexture(0, vector_x, devX, vec_size * sizeof(PetscScalar)));
 
#if _TRACE
    printf("Host diagonals:\n");
    for (int i = 0; i < mat_seq->diagonals->size(); ++i) {
      printf("- %d\n", (*mat_seq->diagonals)[i]);
    }
#endif

    /* Invoke */

#if _TIME
    double start, end;
    start = getclock();
#endif

cudaPrintfInit();
MatMultKernelMPI<<<grid, block, shared_size, mat_seq->stream>>>(mat_seq->deviceData, devY, devX, mat_size, mat_seq->diagonals->size(), mat_seq->deviceDiags, mat_seq->dof,vec_size);
cudaPrintfDisplay(stdout,true);
cudaPrintfEnd();

#if _TIME
    checkCudaError(hipStreamSynchronize(mat_seq->stream));
    end = getclock();
    double elapsed = end - start;
    double gflops = (2.0 * mat_seq->non_zeros / elapsed / 1e9);

    double nos = ((mat_seq->p == 1 ? 2 : 3) * 2 + 1) * (2*mat_seq->dof - 1);
    double nz = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/numprocs;
    double alt_gflops = (2.0 * nos * nz) / ((end - start)*1024*1024*1024);

#if _CSV_OUT
    fprintf(stderr, "%d,%lf,%lf,\n", (mat_seq->m, mat_seq->n, mat_seq->p, mat_seq->dof)/numprocs, elapsed, gflops);
#endif
    printf("SGGPU Kernel Time:           %lf sec\n", elapsed);
    printf("SGGPU Kernel GFlop/s:        %lf\n", gflops);
    printf("SGGPU Kernel GFlop/s (alt):  %lf\n", alt_gflops);
#endif

    /* Cleanup */
    hipUnbindTexture(vector_x);

    ierr = VecCUSPRestoreArrayRead(x, &xgpu); CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(y, &ygpu); CHKERRQ(ierr);
    ierr = WaitForGPU() ; CHKERRCUSP(ierr);
    hipDeviceSynchronize();
  } else {
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Vec must be seqgpu or cusp type");
  }

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValuesBlocked_MPISGGPU"
PetscErrorCode MatSetValuesBlocked_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) {
  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatSetValuesBlocked_MPISGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValues_MPISGGPU"
PetscErrorCode MatSetValues_MPISGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is) {

  int i, j;
  PetscErrorCode ierr;
  PetscBool resizegpu = PETSC_FALSE;
  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;

  PetscInt row, col;	
  PetscInt rank;
  PetscInt numprocs; 	

  PetscInt buf[8192],*bufr=0,*bufc=0,*irowm,*icolm;

  PetscFunctionBegin;
  SGTrace;

  PetscInt rstart = A->rmap->rstart, rend = A->rmap->rend;
  PetscInt cstart = A->cmap->rstart, cend = A->cmap->rend;  
 
    if ((nrow+ncol) <= (PetscInt)(sizeof(buf)/sizeof(PetscInt))) 
	{
	      irowm = buf; icolm = buf+nrow;
        } 
    else 
	{
	      ierr = PetscMalloc2(nrow,PetscInt,&bufr,ncol,PetscInt,&bufc);CHKERRQ(ierr);
	      irowm = bufr; icolm = bufc;
	}
   
   MPI_Comm_rank(PETSC_COMM_WORLD, &rank);
   MPI_Comm_size(PETSC_COMM_WORLD, &numprocs);

   ierr = ISLocalToGlobalMappingApply(A->rmap->mapping,nrow,irow,irowm);CHKERRQ(ierr);
   ierr = ISLocalToGlobalMappingApply(A->cmap->mapping,ncol,icol,icolm);CHKERRQ(ierr);

   // Handle each element
   for (i = 0; i < nrow; i++) {
	if (irowm[i] < 0) continue;
	        row = irowm[i]; 

   for (j = 0; j < ncol; j++) {
  	
	if(irowm[i] >= rstart && irowm[i] < rend) 
	{
		if (icolm[j] >= cstart && icolm[j] < cend) 
		{
	        	col = icolm[j]; //-cstart;
		} 
		else if (icolm[j] < 0) 
			continue;
		else
			col = icolm[j];

      	      // Compute the diagonal and offset into the diagonal storage
	      // for the element
      
	      int diff = col - row;
	      int left = row % mat_seq->dof;
	      int diag = int(floor((double)(diff + left) / mat_seq->dof));
             
              if(mat->rank > 0)
		diag = rstart + diag;

	      int col_offset = col % mat_seq->dof;
	      int num_elems = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/mat->size;
	      int offset = col_offset * num_elems + row - (rank*num_elems);

#if _TRACE
      printf("- row: %d  col: %d  val: %lf  diag: %d  offset: %d\n", row, col, y[i*ncol+j], diag, offset);
#endif

      std::map<int, int> &diag_starts = *(mat_seq->diag_starts);
      std::map<int, int>::iterator I = diag_starts.find(diag);
      int diag_offset = 0;
      if (I == diag_starts.end()) {
        printf("WARNING: malloc() in MatSetValues\n");
        resizegpu = PETSC_TRUE;
        // The diagonal does not yet exist, so add a new diagonal
        int num_diags = diag_starts.size() + 1;
        int size = num_diags * ((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof * mat_seq->dof)/mat->size);
        PetscScalar *newData;
        ierr = PetscMalloc(size * sizeof(PetscScalar), &newData); CHKERRQ(ierr);
        memset(newData, 0, size * sizeof(PetscScalar));
        size -= ((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof * mat_seq->dof)/mat->size);
        if (num_diags > 1) {
          // This is not the first diagonal, so copy
#if _TRACE
          printf("- Memcpy of %d elements\n", size);
#endif
          memcpy(newData, mat_seq->hostData, size * sizeof(PetscScalar));
        }
        PetscFree(mat_seq->hostData);
        mat_seq->hostData = newData;
        diag_offset = size;
        diag_starts[diag] = diag_offset;
        mat_seq->diagonals->push_back(diag);
      }
	
	else 
      {
        // The diagonal already exists, so get the base offset
        diag_offset = I->second;
      }

      diag_offset += offset;

      if (is == INSERT_VALUES)
        mat_seq->hostData[diag_offset] = y[i * ncol + j];
      else
        mat_seq->hostData[diag_offset] += y[i * ncol + j];

      mat_seq->non_zeros++;
	}
    }
  }

  if (resizegpu) {
    int size,mat_size;
    // Create GPU buffer
    if (mat_seq->deviceData) {
      hipFree(mat_seq->deviceData);
    }
    size = mat_seq->diag_starts->size() * ((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof * mat_seq->dof)/numprocs);
    checkCudaError(hipMalloc(&mat_seq->deviceData, sizeof(PetscScalar) * size));

    mat_size = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/numprocs;

    if (mat_seq->deviceX) {
      hipFree(mat_seq->deviceX);
    }
    if (mat_seq->deviceY) {
      hipFree(mat_seq->deviceY);
    }
    if (mat_seq->deviceDiags) {
      hipFree(mat_seq->deviceDiags);
    }
    // We know the expected size of x, y, so go ahead and allocate them now
    checkCudaError(hipMalloc(&mat_seq->deviceX, mat_size * sizeof(PetscScalar)));
    checkCudaError(hipMalloc(&mat_seq->deviceY, mat_size * sizeof(PetscScalar)));

    // We also know how many diagonals we have, and their indices
    checkCudaError(hipMalloc(&mat_seq->deviceDiags, sizeof(int) * mat_seq->diagonals->size()));
  }

  PetscFunctionReturn(0);

}


//#undef __FUNCT__
//#define __FUNCT__ "MatSetStencil_MPISGGPU"
//PetscErrorCode MatSetStencil_MPISGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof) {
//  MatSetStencil_SeqSGGPU(A, dim, dims, starts, dof);
//  PetscFunctionReturn(0);
//}


#undef __FUNCT__
#define __FUNCT__ "MatSetUp_MPISGGPU"
PetscErrorCode MatSetUp_MPISGGPU(Mat A) {

  PetscFunctionBegin;
  SGTrace;

  PetscPrintf(PETSC_COMM_WORLD,"MatSetUp_MPISGGPU() not implemented\n");
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatZeroEntries_MPISGGPU"
PetscErrorCode MatZeroEntries_MPISGGPU(Mat A) {

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;
  PetscInt size;
  PetscFunctionBegin;
  SGTrace;
  
  size = mat_seq->diag_starts->size() * ((mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof * mat_seq->dof)/mat->size);
  memset(mat_seq->hostData, 0, size * sizeof(PetscScalar));
  
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "MatGetDiagonal_MPISGGPU"
PetscErrorCode MatGetDiagonal_MPISGGPU(Mat A, Vec v) {

  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatGetDiagonal_MPISGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatDiagonalScale_MPISGGPU"
PetscErrorCode MatDiagonalScale_MPISGGPU(Mat A, Vec ll, Vec rr) {

  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatDiagonalScale_MPISGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRow_MPISGGPU"
PetscErrorCode MatGetRow_MPISGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v) {

  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatGetRow_MPISGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatRestoreRow_MPISGGPU"
PetscErrorCode MatRestoreRow_MPISGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v) {

  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatRestoreRow_MPISGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRowMaxAbs_MPISGGPU"
PetscErrorCode MatGetRowMaxAbs_MPISGGPU(Mat A, Vec v, PetscInt idx[]) {

  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatGetRowMaxAbs_MPISGGPU not implemented");
}


void DisplayLocalMatrix(Mat_SeqSGGPU *mat_seq);

#undef __FUNCT__
#define __FUNCT__ "MatView_MPISGGPU"
PetscErrorCode MatView_MPISGGPU(Mat A, PetscViewer viewer) 
{
  PetscInt i;
  Mat_MPISGGPU* mat = (Mat_MPISGGPU*)A->data;
  
  for(i = 0; i < mat->size;++i)
	{
	 if(i == mat->rank)
	    DisplayLocalMatrix(mat->mat_seq);
         MPI_Barrier(PETSC_COMM_WORLD);   
    	}
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyBegin_MPISGGPU"
PetscErrorCode MatAssemblyBegin_MPISGGPU(Mat A, MatAssemblyType type) {

  PetscFunctionBegin;
  SGTrace;
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_MPISGGPU"
PetscErrorCode MatAssemblyEnd_MPISGGPU(Mat A, MatAssemblyType type) {

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;

  PetscInt size;

  PetscFunctionBegin;

#if _TRACE
  printf("[SeqSGGPU] MatAssemblyEnd_SeqSGGPU\n");

  for (std::map<int, int>::iterator I = mat_seq->diag_starts->begin(),
       E = mat_seq->diag_starts->end(); I != E; ++I) {
    printf("- Diag %d:\n", I->first);
    for (int i = 0; i < mat_seq->dof; ++i) {
      for (int j = 0; j < (mat_seq->dof * mat_seq->m * mat_seq->n * mat_seq->p)/numprocs; ++j) {
        int offset = i * ((mat_seq->dof * mat_seq->m * mat_seq->n * mat_seq->p)/numprocs) + j;
        printf(" %lf ", mat_seq->hostData[offset + I->second]);
      }
      printf("\n");
    }
  }
#endif

  size = (mat_seq->diag_starts->size()*mat_seq->m*mat_seq->n*mat_seq->p*mat_seq->dof*mat_seq->dof)/mat->size;

  checkCudaError(hipMemcpyAsync(mat_seq->deviceDiags, &(*mat_seq->diagonals)[0], sizeof(int) * mat_seq->diagonals->size(), hipMemcpyHostToDevice, mat_seq->stream));

  checkCudaError(hipMemcpy(mat_seq->deviceData, mat_seq->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));

  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "MatFDColoringApply_MPISGGPU"
PetscErrorCode  MatFDColoringApply_MPISGGPU(Mat A,MatFDColoring coloring,Vec x1,MatStructure *flag,void *sctx) {

  MatFDColoringApply_SeqSGGPU(A, coloring, x1, flag, sctx);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatFDColoringCreate_MPISGGPU"
PetscErrorCode MatFDColoringCreate_MPISGGPU(Mat A,ISColoring iscoloring,MatFDColoring c) {
 
//  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
//  Mat_SeqSGGPU *mat_seq = mat->mat_seq;

  PetscErrorCode ierr;
  PetscInt       i,n,nrows,N,j,k,m,ncols,col;
  const PetscInt *is,*ci,*cj,*rows;
  PetscInt       nis = iscoloring->n,*rowhit,*columnsforrow,l,bs = 1;
  IS             *isa;
  PetscBool      done,flg = PETSC_FALSE;

  PetscFunctionBegin;


	PetscPrintf(PETSC_COMM_WORLD,"MatFDColoringCreate_SeqSGGPU\n");


  ierr = ISColoringGetIS(iscoloring,PETSC_IGNORE,&isa);CHKERRQ(ierr);
  /* this is ugly way to get blocksize but cannot call MatGetBlockSize() because AIJ can have bs > 1 */

  N          = A->cmap->N/bs;
  c->M       = A->rmap->N/bs;  /* set total rows, columns and local rows */
  c->N       = A->cmap->N/bs;
  c->m       = A->rmap->N/bs;
  c->rstart  = 0;

  c->ncolors = nis;
  ierr       = PetscMalloc(nis*sizeof(PetscInt),&c->ncolumns);CHKERRQ(ierr);
  ierr       = PetscMalloc(nis*sizeof(PetscInt*),&c->columns);CHKERRQ(ierr); 
  ierr       = PetscMalloc(nis*sizeof(PetscInt),&c->nrows);CHKERRQ(ierr);
  ierr       = PetscMalloc(nis*sizeof(PetscInt*),&c->rows);CHKERRQ(ierr);
  ierr       = PetscMalloc(nis*sizeof(PetscInt*),&c->columnsforrow);CHKERRQ(ierr);

  ierr = MatGetColumnIJ(A,0,PETSC_FALSE,PETSC_FALSE,&ncols,&ci,&cj,&done);CHKERRQ(ierr);
  if (!done) SETERRQ1(((PetscObject)A)->comm,PETSC_ERR_SUP,"MatGetColumnIJ() not supported for matrix type %s",((PetscObject)A)->type_name);

  /*
     Temporary option to allow for debugging/testing
  */
  ierr = PetscOptionsGetBool(PETSC_NULL,"-matfdcoloring_slow",&flg,PETSC_NULL);CHKERRQ(ierr);

  ierr = PetscMalloc((N+1)*sizeof(PetscInt),&rowhit);CHKERRQ(ierr);
  ierr = PetscMalloc((N+1)*sizeof(PetscInt),&columnsforrow);CHKERRQ(ierr);

  for (i=0; i<nis; i++) {
    ierr = ISGetLocalSize(isa[i],&n);CHKERRQ(ierr);
    ierr = ISGetIndices(isa[i],&is);CHKERRQ(ierr);
    c->ncolumns[i] = n;
    if (n) {
      ierr = PetscMalloc(n*sizeof(PetscInt),&c->columns[i]);CHKERRQ(ierr);
      ierr = PetscMemcpy(c->columns[i],is,n*sizeof(PetscInt));CHKERRQ(ierr);
    } else {
      c->columns[i]  = 0;
    }

    if (!flg) { /* ------------------------------------------------------------------------------*/
      /* fast, crude version requires O(N*N) work */
      ierr = PetscMemzero(rowhit,N*sizeof(PetscInt));CHKERRQ(ierr);
      /* loop over columns*/
      for (j=0; j<n; j++) {
        col  = is[j];
        rows = cj + ci[col]; 
        m    = ci[col+1] - ci[col];
        /* loop over columns marking them in rowhit */
        for (k=0; k<m; k++) {
          rowhit[*rows++] = col + 1;
        }
      }
      /* count the number of hits */
      nrows = 0;
      for (j=0; j<N; j++) {
        if (rowhit[j]) nrows++;
      }
      c->nrows[i] = nrows;
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->rows[i]);CHKERRQ(ierr);
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->columnsforrow[i]);CHKERRQ(ierr);
      nrows       = 0;
      for (j=0; j<N; j++) {
        if (rowhit[j]) {
          c->rows[i][nrows]          = j;
          c->columnsforrow[i][nrows] = rowhit[j] - 1;
          nrows++;
        }
      }
    } else {  /*-------------------------------------------------------------------------------*/
      /* slow version, using rowhit as a linked list */
      PetscInt currentcol,fm,mfm;
      rowhit[N] = N;
      nrows     = 0;
      /* loop over columns */
      for (j=0; j<n; j++) {
        col   = is[j];
        rows  = cj + ci[col]; 
        m     = ci[col+1] - ci[col];
        /* loop over columns marking them in rowhit */
        fm    = N; /* fm points to first entry in linked list */
        for (k=0; k<m; k++) {
          currentcol = *rows++;
	  /* is it already in the list? */
          do {
            mfm  = fm;
            fm   = rowhit[fm];
          } while (fm < currentcol);
          /* not in list so add it */
          if (fm != currentcol) {
            nrows++;
            columnsforrow[currentcol] = col;
            /* next three lines insert new entry into linked list */
            rowhit[mfm]               = currentcol;
            rowhit[currentcol]        = fm;
            fm                        = currentcol; 
            /* fm points to present position in list since we know the columns are sorted */
          } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Detected invalid coloring");
        }
      }
      c->nrows[i] = nrows;
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->rows[i]);CHKERRQ(ierr);
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->columnsforrow[i]);CHKERRQ(ierr);
      /* now store the linked list of rows into c->rows[i] */
      nrows       = 0;
      fm          = rowhit[N];
      do {
        c->rows[i][nrows]            = fm;
        c->columnsforrow[i][nrows++] = columnsforrow[fm];
        fm                           = rowhit[fm];
      } while (fm < N);
    } /* ---------------------------------------------------------------------------------------*/
    ierr = ISRestoreIndices(isa[i],&is);CHKERRQ(ierr);  
  }
  ierr = MatRestoreColumnIJ(A,0,PETSC_FALSE,PETSC_FALSE,&ncols,&ci,&cj,&done);CHKERRQ(ierr);

  ierr = PetscFree(rowhit);CHKERRQ(ierr);
  ierr = PetscFree(columnsforrow);CHKERRQ(ierr);

  /* Optimize by adding the vscale, and scaleforrow[][] fields */
  /*
       see the version for MPIAIJ
  */
  ierr = VecCreateGhost(((PetscObject)A)->comm,A->rmap->n,PETSC_DETERMINE,0,PETSC_NULL,&c->vscale);CHKERRQ(ierr);
  ierr = PetscMalloc(c->ncolors*sizeof(PetscInt*),&c->vscaleforrow);CHKERRQ(ierr);
  for (k=0; k<c->ncolors; k++) { 
    ierr = PetscMalloc((c->nrows[k]+1)*sizeof(PetscInt),&c->vscaleforrow[k]);CHKERRQ(ierr);
    for (l=0; l<c->nrows[k]; l++) {
      col = c->columnsforrow[k][l];
      c->vscaleforrow[k][l] = col;
    }
  }
  ierr = ISColoringRestoreIS(iscoloring,&isa);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetColumnIJ_MPISGGPU"
PetscErrorCode MatGetColumnIJ_MPISGGPU(Mat A,PetscInt oshift,PetscBool  symmetric,PetscBool  inodecompressed,PetscInt *nn, const PetscInt *ia[], const PetscInt *ja[],PetscBool  *done) {

  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *a = mat->mat_seq;  

  PetscErrorCode ierr;
  PetscInt       n = A->cmap->n;
  PetscInt       ndiag = a->diagonals->size();
  PetscInt       nrows = a->m*a->n*a->p*a->dof;
  PetscInt       nz=a->dof*ndiag*nrows;
  PetscInt       iblock,i,j,col,index,colblock,offset;

  PetscFunctionBegin;  

  *nn = nrows;

  if (!ia) PetscFunctionReturn(0);
  if (a->ja) {
    ierr = PetscFree(a->ja); CHKERRQ(ierr);
  }
  if (a->ia) {
    ierr = PetscFree(a->ia); CHKERRQ(ierr);
  }
  ierr = PetscMalloc((n+1)*sizeof(PetscInt),&a->ia);CHKERRQ(ierr);
  ierr = PetscMalloc((nz+1)*sizeof(PetscInt),&a->ja);CHKERRQ(ierr);

  /* Assuming symmetric nonzero structure */
  index=0;
  for (iblock=0;iblock<nrows/a->dof;iblock++) {
    for (i=iblock*a->dof;i<(iblock+1)*a->dof;i++) {
      a->ia[i] = index;
      for (std::map<int, int>::iterator I = a->diag_starts->begin(),
             E = a->diag_starts->end(); I != E; ++I) {
        offset = I->first;
        colblock = offset + iblock;
        /* Ignore 0 padding */
        if (colblock < 0) {
          continue;
        }
        if (colblock >= (nrows/a->dof)) {
          break;
        }
        /* skip some blocks for nonperiodic da */
        if (a->stencil_type==0 &&  a->dim==2 && 
            ((colblock - iblock == 1 && !(colblock % a->n)) ||
             (iblock - colblock == 1 && !(iblock % a->n)))) {
          continue;

        }
      
        for (j=0;j<a->dof;j++) {
          col = (colblock*a->dof)  + j;
          a->ja[index++] = col;
	}

      }
    }
  }
  a->ia[nrows] = index;
  *ia = a->ia;
  *ja = a->ja;

  PetscFunctionReturn(0);
}


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMPISGGPUSetPreallocation"
PetscErrorCode MatMPISGGPUSetPreallocation(Mat A,PetscInt stencil_type, PetscInt dof)
{
  PetscErrorCode ierr;
  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;  

  PetscFunctionBegin;

  mat_seq->stencil_type = stencil_type;
  mat_seq->dof = dof;
  if(A->preallocated)PetscFunctionReturn(0);
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  
  ierr = PetscTryMethod(A,"MatMPISGGPUSetPreallocation_C",(Mat,PetscInt,const PetscInt []),(A,0,0));CHKERRQ(ierr);
  A->preallocated=PETSC_TRUE;
  PetscFunctionReturn(0);
}
EXTERN_C_END


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMPISGGPUSetPreallocation_MPISGGPU"
extern PetscErrorCode MatMPISGGPUSetPreallocation_MPISGGPU(Mat A,PetscInt nz, const PetscInt nnz[])
{
  PetscErrorCode ierr;
  Mat_MPISGGPU *mat = (Mat_MPISGGPU*)A->data;
  Mat_SeqSGGPU *mat_seq = mat->mat_seq;

  PetscInt dim,diag_size,size,num_diags,i,vecsize;

  ierr = PetscLayoutSetBlockSize(A->rmap,1);CHKERRQ(ierr);
  ierr = PetscLayoutSetBlockSize(A->cmap,1);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->cmap);CHKERRQ(ierr);

  dim = A->stencil.dim;
  if (mat_seq->dof > 1) {
    dim--;
  }

  PetscInt rstart = A->rmap->rstart;
  mat_seq->dim = dim;

  if (mat_seq->stencil_type == 0) {
    /* star stencil */
    num_diags = 2*mat_seq->dim + 1;
  } else {
    /* box stencil */
    num_diags =  1;
    for (i=0;i<mat_seq->dim;i++) num_diags*=3;
  }

  diag_size = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof * mat_seq->dof)/mat->size;
  size = num_diags * diag_size;

  if (mat_seq->m == 0 || mat_seq->n == 0 || mat_seq->p == 0 || mat_seq->dof == 0) {
    SETERRQ(PETSC_COMM_SELF,0,"MatSetPreallocation_SeqSGGPU called without valid m, n, p, and dof!");
  }

  ierr = PetscMalloc(sizeof(PetscInt)*num_diags,&mat_seq->diag_offsets);
  ierr = PetscMalloc(size * sizeof(PetscScalar), &mat_seq->hostData); CHKERRQ(ierr);
  memset(mat_seq->hostData, 0, size * sizeof(PetscScalar));

  (*mat_seq->diag_starts)[rstart + 0]  = 0 * diag_size;
  (*mat_seq->diagonals).push_back(rstart + 0);
  (*mat_seq->diag_starts)[rstart + 1]  = 1 * diag_size;
  (*mat_seq->diagonals).push_back(rstart + 1);
  (*mat_seq->diag_starts)[rstart - 1] = 2 * diag_size;
  (*mat_seq->diagonals).push_back(rstart - 1);
  if (mat_seq->stencil_type == 0) {
    if (mat_seq->dim == 2) 
    {
      (*mat_seq->diag_starts)[rstart + mat_seq->m] = 3 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m);
      (*mat_seq->diag_starts)[rstart - mat_seq->m] = 4 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m);

    } 
    else if (mat_seq->dim == 3) 
    {
      (*mat_seq->diag_starts)[rstart + mat_seq->m] = 3 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m);
      (*mat_seq->diag_starts)[rstart - mat_seq->m] = 4 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m);

      (*mat_seq->diag_starts)[rstart + mat_seq->m*mat_seq->n] = 5 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m*mat_seq->n);
      (*mat_seq->diag_starts)[rstart - mat_seq->m*mat_seq->n] = 6 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m*mat_seq->n);
    }
  } else {
    if (mat_seq->dim == 2) {
      (*mat_seq->diag_starts)[rstart + mat_seq->n-1] = 3 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m);
      (*mat_seq->diag_starts)[rstart - mat_seq->n-1] = 4 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m);
      (*mat_seq->diag_starts)[rstart + mat_seq->n] = 5 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m);
      (*mat_seq->diag_starts)[rstart - mat_seq->n] = 6 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m);
      (*mat_seq->diag_starts)[rstart + mat_seq->n+1] = 7 * diag_size;
      (*mat_seq->diagonals).push_back(rstart + mat_seq->m);
      (*mat_seq->diag_starts)[rstart - mat_seq->n+1] = 8 * diag_size;
      (*mat_seq->diagonals).push_back(rstart - mat_seq->m);
    }
  }
  /*
  printf("Diagonals preallocated:\n");
  for (std::map<int, int>::iterator I = mat_seq->diag_starts->begin(),
         E = mat_seq->diag_starts->end(); I != E; ++I) {
    printf("%4d --> %4d\n",I->first,I->second);
  }
   */
  
  
  // Create GPU buffer
  if (mat_seq->deviceData) {
    hipFree(mat_seq->deviceData);
  }
  checkCudaError(hipMalloc(&mat_seq->deviceData, sizeof(PetscScalar) * size));
  checkCudaError(hipMemset(mat_seq->deviceData,0,sizeof(PetscScalar)*size));

  // Copy data to device
  checkCudaError(hipMemcpy(mat_seq->deviceData, mat_seq->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));

  vecsize = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/mat->size;

  // We know the expected size of x, y, so go ahead and allocate them now
  checkCudaError(hipMalloc(&mat_seq->deviceX, vecsize * sizeof(PetscScalar)));
  checkCudaError(hipMalloc(&mat_seq->deviceY, vecsize * sizeof(PetscScalar)));

  // We also know how many diagonals we have, and their indices
  checkCudaError(hipMalloc(&mat_seq->deviceDiags, sizeof(int) * mat_seq->diagonals->size()));
  A->preallocated = PETSC_TRUE;
  ierr = MatAssemblyBegin(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyEnd(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  
  MatSetUpMultiply_MPISGGPU(A);

  PetscFunctionReturn(0);
}
EXTERN_C_END



void DisplayLocalMatrix(Mat_SeqSGGPU *mat_seq)
{

  PetscInt nrows,ndiag,dof,i,j,iblock,col,index,offset;
  std::map<int, int> &diag_starts = *(mat_seq->diag_starts);

  PetscInt numprocs, rank;
  MPI_Comm_rank(PETSC_COMM_WORLD,&rank);
  MPI_Comm_size(PETSC_COMM_WORLD,&numprocs);

  nrows = (mat_seq->m * mat_seq->n * mat_seq->p * mat_seq->dof)/numprocs;
  ndiag = mat_seq->diagonals->size();
  dof = mat_seq->dof;

  for (std::map<int, int>::iterator I = diag_starts.begin(),
         E = diag_starts.end(); I != E; ++I) 
	{
		fprintf(stdout,"- Diag %d:%d\n", I->first, I->second);
  	}

	fprintf(stdout,"\n");
	fprintf(stdout,"hostData:\n");

  for (i=0;i<nrows;i++) 
	{
	fprintf(stdout,"row %2.2d:",rank*nrows + i); 

	    for (j=0;j<ndiag*dof;j++) 
		{
			fprintf(stdout," %4g ",mat_seq->hostData[i+j*nrows]);
		}
	fprintf(stdout,"\n");
  	}
	fprintf(stdout,"\n\n");


  for (iblock=0;iblock<(nrows/dof);iblock++)  
	{
	    for (i=iblock*dof;i<(iblock+1)*dof;i++) 
		{
		fprintf(stdout,"row %d:",rank*nrows + i);
	
		  for (std::map<int, int>::iterator I = mat_seq->diag_starts->begin(),
        	     E = mat_seq->diag_starts->end(); I != E; ++I) 
		  {
	        	/* Ignore 0 padding */
		        offset = I->first;

		        if (offset + iblock + (rank*nrows) < 0) 
			{
	        	  continue;
        		}

	        	if (offset + iblock + (rank*nrows) >= ((nrows*numprocs)/dof)) 
			{
        	  	break;
        		}
	        
	        	for (j=0;j<dof;j++) 
			{
	        	  index = i + I->second + j*nrows; // column-major
	        	  col = offset*dof+((iblock+(rank*nrows))*dof) + j;
	                  fprintf(stdout," (%d, %g) ",col,mat_seq->hostData[index]);
        		}
      	 	  }
		fprintf(stdout,"\n");
    		}
  	}

}
