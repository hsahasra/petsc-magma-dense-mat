#include "hip/hip_runtime.h"
/// SGGPU Matrix Type

#define PETSCMAT_DLL

#include "petsc-private/matimpl.h"
#include "sggpu.h"

// Direct access to seqgpu vector type
#include "../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

// C++ library headers
#include <map>

// Debugging flags
#define _TRACE 0

// Hard-coded block size
#define BLOCKWIDTH_X 256
#define BLOCKWIDTH_Y 1

// Prototypes
PetscErrorCode MatDestroy_SeqSGGPU(Mat A);
PetscErrorCode MatSetGrid_SeqSGGPU(Mat B, PetscInt m, PetscInt n, PetscInt p);
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y);
PetscErrorCode MatSetValuesBlocked_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is);
PetscErrorCode MatSetValues_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is);
PetscErrorCode MatSetStencil_SeqSGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof);
PetscErrorCode MatSetUpPreallocation_SeqSGGPU(Mat mat);
PetscErrorCode MatZeroEntries_SeqSGGPU(Mat A);
PetscErrorCode MatGetDiagonal_SeqSGGPU(Mat A, Vec v);
PetscErrorCode MatDiagonalScale_SeqSGGPU(Mat A, Vec ll, Vec rr);
PetscErrorCode MatGetRow_SeqSGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v);
PetscErrorCode MatRestoreRow_SeqSGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v);
PetscErrorCode MatGetRowMaxAbs_SeqSGGPU(Mat A, Vec v, PetscInt idx[]);
PetscErrorCode MatView_SeqSGGPU(Mat A, PetscViewer viewer);
PetscErrorCode MatAssemblyBegin_SeqSGGPU(Mat A, MatAssemblyType type);
PetscErrorCode MatAssemblyEnd_SeqSGGPU(Mat A, MatAssemblyType type);


// ----------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
void checkCudaError(hipError_t err) {
  if(hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


//===-- CUDA Device Code -------------------------------------------------===//

texture<int2, 1> vector_x;

static __inline__ __device__ double fetch_double(texture<int2, 1> tex, int i)
{
  int2 v = tex1Dfetch(tex, i);
  return __hiloint2double(v.y, v.x);
}


__global__ void MatMultKernel(PetscScalar * coeff, PetscScalar * y, PetscInt mat_size, PetscInt num_diags, int * diagonals, PetscInt dof) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx >= mat_size)
    return;

#if _TRACE
  if (threadIdx.x == 0) {
    printf("Diagonals:\n");
    for (int i = 0; i < num_diags; ++i) {
      printf("- %d\n", diagonals[i]);
    }
    printf("Foo: %d\n", -42);
  }
#endif

  int diag_size = mat_size * dof;
  PetscScalar yval = 0.0;

  for (int i = 0; i < num_diags; ++i) {
    int d = diagonals[i];
    int offset = diag_size * i + idx;
    int block = (idx / dof + d) * dof;
    for (int j = 0; j < dof; ++j) {
      // Get coefficient
#if _TRACE
      if (threadIdx.x == 0) {
        printf("diag: %d  offset: %d  index: %d\n", d, offset, offset + mat_size*j);
      }
#endif

      PetscScalar aval = coeff[offset + mat_size*j];

#if _TRACE
      if (threadIdx.x == 0) {
        printf("aval: %lf\n", aval);
      }
#endif

      // Get x value
      int this_block = block + j;

#if _TRACE
      if (threadIdx.x == 0) {
        printf("this_block: %d\n", this_block);
      }
#endif

      bool in_bounds = this_block >= 0 && this_block < mat_size;
      PetscScalar xval = in_bounds ? fetch_double(vector_x, this_block) : 0.0;

#if _TRACE
      if (threadIdx.x == 0) {
        printf("xval: %lf\n", xval);
      }
#endif

      yval += aval * xval;
    }
  }

  y[idx] = yval;
}

//===-- Host Code --------------------------------------------------------===//


// Matrix function table
static struct _MatOps MatOps_Values = {
/*0*/ MatSetValues_SeqSGGPU,MatGetRow_SeqSGGPU,MatRestoreRow_SeqSGGPU,MatMult_SeqSGGPU,0,
/*5*/0,0,0,0,0,
/*10*/0,0,0,0,0,
/*15*/0,0,MatGetDiagonal_SeqSGGPU,MatDiagonalScale_SeqSGGPU,0,
/*20*/MatAssemblyBegin_SeqSGGPU,MatAssemblyEnd_SeqSGGPU,0,MatZeroEntries_SeqSGGPU,0,
/*25*/0,0,0,0,MatSetUpPreallocation_SeqSGGPU,
/*30*/0,0,0,0,0,
/*35*/0,0,0,0,0,
/*40*/0,0,0,0,0,
/*45*/0,0,0,0,0,
/*50*/0,0,0,0,0,
/*55*/0,0,0,MatSetValuesBlocked_SeqSGGPU,0,
/*60*/MatDestroy_SeqSGGPU,MatView_SeqSGGPU,0,0,0,
/*65*/0,0,MatSetValues_SeqSGGPU,0,MatGetRowMaxAbs_SeqSGGPU,
/*70*/0,0,0,0,0,
/*75*/0,0,0,0,0,
/*80*/0,0,0,0,0,
/*85*/0,0,MatSetValuesBlocked_SeqSGGPU,0,0,
/*90*/0,0,0,0,0,
/*95*/0,0,0,0,0,
/*100*/0,0,0,0,0,
/*105*/0,0,0,0,0,
/*110*/0,0,0,0,0,
/*115*/MatCreate_SeqSGGPU,0,0,0,0,
/*120*/0,0,0,0,0,
/*125*/0,0,0,0,0,
/*130*/0,0,0,0,0,
/*135*/0,0,0,0,MatSetStencil_SeqSGGPU,
/*140*/MatSetGrid_SeqSGGPU
};


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode MatCreate_SeqSGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;
  PetscMPIInt size;

  PetscFunctionBegin;

#if _TRACE
  printf("[SeqSGGPU] MatCreate_SeqSGGPU\n");
#endif

  ierr = MPI_Comm_size(((PetscObject)A)->comm, &size); CHKERRQ(ierr);
  if (size > 1)
    SETERRQ(PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "Comm must be size 1");

  // Create internal matrix structure
  ierr = PetscMalloc(sizeof(Mat_SeqSGGPU), &mat); CHKERRQ(ierr);
  memset(mat, 0, sizeof(Mat_SeqSGGPU));
  mat->diag_starts = new std::map<int, int>();
  mat->diagonals = new std::vector<int>();

  // Fill out PETSc matrix structure
  A->data = mat;
  memcpy(A->ops, &MatOps_Values, sizeof(struct _MatOps));
  A->same_nonzero= PETSC_FALSE;
  A->spptr = 0;

  // Set object type
  ierr = PetscObjectChangeTypeName((PetscObject)A, MATSEQSGGPU); CHKERRQ(ierr);

  PetscFunctionReturn(0);
}
EXTERN_C_END


#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqSGGPU"
PetscErrorCode MatDestroy_SeqSGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;

  PetscFunctionBegin;

#if _TRACE
  printf("[SeqSGGPU] MatDestroy_SeqSGGPU\n");
#endif

  mat = (Mat_SeqSGGPU*)A->data;

  if (mat->hostData) {
    ierr = PetscFree(mat->hostData); CHKERRQ(ierr);
  }
  if (mat->deviceData) {
    hipFree(mat->deviceData);
  }
  if (mat->diag_starts) {
    delete mat->diag_starts;
  }
  if (mat->diagonals) {
    delete mat->diagonals;
  }
  PetscFree(mat); CHKERRQ(ierr);

  ierr = PetscObjectChangeTypeName((PetscObject)A, 0); CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetGrid_SeqSGGPU"
PetscErrorCode MatSetGrid_SeqSGGPU(Mat B, PetscInt m, PetscInt n, PetscInt p)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)B->data;

  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetGrid_SeqSGGPU\n");
#endif

  mat->m = m;
  mat->n = n;
  mat->p = p;

  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat A, Vec x, Vec y)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;
  PetscErrorCode ierr;
  PetscScalar * deviceX;
  PetscScalar * deviceY;

  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatMult_SeqSGGPU\n");
#endif

  // Initialize y to zero
  ierr = VecSet(y, 0.0); CHKERRQ(ierr);

  // NOTE: The seqgpu vector type is not really working here...

  //const VecType vec_type;

  // Get device pointer for X
  /*ierr = PetscObjectGetType((PetscObject)x, &vec_type); CHKERRQ(ierr);
  if (!strcmp(vec_type, "seqgpu")) {
    // We have a GPU vector type, so just use the existing pointer
    deviceX = ((Vec_SeqGPU*)x->data)->devptr;
  } else {
    fprintf(stderr, "Non-GPU vector types are not implemented!");
    exit(EXIT_FAILURE);
  }

  // Get device pointer for Y
  ierr = PetscObjectGetType((PetscObject)y, &vec_type); CHKERRQ(ierr);
  if (!strcmp(vec_type, "seqgpu")) {
    // We have a GPU vector type, so just use the existing pointer
    deviceY = ((Vec_SeqGPU*)y->data)->devptr;
  } else {
    fprintf(stderr, "Non-GPU vector types are not implemented!");
    exit(EXIT_FAILURE);
  }*/

  PetscScalar * hostX;
  PetscScalar * hostY;

  int mat_size = mat->m * mat->n * mat->p * mat->dof;

  ierr = VecGetArray(x, &hostX); CHKERRQ(ierr);
  ierr = VecGetArray(y, &hostY); CHKERRQ(ierr);

  checkCudaError(hipMalloc(&deviceX, mat_size * sizeof(PetscScalar)));
  checkCudaError(hipMalloc(&deviceY, mat_size * sizeof(PetscScalar)));
  checkCudaError(hipMemcpy(deviceX, hostX, mat_size * sizeof(PetscScalar), hipMemcpyHostToDevice));

  // Bind X to device texture
  checkCudaError(hipBindTexture(0, vector_x, deviceX, mat_size * sizeof(PetscScalar)));


  // Get diagonals array
  PetscInt * device_diagonals;

#if _TRACE
  printf("Host diagonals:\n");
  for (int i = 0; i < mat->diagonals->size(); ++i) {
    printf("- %d\n", (*mat->diagonals)[i]);
  }
#endif

  checkCudaError(hipMalloc(&device_diagonals, sizeof(int) * mat->diagonals->size()));
  checkCudaError(hipMemcpy(device_diagonals, &(*mat->diagonals)[0], sizeof(int) * mat->diagonals->size(), hipMemcpyHostToDevice));

  // Invoke
  dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
  dim3 grid((int)ceil((float)(mat->m * mat->n * mat->p * mat->dof)/(float)BLOCKWIDTH_X), 1);

  MatMultKernel<<<grid, block>>>(mat->deviceData, deviceY, mat_size, mat->diagonals->size(), device_diagonals, mat->dof);
  hipDeviceSynchronize();
  checkCudaError(hipGetLastError());

  checkCudaError(hipMemcpy(hostY, deviceY, mat_size * sizeof(PetscScalar), hipMemcpyDeviceToHost));

  // Cleanup
  hipFree(device_diagonals);
  hipFree(deviceX);
  hipFree(deviceY);
  hipUnbindTexture(vector_x);

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValuesBlocked_SeqSGGPU"
PetscErrorCode MatSetValuesBlocked_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetValuesBlocked_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValues_SeqSGGPU"
PetscErrorCode MatSetValues_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is)
{
  int i, j;
  PetscErrorCode ierr;
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;

  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetValues_SeqSGGPU\n");
#endif

  // Handle each element
  for (i = 0; i < nrow; i++) {
    for (j = 0; j < ncol; j++) {
      // Compute the diagonal and offset into the diagonal storage
      // for the element
      int row = irow[i];
      int col = icol[j];
      int diff = col - row;
      int left = row % mat->dof;
      int diag = int(floor((double)(diff + left) / mat->dof));
      int col_offset = col % mat->dof;
      int num_elems = mat->m * mat->n * mat->p * mat->dof;
      int offset = col_offset * num_elems + row;

#if _TRACE
      printf("- row: %d  col: %d  val: %lf  diag: %d  offset: %d\n", row, col, y[i*ncol+j], diag, offset);
#endif

      std::map<int, int> &diag_starts = *(mat->diag_starts);
      std::map<int, int>::iterator I = diag_starts.find(diag);
      int diag_offset = 0;
      if (I == diag_starts.end()) {
        // The diagonal does not yet exist, so add a new diagonal
        int num_diags = diag_starts.size() + 1;
        int size = num_diags * mat->m * mat->n * mat->p * mat->dof * mat->dof;
        PetscScalar *newData;
        ierr = PetscMalloc(size * sizeof(PetscScalar), &newData); CHKERRQ(ierr);
        memset(newData, 0, size * sizeof(PetscScalar));
        size -= mat->m * mat->n * mat->p * mat->dof * mat->dof;
        if (num_diags > 1) {
          // This is not the first diagonal, so copy
#if _TRACE
          printf("- Memcpy of %d elements\n", size);
#endif
          memcpy(newData, mat->hostData, size * sizeof(PetscScalar));
        }
        PetscFree(mat->hostData);
        mat->hostData = newData;
        diag_offset = size;
        diag_starts[diag] = diag_offset;
        mat->diagonals->push_back(diag);
      } else {
        // The diagonal already exists, so get the base offset
        diag_offset = I->second;
      }

      diag_offset += offset;

      if (is == INSERT_VALUES)
        mat->hostData[diag_offset] = y[i * ncol + j];
      else
        mat->hostData[diag_offset] += y[i * ncol + j];
    }
  }

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetStencil_SeqSGGPU"
PetscErrorCode MatSetStencil_SeqSGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetStencil_SeqSGGPU  (%p)\n", A);
#endif

  if (dim < 1 || dim > 3) {
    SETERRQ(PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "Dim must be between 1 and 3.");
  }

  mat->m = dims[0];
  if (dim > 1) {
    mat->n = dims[1];
    if (dim > 2) {
      mat->p = dims[2];
    } else {
      mat->p = 1;
    }
  } else {
    mat->n = 1;
    mat->p = 1;
  }

  mat->dof = dof;

#if _TRACE
  printf("- m: %d  n: %d  p: %d  dof: %d\n", mat->m, mat->n, mat->p, mat->dof);
#endif

  // It appears that we are responsible for pre-allocating
  if (!A->preallocated) {
    ierr = MatSetUpPreallocation_SeqSGGPU(A); CHKERRQ(ierr);
  }

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetUpPreallocation_SeqSGGPU"
PetscErrorCode MatSetUpPreallocation_SeqSGGPU(Mat A)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetUpPreallocation_SeqSGGPU\n");
#endif

  A->preallocated = PETSC_TRUE;

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatZeroEntries_SeqSGGPU"
PetscErrorCode MatZeroEntries_SeqSGGPU(Mat A)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatZeroEntries_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetDiagonal_SeqSGGPU"
PetscErrorCode MatGetDiagonal_SeqSGGPU(Mat A, Vec v)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatGetDiagonal_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatDiagonalScale_SeqSGGPU"
PetscErrorCode MatDiagonalScale_SeqSGGPU(Mat A, Vec ll, Vec rr)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatDiagonalScale_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRow_SeqSGGPU"
PetscErrorCode MatGetRow_SeqSGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatGetRow_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatRestoreRow_SeqSGGPU"
PetscErrorCode MatRestoreRow_SeqSGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatRestoreRow_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRowMaxAbs_SeqSGGPU"
PetscErrorCode MatGetRowMaxAbs_SeqSGGPU(Mat A, Vec v, PetscInt idx[])
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatGetRowMaxAbs_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatView_SeqSGGPU"
PetscErrorCode MatView_SeqSGGPU(Mat A, PetscViewer viewer)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatView_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyBegin_SeqSGGPU"
PetscErrorCode MatAssemblyBegin_SeqSGGPU(Mat A, MatAssemblyType type)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatAssemblyBegin_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_SeqSGGPU"
PetscErrorCode MatAssemblyEnd_SeqSGGPU(Mat A, MatAssemblyType type)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatAssemblyEnd_SeqSGGPU\n");

  for (std::map<int, int>::iterator I = mat->diag_starts->begin(),
       E = mat->diag_starts->end(); I != E; ++I) {
    printf("- Diag %d:\n", I->first);
    for (int i = 0; i < mat->dof; ++i) {
      for (int j = 0; j < mat->dof * mat->m * mat->n * mat->p; ++j) {
        int offset = i * mat->dof * mat->m * mat->n * mat->p + j;
        printf(" %lf ", mat->hostData[offset + I->second]);
      }
      printf("\n");
    }
  }
#endif

  // Create GPU buffer
  if (mat->deviceData) {
    hipFree(mat->deviceData);
  }
  int size = mat->diag_starts->size() * mat->m * mat->n * mat->p * mat->dof * mat->dof;
  checkCudaError(hipMalloc(&mat->deviceData, sizeof(PetscScalar) * size));

  // Copy data to device
  checkCudaError(hipMemcpy(mat->deviceData, mat->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));

  PetscFunctionReturn(0);
}

