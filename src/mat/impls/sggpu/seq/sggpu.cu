#include "hip/hip_runtime.h"
/// SGGPU Matrix Type

#define PETSCMAT_DLL

#include "petsc-private/matimpl.h"
#include "sggpu.h"

// Direct access to seqgpu vector type
//#include "../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h"

// Interop with CUSP vector
#include "../src/vec/vec/impls/seq/seqcusp/cuspvecimpl.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

// C++ library headers
#include <map>

// Debugging flags
#define _TRACE 0
#define _TIME 0
#define _CSV_OUT 0

// Hard-coded block size
#define BLOCKWIDTH_X 128
#define BLOCKWIDTH_Y 1

// Prototypes
PetscErrorCode MatDestroy_SeqSGGPU(Mat A);
PetscErrorCode MatSetGrid_SeqSGGPU(Mat B, PetscInt m, PetscInt n, PetscInt p);
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y);
PetscErrorCode MatSetValuesBlocked_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is);
PetscErrorCode MatSetValues_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is);
PetscErrorCode MatSetStencil_SeqSGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof);
PetscErrorCode MatSetUpPreallocation_SeqSGGPU(Mat mat);
PetscErrorCode MatZeroEntries_SeqSGGPU(Mat A);
PetscErrorCode MatGetDiagonal_SeqSGGPU(Mat A, Vec v);
PetscErrorCode MatDiagonalScale_SeqSGGPU(Mat A, Vec ll, Vec rr);
PetscErrorCode MatGetRow_SeqSGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v);
PetscErrorCode MatRestoreRow_SeqSGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v);
PetscErrorCode MatGetRowMaxAbs_SeqSGGPU(Mat A, Vec v, PetscInt idx[]);
PetscErrorCode MatView_SeqSGGPU(Mat A, PetscViewer viewer);
PetscErrorCode MatAssemblyBegin_SeqSGGPU(Mat A, MatAssemblyType type);
PetscErrorCode MatAssemblyEnd_SeqSGGPU(Mat A, MatAssemblyType type);


// ----------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
void checkCudaError(hipError_t err) {
  if(hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

//------------------------------------------------------
// general timer function using unix system call
// dlowell ANL-MCS
//------------------------------------------------------
double getclock() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return (tp.tv_sec + tp.tv_usec*1.0e-6);
}

//===-- CUDA Device Code -------------------------------------------------===//

texture<int2, 1> vector_x;

static __inline__ __device__ double fetch_double(texture<int2, 1> tex, int i)
{
  int2 v = tex1Dfetch(tex, i);
  return __hiloint2double(v.y, v.x);
}

__global__ void MatMultKernel(PetscScalar * coeff, PetscScalar * y, PetscInt mat_size, PetscInt num_diags, int * diagonals, PetscInt dof) {
  int idx = blockDim.x * blockIdx.x * 1 + threadIdx.x * 1;

  if (idx >= mat_size)
    return;

  int diag_size = mat_size * dof;

  PetscScalar yval0 = 0.0;
  int idx0 = idx;

  //#pragma unroll 4
  for (int i = 0; i < num_diags; ++i) {
    int d = diagonals[i];

    int offset0 = diag_size * i + idx0;
    int block0 = (idx0 / dof + d) * dof;

    //#pragma unroll 12
    for (int j = 0; j < dof; ++j) {
      // Get coefficient
      PetscScalar aval0 = coeff[offset0 + mat_size*j];
      // Get X value
      PetscScalar xval0 = fetch_double(vector_x, block0 + j);

      yval0 += aval0 * xval0;
    }
  }

  y[idx0] = yval0;
}

//===-- Host Code --------------------------------------------------------===//


// Matrix function table
static struct _MatOps MatOps_Values = {
/*0*/ MatSetValues_SeqSGGPU,MatGetRow_SeqSGGPU,MatRestoreRow_SeqSGGPU,MatMult_SeqSGGPU,0,
/*5*/0,0,0,0,0,
/*10*/0,0,0,0,0,
/*15*/0,0,MatGetDiagonal_SeqSGGPU,MatDiagonalScale_SeqSGGPU,0,
/*20*/MatAssemblyBegin_SeqSGGPU,MatAssemblyEnd_SeqSGGPU,0,MatZeroEntries_SeqSGGPU,0,
/*25*/0,0,0,0,MatSetUpPreallocation_SeqSGGPU,
/*30*/0,0,0,0,0,
/*35*/0,0,0,0,0,
/*40*/0,0,0,0,0,
/*45*/0,0,0,0,0,
/*50*/0,0,0,0,0,
/*55*/0,0,0,MatSetValuesBlocked_SeqSGGPU,0,
/*60*/MatDestroy_SeqSGGPU,MatView_SeqSGGPU,0,0,0,
/*65*/0,0,MatSetValues_SeqSGGPU,0,MatGetRowMaxAbs_SeqSGGPU,
/*70*/0,0,0,0,0,
/*75*/0,0,0,0,0,
/*80*/0,0,0,0,0,
/*85*/0,0,MatSetValuesBlocked_SeqSGGPU,0,0,
/*90*/0,0,0,0,0,
/*95*/0,0,0,0,0,
/*100*/0,0,0,0,0,
/*105*/0,0,0,0,0,
/*110*/0,0,0,0,0,
/*115*/MatCreate_SeqSGGPU,0,0,0,0,
/*120*/0,0,0,0,0,
/*125*/0,0,0,0,0,
/*130*/0,0,0,0,0,
/*135*/0,0,0,0,MatSetStencil_SeqSGGPU,
/*140*/MatSetGrid_SeqSGGPU
};


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode MatCreate_SeqSGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;
  PetscMPIInt size;

  PetscFunctionBegin;

#if _TRACE
  printf("[SeqSGGPU] MatCreate_SeqSGGPU\n");
#endif

  ierr = MPI_Comm_size(((PetscObject)A)->comm, &size); CHKERRQ(ierr);
  if (size > 1)
    SETERRQ(PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "Comm must be size 1");

  // Create internal matrix structure
  ierr = PetscMalloc(sizeof(Mat_SeqSGGPU), &mat); CHKERRQ(ierr);
  memset(mat, 0, sizeof(Mat_SeqSGGPU));
  mat->diag_starts = new std::map<int, int>();
  mat->diagonals = new std::vector<int>();

  checkCudaError(hipStreamCreate(&mat->stream));

  // Fill out PETSc matrix structure
  A->data = mat;
  memcpy(A->ops, &MatOps_Values, sizeof(struct _MatOps));
  A->same_nonzero= PETSC_FALSE;
  A->spptr = 0;

  // Set object type
  ierr = PetscObjectChangeTypeName((PetscObject)A, MATSEQSGGPU); CHKERRQ(ierr);

  PetscFunctionReturn(0);
}
EXTERN_C_END


#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqSGGPU"
PetscErrorCode MatDestroy_SeqSGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;

  PetscFunctionBegin;

#if _TRACE
  printf("[SeqSGGPU] MatDestroy_SeqSGGPU\n");
#endif

  mat = (Mat_SeqSGGPU*)A->data;

  if (mat->hostData) {
    ierr = PetscFree(mat->hostData); CHKERRQ(ierr);
  }
  if (mat->deviceData) {
    hipFree(mat->deviceData);
  }
  if (mat->diag_starts) {
    delete mat->diag_starts;
  }
  if (mat->diagonals) {
    delete mat->diagonals;
  }
  if (mat->deviceX) {
    hipFree(mat->deviceX);
  }
  if (mat->deviceY) {
    hipFree(mat->deviceY);
  }
  if (mat->deviceDiags) {
    hipFree(mat->deviceDiags);
  }

  checkCudaError(hipStreamDestroy(mat->stream));
  PetscFree(mat); CHKERRQ(ierr);

  ierr = PetscObjectChangeTypeName((PetscObject)A, 0); CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetGrid_SeqSGGPU"
PetscErrorCode MatSetGrid_SeqSGGPU(Mat B, PetscInt m, PetscInt n, PetscInt p)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)B->data;

  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetGrid_SeqSGGPU\n");
#endif

  mat->m = m;
  mat->n = n;
  mat->p = p;

  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat A, Vec x, Vec y)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatMult_SeqSGGPU\n");
#endif

  // Initialize y to zero
  ierr = VecSet(y, 0.0); CHKERRQ(ierr);

  // NOTE: The seqgpu vector type is not really working here...

  //const VecType vec_type;

  // Get device pointer for X
  /*ierr = PetscObjectGetType((PetscObject)x, &vec_type); CHKERRQ(ierr);
  if (!strcmp(vec_type, "seqgpu")) {
    // We have a GPU vector type, so just use the existing pointer
    deviceX = ((Vec_SeqGPU*)x->data)->devptr;
  } else {
    fprintf(stderr, "Non-GPU vector types are not implemented!");
    exit(EXIT_FAILURE);
  }

  // Get device pointer for Y
  ierr = PetscObjectGetType((PetscObject)y, &vec_type); CHKERRQ(ierr);
  if (!strcmp(vec_type, "seqgpu")) {
    // We have a GPU vector type, so just use the existing pointer
    deviceY = ((Vec_SeqGPU*)y->data)->devptr;
  } else {
    fprintf(stderr, "Non-GPU vector types are not implemented!");
    exit(EXIT_FAILURE);
  }*/

  //PetscScalar * hostX;
  //PetscScalar * hostY;

  int mat_size = mat->m * mat->n * mat->p * mat->dof;

  //ierr = VecGetArray(x, &hostX); CHKERRQ(ierr);
  //ierr = VecGetArray(y, &hostY); CHKERRQ(ierr);

  //checkCudaError(hipMemcpyAsync(mat->deviceX, hostX, mat_size * sizeof(PetscScalar), hipMemcpyHostToDevice, mat->stream));

  CUSPARRAY * xgpu;
  CUSPARRAY * ygpu;

  ierr = VecCUSPGetArrayWrite(y, &ygpu); CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(x, &xgpu); CHKERRQ(ierr);

  PetscScalar * devX = thrust::raw_pointer_cast(&(*xgpu)[0]);
  PetscScalar * devY = thrust::raw_pointer_cast(&(*ygpu)[0]);

  // Bind X to device texture
  checkCudaError(hipBindTexture(0, vector_x, devX, mat_size * sizeof(PetscScalar)));

#if _TRACE
  printf("Host diagonals:\n");
  for (int i = 0; i < mat->diagonals->size(); ++i) {
    printf("- %d\n", (*mat->diagonals)[i]);
  }
#endif

  // Invoke
  dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
  dim3 grid((int)ceil((float)(mat->m * mat->n * mat->p * mat->dof)/(float)BLOCKWIDTH_X / 1.0), 1);

  //int shared_size = mat->diagonals->size() * sizeof(int);
  int shared_size = 0;

  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(MatMultKernel), hipFuncCachePreferL1);


#if _TIME
  double start, end;
  start = getclock();
#endif
  MatMultKernel<<<grid, block, shared_size, mat->stream>>>(mat->deviceData, devY, mat_size, mat->diagonals->size(), mat->deviceDiags, mat->dof);
#if _TIME
  checkCudaError(hipStreamSynchronize(mat->stream));
  end = getclock();
  double elapsed = end - start;
  double gflops = (2.0 * mat->non_zeros / elapsed / 1e9);

  double nos = ((mat->p == 1 ? 2 : 3) * 2 + 1) * (2*mat->dof - 1);
  double nz = mat->m * mat->n * mat->p * mat->dof;
  double alt_gflops = (2.0 * nos * nz) / ((end - start)*1024*1024*1024);

#if _CSV_OUT
  fprintf(stderr, "%d,%d,%d,%d,%lf,%lf,\n", mat->m, mat->n, mat->p, mat->dof, elapsed, gflops);
#endif
  printf("SGGPU Kernel Time:           %lf sec\n", elapsed);
  printf("SGGPU Kernel GFlop/s:        %lf\n", gflops);
  printf("SGGPU Kernel GFlop/s (alt):  %lf\n", alt_gflops);
#endif

  //checkCudaError(hipMemcpyAsync(hostY, mat->deviceY, mat_size * sizeof(PetscScalar), hipMemcpyDeviceToHost, mat->stream));

  // Cleanup
  hipUnbindTexture(vector_x);

  ierr = VecCUSPRestoreArrayRead(x, &xgpu); CHKERRQ(ierr);
  ierr = VecCUSPRestoreArrayWrite(y, &ygpu); CHKERRQ(ierr);

  ierr = WaitForGPU() ; CHKERRCUSP(ierr);

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValuesBlocked_SeqSGGPU"
PetscErrorCode MatSetValuesBlocked_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetValuesBlocked_SeqSGGPU\n");
#endif
  SETERRQ(PETSC_COMM_SELF,0,"MatSetValuesBlocked_SeqSGGPU not implemented");

}


#undef __FUNCT__
#define __FUNCT__ "MatSetValues_SeqSGGPU"
PetscErrorCode MatSetValues_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is)
{
  int i, j;
  PetscErrorCode ierr;
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;

  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetValues_SeqSGGPU\n");
#endif

  // Handle each element
  for (i = 0; i < nrow; i++) {
    for (j = 0; j < ncol; j++) {
      // Compute the diagonal and offset into the diagonal storage
      // for the element
      int row = irow[i];
      int col = icol[j];
      int diff = col - row;
      int left = row % mat->dof;
      int diag = int(floor((double)(diff + left) / mat->dof));
      int col_offset = col % mat->dof;
      int num_elems = mat->m * mat->n * mat->p * mat->dof;
      int offset = col_offset * num_elems + row;

#if _TRACE
      printf("- row: %d  col: %d  val: %lf  diag: %d  offset: %d\n", row, col, y[i*ncol+j], diag, offset);
#endif

      std::map<int, int> &diag_starts = *(mat->diag_starts);
      std::map<int, int>::iterator I = diag_starts.find(diag);
      int diag_offset = 0;
      if (I == diag_starts.end()) {
        printf("WARNING: malloc() in MatSetValues\n");
        // The diagonal does not yet exist, so add a new diagonal
        int num_diags = diag_starts.size() + 1;
        int size = num_diags * mat->m * mat->n * mat->p * mat->dof * mat->dof;
        PetscScalar *newData;
        ierr = PetscMalloc(size * sizeof(PetscScalar), &newData); CHKERRQ(ierr);
        memset(newData, 0, size * sizeof(PetscScalar));
        size -= mat->m * mat->n * mat->p * mat->dof * mat->dof;
        if (num_diags > 1) {
          // This is not the first diagonal, so copy
#if _TRACE
          printf("- Memcpy of %d elements\n", size);
#endif
          memcpy(newData, mat->hostData, size * sizeof(PetscScalar));
        }
        PetscFree(mat->hostData);
        mat->hostData = newData;
        diag_offset = size;
        diag_starts[diag] = diag_offset;
        mat->diagonals->push_back(diag);
      } else {
        // The diagonal already exists, so get the base offset
        diag_offset = I->second;
      }

      diag_offset += offset;

      if (is == INSERT_VALUES)
        mat->hostData[diag_offset] = y[i * ncol + j];
      else
        mat->hostData[diag_offset] += y[i * ncol + j];

      mat->non_zeros++;
    }
  }

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetStencil_SeqSGGPU"
PetscErrorCode MatSetStencil_SeqSGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetStencil_SeqSGGPU  (%p)\n", A);
#endif

  if (dim < 1 || dim > 3) {
    SETERRQ(PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "Dim must be between 1 and 3.");
  }

  mat->m = dims[0];
  if (dim > 1) {
    mat->n = dims[1];
    if (dim > 2) {
      mat->p = dims[2];
    } else {
      mat->p = 1;
    }
  } else {
    mat->n = 1;
    mat->p = 1;
  }

  mat->dof = dof;
  mat->dim = dim;

#if _TRACE
  printf("- m: %d  n: %d  p: %d  dof: %d\n", mat->m, mat->n, mat->p, mat->dof);
#endif

  // It appears that we are responsible for pre-allocating
  if (!A->preallocated) {
    ierr = MatSetUpPreallocation_SeqSGGPU(A); CHKERRQ(ierr);
  }

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetUpPreallocation_SeqSGGPU"
PetscErrorCode MatSetUpPreallocation_SeqSGGPU(Mat A)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;
  PetscErrorCode ierr;

  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetUpPreallocation_SeqSGGPU\n");
#endif

  if (mat->m == 0 || mat->n == 0 || mat->p == 0 || mat->dof == 0) {
    printf("MatSetPreallocation_SeqSGGPU called without valid m, n, p, and dof!");
    exit(0);
  }

  // Determine how many diagonals we should pre-allocate
  int num_diags = (2*mat->dim+1);
  int diag_size = mat->m * mat->n * mat->p * mat->dof * mat->dof;
  int size = num_diags * diag_size;

  ierr = PetscMalloc(size * sizeof(PetscScalar), &mat->hostData); CHKERRQ(ierr);
  memset(mat->hostData, 0, size * sizeof(PetscScalar));

  (*mat->diag_starts)[0]  = 0 * diag_size;
  (*mat->diagonals).push_back(0);
  (*mat->diag_starts)[1]  = 1 * diag_size;
  (*mat->diagonals).push_back(1);
  (*mat->diag_starts)[-1] = 2 * diag_size;
  (*mat->diagonals).push_back(-1);
  if (mat->dim > 1) {
    (*mat->diag_starts)[mat->m] = 3 * diag_size;
    (*mat->diagonals).push_back(mat->m);
    (*mat->diag_starts)[-mat->m] = 4 * diag_size;
    (*mat->diagonals).push_back(-mat->m);
  }
  if (mat->dim > 2) {
    (*mat->diag_starts)[mat->m*mat->n] = 5 * diag_size;
    (*mat->diagonals).push_back(mat->m*mat->n);
    (*mat->diag_starts)[-mat->m*mat->n] = 6 * diag_size;
    (*mat->diagonals).push_back(-mat->m*mat->n);
  }

  A->preallocated = PETSC_TRUE;

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatZeroEntries_SeqSGGPU"
PetscErrorCode MatZeroEntries_SeqSGGPU(Mat A)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatZeroEntries_SeqSGGPU\n");
#endif
  SETERRQ(PETSC_COMM_SELF,0,"MatZeroEntries_SeqSGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatGetDiagonal_SeqSGGPU"
PetscErrorCode MatGetDiagonal_SeqSGGPU(Mat A, Vec v)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatGetDiagonal_SeqSGGPU\n");
#endif
  SETERRQ(PETSC_COMM_SELF,0,"MatGetDiagonal_SeqSGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatDiagonalScale_SeqSGGPU"
PetscErrorCode MatDiagonalScale_SeqSGGPU(Mat A, Vec ll, Vec rr)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatDiagonalScale_SeqSGGPU\n");
#endif
  SETERRQ(PETSC_COMM_SELF,0,"MatDiagonalScale_SeqSGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRow_SeqSGGPU"
PetscErrorCode MatGetRow_SeqSGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatGetRow_SeqSGGPU\n");
#endif
  SETERRQ(PETSC_COMM_SELF,0,"MatGetRow_SeqSGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatRestoreRow_SeqSGGPU"
PetscErrorCode MatRestoreRow_SeqSGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatRestoreRow_SeqSGGPU\n");
#endif
  SETERRQ(PETSC_COMM_SELF,0,"MatRestoreRow_SeqSGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRowMaxAbs_SeqSGGPU"
PetscErrorCode MatGetRowMaxAbs_SeqSGGPU(Mat A, Vec v, PetscInt idx[])
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatGetRowMaxAbs_SeqSGGPU\n");
#endif
  SETERRQ(PETSC_COMM_SELF,0,"MatGetRowMaxAbs_SeqSGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatView_SeqSGGPU"
PetscErrorCode MatView_SeqSGGPU(Mat A, PetscViewer viewer)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatView_SeqSGGPU\n");
#endif
  SETERRQ(PETSC_COMM_SELF,0,"MatView_SeqSGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyBegin_SeqSGGPU"
PetscErrorCode MatAssemblyBegin_SeqSGGPU(Mat A, MatAssemblyType type)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatAssemblyBegin_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_SeqSGGPU"
PetscErrorCode MatAssemblyEnd_SeqSGGPU(Mat A, MatAssemblyType type)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatAssemblyEnd_SeqSGGPU\n");

  for (std::map<int, int>::iterator I = mat->diag_starts->begin(),
       E = mat->diag_starts->end(); I != E; ++I) {
    printf("- Diag %d:\n", I->first);
    for (int i = 0; i < mat->dof; ++i) {
      for (int j = 0; j < mat->dof * mat->m * mat->n * mat->p; ++j) {
        int offset = i * mat->dof * mat->m * mat->n * mat->p + j;
        printf(" %lf ", mat->hostData[offset + I->second]);
      }
      printf("\n");
    }
  }
#endif

  // Create GPU buffer
  if (mat->deviceData) {
    hipFree(mat->deviceData);
  }
  int size = mat->diag_starts->size() * mat->m * mat->n * mat->p * mat->dof * mat->dof;
  checkCudaError(hipMalloc(&mat->deviceData, sizeof(PetscScalar) * size));

  // Copy data to device
  checkCudaError(hipMemcpy(mat->deviceData, mat->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));


  int mat_size = mat->m * mat->n * mat->p * mat->dof;

  // We know the expected size of x, y, so go ahead and allocate them now
  checkCudaError(hipMalloc(&mat->deviceX, mat_size * sizeof(PetscScalar)));
  checkCudaError(hipMalloc(&mat->deviceY, mat_size * sizeof(PetscScalar)));

  // We also know how many diagonals we have, and their indices
  checkCudaError(hipMalloc(&mat->deviceDiags, sizeof(int) * mat->diagonals->size()));
  checkCudaError(hipMemcpyAsync(mat->deviceDiags, &(*mat->diagonals)[0], sizeof(int) * mat->diagonals->size(), hipMemcpyHostToDevice, mat->stream));

  PetscFunctionReturn(0);
}

