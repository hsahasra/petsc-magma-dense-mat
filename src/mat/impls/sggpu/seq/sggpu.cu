/// SGGPU Matrix Type

#define PETSCMAT_DLL

#include "petsc-private/matimpl.h"
#include "sggpu.h"

#include <stdio.h>
#include <hip/hip_runtime.h>


// Debugging flags
#define _TRACE 1

// Prototypes
PetscErrorCode MatDestroy_SeqSGGPU(Mat A);
PetscErrorCode MatSetGrid_SeqSGGPU(Mat B, PetscInt m, PetscInt n, PetscInt p);
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y);
PetscErrorCode MatSetValuesBlocked_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is);
PetscErrorCode MatSetValues_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is);
PetscErrorCode MatSetStencil_SeqSGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof);
PetscErrorCode MatSetUpPreallocation_SeqSGGPU(Mat mat);
PetscErrorCode MatZeroEntries_SeqSGGPU(Mat A);
PetscErrorCode MatGetDiagonal_SeqSGGPU(Mat A, Vec v);
PetscErrorCode MatDiagonalScale_SeqSGGPU(Mat A, Vec ll, Vec rr);
PetscErrorCode MatGetRow_SeqSGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v);
PetscErrorCode MatRestoreRow_SeqSGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v);
PetscErrorCode MatGetRowMaxAbs_SeqSGGPU(Mat A, Vec v, PetscInt idx[]);
PetscErrorCode MatView_SeqSGGPU(Mat A, PetscViewer viewer);


// Matrix function table
static struct _MatOps MatOps_Values = {
/*0*/ MatSetValues_SeqSGGPU,MatGetRow_SeqSGGPU,MatRestoreRow_SeqSGGPU,MatMult_SeqSGGPU,0,
/*5*/0,0,0,0,0,
/*10*/0,0,0,0,0,
/*15*/0,0,MatGetDiagonal_SeqSGGPU,MatDiagonalScale_SeqSGGPU,0,
/*20*/0,0,0,MatZeroEntries_SeqSGGPU,0,
/*25*/0,0,0,0,MatSetUpPreallocation_SeqSGGPU,
/*30*/0,0,0,0,0,
/*35*/0,0,0,0,0,
/*40*/0,0,0,0,0,
/*45*/0,0,0,0,0,
/*50*/0,0,0,0,0,
/*55*/0,0,0,MatSetValuesBlocked_SeqSGGPU,0,
/*60*/MatDestroy_SeqSGGPU,MatView_SeqSGGPU,0,0,0,
/*65*/0,0,MatSetValues_SeqSGGPU,0,MatGetRowMaxAbs_SeqSGGPU,
/*70*/0,0,0,0,0,
/*75*/0,0,0,0,0,
/*80*/0,0,0,0,0,
/*85*/0,0,MatSetValuesBlocked_SeqSGGPU,0,0,
/*90*/0,0,0,0,0,
/*95*/0,0,0,0,0,
/*100*/0,0,0,0,0,
/*105*/0,0,0,0,0,
/*110*/0,0,0,0,0,
/*115*/MatCreate_SeqSGGPU,0,0,0,0,
/*120*/0,0,0,0,0,
/*125*/0,0,0,0,0,
/*130*/0,0,0,0,0,
/*135*/0,0,0,0,MatSetStencil_SeqSGGPU,
/*140*/MatSetGrid_SeqSGGPU
};


EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode MatCreate_SeqSGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;
  PetscMPIInt size;

  PetscFunctionBegin;

#if _TRACE
  printf("[SeqSGGPU] MatCreate_SeqSGGPU\n");
#endif

  ierr = MPI_Comm_size(((PetscObject)A)->comm, &size); CHKERRQ(ierr);
  if (size > 1)
    SETERRQ(PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "Comm must be size 1");

  // Create internal matrix structure
  ierr = PetscMalloc(sizeof(Mat_SeqSGGPU), &mat); CHKERRQ(ierr);
  memset(mat, 0, sizeof(Mat_SeqSGGPU));

  // Fill out PETSc matrix structure
  A->data = mat;
  memcpy(A->ops, &MatOps_Values, sizeof(struct _MatOps));
  A->same_nonzero= PETSC_FALSE;
  A->spptr = 0;

  // Set object type
  ierr = PetscObjectChangeTypeName((PetscObject)A, MATSEQSGGPU); CHKERRQ(ierr);

  PetscFunctionReturn(0);
}
EXTERN_C_END


#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqSGGPU"
PetscErrorCode MatDestroy_SeqSGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;

  PetscFunctionBegin;

#if _TRACE
  printf("[SeqSGGPU] MatDestroy_SeqSGGPU\n");
#endif

  mat = (Mat_SeqSGGPU*)A->data;

  if (mat->hostData) {
    ierr = PetscFree(mat->hostData); CHKERRQ(ierr);
  }
  if (mat->deviceData) {
    hipFree(mat->deviceData);
  }
  PetscFree(mat); CHKERRQ(ierr);

  ierr = PetscObjectChangeTypeName((PetscObject)A, 0); CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetGrid_SeqSGGPU"
PetscErrorCode MatSetGrid_SeqSGGPU(Mat B, PetscInt m, PetscInt n, PetscInt p)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetGrid_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatMult_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValuesBlocked_SeqSGGPU"
PetscErrorCode MatSetValuesBlocked_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetValuesBlocked_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValues_SeqSGGPU"
PetscErrorCode MatSetValues_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetValues_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetStencil_SeqSGGPU"
PetscErrorCode MatSetStencil_SeqSGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetStencil_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetUpPreallocation_SeqSGGPU"
PetscErrorCode MatSetUpPreallocation_SeqSGGPU(Mat mat)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatSetUpPreallocation_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatZeroEntries_SeqSGGPU"
PetscErrorCode MatZeroEntries_SeqSGGPU(Mat A)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatZeroEntries_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetDiagonal_SeqSGGPU"
PetscErrorCode MatGetDiagonal_SeqSGGPU(Mat A, Vec v)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatGetDiagonal_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatDiagonalScale_SeqSGGPU"
PetscErrorCode MatDiagonalScale_SeqSGGPU(Mat A, Vec ll, Vec rr)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatDiagonalScale_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRow_SeqSGGPU"
PetscErrorCode MatGetRow_SeqSGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatGetRow_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatRestoreRow_SeqSGGPU"
PetscErrorCode MatRestoreRow_SeqSGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatRestoreRow_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRowMaxAbs_SeqSGGPU"
PetscErrorCode MatGetRowMaxAbs_SeqSGGPU(Mat A, Vec v, PetscInt idx[])
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatGetRowMaxAbs_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatView_SeqSGGPU"
PetscErrorCode MatView_SeqSGGPU(Mat A, PetscViewer viewer)
{
  PetscFunctionBegin;
#if _TRACE
  printf("[SeqSGGPU] MatView_SeqSGGPU\n");
#endif
  PetscFunctionReturn(0);
}
