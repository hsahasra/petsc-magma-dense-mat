#include "hip/hip_runtime.h"
/// SGGPU Matrix Type

#define PETSCMAT_DLL

// Debugging flags
#define _TRACE 0
#define _TIME 0
#define _CSV_OUT 0

#if _TRACE
#define SGTrace printf("[SeqSGGPU] %s\n",__FUNCT__);
#else
#define SGTrace
#endif

#include "petsc-private/matimpl.h"
#include "sggpu.h"

// Direct access to seqgpu vector type
#include "../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h"

// Interop with CUSP vector
#include "../src/vec/vec/impls/seq/seqcusp/cuspvecimpl.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

// C++ library headers
#include <map>


// Hard-coded block size
#define BLOCKWIDTH_X 128
#define BLOCKWIDTH_Y 1


// ----------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
EXTERN_C_BEGIN
void checkCudaError(hipError_t err) {
  if(hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}
EXTERN_C_END
// ----------------------------------------------------------



//------------------------------------------------------
// general timer function using unix system call
// dlowell ANL-MCS
//------------------------------------------------------
double getclock() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return (tp.tv_sec + tp.tv_usec*1.0e-6);
}

//===-- CUDA Device Code -------------------------------------------------===//

texture<int2, 1> vector_x;

static __inline__ __device__ double fetch_double(texture<int2, 1> tex, int i)
{
  int2 v = tex1Dfetch(tex, i);
  return __hiloint2double(v.y, v.x);
}

__global__ void MatMultKernel(PetscScalar * coeff, PetscScalar * y, PetscInt mat_size, PetscInt num_diags, int * diagonals, PetscInt dof) {
  int idx = blockDim.x * blockIdx.x * 1 + threadIdx.x * 1;

  if (idx >= mat_size)
    return;

  int diag_size = mat_size * dof;

  PetscScalar yval0 = 0.0;
  int idx0 = idx;

  //#pragma unroll 4
  for (int i = 0; i < num_diags; ++i) {
    int d = diagonals[i];

    int offset0 = diag_size * i + idx0;
    int block0 = (idx0 / dof + d) * dof;

    //#pragma unroll 12
    for (int j = 0; j < dof; ++j) {
      // Get coefficient
      PetscScalar aval0 = coeff[offset0 + mat_size*j];
      // Get X value
      PetscScalar xval0 = fetch_double(vector_x, block0 + j);

      yval0 += aval0 * xval0;
    }
  }

  y[idx0] = yval0;
}

//===-- Host Code --------------------------------------------------------===//


// Matrix function table
static struct _MatOps MatOps_Values = {
/*0*/ MatSetValues_SeqSGGPU,MatGetRow_SeqSGGPU,MatRestoreRow_SeqSGGPU,MatMult_SeqSGGPU,0,
/*5*/0,0,0,0,0,
/*10*/0,0,0,0,0,
/*15*/0,0,MatGetDiagonal_SeqSGGPU,MatDiagonalScale_SeqSGGPU,0,
/*20*/MatAssemblyBegin_SeqSGGPU,MatAssemblyEnd_SeqSGGPU,0,MatZeroEntries_SeqSGGPU,0,
/*25*/0,0,0,0,MatSetUp_SeqSGGPU,
/*30*/0,0,0,0,0,
/*35*/0,0,0,0,0,
/*40*/0,0,0,0,0,
/*45*/0,0,0,0,0,
/*50*/0,0,MatGetColumnIJ_SeqSGGPU,0,MatFDColoringCreate_SeqSGGPU,
/*55*/0,0,0,MatSetValuesBlocked_SeqSGGPU,0,
/*60*/MatDestroy_SeqSGGPU,MatView_SeqSGGPU,0,0,0,
/*65*/0,0,MatSetValues_SeqSGGPU,0,MatGetRowMaxAbs_SeqSGGPU,
/*70*/0,0,0,0,0,
/*75*/MatFDColoringApply_SeqSGGPU,0,0,0,0,
/*80*/0,0,0,0,0,
/*85*/0,0,MatSetValuesBlocked_SeqSGGPU,0,0,
/*90*/0,0,0,0,0,
/*95*/0,0,0,0,0,
/*100*/0,0,0,0,0,
/*105*/0,0,0,0,0,
/*110*/0,0,0,0,0,
/*115*/MatCreate_SeqSGGPU,0,0,0,0,
/*120*/0,0,0,0,0,
/*125*/0,0,0,0,0,
/*130*/0,0,0,0,0,
/*135*/0,0,0,0,0,
/*140*/0,0,
/*142*/MatSetGrid_SeqSGGPU
};




EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode MatCreate_SeqSGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;
  PetscMPIInt size;

  PetscFunctionBegin;
  SGTrace;

	PetscPrintf(PETSC_COMM_WORLD,"MatCreate_SeqSGGPU\n");
	

  ierr = MPI_Comm_size(((PetscObject)A)->comm, &size); CHKERRQ(ierr);
  if (size > 1)
    SETERRQ(PETSC_COMM_SELF, PETSC_ERR_ARG_OUTOFRANGE, "Comm must be size 1");

  // Create internal matrix structure
  ierr = PetscMalloc(sizeof(Mat_SeqSGGPU), &mat); CHKERRQ(ierr);
  memset(mat, 0, sizeof(Mat_SeqSGGPU));
  mat->diag_starts = new std::map<int, int>();
  mat->diagonals = new std::vector<int>();

  checkCudaError(hipStreamCreate(&mat->stream));

  // Fill out PETSc matrix structure
  A->data = mat;
  memcpy(A->ops, &MatOps_Values, sizeof(struct _MatOps));
  A->same_nonzero= PETSC_FALSE;
  A->spptr = 0;

  // Set object type
  ierr = PetscObjectChangeTypeName((PetscObject)A, MATSEQSGGPU); CHKERRQ(ierr);

  ierr = PetscObjectComposeFunctionDynamic((PetscObject)A,
        "MatSeqSGGPUSetPreallocation_C","MatSeqSGGPUSetPreallocation_SeqDIA",
        MatSeqSGGPUSetPreallocation_SeqSGGPU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
EXTERN_C_END


#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqSGGPU"
PetscErrorCode MatDestroy_SeqSGGPU(Mat A)
{
  Mat_SeqSGGPU * mat;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  SGTrace;


	PetscPrintf(PETSC_COMM_WORLD,"MatDestroy_SeqSGGPU\n");

  mat = (Mat_SeqSGGPU*)A->data;

  if (mat->hostData) {
    ierr = PetscFree(mat->hostData); CHKERRQ(ierr);
  }
  if (mat->deviceData) {
    hipFree(mat->deviceData);
  }
  if (mat->diag_starts) {
    delete mat->diag_starts;
  }
  ierr = PetscFree(mat->diag_offsets); CHKERRQ(ierr);
  if (mat->diagonals) {
    delete mat->diagonals;
  }
  if (mat->deviceX) {
    hipFree(mat->deviceX);
  }
  if (mat->deviceY) {
    hipFree(mat->deviceY);
  }
  if (mat->deviceDiags) {
    hipFree(mat->deviceDiags);
  }
  if(mat->ja)       { ierr = PetscFree(mat->ja); CHKERRQ(ierr);       }
  if(mat->ia)       { ierr = PetscFree(mat->ia); CHKERRQ(ierr);       }
  checkCudaError(hipStreamDestroy(mat->stream));
  PetscFree(mat); CHKERRQ(ierr);

  ierr = PetscObjectChangeTypeName((PetscObject)A, 0); CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetGrid_SeqSGGPU"
PetscErrorCode MatSetGrid_SeqSGGPU(Mat B, PetscInt m, PetscInt n, PetscInt p)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)B->data;

  PetscFunctionBegin;
  SGTrace;

		PetscPrintf(PETSC_COMM_WORLD,"MatSetGrid_SeqSGGPU\n");

  mat->m = m;
  mat->n = n;
  mat->p = p;

  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat A, Vec x, Vec y)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;
  PetscBool isseqcusp,isseqgpu,ismpicusp,iscusp;
  PetscErrorCode ierr;
  PetscInt mat_size;
  CUSPARRAY *xgpu,*ygpu;
  PetscScalar *devX,*devY;


  PetscFunctionBegin;
  SGTrace;


	PetscPrintf(PETSC_COMM_WORLD,"MatMult_SeqSGGPU\n");


  // Initialize y to zero
  ierr = VecSet(y, 0.0); CHKERRQ(ierr);

  ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQCUSP,&isseqcusp);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)x,VECMPICUSP,&ismpicusp);CHKERRQ(ierr);
  iscusp = (isseqcusp || ismpicusp) ? PETSC_TRUE : PETSC_FALSE;
  ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQGPU,&isseqgpu);CHKERRQ(ierr);
  if (isseqgpu) {
    dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
    dim3 grid((int)ceil((float)(mat->m * mat->n * mat->p * mat->dof)/(float)BLOCKWIDTH_X / 1.0), 1);

    int shared_size = 0;
    Vec_SeqGPU *vx = (Vec_SeqGPU*) x->data;
    Vec_SeqGPU *vy = (Vec_SeqGPU*) y->data;
    /* Make sure y is also VECSEQGPU */
    ierr = PetscObjectTypeCompare((PetscObject)x,VECSEQGPU,&isseqgpu);CHKERRQ(ierr);
    if (!isseqgpu) {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Both x and y must be same type");
    }
    /* synch up x */
    if (vx->syncState==VEC_CPU) {
      ierr = VecCopyOverH2D(x,vx->cpuptr);CHKERRQ(ierr);
      vx->syncState=VEC_SYNCHED;
    }
    /* Get device pointer for X */
    devX = vx->devptr;
    devY = vy->devptr;
    /* Bind X to device texture */
    mat_size = mat->m * mat->n * mat->p * mat->dof;

    checkCudaError(hipBindTexture(0, vector_x, devX, mat_size * sizeof(PetscScalar)));
    MatMultKernel<<<grid, block, shared_size, mat->stream>>>(mat->deviceData, devY, mat_size, mat->diagonals->size(), mat->deviceDiags, mat->dof);

    hipUnbindTexture(vector_x);
    hipDeviceSynchronize();


    

  } else if (iscusp) {
    dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
    dim3 grid((int)ceil((float)(mat->m * mat->n * mat->p * mat->dof)/(float)BLOCKWIDTH_X / 1.0), 1);

    int shared_size = 0;
    /* Make sure y is also VECCUSP */
    ierr = PetscObjectTypeCompare((PetscObject)x,VECCUSP,&isseqgpu);CHKERRQ(ierr);
    if (!iscusp) {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Both x and y must be same type");
    }

    mat_size = mat->m * mat->n * mat->p * mat->dof;
    ierr = VecCUSPGetArrayWrite(y, &ygpu); CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(x, &xgpu); CHKERRQ(ierr);
    devY = thrust::raw_pointer_cast(&(*ygpu)[0]);
    devX = thrust::raw_pointer_cast(&(*xgpu)[0]);

    /* Bind X to device texture */
    checkCudaError(hipBindTexture(0, vector_x, devX, mat_size * sizeof(PetscScalar)));

#if _TRACE
    printf("Host diagonals:\n");
    for (int i = 0; i < mat->diagonals->size(); ++i) {
      printf("- %d\n", (*mat->diagonals)[i]);
    }
#endif

    /* Invoke */

#if _TIME
    double start, end;
    start = getclock();
#endif
      MatMultKernel<<<grid, block, shared_size, mat->stream>>>(mat->deviceData, devY, mat_size, mat->diagonals->size(), mat->deviceDiags, mat->dof);
#if _TIME
    checkCudaError(hipStreamSynchronize(mat->stream));
    end = getclock();
    double elapsed = end - start;
    double gflops = (2.0 * mat->non_zeros / elapsed / 1e9);

    double nos = ((mat->p == 1 ? 2 : 3) * 2 + 1) * (2*mat->dof - 1);
    double nz = mat->m * mat->n * mat->p * mat->dof;
    double alt_gflops = (2.0 * nos * nz) / ((end - start)*1024*1024*1024);

#if _CSV_OUT
    fprintf(stderr, "%d,%d,%d,%d,%lf,%lf,\n", mat->m, mat->n, mat->p, mat->dof, elapsed, gflops);
#endif
    printf("SGGPU Kernel Time:           %lf sec\n", elapsed);
    printf("SGGPU Kernel GFlop/s:        %lf\n", gflops);
    printf("SGGPU Kernel GFlop/s (alt):  %lf\n", alt_gflops);
#endif

    /* Cleanup */
    hipUnbindTexture(vector_x);

    ierr = VecCUSPRestoreArrayRead(x, &xgpu); CHKERRQ(ierr);
    ierr = VecCUSPRestoreArrayWrite(y, &ygpu); CHKERRQ(ierr);
    ierr = WaitForGPU() ; CHKERRCUSP(ierr);
    hipDeviceSynchronize();
  } else {
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Vec must be seqgpu or cusp type");
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatSetValuesBlocked_SeqSGGPU"
PetscErrorCode MatSetValuesBlocked_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is)
{
  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatSetValuesBlocked_SeqSGGPU not implemented");

}


#undef __FUNCT__
#define __FUNCT__ "MatSetValues_SeqSGGPU"
PetscErrorCode MatSetValues_SeqSGGPU(Mat A, PetscInt nrow, const PetscInt irow[], PetscInt ncol, const PetscInt icol[], const PetscScalar y[], InsertMode is)
{
  int i, j;
  PetscErrorCode ierr;
  PetscBool resizegpu = PETSC_FALSE;
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;

  PetscFunctionBegin;
  SGTrace;


	PetscPrintf(PETSC_COMM_WORLD,"MatSetValues_SeqSGGPU\n");


  // Handle each element
  for (i = 0; i < nrow; i++) {
    for (j = 0; j < ncol; j++) {
      // Compute the diagonal and offset into the diagonal storage
      // for the element
      int row = irow[i];
      int col = icol[j];
      int diff = col - row;
      int left = row % mat->dof;
      int diag = int(floor((double)(diff + left) / mat->dof));
      int col_offset = col % mat->dof;
      int num_elems = mat->m * mat->n * mat->p * mat->dof;
      int offset = col_offset * num_elems + row;

#if _TRACE
      printf("- row: %d  col: %d  val: %lf  diag: %d  offset: %d\n", row, col, y[i*ncol+j], diag, offset);
#endif

      std::map<int, int> &diag_starts = *(mat->diag_starts);
      std::map<int, int>::iterator I = diag_starts.find(diag);
      int diag_offset = 0;
      if (I == diag_starts.end()) {
        printf("WARNING: malloc() in MatSetValues\n");
        resizegpu = PETSC_TRUE;
        // The diagonal does not yet exist, so add a new diagonal
        int num_diags = diag_starts.size() + 1;
        int size = num_diags * mat->m * mat->n * mat->p * mat->dof * mat->dof;
        PetscScalar *newData;
        ierr = PetscMalloc(size * sizeof(PetscScalar), &newData); CHKERRQ(ierr);
        memset(newData, 0, size * sizeof(PetscScalar));
        size -= mat->m * mat->n * mat->p * mat->dof * mat->dof;
        if (num_diags > 1) {
          // This is not the first diagonal, so copy
#if _TRACE
          printf("- Memcpy of %d elements\n", size);
#endif
          memcpy(newData, mat->hostData, size * sizeof(PetscScalar));
        }
        PetscFree(mat->hostData);
        mat->hostData = newData;
        diag_offset = size;
        diag_starts[diag] = diag_offset;
        mat->diagonals->push_back(diag);
      } else {
        // The diagonal already exists, so get the base offset
        diag_offset = I->second;
      }

      diag_offset += offset;

      if (is == INSERT_VALUES)
        mat->hostData[diag_offset] = y[i * ncol + j];
      else
        mat->hostData[diag_offset] += y[i * ncol + j];

      mat->non_zeros++;
    }
  }
  if (resizegpu) {
    int size,mat_size;
    // Create GPU buffer
    if (mat->deviceData) {
      hipFree(mat->deviceData);
    }
    size = mat->diag_starts->size() * mat->m * mat->n * mat->p * mat->dof * mat->dof;
    checkCudaError(hipMalloc(&mat->deviceData, sizeof(PetscScalar) * size));


    mat_size = mat->m * mat->n * mat->p * mat->dof;

    if (mat->deviceX) {
      hipFree(mat->deviceX);
    }
    if (mat->deviceY) {
      hipFree(mat->deviceY);
    }
    if (mat->deviceDiags) {
      hipFree(mat->deviceDiags);
    }
    // We know the expected size of x, y, so go ahead and allocate them now
    checkCudaError(hipMalloc(&mat->deviceX, mat_size * sizeof(PetscScalar)));
    checkCudaError(hipMalloc(&mat->deviceY, mat_size * sizeof(PetscScalar)));

    // We also know how many diagonals we have, and their indices
    checkCudaError(hipMalloc(&mat->deviceDiags, sizeof(int) * mat->diagonals->size()));
  }

  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "MatFDColoringView_Private"
PetscErrorCode MatFDColoringView_Private(MatFDColoring fd)
{
  PetscErrorCode ierr;
  PetscBool      flg = PETSC_FALSE;
  PetscViewer    viewer;

  PetscFunctionBegin;

	PetscPrintf(PETSC_COMM_WORLD,"MatFDColoringView_Private\n");


  ierr = PetscViewerASCIIGetStdout(((PetscObject)fd)->comm,&viewer);CHKERRQ(ierr);
  ierr = PetscOptionsGetBool(PETSC_NULL,"-mat_fd_coloring_view",&flg,PETSC_NULL);CHKERRQ(ierr);
  if (flg) {
    ierr = MatFDColoringView(fd,viewer);CHKERRQ(ierr);
  }
  flg  = PETSC_FALSE;
  ierr = PetscOptionsGetBool(PETSC_NULL,"-mat_fd_coloring_view_info",&flg,PETSC_NULL);CHKERRQ(ierr);
  if (flg) {
    ierr = PetscViewerPushFormat(viewer,PETSC_VIEWER_ASCII_INFO);CHKERRQ(ierr);
    ierr = MatFDColoringView(fd,viewer);CHKERRQ(ierr);
    ierr = PetscViewerPopFormat(viewer);CHKERRQ(ierr);
  }
  flg  = PETSC_FALSE;
  ierr = PetscOptionsGetBool(PETSC_NULL,"-mat_fd_coloring_view_draw",&flg,PETSC_NULL);CHKERRQ(ierr);
  if (flg) {
    ierr = MatFDColoringView(fd,PETSC_VIEWER_DRAW_(((PetscObject)fd)->comm));CHKERRQ(ierr);
    ierr = PetscViewerFlush(PETSC_VIEWER_DRAW_(((PetscObject)fd)->comm));CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "MatSetUp_SeqSGGPU"
PetscErrorCode MatSetUp_SeqSGGPU(Mat A)
{

  PetscFunctionBegin;
  SGTrace;

	PetscPrintf(PETSC_COMM_WORLD,"MatSetUP_SeqSGGPU\n");


  //  ierr =  MatSeqSGGPUSetPreallocation(A,PETSC_DEFAULT,0);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "MatSeqSGGPUSetPreallocation"
PetscErrorCode MatSeqSGGPUSetPreallocation(Mat A,PetscInt stencil_type, PetscInt dof)
{
  PetscErrorCode ierr;
  Mat_SeqSGGPU *mat = (Mat_SeqSGGPU*)A->data;
  PetscFunctionBegin;

	PetscPrintf(PETSC_COMM_WORLD,"MatSeqSGGPUSetPreallocation\n");


  mat->stencil_type = stencil_type;
  mat->dof = dof;
  if(A->preallocated)PetscFunctionReturn(0);
  PetscValidHeaderSpecific(A,MAT_CLASSID,1);
  
  ierr = PetscTryMethod(A,"MatSeqSGGPUSetPreallocation_C",(Mat,PetscInt,const PetscInt []),(A,0,0));CHKERRQ(ierr);
  A->preallocated=PETSC_TRUE;
  PetscFunctionReturn(0);
}

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatSeqSGGPUSetPreallocation_SeqSGGPU"
extern PetscErrorCode MatSeqSGGPUSetPreallocation_SeqSGGPU(Mat A,PetscInt nz, const PetscInt nnz[])
{
  PetscErrorCode ierr;
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;
  PetscInt dim,diag_size,size,num_diags,i,vecsize;

	PetscPrintf(PETSC_COMM_WORLD,"MateqSGGPUSetPreallocation_SeqSGGPU\n");


  ierr = PetscLayoutSetBlockSize(A->rmap,1);CHKERRQ(ierr);
  ierr = PetscLayoutSetBlockSize(A->cmap,1);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->rmap);CHKERRQ(ierr);
  ierr = PetscLayoutSetUp(A->cmap);CHKERRQ(ierr);

  dim = A->stencil.dim;
  if (mat->dof > 1) {
    dim--;
  }
  mat->m = mat->n = mat->p = 1;
  mat->dim = dim;
  if (mat->dim > 0) mat->m = A->stencil.dims[dim-1];
  if (mat->dim > 1) mat->n = A->stencil.dims[dim-2];
  if (mat->dim > 2) mat->p = A->stencil.dims[dim-3];

  if (mat->stencil_type == 0) {
    /* star stencil */
    num_diags = 2*mat->dim + 1;
  } else {
    /* box stencil */
    num_diags =  1;
    for (i=0;i<mat->dim;i++) num_diags*=3;
  }

  diag_size = mat->m * mat->n * mat->p * mat->dof * mat->dof;
  size = num_diags * diag_size;

  if (mat->m == 0 || mat->n == 0 || mat->p == 0 || mat->dof == 0) {
    SETERRQ(PETSC_COMM_SELF,0,"MatSetPreallocation_SeqSGGPU called without valid m, n, p, and dof!");
  }


  ierr = PetscMalloc(sizeof(PetscInt)*num_diags,&mat->diag_offsets);
  ierr = PetscMalloc(size * sizeof(PetscScalar), &mat->hostData); CHKERRQ(ierr);
  memset(mat->hostData, 0, size * sizeof(PetscScalar));

  (*mat->diag_starts)[0]  = 0 * diag_size;
  (*mat->diagonals).push_back(0);
  (*mat->diag_starts)[1]  = 1 * diag_size;
  (*mat->diagonals).push_back(1);
  (*mat->diag_starts)[-1] = 2 * diag_size;
  (*mat->diagonals).push_back(-1);
  if (mat->stencil_type == 0) {
    if (mat->dim == 2) {
      (*mat->diag_starts)[mat->m] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->m] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
    } else if (mat->dim == 3) {
      (*mat->diag_starts)[mat->m] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->m] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);

      (*mat->diag_starts)[mat->m*mat->n] = 5 * diag_size;
      (*mat->diagonals).push_back(mat->m*mat->n);
      (*mat->diag_starts)[-mat->m*mat->n] = 6 * diag_size;
      (*mat->diagonals).push_back(-mat->m*mat->n);
    }
  } else {
    if (mat->dim == 2) {
      (*mat->diag_starts)[mat->n-1] = 3 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n-1] = 4 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
      (*mat->diag_starts)[mat->n] = 5 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n] = 6 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
      (*mat->diag_starts)[mat->n+1] = 7 * diag_size;
      (*mat->diagonals).push_back(mat->m);
      (*mat->diag_starts)[-mat->n+1] = 8 * diag_size;
      (*mat->diagonals).push_back(-mat->m);
    }
  }
  /*
  printf("Diagonals preallocated:\n");
  for (std::map<int, int>::iterator I = mat->diag_starts->begin(),
         E = mat->diag_starts->end(); I != E; ++I) {
    printf("%4d --> %4d\n",I->first,I->second);
  }
   */
  
  
  // Create GPU buffer
  if (mat->deviceData) {
    hipFree(mat->deviceData);
  }
  checkCudaError(hipMalloc(&mat->deviceData, sizeof(PetscScalar) * size));
  checkCudaError(hipMemset(mat->deviceData,0.0,sizeof(PetscScalar)*size));

  // Copy data to device
  checkCudaError(hipMemcpy(mat->deviceData, mat->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));


  vecsize = mat->m * mat->n * mat->p * mat->dof;

  // We know the expected size of x, y, so go ahead and allocate them now
  checkCudaError(hipMalloc(&mat->deviceX, vecsize * sizeof(PetscScalar)));
  checkCudaError(hipMalloc(&mat->deviceY, vecsize * sizeof(PetscScalar)));

  // We also know how many diagonals we have, and their indices
  checkCudaError(hipMalloc(&mat->deviceDiags, sizeof(int) * mat->diagonals->size()));
  A->preallocated = PETSC_TRUE;
  ierr = MatAssemblyBegin(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyEnd(A,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  

  PetscFunctionReturn(0);
}
EXTERN_C_END


#undef __FUNCT__
#define __FUNCT__ "MatZeroEntries_SeqSGGPU"
PetscErrorCode MatZeroEntries_SeqSGGPU(Mat A)
{
  Mat_SeqSGGPU *mat = (Mat_SeqSGGPU*)A->data;
  PetscInt size;
  PetscFunctionBegin;
  SGTrace;

	PetscPrintf(PETSC_COMM_WORLD,"MatZeroEntries_SeqSGGPU\n");


  size = mat->diag_starts->size() * mat->m * mat->n * mat->p * mat->dof * mat->dof;
  memset(mat->hostData, 0, size * sizeof(PetscScalar));
  
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatGetDiagonal_SeqSGGPU"
PetscErrorCode MatGetDiagonal_SeqSGGPU(Mat A, Vec v)
{
  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatGetDiagonal_SeqSGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatDiagonalScale_SeqSGGPU"
PetscErrorCode MatDiagonalScale_SeqSGGPU(Mat A, Vec ll, Vec rr)
{
  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatDiagonalScale_SeqSGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRow_SeqSGGPU"
PetscErrorCode MatGetRow_SeqSGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v)
{
  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatGetRow_SeqSGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatRestoreRow_SeqSGGPU"
PetscErrorCode MatRestoreRow_SeqSGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v)
{
  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatRestoreRow_SeqSGGPU not implemented");
}


#undef __FUNCT__
#define __FUNCT__ "MatGetRowMaxAbs_SeqSGGPU"
PetscErrorCode MatGetRowMaxAbs_SeqSGGPU(Mat A, Vec v, PetscInt idx[])
{
  PetscFunctionBegin;
  SGTrace;
  SETERRQ(PETSC_COMM_SELF,0,"MatGetRowMaxAbs_SeqSGGPU not implemented");
}

#undef __FUNCT__
#define __FUNCT__ "MatView_SeqSGGPU_ASCII"
PetscErrorCode MatView_SeqSGGPU_ASCII(Mat A, PetscViewer viewer)
{
  Mat_SeqSGGPU *a = (Mat_SeqSGGPU*)A->data;
  PetscErrorCode ierr;
  PetscInt nrows,ndiag,dof,i,j,iblock,col,index,offset;
  std::map<int, int> &diag_starts = *(a->diag_starts);
  

  PetscFunctionBegin;  



  hipDeviceSynchronize();
  ierr = PetscViewerASCIIUseTabs(viewer,PETSC_FALSE);CHKERRQ(ierr);
  ierr = PetscObjectPrintClassNamePrefixType((PetscObject)A,viewer,"Matrix Object");CHKERRQ(ierr);
  ierr = PetscViewerASCIIPrintf(viewer,"MatView_SeqSGGPU_ASCII still in development\n");

  nrows = a->m * a->n * a->p * a->dof;
  ndiag = a->diagonals->size();
  dof = a->dof;

  ierr = PetscViewerASCIIPrintf(viewer,"offsets: \n"); CHKERRQ(ierr);
  for (std::map<int, int>::iterator I = diag_starts.begin(),
         E = diag_starts.end(); I != E; ++I) {
    PetscViewerASCIIPrintf(viewer,"- Diag %d:%d\n", I->first, I->second);
  }
  ierr = PetscViewerASCIIPrintf(viewer,"\n"); CHKERRQ(ierr);

  ierr = PetscViewerASCIIPrintf(viewer,"hostData:\n"); CHKERRQ(ierr);
  for (i=0;i<nrows;i++) {
    ierr = PetscViewerASCIIPrintf(viewer,"row %2.2D:",i); CHKERRQ(ierr);
    for (j=0;j<ndiag*dof;j++) {
      ierr = PetscViewerASCIIPrintf(viewer," %4G ",a->hostData[i+j*nrows]);CHKERRQ(ierr);
    }
    ierr = PetscViewerASCIIPrintf(viewer,"\n");
  }
  ierr = PetscViewerASCIIPrintf(viewer,"\n\n");



  for (iblock=0;iblock<nrows/dof;iblock++)  {
    for (i=iblock*dof;i<(iblock+1)*dof;i++) {
      ierr = PetscViewerASCIIPrintf(viewer,"row %D:",i);CHKERRQ(ierr);
      for (std::map<int, int>::iterator I = a->diag_starts->begin(),
             E = a->diag_starts->end(); I != E; ++I) {
        /* Ignore 0 padding */
        offset = I->first;
        if (offset + iblock < 0) {
          continue;
        }
        if (offset + iblock >= (nrows/dof)) {
          break;
        }
        
        for (j=0;j<dof;j++) {
          index = i + I->second + j*nrows; // column-major
          col = offset*dof+(iblock*dof) + j;
#if defined(PETSC_USE_COMPLEX)
          if (PetscImaginaryPart(a->hostData[index]) > 0.0) {
            ierr = PetscViewerASCIIPrintf(viewer," (%D, %G + %G i)",col,PetscRealPart(a->hostData[index]),PetscImaginaryPart(a->hostData[index]));CHKERRQ(ierr);
          } else if (PetscImaginaryPart(a->hostData[index]) < 0.0) {
            ierr = PetscViewerASCIIPrintf(viewer," (%D, %G - %G i)",col,PetscRealPart(a->hostData[index]),-PetscImaginaryPart(a->hostData[index]));CHKERRQ(ierr);
          } else {
            ierr = PetscViewerASCIIPrintf(viewer," (%D, %G) ",col,PetscRealPart(a->hostData[index]));CHKERRQ(ierr);
          }
#else
          ierr = PetscViewerASCIIPrintf(viewer," (%D, %G) ",col,a->hostData[index]);CHKERRQ(ierr);
#endif
        }
      }
      ierr = PetscViewerASCIIPrintf(viewer,"\n");CHKERRQ(ierr);
    }
  }

  ierr = PetscViewerASCIIUseTabs(viewer,PETSC_TRUE);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatView_SeqSGGPU"
PetscErrorCode MatView_SeqSGGPU(Mat A, PetscViewer viewer)
{
  PetscErrorCode ierr;
  PetscBool isascii;
  PetscFunctionBegin;

  SGTrace;
  ierr = PetscObjectTypeCompare((PetscObject)viewer,PETSCVIEWERASCII,&isascii);CHKERRQ(ierr);
  if (isascii) {
    ierr = MatView_SeqSGGPU_ASCII(A,viewer);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "MatAssemblyBegin_SeqSGGPU"
PetscErrorCode MatAssemblyBegin_SeqSGGPU(Mat A, MatAssemblyType type)
{
  PetscFunctionBegin;
  SGTrace;

	PetscPrintf(PETSC_COMM_WORLD,"MatAssemblyBegin_SeqSGGPU\n");

  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatAssemblyEnd_SeqSGGPU"
PetscErrorCode MatAssemblyEnd_SeqSGGPU(Mat A, MatAssemblyType type)
{
  Mat_SeqSGGPU * mat = (Mat_SeqSGGPU*)A->data;
  PetscInt size;
  PetscFunctionBegin;

	PetscPrintf(PETSC_COMM_WORLD,"MatAssemblyEnd_SeqSGGPU\n");


#if _TRACE
  printf("[SeqSGGPU] MatAssemblyEnd_SeqSGGPU\n");

  for (std::map<int, int>::iterator I = mat->diag_starts->begin(),
       E = mat->diag_starts->end(); I != E; ++I) {
    printf("- Diag %d:\n", I->first);
    for (int i = 0; i < mat->dof; ++i) {
      for (int j = 0; j < mat->dof * mat->m * mat->n * mat->p; ++j) {
        int offset = i * mat->dof * mat->m * mat->n * mat->p + j;
        printf(" %lf ", mat->hostData[offset + I->second]);
      }
      printf("\n");
    }
  }
#endif
  size = mat->diag_starts->size()*mat->m*mat->n*mat->p*mat->dof*mat->dof;

  checkCudaError(hipMemcpyAsync(mat->deviceDiags, &(*mat->diagonals)[0], sizeof(int) * mat->diagonals->size(), hipMemcpyHostToDevice, mat->stream));

  checkCudaError(hipMemcpy(mat->deviceData, mat->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));

  hipDeviceSynchronize();
  PetscFunctionReturn(0);

}



#undef __FUNCT__  
#define __FUNCT__ "MatFDColoringApply_SeqSGGPU"
PetscErrorCode  MatFDColoringApply_SeqSGGPU(Mat J,MatFDColoring coloring,Vec x1,MatStructure *flag,void *sctx)
{
  PetscErrorCode (*f)(void*,Vec,Vec,void*) = (PetscErrorCode (*)(void*,Vec,Vec,void *))coloring->f;
  PetscErrorCode ierr;
  PetscInt       k,start,end,l,row,col,srow,**vscaleforrow;
  PetscScalar    dx,*y,*xx,*w3_array;
  PetscScalar    *vscale_array;
  PetscReal      epsilon = coloring->error_rel,umin = coloring->umin,unorm; 
  Vec            w1,w2,w3;
  void           *fctx = coloring->fctx;
  PetscBool      flg = PETSC_FALSE;
  PetscInt       ctype=coloring->ctype,N,col_start=0,col_end=0;
  Vec            x1_tmp;

  PetscFunctionBegin;    

	PetscPrintf(PETSC_COMM_WORLD,"MatFDColoringApply_SeqSGGPU\n");


  PetscValidHeaderSpecific(J,MAT_CLASSID,1);
  PetscValidHeaderSpecific(coloring,MAT_FDCOLORING_CLASSID,2);
  PetscValidHeaderSpecific(x1,VEC_CLASSID,3);
  if (!f) SETERRQ(((PetscObject)J)->comm,PETSC_ERR_ARG_WRONGSTATE,"Must call MatFDColoringSetFunction()");

  ierr = PetscLogEventBegin(MAT_FDColoringApply,coloring,J,x1,0);CHKERRQ(ierr);

  ierr = PetscOptionsGetBool(PETSC_NULL,"-mat_fd_coloring_dont_rezero",&flg,PETSC_NULL);CHKERRQ(ierr);
  if (flg) {
    ierr = PetscInfo(coloring,"Not calling MatZeroEntries()\n");CHKERRQ(ierr);
  } else {
    PetscBool  assembled;
    ierr = MatAssembled(J,&assembled);CHKERRQ(ierr);
    if (assembled) {
      ierr = MatZeroEntries(J);CHKERRQ(ierr);
    }
  }

  x1_tmp = x1; 
  if (!coloring->vscale){ 
    ierr = VecDuplicate(x1_tmp,&coloring->vscale);CHKERRQ(ierr);
  }

  if (coloring->htype[0] == 'w') { /* tacky test; need to make systematic if we add other approaches to computing h*/
    ierr = VecNorm(x1_tmp,NORM_2,&unorm);CHKERRQ(ierr);
  }

  if (!coloring->w3) {
    /*
    ierr = VecDestroy(&coloring->w1); CHKERRQ(ierr);
    ierr = VecDestroy(&coloring->w2); CHKERRQ(ierr);
    ierr = VecDuplicate(x1_tmp,&coloring->w1);CHKERRQ(ierr);
     ierr = VecDuplicate(x1_tmp,&coloring->w2);CHKERRQ(ierr);
    ierr = PetscLogObjectParent(coloring,coloring->w1);CHKERRQ(ierr);
     ierr = PetscLogObjectParent(coloring,coloring->w2);CHKERRQ(ierr);*/
    ierr = VecDuplicate(x1_tmp,&coloring->w3);CHKERRQ(ierr);
    ierr = PetscLogObjectParent(coloring,coloring->w3);CHKERRQ(ierr);
  }
  w1 = coloring->w1;
  w2 = coloring->w2;
  w3 = coloring->w3;
  ierr = VecGetOwnershipRange(w1,&start,&end);CHKERRQ(ierr); /* OwnershipRange is used by ghosted x! */

  /* Set w1 = F(x1) */
  if (!coloring->fset) {
    ierr = PetscLogEventBegin(MAT_FDColoringFunction,0,0,0,0);CHKERRQ(ierr);
    ierr = (*f)(sctx,x1_tmp,w1,fctx);CHKERRQ(ierr);
    ierr = PetscLogEventEnd(MAT_FDColoringFunction,0,0,0,0);CHKERRQ(ierr);
  } else {
    coloring->fset = PETSC_FALSE;
  }


    /* Compute all the local scale factors, including ghost points */
  ierr = VecGetLocalSize(x1_tmp,&N);CHKERRQ(ierr);
  ierr = VecGetArray(x1_tmp,&xx);CHKERRQ(ierr);
  ierr = VecGetArray(coloring->vscale,&vscale_array);CHKERRQ(ierr);
  if (ctype == IS_COLORING_GHOSTED){
    col_start = 0; col_end = N;
  } else if (ctype == IS_COLORING_GLOBAL){
    xx = xx - start;
    vscale_array = vscale_array - start;
    col_start = start; col_end = N + start;
  }
  for (col=col_start; col<col_end; col++){ 
    /* Loop over each local column, vscale[col] = 1./(epsilon*dx[col]) */      
    if (coloring->htype[0] == 'w') {
      dx = 1.0 + unorm;
    } else {
      dx  = xx[col];
    }
    if (dx == (PetscScalar)0.0) dx = 1.0;
#if !defined(PETSC_USE_COMPLEX)
    if (dx < umin && dx >= 0.0)      dx = umin;
    else if (dx < 0.0 && dx > -umin) dx = -umin;
#else
    if (PetscAbsScalar(dx) < umin && PetscRealPart(dx) >= 0.0)     dx = umin;
    else if (PetscRealPart(dx) < 0.0 && PetscAbsScalar(dx) < umin) dx = -umin;
#endif
    dx               *= epsilon;
    vscale_array[col] = (PetscScalar)1.0/dx;
  } 
  if (ctype == IS_COLORING_GLOBAL)  vscale_array = vscale_array + start;      
  ierr = VecRestoreArray(coloring->vscale,&vscale_array);CHKERRQ(ierr);
  if (ctype == IS_COLORING_GLOBAL){
    ierr = VecGhostUpdateBegin(coloring->vscale,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
    ierr = VecGhostUpdateEnd(coloring->vscale,INSERT_VALUES,SCATTER_FORWARD);CHKERRQ(ierr);
  }
    
  if (coloring->vscaleforrow) {
    vscaleforrow = coloring->vscaleforrow;
  } else SETERRQ(((PetscObject)J)->comm,PETSC_ERR_ARG_NULL,"Null Object: coloring->vscaleforrow");

  /*
    Loop over each color
  */
  ierr = VecGetArray(coloring->vscale,&vscale_array);CHKERRQ(ierr);
  for (k=0; k<coloring->ncolors; k++) { 
    coloring->currentcolor = k;
    ierr = VecCopy(x1_tmp,w3);CHKERRQ(ierr);
    ierr = VecGetArray(w3,&w3_array);CHKERRQ(ierr);
    if (ctype == IS_COLORING_GLOBAL) w3_array = w3_array - start;
    /*
      Loop over each column associated with color 
      adding the perturbation to the vector w3.
    */
    for (l=0; l<coloring->ncolumns[k]; l++) {
      col = coloring->columns[k][l];    /* local column of the matrix we are probing for */
      if (coloring->htype[0] == 'w') {
        dx = 1.0 + unorm;
      } else {
        dx  = xx[col];
      }
      if (dx == (PetscScalar)0.0) dx = 1.0;
#if !defined(PETSC_USE_COMPLEX)
      if (dx < umin && dx >= 0.0)      dx = umin;
      else if (dx < 0.0 && dx > -umin) dx = -umin;
#else
      if (PetscAbsScalar(dx) < umin && PetscRealPart(dx) >= 0.0)     dx = umin;
      else if (PetscRealPart(dx) < 0.0 && PetscAbsScalar(dx) < umin) dx = -umin;
#endif
      dx            *= epsilon;
      if (!PetscAbsScalar(dx)) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Computed 0 differencing parameter");
      w3_array[col] += dx;

    } 
    if (ctype == IS_COLORING_GLOBAL) w3_array = w3_array + start;
    ierr = VecRestoreArray(w3,&w3_array);CHKERRQ(ierr);

    /*
      Evaluate function at w3 = x1 + dx (here dx is a vector of perturbations)
                           w2 = F(x1 + dx) - F(x1)
    */
    ierr = PetscLogEventBegin(MAT_FDColoringFunction,0,0,0,0);CHKERRQ(ierr);
    ierr = (*f)(sctx,w3,w2,fctx);CHKERRQ(ierr);        
    ierr = PetscLogEventEnd(MAT_FDColoringFunction,0,0,0,0);CHKERRQ(ierr);
    ierr = VecAXPY(w2,-1.0,w1);CHKERRQ(ierr); 
        
    /*
      Loop over rows of vector, putting results into Jacobian matrix
    */
    ierr = VecGetArray(w2,&y);CHKERRQ(ierr);
    for (l=0; l<coloring->nrows[k]; l++) {
      row    = coloring->rows[k][l];             /* local row index */
      col    = coloring->columnsforrow[k][l];    /* global column index */
      y[row] *= vscale_array[vscaleforrow[k][l]];
      srow   = row + start;
      ierr   = MatSetValues(J,1,&srow,1,&col,y+row,INSERT_VALUES);CHKERRQ(ierr);
    }
    ierr = VecRestoreArray(w2,&y);CHKERRQ(ierr);
  } /* endof for each color */
  if (ctype == IS_COLORING_GLOBAL) xx = xx + start; 
  ierr = VecRestoreArray(coloring->vscale,&vscale_array);CHKERRQ(ierr);
  ierr = VecRestoreArray(x1_tmp,&xx);CHKERRQ(ierr);
   
  coloring->currentcolor = -1;
  ierr  = MatAssemblyBegin(J,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr  = MatAssemblyEnd(J,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = PetscLogEventEnd(MAT_FDColoringApply,coloring,J,x1,0);CHKERRQ(ierr);

  flg  = PETSC_FALSE;
  ierr = PetscOptionsGetBool(PETSC_NULL,"-mat_null_space_test",&flg,PETSC_NULL);CHKERRQ(ierr);
  if (flg) {
    ierr = MatNullSpaceTest(J->nullsp,J,PETSC_NULL);CHKERRQ(ierr);
  }
  ierr = MatFDColoringView_Private(coloring);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "MatGetColumnIJ_SeqSGGPU"
PetscErrorCode MatGetColumnIJ_SeqSGGPU(Mat A,PetscInt oshift,PetscBool  symmetric,PetscBool  inodecompressed,PetscInt *nn, const PetscInt *ia[], const PetscInt *ja[],PetscBool  *done)
{
  Mat_SeqSGGPU     *a = (Mat_SeqSGGPU*)A->data;
  PetscErrorCode ierr;
  PetscInt       n = A->cmap->n;
  PetscInt       ndiag = a->diagonals->size();
  PetscInt       nrows = a->m*a->n*a->p*a->dof;
  PetscInt       nz=a->dof*ndiag*nrows;
  PetscInt       iblock,i,j,col,index,colblock,offset;

  PetscFunctionBegin;  

	PetscPrintf(PETSC_COMM_WORLD,"MatGetColumnIJ_SeqSGGPU\n");


  *nn = nrows;

  if (!ia) PetscFunctionReturn(0);
  if (a->ja) {
    ierr = PetscFree(a->ja); CHKERRQ(ierr);
  }
  if (a->ia) {
    ierr = PetscFree(a->ia); CHKERRQ(ierr);
  }
  ierr = PetscMalloc((n+1)*sizeof(PetscInt),&a->ia);CHKERRQ(ierr);
  ierr = PetscMalloc((nz+1)*sizeof(PetscInt),&a->ja);CHKERRQ(ierr);

  /* Assuming symmetric nonzero structure */
  index=0;
  for (iblock=0;iblock<nrows/a->dof;iblock++) {
    for (i=iblock*a->dof;i<(iblock+1)*a->dof;i++) {
      a->ia[i] = index;
      for (std::map<int, int>::iterator I = a->diag_starts->begin(),
             E = a->diag_starts->end(); I != E; ++I) {
        offset = I->first;
        colblock = offset + iblock;
        /* Ignore 0 padding */
        if (colblock < 0) {
          continue;
        }
        if (colblock >= (nrows/a->dof)) {
          break;
        }
        /* skip some blocks for nonperiodic da */
        if (a->stencil_type==0 &&  a->dim==2 && 
            ((colblock - iblock == 1 && !(colblock % a->n)) ||
             (iblock - colblock == 1 && !(iblock % a->n)))) {
          continue;

        }
      
        for (j=0;j<a->dof;j++) {
          col = (colblock*a->dof)  + j;
          a->ja[index++] = col;
	}

      }
    }
  }
  a->ia[nrows] = index;
  *ia = a->ia;
  *ja = a->ja;

  PetscFunctionReturn(0); 
}

#undef __FUNCT__
#define __FUNCT__ "MatFDColoringCreate_SeqSGGPU"
PetscErrorCode MatFDColoringCreate_SeqSGGPU(Mat mat,ISColoring iscoloring,MatFDColoring c)
{
  PetscErrorCode ierr;
  PetscInt       i,n,nrows,N,j,k,m,ncols,col;
  const PetscInt *is,*ci,*cj,*rows;
  PetscInt       nis = iscoloring->n,*rowhit,*columnsforrow,l,bs = 1;
  IS             *isa;
  PetscBool      done,flg = PETSC_FALSE;

  PetscFunctionBegin;


	PetscPrintf(PETSC_COMM_WORLD,"MatFDColoringCreate_SeqSGGPU\n");


  ierr = ISColoringGetIS(iscoloring,PETSC_IGNORE,&isa);CHKERRQ(ierr);
  /* this is ugly way to get blocksize but cannot call MatGetBlockSize() because AIJ can have bs > 1 */

  N          = mat->cmap->N/bs;
  c->M       = mat->rmap->N/bs;  /* set total rows, columns and local rows */
  c->N       = mat->cmap->N/bs;
  c->m       = mat->rmap->N/bs;
  c->rstart  = 0;

  c->ncolors = nis;
  ierr       = PetscMalloc(nis*sizeof(PetscInt),&c->ncolumns);CHKERRQ(ierr);
  ierr       = PetscMalloc(nis*sizeof(PetscInt*),&c->columns);CHKERRQ(ierr); 
  ierr       = PetscMalloc(nis*sizeof(PetscInt),&c->nrows);CHKERRQ(ierr);
  ierr       = PetscMalloc(nis*sizeof(PetscInt*),&c->rows);CHKERRQ(ierr);
  ierr       = PetscMalloc(nis*sizeof(PetscInt*),&c->columnsforrow);CHKERRQ(ierr);

  ierr = MatGetColumnIJ(mat,0,PETSC_FALSE,PETSC_FALSE,&ncols,&ci,&cj,&done);CHKERRQ(ierr);
  if (!done) SETERRQ1(((PetscObject)mat)->comm,PETSC_ERR_SUP,"MatGetColumnIJ() not supported for matrix type %s",((PetscObject)mat)->type_name);

  /*
     Temporary option to allow for debugging/testing
  */
  ierr = PetscOptionsGetBool(PETSC_NULL,"-matfdcoloring_slow",&flg,PETSC_NULL);CHKERRQ(ierr);

  ierr = PetscMalloc((N+1)*sizeof(PetscInt),&rowhit);CHKERRQ(ierr);
  ierr = PetscMalloc((N+1)*sizeof(PetscInt),&columnsforrow);CHKERRQ(ierr);

  for (i=0; i<nis; i++) {
    ierr = ISGetLocalSize(isa[i],&n);CHKERRQ(ierr);
    ierr = ISGetIndices(isa[i],&is);CHKERRQ(ierr);
    c->ncolumns[i] = n;
    if (n) {
      ierr = PetscMalloc(n*sizeof(PetscInt),&c->columns[i]);CHKERRQ(ierr);
      ierr = PetscMemcpy(c->columns[i],is,n*sizeof(PetscInt));CHKERRQ(ierr);
    } else {
      c->columns[i]  = 0;
    }

    if (!flg) { /* ------------------------------------------------------------------------------*/
      /* fast, crude version requires O(N*N) work */
      ierr = PetscMemzero(rowhit,N*sizeof(PetscInt));CHKERRQ(ierr);
      /* loop over columns*/
      for (j=0; j<n; j++) {
        col  = is[j];
        rows = cj + ci[col]; 
        m    = ci[col+1] - ci[col];
        /* loop over columns marking them in rowhit */
        for (k=0; k<m; k++) {
          rowhit[*rows++] = col + 1;
        }
      }
      /* count the number of hits */
      nrows = 0;
      for (j=0; j<N; j++) {
        if (rowhit[j]) nrows++;
      }
      c->nrows[i] = nrows;
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->rows[i]);CHKERRQ(ierr);
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->columnsforrow[i]);CHKERRQ(ierr);
      nrows       = 0;
      for (j=0; j<N; j++) {
        if (rowhit[j]) {
          c->rows[i][nrows]          = j;
          c->columnsforrow[i][nrows] = rowhit[j] - 1;
          nrows++;
        }
      }
    } else {  /*-------------------------------------------------------------------------------*/
      /* slow version, using rowhit as a linked list */
      PetscInt currentcol,fm,mfm;
      rowhit[N] = N;
      nrows     = 0;
      /* loop over columns */
      for (j=0; j<n; j++) {
        col   = is[j];
        rows  = cj + ci[col]; 
        m     = ci[col+1] - ci[col];
        /* loop over columns marking them in rowhit */
        fm    = N; /* fm points to first entry in linked list */
        for (k=0; k<m; k++) {
          currentcol = *rows++;
	  /* is it already in the list? */
          do {
            mfm  = fm;
            fm   = rowhit[fm];
          } while (fm < currentcol);
          /* not in list so add it */
          if (fm != currentcol) {
            nrows++;
            columnsforrow[currentcol] = col;
            /* next three lines insert new entry into linked list */
            rowhit[mfm]               = currentcol;
            rowhit[currentcol]        = fm;
            fm                        = currentcol; 
            /* fm points to present position in list since we know the columns are sorted */
          } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_PLIB,"Detected invalid coloring");
        }
      }
      c->nrows[i] = nrows;
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->rows[i]);CHKERRQ(ierr);
      ierr        = PetscMalloc((nrows+1)*sizeof(PetscInt),&c->columnsforrow[i]);CHKERRQ(ierr);
      /* now store the linked list of rows into c->rows[i] */
      nrows       = 0;
      fm          = rowhit[N];
      do {
        c->rows[i][nrows]            = fm;
        c->columnsforrow[i][nrows++] = columnsforrow[fm];
        fm                           = rowhit[fm];
      } while (fm < N);
    } /* ---------------------------------------------------------------------------------------*/
    ierr = ISRestoreIndices(isa[i],&is);CHKERRQ(ierr);  
  }
  ierr = MatRestoreColumnIJ(mat,0,PETSC_FALSE,PETSC_FALSE,&ncols,&ci,&cj,&done);CHKERRQ(ierr);

  ierr = PetscFree(rowhit);CHKERRQ(ierr);
  ierr = PetscFree(columnsforrow);CHKERRQ(ierr);

  /* Optimize by adding the vscale, and scaleforrow[][] fields */
  /*
       see the version for MPIAIJ
  */
  ierr = VecCreateGhost(((PetscObject)mat)->comm,mat->rmap->n,PETSC_DETERMINE,0,PETSC_NULL,&c->vscale);CHKERRQ(ierr);
  ierr = PetscMalloc(c->ncolors*sizeof(PetscInt*),&c->vscaleforrow);CHKERRQ(ierr);
  for (k=0; k<c->ncolors; k++) { 
    ierr = PetscMalloc((c->nrows[k]+1)*sizeof(PetscInt),&c->vscaleforrow[k]);CHKERRQ(ierr);
    for (l=0; l<c->nrows[k]; l++) {
      col = c->columnsforrow[k][l];
      c->vscaleforrow[k][l] = col;
    }
  }
  ierr = ISColoringRestoreIS(iscoloring,&isa);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}



