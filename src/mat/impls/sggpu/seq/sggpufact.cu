#include "hip/hip_runtime.h"
/// SGGPU Matrix Type

#define PETSCMAT_DLL

#include "petsc-private/matimpl.h"
#include "../src/mat/impls/sggpu/seq/sggpu.h"
#include "../src/mat/impls/aij/seq/aij.h"  

// Direct access to seqgpu vector type
//#include "../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h"

// Interop with CUSP vector
#include "../src/vec/vec/impls/seq/seqcusp/cuspvecimpl.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

// C++ library headers
#include <map>

// Hard-coded block size
#define BLOCKWIDTH_X 128
#define BLOCKWIDTH_Y 1

// Which ILU algorithm to use
#define _ILU_GPU 0

// Debugging flags
#define _TRACE 0
#define _TIME 0
#define _CHECK_SOLVE 0
#define _CHECK_DIAG_INVERT 0

// Hard-coded sizes for cuda shared memory arrays
#define MAXBLOCKSIZE 256
#define BETAMAX 512

// max dof
#define MAXDOF 32

// ----------------------------------------------------------
// helper functions from sggpu.cu
// ----------------------------------------------------------
EXTERN_C_BEGIN
void checkCudaError(hipError_t err);
EXTERN_C_END
double getclock();

PetscBool dumpMat;
PetscBool dumpVec;

//===-- CUDA Device Code -------------------------------------------------===//

texture<int2, 1> vector_rhs;

static __inline__ __device__ double fetch_double(texture<int2, 1> tex, int i)
{
  int2 v = tex1Dfetch(tex, i);
  return __hiloint2double(v.y, v.x);
}

__global__ void MatInvertDiagBlocks_Kernel(PetscScalar * coeff, PetscInt mat_size, PetscInt centerDiagIndex, PetscInt dof) {
  // idx is the index of a dof*dof block on the center diagonal of the matrix
  // the input parameter mat_size is the number of such blocks on the diagonal
  int idx = blockDim.x * blockIdx.x * 1 + threadIdx.x * 1;
  if (idx >= mat_size)
    return;

  // idx0 is the index within the subarray of elements that make up the center
  // diagonal of the [0,0] element of the dof*dof block this thread is inverting
  int idx0 = centerDiagIndex + idx * dof;
  //can we use __shared__ memory for this?
  PetscScalar block_inv[MAXBLOCKSIZE];

  int stripeSize = mat_size * dof;
  int i,j,k;

  // invert lower triangular matrix in block
  // diagonal elements are all 1.0 but are not stored
  block_inv[0] = 1.0;
  for ( i = 1; i < dof; i++ ) {
    block_inv[dof*i+i] = 1.0;
    for ( j = 0; j < i; j++ ) {
      // dot product of i'th row of block with j'th col of inv is 0
      PetscScalar dot = -coeff[idx0 + j*stripeSize + i];
      for ( k = j+1; k < i; k++ )
 	dot -= coeff[idx0 + k*stripeSize + i] * block_inv[j*dof + k];
      block_inv[j*dof + i] = dot;
    }
  }
  
  for ( j = 0; j < dof-1; j++ )
    for ( i = j+1; i < dof; i++ )
      coeff[idx0 + j*stripeSize + i] = block_inv[j*dof + i];

  // invert upper triangular matrix in block
  block_inv[dof*dof-1] = 1.0 / coeff[idx0 + (dof-1)*stripeSize + (dof-1)];
  for ( i = dof-2; i >= 0; i-- ) {
    block_inv[dof*i+i] = 1.0 / coeff[idx0 + i*stripeSize + i];
    for ( j = i+1; j < dof; j++ ) {
      // dot product of i'th row of block with j'th col of inv is 0
      PetscScalar dot = 0;
      for ( k = i+1; k <= j; k++ )
 	dot -= coeff[idx0 + k*stripeSize + i] * block_inv[j*dof + k];
      block_inv[j*dof + i] = dot / coeff[idx0 + i*stripeSize + i];
    }
  }
  
  for ( j = 0; j < dof; j++ )
    for ( i = 0; i <=j; i++ )
      coeff[idx0 + j*stripeSize + i] = block_inv[j*dof + i];
}

//---
//
// This kernel is invoked with input parameters
//     chunk     -- indicates the current portion of the vector of unknowns being solved for
//     chunkSize -- indicates how many unknowns are solved by one invocation of this kernel
//     Note that the unit of a chunk is a block of size dof.
//
// The unknowns already solved for in previous invocations have indices in the range [ 0 , ... , chunk*chunkSize*dof )
// Unknowns being solved for in this invocation have indices in the range [ chunk*chunkSize*dof , ... , (chunk+1)*chunkSize*dof )
// A given thread is solving for the unknown with index idx = chunk*chunkSize*dof + blockDim.x * blockIdx.x + threadIdx.x
//
// The diagonals that come into play for this thread are determined as follows:
// First note that the vector of unknowns is divided into blocks of size dof.
// Let blockIdx = idx / dof;  -- this is the index within the vector of unknowns of
//                               the dof-size block the working thread's unknown belongs to.
// Suppose a diagonal has offset -d  (this is a lower triangular solve so all diagonals have negative offset)
//
// Then the diagonal contributes to the equation involving the working thread's unknown in two possible ways:
// 1. It contains a block with block indices [blockIdx,col] where 0 <= col < chunk*chunkSize. This means it
//    contributes to the current equation with factors involving unknowns already solved for.
//    Since col = blockIdx - d, to determine if a diagonal falls into this category we need to determine if
//                                  d <= blockIdx < d + chunk*chunkSize
//                                                OR
//                                  blockIdx - chunk*chunkSize < d <= blockIdx
//
// 2. It contains a block with block indices [blockIdx,col] where  chunk*chunkSize <= col < (chunk+1)*chunkSize.
//    This means it contributes to the current equation with factors involving unknows that are being solved
//    for concurrently. To fall into this category we must have
//                                  chunk*chunkSize + d <= blockIdx < (chunk+1)*chunkSize + d
//                                                OR
//                                  d <= blockIdx - chunk*chunkSize
//---
__global__ void MatSolveLowerKernel(PetscScalar * coeff, PetscScalar * y, PetscInt mat_size,
				    int * diagonals, PetscInt center_diag,
				    PetscInt chunk, PetscInt chunkSize, PetscInt dof) {
  int idx = blockDim.x * blockIdx.x * 1 + threadIdx.x * 1;

  if (idx >= mat_size)
    return;

  int locidx0 = idx;
  int idx0 = locidx0 + chunk*chunkSize*dof;
  if (idx0 >= mat_size)
    return;

  int blockIdx = idx0 / dof;
  int offset0;
  int block0; 
  __shared__ double beta[BETAMAX];
  __shared__ double temp[BETAMAX];

  // Initialize the solution to corresponding RHS value
  PetscScalar rhs_val0 = fetch_double(vector_rhs, idx0);
  beta[locidx0] = rhs_val0;
  // do we need to sync threads here?

  int diag_size = mat_size * dof;

  // determine the diagonals that contribute already solved values to the RHS for this thread
  PetscInt min_solved_diag = 0, max_solved_diag = center_diag;
  while ( (min_solved_diag < max_solved_diag) && (diagonals[min_solved_diag] < -blockIdx) )
    ++min_solved_diag;
  while ( (0 <= max_solved_diag) && (diagonals[max_solved_diag] >= chunk*chunkSize - blockIdx) )
    --max_solved_diag;

  // multiply matrix*solved and subtract from RHS
  for (int i = min_solved_diag; i <= max_solved_diag; ++i) {
    int d = diagonals[i];

    offset0 = diag_size * i + idx0;
    block0 = (idx0 / dof + d) * dof;

    for (int j = 0; j < dof; ++j) {
      PetscScalar aval0 = coeff[offset0 + mat_size*j];
      PetscScalar yval0 = y[block0 + j];
      beta[locidx0] -= aval0 * yval0;
    }
  }

  // solve the system in the (chunkSize*dof)X(chunkSize*dof) block on the diagonal
  // there is not much parallelism in this loop
  // only certain threads have work to do
  offset0 = diag_size * center_diag + idx0;
  block0 = locidx0 / dof;
  int min_unsolved_diag = 0;
  while ( (min_unsolved_diag < center_diag) && (diagonals[min_unsolved_diag] < chunk*chunkSize - blockIdx) )
    ++min_unsolved_diag;

  for ( int i = 0; i < chunkSize; i++ ) {

    // solve for the next dof unknowns -- dof*dof block on diagonal is stored inverted
    // threads involved: i*dof,...,(i+1)*dof-1
    // the lower triangular elements on the center diagonal have been inverted
    // and the 1's on the diagonal are not stored.
    if ( block0 == i ) {
      temp[locidx0] = beta[locidx0];
      for ( int j = 0; j < locidx0-(i*dof); j++ ) {
 	PetscScalar aval0 = coeff[offset0 + mat_size*j];
 	PetscScalar bval0 = beta[i*dof + j];
 	temp[locidx0] += aval0 * bval0;
      }
      beta[locidx0] = temp[locidx0];
    }
 
#ifndef _IGNORE
    // update rhs for each diagonal
    // threads involved: (i-d)*dof,...,(i-d+1)*dof-1
    // for every non-zero diagonal d in the (chunkSize*dof)X(chunkSize*dof) block on the diagonal
    for ( int di = min_unsolved_diag; di < center_diag; di++ ) {
      int d = diagonals[di];
      // is this thread in a diagonal block?
      if ( block0 == (i-d) ) {
   	PetscInt offset1 = diag_size * di + idx0;
   	for ( int j = 0; j < dof; j++ ) {
   	  PetscScalar aval1 = coeff[offset1 + mat_size*j];
   	  PetscScalar bval1 = beta[i*dof + j];
   	  beta[locidx0] -= aval1 * bval1;
   	}	
      }
    }
#endif
    
  }
  
  y[idx0] = beta[locidx0];
  //y[idx0] = max_solved_diag;
}

//---
//
// This kernel is invoked with input parameters
//     chunk     -- indicates the current portion of the vector of unknowns being solved for
//     chunkSize -- indicates how many unknowns are solved by one invocation of this kernel
//     Note that the unit of a chunk is a block of size dof.
//
// The unknowns already solved for in previous invocations have indices in the range [ 0 , ... , chunk*chunkSize*dof )
// Unknowns being solved for in this invocation have indices in the range [ chunk*chunkSize*dof , ... , (chunk+1)*chunkSize*dof )
// A given thread is solving for the unknown with index idx = chunk*chunkSize*dof + blockDim.x * blockIdx.x + threadIdx.x
//
// The diagonals that come into play for this thread are determined as follows:
// First note that the vector of unknowns is divided into blocks of size dof.
// Let blockIdx = idx / dof;  -- this is the index within the vector of unknowns of
//                               the dof-size block the working thread's unknown belongs to.
// Suppose a diagonal has offset d  (this is an upper triangular solve so all diagonals have positive offset)
//
// Then the diagonal contributes to the equation involving the working thread's unknown in two possible ways:
// 1. It contains a block with block indices [blockIdx,col] where (chunk+1)*chunkSize <= col < mat_size/dof. This means it
//    contributes to the current equation with factors involving unknowns already solved for.
//    Since col = blockIdx + d, to determine if a diagonal falls into this category we need to determine if
//
//                            (chunk+1)*chunkSize - blockIdx <= d < mat_size/dof - blockIdx
//
// 2. It contains a block with block indices [blockIdx,col] where  chunk*chunkSize <= col < (chunk+1)*chunkSize.
//    This means it contributes to the current equation with factors involving unknows that are being solved
//    for concurrently. To fall into this category we must have
//                                  chunk*chunkSize - d <= blockIdx < (chunk+1)*chunkSize - d
//                                                OR
//                                  d < (chunk+1)*chunkSize - blockIdx
//---
__global__ void MatSolveUpperKernel(PetscScalar * coeff, PetscScalar * y, // PetscScalar * rhs,
				    PetscInt grid_size,
				    int * diagonals, PetscInt center_diag,
				    PetscInt chunk, PetscInt chunkSize, PetscInt dof) {
  int idx = blockDim.x * blockIdx.x * 1 + threadIdx.x * 1;

  PetscInt mat_size = grid_size * dof;

  if (idx >= mat_size)
    return;

  int locidx0 = idx;
  int idx0 = locidx0 + chunk*chunkSize*dof;
  if (idx0 >= mat_size)
    return;

  int blockIdx = idx0 / dof;
  int offset0;
  int block0; 
  __shared__ double beta[BETAMAX];
  __shared__ double temp[BETAMAX];

  // Initialize the solution to corresponding RHS value
  PetscScalar rhs_val0 = y[idx0]; //fetch_double(vector_rhs, idx0);
  beta[locidx0] = rhs_val0;
  // do we need to sync threads here?

  int diag_size = mat_size * dof;

  // determine the diagonals that contribute already solved values to the RHS for this thread
  PetscInt min_solved_diag = center_diag, max_solved_diag = 2*center_diag;
  while ( (min_solved_diag <= max_solved_diag) && (diagonals[min_solved_diag] < (chunk+1)*chunkSize - blockIdx) )
    ++min_solved_diag;
  while ( (center_diag <= max_solved_diag) && (diagonals[max_solved_diag] >= grid_size - blockIdx) )
    --max_solved_diag;

  // multiply matrix*solved and subtract from RHS
  for (int i = min_solved_diag; i <= max_solved_diag; ++i) {
    int d = diagonals[i];

    offset0 = diag_size * i + idx0;
    block0 = (idx0 / dof + d) * dof;

    for (int j = 0; j < dof; ++j) {
      PetscScalar aval0 = coeff[offset0 + mat_size*j];
      PetscScalar yval0 = y[block0 + j];
      beta[locidx0] -= aval0 * yval0;
    }
  }

  // solve the system in the (chunkSize*dof)X(chunkSize*dof) block on the diagonal
  // there is not much parallelism in this loop
  // only certain threads have work to do
  offset0 = diag_size * center_diag + idx0;
  block0 = locidx0 / dof;
  int max_unsolved_diag = 2*center_diag;
  while ( (max_unsolved_diag > center_diag) && (diagonals[max_unsolved_diag] >= (chunk+1)*chunkSize - blockIdx) )
    --max_unsolved_diag;

  for ( int i = chunkSize-1; i >= 0; i-- ) {

    // solve for the next dof unknowns -- dof*dof block on diagonal is stored inverted
    // threads involved: i*dof,...,(i+1)*dof-1
    if ( block0 == i ) {
      temp[locidx0] = 0;
      for ( int j = locidx0-(i*dof); j < dof; j++ ) {
 	PetscScalar aval0 = coeff[offset0 + mat_size*j];
 	PetscScalar bval0 = beta[i*dof + j];
 	temp[locidx0] += aval0 * bval0;
      }
      beta[locidx0] = temp[locidx0];
    }
 
    // update rhs for each diagonal
    // threads involved: (i-d)*dof,...,(i-d+1)*dof-1
    // for every non-zero diagonal d in the (chunkSize*dof)X(chunkSize*dof) block on the diagonal
    for ( int di = max_unsolved_diag; di > center_diag; di-- ) {
      int d = diagonals[di];
      // is this thread in a diagonal block?
      if ( block0 == (i-d) ) {
   	PetscInt offset1 = diag_size * di + idx0;
   	for ( int j = 0; j < dof; j++ ) {
   	  PetscScalar aval1 = coeff[offset1 + mat_size*j];
   	  PetscScalar bval1 = beta[i*dof + j];
   	  beta[locidx0] -= aval1 * bval1;
   	}	
      }
    }
  }
  
  // y[idx0] = rhs_val0;
  y[idx0] = beta[locidx0];
  // y[idx0] = max_unsolved_diag;
}


//===-- Host Code --------------------------------------------------------===//

static PetscErrorCode InvertFactoredDiagBlocks( Mat_SeqSGGPU *a, PetscScalar *data )
{
  PetscErrorCode ierr = 0;
  PetscInt dof = a->dof, numBlocks = a->m * a->n * a->p;
  PetscInt numElements = numBlocks * dof;
  PetscInt i,j,k, offset;
  PetscInt          num_diags = a->diagonals->size();

  PetscScalar block_inv[MAXBLOCKSIZE];

  for ( int row = 0; row < numBlocks; row++ ) {

    offset = (num_diags/2) * numElements * dof + row*dof;

    // invert lower triangular matrix in block
    // diagonal elements are all 1.0 but are not stored
    block_inv[0] = 1.0;
    for ( i = 1; i < dof; i++ ) {
      block_inv[dof*i+i] = 1.0;
      for ( j = 0; j < i; j++ ) {
	// dot product of i'th row of block with j'th col of inv is 0
	PetscScalar dot = -data[offset + j*numElements + i];
	for ( k = j+1; k < i; k++ )
	  dot -= data[offset + k*numElements + i] * block_inv[j*dof + k];
	block_inv[j*dof + i] = dot;
      }
    }
  
    for ( j = 0; j < dof-1; j++ )
      for ( i = j+1; i < dof; i++ )
	data[offset + j*numElements + i] = block_inv[j*dof + i];

    // invert upper triangular matrix in block
    block_inv[dof*dof-1] = 1.0 / data[offset + (dof-1)*numElements + (dof-1)];
    for ( i = dof-2; i >= 0; i-- ) {
      block_inv[dof*i+i] = 1.0 / data[offset + i*numElements + i];
      for ( j = i+1; j < dof; j++ ) {
	// dot product of i'th row of block with j'th col of inv is 0
	PetscScalar dot = 0;
	for ( k = i+1; k <= j; k++ )
	  dot -= data[offset + k*numElements + i] * block_inv[j*dof + k];
	block_inv[j*dof + i] = dot * block_inv[dof*i+i];
      }
    }
  
    for ( j = 0; j < dof; j++ )
      for ( i = 0; i <=j; i++ )
	data[offset + j*numElements + i] = block_inv[j*dof + i];
  }

  PetscFunctionReturn(ierr);
}

static PetscErrorCode MatGetBlockIJ_SeqSGGPU( Mat_SeqSGGPU * mat, PetscInt i, PetscInt j,
					      PetscScalar** blockPtr )
{
  PetscErrorCode ierr = 0;

  *blockPtr = PETSC_NULL;

  PetscInt numBlockRows = mat->m * mat->n * mat->p;
  if ( (i >= 0) && (i < numBlockRows) && (j >= 0) && (j < numBlockRows) ) {
    PetscInt blockIndex = -1;
    std::map<int, int> &diag_starts = *(mat->diag_starts);
    std::map<int, int>::iterator I = diag_starts.find(j-i);
    if (I != diag_starts.end()) {
      blockIndex = I->second + i*mat->dof;
      *blockPtr = &(mat->hostData[blockIndex]);
    }
  }

  PetscFunctionReturn(ierr);
}


//------------------------------------
// updateMainDiagonalBlock
// does LU decomp on a dofxdof block
// on the main diagonal
//
// Example for dof=3, topRow=0:
//
// INPUT:
// a  b  c   quotients = (*, d/a, g/a)
// d  e  f             = (*, q, r)
// g  h  i
//
// OUTPUT:
// a  b     c
// q  e-qb  f-qc
// r  h-rb  i-rc
//-------------------------------------
static PetscErrorCode MatUpdateMainDiagonalBlock_SeqSGGPU( PetscInt numElems, PetscInt dof,
							   PetscScalar *quotients, PetscScalar *block, int topRow )
{
  PetscErrorCode ierr = 0;
  PetscScalar quot;

  for ( int i = topRow+1; i < dof; i++ ) {
    quot = quotients[i];
    block[topRow*numElems + i] = quot;
    for ( int j = topRow+1; j < dof; j++ )
      block[j*numElems + i] -= quot*block[j*numElems+topRow];
  }

  PetscFunctionReturn(ierr);
}

//------------------------------------------
// MatUpdateTopRowBlock
// extends the LU decomp from a block on the
// main diagonal to a block in the same row
//
// Example for dof=3 and topRow=0:
//
// INPUT:
// a  b  c   quotients = (*, q, r)
// d  e  f
// g  h  i
//
// OUTPUT:
// a     b     c
// d-qa  e-qb  f-qc
// g-ra  h-rb  i-rc
//-------------------------------------
static PetscErrorCode MatUpdateTopRowBlock_SeqSGGPU( PetscInt numElems, PetscInt dof,
						     PetscScalar *quotients, PetscScalar *block, int topRow )
{
  PetscErrorCode ierr = 0;
  PetscScalar quot;

  for ( int i = topRow+1; i < dof; i++ ) {
    quot = quotients[i];
    for ( int j = 0; j < dof; j++ )
      block[j*numElems + i] -= quot*block[j*numElems + topRow];
  }

  return ierr;
}

//--------------------------------------------
// MatUpdateLeftColBlock
// extends the LU decomp from a block on the
// main diagonal to a block in the same column
//
// Example for dof=3 and startCol=0:
//
// INPUT:
// topRowVals = (x, y, z)
//
// a  b  c   quotients = (a/x, d/x, g/x)
// d  e  f             = (p, q, r)
// g  h  i
//
// OUTPUT:
// p  b-py  c-pz
// q  e-qy  f-qz
// r  h-ry  i-rz
//-------------------------------------
static PetscErrorCode MatUpdateLeftColBlock_SeqSGGPU( PetscInt numElems, PetscInt dof,
						      PetscScalar *quotients, PetscScalar *topRowVals,
						      PetscScalar *block, int startCol )
{
  PetscErrorCode ierr = 0;

  PetscScalar quot;

  for ( int i = 0; i < dof; i++ ) {
    quot = quotients[i];
    block[startCol*numElems + i] = quot;
    for ( int j = startCol+1; j < dof; j++ )
      block[j*numElems + i] -= quot*topRowVals[j];
  }

  PetscFunctionReturn(ierr);
}

//--------------------------------------------
// MatUpdateGeneralBlock
// extends the LU decomp from a block on the
// main diagonal to a non-zero block below and
// to the right
//
// Example:
//
// INPUT:
// topRowVals    = (x, y, z)
//
// a  b  c   quotients = (p, q, r)
// d  e  f             
// g  h  i
//
// OUTPUT:
// a-px  b-py  c-pz
// b-qx  e-qy  f-qz
// c-rx  h-ry  i-rz
//-------------------------------------
static PetscErrorCode MatUpdateGeneralBlock_SeqSGGPU( PetscInt numElems, PetscInt dof,
						      PetscScalar *quotients, PetscScalar *topRowVals,
						      PetscScalar *block )
{
  PetscErrorCode ierr = 0;

  PetscScalar quot;

  for ( int i = 0; i < dof; i++ ) {
    quot = quotients[i];
    for ( int j = 0; j < dof; j++ )
      block[j*numElems + i] -= quot*topRowVals[j];
  }

  PetscFunctionReturn(ierr);
}

#undef __FUNCT__
#define __FUNCT__ "MatLUFactorNumeric_SeqSGGPU"
PetscErrorCode MatLUFactorNumeric_SeqSGGPU(Mat B,Mat A,const MatFactorInfo *info)
{
  Mat              C=B;
  Mat_SeqSGGPU       *a=(Mat_SeqSGGPU*)A->data,*b=(Mat_SeqSGGPU *)C->data;
  PetscInt dof = a->dof, numBlocks = a->m * a->n * a->p;
  PetscInt numElements = numBlocks * dof;
  PetscInt diagSize = numElements * dof;
  PetscScalar *topRowValsDiag, *topRowValsOffDiag, *quotients;
  PetscErrorCode ierr;
  PetscScalar *mainDiagonalBlock, *topRowBlock, *leftColBlock, *generalBlock;
  PetscInt i,j,k, offset;
  PetscInt          num_diags = b->diagonals->size();
  PetscInt          size = diagSize * num_diags;
  PetscInt mainDiagStartIndex = numElements * dof * (num_diags/2);
  std::vector<int>::iterator J, I;

  PetscFunctionBegin;

#if _TIME
  double t_start, t_end, elapsed;
  t_start = getclock();
#endif

  // Copy the diagonals from A to B
  int diag_offset = 0;
  for ( J = b->diagonals->begin();
	J != b->diagonals->end(); J++ ) {
    // first store the start index of this diagonal in b->diag_starts
    int d = *J;

    std::map<int, int>::iterator I = a->diag_starts->find(d);
    // if this diag is in A, copy the data into B
    if (I != a->diag_starts->end()) {
      ierr = PetscMemcpy( &(b->hostData[diag_offset]), &(a->hostData[I->second]),
			  diagSize*sizeof(PetscScalar)); CHKERRQ(ierr);
    }
    // otherwise, zero it out
    else {
      ierr = PetscMemzero(&(b->hostData[diag_offset]),
			  diagSize*sizeof(PetscScalar));CHKERRQ(ierr);
    }
    
    diag_offset += diagSize;

  }

  if (b->deviceData) {
    hipFree(b->deviceData);
  }
  checkCudaError(hipMalloc(&b->deviceData, sizeof(PetscScalar) * size));
  //checkCudaError(hipMemset(b->deviceData,0.0,sizeof(PetscScalar)*size));

  if (b->deviceDiags) {
    hipFree(b->deviceDiags);
  }
  checkCudaError(hipMalloc(&b->deviceDiags, sizeof(int) * b->diagonals->size()));

  checkCudaError(hipMemcpyAsync(b->deviceDiags, &(*b->diagonals)[0], sizeof(int) * b->diagonals->size(), hipMemcpyHostToDevice, b->stream));
  checkCudaError(hipMemcpy(b->deviceData, b->hostData, sizeof(PetscScalar) * size, hipMemcpyHostToDevice));
  hipDeviceSynchronize();


  ierr = PetscMalloc(dof * sizeof(PetscScalar), &topRowValsDiag); CHKERRQ(ierr);
  ierr = PetscMalloc(dof * sizeof(PetscScalar), &topRowValsOffDiag); CHKERRQ(ierr);
  ierr = PetscMalloc(dof * sizeof(PetscScalar), &quotients); CHKERRQ(ierr);

#if _TIME
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("factor numeric preamble time %lf\n",elapsed);
  t_start = getclock();
#endif

  // iterate over blocks in main diagonal
  for ( int row = 0; row < numBlocks; row++ ) {
    
    //ierr = MatGetBlockIJ_SeqSGGPU( b, row, row, &mainDiagonalBlock );
    mainDiagonalBlock = &(b->hostData[mainDiagStartIndex + row*dof]);

    // we need to factor block(row,row) and extend this to
    //    -- any nonzero block(row,row+d) on a super-diagonal
    //    -- any nonzero block(row+d,row) on a subdiagonal
    //    -- any nonzero block(row+d1,row+d2)
    //
    // the outer loop here is over the rows of all blocks
    // that need to be updated
    for ( int k = 0; k < dof; k++ ) {

      // copy the values in the current row of the main diagonal block(row,row)
      // these will be used to update any nonzero block(row+d,row)
      for ( int j = k; j < dof; j++ ) {
	topRowValsDiag[j] = mainDiagonalBlock[j*numElements + k];
      }

      quotients[k] = 1.0;
      for ( int j = k+1; j < dof; j++ )
	quotients[j] = mainDiagonalBlock[k*numElements +j] / mainDiagonalBlock[k*numElements + k];

      // update block(row,row)[k+1..dof-1][k+1..dof-1] using the quotients from column k
      ierr = MatUpdateMainDiagonalBlock_SeqSGGPU( numElements, dof, quotients, mainDiagonalBlock, k );

      // update the blocks to the right using the same quotients from the k'th column of block(row,row)
      for (int i = 0; i < num_diags; ++i) {
	int d = (*b->diagonals)[i];
	if ( (d > 0) && (row+d < numBlocks) ) {
	  //ierr = MatGetBlockIJ_SeqSGGPU( b, row, row+d, &topRowBlock );
	  topRowBlock = &(b->hostData[i*dof*numElements + row*dof]);

	  if ( topRowBlock )
	    ierr = MatUpdateTopRowBlock_SeqSGGPU( numElements, dof, quotients, topRowBlock, k );
	}
      }

      for (int i = 0; i < num_diags; ++i) {
	int dSub = (*b->diagonals)[i];
	if ( (dSub < 0) && (row-dSub < numBlocks) ) {
	  //ierr = MatGetBlockIJ_SeqSGGPU( b, row-dSub, row, &leftColBlock );
	  leftColBlock = &(b->hostData[i*dof*numElements + (row-dSub)*dof]);
	  if ( leftColBlock != 0 ) {
	    for ( int j = 0; j < dof; j++ )
	      quotients[j] = leftColBlock[k*numElements +j] / mainDiagonalBlock[k*numElements + k];

	    ierr = MatUpdateLeftColBlock_SeqSGGPU( numElements, dof, quotients, topRowValsDiag, leftColBlock, k );

	    for (int ii = 0; ii < num_diags; ++ii) {
	      int dSuper = (*b->diagonals)[ii];
	      if ( (dSuper > 0) && (row+dSuper < numBlocks) ) {
		//ierr = MatGetBlockIJ_SeqSGGPU( b, row, row+dSuper, &topRowBlock );
		topRowBlock = &(b->hostData[ii*dof*numElements + row*dof]);

		if ( topRowBlock != 0 ) {
		  ierr = MatGetBlockIJ_SeqSGGPU( b, row-dSub, row+dSuper, &generalBlock );
		  if ( generalBlock != 0 ) {
		    for ( int j = 0; j < dof; j++ )
		      topRowValsOffDiag[j] = topRowBlock[j*numElements + k];
		    
		    ierr = MatUpdateGeneralBlock_SeqSGGPU( numElements, dof, quotients, topRowValsOffDiag, generalBlock);
		  }
		}
	      }
	    }
	  }
	}
      }
    }      
  }

#if _TIME
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("factor numeric main loop time %lf\n",elapsed);
  t_start = getclock();
#endif

#if _ILU_GPU>1
  C->ops->solve = MatSolve_SeqSGGPU_compare;
#elif _ILU_GPU>0
  C->ops->solve = MatSolve_SeqSGGPU_gpu;
#else
  C->ops->solve = MatSolve_SeqSGGPU_cpu;
#endif

  //=================================
  //invert the blocks on the diagonal
  //=================================
  PetscScalar block_inv[MAXBLOCKSIZE];

  for ( int row = 0; row < numBlocks; row++ ) {

    offset = (b->diagonals->size()/2) * numElements * dof + row*dof;

    // invert lower triangular matrix in block
    // diagonal elements are all 1.0 but are not stored
    block_inv[0] = 1.0;
    for ( i = 1; i < dof; i++ ) {
      block_inv[dof*i+i] = 1.0;
      for ( j = 0; j < i; j++ ) {
	// dot product of i'th row of block with j'th col of inv is 0
	PetscScalar dot = -b->hostData[offset + j*numElements + i];
	for ( k = j+1; k < i; k++ )
	  dot -= b->hostData[offset + k*numElements + i] * block_inv[j*dof + k];
	block_inv[j*dof + i] = dot;
      }
    }
  
    for ( j = 0; j < dof-1; j++ )
      for ( i = j+1; i < dof; i++ )
	b->hostData[offset + j*numElements + i] = block_inv[j*dof + i];

    // invert upper triangular matrix in block
    block_inv[dof*dof-1] = 1.0 / b->hostData[offset + (dof-1)*numElements + (dof-1)];
    for ( i = dof-2; i >= 0; i-- ) {
      block_inv[dof*i+i] = 1.0 / b->hostData[offset + i*numElements + i];
      for ( j = i+1; j < dof; j++ ) {
	// dot product of i'th row of block with j'th col of inv is 0
	PetscScalar dot = 0;
	for ( k = i+1; k <= j; k++ )
	  dot -= b->hostData[offset + k*numElements + i] * block_inv[j*dof + k];
	block_inv[j*dof + i] = dot * block_inv[dof*i+i];
      }
    }

#if _CHECK_DIAG_INVERT
    printf("Original U block:\n");
    for ( j = 0; j < dof; j++ ) {
      for ( k = 0; k < j; k++ )
	printf("        ");
      for ( k = j; k < dof; k++ )
	printf("%8f", b->hostData[offset + k*numElements + j] );
      printf("\n");
    }
    printf("\n");
    printf("Inverted U block:\n");
    for ( j = 0; j < dof; j++ ) {
      for ( k = 0; k < j; k++ )
	printf("        ");
      for ( k = j; k < dof; k++ )
	printf("%8f", block_inv[k*dof + j] );
      printf("\n");
    }
#endif
  
    for ( j = 0; j < dof; j++ )
      for ( i = 0; i <=j; i++ )
	b->hostData[offset + j*numElements + i] = block_inv[j*dof + i];
  }

  // this will copy hostData to deviceData
  ierr = MatAssemblyEnd(C,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);

  int mat_size = b->m * b->n * b->p;
  PetscInt centerDiag = ( b->diagonals->size() / 2 ) * mat_size * b->dof * b->dof;

  //#if _ILU_GPU
// #if _TRACE
//   printf("Calling kernel to invert diagonal blocks of the L and U matrices\n");
//   PetscScalar *checkData;
//   int stripeSize = b->dof * mat_size;
//   int diagSize = b->dof * stripeSize;
//   int dataSize =  diagSize* b->diagonals->size();
//   ierr = PetscMalloc( dataSize * sizeof(PetscScalar), &checkData ); CHKERRQ(ierr);
//   checkCudaError(hipMemcpy(checkData, b->deviceData, sizeof(PetscScalar) * dataSize, hipMemcpyDeviceToHost));
//   std::map<int, int> &diag_starts = *(b->diag_starts);
//   std::map<int, int>::iterator I = diag_starts.find(0);
//   int mainDiagIndex;
//   if (I != diag_starts.end()) {
//     mainDiagIndex = I->second;
//     printf("m = %d, n = %d, p = %d, dof = %d\n", b->m, b->n, b->p, b->dof);
//     printf("mainDiagIndex = %d\n", mainDiagIndex);
//     printf("centerDiag = %d\n", centerDiag);
//     printf("hostData block:\n");
//     for ( int j = 0; j < dof; j++ ) {
//       for ( int k = 0; k < j; k++ )
// 	printf( "%f ", b->hostData[mainDiagIndex + j + k*stripeSize] );
//       printf("1 \n");
//     }
//     printf("deviceData block:\n");
//     for ( int j = 0; j < dof; j++ ) {
//       for ( int k = 0; k < j; k++ )
// 	printf( "%f ", checkData[mainDiagIndex + j + k*stripeSize] );
//       printf("1 \n");
//     }
//   }
  
// #endif
  // the first parameter is a pointer to the first element of the center diagonal
  // Invoke
//   dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
//   dim3 grid((int)ceil((float)(mat_size)/(float)BLOCKWIDTH_X / 1.0), 1);
//   PetscInt chunkSize = b->dim==3 ? b->p : b->dim==2 ? b->n : 1;
  //  dim3 block(mat_size, BLOCKWIDTH_Y);
  // dim3 grid(1, 1);
//   int shared_size = 0;
//   MatInvertDiagBlocks_Kernel<<<grid, block, shared_size, b->stream>>>( b->deviceData, mat_size, centerDiag, dof);

// #if _TRACE
//   printf("done inverting diagonal blocks of the L and U matrices\n");
//   checkCudaError(hipMemcpy(checkData, b->deviceData, sizeof(PetscScalar) * dataSize, hipMemcpyDeviceToHost));
//   if (I != diag_starts.end()) {
//     mainDiagIndex = I->second;
//     for ( int blockIdx = 0; blockIdx < mat_size; blockIdx++ ) {
//       printf("uninverted L block %d:\n",blockIdx);
//       for ( int j = 0; j < dof; j++ ) {
// 	for ( int k = 0; k < j; k++ )
// 	  printf( "%f ", b->hostData[mainDiagIndex + blockIdx*dof + j + k*stripeSize] );
// 	printf("1 \n");
//       }
//       printf("inverted L block:\n");
//       for ( int j = 0; j < dof; j++ ) {
// 	for ( int k = 0; k < j; k++ )
// 	  printf( "%f ", checkData[mainDiagIndex + blockIdx*dof + j + k*stripeSize] );
// 	printf("1 \n");
//       }

//       printf("uninverted U block:\n");
//       for ( int j = 0; j < dof; j++ ) {
// 	for ( int k = 0; k < j; k++ )
// 	  printf("        ");
// 	for ( int k = j; k < dof; k++ )
// 	  printf( "%8f ", b->hostData[mainDiagIndex + blockIdx*dof + j + k*stripeSize] );
// 	printf("\n");
//       }
//       printf("inverted U block:\n");
//       for ( int j = 0; j < dof; j++ ) {
// 	for ( int k = 0; k < j; k++ )
// 	  printf("        ");
// 	for ( int k = j; k < dof; k++ )
// 	  printf( "%8f ", checkData[mainDiagIndex + blockIdx*dof + j + k*stripeSize] );
// 	printf("\n");
//       }
//     }
//   }
//   PetscFree(checkData);
// #endif
//#endif

#if _TIME
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("factor numeric finalize time %lf\n",elapsed);
#endif

  PetscFunctionReturn(0); 
}

PetscErrorCode MatILUFactor_SeqSGGPU(Mat inA,IS row,IS col,const MatFactorInfo *info)
{
  PetscErrorCode ierr;
  Mat            outA = inA;

  ierr = MatLUFactorNumeric_SeqSGGPU(outA,inA,info);CHKERRQ(ierr);
  
  PetscFunctionReturn(0);
}

extern PetscErrorCode MatDuplicateNoCreate_SeqSGGPU(Mat,Mat,MatDuplicateOption,PetscBool );
extern PetscErrorCode MatGetRow_SeqSGGPU(Mat A, PetscInt row, PetscInt * nz, PetscInt **idx , PetscScalar ** v);
extern PetscErrorCode MatRestoreRow_SeqSGGPU(Mat A, PetscInt row, PetscInt *nz, PetscInt **idx, PetscScalar **v);
extern PetscErrorCode MatSetStencil_SeqSGGPU(Mat A, PetscInt dim, const PetscInt dims[], const PetscInt starts[], PetscInt dof);

#undef __FUNCT__  
#define __FUNCT__ "MatILUFactorSymbolic_SeqSGGPU"
PetscErrorCode MatILUFactorSymbolic_SeqSGGPU(Mat fact,Mat A,IS isrow,IS iscol,const MatFactorInfo *info)
{
  //Mat_SeqSGGPU         *a = (Mat_SeqSGGPU*)A->data; //,*b;
  IS                    isicol;
  PetscErrorCode        ierr;
  PetscInt              iluLevel = (PetscInt)info->levels;
  Mat_SeqSGGPU         *a = (Mat_SeqSGGPU*)A->data, *b;
  std::vector<int>     *iluDiagonals, newDiagonals;
  int                   j, nz, numDiag;
  std::vector<int>::iterator J, I, II;

  PetscFunctionBegin;
  if (A->rmap->n != A->cmap->n) SETERRQ2(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Must be square matrix, rows %D columns %D",A->rmap->n,A->cmap->n);

#if _TIME
  double t_start, t_end, elapsed;
  t_start = getclock();
#endif

  ierr = ISInvertPermutation(iscol,PETSC_DECIDE,&isicol);CHKERRQ(ierr);

  // duplicate the diagonals of the input matrix,
  // then add new diagonals for each level of ilu(k)
  // a new diagonal at offset (d1+d2) is added for
  // every d1>0 and d2<0 in the current set of diagonals.
  // maybe keep some extra information around about the
  // elements of the new diagonals which are known to be 0?
  iluDiagonals = new std::vector<int>( *(a->diagonals) );
  numDiag = a->diagonals->size();

  for ( j = 0; j < iluLevel; j++ ) {
    
    for ( I = iluDiagonals->begin();
	  I != iluDiagonals->end(); I++ ) {
      int d1 = *I;
      if ( d1 < 0 ) {
	for ( J = iluDiagonals->begin();
	      J != iluDiagonals->end(); J++ ) {
	  int d2 = *J;
	  if ( d2 > 0 ) {
	    II = find( iluDiagonals->begin(), iluDiagonals->end(), d1+d2 );
	    if ( II == iluDiagonals->end() ) {
	      II = find( newDiagonals.begin(), newDiagonals.end(), d1+d2 );
	      if ( II == newDiagonals.end() ) {
		newDiagonals.push_back(d1+d2);
	      }
	    }
	  }
	}
      }
    }

    numDiag += newDiagonals.size();
    iluDiagonals->insert( iluDiagonals->end(), newDiagonals.begin(), newDiagonals.end() );
    newDiagonals.clear();

#if _TRACE
    printf("Level %d diagonals:  ", j+1);
    for ( J = iluDiagonals->begin();
	  J != iluDiagonals->end(); J++ )
      printf(" %d ", *J);
    printf("\n");
#endif

  }

  sort( iluDiagonals->begin(), iluDiagonals->end() );
#if _TRACE
  printf("Sorted level %d diagonals:  ", iluLevel);
  for ( J = iluDiagonals->begin();
	J != iluDiagonals->end(); J++ )
    printf(" %d ", *J);
  printf("\n");
#endif

  // Duplicate the input matrix A so that the hostData of fact will be the same
  nz = a->m * a->n * a->p * a->dof;
  ierr = MatSetSizes(fact,nz,nz,nz,nz);CHKERRQ(ierr);
  MatSetType(fact,MATSEQSGGPU);
  b = (Mat_SeqSGGPU*)fact->data;
  int diagSize = nz * a->dof;
  int num_diags = iluDiagonals->size();
  b->stpoints = num_diags;
  b->dim = a->dim;
  b->dof = a->dof;
  b->m = a->m;
  b->n = a->n;
  b->p = a->p;
  int size = diagSize * num_diags;
  int diag_offset = 0;
  ierr = PetscMalloc( size * sizeof(PetscScalar), &b->hostData ); CHKERRQ(ierr);

  b->diagonals = iluDiagonals;

  fact->factortype             = MAT_FACTOR_ILU;
  fact->info.factor_mallocs    = 0;
  fact->info.fill_ratio_given  = info->fill;
  fact->info.fill_ratio_needed = 1.0;
  fact->ops->lufactornumeric   = MatLUFactorNumeric_SeqSGGPU;

  for ( J = b->diagonals->begin();
	J != b->diagonals->end(); J++ ) {
    // store the start index of this diagonal in b->diag_starts
    int d = *J;
    (*b->diag_starts)[d] = diag_offset;
    diag_offset += diagSize;
  }

  // ierr    = PetscMalloc((fact->rmap->n+1)*sizeof(PetscScalar),&b->solve_work);CHKERRQ(ierr);
  ierr    = PetscObjectReference((PetscObject)isrow);CHKERRQ(ierr);
  ierr    = PetscObjectReference((PetscObject)iscol);CHKERRQ(ierr);

#if _TIME
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("factor symbolic time %lf\n",elapsed);
#endif

  PetscFunctionReturn(0);
}

//---
//
// 
//
//---
static PetscErrorCode MatGetLowerSolveBlocks_SeqSGGPU( Mat_SeqSGGPU * mat, PetscScalar *xx, PetscInt blockRow,
						       PetscInt *numSubBlocks,
						       PetscScalar** subBlockPtrs, PetscScalar** knownPtrs,
						       PetscScalar** diagBlockPtr )
{
  PetscErrorCode ierr = 0;
  int num_diags = mat->diagonals->size();
  int dSub, blockCol;

  *numSubBlocks = 0;

  for (int i = 0; i < num_diags; ++i) {
    dSub = (*mat->diagonals)[i];
    blockCol = dSub + blockRow;
    if ( (dSub <= 0) && (blockCol >= 0) ) {
      PetscInt blockIndex = -1;
      std::map<int, int> &diag_starts = *(mat->diag_starts);
      std::map<int, int>::iterator I = diag_starts.find(dSub);
      if (I != diag_starts.end()) {
	blockIndex = I->second + blockRow*mat->dof;
	if ( dSub < 0 ) {
	  subBlockPtrs[*numSubBlocks] = &(mat->hostData[blockIndex]);
	  knownPtrs[(*numSubBlocks)++] = &(xx[blockCol*mat->dof]);
	}
	else
	  *diagBlockPtr =  &(mat->hostData[blockIndex]);
      }
    }
  }

  PetscFunctionReturn(ierr);
}


//---
//
// 
//
//---
static PetscErrorCode MatGetUpperSolveBlocks_SeqSGGPU( Mat_SeqSGGPU * mat, PetscScalar *xx, PetscInt blockRow,
						       PetscInt *numSuperBlocks,
						       PetscScalar** superBlockPtrs, PetscScalar** knownPtrs,
						       PetscScalar** diagBlockPtr )
{
  PetscErrorCode ierr = 0;
  int num_diags = mat->diagonals->size();
  int dSuper, blockCol, maxBlockCol = mat->m * mat->n * mat->p;

  *numSuperBlocks = 0;

  for (int i = 0; i < num_diags; ++i) {
    dSuper = (*mat->diagonals)[i];
    blockCol = dSuper + blockRow;
    if ( (dSuper >= 0) && (blockCol < maxBlockCol) ) {
      PetscInt blockIndex = -1;
      std::map<int, int> &diag_starts = *(mat->diag_starts);
      std::map<int, int>::iterator I = diag_starts.find(dSuper);
      if (I != diag_starts.end()) {
	blockIndex = I->second + blockRow*mat->dof;
	if ( dSuper > 0 ) {
	  superBlockPtrs[*numSuperBlocks] = &(mat->hostData[blockIndex]);
	  knownPtrs[(*numSuperBlocks)++] = &(xx[blockCol*mat->dof]);
	}
	else
	  *diagBlockPtr =  &(mat->hostData[blockIndex]);
      }
    }
  }

  PetscFunctionReturn(ierr);
}


#undef __FUNCT__  
#define __FUNCT__ "MatSolve_SeqSGGPU_compare"
//---
//
// comparison Version
//
//---
PetscErrorCode MatSolve_SeqSGGPU_compare(Mat A,Vec bb,Vec xx)
{
  Mat_SeqSGGPU        *a = (Mat_SeqSGGPU*)A->data;
  PetscErrorCode    ierr;
  PetscInt          i, j, k, sbIndex, n=A->rmap->n, dof = a->dof, dim = a->dim;
  PetscInt          numBlockRows = a->m * a->n * a->p;
  PetscInt          numDiags = a->diagonals->size();
  PetscInt          chunkSize;
  PetscScalar       *x;
  const PetscScalar *b;
  PetscScalar      *diagBlock;
  PetscInt          numOffDiagBlocks;
  PetscInt          numElements = a->m * a->n * a->p * dof;
  PetscInt          chunk, numChunks;
  PetscScalar      *iluSolveVec;  //< host data for result of lower triangular solve
  PetscScalar     **offDiagBlocks;
  PetscScalar     **knowns;
  PetscScalar      *beta;
  PetscInt          num_diags = a->diagonals->size();

  // Allocate work arrays
  ierr = PetscMalloc(numElements * sizeof(PetscScalar), &iluSolveVec);CHKERRQ(ierr);
  // sub_blocks holds pointers to the nonzero blocks in the current working row
  // knowns holds pointers to the corresponding portions of the solution vector (already solved)  
  ierr = PetscMalloc( (num_diags/2) * sizeof(PetscScalar*), &offDiagBlocks); CHKERRQ(ierr);
  ierr = PetscMalloc( (num_diags/2) * sizeof(PetscScalar*), &knowns);    CHKERRQ(ierr);
  // beta holds the RHS for the dof*dof system that is solved for a blockRow
  ierr = PetscMalloc(dof * sizeof(PetscScalar), &beta); CHKERRQ(ierr);

#if defined(PETSC_USE_DEBUG)
  PetscScalar *checkSolve, maxDiff, diff, maxNorm, relDiff;
#endif

  PetscFunctionBegin;
  if (!n) PetscFunctionReturn(0);

  if ( dumpVec ) {
    dumpVec = PETSC_FALSE;
    PetscViewer rhsViewer;
    ierr = PetscViewerBinaryOpen(PETSC_COMM_WORLD,"sgvec.full.bin",FILE_MODE_WRITE, &rhsViewer);CHKERRQ(ierr);
    ierr = VecView(bb,rhsViewer);CHKERRQ(ierr);
    ierr = PetscViewerDestroy(&rhsViewer);CHKERRQ(ierr);
  }

  // set chunkSize -- this determines how many rows of the solve to do as a unit
  chunkSize = dim==3 ? a->p : dim==2 ? a->n : 1;  

  // x is the input RHS, iluSolveVec is the result of lower triangular solve
  ierr = VecGetArray(xx,&x);CHKERRQ(ierr);
  ierr = VecGetArrayRead(bb,&b);CHKERRQ(ierr);
#if defined(PETSC_USE_DEBUG)
  ierr = PetscMalloc( dof * sizeof(PetscScalar*), &checkSolve); CHKERRQ(ierr);
#endif

  CUSPARRAY * rhsgpu;
  CUSPARRAY * xgpu;

  ierr = VecCUSPGetArrayWrite(xx, &xgpu); CHKERRQ(ierr);
  ierr = VecCUSPGetArrayRead(bb, &rhsgpu); CHKERRQ(ierr);

  PetscScalar * devRHS = thrust::raw_pointer_cast(&(*rhsgpu)[0]);
  PetscScalar * devX = thrust::raw_pointer_cast(&(*xgpu)[0]);

  // Bind X to device texture
  int gridSize = a->m * a->n * a->p;
  int mat_size = gridSize * dof;
  checkCudaError(hipBindTexture(0, vector_rhs, devRHS, mat_size * sizeof(PetscScalar)));


  dim3 block(chunkSize*dof, BLOCKWIDTH_Y);
  dim3 grid(1, 1);
  // dim3 grid((int)ceil((float)(mat_size)/(float)BLOCKWIDTH_X / 1.0), 1);
  int shared_size = 0;
  numChunks = ((mat_size/dof) + (chunkSize-1)) / chunkSize;
#if _TRACE
  printf("mat_size = %d, chunkSize = %d, numChunks = %d, dof = %d\n", mat_size, chunkSize, numChunks, dof);
#endif

  for ( chunk = 0; chunk < numChunks; chunk++ ) {
#if _TRACE
    printf("do cuda lower triangular solve for chunk %d\n", chunk);
#endif
    MatSolveLowerKernel<<<grid, block, shared_size, a->stream>>>(a->deviceData, a->deviceY, mat_size, a->deviceDiags,
								 a->diagonals->size() / 2, chunk, chunkSize, dof);
  }

  ierr = WaitForGPU() ; CHKERRCUSP(ierr);
  hipUnbindTexture(vector_rhs);
  ierr = VecCUSPRestoreArrayRead(bb, &rhsgpu); CHKERRQ(ierr);

#if _TRACE
  printf("done cuda lower triangular solve\n");
  PetscScalar *gpuCheckY;
  ierr = PetscMalloc(mat_size * sizeof(PetscScalar), &gpuCheckY); CHKERRQ(ierr);
  checkCudaError(hipMemcpy(gpuCheckY, a->deviceY, mat_size * sizeof(PetscScalar), hipMemcpyDeviceToHost));
#endif

  checkCudaError(hipBindTexture(0, vector_rhs, a->deviceY, mat_size * sizeof(PetscScalar)));

  //  checkCudaError(hipBindTexture(0, vector_rhs, devY, mat_size * sizeof(PetscScalar)));

  for ( chunk = numChunks-1; chunk >= 0; chunk-- ) {
#if _TRACE
    //    printf("do cuda upper triangular solve for chunk %d\n", chunk);
#endif
    MatSolveUpperKernel<<<grid, block, shared_size, a->stream>>>(a->deviceData, devX, // a->deviceY,
								 gridSize, a->deviceDiags,
								 a->diagonals->size() / 2, chunk, chunkSize, dof);
  }

#if _TRACE
  printf("done cuda upper triangular solve\n");
  PetscScalar *gpuCheckX;
  ierr = PetscMalloc(mat_size * sizeof(PetscScalar), &gpuCheckX); CHKERRQ(ierr);
  checkCudaError(hipMemcpy(gpuCheckX, devX, mat_size * sizeof(PetscScalar), hipMemcpyDeviceToHost));
#endif

  hipUnbindTexture(vector_rhs);

  // LOWER TRIANGULAR SOLVE
  // proceed a block row at a time
  // result of LT solve is stored in iluSolveVec
  for ( i = 0; i < numBlockRows; i++ ) {

    // initialize the RHS with the components of b
    PetscMemcpy( beta, &(b[i*dof]), dof*sizeof(PetscScalar) );

    // fetch pointers to the blocks left of diagonal in this row and the
    // corresponding portions of the result which have already been found
    ierr = MatGetLowerSolveBlocks_SeqSGGPU( a, iluSolveVec, i, &numOffDiagBlocks,
					    offDiagBlocks, knowns, &diagBlock );

    // update RHS by subtracting mat-vec products
    for ( sbIndex = 0; sbIndex < numOffDiagBlocks; sbIndex++ ) {
      for ( k = 0; k < dof; k++ ) {
	for ( j = 0; j < dof; j++ ) {
	  beta[j] -= offDiagBlocks[sbIndex][j + k*numElements] * knowns[sbIndex][k];
	}
      }
    }

    // solve diagBlock * x = beta
    for ( j = 0; j < dof; j++ ) {
      for ( k = 0; k < j; k++ )
	beta[j] -= (diagBlock[k*numElements+j]*beta[k]);
    }

    PetscMemcpy( iluSolveVec + i*dof, beta, dof*sizeof(PetscScalar) );    

#if defined(PETSC_USE_DEBUG)
    // CHECK THAT SOLVE IS CORRECT
    PetscMemzero( checkSolve, dof * sizeof(PetscScalar) );    
    // fetch pointers to the blocks left of diagonal in this row and the
    // corresponding portions of the result stored in iluSolveVec
    ierr = MatGetLowerSolveBlocks_SeqSGGPU( a, iluSolveVec, i, &numOffDiagBlocks,
					    offDiagBlocks, knowns, &diagBlock );

    // multiply iluSolveVec by diagBlock
    for ( j = 0; j < dof; j++ ) {
      for ( k = 0; k < j; k++ )
	checkSolve[j] += diagBlock[j + numElements*k]*iluSolveVec[i*dof+k];
      checkSolve[j] += iluSolveVec[i*dof+j];
    }

    // update RHS by adding mat-vec products
    for ( sbIndex = 0; sbIndex < numOffDiagBlocks; sbIndex++ ) {
      for ( j = 0; j < dof; j++ ) {
	for ( k = 0; k < dof; k++ ) {
	  checkSolve[j] += offDiagBlocks[sbIndex][j + numElements * k] * knowns[sbIndex][k];
	}
      }
    }

    maxDiff = 0;
    for ( k = 0; k < dof; k++ ) {
      diff = (b[i*dof+k]-checkSolve[k])*(b[i*dof+k]-checkSolve[k]);
      if (diff > maxDiff) {
	maxDiff = diff;
      }
    }
    if ( maxDiff > 0.001 )
      printf("After Lower Solve row %d, max diff is %f\n", i, maxDiff);
#endif

  }

#if _TRACE
  printf("Compare host and device L solve:\n");
  maxDiff = 0;
  maxDiffIndex = -1;
  for ( i = 0; i < mat_size; i++ ) {
    //printf("hostY[%d] = %f     devY[%d] = %f\n", i, iluSolveVec[i], i, gpuCheckY[i]);
    diff = (iluSolveVec[i] - gpuCheckY[i])*(iluSolveVec[i] - gpuCheckY[i]);
    if ( diff > maxDiff ) {
      maxDiff = diff;
      maxDiffIndex = i;
    }
  }
  printf("max diff between host and device L solve is %f at index %d\n", maxDiff, maxDiffIndex);
  PetscFree(gpuCheckY);
#endif

  //-----------------------
  // UPPER TRIANGULAR SOLVE
  //-----------------------

  // offDiagBlocks holds pointers to the off-diagonal
  // non-zero blocks in the current working row
  for ( i = numBlockRows-1; i >= 0; i-- ) {

    // initialize the RHS with the components of iluSolveVec
    PetscMemcpy( beta, &(iluSolveVec[i*dof]), dof*sizeof(PetscScalar) );    

    // fetch pointers to the blocks below diagonal in this row and the
    // corresponding portions of the result which have already been found
    ierr = MatGetUpperSolveBlocks_SeqSGGPU( a, x, i, &numOffDiagBlocks,
					    offDiagBlocks, knowns, &diagBlock );

    // update RHS by subtracting mat-vec products
    for ( sbIndex = 0; sbIndex < numOffDiagBlocks; sbIndex++ ) {
      for ( k = 0; k < dof; k++ ) {
	for ( j = 0; j < dof; j++ ) {
	  beta[j] -= offDiagBlocks[sbIndex][j + k*numElements] * knowns[sbIndex][k];
	}
      }
    }

    // solve diagBlock * x = beta
    for ( j = dof-1; j >= 0; j-- ) {
      for ( k = j+1; k < dof; k++ )
	beta[j] -= ( diagBlock[j + k*numElements] * beta[k] );
      beta[j] /= diagBlock[j*numElements+j];
    }

    PetscMemcpy( x + i*dof, beta, dof*sizeof(PetscScalar) );    

#if defined(PETSC_USE_DEBUG)
    // CHECK THAT SOLVE IS CORRECT
    PetscMemzero( checkSolve, dof * sizeof(PetscScalar) );    
    // fetch pointers to the blocks right of diagonal in this row and the
    // corresponding portions of the result stored in x
    ierr = MatGetUpperSolveBlocks_SeqSGGPU( a, x, i, &numOffDiagBlocks,
					    offDiagBlocks, knowns, &diagBlock );

    // multiply x by diagBlock
    for ( j = 0; j < dof; j++ )
      for ( k = j; k < dof; k++ )
	checkSolve[j] += diagBlock[j + numElements*k] * x[i*dof+k];

    // update RHS by adding mat-vec products
    for ( sbIndex = 0; sbIndex < numOffDiagBlocks; sbIndex++ ) {
      for ( j = 0; j < dof; j++ ) {
	for ( k = 0; k < dof; k++ ) {
	  checkSolve[j] += offDiagBlocks[sbIndex][j + numElements * k] * knowns[sbIndex][k];
	}
      }
    }

    maxDiff = 0;
    maxNorm = 0;
    for ( k = 0; k < dof; k++ ) {
      diff = sqrt( (iluSolveVec[i*dof+k]-checkSolve[k])*(iluSolveVec[i*dof+k]-checkSolve[k]) );
      if (diff > maxDiff) {
	maxDiff = diff;
	maxNorm = sqrt( iluSolveVec[i*dof+k] * iluSolveVec[i*dof+k] );
      }
    }
    relDiff = maxDiff / maxNorm;
    if ( maxDiff > 0.01 ) {
      printf("After Upper Solve row %d, max diff is %f, rel diff is %f\n", i, maxDiff, relDiff);

      printf("relevant x entries:\n");
      for ( j = 0; j < dof; j++ )      
	printf( "x[%d] = %f\n", j, x[i*dof+j] );

      printf("Diag block entries:\n");
      for ( j = 0; j < dof; j++ ) {
	for ( k = j; k < dof; k++ )
	  printf( "%10f", diagBlock[j + numElements*k]);
	printf("\n");
      }

      for ( sbIndex = 0; sbIndex < numOffDiagBlocks; sbIndex++ ) {
	printf("knowns %d entries:\n",sbIndex);
	for ( j = 0; j < dof; j++ )
	  printf("knowns[%d][%d] = %f\n",knowns[sbIndex][j]);
	
	printf("Upper block %d entries:\n",sbIndex);
	for ( j = 0; j < dof; j++ ) {
	  for ( k = 0; k < dof; k++ )
	    printf( "%10f", offDiagBlocks[sbIndex][j + numElements * k] );
	  printf("\n");
	}
      }
    }
#endif

  }

#if _TRACE
  printf("Compare host and device LU solve:\n");
  maxDiff = 0;
  for ( i = 0; i < mat_size; i++ ) {
    //printf("hostX[%d] = %f     devX[%d] = %f\n", i, x[i], i, gpuCheckX[i]);
    diff = (x[i] - gpuCheckX[i])*(x[i] - gpuCheckX[i]);
    if ( diff > maxDiff ) maxDiff = diff;
  }
  printf("max diff between host and device LU solve is %f\n", maxDiff);
  PetscFree(gpuCheckX);
#endif

//   printf("Factored sggpu solve result\n");
//   for ( i = 0; i < numElements; i++ )
//     printf("x[%d] = %f\n", i, x[i]);

  ierr = VecRestoreArray(xx,&x);CHKERRQ(ierr);
  ierr = VecRestoreArrayRead(bb,&b);CHKERRQ(ierr);

#if defined(PETSC_USE_DEBUG)
  ierr = PetscFree(checkSolve); CHKERRQ(ierr);
  ierr = PetscFree(iluSolveVec);CHKERRQ(ierr);
  ierr = PetscFree(offDiagBlocks); CHKERRQ(ierr);
  ierr = PetscFree(knowns);CHKERRQ(ierr);
  ierr = PetscFree(beta);CHKERRQ(ierr);
#endif

  //ierr = PetscLogFlops(2*a->nz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "MatSolve_SeqSGGPU_gpu"
//---
//
// GPU Version
//
//---
PetscErrorCode MatSolve_SeqSGGPU_gpu(Mat A,Vec bb,Vec xx)
{
  Mat_SeqSGGPU      *a = (Mat_SeqSGGPU*)A->data;
  PetscBool         isseqcusp,isseqgpu,ismpicusp,iscusp;
  PetscErrorCode    ierr;
  PetscInt          n=A->rmap->n, dof = a->dof, dim = a->dim;
  PetscInt          numDiags = a->diagonals->size();
  PetscInt          matSize, chunkSize, chunk, numChunks;
  PetscScalar       *devX, *devB;
#if _CHECK_SOLVE
  PetscScalar       *checkSolve;
#endif

  PetscFunctionBegin;
  if (!n) PetscFunctionReturn(0);

#if _TIME
  double t_start, t_end, elapsed;
  t_start = getclock();
#endif

  if ( dumpVec ) {
    dumpVec = PETSC_FALSE;
    PetscViewer rhsViewer;
    ierr = PetscViewerBinaryOpen(PETSC_COMM_WORLD,"sgvec.full.bin",FILE_MODE_WRITE, &rhsViewer);CHKERRQ(ierr);
    ierr = VecView(bb,rhsViewer);CHKERRQ(ierr);
    ierr = PetscViewerDestroy(&rhsViewer);CHKERRQ(ierr);
  }

  // set chunkSize -- this determines how many rows of the solve to do as a unit
  chunkSize = dim==3 ? a->p : dim==2 ? a->n : 1;

#if _CHECK_SOLVE
  ierr = PetscMalloc( dof * sizeof(PetscScalar*), &checkSolve); CHKERRQ(ierr);
#endif

  CUSPARRAY * rhsgpu;
  CUSPARRAY * xgpu;

  ierr = PetscObjectTypeCompare((PetscObject)xx,VECSEQCUSP,&isseqcusp);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)xx,VECMPICUSP,&ismpicusp);CHKERRQ(ierr);
  iscusp = (isseqcusp || ismpicusp) ? PETSC_TRUE : PETSC_FALSE;
  ierr = PetscObjectTypeCompare((PetscObject)xx,VECSEQGPU,&isseqgpu);CHKERRQ(ierr);
  if (isseqgpu) {
    dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
    dim3 grid((int)ceil((float)(a->m * a->n * a->p * a->dof)/(float)BLOCKWIDTH_X / 1.0), 1);

    Vec_SeqGPU *xd = (Vec_SeqGPU*) xx->data;
    /* Make sure bb is also VECSEQGPU */
    ierr = PetscObjectTypeCompare((PetscObject)bb,VECSEQGPU,&isseqgpu);CHKERRQ(ierr);
    if (!isseqgpu) {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Both x and b must be same type");
    }
    Vec_SeqGPU *bd = (Vec_SeqGPU*) bb->data;
    /* synch up x */
    if (xd->syncState==VEC_CPU) {
      ierr = VecCopyOverH2D(xx,xd->cpuptr);CHKERRQ(ierr);
      xd->syncState=VEC_SYNCHED;
    }
    /* Get device pointer for X */
    devX = xd->devptr;
    devB = bd->devptr;
    /* Bind X to device texture */
    matSize = a->m * a->n * a->p * a->dof;
    
    checkCudaError(hipBindTexture(0, vector_rhs, devB, matSize * sizeof(PetscScalar)));    
  }


  else if (iscusp) {
    dim3 block(BLOCKWIDTH_X, BLOCKWIDTH_Y);
    dim3 grid((int)ceil((float)(a->m * a->n * a->p * a->dof)/(float)BLOCKWIDTH_X / 1.0), 1);

    /* Make sure y is also VECCUSP */
    ierr = PetscObjectTypeCompare((PetscObject)bb,VECSEQCUSP,&isseqcusp);CHKERRQ(ierr);
    ierr = PetscObjectTypeCompare((PetscObject)bb,VECMPICUSP,&ismpicusp);CHKERRQ(ierr);
    iscusp = (isseqcusp || ismpicusp) ? PETSC_TRUE : PETSC_FALSE;
    if (!iscusp) {
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Both x and b must be same type");
    }
    matSize = a->m * a->n * a->p * a->dof;
    ierr = VecCUSPGetArrayWrite(xx, &xgpu); CHKERRQ(ierr);
    ierr = VecCUSPGetArrayRead(bb, &rhsgpu); CHKERRQ(ierr);
    devX = thrust::raw_pointer_cast(&(*xgpu)[0]);
    devB = thrust::raw_pointer_cast(&(*rhsgpu)[0]);

    /* Bind X to device texture */
    checkCudaError(hipBindTexture(0, vector_rhs, devB, matSize * sizeof(PetscScalar)));
  }

  else {
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_INCOMP,"Vec must be seqgpu or cusp type");
  }

#if _TIME
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("gpu solve preamble time %lf\n",elapsed);
  t_start = getclock();
#endif

  dim3 block(chunkSize*dof, BLOCKWIDTH_Y);
  dim3 grid(1, 1);
  // dim3 grid((int)ceil((float)(matSize)/(float)BLOCKWIDTH_X / 1.0), 1);
  int shared_size = 2*chunkSize*dof*sizeof(PetscScalar);
  numChunks = ((matSize/dof) + (chunkSize-1)) / chunkSize;
#if _TRACE
  printf("matSize = %d, chunkSize = %d, numChunks = %d, dof = %d\n", matSize, chunkSize, numChunks, dof);
#endif

  for ( chunk = 0; chunk < numChunks; chunk++ ) {
#if _TRACE
    //    printf("do cuda lower triangular solve for chunk %d\n", chunk);
#endif
    MatSolveLowerKernel<<<grid, block, shared_size, a->stream>>>(a->deviceData, devX, matSize, a->deviceDiags,
								 a->diagonals->size() / 2, chunk, chunkSize, dof);
  }

#if _TIME
  checkCudaError(hipStreamSynchronize(a->stream));
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("gpu solve lower time %lf\n",elapsed);
  t_start = getclock();
#endif

  ierr = WaitForGPU() ; CHKERRCUSP(ierr);
  hipUnbindTexture(vector_rhs);
  ierr = VecCUSPRestoreArrayRead(bb, &rhsgpu); CHKERRQ(ierr);

#if _TIME
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("gpu solve unbind time %lf\n",elapsed);
  t_start = getclock();
#endif

#if _TRACE
  printf("done cuda lower triangular solve\n");
  PetscScalar *gpuCheckY;
  ierr = PetscMalloc(matSize * sizeof(PetscScalar), &gpuCheckY); CHKERRQ(ierr);
  checkCudaError(hipMemcpy(gpuCheckY, a->deviceY, matSize * sizeof(PetscScalar), hipMemcpyDeviceToHost));
#endif

  // checkCudaError(hipBindTexture(0, vector_rhs, a->deviceY, matSize * sizeof(PetscScalar)));

#if _TIME
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("gpu solve bind 2 time %lf\n",elapsed);
  t_start = getclock();
#endif

  for ( chunk = numChunks-1; chunk >= 0; chunk-- ) {
#if _TRACE
    //    printf("do cuda upper triangular solve for chunk %d\n", chunk);
#endif
    MatSolveUpperKernel<<<grid, block, shared_size, a->stream>>>(a->deviceData, devX, matSize, a->deviceDiags,
								 a->diagonals->size() / 2, chunk, chunkSize, dof);
  }


#if _TIME
  checkCudaError(hipStreamSynchronize(a->stream));
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("gpu solve upper time  %lf\n", elapsed);
  t_start = getclock();
#endif

#if _TRACE
  printf("done cuda upper triangular solve\n");
#endif
  hipUnbindTexture(vector_rhs);
  ierr = VecCUSPRestoreArrayWrite(xx, &xgpu); CHKERRQ(ierr);

#if _TIME
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("gpu solve vec restore time %lf\n",elapsed);
  t_start = getclock();
#endif

#if _CHECK_SOLVE
  ierr = PetscFree(checkSolve); CHKERRQ(ierr);
#endif

  //ierr = PetscLogFlops(2*a->nz - A->cmap->n);CHKERRQ(ierr);

#if _TIME
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("gpu cleanup time %lf\n",elapsed);
#endif

  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "MatSolve_SeqSGGPU_cpu"
//---
//
// CPU Version
//
//---
PetscErrorCode MatSolve_SeqSGGPU_cpu(Mat A,Vec bb,Vec xx)
{
  Mat_SeqSGGPU        *a = (Mat_SeqSGGPU*)A->data;
  PetscErrorCode    ierr;
  PetscInt          j, k, n=A->rmap->n, dof = a->dof, dim = a->dim;
  PetscInt          numDiags = a->diagonals->size();
  PetscScalar       *x;
  const PetscScalar *b;
  PetscScalar       sca1, sca2, sca3;
  PetscInt          gridSize = a->m * a->n * a->p;
  PetscInt          numElements = gridSize * dof;
  PetscInt          chunkSize, chunk, numChunks;
  PetscScalar       beta[MAXDOF];
#if _CHECK_SOLVE
  PetscInt          numOffDiagBlocks;
  PetscScalar      *diagBlock;
  PetscInt          sbIndex;
  PetscScalar      *checkSolve, maxDiff, diff, maxNorm, relDiff;
#endif

  PetscFunctionBegin;
  if (!n) PetscFunctionReturn(0);

#if _TIME
  double t_start, t_end, elapsed;
  t_start = getclock();
#endif

  if ( dumpVec ) {
    dumpVec = PETSC_FALSE;
    PetscViewer rhsViewer;
    ierr = PetscViewerBinaryOpen(PETSC_COMM_WORLD,"sgvec.full.bin",FILE_MODE_WRITE, &rhsViewer);CHKERRQ(ierr);
    ierr = VecView(bb,rhsViewer);CHKERRQ(ierr);
    ierr = PetscViewerDestroy(&rhsViewer);CHKERRQ(ierr);
  }  

  // set chunkSize -- this determines how many rows of the solve to do as a unit
  chunkSize = dim==3 ? a->p : dim==2 ? a->n : 1;

  // x is the input RHS, temp is the result of lower triangular solve
  ierr = VecGetArray(xx,&x);CHKERRQ(ierr);
  ierr = VecGetArrayRead(bb,&b);CHKERRQ(ierr);
#if _CHECK_SOLVE
  ierr = PetscMalloc(nz * sizeof(PetscScalar), &iluSolveVec);CHKERRQ(ierr);
  // sub_blocks holds pointers to the nonzero blocks in the current working row
  // knowns holds pointers to the corresponding portions of the solution vector (already solved)  
  ierr = PetscMalloc( (num_diags/2) * sizeof(PetscScalar*), &offDiagBlocks); CHKERRQ(ierr);
  ierr = PetscMalloc( (num_diags/2) * sizeof(PetscScalar*), &knowns);    CHKERRQ(ierr);
  // beta holds the RHS for the dof*dof system that is solved for a blockRow
  ierr = PetscMalloc( dof * sizeof(PetscScalar*), &checkSolve); CHKERRQ(ierr);
#endif

  // LOWER TRIANGULAR SOLVE
  // determine the diagonals that contribute already solved values to the RHS for this thread
  PetscInt center_diag = numDiags / 2;
  PetscInt max_solved_diag = center_diag;

  // initialize the RHS with the components of b
  // PetscMemcpy( a->iluSolveVec, b, numElements*sizeof(PetscScalar) );    
  PetscMemcpy( x, b, numElements*sizeof(PetscScalar) );    

  // PetscScalar sum[MAXBLOCKSIZE];

  numChunks = ((numElements/dof) + (chunkSize-1)) / chunkSize;
#if _TRACE
  printf("matSize = %d, chunkSize = %d, numChunks = %d, dof = %d\n", numElements, chunkSize, numChunks, dof);
#endif

  // proceed a chunk of rows at a time
  // result of LT solve is stored in a->iluSolveVec
  while ( (0 <= max_solved_diag) && ((*a->diagonals)[max_solved_diag] > -chunkSize ) )
    --max_solved_diag;

  for ( chunk = 0; chunk < numChunks; chunk++ ) {
    // offset into portion of iluSolveVec being solved for this chunk
    int offset2 = (chunk*chunkSize)*dof;


    //---------------------------------------------
    // Update the RHS with elements of the solution
    // vector solved in previous chunks
    //---------------------------------------------
    for ( int di = 0; di < center_diag; di++ ) {  //for ( int di = 0; di <= max_solved_diag; di++ ) {
      int d = (*a->diagonals)[di];
      int startBlock = chunk*chunkSize + d >= 0 ? 0 : -(chunk*chunkSize + d);
      int endBlock = chunkSize > -d ? -d : chunkSize;
      int offset1 = (chunk*chunkSize + d)*dof;

      for ( int rowCoord = 0; rowCoord < dof; rowCoord++ ) {

	// offset into matrix coefficients to top of stripe in chunk
	int offset0 = (di*dof + rowCoord)*numElements + chunk*chunkSize*dof;
	// offset into already solved portion of iluSolveVec needed for this chunk/diagonal combination 
	
	for ( int block = startBlock; block < endBlock; block++ ) {

	  for ( int colCoord = 0; colCoord < dof; colCoord++ ) {
	    // x[offset2 + block*dof + colCoord] -= a->hostData[ offset0 + block*dof + colCoord ] * x[offset1 + block*dof + rowCoord];
	    sca1 = a->hostData[ offset0 + block*dof + colCoord ];
	    sca2 = x[offset1 + block*dof + rowCoord];
	    sca3 = x[offset2 + block*dof + colCoord];
	    sca3 -= sca1 * sca2;
	    x[offset2 + block*dof + colCoord] = sca3;
	  }
	}
      }
    }

    for ( int block = 0; block < chunkSize; block++ ) {
      int offset0 = center_diag*dof*numElements + (chunk*chunkSize + block)*dof;
      int offset2 = (chunk*chunkSize + block)*dof;
      // solve diagBlock * x = beta
      for ( j = 1; j < dof; j++ ) {
	for ( k = 0; k < j; k++ )
	  x[offset2 + j] += ( a->hostData[offset0 + k*numElements + j] * x[offset2 + k] );
      }

      // update blocks below this diag block
      for ( int di = max_solved_diag+1; di < center_diag; di++ ) {
	int d = (*a->diagonals)[di];
	// is this thread in a diagonal block?
	if ( block - d < chunkSize ) {
	  int offset0 = di*dof*numElements + (chunk*chunkSize + block - d)*dof;
	  int offset1 = (chunk*chunkSize + block - d)*dof;
	  for ( int j = 0; j < dof; j++ ) {
	    for ( int k = 0; k < dof; k++ ) {
	      x[offset1 + j] -= a->hostData[ offset0 + k*numElements + j ] * x[offset2 + k];
	    }	
	  }
	}
      }

    }

#if _CHECK_SOLVE
    for ( int block = 0; block < chunkSize; block++ ) {
      // CHECK THAT SOLVE IS CORRECT
      PetscMemzero( checkSolve, dof * sizeof(PetscScalar) );    
      // fetch pointers to the blocks left of diagonal in this row and the
      // corresponding portions of the result stored in x
      ierr = MatGetLowerSolveBlocks_SeqSGGPU( a, x, chunk*chunkSize+block, &numOffDiagBlocks,
					      offDiagBlocks, knowns, &diagBlock );

      int offset1 = (chunk*chunkSize + block)*dof;
      // multiply x by inverse of diagBlock, i.e. solve diagBlock*checkSolve = x
      for ( j = 0; j < dof; j++ ) {
	checkSolve[j] = x[offset1 + j];
	for ( k = 0; k < j; k++ )
	  checkSolve[j] -= diagBlock[j + numElements*k]*checkSolve[k];
      }

      // update RHS by adding mat-vec products
      for ( sbIndex = 0; sbIndex < numOffDiagBlocks; sbIndex++ ) {
	for ( j = 0; j < dof; j++ ) {
	  for ( k = 0; k < dof; k++ ) {
	    checkSolve[j] += offDiagBlocks[sbIndex][j + numElements * k] * knowns[sbIndex][k];
	  }
	}
      }

      maxDiff = 0;
      for ( k = 0; k < dof; k++ ) {
	diff = (b[offset1 + k]-checkSolve[k])*(b[offset1 + k]-checkSolve[k]);
	if (diff > maxDiff) {
	  maxDiff = diff;
	}
      }
      if ( maxDiff > 0.001 )
	printf("After Lower Solve row %d, max diff is %f\n", offset1, maxDiff);
    }
#endif
  }

  //-----------------------
  // UPPER TRIANGULAR SOLVE
  //-----------------------
  // PetscMemcpy( x, iluSolveVec, numElements*sizeof(PetscScalar) );    

#if _CHECK_SOLVE
  PetscScalar *saveX;
  ierr = PetscMalloc( dof * chunkSize*sizeof(PetscScalar*), &saveX); CHKERRQ(ierr);
#endif
  
  // proceed a chunk of rows at a time
  PetscInt min_solved_diag = center_diag+1;
  while ( (min_solved_diag < numDiags) && ((*a->diagonals)[min_solved_diag] < chunkSize ) )
    ++min_solved_diag;

  for ( chunk = numChunks-1; chunk >= 0; chunk-- ) {
    // offset into portion of x being solved for this chunk
    int offset2 = (chunk*chunkSize)*dof;
#if _CHECK_SOLVE
    ierr = PetscMemcpy(saveX,&(x[offset2]), chunkSize*dof*sizeof(PetscScalar));CHKERRQ(ierr);
#endif
    // PetscMemcpy( &(x[offset2]), &(iluSolveVec[offset2]), chunkSize*dof*sizeof(PetscScalar) );    

    /* for ( int di = min_solved_diag; di < numDiags; di++ ) { */
    for ( int di = center_diag+1; di < numDiags; di++ ) {
      int d = (*a->diagonals)[di];
      int startBlock = d < chunkSize ? chunkSize - d : 0;
      int endBlock = (chunk+1)*chunkSize -1 + d < gridSize ? chunkSize - 1 : gridSize - chunk*chunkSize - 1 - d;
      // offset into already solved portion of iluSolveVec needed for this chunk/diagonal combination 
      int offset1 = (chunk*chunkSize + d)*dof;

      for ( int rowCoord = 0; rowCoord < dof; rowCoord++ ) {

	// offset into matrix coefficients to top of stripe in chunk
	int offset0 = (di*dof + rowCoord)*numElements + chunk*chunkSize*dof;
	
	for ( int block = endBlock; block >= startBlock; block-- ) {

	  for ( int colCoord = 0; colCoord < dof; colCoord++ ) {
	    x[offset2 + block*dof + colCoord] -= a->hostData[ offset0 + block*dof + colCoord ] * x[offset1 + block*dof + rowCoord];
	  }
	}
      }
    }

    for ( int block = chunkSize-1; block >= 0; block-- ) {
      int offset0 = center_diag*dof*numElements + (chunk*chunkSize + block)*dof;
      int offset2 = (chunk*chunkSize + block)*dof;
      // solve diagBlock * x = beta
      //PetscMemcpy( beta, &(x[offset2]), dof*sizeof(PetscScalar) );
      for ( j = dof-1; j >= 0; j-- ) {
	x[offset2 + j] *= a->hostData[offset0 + j*numElements + j];
	for ( k = j+1; k < dof; k++ )
	  x[offset2 + j] += ( a->hostData[offset0 + k*numElements + j] * x[offset2 + k] );
      }

      // update blocks above this diag block
      for ( int di = min_solved_diag-1; di > center_diag; di-- ) {
	int d = (*a->diagonals)[di];
	// is this thread in a diagonal block?
	if ( block - d >= 0 ) {
	  int offset0 = di*dof*numElements + (chunk*chunkSize + block - d)*dof;
	  int offset1 = (chunk*chunkSize + block)*dof;
	  int offset2 = (chunk*chunkSize + block - d)*dof;
	  for ( int j = 0; j < dof; j++ ) {
	    for ( int k = 0; k < dof; k++ ) {
	      x[offset2 + j] -= a->hostData[ offset0 + k*numElements + j ] * x[offset1 + k];
	    }	
	  }
	}
      }

    }

#if _CHECK_SOLVE
    for ( int block = 0; block < chunkSize; block++ ) {
      // CHECK THAT SOLVE IS CORRECT
      PetscMemzero( checkSolve, dof * sizeof(PetscScalar) );    
      // fetch pointers to the blocks right of diagonal in this row and the
      // corresponding portions of the result stored in x
      ierr = MatGetUpperSolveBlocks_SeqSGGPU( a, x, chunk*chunkSize+block, &numOffDiagBlocks,
					      offDiagBlocks, knowns, &diagBlock );
      
      int offset1 = (chunk*chunkSize + block)*dof;
      // multiply x by inverse of diagBlock, i.e. solve diagBlock*checkSolve = x
      for ( j = dof-1; j >= 0; j-- ) {
	checkSolve[j] = x[offset1 + j];
	for ( k = j+1; k < dof; k++ )
	  checkSolve[j] -= diagBlock[j + numElements*k] * checkSolve[k];
	checkSolve[j] /= diagBlock[j + numElements*j];
      }

      // update RHS by adding mat-vec products
      for ( sbIndex = 0; sbIndex < numOffDiagBlocks; sbIndex++ ) {
	for ( j = 0; j < dof; j++ ) {
	  for ( k = 0; k < dof; k++ ) {
	    checkSolve[j] += offDiagBlocks[sbIndex][j + numElements * k] * knowns[sbIndex][k];
	  }
	}
      }
      
      maxDiff = 0;
      maxNorm = 0;
      for ( k = 0; k < dof; k++ ) {
	diff = sqrt( (saveX[block*dof+k]-checkSolve[k])*(saveX[block*dof + k]-checkSolve[k]) );
	if (diff > maxDiff) {
	  maxDiff = diff;
	  maxNorm = sqrt( saveX[block*dof + k] * saveX[block*dof + k] );
	}
      }
      relDiff = maxDiff / maxNorm;
      if ( maxDiff > 0.01 ) {
	printf("After Upper Solve row %d, max diff is %f, rel diff is %f\n", offset1, maxDiff, relDiff);
	
	printf("saveX entries:\n");
	for ( j = 0; j < dof; j++ )      
	  printf( "saveX[%d] = %f\n", block*dof+j, saveX[block*dof+j] );

	printf("checkSolve entries:\n");
	for ( j = 0; j < dof; j++ )      
	  printf( "checkSolve[%d] = %f\n", j, checkSolve[j] );

	printf("relevant x entries:\n");
	for ( j = 0; j < dof; j++ )
	  printf( "x[%d] = %f\n", offset1+j, x[offset1+j] );

	printf("Diag block entries:\n");
	for ( j = 0; j < dof; j++ ) {
	  for ( k = j; k < dof; k++ )
	    printf( "%10f", diagBlock[j + numElements*k]);
	  printf("\n");
	}

	for ( sbIndex = 0; sbIndex < numOffDiagBlocks; sbIndex++ ) {
	  printf("knowns %d entries:\n",sbIndex);
	  for ( j = 0; j < dof; j++ )
	    printf("knowns[%d][%d] = %f\n",sbIndex,j,knowns[sbIndex][j]);
	  
	  printf("Upper block %d entries:\n",sbIndex);
	  for ( j = 0; j < dof; j++ ) {
	    for ( k = 0; k < dof; k++ )
	      printf( "%10f", offDiagBlocks[sbIndex][j + numElements * k] );
	    printf("\n");
	  }
	}
      }
    }
#endif

  }

  ierr = VecRestoreArray(xx,&x);CHKERRQ(ierr);
  ierr = VecRestoreArrayRead(bb,&b);CHKERRQ(ierr);

#if _CHECK_SOLVE
  ierr = PetscFree(checkSolve); CHKERRQ(ierr);
  ierr = PetscFree(saveX); CHKERRQ(ierr);
  ierr = PetscFree(iluSolveVec);CHKERRQ(ierr);
  ierr = PetscFree(offDiagBlocks); CHKERRQ(ierr);
  ierr = PetscFree(knowns);CHKERRQ(ierr);
#endif

#if _TIME
  t_end = getclock();
  elapsed = t_end - t_start;
  printf("cpu solve time %lf\n",elapsed);
#endif
  //ierr = PetscLogFlops(2*a->nz - A->cmap->n);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


EXTERN_C_BEGIN
#undef __FUNCT__  
#define __FUNCT__ "MatGetFactor_seqsggpu_petsc"
PetscErrorCode MatGetFactor_seqsggpu_petsc(Mat A,MatFactorType ftype,Mat *B)
{
  PetscInt           n = A->rmap->n;
  PetscErrorCode     ierr;

  PetscFunctionBegin;
  dumpMat = PETSC_FALSE;
  dumpVec = PETSC_FALSE;

  if ( dumpMat ) {
    dumpMat = PETSC_FALSE;
    PetscViewer sgmatViewer;
    ierr = PetscViewerBinaryOpen(PETSC_COMM_WORLD,"sgmat.full.bin",FILE_MODE_WRITE, &sgmatViewer);CHKERRQ(ierr);
    ierr = MatView(A,sgmatViewer);CHKERRQ(ierr);
    ierr = PetscViewerDestroy(&sgmatViewer);CHKERRQ(ierr);
  }

  ierr = MatCreate(((PetscObject)A)->comm,B);CHKERRQ(ierr);
  ierr = MatSetSizes(*B,n,n,n,n);CHKERRQ(ierr);
  ierr = MatSetType(*B,MATSEQSGGPU);CHKERRQ(ierr);
  if (!(*B)->preallocated) {
    Mat_SeqSGGPU* a = (Mat_SeqSGGPU*)A->data;
    PetscInt       dims[3], *starts;
    dims[0] = a->m;
    dims[1] = a->n;
    dims[2] = a->p;
    starts = (PetscInt*)malloc(sizeof(PetscInt)*a->dim);
    ierr = MatSetStencil(*B,a->dim,dims,starts,a->dof); CHKERRQ(ierr);
    ierr = MatSeqSGGPUSetPreallocation(*B,0,a->dof);CHKERRQ(ierr);
  }

  if (ftype == MAT_FACTOR_LU || ftype == MAT_FACTOR_ILU || ftype == MAT_FACTOR_ILUDT){
    (*B)->ops->ilufactorsymbolic = MatILUFactorSymbolic_SeqSGGPU;
    // (*B)->ops->lufactorsymbolic  = MatLUFactorSymbolic_SeqSGGPU;
  } else SETERRQ(PETSC_COMM_SELF,PETSC_ERR_SUP,"Factor type not supported");
  
  (*B)->factortype = ftype;
  
  PetscFunctionReturn(0);
}
EXTERN_C_END

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatConvertLU_SeqSGGPU_SeqAIJ"
/*
  MatConvertLU_SeqSGGPU_SeqAIJ - Converts from an LU-factored sggpu format to two seqaij.
  John Eisenlohr*/
PetscErrorCode MatConvertLU_SeqSGGPU_SeqAIJ(Mat A, Mat *AIJ_L, Mat *AIJ_U ){
  //printf(".................MatConvertLU_SeqSGGPU_SeqAIJ() called\n");
  PetscFunctionBegin;
  Mat_SeqSGGPU *a = (Mat_SeqSGGPU *) A->data;
  PetscScalar *hostDataCopy, *hostDataOrig;
  Mat B,C;
  PetscInt i,j, kl, ku;
  PetscInt m = A->rmap->n,n = A->cmap->n;
  PetscScalar *vals, *vals_l, *vals_u;
  PetscInt *cols, *cols_l, *cols_u;
  PetscErrorCode ierr;
  PetscInt *nnz_l, *nnz_u, nnz;
  PetscInt blockRow, rowInBlock;
  PetscInt dof = a->dof, numBlocks = a->m * a->n * a->p;
  PetscInt numDiags = a->diagonals->size();

  // For efficiency, the diagonal blocks of both the
  // lower and upper factors of A have been inverted
  // but we want to return the factors with univerted diagonal blocks.
  // So we copy the host data from A, invert the diagonals and
  // temporarily replace A's host data with the modified data
  ierr = PetscMalloc(numBlocks*dof*dof*numDiags*sizeof(PetscScalar),&hostDataCopy);CHKERRQ(ierr);
  ierr = PetscMemcpy( hostDataCopy, a->hostData, numBlocks*dof*dof*numDiags*sizeof(PetscScalar) );CHKERRQ(ierr);
  ierr = InvertFactoredDiagBlocks(a,hostDataCopy);CHKERRQ(ierr);
  hostDataOrig = a->hostData;
  a->hostData = hostDataCopy;

  // arrays to hold number of zeros in lower and upper portions of each row
  ierr = PetscMalloc(m*sizeof(PetscInt),&nnz_l); CHKERRQ(ierr);
  ierr = PetscMalloc(m*sizeof(PetscInt),&nnz_u); CHKERRQ(ierr);

  // workspace for holding the upper and lower portions of each row
  // these are reused for each row
  ierr = PetscMalloc(n*sizeof(PetscInt),&cols_l); CHKERRQ(ierr);
  ierr = PetscMalloc(n*sizeof(PetscInt),&cols_u); CHKERRQ(ierr);
  ierr = PetscMalloc(n*sizeof(PetscScalar),&vals_l); CHKERRQ(ierr);
  ierr = PetscMalloc(n*sizeof(PetscScalar),&vals_u); CHKERRQ(ierr);

  // count non-zeros in upper and lower triangles
  // there is surely a closed-form way to do this
  for(i=0;i<m;i++){
    blockRow = i / a->dof;
    nnz_l[i]=0;
    nnz_u[i]=0;
    for (j = 0; j < a->diagonals->size(); ++j) {
      int d = (*a->diagonals)[j];
      if ( ((d + blockRow) >= 0) && ((d + blockRow) < numBlocks) ) {
	if ( d < 0 ) // block is below diagonal
	  nnz_l[i] += a->dof;
	else if ( d > 0 ) // block is above diagonal
	  nnz_u[i] += a->dof;
	else { // block is on diagonal
	  rowInBlock = i % a->dof;
	  nnz_l[i] += rowInBlock + 1;
	  nnz_u[i] += (a->dof - rowInBlock);
	}
      }
    }
  }

  printf("m: %d, n: n: %d\n",m,n);
  ierr = MatCreateSeqAIJ(((PetscObject)A)->comm,m,n,PETSC_NULL,nnz_l,&B);CHKERRQ(ierr);
  ierr = MatCreateSeqAIJ(((PetscObject)A)->comm,m,n,PETSC_NULL,nnz_u,&C);CHKERRQ(ierr);

  // get each row from sggpu, divide into L and R pieces,
  // set values in L and R matrices
  for(i=0;i<m;i++){
      nnz = 0;
      ierr = MatGetRow_SeqSGGPU(A,i, &nnz, &cols,&vals); CHKERRQ(ierr);
      kl = 0;
      ku = 0;
      for ( j = 0; j < nnz; j++ ) {
	if ( cols[j] < i ) {
	  vals_l[kl] = vals[j];
	  cols_l[kl++] = cols[j];
	}
	else {
	  vals_u[ku] = vals[j];
	  cols_u[ku++] = cols[j];
	}
      }
      // add the 1 on the diagonal of L
      vals_l[kl] = 1.0;
      cols_l[kl] = i;

      ierr = MatRestoreRow_SeqSGGPU(A,i,&nnz,&cols,&vals);CHKERRQ(ierr);

      ierr = MatSetValues(B,1,&i,nnz_l[i],cols_l,vals_l,INSERT_VALUES);
      ierr = MatSetValues(C,1,&i,nnz_u[i],cols_u,vals_u,INSERT_VALUES);
  }
  ierr = MatAssemblyBegin(B,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyEnd(B,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyBegin(C,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);
  ierr = MatAssemblyEnd(C,MAT_FINAL_ASSEMBLY);CHKERRQ(ierr);

  ierr=PetscFree(nnz_l);CHKERRQ(ierr);
  ierr=PetscFree(nnz_u);CHKERRQ(ierr);
  ierr=PetscFree(cols_l);CHKERRQ(ierr);
  ierr=PetscFree(cols_u);CHKERRQ(ierr);
  ierr=PetscFree(vals_l);CHKERRQ(ierr);
  ierr=PetscFree(vals_u);CHKERRQ(ierr);
  a->hostData = hostDataOrig;
  ierr=PetscFree(hostDataCopy);CHKERRQ(ierr);

  *AIJ_L = B;
  *AIJ_U = C;

  PetscFunctionReturn(0);
}
EXTERN_C_END
