#include "hip/hip_runtime.h"

/*  -------------------------------------------------------------------- 

     This file extends structgrid data type to make use of GPUS. The new data type
     is structgridgpu. The implementation of the new datatype emulates the seqaijcusp
     implementation which is an extension to aij matrix format. 
     Author: Chekuri S. Choudary, RNET
*/

#define PETSCMAT_DLL
#include "../src/mat/impls/structgrid/structgridgpu/matstructgridgpu.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>
#include <omp.h>
#include "../src/mat/impls/structgrid/matstructgrid.h"

#include "private/matimpl.h"
#include "matstructgridgpu.h"
#include "cuPrintf.cu"

// ----------------------------------------------------------
// hardcodiing the shared memory size this should be set
// to give maximum performance, however should be
// replaced soon with a more flexable dynamically allocated
// shared memory scheme
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
#define SHDSIZE 16


#define size 64
// Structure for Constant Device memory
// storing constants and indices and index limits
// stencile size is hard coded
// written by: dlowell ANL-MCS
// -----------------------------------------------
#define STLSIZE 64
struct Stencilparams{
       int m;
       int n;
       int p;
       int vecsize_x;
       int vecsize_y;
       int matsize;
       int nos;
       int dof;
       int lda1;
       int lda2;
       int lda3;
       int idx[STLSIZE];
       int idy[STLSIZE];
       int idz[STLSIZE];
};//812 bytes

__constant__ Stencilparams devparams;//device memory




// ----------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 





//------------------------------------------------------
// general timer function using unix system call
// dlowell ANL-MCS
//------------------------------------------------------
double getclock(){
      struct timezone tzp;
      struct timeval tp;
      gettimeofday (&tp, &tzp);
      return (tp.tv_sec + tp.tv_usec*1.0e-6);
}


/*  --------------------------------------------------------------------
     This function creates a datatype of structgridgpu. It first creates a
     structgrid datatype and overrides the matrix multiplication method.
     Author: Chekuri S. Choudary, RNET
*/

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode  MatCreate_SeqSGGPU(Mat B)
{

  printf("Call to MatCreate_SeqSGGPU(Mat B)\n");
  PetscErrorCode ierr;
  PetscFunctionBegin;

  ierr             = MatCreate_SeqSG(B);CHKERRQ(ierr);
  B->ops->mult     = MatMult_SeqSGGPU;

  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSTRUCTGRIDGPU);CHKERRQ(ierr);
  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
  PetscFunctionReturn(0);
}
EXTERN_C_END




//---------------------------------------------------------------------
//     This function implements matrix vector multiplication for the
//     structgridgpu datatype. It calls a CUDA kernel to do matrix
//     multiplication on the GPU.
//     Author: Daniel Lowell, ANL-MCS, Chekuri S. Choudary, RNET
//---------------------------------------------------------------------
EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y)
{

        int i;
	PetscErrorCode ierr;
	Mat_SeqSG * a = (Mat_SeqSG *) mat->data;
	PetscScalar * v = a->a, *xx,*yy;

	PetscFunctionBegin;
	ierr = VecSet(y,0.0); CHKERRQ(ierr);
	ierr = VecGetArray(x, &xx); CHKERRQ(ierr);
	ierr = VecGetArray(y, &yy); CHKERRQ(ierr);


        //set up parameters for constant memory
        struct Stencilparams sparams;
        for(i=0;i<a->stpoints;i++){
            sparams.idx[i]=a->idx[i];
            sparams.idy[i]=a->idy[i];
            sparams.idz[i]=a->idz[i];
        }
        sparams.m=a->m;
        sparams.n=a->n;
        sparams.p=a->p;
        VecGetLocalSize(x,&sparams.vecsize_x);
        VecGetLocalSize(y,&sparams.vecsize_y);
        sparams.nos = a->stpoints;
        sparams.dof = a->dof;
        sparams.lda1=a->m*a->n*a->p;
        sparams.lda2=a->m*a->n;
        sparams.lda3=a->m;
        sparams.matsize=a->m*a->n*a->p*a->stpoints;

        /// Debugging block .....................................................
            /*int xsize,ysize;
            printf("Matrix A ::: m: %d, n: %d, p: %d, nos: %d dof: %d nz: %d\n",
                a->m,a->n,a->p,a->stpoints,a->dof,a->nz);
            VecGetLocalSize(x,&xsize);
            VecGetLocalSize(y,&ysize);
            printf("Amat size: %d, Xvec size: %d, Yvec size: %d\n",sparams.matsize,xsize,ysize);
            */
            static PetscInt count = 1;// running count of function calls
            printf("MatMult_SeqSGGPU(Mat mat, Vec x, Vec y): %d\n",count++);
        ///....................................................................


// Call to dlowell's version
        ierr = SGCUDA_MatMult_v2(v,xx,yy,sparams); CHKERRQ(ierr);

// Call to Jeswin's version
//        ierr = SGCUDA_MatMult(v,xx,yy,a->idx,a->idy,a->idz,a->m,a->n,a->p,a->stpoints);
//        CHKERRQ(ierr);

       	ierr = VecRestoreArray(x,&xx); CHKERRQ(ierr);
	ierr = VecRestoreArray(y,&yy); CHKERRQ(ierr);
	ierr = PetscLogFlops(2*a->nz*a->stpoints); CHKERRQ(ierr);
	PetscFunctionReturn(0);
}
EXTERN_C_END







//-------------------------------------------------------------------------------
//   This function is the matrix vector multiplication kernel
//   structgridgpu datatype. This version uses shared memory for the write 
//   back vector Y. Constant memory for reused constants and indices. 
//   More offloading to registers might be possible as well.
//   written by: dlowell, ANL-MCS
//-------------------------------------------------------------------------------
__global__ void MatMul_Kernel_v2(double* A, double* X, double* Y){

   // indices for global accesses
   int btx = blockDim.x*blockIdx.x+threadIdx.x;
   int bty = blockDim.y*blockIdx.y+threadIdx.y;
   int btz = blockDim.z*blockIdx.z+threadIdx.z;
   //cuPrintf("Bxyz: %d %d %d\n",btx,bty,btz);

   //indices for local accesses
   int tx = threadIdx.x;
   int ty = threadIdx.y;
   int tz = threadIdx.z;

   int j;
   int nos  = devparams.nos; // set to register
   int lda1 = devparams.lda1;//  "   "
   int lda2 = devparams.lda2;//  "   "
   int lda3 = devparams.lda3;//  "   "

   int Aindex;
   int Yindex;
   int Xindex;

   // Min shared mem byte count: (gridDim.x*gridDim.y*gridDim.z)*SHDSIZE^3*8 byte double
   __shared__ double Ys[SHDSIZE][SHDSIZE][SHDSIZE];
   Ys[tz][ty][tx]=0;
   __syncthreads();

   for(j=0;j<nos;j++){
       Aindex = j*lda1 + btz*lda2 + bty*lda3 + btx;
       Xindex = (btz+devparams.idz[j])*lda2 + (bty+devparams.idy[j])*lda3 + (btx+devparams.idx[j]);
       __syncthreads();

        if (!((j==1 && tx==devparams.m-1 && ty==devparams.n-1)||
             (j==2 && tx==0 && ty==0)||
             (j==3 && ty==devparams.n-1)||
             (j==4 && ty==0))){
                if(Aindex<devparams.matsize && Xindex < devparams.vecsize_x) {
                   Ys[tz][ty][tx]+=A[Aindex]*X[Xindex];
                }else Ys[tz][ty][tx]+=0.;
       }
       __syncthreads();
   }

   __syncthreads();
   Yindex = btz*lda2+bty*lda3+btx;
   if(Yindex<devparams.vecsize_y) Y[Yindex]=Ys[tz][ty][tx];//global write back
}




//------------------------------------------------------------------------------------
//   This function is the wrapper function which sets up the device memory, transfers
//   data to and from the device, and calls the kernel. Error checking is done at 
//   each step. Timing stats are recorded using static vars.
//   written by: Daniel Lowell, ANL-MCS
//------------------------------------------------------------------------------------
PetscErrorCode SGCUDA_MatMult_v2(PetscScalar* A, PetscScalar* X, PetscScalar* Y, struct Stencilparams P){

        // vars for testing
        //int i;
        static double cumtime=0.;//cummalitive call time
        static unsigned int kcalls=0;//number of kernel calls

        // using CUDA device timer
	float elapsedtime;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	double cs,ce,temp;

	hipError_t cudastatus0,cudastatus1,
	            cudastatus2,cudastatus3,
	            cudastatus4,cudastatus5,
           	    cudastatus6,cudastatus7;


        int matsize =P.matsize*sizeof(double);
        int vecsize_x = P.vecsize_x*sizeof(double);
        int vecsize_y = P.vecsize_y*sizeof(double);

// 	allocate GPU device memory
	cs=getclock();

        // Allocate and Memcpy Structured Matrix A
	double* devA;
	cudastatus0=hipMalloc((void**)&devA,matsize);
	cudastatus1=hipMemcpy(devA,A,matsize,hipMemcpyHostToDevice);
	if(cudastatus0!=hipSuccess|cudastatus1!=hipSuccess){
	  printf("Error in devA memory allocation:\nstatus0: %s, status1: %s\n",
  			hipGetErrorString(cudastatus0),
			hipGetErrorString(cudastatus1));
	  if(devA) hipFree(devA);
          PetscFunctionReturn(PETSC_ERR_MEM);
	}

        // Allocate and Memcpy Vector X
	double* devX;
	cudastatus2=hipMalloc((void**)&devX,vecsize_x);
	cudastatus3=hipMemcpy(devX,X,vecsize_x,hipMemcpyHostToDevice);
	if(cudastatus2!=hipSuccess|cudastatus3!=hipSuccess){
	  printf("Error in devX memory allocation:\nstatus2: %s, status3: %s.\n",
  			hipGetErrorString(cudastatus2),
			hipGetErrorString(cudastatus3));
	  if(devA) hipFree(devA);
	  if(devX) hipFree(devX);
          PetscFunctionReturn(PETSC_ERR_MEM);
	}

        // Allocate and Memset(0.) Vector Y
	double* devY;
	cudastatus4=hipMalloc((void**)&devY,vecsize_y);
	cudastatus5=hipMemset(devY,0.0,vecsize_y);
	if(cudastatus4!=hipSuccess|cudastatus5!=hipSuccess){
	  printf("Error in devY memory allocation:\nstatus4: %s, status5: %s\n",
	  		hipGetErrorString(cudastatus4),
			hipGetErrorString(cudastatus5));
	  if(devA) hipFree(devA);
	  if(devY) hipFree(devY);
          if(devX) hipFree(devX);
          PetscFunctionReturn(PETSC_ERR_MEM);
	}

        // update constant memory with structured grid parameters
	cudastatus6=hipMemcpyToSymbol(HIP_SYMBOL("devparams"),&P,sizeof(Stencilparams));
	if(cudastatus6!=hipSuccess){
	  printf("Error in symbol copy: status6: %s.\n",
	  		hipGetErrorString(cudastatus6));
	  if(devA) hipFree(devA);
	  if(devY) hipFree(devY);
	  if(devX) hipFree(devX);
          PetscFunctionReturn(PETSC_ERR_MEM);
	}


	ce=getclock();
	temp=ce-cs;


        int bx,by,bz;//number of blocks in 3-D
        int tx,ty,tz;//number of threads ber block in 3-D

        //Set up blocks and thread numbers
        if(P.m < SHDSIZE){
           tx = P.m;
           bx = 1;
        }else{
           tx = SHDSIZE;
           bx = ceil((float)P.m/(float)SHDSIZE);
        }

        if(P.n < SHDSIZE){
           ty = P.n;
           by = 1;
        }else{
           ty = SHDSIZE;
           by = ceil((float)P.n/(float)SHDSIZE);
        }

        if(P.p < SHDSIZE){
           tz = P.p;
           bz = 1;
        }else{
           tz = SHDSIZE;
           bz = ceil((float)P.p/(float)SHDSIZE);
        }

        dim3 dimGrid(bx,by,bz);
	dim3 dimBlock(tx,ty,tz);
        //dim3 dimBlock(SHDSIZE,SHDSIZE,1);//testing

        //printf("numblocks xyz: (%d, %d, %d), numthreads: (%d, %d, %d)\n",bx,by,bz,tx,ty,tz);

        cudaPrintfInit();//start cuda printf environ.
	hipEventRecord(start,0);//begin recording kernel
	MatMul_Kernel_v2<<<dimGrid,dimBlock>>>(devA,devX,devY);
        checkCUDAError("CUDA Kernel launch...");//check for failure
        cudaPrintfDisplay(stdout, true);//choose output
        cudaPrintfEnd();//kill cuda printf environ
	hipEventRecord(stop,0);
	hipEventSynchronize(stop); // event barrier
	hipEventElapsedTime(&elapsedtime,start,stop);
        hipEventDestroy(start);
	hipEventDestroy(stop);


        // Copy back Vector Y from Kernel
	cs=getclock();
	cudastatus7=hipMemcpy(Y,devY,vecsize_y,hipMemcpyDeviceToHost);
	if(cudastatus7!=hipSuccess){
          printf("Error on copy back Y, kernel status: %s\nExiting...\n\n",hipGetErrorString(cudastatus7));
	  if(devA) hipFree(devA);
	  if(devY) hipFree(devY);
	  if(devX) hipFree(devX);
          PetscFunctionReturn(PETSC_ERR_MEM);
        }
       // for(i=0;i<P.lda1;i++)printf("Y[%d]: %lf\n",i,Y[i]);//for verification


        //Free device memory
	if(devA) hipFree(devA);
	if(devY) hipFree(devY);
	if(devX) hipFree(devX);

	ce=getclock();
	temp+=ce-cs;
        cumtime+=(elapsedtime/1000)+temp;
        kcalls++;
        printf("Cumilative kernel time (including setup): %lf msec.\n", cumtime);
        printf("Kernel call #: %d, setup+teardown: %f msec., elapsed time: %f msec.\n\n",
                       kcalls,temp,elapsedtime/1000);
        PetscFunctionReturn(0);
}







/*  -------------------------------------------------------------------- 
     The following is a CUDA kernel for matrix vector multiplication on 
     the GPU. The matrix is in a custom layout that facilitates better 
     memory accesses and vectorization. 
     Author: Chekuri S. Choudary, RNET
*/
__global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{
int tx=  blockDim.x * blockIdx.x + threadIdx.x;
int ty=  blockDim.y * blockIdx.y + threadIdx.y;
int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

for (l=0;l<nos;l++)
        {
        xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
        if (l==1 && tx==n-1 && ty==m-1)
        {
        	continue;
        }
        if (l==2 && tx==0 && ty==0)
        {
        	continue;
        }
        if (l==3 && ty==m-1)
        {
        	continue;
        }
        if (l==4 && ty==0)
        {
        	continue;
        }
        for(i=0;i<p;i++)
        	ptr_y[ i*lda2 + ty*lda3 + tx]+= (ptr_coeff[offset + i*lda2 + ty*lda3 +tx] * ptr_x[(i+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
        }
}


int SGCUDA_MatMult(PetscScalar* coeff, PetscScalar* x, PetscScalar* y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

//double tbegin3, tbegin4, tend3, tend4;
PetscInt size_coeff, size_xy, size_id; 
PetscScalar* d_coeff;
PetscScalar* d_x;
PetscScalar* d_y;
PetscInt *d_idx, *d_idy, *d_idz;
PetscInt i,j;

//fprintf(stdout,"%d\t%d\t%d\t%d\n",m,n,p,nos);

//loading the coeff, x, y, idx, idy, idz to device memory

  unsigned int timer1 = 0;
  //cutilCheckError(cutCreateTimer(&timer1));
  //cutilCheckError(cutStartTimer(timer1));

  //  fprintf(stdout,"In SGCUDA_MatMult\n");
	
//tbegin3 = rtclock();
size_coeff=nos*m*n*p*sizeof(PetscScalar);
hipMalloc((void**)&d_coeff,size_coeff);
hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);

size_xy = m*n*p*sizeof(PetscScalar);
hipMalloc((void**)&d_x,size_xy); 
hipMemcpy(d_x, x, size_xy, hipMemcpyHostToDevice);

hipMalloc((void**)&d_y,size_xy); 
hipMemcpy(d_y, y, size_xy, hipMemcpyHostToDevice);

size_id = nos*sizeof(PetscInt);
hipMalloc((void**)&d_idx,size_id); 
hipMemcpy(d_idx, idx, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idy,size_id); 
hipMemcpy(d_idy, idy, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idz,size_id); 
hipMemcpy(d_idz, idz, size_id, hipMemcpyHostToDevice);

//cutilCheckError(cutStopTimer(timer1));
// kernel Configuration
if (m > 16){
dim3 dimBlock(16,16);
dim3 dimGrid((m/16),(n/16));

    // cutilCheckError(cutCreateTimer(&timer));
    // cutilCheckError(cutStartTimer(timer));

MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);

}
else
{
dim3 dimBlock(m,n);
dim3 dimGrid(1,1);
   
    // cutilCheckError(cutCreateTimer(&timer));
    // cutilCheckError(cutStartTimer(timer));

MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);


}

//Cuda Printf
//cudaPrintfInit();

//tbegin4 = rtclock();
// create and start timer
    //unsigned int timer = 0;
    //cutilCheckError(cutCreateTimer(&timer));
    //cutilCheckError(cutStartTimer(timer));

   // check if kernel execution generated and error
    	//cutilCheckMsg("Kernel execution failed");

   // stop and destroy timer
    	//cutilCheckError(cutStopTimer(timer));
		
//tend4 = rtclock();
//Read y from the Device Memory

hipMemcpy(y, d_y, size_xy, hipMemcpyDeviceToHost); 
 
// double time_sec=cutGetTimerValue(timer)/1000;
// double time_sec1=cutGetTimerValue(timer1)/1000;
   
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel : %f; time(sec): %f\n",(2*stpoints*csr_size*csr_size*1.0e-6/time_sec),time_sec);
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel setup time(sec) : %f\n",time_sec1);
    
// cutilCheckError(cutDeleteTimer(timer));
// cutilCheckError(cutDeleteTimer(timer1));
// tend3 = rtclock();
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel with copy time : %f; time: %f\n",2*stpoints*csr_size*csr_size*1.0e-6/(tend3-tbegin3),tend3-tbegin3);
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel : %f; time: %f\n",2*stpoints*csr_size*csr_size*1.0e-6/(tend4-tbegin4),tend4-tbegin4);
  
// printf("\n");
// printf("Matrix cuda y\n");

//  for(i=0;i<m;i++)
//  {
//    for(j=0;j<n;j++) 
//    {
//      printf("%.2f\n",y[i*n+j]);
//    }
//   printf("\n");
//  }


//Free Device Memory
hipFree(d_coeff);
hipFree(d_x);
hipFree(d_y);
hipFree(d_idx);
hipFree(d_idy);
hipFree(d_idz);

return 0;
}




