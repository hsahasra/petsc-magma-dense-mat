#include "hip/hip_runtime.h"

/*  -------------------------------------------------------------------- 

     This file extends structgrid data type to make use of GPUS. The new data type
     is structgridgpu. The implementation of the new datatype emulates the seqaijcusp
     implementation which is an extension to aij matrix format. 
     Author: Chekuri S. Choudary, RNET
*/

#define PETSCMAT_DLL
#include "../src/mat/impls/structgrid/structgridgpu/matstructgridgpu.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "../src/mat/impls/structgrid/matstructgrid.h"

#include "private/matimpl.h"
#include "matstructgridgpu.h"


#define size 64


/*  -------------------------------------------------------------------- 
     This function creates a datatype of structgridgpu. It first creates a 
     structgrid datatype and overrides the matrix multiplication method. 
     Author: Chekuri S. Choudary, RNET
*/

EXTERN_C_BEGIN
#undef __FUNCT__  
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode  MatCreate_SeqSGGPU(Mat B)
{
  PetscErrorCode ierr;
  PetscFunctionBegin;
  
  ierr             = MatCreate_SeqSG(B);CHKERRQ(ierr);
  B->ops->mult     = MatMult_SeqSGGPU;
  
  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSTRUCTGRIDGPU);CHKERRQ(ierr);
  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
  PetscFunctionReturn(0);
}
EXTERN_C_END




/*  -------------------------------------------------------------------- 
     This function implements matrix vector multiplication for the 
     structgridgpu datatype. It calls a CUDA kernel to do matrix 
     multiplication on the GPU.  
     Author: Chekuri S. Choudary, RNET
*/
EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y)
{
	PetscErrorCode ierr;
	Mat_SeqSG * a = (Mat_SeqSG *) mat->data;
	PetscScalar * v = a->a, *xx,*yy;
	
	PetscFunctionBegin;
	ierr = VecSet(y,0.0); CHKERRQ(ierr);
	ierr = VecGetArray(x, &xx); CHKERRQ(ierr);
	ierr = VecGetArray(y, &yy); CHKERRQ(ierr);

ierr = SGCUDA_MatMult(v,xx,yy,a->idx,a->idy,a->idz,a->m,a->n,a->p,a->stpoints); 
CHKERRQ(ierr);

       	ierr = VecRestoreArray(x,&xx); CHKERRQ(ierr);
	ierr = VecRestoreArray(y,&yy); CHKERRQ(ierr);
	ierr = PetscLogFlops(2*a->nz*a->stpoints); CHKERRQ(ierr);
	PetscFunctionReturn(0);
}
EXTERN_C_END


/*  -------------------------------------------------------------------- 
     The following is a CUDA kernel for matrix vector multiplication on 
     the GPU. The matrix is in a custom layout that facilitates better 
     memory accesses and vectorization. 
     Author: Chekuri S. Choudary, RNET
*/
__global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{
int tx=  blockDim.x * blockIdx.x + threadIdx.x;
int ty=  blockDim.y * blockIdx.y + threadIdx.y;
int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

for (l=0;l<nos;l++)
        {
        xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
        if (l==1 && tx==size-1 && ty==size-1)
        {
        	continue;
        }
        if (l==2 && tx==0 && ty==0)
        {
        	continue;
        }
        if (l==3 && ty==size-1)
        {
        	continue;
        }
        if (l==4 && ty==0)
        {
        	continue;
        }
        for(i=0;i<p;i++)
        	ptr_y[ i*lda2 + ty*lda3 + tx]+= (ptr_coeff[offset + i*lda2 + ty*lda3 +tx] * ptr_x[(i+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
        }
}


int SGCUDA_MatMult(PetscScalar* coeff, PetscScalar* x, PetscScalar* y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

//double tbegin3, tbegin4, tend3, tend4;
PetscInt size_coeff, size_xy, size_id; 
PetscScalar* d_coeff;
PetscScalar* d_x;
PetscScalar* d_y;
PetscInt *d_idx, *d_idy, *d_idz;
PetscInt i,j;

fprintf(stdout,"%d\t%d\t%d\t%d\n",m,n,p,nos);

//loading the coeff, x, y, idx, idy, idz to device memory

  unsigned int timer1 = 0;
  //cutilCheckError(cutCreateTimer(&timer1));
  //cutilCheckError(cutStartTimer(timer1));

  fprintf(stdout,"In SGCUDA_MatMult\n");
	
//tbegin3 = rtclock();
size_coeff=nos*m*n*p*sizeof(PetscScalar);
hipMalloc((void**)&d_coeff,size_coeff);
hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);

size_xy = m*n*p*sizeof(PetscScalar);
hipMalloc((void**)&d_x,size_xy); 
hipMemcpy(d_x, x, size_xy, hipMemcpyHostToDevice);

hipMalloc((void**)&d_y,size_xy); 
hipMemcpy(d_y, y, size_xy, hipMemcpyHostToDevice);

size_id = nos*sizeof(PetscInt);
hipMalloc((void**)&d_idx,size_id); 
hipMemcpy(d_idx, idx, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idy,size_id); 
hipMemcpy(d_idy, idy, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idz,size_id); 
hipMemcpy(d_idz, idz, size_id, hipMemcpyHostToDevice);

//cutilCheckError(cutStopTimer(timer1));
// kernel Configuration

dim3 dimBlock(16,16);
dim3 dimGrid((size/16),(size/16));

//Cuda Printf
//cudaPrintfInit();

//tbegin4 = rtclock();
// create and start timer
    unsigned int timer = 0;
    //cutilCheckError(cutCreateTimer(&timer));
    //cutilCheckError(cutStartTimer(timer));

	MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);

   // check if kernel execution generated and error
    	//cutilCheckMsg("Kernel execution failed");

   // stop and destroy timer
    	//cutilCheckError(cutStopTimer(timer));
		
//tend4 = rtclock();
//Read y from the Device Memory

hipMemcpy(y, d_y, size_xy, hipMemcpyDeviceToHost); 
 
// double time_sec=cutGetTimerValue(timer)/1000;
// double time_sec1=cutGetTimerValue(timer1)/1000;
   
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel : %f; time(sec): %f\n",(2*stpoints*csr_size*csr_size*1.0e-6/time_sec),time_sec);
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel setup time(sec) : %f\n",time_sec1);
    
// cutilCheckError(cutDeleteTimer(timer));
// cutilCheckError(cutDeleteTimer(timer1));
// tend3 = rtclock();
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel with copy time : %f; time: %f\n",2*stpoints*csr_size*csr_size*1.0e-6/(tend3-tbegin3),tend3-tbegin3);
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel : %f; time: %f\n",2*stpoints*csr_size*csr_size*1.0e-6/(tend4-tbegin4),tend4-tbegin4);
  
// printf("\n");
// printf("Matrix cuda y\n");

  for(i=0;i<size;i++)
  {
    for(j=0;j<size;j++) 
    {
      printf("%.2f\n",y[i*size+j]);
    }
   printf("\n");
  }
 

//Free Device Memory
hipFree(d_coeff);
hipFree(d_x);
hipFree(d_y);
hipFree(d_idx);
hipFree(d_idy);
hipFree(d_idz);

return 0;
}



