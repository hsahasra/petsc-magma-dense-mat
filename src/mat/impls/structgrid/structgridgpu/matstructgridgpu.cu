#include "hip/hip_runtime.h"
/*  -------------------------------------------------------------------- 

     This file extends structgrid data type to make use of GPUS. The new data type
     is structgridgpu. The implementation of the new datatype emulates the seqaijcusp
     implementation which is an extension to aij matrix format. 
     Author: Chekuri S. Choudary, RNET
*/


#define PETSCMAT_DLL
#include "../src/mat/impls/structgrid/structgridgpu/matstructgridgpu.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>
#include <omp.h>
#include "../src/mat/impls/structgrid/matstructgrid.h"

#include "private/matimpl.h"
#include "matstructgridgpu.h"

#define _DBGFLAG 0
#define PRINT
//block size is 1x256. 
#define BLOCKWIDTH_X 256		
#define BLOCKWIDTH_Y 1   
//block size is 1x256. 
#define BLOCKWIDTH_X 256		
#define BLOCKWIDTH_Y 1   

// ----------------------------------------------------------
// hardcodiing the shared memory size this should be set
// to give maximum performance, however should be
// replaced soon with a more flexable dynamically allocated
// shared memory scheme
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
#define SHDSIZE 4


// -----------------------------------------------
// Structure for Constant Device memory
// storing constants and indices and index limits
// stencile size is hard coded
// written by: dlowell ANL-MCS
// -----------------------------------------------
#define STLSIZE 64
struct Stencilparams{
       int m;
       int n;
       int p;
       int vecsize_x;
       int vecsize_y;
       int matsize;
       int nos;
       int dof;
       int lda1;
       int lda2;
       int lda3;
       int idx[STLSIZE];
       int idy[STLSIZE];
       int idz[STLSIZE];
       int tile_x;
       int tile_y;
       int tile_z;
       int tsizex;
       int tsizey;
       int tsizez;
};//836 bytes

__constant__ Stencilparams devparams;//device memory


static double* devA;
static PetscScalar* d_coeff;
static double* devX;
static double* devY;




// ----------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//------------------------------------------------------
// general timer function using unix system call
// dlowell ANL-MCS
//------------------------------------------------------
double getclock(){
      struct timezone tzp;
      struct timeval tp;
      gettimeofday (&tp, &tzp);
      return (tp.tv_sec + tp.tv_usec*1.0e-6);
}


/*  --------------------------------------------------------------------
     This function destroys the matrix of type structgridgpu. It first 
     deallocates the memory on GPU and then calls the MatDestroy_SeqSG 
     function.
     Author: Chekuri S. Choudary, RNET
*/

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqSGGPU"
PetscErrorCode  MatDestroy_SeqSGGPU(Mat B)
{
  //printf("Call to MatDestroy_SeqSGGPU(Mat B)\n");
  PetscErrorCode ierr;
  PetscFunctionBegin;

  if (B->valid_GPU_matrix != PETSC_CUSP_UNALLOCATED) 
	{
  	if (devA) hipFree(devA);
	if (d_coeff) hipFree(d_coeff);
	if(devY) hipFree(devY);
	if(devX) hipFree(devX);
	}

  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;

  ierr             = MatDestroy_SeqSG(B);CHKERRQ(ierr);

  PetscFunctionReturn(0);
}
EXTERN_C_END



/*  --------------------------------------------------------------------
     This function creates a datatype of structgridgpu. It first creates a
     structgrid datatype and overrides the matrix multiplication method.
     Author: Chekuri S. Choudary, RNET
*/

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode  MatCreate_SeqSGGPU(Mat B)
{

   printf("Call to MatCreate_SeqSGGPU(Mat B)\n");

  PetscErrorCode ierr;
  PetscFunctionBegin;

  ierr             = MatCreate_SeqSG(B);CHKERRQ(ierr);
  B->ops->mult     = MatMult_SeqSGGPU;
  B->ops->destroy  = MatDestroy_SeqSGGPU;

  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSTRUCTGRIDGPU);CHKERRQ(ierr);
  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
  PetscFunctionReturn(0);
}
EXTERN_C_END


//---------------------------------------------------------------------
//     This function implements matrix vector multiplication for the
//     structgridgpu datatype. It calls a CUDA kernel to do matrix
//     multiplication on the GPU.
//     Author: Daniel Lowell, ANL-MCS, Chekuri S. Choudary, RNET
//---------------------------------------------------------------------
EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y)
{
        int i;
	PetscErrorCode ierr;
	Mat_SeqSG * a = (Mat_SeqSG *) mat->data;
	PetscScalar * v = a->a, *xx,*yy;

	PetscFunctionBegin;
	ierr = VecSet(y,0.0); CHKERRQ(ierr);
	ierr = VecGetArray(x, &xx); CHKERRQ(ierr);
	ierr = VecGetArray(y, &yy); CHKERRQ(ierr);

        /* Call to Jeswin's version */
        ierr = SGCUDA_MatMult(v,xx,yy,a->idx,a->idy,a->idz,a->m,a->n,a->p,
                              a->stpoints,&(mat->valid_GPU_matrix),a->dof);CHKERRQ(ierr);

        ierr = VecRestoreArray(x,&xx); CHKERRQ(ierr);
	ierr = VecRestoreArray(y,&yy); CHKERRQ(ierr);
	ierr = PetscLogFlops(2*a->nz*a->stpoints); CHKERRQ(ierr);
	PetscFunctionReturn(0);
}
EXTERN_C_END


/*  -------------------------------------------------------------------- 
     The following is a CUDA kernel for matrix vector multiplication on 
     the GPU. The matrix is in a custom layout that facilitates better 
     memory accesses and vectorization. 
     Author: Chekuri S. Choudary, RNET
*/
/* Version with Shared memory for X only supports rectangular tiles. */
 /* __global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

int tx= blockDim.x * blockIdx.x + threadIdx.x;
int ty= blockDim.y * blockIdx.y + threadIdx.y;

int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

__shared__ PetscScalar y_sm[256];

// initializing to the zero
y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x]=0;
for (l=0;l<nos;l++)
	{
	xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
	if (tx > n-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (ty > m-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (l==1 && tx==n-1 && ty==m-1)
	{
	continue;
	}
	if (l==2 && tx==0 && ty==0)
	{
	continue;
	}
	if (l==3 && ty==m-1)
	{
	continue;
	}
	if (l==4 && ty==0)
	{
	continue;
	}
	for(i=0;i<p;i++)
	y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x]+= (ptr_coeff[offset + i*lda2 + ty*lda3 +tx] * ptr_x[(i+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
	}
	
	ptr_y[ty*lda3 + tx]= y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x];
}
  */
/*  #define BLOCKWIDTH 8
#define BLOCKWIDTH_X 8
#define BLOCKWIDTH_Y 8
#define BLOCKWIDTH_Z 8 
  
  __global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

int tx= blockDim.x * blockIdx.x + threadIdx.x;
int ty= blockDim.y * blockIdx.y + threadIdx.y;
int tz= blockDim.z * blockIdx.z + threadIdx.z;
int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

__shared__ PetscScalar y_sm[512];

// initializing to the zero
y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y + threadIdx.y*BLOCKWIDTH_X + threadIdx.x]=0;
for (l=0;l<nos;l++)
	{
	xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
	if (tx > n-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (ty > m-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (tz > p-1)
	{
	break;
	}
	if (l==1 && tx==n-1 && ty==m-1 && tz==p-1)
	{
	continue;
	}
	if (l==2 && tx==0 && ty==0 && tz==0)
	{
	continue;
	}
	if (l==3 && ty==m-1)
	{
	continue;
	}
	if (l==4 && ty==0)
	{
	continue;
	}
	if (l==5 && tz==p-1)
	{
	continue;
	}
	if (l==6 && tz==0)
	{
	continue;
	}
	//for(i=0;i<p;i++)
	y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y + threadIdx.y*BLOCKWIDTH_X + threadIdx.x]+= (ptr_coeff[offset + tz*lda2 + ty*lda3 +tx] * ptr_x[(tz+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
	}
	
	ptr_y[tz*lda2+ ty*lda3 + tx]= y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y +threadIdx.y*BLOCKWIDTH_X + threadIdx.x];
}
 */   


//------------------------------------------------------------------------------------
//   These functions are used to bind and unbind the Vector x to the texture Memory.	   
//------------------------------------------------------------------------------------ 

texture<int2, 1> tex_x_double;

void unbind_x( double * x)
 {   
 hipUnbindTexture(tex_x_double); 
 }

static __inline__ __device__ double fetch_double(texture<int2, 1> tex_x_double, int i)
{
    int2 v = tex1Dfetch(tex_x_double,i);
    return __hiloint2double(v.y, v.x);
}
 
//------------------------------------------------------------------------------------
// Dynamically allocating Shared Memory size	   
//------------------------------------------------------------------------------------ 
 
extern __shared__ PetscScalar idx_sm[]; 
 
//------------------------------------------------------------------------------------
//  Below functions are SPMV kernel functions where x through the Texture Memory, offsets are accesed
//	through the Shared Memory, Y is accessed per thread from registers. Coeff accesses are from the global Memory 
//	but they are coalesced.  	   
//------------------------------------------------------------------------------------ 
 
#define stpoints 7 //Not a best way of doing this, I have to change this.

__global__ void MatMul_Kernel_tex_1_DOF(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos,PetscInt DOF)
	{
		
		//__shared__ float idx_sm[stpoints];
		
		int tx= blockDim.x * blockIdx.x + threadIdx.x;
		int l,offset;
		int lda1=m*n*p*DOF,lda2=m*p*DOF;  //lda3=m*DOF
		PetscInt Index;
		PetscScalar y_reg=0;
		
		if (threadIdx.x < nos)
			{
			idx_sm[threadIdx.x]=idx[threadIdx.x];
			}
			
		/* if (threadIdx.x < nos && blockIdx.x==0 && blockIdx.y==0 )
			{
			cuPrintf("idx_sm =%f idx=%d\n",idx_sm[threadIdx.x],idx[threadIdx.x]);
			} */
		
		int reg2=blockIdx.y*lda2+tx;
		
		//Iterating through the Diagonals
		for (l=0;l<nos;l++)
			{
			Index =reg2 + idx_sm[l];
				
			if (Index >= 0 && Index <lda1)
				{
				offset = l*lda1;
				y_reg+= ptr_coeff[offset + reg2] * fetch_double(tex_x_double,Index);
				
			
                                /*	  if (threadIdx.y==0){
                                				cuPrintf("l= %d ptr_coeff= %f X= %f Index =%d y_sm=%f \n",l,ptr_coeff[offset + reg2],tex1Dfetch(tex_x_double,Index),Index, y_reg);
                                                                } */ 
		      
				}
			}
							
				
				ptr_y[reg2]= y_reg;
	}

	
__global__ void MatMul_Kernel_tex(double * ptr_coeff, double* ptr_x, double* ptr_y, PetscInt *idx, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos,PetscInt DOF)
	{
		
		//__shared__ float idx_sm[stpoints];
		
		int tx= blockDim.x * blockIdx.x + threadIdx.x;
		int l,i,offset;
		int lda1=m*n*p*DOF,lda2=m*p*DOF; //lda3=m*DOF
		PetscInt X_Index,Index;
		double y_reg=0;
		int BAND_SIZE=(DOF-1)*2+1;
		
		if (threadIdx.x < nos)
			{
			idx_sm[threadIdx.x]=idx[threadIdx.x];
			}
		
		
		int reg2=blockIdx.y*lda2+tx;
		
		//Iterating through the Diagonals
		for (l=0;l<nos;l++)
			{
			X_Index =reg2 + idx_sm[l];
				
			if (X_Index >= 0 && X_Index <lda1)
				{
				for (i=0;i<BAND_SIZE;i++)
					{
					offset = (l*BAND_SIZE+i)*lda1;
									
						if (i > DOF-1)
							{
							Index =X_Index-(i-(DOF-1));
							if (Index < 0)
								{
								continue;
								}
							else{
								y_reg+= ptr_coeff[offset + reg2] * fetch_double(tex_x_double,Index) ;
								}
							}	
						else {
							Index=X_Index+i;
							
							y_reg+= ptr_coeff[offset + reg2] * fetch_double(tex_x_double,Index);
							}
			
				  /* if (threadIdx.y==0){
							cuPrintf("l= %d ptr_coeff= %f X= %f X_Index =%d Index =%d y_sm=%f \n",l,ptr_coeff[offset + reg2],tex1Dfetch(tex_x_double,Index),X_Index,Index, y_reg);
						} */  
					}
				}
			}
							
							
							y_reg+= ptr_coeff[offset + reg2] * fetch_double(tex_x_double,Index);
							}
			
				  /* if (threadIdx.y==0){
							cuPrintf("l= %d ptr_coeff= %f X= %f X_Index =%d Index =%d y_sm=%f \n",l,ptr_coeff[offset + reg2],tex1Dfetch(tex_x_double,Index),X_Index,Index, y_reg);
						} */  
					}
				}
			}
							
				
				ptr_y[reg2]= y_reg;
	}
	

//------------------------------------------------------------------------------------
//   The function is a wrapper function which sets up the device memory, transfers
//   data to and from the device, and calls the MatMult kernel. 
//------------------------------------------------------------------------------------ 

//------------------------------------------------------------------------------------
//   The function is a wrapper function which sets up the device memory, transfers
//   data to and from the device, and calls the MatMult kernel. 
//------------------------------------------------------------------------------------ 
 
int SGCUDA_MatMult(PetscScalar* coeff, PetscScalar* x, PetscScalar* y, PetscInt *idx, PetscInt* idy, 
		   PetscInt* idz, PetscInt m, PetscInt n,PetscInt p, PetscInt nos, PetscCUSPFlag* fp,PetscInt DOF)
{
  double tbegin1, tbegin2, tend1, tend2;
  static PetscInt size_coeff; 
  double tsetup,tkernel;
  static unsigned int kcalls=0;
  PetscInt size_xy, size_id; 
  static double temp=0;
  PetscScalar* d_x;
  PetscScalar* d_y;
  PetscInt *d_idx, *d_idy, *d_idz;
  int BLOCK_SIZE;
  int cons=m*DOF;
  int cons1=m*n*DOF;

    {  
      idx[1]=DOF;
      idx[2]=cons;
      idx[3]=-DOF;
      idx[4]=-cons;
    }
  if (nos==7)
    {  
      idx[0]=0;
      idx[1]=DOF;
      idx[2]=cons;
      idx[3]=cons1;
      idx[4]=-DOF;
      idx[5]=-cons;
      idx[6]=-cons1;
    }
	
  //----Single offset instead of using three offsets int the x,y and z direction.  
  if (nos==5)
    {  
      idx[0]=0;
      idx[1]=DOF;
      idx[2]=cons;
      idx[3]=-DOF;
      idx[4]=-cons;
      // 	}
      if (nos==7)
	{  
	  idx[0]=0;
	  idx[1]=DOF;
	  idx[2]=cons;
	  idx[3]=cons1;
	  idx[4]=-DOF;
	  idx[5]=-cons;
	  idx[6]=-cons1;
	}
for(int s=0;s<DOF*nos;s++)
	{
	printf("\n");	
	for(int i=0;i<n;i++)
		{
		for(int j=0;j<m*DOF;j++) 
			{
			printf("%0.2f   ",coeff[s*cons1+i*cons+j]);
			}
		printf("\n");
		}
	
	}
      //--------------------------------------------------------------------------------
	
	//------------Printing Matrices for Debugging---------------------------------
#ifdef PRINT
	printf("offset vector\n");
	for (int i=0;i<nos;i++)
	  {
	    printf("%d  ", idx[i]);
	  }
	printf("\n");

	printf("Matrix X\n");
	for(int i=0;i<n;i++)
	  {
	    for(int j=0;j<m*DOF;j++) 
	      {
		printf("%0.2f   ",x[i*cons+j]);
	      }
	    //printf("\n");
	  }
	printf("\n");
	printf("Matrix A\n");
	for(int s=0;s<DOF*nos;s++)
	  {
	    printf("\n");	
	    for(int i=0;i<n;i++)
	      {
		for(int j=0;j<m*DOF;j++) 
		  {
		    printf("%0.2f   ",coeff[s*cons1+i*cons+j]);
		  }
		printf("\n");
	      }
		
#if(_DBGFLAG) 
	tbegin1=getclock();
#endif
	  
	if ((*fp == PETSC_CUSP_UNALLOCATED) ||
	    (*fp == PETSC_CUSP_CPU) )
	  {
	    if (*fp == PETSC_CUSP_UNALLOCATED)
	      {
		size_coeff=nos*m*n*p*DOF*sizeof(PetscScalar);	
		hipMalloc((void**)&d_coeff,size_coeff);
	
		//cudastatus0=hipMalloc((void**)&devA,matsize);
		//if(cudastatus0!=hipSuccess)
		//	{
		//  printf("Error in devA memory allocation:\nstatus0: %s\n",
		//	hipGetErrorString(cudastatus0));
		//  PetscFunctionReturn(PETSC_ERR_MEM);
		//	}
	      }
	
	    hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);
	
	    //cudastatus1=hipMemcpy(devA,A,matsize,hipMemcpyHostToDevice);
	    //if(cudastatus1!=hipSuccess)
	    //{
	    // if(devA) hipFree(devA);
	    //  printf("Error in devA memory copying:\nstatus1: %s\n",
	    //	hipGetErrorString(cudastatus1));
	    //  PetscFunctionReturn(PETSC_ERR_MEM);
	    //}
	
	    *fp = PETSC_CUSP_BOTH;
	  }


	//size_coeff=nos*m*n*p*sizeof(PetscScalar);
	//hipMalloc((void**)&d_coeff,size_coeff);
	//hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);


	size_xy = m*n*p*DOF*sizeof(PetscScalar);
	hipMalloc((void**)&d_x,size_xy); 
	hipMemcpy(d_x, x, size_xy, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_y,size_xy); 
	hipMemcpy(d_y, y, size_xy, hipMemcpyHostToDevice);

	size_id = nos*sizeof(PetscInt);
	hipMalloc((void**)&d_idx,size_id); 
	hipMemcpy(d_idx, idx, size_id, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_idy,size_id); 
	hipMemcpy(d_idy, idy, size_id, hipMemcpyHostToDevice);

	hipMalloc((void**)&d_idz,size_id); 
	hipMemcpy(d_idz, idz, size_id, hipMemcpyHostToDevice);

	//Binding X to the texture Memory
	hipBindTexture(0, tex_x_double, d_x, size_xy);

#if(_DBGFLAG)
	cudaPrintfInit();
}

	tend1=getclock();
	tsetup=tend1-tbegin1;
	tbegin2=getclock();
#endif

	// Kernel Setup and Configurations
	
	if ((DOF%2)!=0 || (DOF==6))
	  {
	    BLOCK_SIZE=BLOCKWIDTH_X-BLOCKWIDTH_X%DOF;
	  }
	else{
	  BLOCK_SIZE=BLOCKWIDTH_X;  
	}
		
	dim3 dimBlock(BLOCK_SIZE,BLOCKWIDTH_Y);
	dim3 dimGrid((int)ceil((float)(m*p*DOF)/(float)BLOCK_SIZE),((int)ceil((float)(n)/(float)BLOCKWIDTH_Y)));
				
	if (DOF==1)
	  {
	    MatMul_Kernel_tex_1_DOF<<<dimGrid,dimBlock,nos>>>(d_coeff, d_x, d_y, d_idx, m, n, p, nos,DOF);
	  }
	else{
	  MatMul_Kernel_tex<<<dimGrid,dimBlock,nos>>>(d_coeff, d_x, d_y, d_idx, m, n, p, nos, DOF);
	}
   
	// check if kernel execution generated and error
	//cutilCheckMsg("Kernel execution failed");
			
	/* 
		MatMul_Kernel_tex_1_DOF<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, m, n, p, nos,DOF);
		}
	else{
		MatMul_Kernel_tex<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, m, n, p, nos, DOF);
		}
   
// check if kernel execution generated and error
   //cutilCheckMsg("Kernel execution failed");
			
		/* 
	   if (m > BLOCKWIDTH){
	   // dim3 dimBlock(BLOCKWIDTH,BLOCKWIDTH);
	   // dim3 dimGrid((int)ceil((float)m/(float)BLOCKWIDTH),((int)ceil((float)n/(float)BLOCKWIDTH)));

	   dim3 dimBlock(BLOCKWIDTH,BLOCKWIDTH,BLOCKWIDTH);
	   dim3 dimGrid((int)ceil((float)m/(float)BLOCKWIDTH),((int)ceil((float)n/(float)BLOCKWIDTH)),p/BLOCKWIDTH);

	   // cutilCheckError(cutCreateTimer(&timer));
	   // cutilCheckError(cutStartTimer(timer));

	   MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);

	   }
	   else
	   {
	   //dim3 dimBlock(m,n);
	   dim3 dimBlock(m,n,p);
	   dim3 dimGrid(1,1,1);
		   
	   // cutilCheckError(cutCreateTimer(&timer));
	   // cutilCheckError(cutStartTimer(timer));

	   MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);


	   }

	*/
 

#if(_DBGFLAG) 
	hipDeviceSynchronize();
	tend2=getclock();
	tkernel=tend2-tbegin2;
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
#endif
	
	//Read y from the Device Memory

	hipMemcpy(y, d_y, size_xy, hipMemcpyDeviceToHost); 
	/*
	  int i;
	  char *fn = "/homes/dlowell/cudaexprs/dcheck/outfile_SG1.txt";
	  FILE *fptr;
	  fptr=fopen(fn,"a");
	  if(!fptr){
	  printf("file pointer error.\n");
	  PetscFunctionReturn(PETSC_ERR_LIB);
	  }else{
	  //printf("yy->map->n: %d\n",yy->map->n);
	  for(i=0;i<m*n*p*DOF;i++){
	  //printf("printed to file: %d\n",i);
	  if(y[i]!=0.)fprintf(fptr,"%f ",y[i]);
	  }
	  fclose(fptr);
	  }
	*/

#if(_DBGFLAG)
	temp+=tkernel;
	if (kcalls==0)
	  {
	    printf("\n Structured Grid MatrixMul Kernel Permormance for m *%d* and n size *%d* \n",m,n);
	  }
	if (kcalls==1000)
	  {
	    printf("\ncopy time (sec) : %f\n",tsetup);
	    printf("Kernel time (sec): %f\n",tkernel);
	    printf("Performance in Megaflops with for %dth Kernel call\n",kcalls);
	    printf("Performance in Megaflops with copy time = %f\n",(2*nos*n*m*p*1.0e-6)/(tsetup+tkernel));
	    printf("Performance in Megaflops without copy time = %f\n",(2*nos*n*m*p*1.0e-6)/tkernel);
	    printf("Culmative Performance in Megaflops for *%d* calls without copy time = %f\n",kcalls,(2*nos*n*m*p*1.0e-6)/(temp/(kcalls+1)));
	  }
#endif	
	kcalls++;
#ifdef PRINT
//for(int i=0;i<m*n;i++)
//printf("Y[%d]: %lf\n",i,y[i]);
#endif
	//Free Device Memory
	//hipFree(d_coeff);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_idx);
	hipFree(d_idy);
	hipFree(d_idz);

	return 0;
    }




