#include "hip/hip_runtime.h"
/*  -------------------------------------------------------------------- 
     This file extends structgrid data type to make use of GPUS. The new data type
     is structgridgpu. The implementation of the new datatype emulates the seqaijcusp
     implementation which is an extension to aij matrix format. 
     Author: Chekuri S. Choudary, RNET
             Daniel Lowell, ANL-MCS
*/


#define PETSCMAT_DLL
#include "../src/mat/impls/structgrid/structgridgpu/matstructgridgpu.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>
//#include <omp.h>
#include "../src/mat/impls/structgrid/matstructgrid.h"
#include "private/matimpl.h"
#include "private/vecimpl.h"
#include "matstructgridgpu.h"
#include "../include/private/petscimpl.h"


#define _DBGFLAG 0
#define KERNELVERSION 2

// ----------------------------------------------------------
// hardcodiing the shared memory size this should be set
// to give maximum performance, however should be
// replaced soon with a more flexable dynamically allocated
// shared memory scheme
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
#define SHDSIZE 4


// -----------------------------------------------
// Structure for Constant Device memory
// storing constants and indices and index limits
// stencile size is hard coded
// written by: dlowell ANL-MCS
// -----------------------------------------------
#define STLSIZE 64
struct Stencilparams{
       int m;
       int n;
       int p;
       int vecsize_x;
       int vecsize_y;
       int matsize;
       int nos;
       int dof;
       int lda1;
       int lda2;
       int lda3;
       int idx[STLSIZE];
       int idy[STLSIZE];
       int idz[STLSIZE];
       int tile_x;
       int tile_y;
       int tile_z;
       int tsizex;
       int tsizey;
       int tsizez;
};//836 bytes

__constant__ Stencilparams devparams;//device memory


//static PetscScalar* d_coeff;



// ----------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE);//<-------------------use PETScError handle
  }
}



#undef __FUNCT__
#define __FUNCT__ "MatCheckCUDAError"
PetscErrorCode MatCheckCUDAError(const char *msg) {

  PetscFunctionBegin;
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    fflush(NULL);
    PetscFunctionReturn(PETSC_ERR_LIB);
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "MatCheckCUDAStatus"
PetscErrorCode MatCheckCUDAStatus(hipError_t cs,const char *msg){

  PetscFunctionBegin;
  if(cs!=hipSuccess){
    fprintf(stderr, "Cuda error!: %s: %s.\n",msg,hipGetErrorString(cs));
    fflush(NULL);
    PetscFunctionReturn(PETSC_ERR_LIB);
  }
  PetscFunctionReturn(0);
}












//------------------------------------------------------
// general timer function using unix system call
// dlowell ANL-MCS
//------------------------------------------------------
double getclock(){
  struct timezone tzp;
  struct timeval tp;
  gettimeofday (&tp, &tzp);
  return (tp.tv_sec + tp.tv_usec*1.0e-6);
}


/*  --------------------------------------------------------------------
     This function destroys the matrix of type structgridgpu. It first 
     deallocates the memory on GPU and then calls the MatDestroy_SeqSG 
     function.
     Author: Chekuri S. Choudary, RNET
*/

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqSGGPU"
PetscErrorCode  MatDestroy_SeqSGGPU(Mat B){
  PetscFunctionBegin;
  printf("MatDestroy_SeqSGGPU(Mat B)\n");
  PetscErrorCode ierr;
  hipError_t cudastatus;
  Mat_SeqSG* b=(Mat_SeqSG*)B->data;

  if (b->syncState != MAT_UNALLOC){
    //if (d_coeff) hipFree(d_coeff);
    if(b->devptr){
      cudastatus = hipFree(b->devptr); /* if(devX) hipFree(devX); if(devY) hipFree(devY); */
      ierr = MatCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr); 
      b->devptr=PETSC_NULL;
    }
  }
  b->syncState = MAT_UNALLOC;
  ierr = MatDestroy_SeqSG(B);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
EXTERN_C_END


/*  --------------------------------------------------------------------
     This function creates a datatype of structgridgpu. It first creates a
     structgrid datatype and overrides the matrix multiplication method.
     Author: Chekuri S. Choudary, RNET
*/
EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode  MatCreate_SeqSGGPU(Mat B){
  PetscFunctionBegin;
  printf("MatCreate_SeqSGGPU(Mat B)\n");
  PetscErrorCode ierr;
  //hipError_t cudastatus;
  ierr             = MatCreate_SeqSG(B);CHKERRQ(ierr);
  B->ops->mult     = MatMult_SeqSGGPU;
  B->ops->destroy  = MatDestroy_SeqSGGPU;
  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSTRUCTGRIDGPU);CHKERRQ(ierr);
  /* Allocate device memory for matrix A */
  Mat_SeqSG *b = (Mat_SeqSG *) B->data;
  b->syncState = MAT_UNALLOC;
  PetscFunctionReturn(0);
}
EXTERN_C_END


/* --------------------------------------------------------------------
//     This function implements matrix vector multiplication for the
//     structgridgpu datatype. It calls a CUDA kernel to do matrix
//     multiplication on the GPU.
//     Author: Daniel Lowell, ANL-MCS, Chekuri S. Choudary, RNET
//--------------------------------------------------------------------- */
EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y){
  // int i;
  PetscErrorCode ierr;
  Mat_SeqSG *a=(Mat_SeqSG *) mat->data;
  /* PetscScalar *v, *xx,*yy; */
  PetscFunctionBegin;
  if(KERNELVERSION==1){
    /* Call to Jeswin's version */
    /* v = a->a;
       ierr = VecSet(y,0.0); CHKERRQ(ierr);
       ierr = VecGetArray(x, &xx); CHKERRQ(ierr);
       ierr = VecGetArray(y, &yy); CHKERRQ(ierr);
       ierr = SGCUDA_MatMult(v,xx,yy,a->idx,a->idy,
                            a->idz,a->m,a->n,a->p,a->stpoints,
                            &(mat->valid_GPU_matrix));
       CHKERRQ(ierr);
    ierr = SGCUDA_MatMult_v2(v,xx,yy,sparams,&(mat->valid_GPU_matrix));

       ierr = VecRestoreArray(x,&xx); CHKERRQ(ierr);
       ierr = VecRestoreArray(y,&yy); CHKERRQ(ierr); */
  }else if(KERNELVERSION==2){
    /* Call to dlowell's version */
    ierr = SGCUDA_MatMult_v2(mat,x,y); CHKERRQ(ierr);
  }
  ierr = PetscLogFlops(a->nz*a->stpoints); CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
EXTERN_C_END


/* ------------------------------------------------------------------------
//   This function is the matrix vector multiplication kernel
//   structgridgpu datatype. This version uses shared memory for the write
//   back vector Y. Constant memory for reused constants and indices.
//   More offloading to registers might be possible as well.
//   written by: dlowell, ANL-MCS
//------------------------------------------------------------------------- */
EXTERN_C_BEGIN
__global__ void MatMul_Kernel_v2(double* A, double* X, double* Y){
  /* indices for local accesses */
  int tbtx, tbty, tbtz;
  int ix,iy,iz,j;

  int nos  = devparams.nos; /* set to register */
  int lda1 = devparams.lda1;
  int lda2 = devparams.lda2;
  int lda3 = devparams.lda3;
  int tilex = devparams.tile_x*devparams.tsizex;
  int tiley = devparams.tile_y*devparams.tsizey;
  int tilez = devparams.tile_z*devparams.tsizez;
  int Aindex, Xindex, index, offset;
 
  __shared__ double Ys[SHDSIZE][SHDSIZE][SHDSIZE];
  __shared__ double As[SHDSIZE][SHDSIZE][SHDSIZE];

  /* ------------------------------------------------------------------------ */

  for(iz=0;iz<tilez;iz+=devparams.tsizez){/*  tiles Z loop */
    tbtz = blockDim.z*blockIdx.z+threadIdx.z + iz;

    for(iy=0;iy<tiley;iy+=devparams.tsizey){/*  tiles Y loop */
      tbty = blockDim.y*blockIdx.y+threadIdx.y + iy;

      for(ix=0;ix<tilex;ix+=devparams.tsizex){/*  tiles X loop */
        tbtx = blockDim.x*blockIdx.x+threadIdx.x + ix;

        /* initialize current return-tile */
        Ys[threadIdx.z][threadIdx.y][threadIdx.x]=0.;

        /* adjusted index for global access */
        index = tbtz*lda2 + tbty*lda3 + tbtx;

        /* ......STENCIL........................................... */
        for(j=0;j<nos;j++){/* loop over stencil pattern */
	   offset= j*lda1;
           Aindex=offset+index;/* set up Aindex and read from global A into As tile */
           if(Aindex<devparams.matsize){
             if(A[Aindex]>0.)printf("A[%d]: %e\n",Aindex,A[Aindex]);
              As[threadIdx.z][threadIdx.y][threadIdx.x]=A[Aindex];//needs to be coalesced
           }else{
              As[threadIdx.z][threadIdx.y][threadIdx.x]=0.;
           }

           __syncthreads();

           /* set up Xindex for element-wise operation using stencil pattern */
           Xindex=(devparams.idz[j]*lda2 + devparams.idy[j]*lda3 + devparams.idx[j]) + index;
           if(Xindex<devparams.vecsize_x && Xindex>=0){
              Ys[threadIdx.z][threadIdx.y][threadIdx.x]+=As[threadIdx.z][threadIdx.y][threadIdx.x]*X[Xindex];
           }

        }/* end j-for */

        if(index<devparams.vecsize_y){
           Y[index]=Ys[threadIdx.z][threadIdx.y][threadIdx.x];/* global write back */
           if(Y[index]!=0.)printf("YOUT[%d]: %e\n",index,Y[index]);
        }

   }//end ix-for
   }//end iy-for
   }//end iz-for
}//end kernel_v2
EXTERN_C_END



/* ---------------------------------------------------------------------------------
//   This function is the wrapper function which sets up the device memory, transfers
//   data to and from the device, and calls the kernel. Error checking is done at 
//   each step. Timing stats are recorded using static vars.
//   written by: Daniel Lowell, ANL-MCS
//---------------------------------------------------------------------------------- */
EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "SGCUDA_MatMult_v2"
PetscErrorCode SGCUDA_MatMult_v2(Mat mat, Vec x,Vec y){

  PetscErrorCode ierr;
  PetscFunctionBegin;
  printf("Start SGCUDA_MatMult_v2\n");
  PetscInt i;
  Mat_SeqSG *a = (Mat_SeqSG *) mat->data;
  PetscScalar *A = a->a;
  Vec_SeqGPU* xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU* yd=(Vec_SeqGPU*)y->data;
  PetscScalar *X,*Y;

  static struct Stencilparams sparams;
  static double cumktime=0.;/* cummalitive kernel time */
  static double cumtime=0.;/* cummalitive call time */
  static unsigned int kcalls=0;/* number of kernel calls */
  double cs,ce,temp;
  float elapsedtime;        /* using CUDA device timer */
  hipEvent_t start,stop;
  static unsigned char allocflag = 1;
  static double maxshared;
  static int bx,by,bz;/* number of blocks in 3-D */
  static int tx,ty,tz;/* number of threads ber block in 3-D */
  static int maxblocks_xy;
  static int maxblocks_z;
  static int xytile;

  static dim3 dimGrid;
  static dim3 dimBlock;
  hipError_t cudastatus;



  if(_DBGFLAG){/* create CUDA events for timer */
     hipEventCreate(&start);
     hipEventCreate(&stop);
     cs=getclock();
  }

  /* Allocate and Memcpy Structured Matrix A
     The matrix remains the same throughout one iteration
     of the linear solver. The following uses a flag
     defined in the base class to check the status of the
     matrix. The matrix is copied to the GPU only if
     it has been changed on the CPU side
     This feature added by Chekuri S. Choudary */
  if(a->syncState == MAT_UNALLOC){  /* Allocate device memory for matrix A */
    printf("Allocating matrix A on GPU.\n");
    cudastatus=hipSuccess;
    cudastatus=hipMalloc((void**)&(a->devptr),a->matsize*sizeof(PetscScalar));
    ierr = MatCheckCUDAStatus(cudastatus,"a->devptr alloc in MatCreate_SeqSGGPU");CHKERRQ(ierr); 
    a->syncState = MAT_ALLOC;
  }

  if(A && a->syncState == MAT_ALLOC){
    a->syncState = MAT_CPU;
  }else if(!A){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"Matrix unallocated.");
  }

  if(a->syncState == MAT_CPU){
    /* copy over values of A to device memory */
    printf("Copying A to gpu. Size: %d\n",a->matsize*sizeof(PetscScalar));
    cudastatus=hipSuccess;
    cudastatus=hipMemcpy(a->devptr,A,a->matsize*sizeof(PetscScalar),hipMemcpyHostToDevice);
    ierr = MatCheckCUDAStatus(cudastatus,"devA copy to device in SGCUDA_MatMult_v2");CHKERRQ(ierr);
    a->syncState = MAT_SYNCHED;
  }

  /* ..................................................... */

   if(xd->syncState==VEC_UNALLOC){
     SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"VectorX unallocated.");
   }else if(xd->syncState==VEC_CPU){
     /* copy over values of X to device memory */
     printf("Copying X to gpu.\n");
     ierr = VecGetArray(x,&X); CHKERRQ(ierr);
     ierr = VecRestoreArray(x,&X); CHKERRQ(ierr);
     xd->syncState = VEC_GPU;
   }


   /* ..................................................... */
   if(yd->syncState==VEC_UNALLOC){
     SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"VectorY unallocated.");
   }

   /* memset to 0. Vector Y on device */
   printf("Memset Y to gpu.\n");
   ierr = VecSet(y,0);CHKERRQ(ierr);
   printf("Done vector setup.\n");

   /* ..................................................... */

   if(allocflag){
     /* Set up blocks and thread numbers */
     maxshared = 49152.0/(float)(2.0*sizeof(double));
     if(a->p==1){
       xytile = pow(maxshared,0.5);/* square blocks */
       maxblocks_z = 1;
     }else{
       temp=maxshared/a->p;/* lop off z */
       xytile = pow(temp,0.5);/* xyblocks */
       maxblocks_z=ceil((float)SHDSIZE/(float)a->p);
     }
     maxblocks_xy = xytile/SHDSIZE;


     /* Set up blocks and thread numbers for columns */
     if(a->m <= SHDSIZE){
       tx = a->m; bx = 1; sparams.tile_x = 1; sparams.tsizex=1;
     }else{
       tx = SHDSIZE;
       bx = ceil((float)a->m/(float)SHDSIZE);/* create enough blocks */
       if(bx>maxblocks_xy){                 /* too many blocks created */
         bx = maxblocks_xy;                 /* set to max number of blocks allowed */
         sparams.tile_x=ceil((float)a->m/(float)(bx*SHDSIZE)); /* number of tiles */
         sparams.tsizex=bx*SHDSIZE;               /* tilesize is block-thread coverage */
       }else{
         sparams.tile_x=1; sparams.tsizex=1;
       }
     }

     /* Set up blocks and thread numbers for rows */
     if(a->n <= SHDSIZE){
       ty = a->n; by = 1; sparams.tile_y = 1; sparams.tsizey=1;
     }else{
       ty = SHDSIZE;
       by = ceil((float)a->n/(float)SHDSIZE);
       if(by > maxblocks_xy){
         by = maxblocks_xy;
         sparams.tile_y=ceil((float)a->n/(float)(by*SHDSIZE));
         sparams.tsizey=by*SHDSIZE;
       }else{
         sparams.tile_y=1; sparams.tsizey=1;
       }
     }

     /* Set up blocks and thread numbers for z */
     if(a->p <= SHDSIZE){
       tz = a->p;
       bz = 1;
       sparams.tile_z = 1;
       sparams.tsizez=1;
     }else{
       tz = SHDSIZE; bz = ceil((float)a->p/(float)SHDSIZE);
       if(bz > maxblocks_z){
         bz = maxblocks_z;
         sparams.tile_z=ceil((float)a->p/(float)(bz*SHDSIZE));
         sparams.tsizez=bz*SHDSIZE;
       }else{
         sparams.tile_z=1; sparams.tsizez=1;
       }
     }

     /* set grid shape */
     dimGrid.x = bx; dimGrid.y = by; dimGrid.z = bz;
     //dimGrid.x = 1; dimGrid.y = 1; dimGrid.z = 1;// bz;

     /* set block shape */
     dimBlock.x = tx; dimBlock.y = ty; dimBlock.z = tz;
     //dimBlock.x = 1; dimBlock.y = 1; dimBlock.z = 1;// tz;


     for(i=0;i<a->stpoints;i++){
       sparams.idx[i]=a->idx[i];
       sparams.idy[i]=a->idy[i];
       sparams.idz[i]=a->idz[i];
     }
     sparams.m=a->m;
     sparams.n=a->n;
     sparams.p=a->p;
     VecGetLocalSize(x,&sparams.vecsize_x);
     VecGetLocalSize(y,&sparams.vecsize_y);
     sparams.nos = a->stpoints;
     sparams.dof = a->dof;
     sparams.lda3=a->lda3;
     sparams.lda2=a->lda2;
     sparams.lda1=a->lda1;
     sparams.matsize=a->matsize;


     /* update constant memory with structured grid parameters */
     cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("devparams"),&sparams,sizeof(Stencilparams));
     ierr = MatCheckCUDAStatus(cudastatus,"symbol copy to device in SGCUDA_MatMult_v2");CHKERRQ(ierr);

     /* toggle off allocation flag */
     allocflag = 0;

   }/* end allocflag-if */


   /* grid and block shape & device config. debugging.................................. */
   unsigned int sharebytes;
   static unsigned char dbgflag = 1;
   if(_DBGFLAG && dbgflag){
     sharebytes = 2*tx*ty*tz*bx*by*bz*sizeof(double);
     printf("VecsizeX: %d, VecsizeY: %d, Matsize: %d\n",sparams.vecsize_x,sparams.vecsize_y,sparams.matsize);
     printf("(m, n, p, nos): (%d, %d, %d, %d)\n",sparams.m,sparams.n,sparams.p,sparams.nos);
     printf("MAXBLOCKS: %d maxshared: %lf\n",maxblocks_xy+maxblocks_z,maxshared);
     printf("blocks: (%d, %d, %d), threads per block: %d\n", bx,by,bz,tx*ty*tz );
     printf("Shared elements occupied: %0.3f SharedOccupied in Bytes: %d\n",sharebytes/49152.0,sharebytes);

     printf("Blocks x,y,z: (%d, %d, %d)\n",dimGrid.x,dimGrid.y,dimGrid.z);
     printf("Threads x,y,z: (%d, %d, %d)\n",dimBlock.x,dimBlock.y,dimBlock.z);

     printf("Blocks*ThreadsPer size x,y,z: (%d, %d, %d)\n",bx*tx,by*ty,bz*tz);
     printf("Tiles (x,y,z): (%d, %d, %d)\n",sparams.tile_x,sparams.tile_y,sparams.tile_z);
     printf("Tile Size (x,y,z): (%d, %d, %d)\n",sparams.tsizex,sparams.tsizey,sparams.tsizez);
     //dbgflag=0;
   }
   /* ...End config. debug section...................................................... */

   if(_DBGFLAG){//init kernel timer and debug settings
     ce=getclock();//end setup timer
     temp=ce-cs;
     // cudaPrintfInit();//start cuda printf environ.
     hipEventRecord(start,0);//begin recording kernel
   }

   /*   printf("Launching kernel.\n");
   if(!a->devptr){
     printf("Hey! No a->devptr allocated!\n");
   } */


   /* Launch the kernel.......................................... */
   MatMul_Kernel_v2<<<dimGrid,dimBlock>>>(a->devptr,xd->devptr,yd->devptr);
   ierr = MatCheckCUDAError("CUDA Kernel launch status"); CHKERRQ(ierr);/* check for failure */
   /* ........................................................... */

   xd->syncState = VEC_GPU;
   yd->syncState = VEC_GPU;
   hipDeviceSynchronize();

   if(_DBGFLAG){//end kernel timer and debug settings
     // cudaPrintfDisplay(stdout, true);//choose output
     // cudaPrintfEnd();//kill cuda printf environ
     hipEventRecord(stop,0);
     hipEventSynchronize(stop); // event barrier
     hipEventElapsedTime(&elapsedtime,start,stop);
     hipEventDestroy(start);
     hipEventDestroy(stop);
     cs=getclock();
   }
   //ierr = VecRestoreArray(x,PETSC_NULL); CHKERRQ(ierr);
   //ierr = VecRestoreArray(y,PETSC_NULL); CHKERRQ(ierr);
   // Copy back Vector Y from Kernel
   /*if(y->valid_GPU_array==PETSC_CUSP_GPU){
     ierr = VecGetArray(y, &Y); CHKERRQ(ierr);
     cudastatus7=hipMemcpy(Y,devY,vecsize_y,hipMemcpyDeviceToHost);
     if(cudastatus7!=hipSuccess){
       printf("Error on copy back Y, kernel status: %s\nExiting...\n\n",hipGetErrorString(cudastatus7));
       if(devA) hipFree(devA);
       if(devY) hipFree(devY);
       if(devX) hipFree(devX);
       PetscFunctionReturn(PETSC_ERR_MEM);
       }
     y->valid_GPU_array=PETSC_CUSP_BOTH;
     ierr = VecRestoreArray(y,&Y); CHKERRQ(ierr);
     ierr = PetscLogFlops(a->nz*a->stpoints); CHKERRQ(ierr);
   }*/

   if(_DBGFLAG){//final timer and debug settings
     //for(i=0;i<P.vecsize_y;i++)printf("Y[%d]: %lf\n",i,Y[i]);//for verification
     ce=getclock();
     temp+=ce-cs;
     cumktime+=(elapsedtime/1000);
     cumtime+=(elapsedtime/1000)+temp;
     kcalls++;
     printf("Kernel call #: %d\n",kcalls);
     printf("setup+copyback: %f sec.\nelapsed time: %f sec.\ntotal call time: %f sec.\n",
            temp,elapsedtime/1000,(elapsedtime/1000)+temp);
     printf("Cum. kernel time: %lf sec.\n", cumktime);
     printf("Cum. call time (with setup): %lf sec.\n", cumtime);
     printf(".........................................\n\n");
   }//end _DBGFLAG-if

   printf("returning from SGCUDA_MatMult_v2()\n");
   PetscFunctionReturn(0);
}
EXTERN_C_END








/*  -------------------------------------------------------------------- 
     The following is a CUDA kernel for matrix vector multiplication on 
     the GPU. The matrix is in a custom layout that facilitates better 
     memory accesses and vectorization. 
     Author: Chekuri S. Choudary, RNET
*/


 //Version with Shared memory for X only supports rectangular tiles.
 /* __global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

int tx= blockDim.x * blockIdx.x + threadIdx.x;
int ty= blockDim.y * blockIdx.y + threadIdx.y;

int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

__shared__ PetscScalar y_sm[256];

// initializing to the zero
y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x]=0;
for (l=0;l<nos;l++)
	{
	xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
	if (tx > n-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (ty > m-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (l==1 && tx==n-1 && ty==m-1)
	{
	continue;
	}
	if (l==2 && tx==0 && ty==0)
	{
	continue;
	}
	if (l==3 && ty==m-1)
	{
	continue;
	}
	if (l==4 && ty==0)
	{
	continue;
	}
	for(i=0;i<p;i++)
	y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x]+= (ptr_coeff[offset + i*lda2 + ty*lda3 +tx] * ptr_x[(i+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
	}
	
	ptr_y[ty*lda3 + tx]= y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x];
}
  */
 #define BLOCKWIDTH 8
#define BLOCKWIDTH_X 8
#define BLOCKWIDTH_Y 8
#define BLOCKWIDTH_Z 8 


/*
  
  __global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

int tx= blockDim.x * blockIdx.x + threadIdx.x;
int ty= blockDim.y * blockIdx.y + threadIdx.y;
int tz= blockDim.z * blockIdx.z + threadIdx.z;
int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

__shared__ PetscScalar y_sm[512];

// initializing to the zero
y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y + threadIdx.y*BLOCKWIDTH_X + threadIdx.x]=0;
for (l=0;l<nos;l++)
	{
	xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
	if (tx > n-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (ty > m-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (tz > p-1)
	{
	break;
	}
	if (l==1 && tx==n-1 && ty==m-1 && tz==p-1)
	{
	continue;
	}
	if (l==2 && tx==0 && ty==0 && tz==0)
	{
	continue;
	}
	if (l==3 && ty==m-1)
	{
	continue;
	}
	if (l==4 && ty==0)
	{
	continue;
	}
	if (l==5 && tz==p-1)
	{
	continue;
	}
	if (l==6 && tz==0)
	{
	continue;
	}
	//for(i=0;i<p;i++)
	y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y + threadIdx.y*BLOCKWIDTH_X + threadIdx.x]+= (ptr_coeff[offset + tz*lda2 + ty*lda3 +tx] * ptr_x[(tz+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
	}
	
	ptr_y[tz*lda2+ ty*lda3 + tx]= y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y +threadIdx.y*BLOCKWIDTH_X + threadIdx.x];
}
   
  

 
 __global__ void MatMul_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

int tx= blockDim.x * blockIdx.x + threadIdx.x;
int ty= blockDim.y * blockIdx.y + threadIdx.y;
int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;
__shared__ PetscScalar y_sm[256];
__shared__ PetscScalar x_sm[324];

//initializing to the zero

// copying a Tile from Y into the shared Memory
y_sm[threadIdx.y*BLOCKWIDTH + threadIdx.x]=0;

//Copying a tile x into the shared Memory with 2 steps.

//Copying without the Ghost Cells  
x_sm[(threadIdx.y+1)*(BLOCKWIDTH+2) + (threadIdx.x+1)]=ptr_x[ty*lda3 + tx];

//Copying the Ghost Cells
// if (tx!=0)
// {
// if (threadIdx.x==0)
// x_sm[(threadIdx.y+1)*(BLOCKWIDTH+2) + threadIdx.x]=ptr_x[ty*lda3 + tx-1];
// }

// if (ty!=0)
// {
// if (threadIdx.y==0)
// x_sm[(threadIdx.y)* (BLOCKWIDTH+2) + threadIdx.x+1]=ptr_x[(ty-1)*lda3 + tx];
// }

// if (tx != n-1) // not sure about this
// {
// if (threadIdx.x==BLOCKWIDTH-1)
// x_sm[(threadIdx.y+1)*(BLOCKWIDTH+2) + threadIdx.x + 2]=ptr_x[ty*lda3 + tx + 1];
// }

// if (ty != m-1)
// {
// if (threadIdx.y==BLOCKWIDTH-1)
// x_sm[(threadIdx.y+2)*(BLOCKWIDTH+2) + threadIdx.x +1]=ptr_x[(ty+1)*lda3 + tx];
// }

//Copying the Ghost Cells
if (tx!=0 || ty!=0 || tx != n-1 || ty != m-1 || !(tx > n-1) || !(ty > m-1))
{
if (threadIdx.x==0)
x_sm[(threadIdx.y+1)*(BLOCKWIDTH+2) + threadIdx.x]=ptr_x[ty*lda3 + tx-1];

if (threadIdx.y==0)
x_sm[(threadIdx.y)* (BLOCKWIDTH+2) + threadIdx.x+1]=ptr_x[(ty-1)*lda3 + tx];

if (threadIdx.x==BLOCKWIDTH-1)
x_sm[(threadIdx.y+1)*(BLOCKWIDTH+2) + threadIdx.x + 2]=ptr_x[ty*lda3 + tx + 1];

if (threadIdx.y==BLOCKWIDTH-1)
x_sm[(threadIdx.y+2)*(BLOCKWIDTH+2) + threadIdx.x +1]=ptr_x[(ty+1)*lda3 + tx];
}
__syncthreads();

// if (tx==2 && ty==2)
// {
// cuPrintf("\nPrinting the X from Shared Memory \n ");

// for (int j=0;j<324;j++)
// {
// if(j % 16 ==0)
// {
// cuPrintf("\n");
// }
// cuPrintf("%f  ",  x_sm[j]);
// }
// }
//MATMUL
for (l=0;l<nos;l++)
	{
	xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
	if (tx > n-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (ty > m-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (l==1 && tx==n-1 && ty==m-1)
	{
	continue;
	}
	if (l==2 && tx==0 && ty==0)
	{
	continue;
	}
	if (l==3 && ty==m-1)
	{
	continue;
	}
	if (l==4 && ty==0)
	{
	continue;
	}
	for(i=0;i<p;i++)
	y_sm[threadIdx.y*BLOCKWIDTH + threadIdx.x]+= (ptr_coeff[offset + i*lda2 + ty*lda3 +tx] * x_sm[(i+zdisp)*lda2 + (threadIdx.y+ydisp +1)*
	(BLOCKWIDTH+2) + (threadIdx.x+xdisp+1)]); //forgetting Z currently.. I have to Fix it.
	}
	//removing i tempararily
	ptr_y[ty*lda3 + tx]= y_sm[threadIdx.y*BLOCKWIDTH + threadIdx.x];
}
 

 
 
 int SGCUDA_MatMult(PetscScalar* coeff, PetscScalar* x, PetscScalar* y, PetscInt *idx, PetscInt* idy, 
PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos, PetscCUSPFlag* fp)
{

double tbegin1, tbegin2, tend1, tend2;
static PetscInt size_coeff; 
double tsetup,tkernel;
static unsigned int kcalls=0;
PetscInt size_xy, size_id; 
static double temp=0;
PetscScalar* d_x;
PetscScalar* d_y;
PetscInt *d_idx, *d_idy, *d_idz;

  //unsigned int timer1 = 0;
  //cutilCheckError(cutCreateTimer(&timer1));
  //cutilCheckError(cutStartTimer(timer1));

  //  fprintf(stdout,"In SGCUDA_MatMult\n");
	
      if(_DBGFLAG) tbegin1=getclock();
	  
	  if ((*fp == PETSC_CUSP_UNALLOCATED) ||
	  (*fp == PETSC_CUSP_CPU) )
	{
		if (*fp == PETSC_CUSP_UNALLOCATED)
		{
		size_coeff=nos*m*n*p*sizeof(PetscScalar);	
		hipMalloc((void**)&d_coeff,size_coeff);
	
	   	//cudastatus0=hipMalloc((void**)&devA,matsize);
	   	//if(cudastatus0!=hipSuccess)
		//	{
		//  printf("Error in devA memory allocation:\nstatus0: %s\n",
  		//	hipGetErrorString(cudastatus0));
          	//  PetscFunctionReturn(PETSC_ERR_MEM);
		//	}
		}
	
		hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);
	
           	//cudastatus1=hipMemcpy(devA,A,matsize,hipMemcpyHostToDevice);
	   	//if(cudastatus1!=hipSuccess)
		//{
		// if(devA) hipFree(devA);
		//  printf("Error in devA memory copying:\nstatus1: %s\n",
  		//	hipGetErrorString(cudastatus1));
          	//  PetscFunctionReturn(PETSC_ERR_MEM);
		//}
	
	        *fp = PETSC_CUSP_BOTH;
	}


//size_coeff=nos*m*n*p*sizeof(PetscScalar);
//hipMalloc((void**)&d_coeff,size_coeff);
//hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);


size_xy = m*n*p*sizeof(PetscScalar);
hipMalloc((void**)&d_x,size_xy); 
hipMemcpy(d_x, x, size_xy, hipMemcpyHostToDevice);

hipMalloc((void**)&d_y,size_xy); 
hipMemcpy(d_y, y, size_xy, hipMemcpyHostToDevice);

size_id = nos*sizeof(PetscInt);
hipMalloc((void**)&d_idx,size_id); 
hipMemcpy(d_idx, idx, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idy,size_id); 
hipMemcpy(d_idy, idy, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idz,size_id); 
hipMemcpy(d_idz, idz, size_id, hipMemcpyHostToDevice);

if(_DBGFLAG) 
	{
		tend1=getclock();
		tsetup=tend1-tbegin1;
		tbegin2=getclock();
	}

//cutilCheckError(cutStopTimer(timer1));
// kernel Configuration
if (m > BLOCKWIDTH){
// dim3 dimBlock(BLOCKWIDTH,BLOCKWIDTH);
// dim3 dimGrid((int)ceil((float)m/(float)BLOCKWIDTH),((int)ceil((float)n/(float)BLOCKWIDTH)));

dim3 dimBlock(BLOCKWIDTH,BLOCKWIDTH,BLOCKWIDTH);
dim3 dimGrid((int)ceil((float)m/(float)BLOCKWIDTH),((int)ceil((float)n/(float)BLOCKWIDTH)),p/BLOCKWIDTH);

    // cutilCheckError(cutCreateTimer(&timer));
    // cutilCheckError(cutStartTimer(timer));

MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);

}
else
{
//dim3 dimBlock(m,n);
dim3 dimBlock(m,n,p);
dim3 dimGrid(1,1,1);
   
    // cutilCheckError(cutCreateTimer(&timer));
    // cutilCheckError(cutStartTimer(timer));

MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);


}

//Cuda Printf
//cudaPrintfInit();

//tbegin4 = rtclock();
// create and start timer
    //unsigned int timer = 0;
    //cutilCheckError(cutCreateTimer(&timer));
    //cutilCheckError(cutStartTimer(timer));

	if(_DBGFLAG) 
	{
		hipDeviceSynchronize();
		tend2=getclock();
		tkernel=tend2-tbegin2;
	}

	
	
   // check if kernel execution generated and error
    	//cutilCheckMsg("Kernel execution failed");

   // stop and destroy timer
    	//cutilCheckError(cutStopTimer(timer));
		
//Read y from the Device Memory

hipMemcpy(y, d_y, size_xy, hipMemcpyDeviceToHost); 
 
// double time_sec=cutGetTimerValue(timer)/1000;
// double time_sec1=cutGetTimerValue(timer1)/1000;
   
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel : %f; time(sec): %f\n",(2*stpoints*csr_size*csr_size*1.0e-6/time_sec),time_sec);
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel setup time(sec) : %f\n",time_sec1);
    
// cutilCheckError(cutDeleteTimer(timer));
// cutilCheckError(cutDeleteTimer(timer1));
if(_DBGFLAG)
{
temp+=tkernel;
	if (kcalls==0)
			{
			printf("\n Structured Grid MatrixMul Kernel Permormance for m *%d* and n size *%d* \n",m,n);
			}
	if (kcalls==1000)
		{
		printf("\ncopy time (sec) : %f\n",tsetup);
		printf("Kernel time (sec): %f\n",tkernel);
		printf("Performance in Megaflops with for %dth Kernel call\n",kcalls);
		printf("Performance in Megaflops with copy time = %f\n",(2*nos*n*m*p*1.0e-6)/(tsetup+tkernel));
		printf("Performance in Megaflops without copy time = %f\n",(2*nos*n*m*p*1.0e-6)/tkernel);
		printf("Culmative Performance in Megaflops for *%d* calls without copy time = %f\n",kcalls,(2*nos*n*m*p*1.0e-6)/(temp/(kcalls+1)));
		}
}
kcalls++;
//Free Device Memory
//hipFree(d_coeff);
hipFree(d_x);
hipFree(d_y);
hipFree(d_idx);
hipFree(d_idy);
hipFree(d_idz);

return 0;
}



*/

