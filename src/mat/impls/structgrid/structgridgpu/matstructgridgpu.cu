#include "hip/hip_runtime.h"
/*  -------------------------------------------------------------------- 

     This file extends structgrid data type to make use of GPUS. The new data type
     is structgridgpu. The implementation of the new datatype emulates the seqaijcusp
     implementation which is an extension to aij matrix format. 
     Author: Chekuri S. Choudary, RNET
*/

#define PETSCMAT_DLL
#include "../src/mat/impls/structgrid/structgridgpu/matstructgridgpu.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>
#include <omp.h>
#include "../src/mat/impls/structgrid/matstructgrid.h"

#include "private/matimpl.h"
#include "matstructgridgpu.h"
#include "cuPrintf.cu"

#define _DBGFLAG 1

// ----------------------------------------------------------
// hardcodiing the shared memory size this should be set
// to give maximum performance, however should be
// replaced soon with a more flexable dynamically allocated
// shared memory scheme
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
#define SHDSIZE 4


// -----------------------------------------------
// Structure for Constant Device memory
// storing constants and indices and index limits
// stencile size is hard coded
// written by: dlowell ANL-MCS
// -----------------------------------------------
#define STLSIZE 64
struct Stencilparams{
       int m;
       int n;
       int p;
       int vecsize_x;
       int vecsize_y;
       int matsize;
       int nos;
       int dof;
       int lda1;
       int lda2;
       int lda3;
       int idx[STLSIZE];
       int idy[STLSIZE];
       int idz[STLSIZE];
       int tile_x;
       int tile_y;
       int tile_z;
       int tsizex;
       int tsizey;
       int tsizez;
};//836 bytes

__constant__ Stencilparams devparams;//device memory



static double* devA;
static PetscScalar* d_coeff;
static double* devX;
static double* devY;




// ----------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//------------------------------------------------------
// general timer function using unix system call
// dlowell ANL-MCS
//------------------------------------------------------
double getclock(){
      struct timezone tzp;
      struct timeval tp;
      gettimeofday (&tp, &tzp);
      return (tp.tv_sec + tp.tv_usec*1.0e-6);
}


/*  --------------------------------------------------------------------
     This function destroys the matrix of type structgridgpu. It first 
     deallocates the memory on GPU and then calls the MatDestroy_SeqSG 
     function.
     Author: Chekuri S. Choudary, RNET
*/

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqSGGPU"
PetscErrorCode  MatDestroy_SeqSGGPU(Mat B)
{
  printf("Call to MatDestroy_SeqSGGPU(Mat B)\n");
  PetscErrorCode ierr;
  PetscFunctionBegin;

  if (B->valid_GPU_matrix != PETSC_CUSP_UNALLOCATED) 
	{
  	if (devA) hipFree(devA);
	if (d_coeff) hipFree(d_coeff);
	if(devY) hipFree(devY);
	if(devX) hipFree(devX);
	}

  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;

  ierr             = MatDestroy_SeqSG(B);CHKERRQ(ierr);

  PetscFunctionReturn(0);
}
EXTERN_C_END



/*  --------------------------------------------------------------------
     This function creates a datatype of structgridgpu. It first creates a
     structgrid datatype and overrides the matrix multiplication method.
     Author: Chekuri S. Choudary, RNET
*/

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode  MatCreate_SeqSGGPU(Mat B)
{

  printf("Call to MatCreate_SeqSGGPU(Mat B)\n");

  PetscErrorCode ierr;
  PetscFunctionBegin;

  ierr             = MatCreate_SeqSG(B);CHKERRQ(ierr);
  B->ops->mult     = MatMult_SeqSGGPU;
  B->ops->destroy  = MatDestroy_SeqSGGPU;

  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSTRUCTGRIDGPU);CHKERRQ(ierr);
  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
  PetscFunctionReturn(0);
}
EXTERN_C_END


//---------------------------------------------------------------------
//     This function implements matrix vector multiplication for the
//     structgridgpu datatype. It calls a CUDA kernel to do matrix
//     multiplication on the GPU.
//     Author: Daniel Lowell, ANL-MCS, Chekuri S. Choudary, RNET
//---------------------------------------------------------------------
EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y)
{
        int i;
	PetscErrorCode ierr;
	Mat_SeqSG * a = (Mat_SeqSG *) mat->data;
	PetscScalar * v = a->a, *xx,*yy;

	PetscFunctionBegin;
	ierr = VecSet(y,0.0); CHKERRQ(ierr);
	ierr = VecGetArray(x, &xx); CHKERRQ(ierr);
	ierr = VecGetArray(y, &yy); CHKERRQ(ierr);


        //set up parameters for constant memory
        struct Stencilparams sparams;
        for(i=0;i<a->stpoints;i++){
            sparams.idx[i]=a->idx[i];
            sparams.idy[i]=a->idy[i];
            sparams.idz[i]=a->idz[i];
        }
        sparams.m=a->m;
        sparams.n=a->n;
        sparams.p=a->p;
        VecGetLocalSize(x,&sparams.vecsize_x);
        VecGetLocalSize(y,&sparams.vecsize_y);
        sparams.nos = a->stpoints;
        sparams.dof = a->dof;
        sparams.lda1=a->m*a->n*a->p;
        sparams.lda2=a->m*a->n;
        sparams.lda3=a->m;
        sparams.matsize=a->m*a->n*a->p*a->stpoints;

        /// Debugging block .....................................................
            /*int xsize,ysize;
            //printf("Matrix A ::: m: %d, n: %d, p: %d, nos: %d dof: %d nz: %d\n",
            //    a->m,a->n,a->p,a->stpoints,a->dof,a->nz);
            //VecGetLocalSize(x,&xsize);
            //VecGetLocalSize(y,&ysize);
            //printf("Amat size: %d, Xvec size: %d, Yvec size: %d\n",sparams.matsize,xsize,ysize);
            */
            //static PetscInt count = 1;// running count of function calls
            //printf("MatMult_SeqSGGPU(Mat mat, Vec x, Vec y): %d\n",count++);
        ///....................................................................


// Call to dlowell's version
      ierr = SGCUDA_MatMult_v2(v,xx,yy,sparams,&(mat->valid_GPU_matrix));
	// CHKERRQ(ierr);

// Call to Jeswin's version
     //   ierr = SGCUDA_MatMult(v,xx,yy,a->idx,a->idy,a->idz,a->m,a->n,a->p,a->stpoints,&(mat->valid_GPU_matrix));CHKERRQ(ierr);

       	ierr = VecRestoreArray(x,&xx); CHKERRQ(ierr);
	ierr = VecRestoreArray(y,&yy); CHKERRQ(ierr);
	ierr = PetscLogFlops(2*a->nz*a->stpoints); CHKERRQ(ierr);
	PetscFunctionReturn(0);
}
EXTERN_C_END


//-------------------------------------------------------------------------------
//   This function is the matrix vector multiplication kernel
//   structgridgpu datatype. This version uses shared memory for the write
//   back vector Y. Constant memory for reused constants and indices.
//   More offloading to registers might be possible as well.
//   written by: dlowell, ANL-MCS
//-------------------------------------------------------------------------------
__global__ void MatMul_Kernel_v2(double* A, double* X, double* Y){

   //indices for local accesses
   int tbtx, tbty, tbtz;
   int ix,iy,iz,j;

   int nos  = devparams.nos; // set to register
   int lda1 = devparams.lda1;//  "   "
   int lda2 = devparams.lda2;//  "   "
   int lda3 = devparams.lda3;//  "   "
   int tilex = devparams.tile_x*devparams.tsizex;
   int tiley = devparams.tile_y*devparams.tsizey;
   int tilez = devparams.tile_z*devparams.tsizez;

   int Aindex;
   int Xindex;
   int index;


   //Min shared mem byte count: 2*(gridDim.x*gridDim.y*gridDim.z)*SHDSIZE^3*8 byte double
   __shared__ double Ys[SHDSIZE][SHDSIZE][SHDSIZE];
   __shared__ double As[SHDSIZE][SHDSIZE][SHDSIZE];

//------------------------------------------------------------------------

   for(iz=0;iz<tilez;iz+=devparams.tsizez){// tiles Z loop
        tbtz = blockDim.z*blockIdx.z+threadIdx.z + iz;

   for(iy=0;iy<tiley;iy+=devparams.tsizey){// tiles Y loop
        tbty = blockDim.y*blockIdx.y+threadIdx.y + iy;

   for(ix=0;ix<tilex;ix+=devparams.tsizex){// tiles X loop
        tbtx = blockDim.x*blockIdx.x+threadIdx.x + ix;
        __syncthreads();

        //cuPrintf("tid xyz: %d, %d, %d\n",tbtx,tbty,tbtz);
        //initialize current return-tile
        Ys[threadIdx.z][threadIdx.y][threadIdx.x]=0.;

        //adjusted index for global access
        index = tbtz*lda2 + tbty*lda3 + tbtx;

        __syncthreads();

//......STENCIL...........................................
        for(j=0;j<nos;j++){//loop over stencil pattern

           Aindex=j*lda1+index;//set up Aindex and read from global A into As tile
           if(Aindex<devparams.matsize) As[threadIdx.z][threadIdx.y][threadIdx.x]=A[Aindex];//needs to be coalesced
           else As[threadIdx.z][threadIdx.y][threadIdx.x]=0.;

           __syncthreads();

           if(!((j==1 && tbtx==devparams.m-1 && tbty==devparams.n-1)||
                (j==2 && tbtx==0 && tbty==0)|| (j==3 && tbty==devparams.n-1)||
                (j==4 && tbty==0))){

              //set up Xindex for element-wise operation using stencil pattern
              Xindex=(devparams.idz[j]*lda2 + devparams.idy[j]*lda3 + devparams.idx[j]) + index;
              if(Xindex<devparams.vecsize_x)
                 Ys[threadIdx.z][threadIdx.y][threadIdx.x]+=As[threadIdx.z][threadIdx.y][threadIdx.x]*X[Xindex];

           }//end if
           __syncthreads();

        }//end j-for

        __syncthreads();
        if(index<devparams.vecsize_y) Y[index]=Ys[threadIdx.z][threadIdx.y][threadIdx.x];//global write back

   }//end ix-for
   }//end iy-for
   }//end iz-for
}//end kernel_v2






//   int Xoffset;
              //Xoffset=(devparams.idz[j]*lda2+devparams.idy[j]*lda3+devparams.idx[j]);
              //  cuPrintf("Xindex: %d, Xoffset: %d, Xsize: %d idx: %d, idy: %d, idz: %d, j: %d nos: %d\n",Xindex,Xoffset,devparams.vecsize_x,devparams.idx[j],devparams.idy[j],devparams.idz[j],j,nos);


//------------------------------------------------------------------------------------
//   This function is the wrapper function which sets up the device memory, transfers
//   data to and from the device, and calls the kernel. Error checking is done at 
//   each step. Timing stats are recorded using static vars.
//   written by: Daniel Lowell, ANL-MCS
//------------------------------------------------------------------------------------
PetscErrorCode SGCUDA_MatMult_v2(PetscScalar* A, PetscScalar* X, 
PetscScalar* Y, struct Stencilparams P, PetscCUSPFlag* fp){

        // vars for testing
        int i;
        static double cumktime=0.;//cummalitive kernel time
        static double cumtime=0.;//cummalitive call time
        static unsigned int kcalls=0;//number of kernel calls
	double cs,ce,temp;
	float elapsedtime;        // using CUDA device timer
	hipEvent_t start,stop;

        static unsigned char allocflag = 1;
        static double maxshared;
        static int bx,by,bz;//number of blocks in 3-D
        static int tx,ty,tz;//number of threads ber block in 3-D
        static int maxblocks_xy;
        static int maxblocks_z;
        static dim3 dimGrid;
	static dim3 dimBlock;
        static int xytile;

	hipError_t cudastatus0,cudastatus1,
	            cudastatus2,cudastatus3,
	            cudastatus4,cudastatus5,
           	    cudastatus6,cudastatus7;


        //size in bytes to be allocated onto device
        int matsize =P.matsize*sizeof(double);
        int vecsize_x = P.vecsize_x*sizeof(double);
        int vecsize_y = P.vecsize_y*sizeof(double);


        if(_DBGFLAG){//create CUDA events for timer
           hipEventCreate(&start);
           hipEventCreate(&stop);
        }


	if(_DBGFLAG) cs=getclock();

        //Allocate and Memcpy Structured Matrix A
	//The matrix remains the same throughout one iteration
        //of the linear solver. The following uses a flag
        //defined in the base class to check the status of the
        //matrix. The matrix is copied to the GPU only if
        //it has been changed on the CPU side
        //This feature added by Chekuri S. Choudary

        if ((*fp == PETSC_CUSP_UNALLOCATED) || (*fp == PETSC_CUSP_CPU)){
		if (*fp == PETSC_CUSP_UNALLOCATED){
	   	   cudastatus0=hipMalloc((void**)&devA,matsize);
	   	   if(cudastatus0!=hipSuccess){
                        printf("Error in devA memory allocation:\nstatus0: %s\n",
  			hipGetErrorString(cudastatus0));
          	        PetscFunctionReturn(PETSC_ERR_MEM);
		   }
		}

           	cudastatus1=hipMemcpy(devA,A,matsize,hipMemcpyHostToDevice);
	   	if(cudastatus1!=hipSuccess){
		  if(devA) hipFree(devA);
		  printf("Error in devA memory copying:\nstatus1: %s\n",
  			hipGetErrorString(cudastatus1));
          	  PetscFunctionReturn(PETSC_ERR_MEM);
		}

	       *fp = PETSC_CUSP_BOTH;
	}


        //Allocate device memory for X and Y, and shape grid and blocks
        if(allocflag){
                cudastatus2=hipMalloc((void**)&devX,vecsize_x);//allocate X on device
	        if(cudastatus2!=hipSuccess){
                        printf("Error in devX memory allocation: %s\n",hipGetErrorString(cudastatus2));
	                if(devA) hipFree(devA);
                        PetscFunctionReturn(PETSC_ERR_MEM);
                }

                cudastatus3=hipMalloc((void**)&devY,vecsize_y);//allocate Y on device
                if(cudastatus3!=hipSuccess){
                        printf("Error in devY memory allocation: %s\n",hipGetErrorString(cudastatus3));
	                if(devA) hipFree(devA);
	                if(devX) hipFree(devX);
                        PetscFunctionReturn(PETSC_ERR_MEM);
	        }



                //Set up blocks and thread numbers
                maxshared = 49152.0/(double)(2.0*sizeof(double));
                if(P.p==1){
                    xytile = pow(maxshared,0.5);//square blocks
                    maxblocks_z = 1;
                }else{
                    temp=maxshared/P.p;//lop off z
                    xytile = pow(temp,0.5);//xyblocks
                    maxblocks_z=ceil((float)SHDSIZE/(float)P.p);
                }
                maxblocks_xy = xytile/SHDSIZE;


                //Set up blocks and thread numbers for columns
                if(P.m <= SHDSIZE){
                       tx = P.m;
                       bx = 1;
                       P.tile_x = 1;
                       P.tsizex=1;
                }else{
                       tx = SHDSIZE;
                       bx = ceil((float)P.m/(float)SHDSIZE);//create enough blocks
                       if(bx>maxblocks_xy){                    //too many blocks created
                          bx = maxblocks_xy;                   //set to max number of blocks allowed
                          P.tile_x=ceil((float)P.m/(float)(bx*SHDSIZE));//number of tiles
                          P.tsizex=bx*SHDSIZE;              //tilesize is block-thread coverage
                       }else{
                          P.tile_x=1;
                          P.tsizex=1;
                       }
                }

                //Set up blocks and thread numbers for rows
                if(P.n <= SHDSIZE){
                       ty = P.n;
                       by = 1;
                       P.tile_y = 1;
                       P.tsizey=1;
                }else{
                       ty = SHDSIZE;
                       by = ceil((float)P.n/(float)SHDSIZE);
                       if(by > maxblocks_xy){
                          by = maxblocks_xy;
                          P.tile_y=ceil((float)P.n/(float)(by*SHDSIZE));
                          P.tsizey=by*SHDSIZE;
                       }else{
                          P.tile_y=1;
                          P.tsizey=1;
                       }
                }

                //Set up blocks and thread numbers for z
                if(P.p <= SHDSIZE){
                       tz = P.p;
                       bz = 1;
                       P.tile_z = 1;
                       P.tsizez=1;
                }else{
                       tz = SHDSIZE;
                       bz = ceil((float)P.p/(float)SHDSIZE);
                       if(bz > maxblocks_z){
                          bz = maxblocks_z;
                          P.tile_z=ceil((float)P.p/(float)(bz*SHDSIZE));
                          P.tsizez=bz*SHDSIZE;
                       }else{
                          P.tile_z=1;
                          P.tsizez=1;
                       }
                }

                //set grid shape
                dimGrid.x = bx;
                dimGrid.y = by;
                dimGrid.z = bz;

                //set block shape
                dimBlock.x = tx;
                dimBlock.y = ty;
                dimBlock.z = tz;

                // update constant memory with structured grid parameters
	        cudastatus6=hipMemcpyToSymbol(HIP_SYMBOL("devparams"),&P,sizeof(Stencilparams));
	        if(cudastatus6!=hipSuccess){
                        printf("Error in symbol copy to device: %s.\n",hipGetErrorString(cudastatus6));
                        if(devA) hipFree(devA);
	                if(devY) hipFree(devY);
	                if(devX) hipFree(devX);
                        PetscFunctionReturn(PETSC_ERR_MEM);
	        }

                //toggle off allocation flag
                allocflag = 0;

        }//end allocflag-if

        //grid and block shape & device config. debugging.....................................
        unsigned int sharebytes = 2*tx*ty*tz*bx*by*bz*sizeof(double);
        static unsigned char dbgflag = 1;
        if(dbgflag){
           printf("(m, n, p, nos): (%d, %d, %d, %d)\n",P.m,P.n,P.p,P.nos);
           printf("MAXBLOCKS: %d maxshared: %lf\n",maxblocks_xy+maxblocks_z,maxshared);
           printf("blocks: (%d, %d, %d), threads per block: %d\n", bx,by,bz,tx*ty*tz );
           printf("Shared elements occupied: %0.3f SharedOccupied in Bytes: %d\n",sharebytes/49152.0,sharebytes);
           printf("Blocks x,y,z: (%d, %d, %d)\n",bx,by,bz);
           printf("Blocks*ThreadsPer size x,y,z: (%d, %d, %d)\n",bx*tx,by*ty,bz*tz);
           printf("Tiles (x,y,z): (%d, %d, %d)\n",P.tile_x,P.tile_y,P.tile_z);
           printf("Tile Size (x,y,z): (%d, %d, %d)\n",P.tsizex,P.tsizey,P.tsizez);
           dbgflag=0;
        }
        //...End config. debug section.................................................................




       //copy over values of X to device memory
	cudastatus4=hipMemcpy(devX,X,vecsize_x,hipMemcpyHostToDevice);
	if(cudastatus4!=hipSuccess){
                printf("Error in devX memory copy to device: status: %s\n",hipGetErrorString(cudastatus4));
	        if(devA) hipFree(devA);
	        if(devX) hipFree(devX);
	        if(devY) hipFree(devY);
                PetscFunctionReturn(PETSC_ERR_MEM);
	}

/*//probably an unnecessary step.
        // memset to 0. Vector Y on device
	cudastatus5=hipMemset(devY,0.0,vecsize_y);
	if(cudastatus5!=hipSuccess){
                printf("Error in devY memset to device: %s\n",hipGetErrorString(cudastatus5));
	        if(devA) hipFree(devA);
                if(devY) hipFree(devY);
                if(devX) hipFree(devX);
                PetscFunctionReturn(PETSC_ERR_MEM);
	}
*/

        //toggle timer and debug settings
        if(_DBGFLAG){
                ce=getclock();//end setup timer
	        temp=ce-cs;
                cudaPrintfInit();//start cuda printf environ.
	        hipEventRecord(start,0);//begin recording kernel
        }

        //Launch the kernel..........................................
	MatMul_Kernel_v2<<<dimGrid,dimBlock>>>(devA,devX,devY);
        checkCUDAError("CUDA Kernel launch...");//check for failure
        //...........................................................

        //toggle timer and debug settings
        if(_DBGFLAG){
                cudaPrintfDisplay(stdout, true);//choose output
                cudaPrintfEnd();//kill cuda printf environ
	        hipEventRecord(stop,0);
	        hipEventSynchronize(stop); // event barrier
	        hipEventElapsedTime(&elapsedtime,start,stop);
                hipEventDestroy(start);
	        hipEventDestroy(stop);
        }

        // Copy back Vector Y from Kernel
	cs=getclock();
	cudastatus7=hipMemcpy(Y,devY,vecsize_y,hipMemcpyDeviceToHost);
	if(cudastatus7!=hipSuccess){
          printf("Error on copy back Y, kernel status: %s\nExiting...\n\n",hipGetErrorString(cudastatus7));
	  if(devA) hipFree(devA);
	  if(devY) hipFree(devY);
	  if(devX) hipFree(devX);
          PetscFunctionReturn(PETSC_ERR_MEM);
        }



        if(_DBGFLAG){
          //for(i=0;i<P.lda1;i++)printf("Y[%d]: %lf\n",i,Y[i]);//for verification
	  ce=getclock();
	  temp+=ce-cs;
          cumktime+=(elapsedtime/1000);
          cumtime+=(elapsedtime/1000)+temp;
          kcalls++;
          printf("Kernel call #: %d\n",kcalls);
          printf("setup+copyback: %f sec.\nelapsed time: %f sec.\ntotal call time: %f sec.\n",
                  temp,elapsedtime/1000,(elapsedtime/1000)+temp);
          printf("Cum. kernel time: %lf sec.\n", cumktime);
          printf("Cum. call time (with setup): %lf sec.\n", cumtime);
          printf(".........................................\n\n");
        }//end _DBGFLAG-if

        PetscFunctionReturn(0);
}





/*  -------------------------------------------------------------------- 
     The following is a CUDA kernel for matrix vector multiplication on 
     the GPU. The matrix is in a custom layout that facilitates better 
     memory accesses and vectorization. 
     Author: Chekuri S. Choudary, RNET
*/
/*  __global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{
int tx=  blockDim.x * blockIdx.x + threadIdx.x;
int ty=  blockDim.y * blockIdx.y + threadIdx.y;
int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

	for (l=0;l<nos;l++)
		{
			xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
			if (l==1 && tx==n-1 && ty==m-1)
				{
				continue;
				}
			if (l==2 && tx==0 && ty==0)
				{
				continue;
				}
			if (l==3 && ty==m-1)
				{
				continue;
				}
			if (l==4 && ty==0)
				{
				continue;
				}
			for(i=0;i<p;i++)
				ptr_y[ i*lda2 + ty*lda3 + tx]+= (ptr_coeff[offset + i*lda2 + ty*lda3 +tx] * ptr_x[(i+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
		}
}
 */ 
#define BLOCKWIDTH 16
 
 __global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt* idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

int tx= blockDim.x * blockIdx.x + threadIdx.x;
int ty= blockDim.y * blockIdx.y + threadIdx.y;
int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;
__shared__ double y_sm[256];
__shared__ double x_sm[324];

// copying a Tile from Y into the shared Memory
y_sm[threadIdx.y*BLOCKWIDTH + threadIdx.x]=0;

// Copying a tile x into the shared Memory with 2 steps.

// Copying without the Ghost Cells  
x_sm[(threadIdx.y+1)*(BLOCKWIDTH+2) + (threadIdx.x+1)]=ptr_x[ty*lda3 + tx];


// Copying the Ghost Cells
if (tx!=0 || ty!=0 || tx != n-1 || ty != m-1)
{
	if (threadIdx.x==0)
	x_sm[(threadIdx.y+1)*(BLOCKWIDTH+2) + threadIdx.x]=ptr_x[ty*lda3 + tx-1];

	if (threadIdx.y==0)
	x_sm[(threadIdx.y)* (BLOCKWIDTH+2) + threadIdx.x+1]=ptr_x[(ty-1)*lda3 + tx];

	if (threadIdx.x==BLOCKWIDTH-1)
	x_sm[(threadIdx.y+1)*(BLOCKWIDTH+2) + threadIdx.x + 2]=ptr_x[ty*lda3 + tx + 1];

	if (threadIdx.y==BLOCKWIDTH-1)
	x_sm[(threadIdx.y+2)*(BLOCKWIDTH+2) + threadIdx.x +1]=ptr_x[(ty+1)*lda3 + tx];
}
__syncthreads();

// if (tx==2 && ty==2)
// {
// cuPrintf("\nPrinting the X from Shared Memory \n ");

// for (int j=0;j<324;j++)
// {
// if(j % 16 ==0)
// {
// cuPrintf("\n");
// }
// cuPrintf("%f  ",  x_sm[j]);
// }
// }
// MATMUL
for (l=0;l<nos;l++)
	{
	xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
	if (tx > n-1)
	{
		break; //use Break and test performance later(divergence)
	}
	if (ty > m-1)
	{
		break; //use Break and test performance later(divergence)
	}
	if (l==1 && tx==n-1 && ty==m-1)
	{
		continue;
	}
	if (l==2 && tx==0 && ty==0)
	{
		continue;
	}
	if (l==3 && ty==m-1)
	{
		continue;
	}
	if (l==4 && ty==0)
	{
		continue;
	}
	for(i=0;i<p;i++)
	y_sm[threadIdx.y*BLOCKWIDTH + threadIdx.x]+= (ptr_coeff[offset + i*lda2 + ty*lda3 +tx] * x_sm[(i+zdisp)*lda2 + (threadIdx.y+ydisp +1)*(BLOCKWIDTH+2) + (threadIdx.x+xdisp+1)]); //forgetting Z currently.. I have to Fix it.
	}
	// removing i tempararily
	ptr_y[ty*lda3 + tx]= y_sm[threadIdx.y*BLOCKWIDTH + threadIdx.x];
}
  
int SGCUDA_MatMult(PetscScalar* coeff, PetscScalar* x, PetscScalar* y, PetscInt *idx, PetscInt* idy, 
PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos, PetscCUSPFlag* fp)
{

double tbegin1, tbegin2, tend1, tend2;
static PetscInt size_coeff; 
double tsetup,tkernel;
static unsigned int kcalls=0;
PetscInt size_xy, size_id; 
static double temp=0;
PetscScalar* d_x;
PetscScalar* d_y;
PetscInt *d_idx, *d_idy, *d_idz;

  //unsigned int timer1 = 0;
  //cutilCheckError(cutCreateTimer(&timer1));
  //cutilCheckError(cutStartTimer(timer1));

  //  fprintf(stdout,"In SGCUDA_MatMult\n");
	
      if(_DBGFLAG) tbegin1=getclock();
	  
	  if ((*fp == PETSC_CUSP_UNALLOCATED) ||
	  (*fp == PETSC_CUSP_CPU) )
	{
		if (*fp == PETSC_CUSP_UNALLOCATED)
		{
		size_coeff=nos*m*n*p*sizeof(PetscScalar);	
		hipMalloc((void**)&d_coeff,size_coeff);
	
	   	//cudastatus0=hipMalloc((void**)&devA,matsize);
	   	//if(cudastatus0!=hipSuccess)
		//	{
		//  printf("Error in devA memory allocation:\nstatus0: %s\n",
  		//	hipGetErrorString(cudastatus0));
          	//  PetscFunctionReturn(PETSC_ERR_MEM);
		//	}
		}
	
		hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);
	
           	//cudastatus1=hipMemcpy(devA,A,matsize,hipMemcpyHostToDevice);
	   	//if(cudastatus1!=hipSuccess)
		//{
		// if(devA) hipFree(devA);
		//  printf("Error in devA memory copying:\nstatus1: %s\n",
  		//	hipGetErrorString(cudastatus1));
          	//  PetscFunctionReturn(PETSC_ERR_MEM);
		//}
	
	        *fp = PETSC_CUSP_BOTH;
	}


//size_coeff=nos*m*n*p*sizeof(PetscScalar);
//hipMalloc((void**)&d_coeff,size_coeff);
//hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);


size_xy = m*n*p*sizeof(PetscScalar);
hipMalloc((void**)&d_x,size_xy); 
hipMemcpy(d_x, x, size_xy, hipMemcpyHostToDevice);

hipMalloc((void**)&d_y,size_xy); 
hipMemcpy(d_y, y, size_xy, hipMemcpyHostToDevice);

size_id = nos*sizeof(PetscInt);
hipMalloc((void**)&d_idx,size_id); 
hipMemcpy(d_idx, idx, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idy,size_id); 
hipMemcpy(d_idy, idy, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idz,size_id); 
hipMemcpy(d_idz, idz, size_id, hipMemcpyHostToDevice);

if(_DBGFLAG) 
	{
		tend1=getclock();
		tsetup=tend1-tbegin1;
		tbegin2=getclock();
	}

//cutilCheckError(cutStopTimer(timer1));
// kernel Configuration
if (m > BLOCKWIDTH){
dim3 dimBlock(BLOCKWIDTH,BLOCKWIDTH);
dim3 dimGrid((int)ceil((float)m/(float)BLOCKWIDTH),((int)ceil((float)n/(float)BLOCKWIDTH)));

    // cutilCheckError(cutCreateTimer(&timer));
    // cutilCheckError(cutStartTimer(timer));

MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);

}
else
{
dim3 dimBlock(m,n);
dim3 dimGrid(1,1);
   
    // cutilCheckError(cutCreateTimer(&timer));
    // cutilCheckError(cutStartTimer(timer));

MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);


}

//Cuda Printf
//cudaPrintfInit();

//tbegin4 = rtclock();
// create and start timer
    //unsigned int timer = 0;
    //cutilCheckError(cutCreateTimer(&timer));
    //cutilCheckError(cutStartTimer(timer));

	if(_DBGFLAG) 
	{
		hipDeviceSynchronize();
		tend2=getclock();
		tkernel=tend2-tbegin2;
	}

	
	
   // check if kernel execution generated and error
    	//cutilCheckMsg("Kernel execution failed");

   // stop and destroy timer
    	//cutilCheckError(cutStopTimer(timer));
		
//Read y from the Device Memory

hipMemcpy(y, d_y, size_xy, hipMemcpyDeviceToHost); 
 
// double time_sec=cutGetTimerValue(timer)/1000;
// double time_sec1=cutGetTimerValue(timer1)/1000;
   
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel : %f; time(sec): %f\n",(2*stpoints*csr_size*csr_size*1.0e-6/time_sec),time_sec);
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel setup time(sec) : %f\n",time_sec1);
    
// cutilCheckError(cutDeleteTimer(timer));
// cutilCheckError(cutDeleteTimer(timer1));
if(_DBGFLAG)
{
temp+=tkernel;
	if (kcalls==0)
			{
			printf("\n Structured Grid MatrixMul Kernel Permormance for m *%d* and n size *%d* \n",m,n);
			}
	if (kcalls==1000)
		{
		printf("\ncopy time (sec) : %f\n",tsetup);
		printf("Kernel time (sec): %f\n",tkernel);
		printf("Performance in Megaflops with for %dth Kernel call\n",kcalls);
		printf("Performance in Megaflops with copy time = %f\n",(2*nos*n*m*p*1.0e-6)/(tsetup+tkernel));
		printf("Performance in Megaflops without copy time = %f\n",(2*nos*n*m*p*1.0e-6)/tkernel);
		printf("Culmative Performance in Megaflops for *%d* calls without copy time = %f\n",kcalls,(2*nos*n*m*p*1.0e-6)/(temp/(kcalls+1)));
		}
}
kcalls++;
//Free Device Memory
//hipFree(d_coeff);
hipFree(d_x);
hipFree(d_y);
hipFree(d_idx);
hipFree(d_idy);
hipFree(d_idz);

return 0;
}




