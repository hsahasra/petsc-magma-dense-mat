#include "hip/hip_runtime.h"
/*  --------------------------------------------------------------------

     This file extends structgrid data type to make use of GPUS. The new data type
     is structgridgpu. The implementation of the new datatype emulates the seqaijcusp
     implementation which is an extension to aij matrix format.
     Author: Chekuri S. Choudary, RNET
 */


#define PETSCMAT_DLL
#include "../src/mat/impls/structgrid/structgridgpu/matstructgridgpu.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>
#include <omp.h>
#include "../src/mat/impls/structgrid/matstructgrid.h"
#include <../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h>
#include "petsc-private/matimpl.h"

#include "matstructgridgpu.h"

#define _DBGFLAG 0
//#define PRINT
//block size is 1x256.
#define BLOCKWIDTH_X 256
#define BLOCKWIDTH_Y 1


// ----------------------------------------------------------
// hardcodiing the shared memory size this should be set
// to give maximum performance, however should be
// replaced soon with a more flexable dynamically allocated
// shared memory scheme
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
#define SHDSIZE 4


#define size 64
// Structure for Constant Device memory
// storing constants and indices and index limits
// stencile size is hard coded
// written by: dlowell ANL-MCS
// -----------------------------------------------
#define STLSIZE 64
struct Stencilparams{
  int m;
  int n;
  int p;
  int vecsize_x;
  int vecsize_y;
  int matsize;
  int nos;
  int dof;
  int lda1;
  int lda2;
  int lda3;
  int idx[STLSIZE];
  int idy[STLSIZE];
  int idz[STLSIZE];
  int tile_x;
  int tile_y;
  int tile_z;
  int tsizex;
  int tsizey;
  int tsizez;
};//836 bytes

__constant__ Stencilparams devparams;//device memory



// ----------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
    exit(EXIT_FAILURE);
  }
}

//------------------------------------------------------
// general timer function using unix system call
// dlowell ANL-MCS
//------------------------------------------------------
double getclock(){
  struct timezone tzp;
  struct timeval tp;
  gettimeofday (&tp, &tzp);
  return (tp.tv_sec + tp.tv_usec*1.0e-6);
}


/*  --------------------------------------------------------------------
     This function destroys the matrix of type structgridgpu. It first
     deallocates the memory on GPU and then calls the MatDestroy_SeqSG
     function.
     Author: Chekuri S. Choudary, RNET
 */

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqSGGPU"
PetscErrorCode  MatDestroy_SeqSGGPU(Mat B)
{
  //printf("Call to MatDestroy_SeqSGGPU(Mat B)\n");
  PetscErrorCode ierr;
  PetscFunctionBegin;

  Mat_SeqSG * a = (Mat_SeqSG *) B->data;

  if (B->valid_GPU_matrix != PETSC_CUSP_UNALLOCATED)
  {
#if _DBGFLAG
  	  printf("Free GPU matrix(%p)\n", a->gpuMat);
  	  fflush(stdout);
#endif
    if (a->gpuMat) hipFree(a->gpuMat);
#if _DBGFLAG
    printf("After free\n", a->gpuMat);
    fflush(stdout);
#endif
  }

  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;

  ierr             = MatDestroy_SeqSG(B);CHKERRQ(ierr);

  PetscFunctionReturn(0);
}
EXTERN_C_END



/*  --------------------------------------------------------------------
     This function creates a datatype of structgridgpu. It first creates a
     structgrid datatype and overrides the matrix multiplication method.
     Author: Chekuri S. Choudary, RNET
 */

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode  MatCreate_SeqSGGPU(Mat B)
{
#if _DBGFLAG
  printf("Call to MatCreate_SeqSGGPU(Mat B)\n");
#endif

  PetscErrorCode ierr;
  PetscFunctionBegin;

  ierr             = MatCreate_SeqSG(B);CHKERRQ(ierr);
  B->ops->mult     = MatMult_SeqSGGPU;
  B->ops->destroy  = MatDestroy_SeqSGGPU;

  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSTRUCTGRIDGPU);CHKERRQ(ierr);
  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
  PetscFunctionReturn(0);
}
EXTERN_C_END


//---------------------------------------------------------------------
//     This function implements matrix vector multiplication for the
//     structgridgpu datatype. It calls a CUDA kernel to do matrix
//     multiplication on the GPU.
//     Author: Daniel Lowell, ANL-MCS, Chekuri S. Choudary, RNET
//---------------------------------------------------------------------
EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y)
{
  int i;
  PetscErrorCode ierr;
  Mat_SeqSG * a = (Mat_SeqSG *) mat->data;
  PetscScalar * v = a->a, *xx,*yy;

  PetscFunctionBegin;
  ierr = VecSet(y,0.0); CHKERRQ(ierr);
  //  ierr = VecGetArray(x, &xx); CHKERRQ(ierr);
  //  ierr = VecGetArray(y, &yy); CHKERRQ(ierr);

  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  xx = xd->devptr;
  yy = yd->devptr;

#if _DBGFLAG
  printf("MatMult call (%p)\n", a);
#endif
  /* Call to Jeswin's version */
  ierr = SGCUDA_MatMult(v,xx,yy,a->idx,a->idy,a->idz,a->m,a->n,a->p,
      a->stpoints,&(mat->valid_GPU_matrix),a->dof, &a->gpuMat);CHKERRQ(ierr);

  yd->syncState=VEC_GPU;
  //  ierr = VecRestoreArray(x,&xx); CHKERRQ(ierr);
  //  ierr = VecRestoreArray(y,&yy); CHKERRQ(ierr);
  ierr = PetscLogFlops(2*a->nz*a->stpoints); CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
EXTERN_C_END


/*  --------------------------------------------------------------------
     The following is a CUDA kernel for matrix vector multiplication on
     the GPU. The matrix is in a custom layout that facilitates better
     memory accesses and vectorization.
     Author: Chekuri S. Choudary, RNET
 */
/* Version with Shared memory for X only supports rectangular tiles. */
/* __global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

int tx= blockDim.x * blockIdx.x + threadIdx.x;
int ty= blockDim.y * blockIdx.y + threadIdx.y;

int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

__shared__ PetscScalar y_sm[256];

// initializing to the zero
y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x]=0;
for (l=0;l<nos;l++)
	{
	xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
	if (tx > n-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (ty > m-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (l==1 && tx==n-1 && ty==m-1)
	{
	continue;
	}
	if (l==2 && tx==0 && ty==0)
	{
	continue;
	}
	if (l==3 && ty==m-1)
	{
	continue;
	}
	if (l==4 && ty==0)
	{
	continue;
	}
	for(i=0;i<p;i++)
	y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x]+= (ptr_coeff[offset + i*lda2 + ty*lda3 +tx] * ptr_x[(i+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
	}

	ptr_y[ty*lda3 + tx]= y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x];
}
 */
/*  #define BLOCKWIDTH 8
#define BLOCKWIDTH_X 8
#define BLOCKWIDTH_Y 8
#define BLOCKWIDTH_Z 8

  __global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

int tx= blockDim.x * blockIdx.x + threadIdx.x;
int ty= blockDim.y * blockIdx.y + threadIdx.y;
int tz= blockDim.z * blockIdx.z + threadIdx.z;
int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

__shared__ PetscScalar y_sm[512];

// initializing to the zero
y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y + threadIdx.y*BLOCKWIDTH_X + threadIdx.x]=0;
for (l=0;l<nos;l++)
	{
	xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
	if (tx > n-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (ty > m-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (tz > p-1)
	{
	break;
	}
	if (l==1 && tx==n-1 && ty==m-1 && tz==p-1)
	{
	continue;
	}
	if (l==2 && tx==0 && ty==0 && tz==0)
	{
	continue;
	}
	if (l==3 && ty==m-1)
	{
	continue;
	}
	if (l==4 && ty==0)
	{
	continue;
	}
	if (l==5 && tz==p-1)
	{
	continue;
	}
	if (l==6 && tz==0)
	{
	continue;
	}
	//for(i=0;i<p;i++)
	y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y + threadIdx.y*BLOCKWIDTH_X + threadIdx.x]+= (ptr_coeff[offset + tz*lda2 + ty*lda3 +tx] * ptr_x[(tz+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
	}

	ptr_y[tz*lda2+ ty*lda3 + tx]= y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y +threadIdx.y*BLOCKWIDTH_X + threadIdx.x];
}
 */


//------------------------------------------------------------------------------------
//   These functions are used to bind and unbind the Vector x to the texture Memory.
//------------------------------------------------------------------------------------
texture<int2, 1> tex_x_double;

void unbind_x( double * x)
{
  hipUnbindTexture(tex_x_double);
}


static __inline__ __device__ double fetch_double(texture<int2, 1> tex_x_double, int i)
{
  int2 v = tex1Dfetch(tex_x_double,i);
  return __hiloint2double(v.y, v.x);
}

//------------------------------------------------------------------------------------
// Dynamically allocating Shared Memory size
//------------------------------------------------------------------------------------

extern __shared__ PetscInt idx_sm[];

//------------------------------------------------------------------------------------
//  Below functions are SPMV kernel functions where x through the Texture Memory, offsets are accesed
//	through the Shared Memory, Y is accessed per thread from registers. Coeff accesses are from the global Memory
//	but they are coalesced.
//------------------------------------------------------------------------------------


__global__ void MatMul_Kernel_tex_1_DOF(PetscScalar * ptr_coeff,
    PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt m,
    PetscInt n, PetscInt p, PetscInt nos, PetscInt DOF) {
  int tx = blockDim.x * blockIdx.x + threadIdx.x;
  int l, offset;
  int lda1 = m * n * p * DOF;
  int lda2 = m * p * DOF; //lda3=m*DOF
  PetscInt Index;
  PetscScalar y_reg = 0;

  if (threadIdx.x < nos) {
    idx_sm[threadIdx.x] = idx[threadIdx.x];
  }

  int reg2 = blockIdx.y * lda2 + tx;

  //Iterating through the Diagonals
  for (l = 0; l < nos; l++) {
    Index = reg2 + idx_sm[l];

    if (Index >= 0 && Index < lda1) {
      offset = l * lda1;
      y_reg += ptr_coeff[offset + reg2] * fetch_double(tex_x_double, Index);
    }
  }

  ptr_y[reg2] = y_reg;
}


// Jeswin's uncommitted kernel
__global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x,
    PetscScalar* ptr_y, PetscInt *idx, PetscInt m, PetscInt n, PetscInt p,
    PetscInt nos, PetscInt stpoints, PetscInt DOF) {
  int tx = blockDim.x * blockIdx.x + threadIdx.x;
  int l, i;
  int offset;
  int lda1 = m * n * p * DOF, lda2 = m * p * DOF, lda3 = m * DOF;
  int Index;
  double y_reg = 0;

  if (tx >= lda1)
    return;

  if (threadIdx.x < stpoints) {
    idx_sm[threadIdx.x] = idx[threadIdx.x];
  }
  __syncthreads();

  int reg2 = blockIdx.y * lda2 + tx;

  //Divergence in Computation
  for (l = 0; l < stpoints; l++) {
    Index = reg2 + idx_sm[l];
    if ((Index >= 0) && (Index < lda1)) {
      for (i = 0; i < DOF; i++) {
        offset = (l * DOF + i) * lda1;
        if (threadIdx.x == 0) {
          printf("index: %d\n", offset + reg2);
        }
        //if ((offset + reg2) >= (m*n*p*DOF*stpoints*DOF)) {
        //	  printf("Out of range!\n");
        //}
        y_reg += ptr_coeff[offset + reg2]
            * fetch_double(tex_x_double, Index - (threadIdx.x % DOF) + i);
      }
    }
  }
  ptr_y[reg2] = y_reg;
}



// Jeswin's kernel as of HG-tip
#if 0
__global__ void MatMul_Kernel_tex(double * ptr_coeff, double* ptr_x,
    double* ptr_y, PetscInt *idx, PetscInt m, PetscInt n, PetscInt p,
    PetscInt nos, PetscInt DOF) {

  int tx = blockDim.x * blockIdx.x + threadIdx.x;
  int l, i, offset;
  int lda1 = m * n * p * DOF, lda2 = m * p * DOF; //lda3=m*DOF
  PetscInt X_Index, Index;
  double y_reg = 0;
  int BAND_SIZE = (DOF - 1) * 2 + 1;

  if (threadIdx.x < nos) {
    idx_sm[threadIdx.x] = idx[threadIdx.x];
  }

  int reg2 = blockIdx.y * lda2 + tx;

  //Iterating through the Diagonals
  for (l = 0; l < nos; l++) {
    X_Index = reg2 + idx_sm[l];

    if (X_Index >= 0 && X_Index < lda1) {
      for (i = 0; i < BAND_SIZE; i++) {
        offset = (l * BAND_SIZE + i) * lda1;

        if (i > DOF - 1) {
          Index = X_Index - (i - (DOF - 1));
          if (Index < 0) {
            continue;
          } else {
            y_reg += ptr_coeff[offset + reg2]
                * fetch_double(tex_x_double, Index);
          }
        } else {
          Index = X_Index + i;

          y_reg += ptr_coeff[offset + reg2] * fetch_double(tex_x_double, Index);
        }
      }
    }
  }
  ptr_y[reg2] = y_reg;
}
#endif

//------------------------------------------------------------------------------------
//   The function is a wrapper function which sets up the device memory, transfers
//   data to and from the device, and calls the MatMult kernel.
//------------------------------------------------------------------------------------

int SGCUDA_MatMult(PetscScalar* coeff, PetscScalar* x, PetscScalar* y,
    PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n,
    PetscInt p, PetscInt stpoints, PetscCUSPFlag* fp, PetscInt DOF,
    PetscScalar** gpuMat) {
  double tbegin1, tbegin2, tend1, tend2;
  static PetscInt size_coeff;
  double tsetup,tkernel;
  static unsigned int kcalls=0;
  PetscInt size_xy, size_id;
  static double temp=0;
  //PetscScalar* d_x;
  //PetscScalar* d_y;
  PetscInt *d_linear_idx;
  int nos;

  int BLOCK_SIZE;
  int cons=m*DOF;
  int cons1=m*n*DOF;

  nos = DOF*stpoints;

#if _DBGFLAG
  printf("m: %d  n: %d  p: %d  nos: %d  stpoints: %d  DOF: %d\n", m, n, p, nos, stpoints, DOF);

  for (unsigned i = 0; i < stpoints; ++i) {
    printf("Stencil Point:  %d, %d, %d\n", idx[i], idy[i], idz[i]);
  }
#endif

  PetscInt *linear_idx;
  PetscMalloc(stpoints * sizeof(PetscInt), &linear_idx);

  for (unsigned i = 0; i < stpoints; ++i) {
    linear_idx[i] = idx[i]*DOF + idy[i]*cons + idz[i]*cons1;
  }
  //----Single offset instead of using three offsets int the x,y and z direction.
  //if (stpoints==5)
  //{
  //  idx[0]=0;
  //  idx[1]=DOF;
  //  idx[2]=cons;
  //  idx[3]=-DOF;
  //  idx[4]=-cons;
  //}
  //else if (stpoints==7)
  //{
  //  idx[0]=0;
  //  idx[1]=DOF;
  //  idx[2]=cons;
  //  idx[3]=cons1;
  //  idx[4]=-DOF;
  //  idx[5]=-cons;
  //  idx[6]=-cons1;
  //} else {
  //  printf("Bad value for stpoints\n");
  //  exit(EXIT_FAILURE);
  //}


  //------------Printing Matrices for Debugging---------------------------------
#ifdef PRINT
  printf("offset vector\n");
  for (int i=0;i<nos;i++)
  {
    printf("%d  ", idx[i]);
  }
  printf("\n");

  printf("Matrix X\n");
  for(int i=0;i<n;i++)
  {
    for(int j=0;j<m*DOF;j++)
    {
      printf("%0.2f   ",x[i*cons+j]);
    }
    //printf("\n");
  }
  printf("\n");
  printf("Matrix A\n");
  for(int s=0;s<DOF*nos;s++)
  {
    printf("\n");
    for(int i=0;i<n;i++)
    {
      for(int j=0;j<m*DOF;j++)
      {
        printf("%0.2f   ",coeff[s*cons1+i*cons+j]);
      }
      printf("\n");
    }

  }
#endif
  //------------------------------------------------------------------------------------------


#if(_DBGFLAG)
  tbegin1=getclock();
#endif

  if ((*fp == PETSC_CUSP_UNALLOCATED) ||
      (*fp == PETSC_CUSP_CPU) )
  {
    if (*fp == PETSC_CUSP_UNALLOCATED)
    {
#if _DBGFLAG
    		printf("Allocate GPU matrix (%p) container (%p)\n", *gpuMat, gpuMat);
    		fflush(stdout);
#endif
      size_coeff=nos*m*n*p*DOF*sizeof(PetscScalar);
      hipMalloc((void**)gpuMat,size_coeff);
      checkCUDAError("hipMalloc (d_coeff)");
#if _DBGFLAG
      printf("After malloc (%p)\n", *gpuMat);
      fflush(stdout);
#endif


      //cudastatus0=hipMalloc((void**)&devA,matsize);
      //if(cudastatus0!=hipSuccess)
      //	{
      //  printf("Error in devA memory allocation:\nstatus0: %s\n",
      //	hipGetErrorString(cudastatus0));
      //  PetscFunctionReturn(PETSC_ERR_MEM);
      //	}
    }

#if _DBGFLAG
    printf("- Memcpy host ptr %p and size %ld\n", coeff, size_coeff);
#endif
    hipMemcpy(*gpuMat, coeff, size_coeff, hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy (d_coeff)");

    //cudastatus1=hipMemcpy(devA,A,matsize,hipMemcpyHostToDevice);
    //if(cudastatus1!=hipSuccess)
    //{
    // if(devA) hipFree(devA);
    //  printf("Error in devA memory copying:\nstatus1: %s\n",
    //	hipGetErrorString(cudastatus1));
    //  PetscFunctionReturn(PETSC_ERR_MEM);
    //}

    *fp = PETSC_CUSP_BOTH;
  }


  //size_coeff=nos*m*n*p*sizeof(PetscScalar);
  //hipMalloc((void**)&d_coeff,size_coeff);
  //hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);


  size_xy = m*n*p*DOF*sizeof(PetscScalar);
  /*  hipMalloc((void**)&d_x,size_xy);
  checkCUDAError("hipMalloc (d_x)");
  hipMemcpy(d_x, x, size_xy, hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy (d_x)");

  hipMalloc((void**)&d_y,size_xy);
  checkCUDAError("hipMalloc (d_y)");
  hipMemcpy(d_y, y, size_xy, hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy (d_y)");
  */
  size_id = stpoints*sizeof(PetscInt);
  hipMalloc((void**)&d_linear_idx,size_id);
  checkCUDAError("hipMalloc (d_idx)");
  hipMemcpy(d_linear_idx, linear_idx, size_id, hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy (d_idx)");


  //Binding X to the texture Memory
  hipBindTexture(0, tex_x_double, x, size_xy);
  checkCUDAError("hipBindTexture");

#if(_DBGFLAG)
  //cudaPrintfInit();
  tend1=getclock();
  tsetup=tend1-tbegin1;
  tbegin2=getclock();
#endif


  // Kernel Setup and Configurations

  if ((DOF%2)!=0 || (DOF==6))
  {
    BLOCK_SIZE=BLOCKWIDTH_X-BLOCKWIDTH_X%DOF;
  }
  else{
    BLOCK_SIZE=BLOCKWIDTH_X;
  }

  //dim3 dimBlock(BLOCK_SIZE,BLOCKWIDTH_Y);
  //dim3 dimGrid((int)ceil((float)(m*p*DOF)/(float)BLOCK_SIZE),((int)ceil((float)(n)/(float)BLOCKWIDTH_Y)));

  dim3 dimBlock(BLOCK_SIZE,BLOCKWIDTH_Y);
  dim3 dimGrid((int)ceil((float)(m*n*p*DOF)/(float)BLOCK_SIZE), 1);

  PetscInt shared_size = stpoints * sizeof(PetscInt);

#if _DBGFLAG
  printf("Launch Bounds:\n");
  printf("dimBlock: %d, %d\n", dimBlock.x, dimBlock.y);
  printf("dimGrid:  %d, %d\n", dimGrid.x, dimGrid.y);

  for (unsigned i = 0; i < stpoints; ++i) {
    printf("linear_idx[%d] = %d\n", i, linear_idx[i]);
  }
#endif

  //if (DOF==1)
 // {
  //  MatMul_Kernel_tex_1_DOF<<<dimGrid,dimBlock,shared_size>>>(d_coeff, d_x, d_y, d_idx, m, n, p, nos,DOF);
  //}
  //else{
  printf("nos: %d  stpoints: %d  DOF: %d\n", nos, stpoints, DOF);
#if _DBGFLAG
  printf("Launch stpoints: %d\n", stpoints);
#endif
  MatMult_Kernel<<<dimGrid,dimBlock,shared_size>>>(*gpuMat, x, y, d_linear_idx, m, n, p, nos, stpoints, DOF);
  //}

  // check if kernel execution generated and error
  //cutilCheckMsg("Kernel execution failed");

  /*
	   if (m > BLOCKWIDTH){
	   // dim3 dimBlock(BLOCKWIDTH,BLOCKWIDTH);
	   // dim3 dimGrid((int)ceil((float)m/(float)BLOCKWIDTH),((int)ceil((float)n/(float)BLOCKWIDTH)));

	   dim3 dimBlock(BLOCKWIDTH,BLOCKWIDTH,BLOCKWIDTH);
	   dim3 dimGrid((int)ceil((float)m/(float)BLOCKWIDTH),((int)ceil((float)n/(float)BLOCKWIDTH)),p/BLOCKWIDTH);

	   // cutilCheckError(cutCreateTimer(&timer));
	   // cutilCheckError(cutStartTimer(timer));

	   MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);

	   }
	   else
	   {
	   //dim3 dimBlock(m,n);
	   dim3 dimBlock(m,n,p);
	   dim3 dimGrid(1,1,1);

	   // cutilCheckError(cutCreateTimer(&timer));
	   // cutilCheckError(cutStartTimer(timer));

	   MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);


	   }

   */


#if(_DBGFLAG)
  hipDeviceSynchronize();
  checkCUDAError("Launch/Sync");
  tend2=getclock();
  tkernel=tend2-tbegin2;
  //cudaPrintfDisplay(stdout, true);
  //cudaPrintfEnd();
#endif

  //Read y from the Device Memory

  //  hipMemcpy(y, d_y, size_xy, hipMemcpyDeviceToHost);
  // checkCUDAError("hipMemcpy (d_y) OUT");
  /*
	  int i;
	  char *fn = "/homes/dlowell/cudaexprs/dcheck/outfile_SG1.txt";
	  FILE *fptr;
	  fptr=fopen(fn,"a");
	  if(!fptr){
	  printf("file pointer error.\n");
	  PetscFunctionReturn(PETSC_ERR_LIB);
	  }else{
	  //printf("yy->map->n: %d\n",yy->map->n);
	  for(i=0;i<m*n*p*DOF;i++){
	  //printf("printed to file: %d\n",i);
	  if(y[i]!=0.)fprintf(fptr,"%f ",y[i]);
	  }
	  fclose(fptr);
	  }
   */

#if(_DBGFLAG)
  temp+=tkernel;
  if (kcalls==0)
  {
    printf("\n Structured Grid MatrixMul Kernel Permormance for m *%d* and n size *%d* \n",m,n);
  }
  //if (kcalls==1000)
  {
    printf("\ncopy time (sec) : %f\n",tsetup);
    printf("Kernel time (sec): %f\n",tkernel);
    printf("Performance in Megaflops with for %dth Kernel call\n",kcalls);
    printf("Performance in Megaflops with copy time = %f\n",(2*nos*n*m*p*1.0e-6)/(tsetup+tkernel));
    printf("Performance in Megaflops without copy time = %f\n",(2*nos*n*m*p*1.0e-6)/tkernel);
    printf("Culmative Performance in Megaflops for *%d* calls without copy time = %f\n",kcalls,(2*nos*n*m*p*1.0e-6)/(temp/(kcalls+1)));
  }
#endif
  kcalls++;


#ifdef PRINT
  for(int i=0;i<m*n;i++)
    printf("Y[%d]: %lf\n",i,y[i]);
#endif

  PetscFree(linear_idx);

  //Free Device Memory
  //hipFree(d_coeff);
  //hipFree(d_x);
  //hipFree(d_y);
  hipFree(d_linear_idx);

  return 0;
}




