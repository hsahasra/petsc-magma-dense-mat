#include "hip/hip_runtime.h"

/*  -------------------------------------------------------------------- 

     This file extends structgrid data type to make use of GPUS. The new data type
     is structgridgpu. The implementation of the new datatype emulates the seqaijcusp
     implementation which is an extension to aij matrix format. 
     Author: Chekuri S. Choudary, RNET
*/

#define PETSCMAT_DLL
#include "../src/mat/impls/structgrid/structgridgpu/matstructgridgpu.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>
#include <omp.h>
#include "../src/mat/impls/structgrid/matstructgrid.h"

#include "private/matimpl.h"
#include "matstructgridgpu.h"
#include "cuPrintf.cu"

// ----------------------------------------------------------
// hardcodiing the shared memory size this should be set
// to give maximum performance, however should be
// replaced soon with a more flexable dynamically allocated
// shared memory scheme
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
#define SHDSIZE 4


// -----------------------------------------------
// Structure for Constant Device memory
// storing constants and indices and index limits
// stencile size is hard coded
// written by: dlowell ANL-MCS
// -----------------------------------------------
#define STLSIZE 64
struct Stencilparams{
       int m;
       int n;
       int p;
       int vecsize_x;
       int vecsize_y;
       int matsize;
       int nos;
       int dof;
       int lda1;
       int lda2;
       int lda3;
       int idx[STLSIZE];
       int idy[STLSIZE];
       int idz[STLSIZE];
       int tile_x;
       int tile_y;
       int tile_z;
       int tilesize;
};//820 bytes

__constant__ Stencilparams devparams;//device memory

static double* devA;
static PetscScalar* d_coeff;



// ----------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//------------------------------------------------------
// general timer function using unix system call
// dlowell ANL-MCS
//------------------------------------------------------
double getclock(){
      struct timezone tzp;
      struct timeval tp;
      gettimeofday (&tp, &tzp);
      return (tp.tv_sec + tp.tv_usec*1.0e-6);
}


/*  --------------------------------------------------------------------
     This function destroys the matrix of type structgridgpu. It first 
     deallocates the memory on GPU and then calls the MatDestroy_SeqSG 
     function.
     Author: Chekuri S. Choudary, RNET
*/

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqSGGPU"
PetscErrorCode  MatDestroy_SeqSGGPU(Mat B)
{
  printf("Call to MatDestroy_SeqSGGPU(Mat B)\n");
  PetscErrorCode ierr;
  PetscFunctionBegin;

  if (B->valid_GPU_matrix != PETSC_CUSP_UNALLOCATED) 
	{
  	if (devA) hipFree(devA);
	if (d_coeff) hipFree(d_coeff);
	}

  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;

  ierr             = MatDestroy_SeqSG(B);CHKERRQ(ierr);

  PetscFunctionReturn(0);
}
EXTERN_C_END



/*  --------------------------------------------------------------------
     This function creates a datatype of structgridgpu. It first creates a
     structgrid datatype and overrides the matrix multiplication method.
     Author: Chekuri S. Choudary, RNET
*/

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode  MatCreate_SeqSGGPU(Mat B)
{

  printf("Call to MatCreate_SeqSGGPU(Mat B)\n");

  PetscErrorCode ierr;
  PetscFunctionBegin;

  ierr             = MatCreate_SeqSG(B);CHKERRQ(ierr);
  B->ops->mult     = MatMult_SeqSGGPU;
  B->ops->destroy  = MatDestroy_SeqSGGPU;

  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSTRUCTGRIDGPU);CHKERRQ(ierr);
  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
  PetscFunctionReturn(0);
}
EXTERN_C_END


//---------------------------------------------------------------------
//     This function implements matrix vector multiplication for the
//     structgridgpu datatype. It calls a CUDA kernel to do matrix
//     multiplication on the GPU.
//     Author: Daniel Lowell, ANL-MCS, Chekuri S. Choudary, RNET
//---------------------------------------------------------------------
EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y)
{
        int i;
	PetscErrorCode ierr;
	Mat_SeqSG * a = (Mat_SeqSG *) mat->data;
	PetscScalar * v = a->a, *xx,*yy;

	PetscFunctionBegin;
	ierr = VecSet(y,0.0); CHKERRQ(ierr);
	ierr = VecGetArray(x, &xx); CHKERRQ(ierr);
	ierr = VecGetArray(y, &yy); CHKERRQ(ierr);


        //set up parameters for constant memory
        struct Stencilparams sparams;
        for(i=0;i<a->stpoints;i++){
            sparams.idx[i]=a->idx[i];
            sparams.idy[i]=a->idy[i];
            sparams.idz[i]=a->idz[i];
        }
        sparams.m=a->m;
        sparams.n=a->n;
        sparams.p=a->p;
        VecGetLocalSize(x,&sparams.vecsize_x);
        VecGetLocalSize(y,&sparams.vecsize_y);
        sparams.nos = a->stpoints;
        sparams.dof = a->dof;
        sparams.lda1=a->m*a->n*a->p;
        sparams.lda2=a->m*a->n;
        sparams.lda3=a->m;
        sparams.matsize=a->m*a->n*a->p*a->stpoints;

        /// Debugging block .....................................................
            /*int xsize,ysize;
            //printf("Matrix A ::: m: %d, n: %d, p: %d, nos: %d dof: %d nz: %d\n",
            //    a->m,a->n,a->p,a->stpoints,a->dof,a->nz);
            //VecGetLocalSize(x,&xsize);
            //VecGetLocalSize(y,&ysize);
            //printf("Amat size: %d, Xvec size: %d, Yvec size: %d\n",sparams.matsize,xsize,ysize);
            */
            //static PetscInt count = 1;// running count of function calls
            //printf("MatMult_SeqSGGPU(Mat mat, Vec x, Vec y): %d\n",count++);
        ///....................................................................


// Call to dlowell's version
      ierr = SGCUDA_MatMult_v2(v,xx,yy,sparams,&(mat->valid_GPU_matrix)); 
//	CHKERRQ(ierr);

// Call to Jeswin's version
//        ierr = SGCUDA_MatMult(v,xx,yy,a->idx,a->idy,a->idz,a->m,a->n,a->p,a->stpoints, 
//	                     &(mat->valid_GPU_matrix));
        CHKERRQ(ierr);

       	ierr = VecRestoreArray(x,&xx); CHKERRQ(ierr);
	ierr = VecRestoreArray(y,&yy); CHKERRQ(ierr);
	ierr = PetscLogFlops(2*a->nz*a->stpoints); CHKERRQ(ierr);
	PetscFunctionReturn(0);
}
EXTERN_C_END


//-------------------------------------------------------------------------------
//   This function is the matrix vector multiplication kernel
//   structgridgpu datatype. This version uses shared memory for the write
//   back vector Y. Constant memory for reused constants and indices.
//   More offloading to registers might be possible as well.
//   written by: dlowell, ANL-MCS
//-------------------------------------------------------------------------------
__global__ void MatMul_Kernel_v2(double* A, double* X, double* Y){

   // indices for global accesses
   int btx = blockDim.x*blockIdx.x+threadIdx.x;
   int bty = blockDim.y*blockIdx.y+threadIdx.y;
   int btz = blockDim.z*blockIdx.z+threadIdx.z;

   //indices for local accesses
   int tx = threadIdx.x;
   int ty = threadIdx.y;
   int tz = threadIdx.z;
   int tbtx, tbty, tbtz;
   int ix,iy,iz,j;

   int nos  = devparams.nos; // set to register
   int lda1 = devparams.lda1;//  "   "
   int lda2 = devparams.lda2;//  "   "
   int lda3 = devparams.lda3;//  "   "
   int tilex = devparams.tile_x;
   int tiley = devparams.tile_y;
   int tilez = devparams.tile_z;

   int Aindex;
   int Yindex;
   int Xindex;

   int index;


   // Min shared mem byte count: (gridDim.x*gridDim.y*gridDim.z)*SHDSIZE^3*8 byte double
   __shared__ double Ys[SHDSIZE][SHDSIZE][SHDSIZE];
   __shared__ double As[SHDSIZE][SHDSIZE][SHDSIZE];

//------------------------------------------------------------------------

   for(j=0;j<nos;j++){

       for(ix=0;ix<tilex;ix+=devparams.tilesize){
       for(iy=0;iy<tiley;iy+=devparams.tilesize){
       for(iz=0;iz<tilez;iz+=devparams.tilesize){

           tbtx = btx + ix;
           tbty = bty + iy;
           tbtz = btz + iz;
           index = tbtz*lda2 + tbty*lda3 + tbtx;//tile width indices

           Ys[tz][ty][tx]=0.;
           Aindex = j*lda1 + index;
           As[tz][ty][tx]=A[Aindex];
           Xindex = (devparams.idz[j]*lda2 + devparams.idy[j]*lda3 + devparams.idx[j]) + index;
           __syncthreads();

           if (!((j==1 && tbtx==devparams.n-1 && tbty==devparams.m-1)||
                (j==2 && tbtx==0 && tbty==0)||
                (j==3 && tbty==devparams.m-1)||
                (j==4 && tbty==0))){
                   if(Aindex<devparams.matsize && Xindex < devparams.vecsize_x) {
                      Ys[tz][ty][tx]=As[tz][ty][tx]*X[Xindex];
                   }else Ys[tz][ty][tx]=0.;
           }//end if

           Yindex = index;
           if(Yindex<devparams.vecsize_y) Y[Yindex]+=Ys[tz][ty][tx];//global write back

       }//end ix-for
           __syncthreads();
       }//end iy-for
           __syncthreads();
       }//end iz-for
           __syncthreads();
   }//end j-for

}




//------------------------------------------------------------------------------------
//   This function is the wrapper function which sets up the device memory, transfers
//   data to and from the device, and calls the kernel. Error checking is done at 
//   each step. Timing stats are recorded using static vars.
//   written by: Daniel Lowell, ANL-MCS
//------------------------------------------------------------------------------------
PetscErrorCode SGCUDA_MatMult_v2(PetscScalar* A, PetscScalar* X, 
PetscScalar* Y, struct Stencilparams P, PetscCUSPFlag* fp){

        // vars for testing
        int i;
        static double cumtime=0.;//cummalitive call time
        static unsigned int kcalls=0;//number of kernel calls

        // using CUDA device timer
	float elapsedtime;
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	double cs,ce,temp;

	hipError_t cudastatus0,cudastatus1,
	            cudastatus2,cudastatus3,
	            cudastatus4,cudastatus5,
           	    cudastatus6,cudastatus7;


        int matsize =P.matsize*sizeof(double);
        int vecsize_x = P.vecsize_x*sizeof(double);
        int vecsize_y = P.vecsize_y*sizeof(double);

// 	allocate GPU device memory
	cs=getclock();

        //Allocate and Memcpy Structured Matrix A
	//The matrix remains the same throughout one iteration
        //of the linear solver. The following uses a flag
        //defined in the base class to check the status of the
        //matrix. The matrix is copied to the GPU only if
        //it has been changed on the CPU side
        //This feature added by Chekuri S. Choudary  
	
      if ((*fp == PETSC_CUSP_UNALLOCATED) ||
	  (*fp == PETSC_CUSP_CPU) )
	{
		if (*fp == PETSC_CUSP_UNALLOCATED)
		{
	   	cudastatus0=hipMalloc((void**)&devA,matsize);
	   	if(cudastatus0!=hipSuccess)
			{
		  printf("Error in devA memory allocation:\nstatus0: %s\n",
  			hipGetErrorString(cudastatus0));
          	  PetscFunctionReturn(PETSC_ERR_MEM);
			}
		}

           	cudastatus1=hipMemcpy(devA,A,matsize,hipMemcpyHostToDevice);
	   	if(cudastatus1!=hipSuccess)
		{
		  if(devA) hipFree(devA);
		  printf("Error in devA memory copying:\nstatus1: %s\n",
  			hipGetErrorString(cudastatus1));
          	  PetscFunctionReturn(PETSC_ERR_MEM);
		}

	       *fp = PETSC_CUSP_BOTH;
	}

        // Allocate and Memcpy Vector X
	double* devX;
	cudastatus2=hipMalloc((void**)&devX,vecsize_x);
	cudastatus3=hipMemcpy(devX,X,vecsize_x,hipMemcpyHostToDevice);
	if(cudastatus2!=hipSuccess|cudastatus3!=hipSuccess){
	  printf("Error in devX memory allocation:\nstatus2: %s, status3: %s.\n",
  			hipGetErrorString(cudastatus2),
			hipGetErrorString(cudastatus3));
	  if(devA) hipFree(devA);
	  if(devX) hipFree(devX);
          PetscFunctionReturn(PETSC_ERR_MEM);
	}

        // Allocate and Memset(0.) Vector Y
	double* devY;
	cudastatus4=hipMalloc((void**)&devY,vecsize_y);
	cudastatus5=hipMemset(devY,0.0,vecsize_y);
	if(cudastatus4!=hipSuccess|cudastatus5!=hipSuccess){
	  printf("Error in devY memory allocation:\nstatus4: %s, status5: %s\n",
	  		hipGetErrorString(cudastatus4),
			hipGetErrorString(cudastatus5));
	  if(devA) hipFree(devA);
	  if(devY) hipFree(devY);
          if(devX) hipFree(devX);
          PetscFunctionReturn(PETSC_ERR_MEM);
	}


        //Set up blocks and thread numbers
        unsigned int maxshared = floor(49152.0/(float)2*sizeof(double));///max shared elements (2*arrays)
        P.tilesize = ceil(pow(maxshared,(1.0/3.0)));

        int bx,by,bz;//number of blocks in 3-D
        int tx,ty,tz;//number of threads ber block in 3-D

        //Set up blocks and thread numbers
        if(P.m < SHDSIZE){
           tx = P.m;
           bx = 1;
           P.tile_x = 1;
        }else{
           tx = SHDSIZE;
           bx = ceil((float)P.m/(float)SHDSIZE);
           P.tile_x=ceil((float)bx*tx/(float)P.tilesize);
        }

        if(P.n < SHDSIZE){
           ty = P.n;
           by = 1;
           P.tile_y = 1;
        }else{
           ty = SHDSIZE;
           by = ceil((float)P.n/(float)SHDSIZE);
           P.tile_y=ceil((float)by*ty/(float)P.tilesize);
        }

        if(P.p < SHDSIZE){
           tz = P.p;
           bz = 1;
           P.tile_z = 1;
        }else{
           tz = SHDSIZE;
           bz = ceil((float)P.p/(float)SHDSIZE);
           P.tile_z=ceil((float)bz*tz/(float)P.tilesize);
        }

        dim3 dimGrid(bx,by,bz);
	dim3 dimBlock(tx,ty,tz);

        unsigned int sharesize = tx*ty*tz*bx*by*bz;
        unsigned int sharebytes = tx*ty*tz*bx*by*bz*sizeof(double);



        static unsigned char dbgflag = 1;
        if(dbgflag){
           printf("blocks: %d, threads: %d, Snum: %d SB: %d, Tile x,y,z: (%d, %d, %d),  tileEdge: %d\n",
                        bx*by*bz,tx*ty*tz,sharesize,sharebytes,P.tile_x,P.tile_y,P.tile_z,P.tilesize);
                dbgflag=0;
        }
        // update constant memory with structured grid parameters
	cudastatus6=hipMemcpyToSymbol(HIP_SYMBOL("devparams"),&P,sizeof(Stencilparams));
	if(cudastatus6!=hipSuccess){
	  printf("Error in symbol copy: status6: %s.\n",
	  		hipGetErrorString(cudastatus6));
	  if(devA) hipFree(devA);
	  if(devY) hipFree(devY);
	  if(devX) hipFree(devX);
          PetscFunctionReturn(PETSC_ERR_MEM);
	}


	ce=getclock();//end setup timer
	temp=ce-cs;

        cudaPrintfInit();//start cuda printf environ.
	hipEventRecord(start,0);//begin recording kernel
	MatMul_Kernel_v2<<<dimGrid,dimBlock>>>(devA,devX,devY);
        checkCUDAError("CUDA Kernel launch...");//check for failure
        cudaPrintfDisplay(stdout, true);//choose output
        cudaPrintfEnd();//kill cuda printf environ
	hipEventRecord(stop,0);
	hipEventSynchronize(stop); // event barrier
	hipEventElapsedTime(&elapsedtime,start,stop);
        hipEventDestroy(start);
	hipEventDestroy(stop);


        // Copy back Vector Y from Kernel
	cs=getclock();
	cudastatus7=hipMemcpy(Y,devY,vecsize_y,hipMemcpyDeviceToHost);
	if(cudastatus7!=hipSuccess){
          printf("Error on copy back Y, kernel status: %s\nExiting...\n\n",hipGetErrorString(cudastatus7));
	  if(devA) hipFree(devA);
	  if(devY) hipFree(devY);
	  if(devX) hipFree(devX);
          PetscFunctionReturn(PETSC_ERR_MEM);
        }
        //for(i=0;i<P.lda1;i++)printf("Y[%d]: %lf\n",i,Y[i]);//for verification


        //Free device memory
	//if(devA) hipFree(devA);
	if(devY) hipFree(devY);
	if(devX) hipFree(devX);

	ce=getclock();
	temp+=ce-cs;
        cumtime+=(elapsedtime/1000)+temp;
       // kcalls++;
       // printf("Cumilative kernel time (including setup): %lf msec.\n", cumtime);
      //  printf("Kernel call #: %d, setup+teardown: %f msec., elapsed time: %f msec.\n\n",
      //                 kcalls,temp,elapsedtime/1000);
        PetscFunctionReturn(0);
}


/*  -------------------------------------------------------------------- 
     The following is a CUDA kernel for matrix vector multiplication on 
     the GPU. The matrix is in a custom layout that facilitates better 
     memory accesses and vectorization. 
     Author: Chekuri S. Choudary, RNET
*/
__global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{
int tx=  blockDim.x * blockIdx.x + threadIdx.x;
int ty=  blockDim.y * blockIdx.y + threadIdx.y;
int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

for (l=0;l<nos;l++)
        {
        xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
        if (l==1 && tx==n-1 && ty==m-1)
        {
        	continue;
        }
        if (l==2 && tx==0 && ty==0)
        {
        	continue;
        }
        if (l==3 && ty==m-1)
        {
        	continue;
        }
        if (l==4 && ty==0)
        {
        	continue;
        }
        for(i=0;i<p;i++)
        	ptr_y[ i*lda2 + ty*lda3 + tx]+= (ptr_coeff[offset + i*lda2 + ty*lda3 +tx] * ptr_x[(i+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
        }
}


int SGCUDA_MatMult(PetscScalar* coeff, PetscScalar* x, PetscScalar* y, PetscInt *idx, PetscInt* idy, 
PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos, PetscCUSPFlag* fp)
{

//double tbegin3, tbegin4, tend3, tend4;
static PetscInt size_coeff; 
PetscInt size_xy, size_id; 

PetscScalar* d_x;
PetscScalar* d_y;
PetscInt *d_idx, *d_idy, *d_idz;

//fprintf(stdout,"%d\t%d\t%d\t%d\n",m,n,p,nos);

//loading the coeff, x, y, idx, idy, idz to device memory

  //unsigned int timer1 = 0;
  //cutilCheckError(cutCreateTimer(&timer1));
  //cutilCheckError(cutStartTimer(timer1));

  //  fprintf(stdout,"In SGCUDA_MatMult\n");
	
      if ((*fp == PETSC_CUSP_UNALLOCATED) ||
	  (*fp == PETSC_CUSP_CPU) )
	{
		if (*fp == PETSC_CUSP_UNALLOCATED)
		{
		size_coeff=nos*m*n*p*sizeof(PetscScalar);	
		hipMalloc((void**)&d_coeff,size_coeff);
	
	   	//cudastatus0=hipMalloc((void**)&devA,matsize);
	   	//if(cudastatus0!=hipSuccess)
		//	{
		//  printf("Error in devA memory allocation:\nstatus0: %s\n",
  		//	hipGetErrorString(cudastatus0));
          	//  PetscFunctionReturn(PETSC_ERR_MEM);
		//	}
		}
	
		hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);
	
           	//cudastatus1=hipMemcpy(devA,A,matsize,hipMemcpyHostToDevice);
	   	//if(cudastatus1!=hipSuccess)
		//{
		// if(devA) hipFree(devA);
		//  printf("Error in devA memory copying:\nstatus1: %s\n",
  		//	hipGetErrorString(cudastatus1));
          	//  PetscFunctionReturn(PETSC_ERR_MEM);
		//}
	
	        *fp = PETSC_CUSP_BOTH;
	}

//tbegin3 = rtclock();
//size_coeff=nos*m*n*p*sizeof(PetscScalar);
//hipMalloc((void**)&d_coeff,size_coeff);
//hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);


size_xy = m*n*p*sizeof(PetscScalar);
hipMalloc((void**)&d_x,size_xy); 
hipMemcpy(d_x, x, size_xy, hipMemcpyHostToDevice);

hipMalloc((void**)&d_y,size_xy); 
hipMemcpy(d_y, y, size_xy, hipMemcpyHostToDevice);

size_id = nos*sizeof(PetscInt);
hipMalloc((void**)&d_idx,size_id); 
hipMemcpy(d_idx, idx, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idy,size_id); 
hipMemcpy(d_idy, idy, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idz,size_id); 
hipMemcpy(d_idz, idz, size_id, hipMemcpyHostToDevice);

//cutilCheckError(cutStopTimer(timer1));
// kernel Configuration
if (m > 16){
dim3 dimBlock(16,16);
dim3 dimGrid((m/16),(n/16));

    // cutilCheckError(cutCreateTimer(&timer));
    // cutilCheckError(cutStartTimer(timer));

MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);

}
else
{
dim3 dimBlock(m,n);
dim3 dimGrid(1,1);
   
    // cutilCheckError(cutCreateTimer(&timer));
    // cutilCheckError(cutStartTimer(timer));

MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);


}

//Cuda Printf
//cudaPrintfInit();

//tbegin4 = rtclock();
// create and start timer
    //unsigned int timer = 0;
    //cutilCheckError(cutCreateTimer(&timer));
    //cutilCheckError(cutStartTimer(timer));

   // check if kernel execution generated and error
    	//cutilCheckMsg("Kernel execution failed");

   // stop and destroy timer
    	//cutilCheckError(cutStopTimer(timer));
		
//tend4 = rtclock();
//Read y from the Device Memory

hipMemcpy(y, d_y, size_xy, hipMemcpyDeviceToHost); 
 
// double time_sec=cutGetTimerValue(timer)/1000;
// double time_sec1=cutGetTimerValue(timer1)/1000;
   
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel : %f; time(sec): %f\n",(2*stpoints*csr_size*csr_size*1.0e-6/time_sec),time_sec);
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel setup time(sec) : %f\n",time_sec1);
    
// cutilCheckError(cutDeleteTimer(timer));
// cutilCheckError(cutDeleteTimer(timer1));
// tend3 = rtclock();
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel with copy time : %f; time: %f\n",2*stpoints*csr_size*csr_size*1.0e-6/(tend3-tbegin3),tend3-tbegin3);
// printf("MFLOPS: GPU Structured Grid Matrix Mult kernel : %f; time: %f\n",2*stpoints*csr_size*csr_size*1.0e-6/(tend4-tbegin4),tend4-tbegin4);
  
// printf("\n");
// printf("Matrix cuda y\n");

//  for(i=0;i<m;i++)
//  {
//    for(j=0;j<n;j++) 
//    {
//      printf("%.2f\n",y[i*n+j]);
//    }
//   printf("\n");
//  }


//Free Device Memory
//hipFree(d_coeff);
hipFree(d_x);
hipFree(d_y);
hipFree(d_idx);
hipFree(d_idy);
hipFree(d_idz);

return 0;
}




