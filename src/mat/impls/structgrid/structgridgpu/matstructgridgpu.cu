#include "hip/hip_runtime.h"
/*  -------------------------------------------------------------------- 

     This file extends structgrid data type to make use of GPUS. The new data type
     is structgridgpu. The implementation of the new datatype emulates the seqaijcusp
     implementation which is an extension to aij matrix format. 
     Author: Chekuri S. Choudary, RNET
*/

#define PETSCMAT_DLL
#include "../src/mat/impls/structgrid/structgridgpu/matstructgridgpu.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>
#include <string.h>
#include <omp.h>
#include "../src/mat/impls/structgrid/matstructgrid.h"

#include "private/matimpl.h"
#include "matstructgridgpu.h"
#include "cuPrintf.cu"

#define _DBGFLAG 1

//block size is 1x256. 
#define BLOCKWIDTH_X 4		
#define BLOCKWIDTH_Y 1   

// ----------------------------------------------------------
// hardcodiing the shared memory size this should be set
// to give maximum performance, however should be
// replaced soon with a more flexable dynamically allocated
// shared memory scheme
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
#define SHDSIZE 4


// -----------------------------------------------
// Structure for Constant Device memory
// storing constants and indices and index limits
// stencile size is hard coded
// written by: dlowell ANL-MCS
// -----------------------------------------------
#define STLSIZE 64
struct Stencilparams{
       int m;
       int n;
       int p;
       int vecsize_x;
       int vecsize_y;
       int matsize;
       int nos;
       int dof;
       int lda1;
       int lda2;
       int lda3;
       int idx[STLSIZE];
       int idy[STLSIZE];
       int idz[STLSIZE];
       int tile_x;
       int tile_y;
       int tile_z;
       int tsizex;
       int tsizey;
       int tsizez;
};//836 bytes

__constant__ Stencilparams devparams;//device memory



static double* devA;
static PetscScalar* d_coeff;
static double* devX;
static double* devY;




// ----------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// ----------------------------------------------------------
void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//------------------------------------------------------
// general timer function using unix system call
// dlowell ANL-MCS
//------------------------------------------------------
double getclock(){
      struct timezone tzp;
      struct timeval tp;
      gettimeofday (&tp, &tzp);
      return (tp.tv_sec + tp.tv_usec*1.0e-6);
}


/*  --------------------------------------------------------------------
     This function destroys the matrix of type structgridgpu. It first 
     deallocates the memory on GPU and then calls the MatDestroy_SeqSG 
     function.
     Author: Chekuri S. Choudary, RNET
*/

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatDestroy_SeqSGGPU"
PetscErrorCode  MatDestroy_SeqSGGPU(Mat B)
{
  printf("Call to MatDestroy_SeqSGGPU(Mat B)\n");
  PetscErrorCode ierr;
  PetscFunctionBegin;

  if (B->valid_GPU_matrix != PETSC_CUSP_UNALLOCATED) 
	{
  	if (devA) hipFree(devA);
	if (d_coeff) hipFree(d_coeff);
	if(devY) hipFree(devY);
	if(devX) hipFree(devX);
	}

  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;

  ierr             = MatDestroy_SeqSG(B);CHKERRQ(ierr);

  PetscFunctionReturn(0);
}
EXTERN_C_END



/*  --------------------------------------------------------------------
     This function creates a datatype of structgridgpu. It first creates a
     structgrid datatype and overrides the matrix multiplication method.
     Author: Chekuri S. Choudary, RNET
*/

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatCreate_SeqSGGPU"
PetscErrorCode  MatCreate_SeqSGGPU(Mat B)
{

  printf("Call to MatCreate_SeqSGGPU(Mat B)\n");

  PetscErrorCode ierr;
  PetscFunctionBegin;

  ierr             = MatCreate_SeqSG(B);CHKERRQ(ierr);
  B->ops->mult     = MatMult_SeqSGGPU;
  B->ops->destroy  = MatDestroy_SeqSGGPU;

  ierr = PetscObjectChangeTypeName((PetscObject)B,MATSTRUCTGRIDGPU);CHKERRQ(ierr);
  B->valid_GPU_matrix = PETSC_CUSP_UNALLOCATED;
  PetscFunctionReturn(0);
}
EXTERN_C_END


//---------------------------------------------------------------------
//     This function implements matrix vector multiplication for the
//     structgridgpu datatype. It calls a CUDA kernel to do matrix
//     multiplication on the GPU.
//     Author: Daniel Lowell, ANL-MCS, Chekuri S. Choudary, RNET
//---------------------------------------------------------------------
EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "MatMult_SeqSGGPU"
PetscErrorCode MatMult_SeqSGGPU(Mat mat, Vec x, Vec y)
{
        int i;
	PetscErrorCode ierr;
	Mat_SeqSG * a = (Mat_SeqSG *) mat->data;
	PetscScalar * v = a->a, *xx,*yy;

	PetscFunctionBegin;
	ierr = VecSet(y,0.0); CHKERRQ(ierr);
	ierr = VecGetArray(x, &xx); CHKERRQ(ierr);
	ierr = VecGetArray(y, &yy); CHKERRQ(ierr);


        //set up parameters for constant memory
        struct Stencilparams sparams;
        for(i=0;i<a->stpoints;i++){
            sparams.idx[i]=a->idx[i];
            sparams.idy[i]=a->idy[i];
            sparams.idz[i]=a->idz[i];
        }
        sparams.m=a->m;
        sparams.n=a->n;
        sparams.p=a->p;
        VecGetLocalSize(x,&sparams.vecsize_x);
        VecGetLocalSize(y,&sparams.vecsize_y);
        sparams.nos = a->stpoints;
        sparams.dof = a->dof;
	sparams.lda3=a->m*a->dof;
	sparams.lda2=sparams.lda3*a->n;
        sparams.lda1=sparams.lda2*a->p;
        sparams.matsize=sparams.lda1*a->stpoints;

        /// Debugging block .....................................................
            /*int xsize,ysize;
            //printf("Matrix A ::: m: %d, n: %d, p: %d, nos: %d dof: %d nz: %d\n",
            //    a->m,a->n,a->p,a->stpoints,a->dof,a->nz);
            //VecGetLocalSize(x,&xsize);
            //VecGetLocalSize(y,&ysize);
            //printf("Amat size: %d, Xvec size: %d, Yvec size: %d\n",sparams.matsize,xsize,ysize);
            */
            //static PetscInt count = 1;// running count of function calls
            //printf("MatMult_SeqSGGPU(Mat mat, Vec x, Vec y): %d\n",count++);
        ///....................................................................


// Call to dlowell's version
    //ierr = SGCUDA_MatMult_v2(v,xx,yy,sparams,&(mat->valid_GPU_matrix));
	// CHKERRQ(ierr);

// Call to Jeswin's version
    ierr = SGCUDA_MatMult(v,xx,yy,a->idx,a->idy,a->idz,a->m,a->n,a->p,a->stpoints,&(mat->valid_GPU_matrix),a->dof);CHKERRQ(ierr);

    ierr = VecRestoreArray(x,&xx); CHKERRQ(ierr);
	ierr = VecRestoreArray(y,&yy); CHKERRQ(ierr);
	ierr = PetscLogFlops(2*a->nz*a->stpoints); CHKERRQ(ierr);
	PetscFunctionReturn(0);
}
EXTERN_C_END


//-------------------------------------------------------------------------------
//   This function is the matrix vector multiplication kernel
//   structgridgpu datatype. This version uses shared memory for the write
//   back vector Y. Constant memory for reused constants and indices.
//   More offloading to registers might be possible as well.
//   written by: dlowell, ANL-MCS
//-------------------------------------------------------------------------------
__global__ void MatMul_Kernel_v2(double* A, double* X, double* Y){

   //indices for local accesses
   int tbtx, tbty, tbtz;
   int ix,iy,iz,j;

   int nos  = devparams.nos; // set to register
   int lda1 = devparams.lda1;//  "   "
   int lda2 = devparams.lda2;//  "   "
   int lda3 = devparams.lda3;//  "   "
   int tilex = devparams.tile_x*devparams.tsizex;
   int tiley = devparams.tile_y*devparams.tsizey;
   int tilez = devparams.tile_z*devparams.tsizez;
   //int offset = 0;

   int Aindex;
   int Xindex;
   int index;
   int offset;

   //Min shared mem byte count: 2*(gridDim.x*gridDim.y*gridDim.z)*SHDSIZE^3*8 byte double
   __shared__ double Ys[SHDSIZE][SHDSIZE][SHDSIZE];
   __shared__ double As[SHDSIZE][SHDSIZE][SHDSIZE];

//------------------------------------------------------------------------

   for(iz=0;iz<tilez;iz+=devparams.tsizez){// tiles Z loop
        tbtz = blockDim.z*blockIdx.z+threadIdx.z + iz;

   for(iy=0;iy<tiley;iy+=devparams.tsizey){// tiles Y loop
        tbty = blockDim.y*blockIdx.y+threadIdx.y + iy;

   for(ix=0;ix<tilex;ix+=devparams.tsizex){// tiles X loop
        tbtx = blockDim.x*blockIdx.x+threadIdx.x + ix;

        //cuPrintf("tid xyz: %d, %d, %d\n",tbtx,tbty,tbtz);
        //initialize current return-tile
        Ys[threadIdx.z][threadIdx.y][threadIdx.x]=0.;

        //adjusted index for global access
        index = tbtz*lda2 + tbty*lda3 + tbtx;

//......STENCIL...........................................
        for(j=0;j<nos;j++){//loop over stencil pattern
	   offset= j*lda1;
           Aindex=offset+index;//set up Aindex and read from global A into As tile
           if(Aindex<devparams.matsize) As[threadIdx.z][threadIdx.y][threadIdx.x]=A[Aindex];//needs to be coalesced
           else As[threadIdx.z][threadIdx.y][threadIdx.x]=0.;

           __syncthreads();

           //set up Xindex for element-wise operation using stencil pattern
           Xindex=(devparams.idz[j]*lda2 + devparams.idy[j]*lda3 + devparams.idx[j]) + index;
           if(Xindex<devparams.vecsize_x && Xindex>=0){
              Ys[threadIdx.z][threadIdx.y][threadIdx.x]+=As[threadIdx.z][threadIdx.y][threadIdx.x]*X[Xindex];
           }

        }//end j-for

        if(index<devparams.vecsize_y) Y[index]=Ys[threadIdx.z][threadIdx.y][threadIdx.x];//global write back

   }//end ix-for
   }//end iy-for
   }//end iz-for
}//end kernel_v2






//   int Xoffset;
              //Xoffset=(devparams.idz[j]*lda2+devparams.idy[j]*lda3+devparams.idx[j]);
              //  cuPrintf("Xindex: %d, Xoffset: %d, Xsize: %d idx: %d, idy: %d, idz: %d, j: %d nos: %d\n",Xindex,Xoffset,devparams.vecsize_x,devparams.idx[j],devparams.idy[j],devparams.idz[j],j,nos);


//------------------------------------------------------------------------------------
//   This function is the wrapper function which sets up the device memory, transfers
//   data to and from the device, and calls the kernel. Error checking is done at 
//   each step. Timing stats are recorded using static vars.
//   written by: Daniel Lowell, ANL-MCS
//------------------------------------------------------------------------------------
PetscErrorCode SGCUDA_MatMult_v2(PetscScalar* A, PetscScalar* X, 
PetscScalar* Y, struct Stencilparams P, PetscCUSPFlag* fp){

        // vars for testing
        int i;
        static double cumktime=0.;//cummalitive kernel time
        static double cumtime=0.;//cummalitive call time
        static unsigned int kcalls=0;//number of kernel calls
	double cs,ce,temp;
	float elapsedtime;        // using CUDA device timer
	hipEvent_t start,stop;

        static unsigned char allocflag = 1;
        static double maxshared;
        static int bx,by,bz;//number of blocks in 3-D
        static int tx,ty,tz;//number of threads ber block in 3-D
        static int maxblocks_xy;
        static int maxblocks_z;
        static dim3 dimGrid;
	static dim3 dimBlock;
        static int xytile;

	hipError_t cudastatus0,cudastatus1,
	            cudastatus2,cudastatus3,
	            cudastatus4,cudastatus5,
           	    cudastatus6,cudastatus7;


        //size in bytes to be allocated onto device
        int matsize =P.matsize*sizeof(double);
        int vecsize_x = P.vecsize_x*sizeof(double);
        int vecsize_y = P.vecsize_y*sizeof(double);


        if(_DBGFLAG){//create CUDA events for timer
           hipEventCreate(&start);
           hipEventCreate(&stop);
        }


	if(_DBGFLAG) cs=getclock();

        //Allocate and Memcpy Structured Matrix A
	//The matrix remains the same throughout one iteration
        //of the linear solver. The following uses a flag
        //defined in the base class to check the status of the
        //matrix. The matrix is copied to the GPU only if
        //it has been changed on the CPU side
        //This feature added by Chekuri S. Choudary

        if ((*fp == PETSC_CUSP_UNALLOCATED) || (*fp == PETSC_CUSP_CPU)){
		if (*fp == PETSC_CUSP_UNALLOCATED){
	   	   cudastatus0=hipMalloc((void**)&devA,matsize);
	   	   if(cudastatus0!=hipSuccess){
                        printf("Error in devA memory allocation:\nstatus0: %s\n",
  			hipGetErrorString(cudastatus0));
          	        PetscFunctionReturn(PETSC_ERR_MEM);
		   }
		}

           	cudastatus1=hipMemcpy(devA,A,matsize,hipMemcpyHostToDevice);
	   	if(cudastatus1!=hipSuccess){
		  if(devA) hipFree(devA);
		  printf("Error in devA memory copying:\nstatus1: %s\n",
  			hipGetErrorString(cudastatus1));
          	  PetscFunctionReturn(PETSC_ERR_MEM);
		}

	       *fp = PETSC_CUSP_BOTH;
	}


        //Allocate device memory for X and Y, and shape grid and blocks
        if(allocflag){
                cudastatus2=hipMalloc((void**)&devX,vecsize_x);//allocate X on device
	        if(cudastatus2!=hipSuccess){
                        printf("Error in devX memory allocation: %s\n",hipGetErrorString(cudastatus2));
	                if(devA) hipFree(devA);
                        PetscFunctionReturn(PETSC_ERR_MEM);
                }

                cudastatus3=hipMalloc((void**)&devY,vecsize_y);//allocate Y on device
                if(cudastatus3!=hipSuccess){
                        printf("Error in devY memory allocation: %s\n",hipGetErrorString(cudastatus3));
	                if(devA) hipFree(devA);
	                if(devX) hipFree(devX);
                        PetscFunctionReturn(PETSC_ERR_MEM);
	        }



                //Set up blocks and thread numbers
                maxshared = 49152.0/(double)(2.0*sizeof(double));
                if(P.p==1){
                    xytile = pow(maxshared,0.5);//square blocks
                    maxblocks_z = 1;
                }else{
                    temp=maxshared/P.p;//lop off z
                    xytile = pow(temp,0.5);//xyblocks
                    maxblocks_z=ceil((float)SHDSIZE/(float)P.p);
                }
                maxblocks_xy = xytile/SHDSIZE;


                //Set up blocks and thread numbers for columns
                if(P.m <= SHDSIZE){
                       tx = P.m;
                       bx = 1;
                       P.tile_x = 1;
                       P.tsizex=1;
                }else{
                       tx = SHDSIZE;
                       bx = ceil((float)P.m/(float)SHDSIZE);//create enough blocks
                       if(bx>maxblocks_xy){                    //too many blocks created
                          bx = maxblocks_xy;                   //set to max number of blocks allowed
                          P.tile_x=ceil((float)P.m/(float)(bx*SHDSIZE));//number of tiles
                          P.tsizex=bx*SHDSIZE;              //tilesize is block-thread coverage
                       }else{
                          P.tile_x=1;
                          P.tsizex=1;
                       }
                }

                //Set up blocks and thread numbers for rows
                if(P.n <= SHDSIZE){
                       ty = P.n;
                       by = 1;
                       P.tile_y = 1;
                       P.tsizey=1;
                }else{
                       ty = SHDSIZE;
                       by = ceil((float)P.n/(float)SHDSIZE);
                       if(by > maxblocks_xy){
                          by = maxblocks_xy;
                          P.tile_y=ceil((float)P.n/(float)(by*SHDSIZE));
                          P.tsizey=by*SHDSIZE;
                       }else{
                          P.tile_y=1;
                          P.tsizey=1;
                       }
                }

                //Set up blocks and thread numbers for z
                if(P.p <= SHDSIZE){
                       tz = P.p;
                       bz = 1;
                       P.tile_z = 1;
                       P.tsizez=1;
                }else{
                       tz = SHDSIZE;
                       bz = ceil((float)P.p/(float)SHDSIZE);
                       if(bz > maxblocks_z){
                          bz = maxblocks_z;
                          P.tile_z=ceil((float)P.p/(float)(bz*SHDSIZE));
                          P.tsizez=bz*SHDSIZE;
                       }else{
                          P.tile_z=1;
                          P.tsizez=1;
                       }
                }

                //set grid shape
                dimGrid.x = bx;
                dimGrid.y = by;
                dimGrid.z = bz;

                //set block shape
                dimBlock.x = tx;
                dimBlock.y = ty;
                dimBlock.z = tz;

                // update constant memory with structured grid parameters
	        cudastatus6=hipMemcpyToSymbol(HIP_SYMBOL("devparams"),&P,sizeof(Stencilparams));
	        if(cudastatus6!=hipSuccess){
                        printf("Error in symbol copy to device: %s.\n",hipGetErrorString(cudastatus6));
                        if(devA) hipFree(devA);
	                if(devY) hipFree(devY);
	                if(devX) hipFree(devX);
                        PetscFunctionReturn(PETSC_ERR_MEM);
	        }

                //toggle off allocation flag
                allocflag = 0;

        }//end allocflag-if

        //grid and block shape & device config. debugging.....................................
        unsigned int sharebytes = 2*tx*ty*tz*bx*by*bz*sizeof(double);
        static unsigned char dbgflag = 1;
        if(dbgflag){
           printf("(m, n, p, nos): (%d, %d, %d, %d)\n",P.m,P.n,P.p,P.nos);
           printf("MAXBLOCKS: %d maxshared: %lf\n",maxblocks_xy+maxblocks_z,maxshared);
           printf("blocks: (%d, %d, %d), threads per block: %d\n", bx,by,bz,tx*ty*tz );
           printf("Shared elements occupied: %0.3f SharedOccupied in Bytes: %d\n",sharebytes/49152.0,sharebytes);
           printf("Blocks x,y,z: (%d, %d, %d)\n",bx,by,bz);
           printf("Blocks*ThreadsPer size x,y,z: (%d, %d, %d)\n",bx*tx,by*ty,bz*tz);
           printf("Tiles (x,y,z): (%d, %d, %d)\n",P.tile_x,P.tile_y,P.tile_z);
           printf("Tile Size (x,y,z): (%d, %d, %d)\n",P.tsizex,P.tsizey,P.tsizez);
           dbgflag=0;
        }
        //...End config. debug section.................................................................




       //copy over values of X to device memory
	cudastatus4=hipMemcpy(devX,X,vecsize_x,hipMemcpyHostToDevice);
	if(cudastatus4!=hipSuccess){
                printf("Error in devX memory copy to device: status: %s\n",hipGetErrorString(cudastatus4));
	        if(devA) hipFree(devA);
	        if(devX) hipFree(devX);
	        if(devY) hipFree(devY);
                PetscFunctionReturn(PETSC_ERR_MEM);
	}

//probably an unnecessary step.
        // memset to 0. Vector Y on device
	cudastatus5=hipMemset(devY,0.0,vecsize_y);
	if(cudastatus5!=hipSuccess){
                printf("Error in devY memset to device: %s\n",hipGetErrorString(cudastatus5));
	        if(devA) hipFree(devA);
                if(devY) hipFree(devY);
                if(devX) hipFree(devX);
                PetscFunctionReturn(PETSC_ERR_MEM);
	}


        //toggle timer and debug settings
        if(_DBGFLAG){
                ce=getclock();//end setup timer
	        temp=ce-cs;
                cudaPrintfInit();//start cuda printf environ.
	        hipEventRecord(start,0);//begin recording kernel
        }

        //Launch the kernel..........................................
	MatMul_Kernel_v2<<<dimGrid,dimBlock>>>(devA,devX,devY);
        checkCUDAError("CUDA Kernel launch...");//check for failure
        //...........................................................

        //toggle timer and debug settings
        if(_DBGFLAG){
                cudaPrintfDisplay(stdout, true);//choose output
                cudaPrintfEnd();//kill cuda printf environ
	        hipEventRecord(stop,0);
	        hipEventSynchronize(stop); // event barrier
	        hipEventElapsedTime(&elapsedtime,start,stop);
                hipEventDestroy(start);
	        hipEventDestroy(stop);
        }

        // Copy back Vector Y from Kernel
	cs=getclock();
	cudastatus7=hipMemcpy(Y,devY,vecsize_y,hipMemcpyDeviceToHost);
	if(cudastatus7!=hipSuccess){
          printf("Error on copy back Y, kernel status: %s\nExiting...\n\n",hipGetErrorString(cudastatus7));
	  if(devA) hipFree(devA);
	  if(devY) hipFree(devY);
	  if(devX) hipFree(devX);
          PetscFunctionReturn(PETSC_ERR_MEM);
        }

	//for(i=0;i<P.lda1;i++)printf("Y[%d]: %lf\n",i,Y[i]);//for verification


        if(_DBGFLAG){
          for(i=0;i<P.lda1;i++)printf("Y[%d]: %lf\n",i,Y[i]);//for verification
	  ce=getclock();
	  temp+=ce-cs;
          cumktime+=(elapsedtime/1000);
          cumtime+=(elapsedtime/1000)+temp;
          kcalls++;
          printf("Kernel call #: %d\n",kcalls);
          printf("setup+copyback: %f sec.\nelapsed time: %f sec.\ntotal call time: %f sec.\n",
                  temp,elapsedtime/1000,(elapsedtime/1000)+temp);
          printf("Cum. kernel time: %lf sec.\n", cumktime);
          printf("Cum. call time (with setup): %lf sec.\n", cumtime);
          printf(".........................................\n\n");
        }//end _DBGFLAG-if

        PetscFunctionReturn(0);
}





/*  -------------------------------------------------------------------- 
     The following is a CUDA kernel for matrix vector multiplication on 
     the GPU. The matrix is in a custom layout that facilitates better 
     memory accesses and vectorization. 
     Author: Chekuri S. Choudary, RNET
*/


 //Version with Shared memory for X only supports rectangular tiles.
 /* __global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

int tx= blockDim.x * blockIdx.x + threadIdx.x;
int ty= blockDim.y * blockIdx.y + threadIdx.y;

int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

__shared__ PetscScalar y_sm[256];

// initializing to the zero
y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x]=0;
for (l=0;l<nos;l++)
	{
	xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
	if (tx > n-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (ty > m-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (l==1 && tx==n-1 && ty==m-1)
	{
	continue;
	}
	if (l==2 && tx==0 && ty==0)
	{
	continue;
	}
	if (l==3 && ty==m-1)
	{
	continue;
	}
	if (l==4 && ty==0)
	{
	continue;
	}
	for(i=0;i<p;i++)
	y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x]+= (ptr_coeff[offset + i*lda2 + ty*lda3 +tx] * ptr_x[(i+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
	}
	
	ptr_y[ty*lda3 + tx]= y_sm[threadIdx.y*BLOCKWIDTH_X + threadIdx.x];
}
  */
/*  #define BLOCKWIDTH 8
#define BLOCKWIDTH_X 8
#define BLOCKWIDTH_Y 8
#define BLOCKWIDTH_Z 8 
  
  __global__ void MatMult_Kernel(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt* idy, PetscInt* idz, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos)
{

int tx= blockDim.x * blockIdx.x + threadIdx.x;
int ty= blockDim.y * blockIdx.y + threadIdx.y;
int tz= blockDim.z * blockIdx.z + threadIdx.z;
int l,i;
int xdisp,ydisp,zdisp,offset;
int lda1=m*n*p,lda2=m*n,lda3=m;

__shared__ PetscScalar y_sm[512];

// initializing to the zero
y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y + threadIdx.y*BLOCKWIDTH_X + threadIdx.x]=0;
for (l=0;l<nos;l++)
	{
	xdisp = idx[l]; ydisp = idy[l]; zdisp = idz[l]; offset = l*lda1;
	if (tx > n-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (ty > m-1)
	{
	break; //use Break and test performance later(divergence)
	}
	if (tz > p-1)
	{
	break;
	}
	if (l==1 && tx==n-1 && ty==m-1 && tz==p-1)
	{
	continue;
	}
	if (l==2 && tx==0 && ty==0 && tz==0)
	{
	continue;
	}
	if (l==3 && ty==m-1)
	{
	continue;
	}
	if (l==4 && ty==0)
	{
	continue;
	}
	if (l==5 && tz==p-1)
	{
	continue;
	}
	if (l==6 && tz==0)
	{
	continue;
	}
	//for(i=0;i<p;i++)
	y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y + threadIdx.y*BLOCKWIDTH_X + threadIdx.x]+= (ptr_coeff[offset + tz*lda2 + ty*lda3 +tx] * ptr_x[(tz+zdisp)*lda2 + (ty+ydisp)*lda3 + (tx+xdisp)]);
	}
	
	ptr_y[tz*lda2+ ty*lda3 + tx]= y_sm[threadIdx.z*BLOCKWIDTH_X*BLOCKWIDTH_Y +threadIdx.y*BLOCKWIDTH_X + threadIdx.x];
}
 */   


//------------------------------------------------------------------------------------
//   These functions are used to bind and unbind the Vector x to the texture Memory.	   
//------------------------------------------------------------------------------------ 
texture<int2, 1> tex_x_double;

void unbind_x( double * x)
 {   
 hipUnbindTexture(tex_x_double); 
 }

static __inline__ __device__ double fetch_double(texture<int2, 1> tex_x_double, int i)
{
    int2 v = tex1Dfetch(tex_x_double,i);
    return __hiloint2double(v.y, v.x);
}
 
 
 
 
//------------------------------------------------------------------------------------
//  Below functions are SPMV kernel functions where x through the Texture Memory, offsets are accesed
//	through the Shared Memory, Y is accessed per thread from registers. Coeff accesses are from the global Memory 
//	but they are coalesced.  	   
//------------------------------------------------------------------------------------ 
#define stpoints 5 // I have to fix this 

__global__ void MatMul_Kernel_tex_1_DOF(PetscScalar * ptr_coeff, PetscScalar* ptr_x, PetscScalar* ptr_y, PetscInt *idx, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos,PetscInt DOF)
	{
		
		__shared__ float idx_sm[stpoints];
		
		int tx= blockDim.x * blockIdx.x + threadIdx.x;
		int l,offset;
		int lda1=m*n*p*DOF,lda2=m*p*DOF;  //lda3=m*DOF
		PetscInt Index;
		PetscScalar y_reg=0;
		
		if (threadIdx.x < stpoints)
			{
			idx_sm[threadIdx.x]=idx[threadIdx.x];
			}
		int reg2=blockIdx.y*lda2+tx;
		
		//Iterating through the Diagonals
		for (l=0;l<stpoints;l++)
			{
			Index =reg2 + idx_sm[l];
				
			if (Index >= 0 && Index <lda1)
				{
				offset = l*lda1;
				y_reg+= ptr_coeff[offset + reg2] * fetch_double(tex_x_double,Index);
				
			
				  if (threadIdx.y==0){
							cuPrintf("l= %d ptr_coeff= %f X= %f Index =%d y_sm=%f \n",l,ptr_coeff[offset + reg2],tex1Dfetch(tex_x_double,Index),Index, y_reg);
						}  
					
				}
			}
							
				
				ptr_y[reg2]= y_reg;
	}

	
__global__ void MatMul_Kernel_tex(double * ptr_coeff, double* ptr_x, double* ptr_y, PetscInt *idx, PetscInt m, PetscInt n ,PetscInt p, PetscInt nos,PetscInt DOF)
	{
		
		__shared__ float idx_sm[stpoints];
		
		int tx= blockDim.x * blockIdx.x + threadIdx.x;
		int l,i,offset;
		int lda1=m*n*p*DOF,lda2=m*p*DOF; //lda3=m*DOF
		PetscInt X_Index,Index;
		double y_reg=0;
		int BAND_SIZE=(DOF-1)*2+1;
		
		if (threadIdx.x < stpoints)
			{
			idx_sm[threadIdx.x]=idx[threadIdx.x];
			}
		
		
		int reg2=blockIdx.y*lda2+tx;
		
		//Iterating through the Diagonals
		for (l=0;l<stpoints;l++)
			{
			X_Index =reg2 + idx_sm[l];
				
			if (X_Index >= 0 && X_Index <lda1)
				{
				for (i=0;i<BAND_SIZE;i++)
					{
					offset = (l*BAND_SIZE+i)*lda1;
									
						if (i > DOF-1)
							{
							Index =X_Index-(i-(DOF-1));
							if (Index < 0)
								{
								continue;
								}
							else{
								y_reg+= ptr_coeff[offset + reg2] * fetch_double(tex_x_double,Index) ;
								}
							}	
						else {
							Index=X_Index+i;
							
							y_reg+= ptr_coeff[offset + reg2] * fetch_double(tex_x_double,Index);
							}
			
				  /* if (threadIdx.y==0){
							cuPrintf("l= %d ptr_coeff= %f X= %f X_Index =%d Index =%d y_sm=%f \n",l,ptr_coeff[offset + reg2],tex1Dfetch(tex_x_double,Index),X_Index,Index, y_reg);
						} */  
					}
				}
			}
							
				
				ptr_y[reg2]= y_reg;
	}
	

//------------------------------------------------------------------------------------
//   The function is a wrapper function which sets up the device memory, transfers
//   data to and from the device, and calls the MatMult kernel. 
//------------------------------------------------------------------------------------ 
 
 int SGCUDA_MatMult(PetscScalar* coeff, PetscScalar* x, PetscScalar* y, PetscInt *idx, PetscInt* idy, 
PetscInt* idz, PetscInt m, PetscInt n,PetscInt p, PetscInt nos, PetscCUSPFlag* fp,PetscInt DOF)
{

double tbegin1, tbegin2, tend1, tend2;
static PetscInt size_coeff; 
double tsetup,tkernel;
static unsigned int kcalls=0;
PetscInt size_xy, size_id; 
static double temp=0;
PetscScalar* d_x;
PetscScalar* d_y;
PetscInt *d_idx, *d_idy, *d_idz;
int BLOCK_SIZE;
int cons=m*DOF;
int cons1=m*n*DOF;

 //Reducing to a Single offset instead of using three offsets int the x,y and z direction.  
  
  idx[0]=0;
  idx[1]=DOF;
  idx[2]=-DOF;
  idx[3]=cons;
  idx[4]=-cons;
  if(nos==7)
    {
     idx[5]=cons1;
     idx[6]=-cons1;       
    }

	
      if(_DBGFLAG) tbegin1=getclock();
	  
	  if ((*fp == PETSC_CUSP_UNALLOCATED) ||
	  (*fp == PETSC_CUSP_CPU) )
	{
		if (*fp == PETSC_CUSP_UNALLOCATED)
		{
		size_coeff=nos*m*n*p*DOF*sizeof(PetscScalar);	
		hipMalloc((void**)&d_coeff,size_coeff);
	
	   	//cudastatus0=hipMalloc((void**)&devA,matsize);
	   	//if(cudastatus0!=hipSuccess)
		//	{
		//  printf("Error in devA memory allocation:\nstatus0: %s\n",
  		//	hipGetErrorString(cudastatus0));
          	//  PetscFunctionReturn(PETSC_ERR_MEM);
		//	}
		}
	
		hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);
	
           	//cudastatus1=hipMemcpy(devA,A,matsize,hipMemcpyHostToDevice);
	   	//if(cudastatus1!=hipSuccess)
		//{
		// if(devA) hipFree(devA);
		//  printf("Error in devA memory copying:\nstatus1: %s\n",
  		//	hipGetErrorString(cudastatus1));
          	//  PetscFunctionReturn(PETSC_ERR_MEM);
		//}
	
	        *fp = PETSC_CUSP_BOTH;
	}


//size_coeff=nos*m*n*p*sizeof(PetscScalar);
//hipMalloc((void**)&d_coeff,size_coeff);
//hipMemcpy(d_coeff, coeff, size_coeff, hipMemcpyHostToDevice);


size_xy = m*n*p*DOF*sizeof(PetscScalar);
hipMalloc((void**)&d_x,size_xy); 
hipMemcpy(d_x, x, size_xy, hipMemcpyHostToDevice);

hipMalloc((void**)&d_y,size_xy); 
hipMemcpy(d_y, y, size_xy, hipMemcpyHostToDevice);

size_id = nos*sizeof(PetscInt);
hipMalloc((void**)&d_idx,size_id); 
hipMemcpy(d_idx, idx, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idy,size_id); 
hipMemcpy(d_idy, idy, size_id, hipMemcpyHostToDevice);

hipMalloc((void**)&d_idz,size_id); 
hipMemcpy(d_idz, idz, size_id, hipMemcpyHostToDevice);

//Binding X to the texture Memory
hipBindTexture(0, tex_x_double, d_x, size_xy);

if (_DBGFLAG){
cudaPrintfInit();
}

if(_DBGFLAG) 
	{
		tend1=getclock();
		tsetup=tend1-tbegin1;
		tbegin2=getclock();
	}

// Kernel Setup and Configurations
	
	if ((DOF%2)!=0 || (DOF==6))
		{
		BLOCK_SIZE=BLOCKWIDTH_X-BLOCKWIDTH_X%DOF;
		}
	else{
		BLOCK_SIZE=BLOCKWIDTH_X;  
		}
		
	dim3 dimBlock(BLOCK_SIZE,BLOCKWIDTH_Y);
	dim3 dimGrid((int)ceil((float)(m*p*DOF)/(float)BLOCK_SIZE),((int)ceil((float)(n)/(float)BLOCKWIDTH_Y)));
				
	if (DOF==1)
		{
		MatMul_Kernel_tex_1_DOF<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, m, n, p, nos,DOF);
		}
	else{
		MatMul_Kernel_tex<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, m, n, p, nos, DOF);
		}
   
// check if kernel execution generated and error
   //cutilCheckMsg("Kernel execution failed");
			
		/* 
		if (m > BLOCKWIDTH){
		// dim3 dimBlock(BLOCKWIDTH,BLOCKWIDTH);
		// dim3 dimGrid((int)ceil((float)m/(float)BLOCKWIDTH),((int)ceil((float)n/(float)BLOCKWIDTH)));

		dim3 dimBlock(BLOCKWIDTH,BLOCKWIDTH,BLOCKWIDTH);
		dim3 dimGrid((int)ceil((float)m/(float)BLOCKWIDTH),((int)ceil((float)n/(float)BLOCKWIDTH)),p/BLOCKWIDTH);

			// cutilCheckError(cutCreateTimer(&timer));
			// cutilCheckError(cutStartTimer(timer));

		MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);

		}
		else
		{
		//dim3 dimBlock(m,n);
		dim3 dimBlock(m,n,p);
		dim3 dimGrid(1,1,1);
		   
			// cutilCheckError(cutCreateTimer(&timer));
			// cutilCheckError(cutStartTimer(timer));

		MatMult_Kernel<<<dimGrid,dimBlock>>>(d_coeff, d_x, d_y, d_idx, d_idy, d_idz, m, n, p, nos);


		}

		 */
 

	if(_DBGFLAG) 
	{
		hipDeviceSynchronize();
		tend2=getclock();
		tkernel=tend2-tbegin2;
	}

	if (_DBGFLAG){
	cudaPrintfDisplay(stdout, true);
	cudaPrintfEnd();
	}
	
//Read y from the Device Memory

hipMemcpy(y, d_y, size_xy, hipMemcpyDeviceToHost); 
 
if(_DBGFLAG)
	{
	temp+=tkernel;
	if (kcalls==0)
		{
		printf("\n Structured Grid MatrixMul Kernel Permormance for m *%d* and n size *%d* \n",m,n);
		}
	if (kcalls==1000)
		{
		printf("\ncopy time (sec) : %f\n",tsetup);
		printf("Kernel time (sec): %f\n",tkernel);
		printf("Performance in Megaflops with for %dth Kernel call\n",kcalls);
		printf("Performance in Megaflops with copy time = %f\n",(2*nos*n*m*p*1.0e-6)/(tsetup+tkernel));
		printf("Performance in Megaflops without copy time = %f\n",(2*nos*n*m*p*1.0e-6)/tkernel);
		printf("Culmative Performance in Megaflops for *%d* calls without copy time = %f\n",kcalls,(2*nos*n*m*p*1.0e-6)/(temp/(kcalls+1)));
		}
	}
kcalls++;

for(int i=0;i<m*n;i++)
printf("Y[%d]: %lf\n",i,y[i]);
//Free Device Memory
//hipFree(d_coeff);
hipFree(d_x);
hipFree(d_y);
hipFree(d_idx);
hipFree(d_idy);
hipFree(d_idz);

return 0;
}




