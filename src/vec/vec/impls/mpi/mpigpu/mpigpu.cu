#include "hip/hip_runtime.h"

/*
   This file contains routines for Parallel vector operations.
 */
#include <petscconf.h>
PETSC_CUDA_EXTERN_C_BEGIN
#include <../src/vec/vec/impls/mpi/pvecimpl.h>   /*I  "petscvec.h"   I*/
PETSC_CUDA_EXTERN_C_END
#include <../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h>
static hipError_t ccs[16];
static hipError_t cms[16];

extern MPI_Op VecMax_Local_Op;
extern MPI_Op VecMin_Local_Op;



#undef __FUNCT__
#define __FUNCT__ "VecDestroy_MPIGPU"
PetscErrorCode VecDestroy_MPIGPU(Vec v)
{
  PetscErrorCode ierr;

  PetscFunctionBegin;
  /* Destroy the stashes: note the order - so that the tags are freed properly */
  ierr = VecStashDestroy_Private(&v->bstash);CHKERRQ(ierr);
  ierr = VecStashDestroy_Private(&v->stash);CHKERRQ(ierr);
  ierr = VecDestroy_SeqGPU(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecSetValues_MPIGPU"
PetscErrorCode VecSetValues_MPIGPU(Vec xin,PetscInt ni,const PetscInt ix[],const PetscScalar y[],InsertMode addv)
{
  PetscErrorCode ierr;
  Vec_SeqGPU *x = (Vec_SeqGPU*)xin->data;
  PetscFunctionBegin;
  if (addv == ADD_VALUES && x->syncState == VEC_GPU) {
    ierr = VecCopyOverD2H(xin,x->cpuptr); CHKERRQ(ierr);
    hipDeviceSynchronize();
  }
  ierr = VecSetValues_MPI(xin,ni,ix,y,addv);CHKERRQ(ierr);
  x->syncState = VEC_CPU;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecAssemblyEnd_MPIGPU"
PetscErrorCode VecAssemblyEnd_MPIGPU(Vec xin)
{
  PetscErrorCode ierr;
  Vec_SeqGPU *x = (Vec_SeqGPU*)xin->data;

  PetscFunctionBegin;
  ierr = VecAssemblyEnd_MPI(xin);CHKERRQ(ierr);
  x->syncState = VEC_CPU;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecMax_MPIGPU"
PetscErrorCode VecMax_MPIGPU(Vec xin, PetscInt *idx, PetscReal *z)
{
  PetscErrorCode ierr;
  PetscReal work;

  PetscFunctionBegin;
  ierr = VecMax_SeqGPU(xin,idx,&work);CHKERRQ(ierr); 
  if (!idx) {
    ierr = MPI_Allreduce(&work,z,1,MPIU_SCALAR,MPIU_MAX,((PetscObject)xin)->comm);CHKERRQ(ierr);
  } else {
    PetscReal work2[2],z2[2];
    PetscInt  rstart;
    rstart = xin->map->rstart;
    work2[0] = work;
    work2[1] = *idx + rstart;
    ierr = MPI_Allreduce(work2,z2,2,MPIU_REAL,VecMax_Local_Op,((PetscObject)xin)->comm);CHKERRQ(ierr);
    *z   = z2[0];
    *idx = (PetscInt)z2[1];
  }
  PetscFunctionReturn(0);

}


#undef __FUNCT__
#define __FUNCT__ "VecMin_MPIGPU"
PetscErrorCode VecMin_MPIGPU(Vec xin,PetscInt *idx,PetscReal *z)
{
  PetscErrorCode ierr;
  PetscReal      work;

  PetscFunctionBegin;
  /* Find the local Min */
  ierr = VecMin_SeqGPU(xin,idx,&work);CHKERRQ(ierr);

  /* Find the global Min */
  if (!idx) {
    ierr = MPI_Allreduce(&work,z,1,MPIU_REAL,MPIU_MIN,((PetscObject)xin)->comm);CHKERRQ(ierr);
  } else {
    PetscReal work2[2],z2[2];
    PetscInt  rstart;

    ierr = VecGetOwnershipRange(xin,&rstart,PETSC_NULL);CHKERRQ(ierr);
    work2[0] = work;
    work2[1] = *idx + rstart;
    ierr = MPI_Allreduce(work2,z2,2,MPIU_REAL,VecMin_Local_Op,((PetscObject)xin)->comm);CHKERRQ(ierr);
    *z   = z2[0];
    *idx = (PetscInt)z2[1];
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__  
#define __FUNCT__ "VecNorm_MPIGPU"
PetscErrorCode VecNorm_MPIGPU(Vec xin,NormType type,PetscReal *z)
{
  PetscReal      sum,work = 0.0;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (type == NORM_2 || type == NORM_FROBENIUS) {
    ierr = VecNorm_SeqGPU(xin,NORM_2,&work);
    work *= work;
    ierr = MPI_Allreduce(&work,&sum,1,MPIU_REAL,MPIU_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
    *z = PetscSqrtReal(sum);
    //printf("VecNorm_MPIGPU : z=%1.5g\n",*z);
  } else if (type == NORM_1) {
    /* Find the local part */
    ierr = VecNorm_SeqGPU(xin,NORM_1,&work);CHKERRQ(ierr);
    /* Find the global max */
    ierr = MPI_Allreduce(&work,z,1,MPIU_REAL,MPIU_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
  } else if (type == NORM_INFINITY) {
    /* Find the local max */
    ierr = VecNorm_SeqGPU(xin,NORM_INFINITY,&work);CHKERRQ(ierr);
    /* Find the global max */
    ierr = MPI_Allreduce(&work,z,1,MPIU_REAL,MPIU_MAX,((PetscObject)xin)->comm);CHKERRQ(ierr);
  } else if (type == NORM_1_AND_2) {
    PetscReal temp[2];
    ierr = VecNorm_SeqGPU(xin,NORM_1,temp);CHKERRQ(ierr);
    ierr = VecNorm_SeqGPU(xin,NORM_2,temp+1);CHKERRQ(ierr);
    temp[1] = temp[1]*temp[1];
    ierr = MPI_Allreduce(temp,z,2,MPIU_REAL,MPIU_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
    z[1] = PetscSqrtReal(z[1]);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecView_MPIGPU_ASCII"
PetscErrorCode VecView_MPIGPU_ASCII(Vec xin,PetscViewer viewer)
{
  PetscErrorCode ierr;
  PetscInt          i,work = xin->map->n,cnt,len;
  PetscMPIInt       j,n = 0,size,rank,tag = ((PetscObject)viewer)->tag;
  MPI_Status        status;
  PetscScalar       *values;
  PetscScalar *xarray;
  const char        *name;
  Vec_SeqGPU  *x=(Vec_SeqGPU*)xin->data;
  PetscViewerFormat format;

  PetscFunctionBegin;
  if (x->syncState == VEC_GPU) {
    ierr = VecCopyOverD2H(xin,x->cpuptr); CHKERRQ(ierr);
    hipDeviceSynchronize();
  }
  xarray = x->cpuptr;
  /* determine maximum message to arrive */
  ierr = MPI_Comm_rank(((PetscObject)xin)->comm,&rank);CHKERRQ(ierr);
  ierr = MPI_Reduce(&work,&len,1,MPIU_INT,MPI_MAX,0,((PetscObject)xin)->comm);CHKERRQ(ierr);
  ierr = MPI_Comm_size(((PetscObject)xin)->comm,&size);CHKERRQ(ierr);

  if (!rank) {
    ierr = PetscMalloc(len*sizeof(PetscScalar),&values);CHKERRQ(ierr);
    ierr = PetscViewerGetFormat(viewer,&format);CHKERRQ(ierr);
    /*
        MATLAB format and ASCII format are very similar except
        MATLAB uses %18.16e format while ASCII uses %g
    */
    if (format == PETSC_VIEWER_ASCII_MATLAB) {
      ierr = PetscObjectGetName((PetscObject)xin,&name);CHKERRQ(ierr);
      ierr = PetscViewerASCIIPrintf(viewer,"%s = [\n",name);CHKERRQ(ierr);
      for (i=0; i<xin->map->n; i++) {
#if defined(PETSC_USE_COMPLEX)
        if (PetscImaginaryPart(xarray[i]) > 0.0) {
          ierr = PetscViewerASCIIPrintf(viewer,"%18.16e + %18.16ei\n",PetscRealPart(xarray[i]),PetscImaginaryPart(xarray[i]));CHKERRQ(ierr);
        } else if (PetscImaginaryPart(xarray[i]) < 0.0) {
          ierr = PetscViewerASCIIPrintf(viewer,"%18.16e - %18.16ei\n",PetscRealPart(xarray[i]),-PetscImaginaryPart(xarray[i]));CHKERRQ(ierr);
        } else {
          ierr = PetscViewerASCIIPrintf(viewer,"%18.16e\n",PetscRealPart(xarray[i]));CHKERRQ(ierr);
        }
#else
        ierr = PetscViewerASCIIPrintf(viewer,"%18.16e\n",(double)xarray[i]);CHKERRQ(ierr);
#endif
      }
      /* receive and print messages */
      for (j=1; j<size; j++) {
        ierr = MPI_Recv(values,(PetscMPIInt)len,MPIU_SCALAR,j,tag,((PetscObject)xin)->comm,&status);CHKERRQ(ierr);
        ierr = MPI_Get_count(&status,MPIU_SCALAR,&n);CHKERRQ(ierr);
        for (i=0; i<n; i++) {
#if defined(PETSC_USE_COMPLEX)
          if (PetscImaginaryPart(values[i]) > 0.0) {
            ierr = PetscViewerASCIIPrintf(viewer,"%18.16e + %18.16e i\n",PetscRealPart(values[i]),PetscImaginaryPart(values[i]));CHKERRQ(ierr);
          } else if (PetscImaginaryPart(values[i]) < 0.0) {
            ierr = PetscViewerASCIIPrintf(viewer,"%18.16e - %18.16e i\n",PetscRealPart(values[i]),-PetscImaginaryPart(values[i]));CHKERRQ(ierr);
          } else {
            ierr = PetscViewerASCIIPrintf(viewer,"%18.16e\n",PetscRealPart(values[i]));CHKERRQ(ierr);
          }
#else
          ierr = PetscViewerASCIIPrintf(viewer,"%18.16e\n",values[i]);CHKERRQ(ierr);
#endif
        }
      }
      ierr = PetscViewerASCIIPrintf(viewer,"];\n");CHKERRQ(ierr);

    } else {
      ierr = PetscObjectPrintClassNamePrefixType((PetscObject)xin,viewer,"Vector Object");CHKERRQ(ierr);
      if (format != PETSC_VIEWER_ASCII_COMMON) {ierr = PetscViewerASCIIPrintf(viewer,"Process [%d]\n",rank);CHKERRQ(ierr);}
      cnt = 0;
      for (i=0; i<xin->map->n; i++) {
        if (format == PETSC_VIEWER_ASCII_INDEX) {
          ierr = PetscViewerASCIIPrintf(viewer,"%D: ",cnt++);CHKERRQ(ierr);
        }
#if defined(PETSC_USE_COMPLEX)
        if (PetscImaginaryPart(xarray[i]) > 0.0) {
          ierr = PetscViewerASCIIPrintf(viewer,"%g + %g i\n",PetscRealPart(xarray[i]),PetscImaginaryPart(xarray[i]));CHKERRQ(ierr);
        } else if (PetscImaginaryPart(xarray[i]) < 0.0) {
          ierr = PetscViewerASCIIPrintf(viewer,"%g - %g i\n",PetscRealPart(xarray[i]),-PetscImaginaryPart(xarray[i]));CHKERRQ(ierr);
        } else {
          ierr = PetscViewerASCIIPrintf(viewer,"%g\n",PetscRealPart(xarray[i]));CHKERRQ(ierr);
        }
#else
        ierr = PetscViewerASCIIPrintf(viewer,"%g\n",(double)xarray[i]);CHKERRQ(ierr);
#endif
      }
      /* receive and print messages */
      for (j=1; j<size; j++) {
        ierr = MPI_Recv(values,(PetscMPIInt)len,MPIU_SCALAR,j,tag,((PetscObject)xin)->comm,&status);CHKERRQ(ierr);
        ierr = MPI_Get_count(&status,MPIU_SCALAR,&n);CHKERRQ(ierr);
        if (format != PETSC_VIEWER_ASCII_COMMON) {
          ierr = PetscViewerASCIIPrintf(viewer,"Process [%d]\n",j);CHKERRQ(ierr);
        }
        for (i=0; i<n; i++) {
          if (format == PETSC_VIEWER_ASCII_INDEX) {
            ierr = PetscViewerASCIIPrintf(viewer,"%D: ",cnt++);CHKERRQ(ierr);
          }
#if defined(PETSC_USE_COMPLEX)
          if (PetscImaginaryPart(values[i]) > 0.0) {
            ierr = PetscViewerASCIIPrintf(viewer,"%g + %g i\n",PetscRealPart(values[i]),PetscImaginaryPart(values[i]));CHKERRQ(ierr);
          } else if (PetscImaginaryPart(values[i]) < 0.0) {
            ierr = PetscViewerASCIIPrintf(viewer,"%g - %g i\n",PetscRealPart(values[i]),-PetscImaginaryPart(values[i]));CHKERRQ(ierr);
          } else {
            ierr = PetscViewerASCIIPrintf(viewer,"%g\n",PetscRealPart(values[i]));CHKERRQ(ierr);
          }
#else
          ierr = PetscViewerASCIIPrintf(viewer,"%g\n",(double)values[i]);CHKERRQ(ierr);
#endif
        }
      }
    }
    ierr = PetscFree(values);CHKERRQ(ierr);
  } else {
    ierr = PetscViewerGetFormat(viewer,&format);CHKERRQ(ierr);
    if (format == PETSC_VIEWER_ASCII_MATLAB) {
      /* this may be a collective operation so make sure everyone calls it */
      ierr = PetscObjectGetName((PetscObject)xin,&name);CHKERRQ(ierr);
    }
    /* send values */
    ierr = MPI_Send((void*)xarray,xin->map->n,MPIU_SCALAR,0,tag,((PetscObject)xin)->comm);CHKERRQ(ierr);
  }
  ierr = PetscViewerFlush(viewer);CHKERRQ(ierr);

  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecView_MPIGPU"
PetscErrorCode VecView_MPIGPU(Vec xin,PetscViewer viewer)
{
  PetscErrorCode ierr;
  PetscBool      iascii,isbinary,isdraw;

  PetscFunctionBegin;
  ierr = PetscObjectTypeCompare((PetscObject)viewer,PETSCVIEWERASCII,&iascii);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)viewer,PETSCVIEWERBINARY,&isbinary);CHKERRQ(ierr);
  ierr = PetscObjectTypeCompare((PetscObject)viewer,PETSCVIEWERDRAW,&isdraw);CHKERRQ(ierr);

  if (iascii){
    ierr = VecView_MPIGPU_ASCII(xin,viewer);CHKERRQ(ierr);
    /*
  } else if (isbinary) {
    ierr = VecView_MPIGPU_Binary(xin,viewer);CHKERRQ(ierr);

  } else if (isdraw) {
    PetscViewerFormat format;

    ierr = PetscViewerGetFormat(viewer,&format);CHKERRQ(ierr);
    if (format == PETSC_VIEWER_DRAW_LG) {
      ierr = VecView_MPIGPU_Draw_LG(xin,viewer);CHKERRQ(ierr);
    } else {
      ierr = VecView_MPIGPU_Draw(xin,viewer);CHKERRQ(ierr);
    }
     */
  } else SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_SUP,"Viewer type %s not supported for this object",((PetscObject)viewer)->type_name);
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecDot_MPIGPU"
PetscErrorCode VecDot_MPIGPU(Vec xin,Vec yin,PetscScalar *z)
{
  PetscScalar    sum,work;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecDot_SeqGPU(xin,yin,&work);CHKERRQ(ierr);
  ierr = MPI_Allreduce(&work,&sum,1,MPIU_SCALAR,MPIU_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
  *z = sum;
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecTDot_MPIGPU"
PetscErrorCode VecTDot_MPIGPU(Vec xin,Vec yin,PetscScalar *z)
{
  PetscScalar    sum,work;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = VecTDot_SeqGPU(xin,yin,&work);CHKERRQ(ierr);
  ierr = MPI_Allreduce(&work,&sum,1,MPIU_SCALAR,MPIU_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
  *z   = sum;
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecMDot_MPIGPU"
PetscErrorCode VecMDot_MPIGPU(Vec xin,PetscInt nv,const Vec y[],PetscScalar *z)
{
  PetscScalar    awork[128],*work = awork;
  PetscErrorCode ierr;

  PetscFunctionBegin;
  if (nv > 128) {
    ierr = PetscMalloc(nv*sizeof(PetscScalar),&work);CHKERRQ(ierr);
  }
  ierr = VecMDot_SeqGPU(xin,nv,y,work);CHKERRQ(ierr);
  ierr = MPI_Allreduce(work,z,nv,MPIU_SCALAR,MPIU_SUM,((PetscObject)xin)->comm);CHKERRQ(ierr);
  if (nv > 128) {
    ierr = PetscFree(work);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

/*MC
   VECMPIGPU - VECMPIGPU = "mpicusp" - The basic parallel vector, modified to use CUSP

   Options Database Keys:
. -vec_type mpicusp - sets the vector type to VECMPIGPU during a call to VecSetFromOptions()

  Level: beginner

.seealso: VecCreate(), VecSetType(), VecSetFromOptions(), VecCreateMpiWithArray(), VECMPI, VecType, VecCreateMPI(), VecCreateMpi()
M*/


#undef __FUNCT__  
#define __FUNCT__ "VecDuplicate_MPIGPU"
PetscErrorCode VecDuplicate_MPIGPU(Vec win,Vec *v)
{
  PetscErrorCode ierr;
  Vec_SeqGPU     *s;
  Vec V; 

  PetscFunctionBegin;
  ierr = VecCreate(((PetscObject)win)->comm,&V);CHKERRQ(ierr);
  ierr = PetscLayoutReference(win->map,&V->map);CHKERRQ(ierr);
  ierr = PetscNewLog(V,Vec_SeqGPU,&s);CHKERRQ(ierr);

  V->data = (void*)s;

  ierr = PetscMemcpy(V->ops,win->ops,sizeof(struct _VecOps));CHKERRQ(ierr);


  /* New vector should inherit stashing property of parent */
  V->stash.donotstash = win->stash.donotstash;
  V->stash.ignorenegidx = win->stash.ignorenegidx;

  ierr = PetscOListDuplicate(((PetscObject)win)->olist,&((PetscObject)V)->olist);CHKERRQ(ierr);
  ierr = PetscFListDuplicate(((PetscObject)win)->qlist,&((PetscObject)V)->qlist);CHKERRQ(ierr);
  V->map->bs    = win->map->bs;
  V->bstash.bs = win->bstash.bs;


  /* Set up local and device storage */
  s->syncState      = VEC_UNALLOC;
  s->unplacedarray=PETSC_NULL;
  s->array_allocated=PETSC_NULL;
  s->array=PETSC_NULL;
  /* create an associated stream */
  cms[0] = hipStreamCreate(&(s->streamid));
  /* allocate the variable for vector size */
  cms[1]=hipMalloc((void**)&(s->length),sizeof(int));
  /* send vec length size to device */
  ccs[0]=hipMemcpyAsync((void*)s->length,
               (void*)&(V->map->n),sizeof(int),hipMemcpyHostToDevice,s->streamid);
  /* allocate the vector on device */
  cms[2]=hipMalloc((void**)&(s->devptr),V->map->n*sizeof(double));
  ccs[1]=hipMemsetAsync((void*)s->devptr,0,V->map->n*sizeof(double),s->streamid);
  /* allocate the variable for vector offsets */
  cms[3]=hipMalloc((void**)&(s->offset),sizeof(int));
  /* allocate the variable for vector segment length */
  cms[4]=hipMalloc((void**)&(s->segment),sizeof(int));
  /* allocate the variable for vector single value result */
  cms[5]=hipMalloc((void**)&(s->zval),sizeof(double));
  cms[6]=hipMalloc((void**)&(s->scalar),sizeof(double));
  /* using pinned memory (could be a resource hog with very large arrays) */
  ierr = PinnedMalloc(&(s->cpuptr),V->map->n*sizeof(double));CHKERRQ(ierr);


  ierr = VecCheckCUDAStatus(ccs[0],"Copy H2D devlength in VecCreate_SeqGPU");CHKERRQ(ierr);
  ierr = VecCheckCUDAStatus(ccs[1],"on device cudaMemSet VecCreate_SeqGPU"); CHKERRQ(ierr);
  ierr = VecCheckCUDAStatus(cms[0],"on hipStreamCreate VecCreate_SeqGPU");  CHKERRQ(ierr);
  ierr = VecCheckCUDAStatus(cms[1],"Alloc devlength in VecCreate_SeqGPU");   CHKERRQ(ierr);
  ierr = VecCheckCUDAStatus(cms[2],"Alloc of devptr in VecCreate_SeqGPU");   CHKERRQ(ierr);
  ierr = VecCheckCUDAStatus(cms[3],"Alloc devoffset in VecCreate_SeqGPU");   CHKERRQ(ierr);
  ierr = VecCheckCUDAStatus(cms[4],"Alloc dev segment in VecCreate_SeqGPU"); CHKERRQ(ierr);
  ierr = VecCheckCUDAStatus(cms[5],"Alloc dev zval in VecCreate_SeqGPU");    CHKERRQ(ierr);
  ierr = VecCheckCUDAStatus(cms[6],"Alloc dev scalar in VecCreate_SeqGPU");    CHKERRQ(ierr);


  /* ierr = PetscMalloc(V->map->n*sizeof(PetscScalar),&(s->cpuptr)); */
  ierr = PetscMemzero(s->cpuptr,V->map->n*sizeof(double));CHKERRQ(ierr);
  s->syncState=VEC_ALLOC;


  /* change type_name appropriately */
  ierr = PetscObjectChangeTypeName((PetscObject)V,VECMPIGPU);CHKERRQ(ierr);


  ierr = PetscOListDuplicate(((PetscObject)win)->olist,&((PetscObject)V)->olist);CHKERRQ(ierr);
  ierr = PetscFListDuplicate(((PetscObject)win)->qlist,&((PetscObject)V)->qlist);CHKERRQ(ierr);
  V->map->bs    = win->map->bs;
  V->bstash.bs = win->bstash.bs;

  *v = V;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecDotNorm2_MPIGPU"
PetscErrorCode VecDotNorm2_MPIGPU(Vec s,Vec t,PetscScalar *dp,PetscScalar *nm)
{
  PetscErrorCode  ierr;
  PetscScalar     work[2],sum[2];
  PetscFunctionBegin;
  ierr    = VecDotNorm2_SeqGPU(s,t,work,work+1);CHKERRQ(ierr);
  ierr    = MPI_Allreduce(&work,&sum,2,MPIU_SCALAR,MPIU_SUM,((PetscObject)s)->comm);CHKERRQ(ierr);
  *dp     = sum[0];
  *nm     = sum[1];
  //printf("VecDotNorm2_MPIGPU=%1.5g,%1.5g\n",PetscRealPart(*dp),PetscImaginaryPart(*dp));
  //printf("VecDotNorm2_MPIGPU=%1.5g,%1.5g\n",PetscRealPart(*nm),PetscImaginaryPart(*nm));
  PetscFunctionReturn(0);
}

EXTERN_C_BEGIN
#undef __FUNCT__  
#define __FUNCT__ "VecCreate_MPIGPU"
PetscErrorCode  VecCreate_MPIGPU(Vec V)
{
  PetscErrorCode ierr;
  Vec_SeqGPU* s = PETSC_NULL;
  PetscFunctionBegin;
  ierr = PetscNewLog(V,Vec_SeqGPU,&s);CHKERRQ(ierr);
  V->data = (void*)s;
  ierr = PetscLayoutSetUp(V->map);CHKERRQ(ierr);

  V->stash.insertmode  = NOT_SET_VALUES;
  /* create the stashes. The block-size for bstash is set later when
     VecSetValuesBlocked is called.
  */
  ierr = VecStashCreate_Private(((PetscObject)V)->comm,1,&V->stash);CHKERRQ(ierr);
  ierr = VecStashCreate_Private(((PetscObject)V)->comm,V->map->bs,&V->bstash);CHKERRQ(ierr);

  
  s->syncState      = VEC_UNALLOC;
  s->unplacedarray=PETSC_NULL;
  s->array_allocated=PETSC_NULL;
  s->array=PETSC_NULL;
  /* create an associated stream */
  cms[0] = hipStreamCreate(&(s->streamid));
  /* allocate the variable for vector size */
  cms[1]=hipMalloc((void**)&(s->length),sizeof(int));
  /* send vec length size to device */
  ccs[0]=hipMemcpyAsync((void*)s->length,
               (void*)&(V->map->n),sizeof(int),hipMemcpyHostToDevice,s->streamid);
  /* allocate the vector on device */
  cms[2]=hipMalloc((void**)&(s->devptr),V->map->n*sizeof(double));
  ccs[1]=hipMemsetAsync((void*)s->devptr,0,V->map->n*sizeof(double),s->streamid);
  /* allocate the variable for vector offsets */
  cms[3]=hipMalloc((void**)&(s->offset),sizeof(int));
  /* allocate the variable for vector segment length */
  cms[4]=hipMalloc((void**)&(s->segment),sizeof(int));
  /* allocate the variable for vector single value result */
  cms[5]=hipMalloc((void**)&(s->zval),sizeof(double));
  cms[6]=hipMalloc((void**)&(s->scalar),sizeof(double));
  /* using pinned memory (could be a resource hog with very large arrays) */
  ierr = PinnedMalloc(&(s->cpuptr),V->map->n*sizeof(double));CHKERRQ(ierr);

  /* ierr = PetscMalloc(V->map->n*sizeof(PetscScalar),&(s->cpuptr)); */
  ierr = PetscMemzero(s->cpuptr,V->map->n*sizeof(double));CHKERRQ(ierr);
  s->syncState=VEC_ALLOC;

  ierr = PetscObjectChangeTypeName((PetscObject)V,VECMPIGPU);CHKERRQ(ierr);
  V->ops->dotnorm2        = VecDotNorm2_MPIGPU;
  V->ops->waxpy           = VecWAXPY_SeqGPU;
  V->ops->duplicate       = VecDuplicate_MPIGPU;
  V->ops->dot             = VecDot_MPIGPU;
  V->ops->mdot            = VecMDot_MPIGPU;
  V->ops->tdot            = VecTDot_MPIGPU;
  V->ops->norm            = VecNorm_MPIGPU;
  V->ops->getlocalsize    = VecGetSize_Seq;
  V->ops->view            = VecView_MPIGPU;
  V->ops->max             = VecMax_MPIGPU;
  V->ops->min             = VecMin_MPIGPU;
  V->ops->destroy         = VecDestroy_MPIGPU;

  V->ops->setvalues       = VecSetValues_MPIGPU;
  V->ops->assemblybegin   = VecAssemblyBegin_MPI;
  V->ops->assemblyend     = VecAssemblyEnd_MPIGPU;

  V->ops->getarray        = VecGetArray_SeqGPU;
  V->ops->restorearray    = VecRestoreArray_SeqGPU;
  V->ops->getsize         = VecGetSize_MPI;
  V->ops->duplicatevecs   = VecDuplicateVecs_SeqGPU;
  V->ops->destroyvecs     = VecDestroyVecs_SeqGPU;
  V->ops->scale           = VecScale_SeqGPU;
  V->ops->copy            = VecCopy_SeqGPU;
  V->ops->set             = VecSet_SeqGPU;
  V->ops->swap            = VecSwap_SeqGPU;
  V->ops->axpy            = VecAXPY_SeqGPU;
  V->ops->axpby           = VecAXPBY_SeqGPU;
  V->ops->maxpy           = VecMAXPY_SeqGPU;
  V->ops->aypx            = VecAYPX_SeqGPU;
  V->ops->axpbypcz        = VecAXPBYPCZ_SeqGPU;
  V->ops->pointwisemult   = VecPointwiseMult_SeqGPU;
  V->ops->setrandom       = VecSetRandom_SeqGPU;
  V->ops->replacearray    = VecReplaceArray_SeqGPU;
  V->ops->dot_local       = VecDot_SeqGPU;
  V->ops->tdot_local      = VecTDot_SeqGPU;
  V->ops->norm_local      = VecNorm_SeqGPU;
  V->ops->mdot_local      = VecMDot_SeqGPU;
  V->ops->pointwisedivide = VecPointwiseDivide_SeqGPU;
  /* place array?
     reset array?
     get values?
  */
  ierr = VecSet(V,0.0);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}
EXTERN_C_END

EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "VecCreate_GPU"
PetscErrorCode  VecCreate_GPU(Vec v)
{
  PetscErrorCode ierr;
  PetscMPIInt    size;

  PetscFunctionBegin;
  ierr = MPI_Comm_size(((PetscObject)v)->comm,&size);CHKERRQ(ierr);
  if (size == 1) {
    ierr = VecSetType(v,VECSEQGPU);CHKERRQ(ierr);
  } else {
    ierr = VecSetType(v,VECMPIGPU);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}
EXTERN_C_END





