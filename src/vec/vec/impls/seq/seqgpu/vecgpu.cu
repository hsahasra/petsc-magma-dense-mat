#include "hip/hip_runtime.h"
#include <petscconf.h>
#include <petscsys.h>
PETSC_CUDA_EXTERN_C_BEGIN
#include <string.h>
#include <omp.h>
#include <stdlib.h>
#include <float.h>
#include <private/vecimpl.h>          /*I "petscvec.h" I*/
#include <../src/vec/vec/impls/dvecimpl.h>
#include <../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h>


PETSC_CUDA_EXTERN_C_END




EXTERN_C_BEGIN

__constant__ int     integerSymbol;
__constant__ int2    integer2Symbol;
__constant__ int3    integer3Symbol;
__constant__ int     devN;//vector length
__constant__ double  dblScalarValue;//utility var
__constant__ double2 dblScalar2Value;//utility var
__constant__ float   fltScalarValue;//utility var
__constant__ float2  fltScalar2Value;//utility var


/* Valid pointer check function (probably doesn't work) */
PetscBool valid(void *p){
  extern char _etext;
  if((p != PETSC_NULL) && ((char*) p > &_etext)){
    return PETSC_TRUE;
  }else{
    return PETSC_FALSE;
  }
}




/* ---------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecCheckCUDAError"
PetscErrorCode VecCheckCUDAError(const char *msg){

  PetscFunctionBegin;
  hipError_t err = hipGetLastError();
  if( hipSuccess != err){
    fprintf(stderr, "Cuda kernel error: %s: %s.\n", msg,hipGetErrorString(err));
    fflush(NULL);
    PetscFunctionReturn(PETSC_ERR_LIB);
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCheckCUDAStatus"
PetscErrorCode VecCheckCUDAStatus(hipError_t cs,const char *msg){
  PetscFunctionBegin;
    if(cs!=hipSuccess){
      fprintf(stderr, "Cuda error: %s: %s.\n",msg,hipGetErrorString(cs));
      fflush(NULL);
      PetscFunctionReturn(PETSC_ERR_LIB);
    }
    PetscFunctionReturn(0);
}






/* ****************************************************************************
 * This is a shared memory implementation that keeps the full 625 words of state
 * in shared memory. Faster for heavy random work where you can afford 
 *  the shared memory. */
/* Init by single seed - single threaded as only used once */
__device__ void mt19937si(uint seed){
    int	i;
    if(threadIdx.x == 0){
	mtNexts = 0;
	s_seeds[0] = seed;
	for(i = 1;i < NNN;i++){
	   seed = (INIT_MULT * (seed^(seed >> 30))+i);
	   s_seeds[i] = seed;
	}
    }
    __syncthreads();/* Ensure mtNexts set */
    return;
}

/* Init by array - single threaded as only used once */
__device__ void mt19937sai(uint* seeds,uint length){
    int i,j,k;
    mt19937si(ARRAY_SEED);
    if(threadIdx.x==0){
     i=1; j=0;
     for(k = NNN>length?NNN:length;k!=0;k--){
        s_seeds[i] = (s_seeds[i]^((s_seeds[i-1]^(s_seeds[i-1] >> 30))*1664525)) + seeds[j] + j;
	if(++i >= NNN){
          s_seeds[0] = s_seeds[NNN-1];
	  i = 1;
        }
        if(++j>=length)j = 0;
     }
     for(k = NNN-1; k!=0;k--){
       s_seeds[i] = (s_seeds[i] ^ ((s_seeds[i-1]^(s_seeds[i-1]>>30))*1566083941))-i;
       if(++i >= NNN){
         s_seeds[0] = s_seeds[NNN-1];
	 i=1;
       }
     }
     s_seeds[0] = 0x80000000;/* MSB is 1; assuring non-zero initial array */ 
    }
    __syncthreads();				/* Needed for mt19937w() */
    return;
}

/* Return next MT random by increasing thread ID for 1-227 threads. */
__device__ uint mt19937s(void){
    int		kk;
    uint	x;
    uint	y;
    int		tid = threadIdx.x;

    kk = (mtNexts + tid) % NNN;
    __syncthreads();				/* Finished with mtNexts */

    if (tid == blockDim.x - 1)mtNexts = kk + 1;			/* Will get modded on next call */
    x = s_seeds[kk] & UPPER_MASK;
    if(kk < NNN - MMM){
      x |= (s_seeds[kk+1]&LOWER_MASK);
      y = s_seeds[kk+MMM];
    }else if(kk < NNN-1){
      x |= (s_seeds[kk+1]&LOWER_MASK);
      y = s_seeds[kk + (MMM-NNN)];
    }else{					/* kk == N - 1 */
      x |= (s_seeds[0]&LOWER_MASK);
      y = s_seeds[MMM - 1];
    }
    y ^= x >> 1;
    if (x & 1)y ^= MATRIX_A;
    __syncthreads();				/* All done before we update */

    s_seeds[kk] = y;
    y ^= (y >> 11);				/* Tempering */
    y ^= (y <<  7) & TEMPER1;
    y ^= (y << 15) & TEMPER2;
    y ^= (y >> 18);
    return y;
}

/* General shared memory version for any number of threads.
 * Note only up to 227 threads are run at any one time,
 * the rest loop and block till all are done. */
__device__ uint mt19937sl(void){
  int jj,kk,tid;
  uint x,y;
  tid = threadIdx.x;
  kk = (mtNexts + tid) % NNN;
  __syncthreads();				/* Finished with mtNexts */

  if(tid == blockDim.x - 1)mtNexts = kk + 1;	/* Will get modded on next call */
  jj = 0;
  do{
    if(jj <= tid && tid < jj + NNN - MMM){
      x = s_seeds[kk] & UPPER_MASK;
      if(kk < NNN - MMM){
         x |= (s_seeds[kk+1]&LOWER_MASK);
	 y = s_seeds[kk + MMM];
      }else if (kk < NNN-1){
         x |= (s_seeds[kk + 1]&LOWER_MASK);
	 y = s_seeds[kk + (MMM-NNN)];
      }else{				/* kk == N - 1 */
         x |= (s_seeds[0]&LOWER_MASK);
         y = s_seeds[MMM-1];
      }

      y ^= x >> 1;
      if(x & 1) y ^= MATRIX_A;
    }
    __syncthreads();			/* All done before we update */
    if(jj <= tid && tid < jj+NNN-MMM) s_seeds[kk] = y;
    __syncthreads();

  }while ((jj += NNN-MMM) < blockDim.x);
  y ^= (y >> 11);				/* Tempering */
  y ^= (y <<  7) & TEMPER1;
  y ^= (y << 15) & TEMPER2;
  y ^= (y >> 18);
  return y;
}

#undef __FUNCT__
#define __FUNCT__ "kernRandS"
__global__ void kernRandS(uint* seeds){
  mt19937sai(seeds,gridDim.x);
}

#undef __FUNCT__
#define __FUNCT__ "kernRand"
__global__ void kernRand(double *x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  uint rval;
  if(tid<*n){
    rval = mt19937sl();
    x[tid] = ((double)rval/(double)UINT_MAX);
    /* printf("RAND value[%d]: %0.13f, rval: %u UINT_MAX: %u\n",
       tid,x[tid],rval,UINT_MAX); */
  }
}

#undef __FUNCT__
#define __FUNCT__ "VecSetRandom_SeqGPU"
PetscErrorCode VecSetRandom_SeqGPU(Vec x,PetscRandom r){
  PetscFunctionBegin;
  static PetscBool seed_flag=PETSC_TRUE;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  PetscInt i,bx,tx;
  uint *seeds=PETSC_NULL,*devseeds=PETSC_NULL;
  PetscScalar rval;
  dim3 dimBlock,dimGrid;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  /* assuming width mem load isn't going to be an issue */
  /* printf("Call to VecSetRandom_SeqGPU\n");*/
  if(xd->syncState==VEC_ALLOC || xd->syncState==VEC_CPU){
    for(i=0; i<x->map->n; i++){
       ierr = PetscRandomGetValue(r,&xd->cpuptr[i]);CHKERRQ(ierr);
    }
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }else if(xd->syncState==VEC_SYNCHED || xd->syncState==VEC_GPU){
    bx=ceil((float)x->map->n/(float)TCOUNT);
    ierr = PetscMalloc(bx*sizeof(PetscInt),&seeds);CHKERRQ(ierr);
    tx=TCOUNT;
    dimGrid.x=bx; dimGrid.y=1;
    dimBlock.x=tx; dimBlock.y=1;
    if(seed_flag){
      for(i=0; i<bx; i++){
         ierr = PetscRandomGetValue(r,&rval);CHKERRQ(ierr);
         seeds[i]=(uint)(UINT_MAX*rval);
      }

      cudastatus = hipMalloc((void**)&devseeds,bx*sizeof(uint));
      ierr = VecCheckCUDAStatus(cudastatus,"error in hipMalloc");CHKERRQ(ierr);

      cudastatus=hipMemcpy(devseeds,seeds,bx*sizeof(uint),hipMemcpyHostToDevice);
      ierr = VecCheckCUDAStatus(cudastatus,"on copy H2D in VecSetRandom_SeqGPU");CHKERRQ(ierr);
      xd->vstat.h2d_count++;
      xd->vstat.h2d_bytes+=bx*sizeof(uint);

      kernRandS<<<dimGrid,dimBlock>>>(devseeds);
      ierr = VecCheckCUDAError("kernRandS launch");CHKERRQ(ierr);
      ierr = PetscFree(seeds);CHKERRQ(ierr);
      hipDeviceSynchronize();
      cudastatus = hipFree(devseeds);
      ierr = VecCheckCUDAStatus(cudastatus,"in hipFree()");CHKERRQ(ierr);
      seed_flag=PETSC_FALSE;
    }
    kernRand<<<dimGrid,dimBlock>>>(xd->devptr,xd->length);
    ierr = VecCheckCUDAError("kernRand launch");CHKERRQ(ierr);
  }
  xd->syncState = VEC_GPU;
  PetscFunctionReturn(0);
}

/*------------------------end random generator ------------------------*/



/*------------------------------ compare ------------------------------*/

#undef __FUNCT__
#define __FUNCT__ "VecCompare_SeqGPU"
PetscErrorCode VecCompare_SeqGPU(Vec x, Vec y, PetscBool *same, PetscInt offset, PetscInt blocksize){
  PetscFunctionBegin;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  Vec_SeqGPU* yd = (Vec_SeqGPU*)y->data;
  if(xd->syncState!=yd->syncState||xd->syncState==VEC_ALLOC||yd->syncState==VEC_ALLOC){
    *same=PETSC_FALSE;
    PetscFunctionReturn(0);
  }
  PetscErrorCode ierr;
  int blocks,threads;/* assuming shared memory size is not an issue */
  if(blocksize && !offset){
    blocks=ceil((float)blocksize/(float)TCOUNT);
  } else {
    blocks=ceil((float)x->map->n/(float)TCOUNT);
    blocksize = x->map->n;
  }
  threads=TCOUNT;
  hipError_t cudastatus;
  int *devsame=PETSC_NULL;
  int cpusame=0;
  int2 offset_bsize;
  offset_bsize.x = offset;
  offset_bsize.y = blocksize;


  if(xd->syncState==VEC_CPU && yd->syncState==VEC_CPU){
    ierr = PetscMemcmp((void*)&xd->cpuptr[offset],(void*)&yd->cpuptr[offset],blocksize,same);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }

  cudastatus = hipMalloc((void**)&devsame,sizeof(int));
  ierr = VecCheckCUDAStatus(cudastatus,"error in device malloc");CHKERRQ(ierr);

  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("integer2Symbol"),(void*)&offset_bsize,sizeof(int2),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);

  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("devN"),(void*)&x->map->n,sizeof(int),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);

  dim3 dimGrid; dimGrid.x=blocks; dimGrid.y=1;
  dim3 dimBlock; dimBlock.x=threads; dimBlock.y=1;
  kernCompare<<<dimGrid,dimBlock>>>(xd->devptr,yd->devptr,xd->length,yd->length,devsame);
  ierr = VecCheckCUDAError("kernCompare launch");CHKERRQ(ierr);

  cudastatus=hipMemcpy(&cpusame,devsame,sizeof(int),hipMemcpyDeviceToHost);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H in VecCompare_SeqGPU");CHKERRQ(ierr);

  if(cpusame==1)*same=PETSC_TRUE;
  else *same=PETSC_FALSE;
  cudastatus = hipFree(devsame);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "kernCompare"
__global__ void kernCompare(double* devX, double* devY, int* lx, int* ly, int* devsame){

  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int2 localOBS = integer2Symbol;
  int localn = localOBS.x+localOBS.y;
  int index = tid+localOBS.x;
  double value=0;
  __shared__ unsigned char blockflag;
  __shared__ double chunkX[TCOUNT];
  __shared__ double chunkY[TCOUNT];

  if(threadIdx.x==0)blockflag=0;
  __syncthreads();
  if(index<localn){
    /* read in values to shared */
    chunkX[threadIdx.x]=devX[index];
    chunkY[threadIdx.x]=devY[index];
    value = fabs(chunkX[threadIdx.x]-chunkY[threadIdx.x]);
    if(value>1e-16){
      //printf("In kernCompare found an element mismatch: %e\n",value);
      blockflag=1;
    }
    if(*lx!=*ly){
      //printf("In kernCompare found length mismatch: lx: %d vs ly: %d\n",*lx,*ly);
      blockflag=1;
    }
  }
  __syncthreads();
  if(threadIdx.x==0){
    if(blockflag)*devsame=0;
    else *devsame=1;
  }
  return;
}

/*------------------------------- end compare --------------------------*/


/*---------------------------- Vec info functions ----------------------*/

#undef __FUNCT__
#define __FUNCT__ "VecView_SeqGPU"
PetscErrorCode VecView_SeqGPU(Vec x,PetscViewer viewer){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  if(xd->syncState==VEC_GPU){
    ierr = VecCopyOverD2H(x,xd->cpuptr); CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  ierr= PetscViewerSetType(viewer,PETSCVIEWERASCII);CHKERRQ(ierr);/* forced ASCII for now */
  ierr =VecView_Seq_ASCII(x,viewer);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecGetLocalSize_SeqGPU"
PetscErrorCode VecGetLocalSize_SeqGPU(Vec x, PetscInt *localsize){
  PetscFunctionBegin;
  /* printf("Call to VecGetLocalSize_SeqGPU\n"); */
  PetscValidHeaderSpecific(x,VEC_CLASSID,1);
  PetscValidIntPointer(localsize,2);
  PetscValidType(x,1);
  *localsize=x->map->n;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecGetSize_SeqGPU"
PetscErrorCode VecGetSize_SeqGPU(Vec x, PetscInt *globalsize){
  PetscFunctionBegin;
  /* printf("Call to VecGetSize_SeqGPU\n"); */
  PetscValidHeaderSpecific(x,VEC_CLASSID,1);
  PetscValidIntPointer(globalsize,2);
  PetscValidType(x,1);
  *globalsize=x->map->N;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCheck_SeqGPU"
PetscErrorCode VecCheck_SeqGPU(Vec x){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  dim3 dimgrid(ceil((float)x->map->n/((float)TCOUNT)),1,1);
  dim3 dimblocks(TCOUNT,1,1);
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  printf("******************************************\n");
  kernCheck<<<dimgrid,dimblocks>>>(xd->devptr,xd->length);
  ierr = VecCheckCUDAError("Call to kernCheck. "); CHKERRQ(ierr);
  hipDeviceSynchronize();
  printf("******************************************\n");
  fflush(NULL);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "kernCheck"
__global__ void kernCheck(double* x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if(tid<*n){
    printf("kernCheck: x[%d]: %e, length: %d\n",tid,x[tid],*n);
  }
}

/*------------------------------ end info -------------------------------*/


/*---------------------------- copy functions ---------------------------*/
#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockDevice"
PetscErrorCode VecCopyBlockDevice(Vec d, Vec s, PetscInt doffset, PetscInt soffset, PetscInt blocksize){
  PetscFunctionBegin;
  printf("Call to VecCopyBlockDevice (**** EMPTY ****)\n");
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCopyOverDevice"
PetscErrorCode VecCopyOverDevice(Vec d,Vec s){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* dd = (Vec_SeqGPU*)d->data;
  Vec_SeqGPU* sd = (Vec_SeqGPU*)s->data;
  //printf("Call to VecCopyOverDevice\n");
  hipError_t cudastatus;
  dim3 dimGrid;
  dim3 dimBlock;

  if(s->map->n!=d->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(sd->syncState==VEC_CPU){/* synch y */
    ierr = VecCopyOverH2D(s,sd->cpuptr);CHKERRQ(ierr);
    sd->syncState=VEC_SYNCHED;
    hipStreamSynchronize(sd->stream);
  }
  cudastatus=hipMemcpyAsync(dd->devptr,sd->devptr,
               s->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice,dd->stream);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2D in VecCopyOverDevice");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "kernCopyLen"
__global__ void kernCopyLen(int* ly,int* lx){
  if(threadIdx.x==0)*ly=*lx;
}

#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockH2D"
PetscErrorCode VecCopyBlockH2D(Vec v,PetscScalar *y, PetscInt offset, PetscInt blocksize){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  /* printf("Call to VecCopyBlockH2D\n"); */
  cudastatus=hipMemcpyAsync(&(vd->devptr[offset]),y,
               blocksize*sizeof(PetscScalar),hipMemcpyHostToDevice,vd->stream);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy H2D in VecCopyBlockH2D");CHKERRQ(ierr);
  vd->vstat.h2d_count++;
  vd->vstat.h2d_bytes+=blocksize*sizeof(PetscScalar);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCopyOverH2D"
PetscErrorCode VecCopyOverH2D(Vec v,PetscScalar *y){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  /* printf("Call to VecCopyOverH2D\n"); */
  cudastatus=hipMemcpyAsync(vd->devptr,y,
               v->map->n*sizeof(PetscScalar),hipMemcpyHostToDevice,vd->stream);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy H2D in VecCopyOverH2D");CHKERRQ(ierr);
  vd->vstat.h2d_count++;
  vd->vstat.h2d_bytes+=v->map->n*sizeof(PetscScalar);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockD2H"
PetscErrorCode VecCopyBlockD2H(Vec v,PetscScalar *y,PetscInt offset, PetscInt blocksize){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  /* printf("Call to VecCopyBlockD2H\n"); */
  cudastatus=hipMemcpyAsync(y,&(vd->devptr[offset]),
               blocksize*sizeof(PetscScalar),hipMemcpyDeviceToHost,vd->stream);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H in VecCopyBlockD2H");CHKERRQ(ierr);
  vd->vstat.d2h_count++;
  vd->vstat.d2h_bytes+=blocksize*sizeof(PetscScalar);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCopyOverD2H"
PetscErrorCode VecCopyOverD2H(Vec v,PetscScalar *y){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  /* printf("Call to VecCopyOverD2H\n"); */
  cudastatus=hipMemcpyAsync(y,vd->devptr,
               v->map->n*sizeof(PetscScalar),hipMemcpyDeviceToHost,vd->stream);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H in VecCopyOverD2H");CHKERRQ(ierr);
  vd->vstat.d2h_count++;
  vd->vstat.d2h_bytes+=v->map->n*sizeof(PetscScalar);
  PetscFunctionReturn(0);
}

/*---------------------------- end copy functions --------------------------*/



/*------------------------------ set functions -----------------------------*/
#undef __FUNCT__
#define __FUNCT__ "VecSetValues_SeqGPU"
/*@
   VecSetValues - Inserts or adds values into certain locations of a vector.
@*/
PetscErrorCode VecSetValues_SeqGPU(Vec x,PetscInt ni,const PetscInt ix[],const PetscScalar y[],InsertMode iora){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt i;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  //printf("Call to VecSetValues_SeqGPU\n");
  if(xd->syncState==VEC_CPU || xd->syncState==VEC_SYNCHED){
    if(iora==INSERT_VALUES){
      #pragma omp parallel for
      for(i=0;i<ni;i++){
         xd->cpuptr[i]=y[i];
      }
      ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
      xd->syncState=VEC_SYNCHED;
    }else{
      /* ADD_VALUES not supported now */
      printf("Call to VecSetValues_SeqGPU: ADD_VALUES (*** EMPTY ***)\n");
    }
  }else{
      if(iora==INSERT_VALUES){/* not efficient */
        PetscScalar yval=0;
        for(i=0;i<ni;i++){
          yval=y[i];
          ierr = VecCopyBlockH2D(x,&yval,ix[i],1);CHKERRQ(ierr);
        }
      }
      xd->syncState=VEC_GPU;
  }
  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "VecSet_SeqGPU"
PetscErrorCode VecSet_SeqGPU(Vec xin,PetscScalar alpha){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  dim3 dimgrid(ceil((float)xin->map->n/((float)TCOUNT)),1,1);
  dim3 dimblocks(TCOUNT,1,1);
  Vec_SeqGPU* xd = (Vec_SeqGPU*)xin->data;
  //printf("Call to VecSet_SeqGPU, alpha: %e\n",alpha);
  hipDeviceSynchronize();
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,
            PETSC_ERR_MEM,"*** In VecSet_SeqGPU, Vec not allocated. ***\n");
  }else{
    cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha,sizeof(double),0,hipMemcpyHostToDevice);
    ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);
    kernSet<<<dimgrid,dimblocks>>>(xd->devptr,xd->length);
    ierr = VecCheckCUDAError("Call to kernSet. "); CHKERRQ(ierr);
    xd->syncState=VEC_GPU;
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "kernSet"
__global__ void kernSet(double* x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  __shared__ double chunkX[TCOUNT];
  chunkX[threadIdx.x] = dblScalarValue;
  if(tid<*n){
    x[tid] = chunkX[threadIdx.x]; /* arr[threadIdx.x]; */
    //printf("in kernSet: x[%d]: %e\n",tid,x[tid]);
  }
}




#undef __FUNCT__
#define __FUNCT__ "VecScale_SeqGPU"
PetscErrorCode VecScale_SeqGPU(Vec x, PetscScalar alpha){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  dim3 dimgrid(ceil((float)x->map->n/((float)TCOUNT)),1,1);
  dim3 dimblocks(TCOUNT,1,1);
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  //printf("VecScale_SeqGPU...alpha: %e\n",alpha);
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,
            PETSC_ERR_MEM,
            "*** In call to VecScale_SeqGPU, arg Vec xin has not been allocated. ***\n");
  }else if(xd->syncState==VEC_CPU){
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();
  if(alpha==0.){
    cudastatus = hipMemsetAsync(xd->devptr,0,x->map->n*sizeof(double),xd->stream);
    ierr = VecCheckCUDAStatus(cudastatus,"error in hipMemset");CHKERRQ(ierr);
  }else if (alpha != 1.0){
    cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha,sizeof(double),0,hipMemcpyHostToDevice);
    ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);
    kernScale<<<dimgrid,dimblocks,0,xd->stream>>>(xd->devptr,xd->length);
    ierr = VecCheckCUDAError("Call to kernScale. "); CHKERRQ(ierr);
  }
  xd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "kernScale"
__global__ void kernScale(double* x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  __shared__ double arr[TCOUNT];
  double localdbl=dblScalarValue;
  if(tid<*n){
    arr[threadIdx.x] = x[tid];
    //if(x[tid]!=0)printf("Pre: kernScale: x[%d]: %e, alpha: %e\n",tid,x[tid],localdbl);
    arr[threadIdx.x] *= localdbl;
    x[tid] = arr[threadIdx.x];
    //if(x[tid]!=0)printf("kernScale: x[%d]: %e, alpha: %e\n",tid,x[tid],localdbl);
  }
}

/*---------------------------- end set and scale ---------------------------*/


/*-------------------------- dot product functions -------------------------*/

#undef __FUNCT__
#define __FUNCT__ "VecTDot_SeqGPU"
PetscErrorCode VecTDot_SeqGPU(Vec xin,Vec yin,PetscScalar *z){
  PetscFunctionBegin;
  printf("VecTDot_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDot_SeqGPU"
PetscErrorCode VecDot_SeqGPU(Vec x,Vec y,PetscScalar *z){
 PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  double *devScratch,*devPartial,zhost;
  PetscInt i,chunks=0,segment,partialsize,scratchsize;
  hipStream_t* dotstream;
  dim3 dimGrid, dimBlock;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;

  //printf("Call to VecDot_SeqGPU, chunkwidth: %f xlen: %d\n",(CHUNKWIDTH),x->map->n);

  /* figure out how many chunks will be needed */
  chunks = ceil( ((float)x->map->n) /(float)(CHUNKWIDTH));
  //printf("Number of chunks in Dot: %d\n",chunks);
  dotstream = (hipStream_t*)malloc(chunks*sizeof(hipStream_t));

  if(chunks>1){
    segment = (int) CHUNKWIDTH;
    dimGrid.x=ceil((CHUNKWIDTH)/(float)THRDOTCNT);
  }else{
    segment = x->map->n;
    dimGrid.x=ceil(((float)segment)/(float)THRDOTCNT);
  }
  dimBlock.x = THRDOTCNT;
  partialsize=chunks*sizeof(double);
  scratchsize=partialsize*dimGrid.x;
  /* set up on x stream */
  if(xd->syncState==VEC_CPU){
    printf("xd state VEC_CPU: copying to device.\n");
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  cudastatus = hipMalloc((void**)&devScratch,scratchsize);/* scratch pad */
  ierr = VecCheckCUDAStatus(cudastatus,"devScratch alloc in VecDot_SeqGPU");CHKERRQ(ierr);
  cudastatus = hipMemsetAsync(devScratch,0,scratchsize,xd->stream);
  ierr = VecCheckCUDAStatus(cudastatus,"devScratch memset in VecDot_SeqGPU");CHKERRQ(ierr);

  /* set up on y stream */
  if(yd->syncState==VEC_CPU){
    printf("yd state VEC_CPU: copying to device.\n");
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  cudastatus=hipMalloc((void**)&devPartial,partialsize);/* partial results to be combined */
  ierr = VecCheckCUDAStatus(cudastatus,"devPartial alloc in VecDot_SeqGPU");CHKERRQ(ierr);
  cudastatus=hipMemsetAsync(devPartial,0,partialsize,yd->stream);
  ierr = VecCheckCUDAStatus(cudastatus,"devPartial memset in VecDot_SeqGPU");CHKERRQ(ierr);

  hipDeviceSynchronize();/* make sure everyone is ready */

  for(i=0;i<chunks;i++){  /* streaming async kernel calls */
    cudastatus=hipStreamCreate(&(dotstream[i]));
    ierr = VecCheckCUDAStatus(cudastatus,"on hipStreamCreate");CHKERRQ(ierr);
    cudastatus=hipMemcpyAsync(xd->offset,&i,sizeof(int),hipMemcpyHostToDevice,dotstream[i]);
    ierr = VecCheckCUDAStatus(cudastatus,"on copy array length H2D in VecDot_SeqGPU");CHKERRQ(ierr);
    cudastatus=hipMemcpyAsync(xd->segment,&segment,sizeof(int),hipMemcpyHostToDevice,dotstream[i]);
    ierr = VecCheckCUDAStatus(cudastatus,"on copy segment size H2D in VecDot_SeqGPU");CHKERRQ(ierr);
    /* Overlapping execution */
    kernDot<<<dimGrid,dimBlock,0,dotstream[i]>>>(xd->devptr,yd->devptr,
                                                          xd->segment,
                                                          xd->length,
                                                          xd->offset,
                                                          (devScratch+i*dimGrid.x),
                                                 (devPartial+i));
    ierr = VecCheckCUDAError("kernDot launch in VecDot_SeqGPU");CHKERRQ(ierr);
    xd->vstat.h2d_count++;
    xd->vstat.h2d_bytes+=2*sizeof(int);
    yd->vstat.h2d_count++;
    yd->vstat.h2d_bytes+=2*sizeof(int);
  }

  /* dot product block reduction */
  dimGrid.x  = 1;
  dimBlock.x = chunks;
  hipDeviceSynchronize();/* make sure everyone is caught up */
  kernRedDot<<<dimGrid,dimBlock,chunks*sizeof(double),xd->stream>>>(devPartial,xd->zval);
  ierr = VecCheckCUDAError("kernRedDot launch in VecDot_SeqGPU");CHKERRQ(ierr);
  hipDeviceSynchronize();/* make sure everyone is caught up */

  /* Copy back dot z */
  cudastatus=hipMemcpy(&zhost,xd->zval,sizeof(double),hipMemcpyDeviceToHost);/* copy back z */
  ierr = VecCheckCUDAStatus(cudastatus,"on copy zdot D2H in VecDot_SeqGPU");CHKERRQ(ierr);
  *z=zhost;
  //printf("Zdot: %e\n",*z);

  /* clean up resources */
  cudastatus = hipFree(devPartial);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  for(i=0;i<chunks;i++){
    cudastatus = hipStreamDestroy(dotstream[i]);
    ierr = VecCheckCUDAStatus(cudastatus,"on hipStreamCreate");CHKERRQ(ierr);
  }
  free(dotstream);
  cudastatus = hipFree(devScratch);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


extern __shared__ double arrayDot[];
#undef __FUNCT__
#define __FUNCT__ "kernRedDot"
__global__ void kernRedDot(double* arr,double* z){/* reduction kernel */

  int i = (blockDim.x+1)/2;
  double* zptr=(double*)arrayDot;

  zptr[threadIdx.x]=arr[threadIdx.x];
  __syncthreads();
  while(i>0){
    if(threadIdx.x<i){
      zptr[threadIdx.x]+=zptr[threadIdx.x+i];
    }
    __syncthreads();
    i/=2;
  }
  if(threadIdx.x==0){
    *z=zptr[0];
  }
}



#undef __FUNCT__
#define __FUNCT__ "kernDot"
__global__ void kernDot(double* devX, double* devY,
                        int* segmentsize, int* arrsize,
                        int* offset, double* scratch, double* z){
  __shared__ double chunkX[THRDOTCNT];
  __shared__ double chunkY[THRDOTCNT];
  __shared__ int n;    n   = *arrsize;
  __shared__ int seg;  seg = *segmentsize;
  __shared__ int off;  off = *offset;

  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int i = (blockDim.x+1)/2;
  int j = (gridDim.x+1)/2;
  int item = seg*off+tid;

  if(item<n){
    /* read in values to shared */
    chunkX[threadIdx.x]=devX[item]; /* offset values */
    chunkY[threadIdx.x]=devY[item]; /* offset values */
  }else{
    chunkX[threadIdx.x]=0.;
    chunkY[threadIdx.x]=0.;
  }

  chunkX[threadIdx.x]*=chunkY[threadIdx.x];
  __syncthreads();

  /* block level reduction */
  while(i>0){
     if(threadIdx.x<i){
       chunkX[threadIdx.x]+=chunkX[threadIdx.x+i];
     }
     __syncthreads();
     i/=2;
  }/* end while */

  if(threadIdx.x==0){
    scratch[blockIdx.x]=chunkX[0];
  }
  __syncthreads();


  /* grid level reduction */
  while(j>0){
    if(threadIdx.x==0 && blockIdx.x<j){
      scratch[blockIdx.x]+=scratch[blockIdx.x+j];
    }
    __syncthreads();
    j/=2;
  }

  if(tid==0)*z=scratch[blockIdx.x];
}




#undef __FUNCT__
#define __FUNCT__ "VecMDot_SeqGPU"
PetscErrorCode  VecMDot_SeqGPU(Vec x,PetscInt nv,const Vec y[],PetscScalar val[]){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt       i;
  // printf("VecMDot_SeqGPU\n");
  //printf("Number of vectors in MDot: %d\n",nv);
  for (i=0; i<nv; i++) {
    ierr = VecDot_SeqGPU(x,y[i],&val[i]);CHKERRQ(ierr);
    //hipDeviceSynchronize();
    if(PetscIsInfOrNanScalar(val[i])){
      SETERRQ1(((PetscObject)x)->comm,PETSC_ERR_FP,"Infinite or not-a-number generated in mdot, entry %D",i);
    }
  }
  PetscFunctionReturn(0);
}

/*----------------------------- end dot ----------------------------- */





#undef __FUNCT__
#define __FUNCT__ "VecAXPBY_SeqGPU"
PetscErrorCode VecAXPBY_SeqGPU(Vec yin,PetscScalar beta,PetscScalar alpha,Vec xin){
  /* Y = b*Y + a*X */
  PetscFunctionBegin;
  printf("Call to VecAXPBY_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecWAXPY_SeqGPU"
PetscErrorCode VecWAXPY_SeqGPU(Vec w,PetscScalar alpha,Vec x,Vec y){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt bx,tx;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)w->data;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid;
  dim3 dimBlock;
  hipError_t cudastatus;
  //printf("VecWAXPY_SeqGPU...");
  //printf("alpha: %e\n",alpha);
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();
  /* assuming width mem load isn't going to be an issue */
  bx=ceil((float)y->map->n/(float)AXPYTCOUNT);
  tx=AXPYTCOUNT;
  dimGrid.x=bx; dimGrid.y=1;
  dimBlock.x=tx; dimBlock.y=1;

  if(alpha==0.0){
    ierr = VecCopyOverDevice(w,y);CHKERRQ(ierr);
  }else if(alpha==1.0){
    kernWXPY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
    ierr = VecCheckCUDAError("kernel call to kernWXPY");CHKERRQ(ierr);
  }else if(alpha==-1.0){
    kernWXMY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
    ierr = VecCheckCUDAError("kernel call to kernWXMY");CHKERRQ(ierr);
  }else{
    cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha,sizeof(double),0,hipMemcpyHostToDevice);
    ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);
    kernWAXPY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
    ierr = VecCheckCUDAError("kernel call to kernWAXPY");CHKERRQ(ierr);
  }
  wd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "kernWAXPY"
__global__ void  kernWAXPY(double* devY,double* devX, int* vlen, double* devW){

 /* w <- y + alpha*x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double alphaShared;
  __shared__ double chunkY[AXPYTCOUNT];
  __shared__ double chunkX[AXPYTCOUNT];
  __shared__ double chunkW[AXPYTCOUNT];

  alphaShared = dblScalarValue;
  /* printf("in kernWAXPY:alphaShared: %f, tid: %d, vlen: %d\n",alphaShared,tid,*vlen); */
  if(tid<*vlen){
    //if(devX[tid]!=0)printf("kernWAXPY: devX[%d]: %e\n",tid,devX[tid]);
    //if(devY[tid]!=0)printf("kernWAXPY: devY[%d]: %e\n",tid,devY[tid]);
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkY[threadIdx.x]+(chunkX[threadIdx.x]*alphaShared);
    devW[tid]=chunkW[threadIdx.x];
    //if(devW[tid]!=0)printf("kernWAXPY: devW[%d]: %e, alpha: %e\n",tid,devW[tid],alphaShared);
  }
}

#undef __FUNCT__
#define __FUNCT__ "kernWXPY"
__global__ void  kernWXPY(double* devY,double* devX, int* vlen, double* devW){

 /* w <- y + x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunkY[AXPYTCOUNT];
  __shared__ double chunkX[AXPYTCOUNT];
  __shared__ double chunkW[AXPYTCOUNT];

  /* printf("in kernWAXPY:alphaShared: %f, tid: %d, vlen: %d\n",alphaShared,tid,*vlen); */
  if(tid<*vlen){
    //if(devX[tid]!=0)printf("kernWXPY: devX[%d]: %e\n",tid,devX[tid]);
    //if(devY[tid]!=0)printf("kernWXPY: devY[%d]: %e\n",tid,devY[tid]);
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkY[threadIdx.x]+chunkX[threadIdx.x];
    devW[tid]=chunkW[threadIdx.x];
    //if(devW[tid]!=0)printf("kernWXPY: devW[%d]: %e\n",tid,devW[tid]);
  }
}

#undef __FUNCT__
#define __FUNCT__ "kernWXMY"
__global__ void  kernWXMY(double* devY,double* devX, int* vlen, double* devW){

 /* w <- y + alpha*x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;

  __shared__ double chunkY[AXPYTCOUNT];
  __shared__ double chunkX[AXPYTCOUNT];
  __shared__ double chunkW[AXPYTCOUNT];
  /* printf("in kernWAXPY:alphaShared: %f, tid: %d, vlen: %d\n",alphaShared,tid,*vlen); */
  if(tid<*vlen){
    //if(devX[tid]!=0)printf("kernWXMY: devX[%d]: %e\n",tid,devX[tid]);
    //if(devY[tid]!=0)printf("kernWXMY: devY[%d]: %e\n",tid,devY[tid]);
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkY[threadIdx.x]-chunkX[threadIdx.x];
    devW[tid]=chunkW[threadIdx.x];
    //if(devW[tid]!=0)printf("kernWXMY: devW[%d]: %e\n",tid,devW[tid]);
  }
}

#undef __FUNCT__
#define __FUNCT__ "VecMAXPY_SeqGPU"
PetscErrorCode VecMAXPY_SeqGPU(Vec x,PetscInt nv,const PetscScalar* alpha,Vec *y){
  /* y = y + sum(a[i]*x[i]) */
  PetscFunctionBegin;
  //printf("VecMAXPY_SeqGPU: alpha: %e\n",*alpha);
  PetscErrorCode ierr;
  PetscInt i;  PetscInt bx,tx;
  dim3 dimGrid;
  dim3 dimBlock;
  hipError_t cudastatus;
  PetscScalar *devW;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=PETSC_NULL;

  cudastatus = hipMalloc((void**)&devW,x->map->n*sizeof(double));
  ierr = VecCheckCUDAStatus(cudastatus,"error in device malloc");CHKERRQ(ierr);
  cudastatus = hipMemset(devW,0,x->map->n*sizeof(double));
  ierr = VecCheckCUDAStatus(cudastatus,"error in device memset");CHKERRQ(ierr);

  /* assuming xwidth mem load isn't going to be an issue */
  bx=ceil((float)x->map->n/(float)AXPYTCOUNT);
  tx=AXPYTCOUNT;
  dimGrid.x=bx; dimGrid.y=1;
  dimBlock.x=tx; dimBlock.y=1;

  //printf("Number of vectors in MAXPY: %d\n",nv);
  //ierr = VecCheck_SeqGPU(x);CHKERRQ(ierr);
  for(i=0;i<nv;i++){
     if(y[i]->map->n!=x->map->n){
      SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
    }
    yd=(Vec_SeqGPU*)y[i]->data;
    if(yd->syncState==VEC_CPU){/* synch x */
      ierr = VecCopyOverH2D(y[i],yd->cpuptr);CHKERRQ(ierr);
      yd->syncState=VEC_SYNCHED;
    }
    hipDeviceSynchronize();
    cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha[i],sizeof(double),0,hipMemcpyHostToDevice);
    ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);
    //printf("Alpha[%d]: %e\n", i, alpha[i]);
    if(alpha[i]==0){
      continue;
    }else if(alpha[i]==1.){
      /* assuming width mem load isn't going to be an issue */
      kernXPY<<<dimGrid,dimBlock>>>(devW,yd->devptr,yd->length);
      ierr = VecCheckCUDAError("kernel call to kernXPY");CHKERRQ(ierr);
    }else{
      /* assuming width mem load isn't going to be an issue */
      kernAXPY<<<dimGrid,dimBlock>>>(devW,yd->devptr,yd->length);
      ierr = VecCheckCUDAError("kernel call to kernAXPY");CHKERRQ(ierr);
    }
  }
  if(xd->syncState==VEC_CPU){/* synch x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();
  kernXPY<<<dimGrid,dimBlock>>>(xd->devptr,devW,xd->length);
  ierr = VecCheckCUDAError("kernel call to kernXPY");CHKERRQ(ierr);

  cudastatus = hipFree(devW);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree");CHKERRQ(ierr);
  xd->syncState=VEC_GPU;
  //ierr = VecCheck_SeqGPU(x);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "kernXPY"
__global__ void  kernXPY(double* devY,double* devX, int* vlen){

 /* y <- y + x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;

  __shared__ double chunkY[AXPYTCOUNT];
  __shared__ double chunkX[AXPYTCOUNT];

  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkY[threadIdx.x]+=chunkX[threadIdx.x];
    devY[tid]=chunkY[threadIdx.x];
  }
}

#undef __FUNCT__
#define __FUNCT__ "VecAXPY_SeqGPU"
PetscErrorCode VecAXPY_SeqGPU(Vec y,PetscScalar alpha,Vec x){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt bx,tx;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid;
  dim3 dimBlock;
  hipError_t cudastatus;
  //printf("VecAXPY_SeqGPU\n");

  if(x->map->n!=y->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha,sizeof(double),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);
  hipDeviceSynchronize();
  if(alpha==1.){
    /* assuming width mem load isn't going to be an issue */
    bx=ceil((float)x->map->n/(float)AXPYTCOUNT);
    tx=AXPYTCOUNT;
    dimGrid.x=bx; dimGrid.y=1;
    dimBlock.x=tx; dimBlock.y=1;
    kernXPY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,yd->length);
    ierr = VecCheckCUDAError("kernel call to kernXPY");CHKERRQ(ierr);
  }else if(alpha!=0){
    /* assuming width mem load isn't going to be an issue */
    bx=ceil((float)x->map->n/(float)AXPYTCOUNT);
    tx=AXPYTCOUNT;
    dimGrid.x=bx; dimGrid.y=1;
    dimBlock.x=tx; dimBlock.y=1;
    kernAXPY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,yd->length);
    ierr = VecCheckCUDAError("kernel call to kernAXPY");CHKERRQ(ierr);
  }
  //hipDeviceSynchronize();
  yd->syncState=VEC_GPU;
  //ierr = VecCheck_SeqGPU(y);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "kernAXPY"
__global__ void  kernAXPY(double* devY,double* devX, int* vlen){

 /* y <- y + alpha*x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;

  __shared__ double alphaShared;
  __shared__ double chunkY[AXPYTCOUNT];
  __shared__ double chunkX[AXPYTCOUNT];

  alphaShared = dblScalarValue;

  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];

    chunkY[threadIdx.x]=devY[tid];
    chunkY[threadIdx.x]+=chunkX[threadIdx.x]*alphaShared;
    devY[tid]=chunkY[threadIdx.x];
  }
}

#undef __FUNCT__
#define __FUNCT__ "VecAXPBYPCZ_SeqGPU"
PetscErrorCode VecAXPBYPCZ_SeqGPU(Vec x, PetscScalar alpha, PetscScalar beta,\
                           PetscScalar gamma, Vec y, Vec z){

  PetscFunctionBegin;
  PetscErrorCode ierr;
  int blocks=ceil((float)x->map->n/(float)AXPBYPCZTCOUNT);/* assuming shared memory size is not an issue */
  int threads=AXPBYPCZTCOUNT;
  hipError_t cudastatus;
  Vec_SeqGPU* devX = (Vec_SeqGPU*)x->data;
  Vec_SeqGPU* devY = (Vec_SeqGPU*)y->data;
  Vec_SeqGPU* devZ = (Vec_SeqGPU*)z->data;

  double2 alphabeta;
  alphabeta.x = alpha;
  alphabeta.y = beta;
  hipDeviceSynchronize();
  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalar2Value"),(void*)&alphabeta,sizeof(double2),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);
  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&gamma,sizeof(double),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);

  dim3 dimGrid; dimGrid.x=blocks; dimGrid.y=1;
  dim3 dimBlock; dimBlock.x=threads; dimBlock.y=1;
  kernAXPBYPCZ<<<dimGrid,dimBlock>>>(devX->devptr,devY->devptr,devZ->devptr,devX->length);
  ierr = VecCheckCUDAError("launch kernAXPBYPCZ");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "kernAXPBYPCZ"
__global__ void kernAXPBYPCZ(double* devX, double* devY, double* devZ, int* len){
  /* x <- alpha*x + beta*y + gamma*z */
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int localn = *len;

  __shared__ double work[AXPBYPCZTCOUNT];
  __shared__ double chunkX[AXPBYPCZTCOUNT];
  __shared__ double chunkY[AXPBYPCZTCOUNT];
  __shared__ double chunkZ[AXPBYPCZTCOUNT];

  if(tid<localn){
    /* read in values to shared */
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkZ[threadIdx.x]=devZ[tid];

    /* do flops */
    if(dblScalarValue){
      work[threadIdx.x]=dblScalarValue*chunkZ[threadIdx.x];
    }else{
      work[threadIdx.x]=0.;
    }

    if(dblScalar2Value.y){
      work[threadIdx.x]+=dblScalar2Value.y*chunkY[threadIdx.x];
    }
    if(dblScalar2Value.x){
      work[threadIdx.x]+=dblScalar2Value.x*chunkX[threadIdx.x];
    }

    /* write back */
    devX[tid]=work[threadIdx.x];
  }
  return;
}

/*---------------------------- end level 2 ------------------------------ */

/*------------------------- pointwise functions ------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecPointwiseMult_SeqGPU"
PetscErrorCode VecPointwiseMult_SeqGPU(Vec w,Vec x,Vec y){
  PetscFunctionBegin;
  printf("VecPointwiseMult_SeqGPU\n");
  PetscErrorCode ierr;
  PetscInt bx,tx;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid;
  dim3 dimBlock;
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();
  /* assuming width mem load isn't going to be an issue */
  bx=ceil((float)y->map->n/(float)PMULTCOUNT);
  tx=PMULTCOUNT;
  dimGrid.x=bx; dimGrid.y=1;
  dimBlock.x=tx; dimBlock.y=1;
  kernPMULT<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
  ierr = VecCheckCUDAError("kernel call to kernPMULT");CHKERRQ(ierr);
  //hipDeviceSynchronize();
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "kernPMULT"
__global__ void  kernPMULT(double* devY,double* devX, int* vlen, double* devW){

 /* w <- x./y */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunkY[PMULTCOUNT];
  __shared__ double chunkX[PMULTCOUNT];
  __shared__ double chunkW[PMULTCOUNT];
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkX[threadIdx.x]*chunkY[threadIdx.x];
    devW[tid]=chunkW[threadIdx.x];
  }
}


#undef __FUNCT__
#define __FUNCT__ "VecMaxPointwiseDivide_SeqGPU"
PetscErrorCode VecMaxPointwiseDivide_SeqGPU(Vec x,Vec y,PetscReal *max){
  PetscFunctionBegin;
  printf("VecMaxPointwiseDivide_SeqGPU...");
  PetscErrorCode ierr;
  hipError_t cudastatus;
  PetscInt i,bx,tx;
  PetscScalar *maxlist=PETSC_NULL;
  PetscScalar *devmaxlist=PETSC_NULL;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid;
  dim3 dimBlock;
  if(x->map->n!=y->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();
  /* assuming width mem load isn't going to be an issue */
  bx=ceil((float)y->map->n/(float)PDIVTCOUNT);
  tx=PDIVTCOUNT;
  dimGrid.x=bx; dimGrid.y=1;
  dimBlock.x=tx; dimBlock.y=1;

  ierr = PetscMalloc(bx*sizeof(PetscScalar),&maxlist);CHKERRQ(ierr);

  cudastatus=hipMalloc((void**)&devmaxlist,bx*sizeof(PetscScalar));
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H");CHKERRQ(ierr);

  kernMAXPDIV<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,devmaxlist);
  ierr = VecCheckCUDAError("kernel call to kernPDIV");CHKERRQ(ierr);

  cudastatus=hipMemcpy(maxlist,devmaxlist,bx*sizeof(PetscScalar),hipMemcpyDeviceToHost);/* copy back */
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H");CHKERRQ(ierr);

  *max = maxlist[0];
  if(bx>1){/* final collapse */
    for(i=1;i<bx;i++){
      if(maxlist[i]>*max){
        *max=maxlist[i];
      }
    }
  }
  printf("max: %f\n",*max);
  ierr = PetscFree(maxlist);CHKERRQ(ierr);
  cudastatus = hipFree(devmaxlist);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

extern __shared__ double maxshared[]
#undef __FUNCT__
#define __FUNCT__ "kernMAX"
__global__ void  kernMAX(int* vlen, double* maxlist,double* max){
  int i,tid;
  tid = threadIdx.x;
  i = (blockDim.x+1)/2;
  __shared__  double* slist;
  slist = maxshared;

  slist[tid]=maxlist[tid];
  while(i<0){
    if(tid>i){
      slist[tid] = (slist[tid]>slist[tid+i])?slist[tid]:slist[tid+i];
    }
    __synchthreads();
    i/=2;
  }
  if(tid==0)*max = slist[0];
}




#undef __FUNCT__
#define __FUNCT__ "kernMAXPDIV"
__global__ void  kernMAXPDIV(double* devY,double* devX, int* vlen, double* maxlist){

 /* w <- max(abs(x./y)) */
  int i,tid;
  i = (PDIVTCOUNT+1)/2;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunkY[PDIVTCOUNT];
  __shared__ double chunkX[PDIVTCOUNT];
  __shared__ double chunkW[PDIVTCOUNT];
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    if(chunkY[threadIdx.x]!=0){
      chunkW[threadIdx.x]=fabs(__ddiv_rn(chunkX[threadIdx.x],chunkY[threadIdx.x]));
    }else{
      chunkW[threadIdx.x]=fabs(chunkX[threadIdx.x]);
    }
  }else{
    chunkW[threadIdx.x]=0.0;
  }
  __syncthreads();
  while(i>0){
    if(threadIdx.x<i && chunkW[threadIdx.x]<chunkW[threadIdx.x+i]){
      chunkW[threadIdx.x]=chunkW[threadIdx.x+i];
    }
    i/=2;
    __syncthreads();
  }
  if(threadIdx.x==0)maxlist[blockIdx.x]=chunkW[0];
}


#undef __FUNCT__
#define __FUNCT__ "VecPointwiseDivide_SeqGPU"
PetscErrorCode VecPointwiseDivide_SeqGPU(Vec w,Vec x,Vec y){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt bx,tx;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid;
  dim3 dimBlock;
  printf("Call to VecPointwiseDivide_SeqGPU\n");
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();
  /* assuming width mem load isn't going to be an issue */
  bx=ceil((float)y->map->n/(float)PDIVTCOUNT);
  tx=PDIVTCOUNT;
  dimGrid.x=bx; dimGrid.y=1;
  dimBlock.x=tx; dimBlock.y=1;

  kernPDIV<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
  ierr = VecCheckCUDAError("kernel call to kernPDIV");CHKERRQ(ierr);
  //hipDeviceSynchronize();
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "kernPDIV"
__global__ void  kernPDIV(double* devY,double* devX, int* vlen, double* devW){

 /* w <- x./y */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunkY[PDIVTCOUNT];
  __shared__ double chunkX[PDIVTCOUNT];
  __shared__ double chunkW[PDIVTCOUNT];
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    if(chunkX[threadIdx.x]*chunkY[threadIdx.x]!=0){
      chunkW[threadIdx.x]=__ddiv_rn(chunkX[threadIdx.x],chunkY[threadIdx.x]);
    }else{
      chunkW[threadIdx.x]=0;
    }
    devW[tid]=chunkW[threadIdx.x];
  }
}

/*--------------------------- end pointwise ---------------------------- */


/*-------------------------- norm functions ---------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecDotNorm2_SeqGPU"
PetscErrorCode VecDotNorm2_SeqGPU(Vec s, Vec t, PetscScalar *dp, PetscScalar *nm){
  PetscErrorCode ierr;
  PetscFunctionBegin;
  //printf("VecDotNorm2_SeqGPU\n");
  ierr = VecDot(s,t,dp); CHKERRQ(ierr);
  ierr = VecNorm(t,NORM_2,nm); CHKERRQ(ierr);
  //printf("dp: %e, nm: %e\n",*dp,*nm);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecNorm_SeqGPU"
PetscErrorCode VecNorm_SeqGPU(Vec x,NormType type,PetscReal* z){
  /* NormType: NORM_1=0,NORM_2=1,NORM_FROBENIUS=2,NORM_INFINITY=3,NORM_1_AND_2=4 */
  /* dealing with NORM_2 for now... */
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  double *devScratch,*devPartial,zhost;
  PetscInt i,chunks=0,segment,partialsize,scratchsize;
  hipStream_t* nrmstream;
  dim3 dimGrid, dimBlock;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;

  /* figure out how many chunks will be needed */
  chunks = ceil( ((float)x->map->n) /(float)(CHUNKWIDTH));
  nrmstream = (hipStream_t*)malloc(chunks*sizeof(hipStream_t));

  if(chunks>1){
    segment = (int) CHUNKWIDTH;
    dimGrid.x=ceil((CHUNKWIDTH)/(float)THRNRMCNT);
  }else{
    segment = x->map->n;
    dimGrid.x=ceil(((float)segment)/(float)THRNRMCNT);
  }
  dimBlock.x  = THRNRMCNT;
  partialsize = chunks*sizeof(double);
  scratchsize = partialsize*dimGrid.x;

  printf("chunks: %d, segmentsize: %d, dimGrid.x: %d, partialsize: %d, scratchsize: %d, CHUNKWIDTH: %d\n",
         chunks,xd->segment,dimGrid.x,partialsize,scratchsize,(int)(CHUNKWIDTH));
  cudastatus = hipMalloc((void**)&devScratch,scratchsize);
  ierr = VecCheckCUDAStatus(cudastatus,"devScratch alloc in VecNorm_SeqGPU");CHKERRQ(ierr);
  cudastatus = hipMemsetAsync(devScratch,0,scratchsize,xd->stream);
  ierr = VecCheckCUDAStatus(cudastatus,"devScratch memset in VecNorm_SeqGPU");CHKERRQ(ierr);

  cudastatus=hipMalloc((void**)&devPartial,partialsize);
  ierr = VecCheckCUDAStatus(cudastatus,"devPartial alloc in VecNorm_SeqGPU");CHKERRQ(ierr);
  cudastatus = hipMemsetAsync(devPartial,0,partialsize,xd->stream);
  ierr = VecCheckCUDAStatus(cudastatus,"devPartial memset in VecNorm_SeqGPU");CHKERRQ(ierr);

  if(xd->syncState==VEC_CPU){
    printf("xd state VEC_CPU: copying to device.\n");
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();/* make sure everyone is ready to go */

  for(i=0;i<chunks;i++){/* streaming async kernel calls */
    cudastatus=hipStreamCreate(&(nrmstream[i]));
    ierr = VecCheckCUDAStatus(cudastatus,"on hipStreamCreate");CHKERRQ(ierr);
    cudastatus=hipMemcpyAsync(xd->offset,&i,sizeof(int),hipMemcpyHostToDevice,nrmstream[i]);
    ierr = VecCheckCUDAStatus(cudastatus,"on copy array length H2D in VecNorm_SeqGPU");CHKERRQ(ierr);
    cudastatus=hipMemcpyAsync(xd->segment,&segment,sizeof(int),hipMemcpyHostToDevice,nrmstream[i]);
    ierr = VecCheckCUDAStatus(cudastatus,"on copy segment size H2D in VecNorm_SeqGPU");CHKERRQ(ierr);
    /* Overlapping execution */
    kernNorm2_double<<<dimGrid,dimBlock,0,nrmstream[i]>>>(xd->devptr,
                                                          xd->segment,
                                                          xd->length,
                                                          xd->offset,
                                                          (devScratch+i*dimGrid.x),
                                                          (devPartial+i));
    ierr = VecCheckCUDAError("kernNorm2 launch in VecNorm_SeqGPU");CHKERRQ(ierr);

    xd->vstat.h2d_count++;
    xd->vstat.h2d_bytes+=2*sizeof(int);
  }

  /* norm2 block reduction */
  dimGrid.x  = 1;
  dimBlock.x = chunks;
  hipDeviceSynchronize();/* make sure everyone is caught up */

  kernRedNorm_double<<<dimGrid,dimBlock,chunks*sizeof(double),xd->stream>>>(devPartial,xd->zval);
  ierr = VecCheckCUDAError("kernRedNorm_double launch in VecNorm_SeqGPU");CHKERRQ(ierr);
  hipDeviceSynchronize();/* make sure everyone is caught up */

  /* Copy back norm z */
  cudastatus=hipMemcpy(&zhost,xd->zval,sizeof(double),hipMemcpyDeviceToHost);/* copy back z */
  ierr = VecCheckCUDAStatus(cudastatus,"on copy znorm D2H in VecNorm_SeqGPU");CHKERRQ(ierr);
  *z = PetscSqrtScalar(zhost);
  xd->vstat.h2d_count++;
  xd->vstat.h2d_bytes+=sizeof(double);
  //printf("Znorm: %e, zhost: %e\n",*z,zhost);

  /* clean up resources */
  cudastatus = hipFree(devPartial);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  for(i=0;i<chunks;i++){
    cudastatus = hipStreamDestroy(nrmstream[i]);
    ierr = VecCheckCUDAStatus(cudastatus,"on hipStreamCreate");CHKERRQ(ierr);
  }
  free(nrmstream);
  cudastatus = hipFree(devScratch);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


extern __shared__ double arrayNorm_double[];
#undef __FUNCT__
#define __FUNCT__ "kernRedNorm_double"
__global__ void kernRedNorm_double(double* arr,double* z){/* reduction kernel */

  int i = (blockDim.x+1)/2;
  double* zptr=(double*)arrayNorm_double;

  zptr[threadIdx.x]=arr[threadIdx.x];
  __syncthreads();
  while(i>0){
    if(threadIdx.x<i){
      zptr[threadIdx.x]+=zptr[threadIdx.x+i];
    }
    __syncthreads();
    i/=2;
  }/* end while */
  if(threadIdx.x==0){
    *z=zptr[0];
  }
}

#undef __FUNCT__
#define __FUNCT__ "kernNorm2_double"
__global__ void kernNorm2_double(double* devX,
                                 int* segmentsize,int* arrsize,
                                 int* offset,double *scratch, double *z){

  __shared__ double chunkX[THRNRMCNT];
  __shared__ int n;    n   = *arrsize;
  __shared__ int seg;  seg = *segmentsize;
  __shared__ int off;  off = *offset;

  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int i = (blockDim.x+1)/2;
  int j = (gridDim.x+1)/2;
  int item = seg*off+tid;

  /* block level reduction */
  if(item<n){/* read in values to shared */
    chunkX[threadIdx.x]=devX[item]; /* offset values */
  }else{
    chunkX[threadIdx.x]=0.;
  }

  chunkX[threadIdx.x]*=chunkX[threadIdx.x];
  __syncthreads();

  while(i>0){
     if(threadIdx.x<i){
       chunkX[threadIdx.x]+=chunkX[threadIdx.x+i];
     }
     __syncthreads();
     i/=2;
  }/* end while */

  if(threadIdx.x==0){
    scratch[blockIdx.x]=chunkX[0];
  }
  __syncthreads();

  /* grid level reduction */
  while(j>0){
    if(threadIdx.x==0 && blockIdx.x<j){
      scratch[blockIdx.x]+=scratch[blockIdx.x+j];
    }
    __syncthreads();
    j/=2;
  }/* end while */
  if(tid==0){
     *z=scratch[blockIdx.x];
  }
  return;
}

/*
#undef __FUNCT__
#define __FUNCT__ "VecNorm1_SeqGPU"
PetscErrorCode VecNorm1_SeqGPU(Vec xin,NormType type,PetscReal* z)
{*/

/* NormType: NORM_1=0,NORM_2=1,NORM_FROBENIUS=2,NORM_INFINITY=3,NORM_1_AND_2=4 */
/* dealing with NORM_2 for now... */
/* z has 2 elements */

/*
  PetscErrorCode ierr;
  PetscFunctionBegin;
  printf("Call to VecNorm_SeqGPU\n");
  ierr = VecDot_SeqGPU(xin,xin,&z[0]);CHKERRQ(ierr);
  z[0]=PetscSqrtScalar(z[0]);
  printf("ZNORM: %f\n\n",*z);
  PetscFunctionReturn(0);
}*/


/*
#undef __FUNCT__
#define __FUNCT__ "kernReduceAbsSum"
PetscErrorCode kernReduceAbsSum(double * x, PetscReal* z){

}
*/

/* ------------------------------ end norms -------------------------------- */



#undef __FUNCT__
#define __FUNCT__ "VecGetArray_SeqGPU"
PetscErrorCode VecGetArray_SeqGPU(Vec v,PetscScalar **a){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *vd=(Vec_SeqGPU*)v->data;
  if(vd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecGetArray_SeqGPU the vector has not been created.");
  }
  //printf("Call to VecGetArray_SeqGPU\n");
  PetscInt flg1=0,flg2=0;
  PetscStackCheckByName(4,"DMDAVecGetArray",flg1);
  PetscStackCheckByName(6,"DMGlobalToLocalBegin",flg2);
  if(vd->syncState==VEC_GPU || flg1 || flg2){
    ierr = VecCopyOverD2H(v,vd->cpuptr); CHKERRQ(ierr);
    vd->syncState = VEC_CPU;
  }
  hipDeviceSynchronize();
  *a=vd->cpuptr;
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecRestoreArray_SeqGPU"
PetscErrorCode VecRestoreArray_SeqGPU(Vec v,PetscScalar **a){
  PetscFunctionBegin;
  /* printf("Call to VecRestoreArray_SeqGPU\n"); */
  PetscErrorCode ierr;
  Vec_SeqGPU *vd=(Vec_SeqGPU*)v->data;
  if(vd->syncState==VEC_CPU || vd->syncState==VEC_ALLOC){
    if(a){
      ierr = VecCopyOverH2D(v,*a);CHKERRQ(ierr);
      vd->syncState=VEC_GPU;
    }else{
      ierr = VecCopyOverH2D(v,vd->cpuptr);CHKERRQ(ierr);
      vd->syncState=VEC_SYNCHED;
    }
  }
  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCreateSeqGPU"
PetscErrorCode  VecCreateSeqGPU(MPI_Comm comm,PetscInt n,Vec *v){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  //printf("VecCreateSeqGPU\n");
  ierr = VecCreate(comm,v);CHKERRQ(ierr);
  ierr = VecSetSizes(*v,n,n);CHKERRQ(ierr);
  ierr = VecSetType(*v,VECSEQGPU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCopy_SeqGPU"
PetscErrorCode VecCopy_SeqGPU(Vec s,Vec d){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *sd=(Vec_SeqGPU*)s->data;
  Vec_SeqGPU *dd=(Vec_SeqGPU*)d->data;
  hipError_t cudastatus;

  if(d->map->n!=s->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
   }

  //printf("Call to VecCopy_SeqGPU\n");
  if(dd->syncState==VEC_UNALLOC){
     SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Destination vector unalloced.");
  }

  if(sd->syncState==VEC_ALLOC){
      PetscFunctionReturn(0);/* nothing to do */
  }

  if(sd->syncState==VEC_CPU){
    ierr = PetscMemcpy((void*)dd->cpuptr,(void*)sd->cpuptr,s->map->n*sizeof(PetscScalar));CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }

  //cudastatus = hipMemcpy(dd->length,sd->length,sizeof(int),hipMemcpyDeviceToDevice);
  cudastatus = hipMemcpyAsync(dd->zval,sd->zval,sizeof(double),hipMemcpyDeviceToDevice,dd->stream);
  ierr = VecCopyOverDevice(d,s); CHKERRQ(ierr);
  dd->syncState=sd->syncState;/* synch signal copy */
  hipDeviceSynchronize();
  //PetscBool same=PETSC_FALSE;
  //ierr = VecCompare_SeqGPU(s,d,&same,0,0);CHKERRQ(ierr);
  //if(!same)SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"Vector duplication failed.");
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecSwap_SeqGPU"
PetscErrorCode VecSwap_SeqGPU(Vec xin,Vec yin){
  /* PetscErrorCode ierr; */
  PetscFunctionBegin;
  printf("VecSwap_SeqGPU (***EMPTY***)\n");
  if (xin != yin) {
#if defined(PETSC_USE_REAL_SINGLE)
    //////// hipblasSswap(bn,VecCUSPCastToRawPtr(*xarray),one,VecCUSPCastToRawPtr(*yarray),one);
#else
    //////   hipblasDswap(bn,VecCUSPCastToRawPtr(*xarray),one,VecCUSPCastToRawPtr(*yarray),one);
#endif

  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDuplicate_SeqGPU"
PetscErrorCode VecDuplicate_SeqGPU(Vec win,Vec *V){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  //printf("Call to VecDuplicate_SeqGPU\n");
  ierr = VecCreate(((PetscObject)win)->comm,V);CHKERRQ(ierr);
  ierr = VecSetType(*V,VECSEQGPU);CHKERRQ(ierr);
  ierr = PetscObjectSetPrecision((PetscObject)*V,((PetscObject)win)->precision);CHKERRQ(ierr);
  ierr = VecSetSizes(*V,win->map->n,win->map->N);CHKERRQ(ierr);
  ierr = PetscLayoutReference(win->map,&(*V)->map);CHKERRQ(ierr);
  ierr = PetscOListDuplicate(((PetscObject)win)->olist,&((PetscObject)(*V))->olist);CHKERRQ(ierr);
  ierr = PetscFListDuplicate(((PetscObject)win)->qlist,&((PetscObject)(*V))->qlist);CHKERRQ(ierr);
  (*V)->stash.ignorenegidx = win->stash.ignorenegidx;
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDuplicateVecs_SeqGPU"
PetscErrorCode VecDuplicateVecs_SeqGPU(Vec vin, PetscInt m, Vec **Vlist){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt i=0;
  PetscValidHeaderSpecific(vin,VEC_CLASSID,1);
  PetscValidPointer(Vlist,3);
  if (m <= 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_OUTOFRANGE,"m must be > 0: m = %D",m);
  ierr = PetscMalloc(m*sizeof(Vec),Vlist);CHKERRQ(ierr);
  for(i=0;i<m;i++){
    ierr = VecDuplicate_SeqGPU(vin,*Vlist+i);CHKERRQ(ierr);
    hipDeviceSynchronize();
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecPlaceArray_SeqGPU"
PetscErrorCode  VecPlaceArray_SeqGPU(Vec x,const PetscScalar* array){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecPlaceArray_SeqGPU the vector has not been created.");
  }
  if(xd->unplacedarray){
     SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,
       "VecPlaceArray() was already called on this vector, without a call to VecResetArray()");
  }
  if(xd->syncState==VEC_GPU){/* assuming there is a logical reason for this copy up */
    ierr = VecCopyOverD2H(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  xd->unplacedarray=xd->cpuptr;
  xd->cpuptr=(PetscScalar*)array;
  ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
  xd->syncState=VEC_SYNCHED;
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecResetArray_SeqGPU"
PetscErrorCode  VecResetArray_SeqGPU(Vec x){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecResetArray_SeqGPU the vector has not been created.");
  }
  if(xd->cpuptr){
    ierr = PetscFree(xd->cpuptr);CHKERRQ(ierr);
  }
  xd->cpuptr=xd->unplacedarray;
  xd->unplacedarray=PETSC_NULL;
  ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
  xd->syncState=VEC_SYNCHED;
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecReplaceArray_SeqGPU"
PetscErrorCode  VecReplaceArray_SeqGPU(Vec x,const PetscScalar* array){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecResetArray_SeqGPU the vector has not been created.");
  }
  if(xd->cpuptr){
    ierr = PetscFree(xd->cpuptr);CHKERRQ(ierr);
  }
  xd->cpuptr=(PetscScalar*)array;
  ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
  xd->syncState=VEC_SYNCHED;
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "PinnedMalloc"
static PetscErrorCode  PinnedMalloc(PetscScalar** x,PetscInt n){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  cudastatus=hipHostAlloc((void**)x,n,0);
  ierr=VecCheckCUDAStatus(cudastatus,"in PinnedMalloc");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "PinnedFree"
static PetscErrorCode  PinnedFree(PetscScalar* x){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  cudastatus=hipHostFree(x);
  ierr=VecCheckCUDAStatus(cudastatus,"in PinnedFree");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCreate_SeqGPU"
PetscErrorCode  VecCreate_SeqGPU(Vec V){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  PetscMPIInt    size;
  Vec_SeqGPU* seqgpu=PETSC_NULL;
  //printf("Call to VecCreate_SeqGPU\n");
  /*  ierr = PetscNewLog(V,Vec_SeqGPU,&(V->data));CHKERRQ(ierr); */
  ierr = PetscMalloc(sizeof(Vec_SeqGPU),&seqgpu);
  V->data=(void*)seqgpu;
  ierr = MPI_Comm_size(((PetscObject)V)->comm,&size);CHKERRQ(ierr);
  if  (size > 1) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Cannot create VECSEQGPU on more than one process");

  if (V->map->bs == -1) V->map->bs = 1;
  ierr = PetscLayoutSetUp(V->map);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)V,VECSEQGPU);CHKERRQ(ierr);

  V->ops->dot             = VecDot_SeqGPU;
  V->ops->norm            = VecNorm_SeqGPU;
  V->ops->tdot            = VecTDot_SeqGPU;
  V->ops->scale           = VecScale_SeqGPU;
  V->ops->copy            = VecCopy_SeqGPU;
  V->ops->set             = VecSet_SeqGPU;
  V->ops->setvalues       = VecSetValues_SeqGPU;
  V->ops->swap            = VecSwap_SeqGPU;
  V->ops->axpy            = VecAXPY_SeqGPU;
  V->ops->axpby           = VecAXPBY_SeqGPU;
  V->ops->axpbypcz        = VecAXPBYPCZ_SeqGPU;
  V->ops->pointwisemult   = VecPointwiseMult_SeqGPU;
  V->ops->pointwisedivide = VecPointwiseDivide_SeqGPU;
  V->ops->maxpointwisedivide = VecMaxPointwiseDivide_SeqGPU;
  V->ops->setrandom       = VecSetRandom_SeqGPU;
  V->ops->dot_local       = VecDot_SeqGPU;
  V->ops->tdot_local      = VecTDot_SeqGPU;
  V->ops->norm_local      = VecNorm_SeqGPU;
  V->ops->maxpy           = VecMAXPY_SeqGPU;
  V->ops->mdot            = VecMDot_SeqGPU;
  /* V->ops->aypx            = VecAYPX_SeqGPU; */
  V->ops->waxpy           = VecWAXPY_SeqGPU;
  V->ops->dotnorm2        = VecDotNorm2_SeqGPU;
  V->ops->placearray      = VecPlaceArray_SeqGPU;
  V->ops->replacearray    = VecReplaceArray_SeqGPU;
  V->ops->resetarray      = VecResetArray_SeqGPU;
  V->ops->destroy         = VecDestroy_SeqGPU;
  V->ops->destroyvecs     = VecDestroyVecs_SeqGPU;
  V->ops->duplicate       = VecDuplicate_SeqGPU;
  V->ops->duplicatevecs   = VecDuplicateVecs_SeqGPU;
  V->ops->getarray        = VecGetArray_SeqGPU;
  V->ops->restorearray    = VecRestoreArray_SeqGPU;
  V->ops->getlocalsize    = VecGetLocalSize_SeqGPU;
  V->ops->getsize         = VecGetSize_SeqGPU;
  V->ops->view            = VecView_SeqGPU;
  V->petscnative=PETSC_FALSE;
  seqgpu->syncState      = VEC_UNALLOC;
  seqgpu->lifetime       = VEC_PERSIST;

  seqgpu->vstat.h2d_count=0;
  seqgpu->vstat.d2h_count=0;
  seqgpu->vstat.h2d_bytes=0;
  seqgpu->vstat.d2h_bytes=0;

  seqgpu->unplacedarray=PETSC_NULL;
  seqgpu->array_allocated=PETSC_NULL;
  seqgpu->array=PETSC_NULL;
  hipDeviceSynchronize();

  /* create an associated stream */
  cudastatus = hipStreamCreate(&(seqgpu->stream));
  ierr = VecCheckCUDAStatus(cudastatus,"on device hipStreamCreate");CHKERRQ(ierr);

  /* allocate the variable for vector size */
  cudastatus=hipMalloc((void**)&(seqgpu->length),sizeof(int));
  ierr = VecCheckCUDAStatus(cudastatus,"**** Alloc devlength in VecCreate_SeqGPU");CHKERRQ(ierr);
  /* send vec length size to device */
  cudastatus=hipMemcpyAsync((void*)seqgpu->length,
               (void*)&(V->map->n),sizeof(int),hipMemcpyHostToDevice,seqgpu->stream);
  ierr = VecCheckCUDAStatus(cudastatus,"**** Copy H2D devlength in VecCreate_SeqGPU");CHKERRQ(ierr);
  seqgpu->vstat.h2d_count++;
  seqgpu->vstat.h2d_bytes+=sizeof(int);

  /* allocate the vector on device */
  cudastatus=hipMalloc((void**)&(seqgpu->devptr),V->map->n*sizeof(double));
  ierr = VecCheckCUDAStatus(cudastatus,"**** Alloc of devptr in VecCreate_SeqGPU");CHKERRQ(ierr);

  cudastatus=hipMemsetAsync((void*)seqgpu->devptr,0,V->map->n*sizeof(double),seqgpu->stream);
  ierr = VecCheckCUDAStatus(cudastatus,"on device cudaMemSet");CHKERRQ(ierr);

  /* allocate the variable for vector offsets */
  cudastatus=hipMalloc((void**)&(seqgpu->offset),sizeof(int));
  ierr = VecCheckCUDAStatus(cudastatus,"**** Alloc devoffset in VecCreate_SeqGPU");CHKERRQ(ierr);

  /* allocate the variable for vector segment length */
  cudastatus=hipMalloc((void**)&(seqgpu->segment),sizeof(int));
  ierr = VecCheckCUDAStatus(cudastatus,"**** Alloc dev segment in VecCreate_SeqGPU");CHKERRQ(ierr);

  /* allocate the variable for vector single value result */
  cudastatus=hipMalloc((void**)&(seqgpu->zval),sizeof(double));
  ierr = VecCheckCUDAStatus(cudastatus,"**** Alloc dev zval in VecCreate_SeqGPU");CHKERRQ(ierr);

  /* using pinned memory */
  ierr = PinnedMalloc(&(seqgpu->cpuptr),V->map->n*sizeof(PetscScalar));CHKERRQ(ierr);
  //ierr = PetscMalloc(V->map->n*sizeof(PetscScalar),&(seqgpu->cpuptr));CHKERRQ(ierr);
  ierr = PetscMemzero(seqgpu->cpuptr,V->map->n*sizeof(PetscScalar));CHKERRQ(ierr);
  seqgpu->syncState=VEC_ALLOC;
  
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "VecDestroy_SeqGPU"
PetscErrorCode VecDestroy_SeqGPU(Vec v){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  PetscValidHeaderSpecific(v,VEC_CLASSID,1);
  //printf("VecDestroy_SeqGPU vstats: \n");
  //printf("...................................\n");
  //printf("H2D transfers: %d, byte count: %d\n",vd->vstat.h2d_count,vd->vstat.h2d_bytes);
  //printf("D2H transfers: %d, byte count: %d\n",vd->vstat.d2h_count,vd->vstat.d2h_bytes);
  //printf("...................................\n");
  /* static int counter = 1; */
  if(vd && vd->syncState != VEC_UNALLOC){
    if(vd->devptr){
      cudastatus=hipFree(vd->devptr);
      ierr=VecCheckCUDAStatus(cudastatus,"destroying vd->devptr in VecDestroy_SeqGPU");CHKERRQ(ierr);
      vd->devptr=PETSC_NULL;
    }
    if(vd->length){
      cudastatus=hipFree(vd->length);
      ierr=VecCheckCUDAStatus(cudastatus,"destroying vd->length in VecDestroy_SeqGPU");CHKERRQ(ierr);
      vd->length=PETSC_NULL;
    }
    if(vd->segment){
      cudastatus=hipFree(vd->segment);
      ierr=VecCheckCUDAStatus(cudastatus,"destroying vd->segment in VecDestroy_SeqGPU");CHKERRQ(ierr);
      vd->segment=PETSC_NULL;
    }
    if(vd->zval){
      cudastatus=hipFree(vd->zval);
      ierr=VecCheckCUDAStatus(cudastatus,"destroying vd->zval in VecDestroy_SeqGPU");CHKERRQ(ierr);
      vd->zval=PETSC_NULL;
    }
    if(vd->cpuptr){
      ierr = PinnedFree(vd->cpuptr); CHKERRQ(ierr);
    }
    vd->syncState = VEC_UNALLOC;
  }

  cudastatus = hipStreamDestroy(vd->stream);
  ierr = VecCheckCUDAError("call to hipStreamDestroy");CHKERRQ(ierr);

  ierr = PetscObjectDepublish(v);CHKERRQ(ierr);
#if defined(PETSC_USE_LOG)
  PetscLogObjectState((PetscObject)v,"Length=%D",v->map->n);
#endif
  ierr = PetscFree(v->data);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDestroyVecs_SeqGPU"
PetscErrorCode  VecDestroyVecs_SeqGPU(PetscInt m,Vec *vv){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt i;
   /* destroy the internal part */
  for(i=0;i<m;i++){
    ierr = VecDestroy(&vv[i]);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}





#undef __FUNCT__  
#define __FUNCT__ "VecView_Seq_ASCII"
static PetscErrorCode VecView_Seq_ASCII(Vec xin,PetscViewer viewer){
  PetscErrorCode    ierr;
  PetscInt          i,n = xin->map->n;
  const char        *name;
  PetscViewerFormat format;
  PetscScalar *xv;

  PetscFunctionBegin;
  ierr = VecGetArray_SeqGPU(xin,&xv);CHKERRQ(ierr);
  ierr = PetscViewerGetFormat(viewer,&format);CHKERRQ(ierr);
  if (format == PETSC_VIEWER_ASCII_MATLAB) {
    ierr = PetscObjectGetName((PetscObject)xin,&name);CHKERRQ(ierr);
    ierr = PetscViewerASCIIPrintf(viewer,"%s = [\n",name);CHKERRQ(ierr);
    for (i=0; i<n; i++) {
#if defined(PETSC_USE_COMPLEX)
      if (PetscImaginaryPart(xv[i]) > 0.0) {
        ierr = PetscViewerASCIIPrintf(viewer,"%18.16e + %18.16ei\n",PetscRealPart(xv[i]),PetscImaginaryPart(xv[i]));CHKERRQ(ierr);
      } else if (PetscImaginaryPart(xv[i]) < 0.0) {
        ierr = PetscViewerASCIIPrintf(viewer,"%18.16e - %18.16ei\n",PetscRealPart(xv[i]),-PetscImaginaryPart(xv[i]));CHKERRQ(ierr);
      } else {
        ierr = PetscViewerASCIIPrintf(viewer,"%18.16e\n",PetscRealPart(xv[i]));CHKERRQ(ierr);
      }
#else
      ierr = PetscViewerASCIIPrintf(viewer,"%18.16e\n",(double) xv[i]);CHKERRQ(ierr);
#endif
    }
    ierr = PetscViewerASCIIPrintf(viewer,"];\n");CHKERRQ(ierr);
  } else if (format == PETSC_VIEWER_ASCII_SYMMODU) {
    for (i=0; i<n; i++) {
#if defined(PETSC_USE_COMPLEX)
      ierr = PetscViewerASCIIPrintf(viewer,"%18.16e %18.16e\n",PetscRealPart(xv[i]),PetscImaginaryPart(xv[i]));CHKERRQ(ierr);
#else
      ierr = PetscViewerASCIIPrintf(viewer,"%18.16e\n",xv[i]);CHKERRQ(ierr);
#endif
    }
  } else if (format == PETSC_VIEWER_ASCII_VTK || format == PETSC_VIEWER_ASCII_VTK_CELL) {
    /* 
       state 0: No header has been output
       state 1: Only POINT_DATA has been output
       state 2: Only CELL_DATA has been output
       state 3: Output both, POINT_DATA last
       state 4: Output both, CELL_DATA last 
    */
    static PetscInt stateId = -1;
    int outputState = 0;
    PetscBool  hasState;
    int doOutput = 0;
    PetscInt bs, b;

    if (stateId < 0) {
      ierr = PetscObjectComposedDataRegister(&stateId);CHKERRQ(ierr);
    }
    ierr = PetscObjectComposedDataGetInt((PetscObject) viewer, stateId, outputState, hasState);CHKERRQ(ierr);
    if (!hasState) {
      outputState = 0;
    }
    ierr = PetscObjectGetName((PetscObject) xin, &name);CHKERRQ(ierr);
    ierr = VecGetBlockSize(xin, &bs);CHKERRQ(ierr);
    if ((bs < 1) || (bs > 3)) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE, "VTK can only handle 3D objects, but vector dimension is %d", bs);
    }
    if (format == PETSC_VIEWER_ASCII_VTK) {
      if (outputState == 0) {
        outputState = 1;
        doOutput = 1;
      } else if (outputState == 1) {
        doOutput = 0;
      } else if (outputState == 2) {
        outputState = 3;
        doOutput = 1;
      } else if (outputState == 3) {
        doOutput = 0;
      } else if (outputState == 4) {
        SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE, "Tried to output POINT_DATA again after intervening CELL_DATA");
      }
      if (doOutput) {
        ierr = PetscViewerASCIIPrintf(viewer, "POINT_DATA %d\n", n/bs);CHKERRQ(ierr);
      }
    } else {
      if (outputState == 0) {
        outputState = 2;
        doOutput = 1;
      } else if (outputState == 1) {
        outputState = 4;
        doOutput = 1;
      } else if (outputState == 2) {
        doOutput = 0;
      } else if (outputState == 3) {
        SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE, "Tried to output CELL_DATA again after intervening POINT_DATA");
      } else if (outputState == 4) {
        doOutput = 0;
      }
      if (doOutput) {
        ierr = PetscViewerASCIIPrintf(viewer, "CELL_DATA %d\n", n);CHKERRQ(ierr);
      }
    }
    ierr = PetscObjectComposedDataSetInt((PetscObject) viewer, stateId, outputState);CHKERRQ(ierr);
    if (name) {
      if (bs == 3) {
        ierr = PetscViewerASCIIPrintf(viewer, "VECTORS %s double\n", name);CHKERRQ(ierr);
      } else {
        ierr = PetscViewerASCIIPrintf(viewer, "SCALARS %s double %d\n", name, bs);CHKERRQ(ierr);
      }
    } else {
      ierr = PetscViewerASCIIPrintf(viewer, "SCALARS scalars double %d\n", bs);CHKERRQ(ierr);
    }
    if (bs != 3) {
      ierr = PetscViewerASCIIPrintf(viewer, "LOOKUP_TABLE default\n");CHKERRQ(ierr);
    }
    for (i=0; i<n/bs; i++) {
      for (b=0; b<bs; b++) {
        if (b > 0) {
          ierr = PetscViewerASCIIPrintf(viewer," ");CHKERRQ(ierr);
        }
#if !defined(PETSC_USE_COMPLEX)
        ierr = PetscViewerASCIIPrintf(viewer,"%G",xv[i*bs+b]);CHKERRQ(ierr);
#endif
      }
      ierr = PetscViewerASCIIPrintf(viewer,"\n");CHKERRQ(ierr);
    }
  } else if (format == PETSC_VIEWER_ASCII_VTK_COORDS) {
    PetscInt bs, b;

    ierr = VecGetBlockSize(xin, &bs);CHKERRQ(ierr);
    if ((bs < 1) || (bs > 3)) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE, "VTK can only handle 3D objects, but vector dimension is %d", bs);
    }
    for (i=0; i<n/bs; i++) {
      for (b=0; b<bs; b++) {
        if (b > 0) {
          ierr = PetscViewerASCIIPrintf(viewer," ");CHKERRQ(ierr);
        }
#if !defined(PETSC_USE_COMPLEX)
        ierr = PetscViewerASCIIPrintf(viewer,"%G",xv[i*bs+b]);CHKERRQ(ierr);
#endif
      }
      for (b=bs; b<3; b++) {
        ierr = PetscViewerASCIIPrintf(viewer," 0.0");CHKERRQ(ierr);
      }
      ierr = PetscViewerASCIIPrintf(viewer,"\n");CHKERRQ(ierr);
    }
  } else if (format == PETSC_VIEWER_ASCII_PCICE) {
    PetscInt bs, b;

    ierr = VecGetBlockSize(xin, &bs);CHKERRQ(ierr);
    if ((bs < 1) || (bs > 3)) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE, "PCICE can only handle up to 3D objects, but vector dimension is %d", bs);
    }
    ierr = PetscViewerASCIIPrintf(viewer,"%D\n", xin->map->N/bs);CHKERRQ(ierr);
    for (i=0; i<n/bs; i++) {
      ierr = PetscViewerASCIIPrintf(viewer,"%7D   ", i+1);CHKERRQ(ierr);
      for (b=0; b<bs; b++) {
        if (b > 0) {
          ierr = PetscViewerASCIIPrintf(viewer," ");CHKERRQ(ierr);
        }
#if !defined(PETSC_USE_COMPLEX)
        ierr = PetscViewerASCIIPrintf(viewer,"% 12.5E",xv[i*bs+b]);CHKERRQ(ierr);
#endif
      }
      ierr = PetscViewerASCIIPrintf(viewer,"\n");CHKERRQ(ierr);
    }
  } else {
    ierr = PetscObjectPrintClassNamePrefixType((PetscObject)xin,viewer,"Vector Object");CHKERRQ(ierr);
    for (i=0; i<n; i++) {
      if (format == PETSC_VIEWER_ASCII_INDEX) {
        ierr = PetscViewerASCIIPrintf(viewer,"%D: ",i);CHKERRQ(ierr);
      }
#if defined(PETSC_USE_COMPLEX)
      if (PetscImaginaryPart(xv[i]) > 0.0) {
        ierr = PetscViewerASCIIPrintf(viewer,"%G + %G i\n",PetscRealPart(xv[i]),PetscImaginaryPart(xv[i]));CHKERRQ(ierr);
      } else if (PetscImaginaryPart(xv[i]) < 0.0) {
        ierr = PetscViewerASCIIPrintf(viewer,"%G - %G i\n",PetscRealPart(xv[i]),-PetscImaginaryPart(xv[i]));CHKERRQ(ierr);
      } else {
        ierr = PetscViewerASCIIPrintf(viewer,"%G\n",PetscRealPart(xv[i]));CHKERRQ(ierr);
      }
#else
      ierr = PetscViewerASCIIPrintf(viewer,"%G\n",(double) xv[i]);CHKERRQ(ierr);
#endif
    }
  }
  ierr = PetscViewerFlush(viewer);CHKERRQ(ierr);
  ierr = VecRestoreArray_SeqGPU(xin,&xv);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}




















EXTERN_C_END
