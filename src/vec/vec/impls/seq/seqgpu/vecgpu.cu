#include "hip/hip_runtime.h"
#include <petscconf.h>
#include <petscsys.h>
//#include <petscerror.h>
PETSC_CUDA_EXTERN_C_BEGIN
#include <string.h>

#include <stdlib.h>
#include <float.h>
#include <private/vecimpl.h>          /*I "petscvec.h" I*/
#include <../src/vec/vec/impls/dvecimpl.h>
#include <../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h>
PETSC_CUDA_EXTERN_C_END




EXTERN_C_BEGIN

/* Misc constant memory variables (rarely used) */
__constant__ int     integerSymbol;
__constant__ int2    integer2Symbol;
__constant__ int3    integer3Symbol;
__constant__ int     devN;/* vector length */
__constant__ double  dblScalarValue;/* utility var */
__constant__ double2 dblScalar2Value;/* utility var */
__constant__ float   fltScalarValue;/* utility var */
__constant__ float2  fltScalar2Value;/* utility var */

/* error check variables */

static hipError_t ccs[16];
static hipError_t cms[16];

/* timer for vector functions */
#undef __FUNCT__
#define __FUNCT__ "vec_clock"
double vec_clock(){
  struct timezone tzp;
  struct timeval tp;
  gettimeofday (&tp, &tzp);
  return (tp.tv_sec + tp.tv_usec*1.0e-6);
}


__device__ void orcu_warpReduce32(int tid, volatile double* reducts) {
  reducts[tid]+=reducts[tid+16];
  reducts[tid]+=reducts[tid+8];
  reducts[tid]+=reducts[tid+4];
  reducts[tid]+=reducts[tid+2];
  reducts[tid]+=reducts[tid+1];
}

__device__ void orcu_warpReduce64(int tid, volatile double* reducts) {
  reducts[tid]+=reducts[tid+32];
  reducts[tid]+=reducts[tid+16];
  reducts[tid]+=reducts[tid+8];
  reducts[tid]+=reducts[tid+4];
  reducts[tid]+=reducts[tid+2];
  reducts[tid]+=reducts[tid+1];
}

/* Function unrolls work of warp for reductions */
__device__ void warpDotReduce(volatile double* sdata, int tid){
  if(blockDim.x>=64)sdata[tid]+= sdata[tid+32];
  if(blockDim.x>=32)sdata[tid]+= sdata[tid+16];
  if(blockDim.x>=16)sdata[tid]+= sdata[tid+8];
  if(blockDim.x>=8) sdata[tid]+= sdata[tid+4];
  if(blockDim.x>=4) sdata[tid]+= sdata[tid+2];
  if(blockDim.x>=2) sdata[tid]+= sdata[tid+1];
}

/* Function unrolls work of warp for reductions */
__device__ void warpReduce(volatile double* sdata, int tid){
  sdata[tid]+= sdata[tid+32];
  sdata[tid]+= sdata[tid+16];
  sdata[tid]+= sdata[tid+8];
  sdata[tid]+= sdata[tid+4];
  sdata[tid]+= sdata[tid+2];
  sdata[tid]+= sdata[tid+1];
}

/* Function unrolls work of warp for reductions */
__device__ void warpMaxReduce(volatile double* sdata, int tid){
  sdata[tid]= fmax(sdata[tid],sdata[tid+32]);
  sdata[tid]= fmax(sdata[tid],sdata[tid+16]);
  sdata[tid]= fmax(sdata[tid],sdata[tid+8]);
  sdata[tid]= fmax(sdata[tid],sdata[tid+4]);
  sdata[tid]= fmax(sdata[tid],sdata[tid+2]);
  sdata[tid]= fmax(sdata[tid],sdata[tid+1]);
}


/* ---------------------------------------------------------
// helper function for error checking from kernel launches
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecCheckCUDAError"
PetscErrorCode VecCheckCUDAError(const char *msg){

  PetscFunctionBegin;
  hipError_t err = hipGetLastError();
  if( hipSuccess != err){
    fprintf(stderr, "Cuda kernel error: %s: %s.\n", msg,hipGetErrorString(err));
    fflush(NULL);
    PetscFunctionReturn(PETSC_ERR_LIB);
  }
  PetscFunctionReturn(0);
}

/* ---------------------------------------------------------
// helper function for error checking from status codes
// exits on nonzero error code, else does nothing
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecCheckCUDAStatus"
PetscErrorCode VecCheckCUDAStatus(hipError_t cs,const char *msg){
  PetscFunctionBegin;
    if(cs!=hipSuccess){
      SETERRQ2(PETSC_COMM_SELF,0,"Cuda error: %s: %s.\n",msg,hipGetErrorString(cs));
    }
  PetscFunctionReturn(0);
}
/* -------------------- end error checkers ------------------- */




/* ****************************************************************************
 This code is now included in CUDA SDK 4.1+ as cuRAND, so it may be obsolete

 *****************************************************************************
 * This is a shared memory implementation that keeps the full 625 words of state
 * in shared memory. Faster for heavy random work where you can afford 
 *  the shared memory. */
/* Init by single seed - single threaded as only used once */
__device__ void mt19937si(uint seed){
    int	i;
    if(threadIdx.x == 0){
	mtNexts = 0;
	s_seeds[0] = seed;
	for(i = 1;i < NNN;i++){
	   seed = (INIT_MULT * (seed^(seed >> 30))+i);
	   s_seeds[i] = seed;
	}
    }
    __syncthreads();/* Ensure mtNexts set */
    return;
}

/* Init by array - single threaded as only used once */
__device__ void mt19937sai(uint* seeds,uint length){
    int i,j,k;
    mt19937si(ARRAY_SEED);
    if(threadIdx.x==0){
     i=1; j=0;
     for(k = NNN>length?NNN:length;k!=0;k--){
        s_seeds[i] = (s_seeds[i]^((s_seeds[i-1]^(s_seeds[i-1] >> 30))*1664525)) + seeds[j] + j;
	if(++i >= NNN){
          s_seeds[0] = s_seeds[NNN-1];
	  i = 1;
        }
        if(++j>=length)j = 0;
     }
     for(k = NNN-1; k!=0;k--){
       s_seeds[i] = (s_seeds[i] ^ ((s_seeds[i-1]^(s_seeds[i-1]>>30))*1566083941))-i;
       if(++i >= NNN){
         s_seeds[0] = s_seeds[NNN-1];
	 i=1;
       }
     }
     s_seeds[0] = 0x80000000;/* MSB is 1; assuring non-zero initial array */ 
    }
    __syncthreads();				/* Needed for mt19937w() */
    return;
}

/* Return next MT random by increasing thread ID for 1-227 threads. */
__device__ uint mt19937s(void){
    int		kk;
    uint	x;
    uint	y;
    int		tid = threadIdx.x;

    kk = (mtNexts + tid) % NNN;
    __syncthreads();				/* Finished with mtNexts */

    if (tid == blockDim.x - 1)mtNexts = kk + 1;			/* Will get modded on next call */
    x = s_seeds[kk] & UPPER_MASK;
    if(kk < NNN - MMM){
      x |= (s_seeds[kk+1]&LOWER_MASK);
      y = s_seeds[kk+MMM];
    }else if(kk < NNN-1){
      x |= (s_seeds[kk+1]&LOWER_MASK);
      y = s_seeds[kk + (MMM-NNN)];
    }else{					/* kk == N - 1 */
      x |= (s_seeds[0]&LOWER_MASK);
      y = s_seeds[MMM - 1];
    }
    y ^= x >> 1;
    if (x & 1)y ^= MATRIX_A;
    __syncthreads();				/* All done before we update */

    s_seeds[kk] = y;
    y ^= (y >> 11);				/* Tempering */
    y ^= (y <<  7) & TEMPER1;
    y ^= (y << 15) & TEMPER2;
    y ^= (y >> 18);
    return y;
}

/* General shared memory version for any number of threads.
 * Note only up to 227 threads are run at any one time,
 * the rest loop and block till all are done. */
__device__ uint mt19937sl(void){
  int jj,kk,tid;
  uint x,y;
  tid = threadIdx.x;
  kk = (mtNexts + tid) % NNN;
  __syncthreads();				/* Finished with mtNexts */

  if(tid == blockDim.x - 1)mtNexts = kk + 1;	/* Will get modded on next call */
  jj = 0;
  do{
    if(jj <= tid && tid < jj + NNN - MMM){
      x = s_seeds[kk] & UPPER_MASK;
      if(kk < NNN - MMM){
         x |= (s_seeds[kk+1]&LOWER_MASK);
	 y = s_seeds[kk + MMM];
      }else if (kk < NNN-1){
         x |= (s_seeds[kk + 1]&LOWER_MASK);
	 y = s_seeds[kk + (MMM-NNN)];
      }else{				/* kk == N - 1 */
         x |= (s_seeds[0]&LOWER_MASK);
         y = s_seeds[MMM-1];
      }

      y ^= x >> 1;
      if(x & 1) y ^= MATRIX_A;
    }
    __syncthreads();			/* All done before we update */
    if(jj <= tid && tid < jj+NNN-MMM) s_seeds[kk] = y;
    __syncthreads();

  }while ((jj += NNN-MMM) < blockDim.x);
  y ^= (y >> 11);				/* Tempering */
  y ^= (y <<  7) & TEMPER1;
  y ^= (y << 15) & TEMPER2;
  y ^= (y >> 18);
  return y;
}

#undef __FUNCT__
#define __FUNCT__ "kernRandS"
__global__ void kernRandS(uint* seeds){
  mt19937sai(seeds,gridDim.x);
}

#undef __FUNCT__
#define __FUNCT__ "kernRand"
__global__ void kernRand(double *x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  uint rval;
  if(tid<*n){
    rval = mt19937sl();
    x[tid] = ((double)rval/(double)UINT_MAX);
    /* printf("RAND value[%d]: %0.13f, rval: %u UINT_MAX: %u\n",
       tid,x[tid],rval,UINT_MAX); */
  }
}

#undef __FUNCT__
#define __FUNCT__ "VecSetRandom_SeqGPU"
PetscErrorCode VecSetRandom_SeqGPU(Vec x,PetscRandom r){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt i;
  uint *seeds=PETSC_NULL,*devseeds=PETSC_NULL;
  PetscScalar rval;
  dim3 dimBlock,dimGrid;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  if(xd->syncState==VEC_ALLOC || xd->syncState==VEC_CPU){
    for(i=0; i<x->map->n; i++){
       ierr = PetscRandomGetValue(r,&xd->cpuptr[i]);CHKERRQ(ierr);
    }
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }else if(xd->syncState==VEC_SYNCHED || xd->syncState==VEC_GPU){
    dimGrid.x=ceil((float)x->map->n/(float)TCOUNT);
    dimBlock.x=TCOUNT;
    ierr = PetscMalloc(dimGrid.x*sizeof(PetscInt),&seeds);CHKERRQ(ierr);
    for(i=0; i<dimGrid.x; i++){
       ierr = PetscRandomGetValue(r,&rval);CHKERRQ(ierr);
       seeds[i]=(uint)(UINT_MAX*rval);
    }
    cms[0] = hipMalloc((void**)&devseeds,dimGrid.x*sizeof(uint));
    ccs[0]=hipMemcpy(devseeds,seeds,dimGrid.x*sizeof(uint),hipMemcpyHostToDevice);
    #if(DEBUGVEC)
      ierr = VecCheckCUDAStatus(cms[0],"error in hipMalloc");CHKERRQ(ierr);
      ierr = VecCheckCUDAStatus(ccs[0],"on copy H2D in VecSetRandom_SeqGPU");CHKERRQ(ierr);
    #endif

    kernRandS<<<dimGrid,dimBlock>>>(devseeds);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernRandS launch");CHKERRQ(ierr);
    #endif
    kernRand<<<dimGrid,dimBlock>>>(xd->devptr,xd->length);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernRand launch");CHKERRQ(ierr);
    #endif
    ierr = PetscFree(seeds);CHKERRQ(ierr);
    hipDeviceSynchronize();
    cms[1] = hipFree(devseeds);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(cms[1],"in hipFree()");CHKERRQ(ierr);
    #endif
  }
  xd->syncState = VEC_GPU;
  PetscFunctionReturn(0);
}
/*------------------------ end random generator ------------------------*/



/*------------------------------ compare ------------------------------*/

#undef __FUNCT__
#define __FUNCT__ "VecCompare_SeqGPU"
PetscErrorCode VecCompare_SeqGPU(Vec x, Vec y, PetscBool *same, PetscInt offset, PetscInt blocksize){
  PetscFunctionBegin;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  Vec_SeqGPU* yd = (Vec_SeqGPU*)y->data;
  if(xd->syncState!=yd->syncState||xd->syncState==VEC_ALLOC||yd->syncState==VEC_ALLOC){
    *same=PETSC_FALSE;
    PetscFunctionReturn(0);
  }
  PetscErrorCode ierr;
  dim3 dimGrid, dimBlock;
  if(blocksize && !offset){
    dimGrid.x=ceil((float)blocksize/(float)TCOUNT);
  } else {
    dimGrid.x=ceil((float)x->map->n/(float)TCOUNT);
  }
  dimBlock.x=TCOUNT;
  hipError_t cudastatus;
  int *devsame=PETSC_NULL;
  int cpusame=0;
  int2 offset_bsize;
  offset_bsize.x = offset;
  offset_bsize.y = blocksize;
  if(xd->syncState==VEC_CPU && yd->syncState==VEC_CPU){
    ierr = PetscMemcmp((void*)&xd->cpuptr[offset],(void*)&yd->cpuptr[offset],blocksize,same);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }

  cudastatus = hipMalloc((void**)&devsame,sizeof(int));
  ierr = VecCheckCUDAStatus(cudastatus,"error in device malloc");CHKERRQ(ierr);

  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("integer2Symbol"),(void*)&offset_bsize,sizeof(int2),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);

  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("devN"),(void*)&x->map->n,sizeof(int),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);

  kernCompare<<<dimGrid,dimBlock,2*dimBlock.x*sizeof(double)>>>(xd->devptr,yd->devptr,xd->length,yd->length,devsame);
  ierr = VecCheckCUDAError("kernCompare launch");CHKERRQ(ierr);

  cudastatus=hipMemcpy(&cpusame,devsame,sizeof(int),hipMemcpyDeviceToHost);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H in VecCompare_SeqGPU");CHKERRQ(ierr);

  if(cpusame==1)*same=PETSC_TRUE;
  else *same=PETSC_FALSE;
  cudastatus = hipFree(devsame);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

extern __shared__ double sharedCompare[];
#undef __FUNCT__
#define __FUNCT__ "kernCompare"
__global__ void kernCompare(double* devX, double* devY, int* lx, int* ly, int* devsame){
  __shared__ unsigned char blockflag;
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int2 localOBS = integer2Symbol;
  int localn = localOBS.x+localOBS.y;
  int index = tid+localOBS.x;
  double value=0;
  double* chunkX = sharedCompare;
  double* chunkY = sharedCompare + blockDim.x;

  if(threadIdx.x==0)blockflag=0;
  __syncthreads();
  if(index<localn){
    /* read in values to shared */
    chunkX[threadIdx.x]=devX[index];
    chunkY[threadIdx.x]=devY[index];
    value = fabs(chunkX[threadIdx.x]-chunkY[threadIdx.x]);
    if(value>1e-16){
      #if(DEBUGVEC && VVERBOSE)
         printf("In kernCompare found an element mismatch: %e\n",value);
      #endif
      blockflag=1;
    }
    if(*lx!=*ly){
      #if(DEBUGVEC && VVERBOSE)
         printf("In kernCompare found length mismatch: lx: %d vs ly: %d\n",*lx,*ly);
      #endif
      blockflag=1;
    }
  }
  __syncthreads();
  if(threadIdx.x==0){
    if(blockflag)*devsame=0;
    else *devsame=1;
  }
  return;
}

/*-------------------------- end compare ----------------------------*/

/*----------------------- Vec info functions ------------------------*/

#undef __FUNCT__
#define __FUNCT__ "VecView_SeqGPU"
PetscErrorCode VecView_SeqGPU(Vec x,PetscViewer viewer){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  if(xd->syncState==VEC_GPU){
    ierr = VecCopyOverD2H(x,xd->cpuptr); CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();
  int i;
  ierr = PetscObjectPrintClassNamePrefixType((PetscObject)x,viewer,"Vector Object");CHKERRQ(ierr);
  for(i=0;i<x->map->n;i++){
    PetscViewerASCIIPrintf(viewer,"%G\n",xd->cpuptr[i]);
  }
  /* ierr= PetscViewerSetType(viewer,PETSCVIEWERASCII);CHKERRQ(ierr);*/
  /* ierr =VecView_Seq_ASCII(x,viewer);CHKERRQ(ierr); */
  ierr = PetscViewerFlush(viewer);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecGetLocalSize_SeqGPU"
PetscErrorCode VecGetLocalSize_SeqGPU(Vec x, PetscInt *localsize){
  PetscFunctionBegin;
  #if(DEBUGVEC && VVERBOSE)
     printf("Call to VecGetLocalSize_SeqGPU\n");
  #endif
  PetscValidHeaderSpecific(x,VEC_CLASSID,1);
  PetscValidIntPointer(localsize,2);
  PetscValidType(x,1);
  *localsize=x->map->n;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecGetSize_SeqGPU"
PetscErrorCode VecGetSize_SeqGPU(Vec x, PetscInt *globalsize){
  PetscFunctionBegin;
  #if(DEBUGVEC && VVERBOSE)
     printf("Call to VecGetSize_SeqGPU\n");
  #endif
  PetscValidHeaderSpecific(x,VEC_CLASSID,1);
  PetscValidIntPointer(globalsize,2);
  PetscValidType(x,1);
  *globalsize=x->map->N;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCheck_SeqGPU"
PetscErrorCode VecCheck_SeqGPU(Vec x){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  dim3 dimgrid(ceil((float)x->map->n/((float)TCOUNT)),1,1);
  dim3 dimblocks(TCOUNT,1,1);
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  printf("******************************************\n");
  kernCheck<<<dimgrid,dimblocks>>>(xd->devptr,xd->length);
  ierr = VecCheckCUDAError("Call to kernCheck. "); CHKERRQ(ierr);
  hipDeviceSynchronize();
  printf("******************************************\n");
  fflush(NULL);
  PetscFunctionReturn(0);
}
#undef __FUNCT__
#define __FUNCT__ "kernCheck"
__global__ void kernCheck(double* x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if(tid<*n){
    #if(DEBUGVEC && VVERBOSE)
       printf("kernCheck: x[%d]: %e, length: %d\n",tid,x[tid],*n);
    #endif
  }
}
/*------------------------------ end info -------------------------------*/

/*---------------------------- copy functions ---------------------------*/


/* ---------------------------------------------------------
// Copies a block of memory from one array to another both 
// of which are on the device
// *** Currently nonfunctional ***
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockDevice"
PetscErrorCode VecCopyBlockDevice(Vec d, Vec s, PetscInt doffset, PetscInt soffset, PetscInt blocksize){
  PetscFunctionBegin;
  printf("Call to VecCopyBlockDevice (**** EMPTY ****)\n");
  PetscFunctionReturn(0);
}


/* ---------------------------------------------------------
// Copies all elements from one allocated array to another.
// This is done asynchronously on the device Vec's streamID.
// Both array must be allocated and be of the same size otherwise
// PETSc will return an error.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecCopyOverDevice"
PetscErrorCode VecCopyOverDevice(Vec d,Vec s){
  PetscErrorCode ierr;

  PetscFunctionBegin;
  ierr = PetscInfo(d,"Copying vector on device only\n"); CHKERRQ(ierr);
  Vec_SeqGPU* dd = (Vec_SeqGPU*)d->data;
  Vec_SeqGPU* sd = (Vec_SeqGPU*)s->data;
  dim3 dimGrid;  dim3 dimBlock;
  if(s->map->n!=d->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  ccs[0]=hipMemcpyAsync(dd->devptr,sd->devptr,
                    s->map->n*sizeof(double),hipMemcpyDeviceToDevice,dd->streamid);
  PetscFunctionReturn(0);
}

/* ---------------------------------------------------------
// Helper function copies an integer array on device
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "kernCopyLen"
__global__ void kernCopyLen(int* ly,int* lx){
  if(threadIdx.x==0)*ly=*lx;
}


/* ---------------------------------------------------------
// Copies a block of elements from one host allocated array to an
// array allocated on the device. Does not check for allocation,
// only for failure if debugging is toggled.
// Copy is done asynchronously on the destination's Vec type's
// streamID.
// Both array must be allocated and be of the same size otherwise
// PETSc will return an error
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockH2D"
PetscErrorCode VecCopyBlockH2D(Vec v,PetscScalar *y, PetscInt offset, PetscInt blocksize){
  PetscErrorCode ierr;
  PetscFunctionBegin;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  ierr = PetscInfo(v,"Copying vec: cpu -> gpu\n"); CHKERRQ(ierr);
  ccs[0]=hipMemcpyAsync(&(vd->devptr[offset]),y,
               blocksize*sizeof(double),hipMemcpyHostToDevice,vd->streamid);
  PetscFunctionReturn(0);
}

/* ---------------------------------------------------------
// Copies all elements from one host allocated array to an
// array allocated on the device. Does not check for allocation,
// only for failure if debugging is toggled.
// Copy is done asynchronously on the destination's Vec type's
// streamID.
// Both array must be allocated and be of the same size otherwise
// PETSc will return an error
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecCopyOverH2D"
PetscErrorCode VecCopyOverH2D(Vec v,PetscScalar *y){
  PetscErrorCode ierr;
  PetscFunctionBegin;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  ccs[0]=hipMemcpyAsync(vd->devptr,y,
               v->map->n*sizeof(double),hipMemcpyHostToDevice,vd->streamid);
  ierr = PetscInfo(v,"Copying vec: cpu -> gpu\n"); CHKERRQ(ierr);
  #if(DEBUGVEC)
    ierr = VecCheckCUDAStatus(ccs[0],"on copy H2D in VecCopyOverH2D");CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}


/* ---------------------------------------------------------
// Copies a block of elements from one device allocated array to an
// array allocated on the host. Does not check for allocation,
// only for failure if debugging is toggled.
// Copy is done asynchronously on the destination's Vec type's
// streamID.
// Both array must be allocated and be of the same size otherwise
// PETSc will return an error
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockD2H"
PetscErrorCode VecCopyBlockD2H(Vec v,PetscScalar *y,PetscInt offset, PetscInt blocksize){
  PetscErrorCode ierr;
  PetscFunctionBegin;
  ierr = PetscInfo(v,"Copying vec: gpu -> cpu\n"); CHKERRQ(ierr);
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  ccs[0]=hipMemcpyAsync(y,&(vd->devptr[offset]),
               blocksize*sizeof(double),hipMemcpyDeviceToHost,vd->streamid);
  #if(DEBUGVEC)
    ierr = VecCheckCUDAStatus(ccs[0],"on copy D2H in VecCopyBlockD2H");CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}


/* ---------------------------------------------------------
// Copies all elements from one device allocated array to an
// array allocated on the host. Does not check for allocation,
// only for failure if debugging is toggled.
// Copy is done asynchronously on the destination's Vec type's
// streamID.
// Both array must be allocated and be of the same size otherwise
// PETSc will return an error
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecCopyOverD2H"
PetscErrorCode VecCopyOverD2H(Vec v,PetscScalar *y){
  PetscErrorCode ierr;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  PetscFunctionBegin;
  ierr = PetscInfo(v,"Copying vec: gpu -> cpu\n"); CHKERRQ(ierr);
  ccs[0]=hipMemcpyAsync(y,vd->devptr,v->map->n*sizeof(double),hipMemcpyDeviceToHost,vd->streamid);
  #if(DEBUGVEC)
    ierr = VecCheckCUDAStatus(ccs[0],"on copy D2H in VecCopyOverD2H");CHKERRQ(ierr); 
  #endif
  PetscFunctionReturn(0);
}

/*---------------------------- end copy functions --------------------------*/





/*------------------------------ set functions -----------------------------*/


/* ---------------------------------------------------------
// VecSetValues - Inserts or adds values into certain locations of a vector.
// INSERT and ADD VALUES both are implemented
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecSetValues_SeqGPU"
PetscErrorCode VecSetValues_SeqGPU(Vec x,PetscInt ni,const PetscInt ix[],const PetscScalar y[],InsertMode iora){
  PetscErrorCode ierr;
  PetscInt i;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  PetscInt *devix;
  PetscScalar *devy;
  dim3 grid,blocks;
  PetscFunctionBegin;
  ierr = PetscInfo(x,"setting gpu values\n"); CHKERRQ(ierr);
  if(xd->syncState==VEC_CPU || xd->syncState==VEC_SYNCHED){
    if(iora==INSERT_VALUES){
      for(i=0;i<ni;i++){
         xd->cpuptr[i]=y[i];
      }
      ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
      xd->syncState=VEC_SYNCHED;
    }else{/* ADD_VALUES */
       ierr = VecCheckCUDAStatus(hipMalloc((void**)&devix,ni*sizeof(PetscInt)),
                     "hipMalloc ADD_VALS");CHKERRQ(ierr);
       ierr = VecCheckCUDAStatus(hipMalloc((void**)&devy,ni*sizeof(PetscScalar)),
                     "hipMalloc ADD_VALS");CHKERRQ(ierr);
       ierr = VecCheckCUDAStatus(hipMemcpy(devix,ix,ni*sizeof(PetscInt),hipMemcpyHostToDevice),
                     "hipMemcpy ADD_VALS");CHKERRQ(ierr);
       ierr = VecCheckCUDAStatus(hipMemcpy(devy,y,ni*sizeof(PetscScalar),hipMemcpyHostToDevice),
                     "hipMemcpy ADD_VALS");CHKERRQ(ierr);
       grid.x = ceil((float)ni/((float)TCOUNT));
       blocks.x = TCOUNT;
       kernAddValues<<<grid,blocks>>>(xd->devptr,x->map->n,devix,ni,devy);
       hipDeviceSynchronize();
       ierr=VecCheckCUDAStatus(hipFree(devy),"hipFree devy");CHKERRQ(ierr);
       ierr=VecCheckCUDAStatus(hipFree(devix),"hipFree devix");CHKERRQ(ierr);
       printf("Call to VecSetValues_SeqGPU 1: ADD_VALUES\n");
    }
  }else{
      if(iora==INSERT_VALUES){/* not efficient */
        PetscScalar yval=0;
        for(i=0;i<ni;i++){
          yval=y[i];
          ierr = VecCopyBlockH2D(x,&yval,ix[i],1);CHKERRQ(ierr);
        }
      }else{/* ADD_VALUES */

        ierr = VecCheckCUDAStatus(hipMalloc((void**)&devix,ni*sizeof(PetscInt)),
                                  "hipMalloc ADD_VALS");CHKERRQ(ierr);
        ierr = VecCheckCUDAStatus(hipMalloc((void**)&devy,ni*sizeof(PetscScalar)),
                                  "hipMalloc ADD_VALS");CHKERRQ(ierr);
        ierr = VecCheckCUDAStatus(hipMemcpy(devix,ix,ni*sizeof(PetscInt),hipMemcpyHostToDevice),
                                  "hipMemcpy ADD_VALS");CHKERRQ(ierr);
        ierr = VecCheckCUDAStatus(hipMemcpy(devy,y,ni*sizeof(PetscScalar),hipMemcpyHostToDevice),
                                  "hipMemcpy ADD_VALS");CHKERRQ(ierr);
        grid.x = ceil((float)ni/((float)TCOUNT));
        blocks.x = TCOUNT;
        kernAddValues<<<grid,blocks>>>(xd->devptr,x->map->n,devix,ni,devy);
        ierr=VecCheckCUDAError("call to kernAddValues.");CHKERRQ(ierr);
        hipDeviceSynchronize();
        ierr=VecCheckCUDAStatus(hipFree(devy),"hipFree devy");CHKERRQ(ierr);
        ierr=VecCheckCUDAStatus(hipFree(devix),"hipFree devix");CHKERRQ(ierr);
        printf("Call to VecSetValues_SeqGPU 2: ADD_VALUES\n");
      }
      xd->syncState=VEC_GPU;
  }
  PetscFunctionReturn(0);
}

/* ---------------------------------------------------------
// VecSet - Sets all values of an allocated device array to
// a scalar alpha. Checks if deivce array is allocated.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecSet_SeqGPU"
PetscErrorCode VecSet_SeqGPU(Vec xin,PetscScalar alpha){
  PetscFunctionBegin;
  #if(DEBUGVEC)
    PetscErrorCode ierr;
  #endif
  dim3 dimGrid, dimBlock;
  dimGrid.x = ceil((float)xin->map->n/((float)TCOUNT));
  dimBlock.x = TCOUNT;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)xin->data;
  #if(DEBUGVEC && VVERBOSE)
     printf("Call to VecSet_SeqGPU, alpha: %e\n",alpha);
  #endif
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,
            PETSC_ERR_MEM,"*** In VecSet_SeqGPU, Vec not allocated. ***\n");
  }else{
    kernSet<<<dimGrid,dimBlock>>>(xd->devptr,alpha,xin->map->n);
    #if(DEBUGVEC)
      #if(VVERBOSE)
         printf("In VecSet_SeqGPU: blocks: %d, threads: %d\n",dimGrid.x, dimBlock.x);
      #endif
      ierr = VecCheckCUDAError("Call to kernSet."); CHKERRQ(ierr);
    #endif
    xd->syncState=VEC_GPU;
  }
  PetscFunctionReturn(0);
}

/* ---------------------------------------------------------
// Device kernel called by VecSetValues when insert type is 
// ADD VALUES.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "kernAddValues"
__global__ void kernAddValues(double* x, int n, int* xi, int ni,double *y){
  unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if(tid<ni) x[xi[tid]] += y[tid];
}

/* ---------------------------------------------------------
// Called by VecSet and set all values of an allocated array
// to a scalar alpha.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "kernSet"
__global__ void kernSet(double* x, double alpha, int n){
  unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if(tid<n) x[tid] = alpha;
}


/* ---------------------------------------------------------
// VecScale - Scales all values of an allocated device array
// by a scalar alpha. Checks if deivce array is allocated.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecScale_SeqGPU"
PetscErrorCode VecScale_SeqGPU(Vec x, PetscScalar alpha){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  dim3 dimGrid,dimBlock;
  dimGrid.x=ceil((float)x->map->n/((float)TCOUNT));
  dimBlock.x=TCOUNT;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  #if(DEBUGVEC && VVERBOSE)
     printf("VecScale_SeqGPU...alpha: %e\n",alpha);
  #endif
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,
            PETSC_ERR_MEM,
            "*** In call to VecScale_SeqGPU, arg Vec xin has not been allocated. ***\n");
  }else if(xd->syncState==VEC_CPU){
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();
  if(alpha==0.){
    ccs[0] = hipMemset(xd->devptr,0,x->map->n*sizeof(double));
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(ccs[0],"error in hipMemset");CHKERRQ(ierr);
    #endif
  }else if (alpha != 1.0){
    kernScale<<<dimGrid,dimBlock>>>(xd->devptr,alpha,x->map->n);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("Call to kernScale."); CHKERRQ(ierr);
    #endif
  }
  xd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}

/* ---------------------------------------------------------
// Device array scaling kernel called by VecScale
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "kernScale"
__global__ void kernScale(double* x, double alpha, int n){
  unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if(tid<n) x[tid] *= alpha;
}
/*---------------------------- end set and scale ---------------------------*/


/*-------------------------- dot product functions -------------------------*/

/* ---------------------------------------------------------
// Does the Dot product of a transposed vector yin...
// *** Currently non-functional ***
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecTDot_SeqGPU"
PetscErrorCode VecTDot_SeqGPU(Vec xin,Vec yin,PetscScalar *z){
  PetscFunctionBegin;
  printf("VecTDot_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}

/* ---------------------------------------------------------
// Computes the dot product of two vectors
// Checks for size mismatch and will synch the vectors to the
// device if needed.
// Orio tuned kernels are implemented for 3 size ranges:
// Manual tuned kernels are also included.
// Timers are available if toggled.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecDot_SeqGPU"
PetscErrorCode VecDot_SeqGPU(Vec x,Vec y,PetscScalar *z){
  PetscErrorCode ierr;
  PetscFunctionBegin;
  if(x->map->n!=y->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
#if(VTIMER)
  double start,finish,elapsed;
  static double mint,maxt=0.,cumt=0.,avg=0.;
  static int ccnt=0;
  //start = vec_clock();
#endif
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  if(xd->syncState==VEC_CPU){
    #if(DEBUGVEC && VVERBOSE)
       printf("xd state VEC_CPU: copying to device.\n");
    #endif
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  if(yd->syncState==VEC_CPU){
    #if(DEBUGVEC && VVERBOSE)
       printf("yd state VEC_CPU: copying to device.\n");
    #endif
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  double *devScratch;
  dim3 dimGrid, dimBlock;
#if(VMANDOT)
hipStream_t* dotstream;
  PetscInt i,chunks=0,segment,scratchsize;
  float threadscale =(DOTMPLIER*CHUNKWIDTH);
  /* figure out how many chunks will be needed */
  chunks = ceil( ((float)x->map->n) /threadscale);
  dotstream = (hipStream_t*)malloc(chunks*sizeof(hipStream_t));
  /* make sure the segment size for each chunk is correct */
  if(chunks>1) segment = (int) threadscale;
  else segment = x->map->n;
  dimGrid.x=ceil(((float)segment)/(float)THRDOTCNT);
  dimBlock.x = THRDOTCNT;
  /* allocate gridwide scratch array */
  scratchsize=chunks*dimGrid.x;
  cms[0] = hipMalloc((void**)&devScratch,scratchsize*sizeof(double));/* scratch pad */
  #if(DEBUGVEC)
    #if(VVERBOSE)
       printf("Call to VecDot_SeqGPU, chunks: %d segsize: %d grid: %d\n",chunks,segment,dimGrid.x);
    #endif
    ierr = VecCheckCUDAStatus(cms[0],"devScratch alloc in VecDot_SeqGPU");CHKERRQ(ierr);
  #endif
  hipDeviceSynchronize();/* make sure everyone is ready  */
  #if(VTIMER)
    start = vec_clock();
  #endif
  for(i=0;i<chunks;i++){  /* streaming async kernel calls */
    hipStreamCreate(&(dotstream[i]));
    /* Overlapping execution */
    kernDot<<<dimGrid,dimBlock,0,dotstream[i]>>>(xd->devptr,yd->devptr,
                                                          segment,
                                                          x->map->n,
                                                          i,
                                                          devScratch+i*dimGrid.x);
  }
  dimBlock.x  = THRDOTCNT2;
  hipDeviceSynchronize();

  while(scratchsize>1){/* begin next reduction */
    dimGrid.x = ceil((float)scratchsize/(float)THRDOTCNT2);
    if(dimGrid.x>MAXBLOCKS){
      SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_LIB,"Grid size too large for gpu capabilities.");
    }
    kernRedDot<<<dimGrid,dimBlock>>>(scratchsize,devScratch);
    scratchsize = dimGrid.x;
  }
  for(i=0;i<chunks;i++){
     hipStreamDestroy(dotstream[i]);
  }
  free(dotstream);
 
#else /* use Orio kernels */


  if(y->map->n>=1e7){
    int nthreads=320;
    int nstreams=4;
    /* Set per function L1 cache config */
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(orcu_dotkernel_1e7),hipFuncCachePreferL1);
    dimBlock.x=nthreads;
    dimGrid.x=112;
    /*create streams*/
    int istream, soffset, boffset;
    hipStream_t stream[nstreams+1];
    for(istream=0;istream<=nstreams;istream++) hipStreamCreate(&stream[istream]);
    /* divide up stream work */
    int chunklen=x->map->n/nstreams;
    int chunkrem=x->map->n%nstreams;
    int blks4chunk=dimGrid.x/nstreams;
    if(dimGrid.x%nstreams!=0) blks4chunk++;
    int blks4chunks=blks4chunk*nstreams;
    /* allocate scratch pad memory */
    hipMalloc((void**)&devScratch,(dimGrid.x+1)*sizeof(double));

    hipDeviceSynchronize();
    for(istream=0; istream<nstreams;istream++) {
      soffset=istream*chunklen;
      boffset=istream*blks4chunk;
      orcu_dotkernel_1e7<<<blks4chunk,dimBlock,0,stream[istream]>>>
                        (chunklen,xd->devptr+soffset,yd->devptr+soffset,devScratch+boffset);
    }
    if(chunkrem!=0){/* do remaining work */
      soffset=istream*chunklen;
      boffset=istream*blks4chunk;
      orcu_dotkernel_1e7<<<blks4chunk,dimBlock,0,stream[istream]>>>
                        (chunkrem,xd->devptr+soffset,yd->devptr+soffset,devScratch+boffset);
      blks4chunks++ ;
    }
    int orcu_blks=blks4chunks;
    int orcu_n;
    while (orcu_blks>1){/* second stage reduction */
      orcu_n=orcu_blks;
      orcu_blks=(orcu_blks+319)/320;
      orcu_dotblksum_1e7<<<orcu_blks,320>>>(orcu_n,devScratch);
    }
    /* destroy streams */
    for (istream=0; istream<=nstreams; istream++)hipStreamDestroy(stream[istream]);


  }else if(y->map->n>=1e6){
    int nthreads=160;
    int nstreams=2;
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(orcu_dotkernel_1e6),hipFuncCachePreferL1);
    dimBlock.x=nthreads;
    dimGrid.x=112;
    /*create streams*/
    int istream, soffset, boffset;
    hipStream_t stream[nstreams+1];
    for(istream=0;istream<=nstreams;istream++) hipStreamCreate(&stream[istream]);
    /* divide up stream work */
    int chunklen=x->map->n/nstreams;
    int chunkrem=x->map->n%nstreams;
    int blks4chunk=dimGrid.x/nstreams;
    if(dimGrid.x%nstreams!=0) blks4chunk++;
    int blks4chunks=blks4chunk*nstreams;
    /* allocate scratch pad memory */
    hipMalloc((void**)&devScratch,(dimGrid.x+1)*sizeof(double));

    hipDeviceSynchronize();
    for(istream=0; istream<nstreams;istream++) {
      soffset=istream*chunklen;
      boffset=istream*blks4chunk;
      orcu_dotkernel_1e6<<<blks4chunk,dimBlock,0,stream[istream]>>>
                        (chunklen,xd->devptr+soffset,yd->devptr+soffset,devScratch+boffset);
    }
    if(chunkrem!=0){/* do remaining work */
      soffset=istream*chunklen;
      boffset=istream*blks4chunk;
      orcu_dotkernel_1e6<<<blks4chunk,dimBlock,0,stream[istream]>>>
                        (chunkrem,xd->devptr+soffset,yd->devptr+soffset,devScratch+boffset);
      blks4chunks++ ;
    }
    int orcu_blks=blks4chunks;
    int orcu_n;
    while (orcu_blks>1){/* second stage reduction */
      orcu_n=orcu_blks;
      orcu_blks=(orcu_blks+159)/160;
      orcu_dotblksum_1e6<<<orcu_blks,160>>>(orcu_n,devScratch);
    }
    /* kill streams */
    for (istream=0; istream<=nstreams; istream++)hipStreamDestroy(stream[istream]);
 }else{
      /*calculate device dimensions*/
      dimBlock.x=256;
      dimGrid.x=112;
      /* allocate scratch pad memory */
      hipMalloc((void**)&devScratch,(dimGrid.x+1)*sizeof(double));
      orcu_dotkernel_1e5<<<dimGrid,dimBlock>>>
                 (y->map->n,yd->devptr,xd->devptr,devScratch);

      int orcu_blks=dimGrid.x;
      int orcu_n;
      while (orcu_blks>1){/* second stage reduction */
        orcu_n=orcu_blks;
        orcu_blks=(orcu_blks+255)/256;
        orcu_dotblksum_1e5<<<orcu_blks,256>>>(orcu_n,devScratch);
      }
  }/*end Orio vecsize if */
#endif /* end VMANUAL */
#if(VTIMER)
  finish=vec_clock();
#endif

  ccs[4]=hipMemcpy(z,devScratch,sizeof(double),hipMemcpyDeviceToHost);/* copy back */
  /* delete scratch memory */
  cms[3] = hipFree(devScratch);
#if(DEBUGVEC)
  ierr = VecCheckCUDAStatus(ccs[4],"on hipMemcpy(devScratch)");CHKERRQ(ierr);
#endif

 #if(DEBUGVEC)
   #if(VVERBOSE)
       printf("Zdot: %e\n",*z);
   #endif
    ierr = VecCheckCUDAStatus(cms[3],"on hipFree(devScratch)");CHKERRQ(ierr);
 #endif

 #if(VTIMER)
    // finish=vec_clock();
    elapsed=finish-start;
    cumt+=elapsed;
    if(!ccnt++){
      maxt=mint=avg=elapsed;
    }else{
      maxt=elapsed>maxt?elapsed:maxt;
      mint=elapsed<mint?elapsed:mint;
      avg=cumt/ccnt;
      if(!(ccnt%ITSHOW)){
        printf("VecDot_SeqGPU calls: %d, max: %e, min: %e, average: %e\n",
               ccnt,maxt,mint,avg);
      }
    }
  #endif
  PetscFunctionReturn(0);
}

#if(VMANDOT)
/* ---------------------------------------------------------
// Manual tuned second stage dot product parallel reduction.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "kernRedDot"
__global__ void kernRedDot(int n,double* scratch){/* reduction kernel */
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunk[THRDOTCNT2];
  double mysum =(tid<n)?scratch[tid]:0.;
  if(threadIdx.x>127)chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<128)mysum+=chunk[threadIdx.x+128];
  else return;
  if(threadIdx.x>63)chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<64)mysum+=chunk[threadIdx.x+64];
  else return;
  chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<32) warpReduce(chunk,threadIdx.x);
  else return;
  if(threadIdx.x==0){
    scratch[blockIdx.x]=chunk[0];
  }else return;
}

/* ---------------------------------------------------------
// Manual tuned first stage dot product parallel reduction.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "kernDot"
__global__ void kernDot(double* devX, double* devY,int segmentsize,
                        int arrsize, int offset, double* scratch){
  __shared__ double chunk[THRDOTCNT];
  unsigned int item = segmentsize*offset+blockDim.x*blockIdx.x+threadIdx.x;
  double mysum=(item<arrsize)?(devX[item]*devY[item]):0.;
  if(threadIdx.x>127)chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<128)mysum+=chunk[threadIdx.x+128];
  else return;
  if(threadIdx.x>63)chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<64)mysum+=chunk[threadIdx.x+64];
  else return;
  chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<32) warpReduce(chunk,threadIdx.x);
  else return;
  if(threadIdx.x==0){
    scratch[blockIdx.x]=chunk[0];
  }else return;
}

#else

/* 1e5 tuned kernels */
#undef __FUNCT__
#define __FUNCT__ "orcu_dotkernel_1e5"
__global__ void orcu_dotkernel_1e5(const int n, double* y, double* x, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  double orcu_var8193=0;
  for (int i=tid; i<=n-1; i+=gsize) {
    orcu_var8193=orcu_var8193+x[i]*y[i];
  }
  /*reduce single-thread results within a block*/
  __shared__ double orcu_vec8194[256];
  orcu_vec8194[threadIdx.x]=orcu_var8193;
  __syncthreads();
  if (threadIdx.x<128) 
    orcu_vec8194[threadIdx.x]+=orcu_vec8194[threadIdx.x+128];
  __syncthreads();
  if (threadIdx.x<64) 
    orcu_vec8194[threadIdx.x]+=orcu_vec8194[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) 
    orcu_warpReduce64(threadIdx.x,orcu_vec8194);
  __syncthreads();
  if (threadIdx.x==0) 
    reducts[blockIdx.x]=orcu_vec8194[0];
}

#undef __FUNCT__
#define __FUNCT__ "orcu_dotblksum_1e5"
__global__ void orcu_dotblksum_1e5(int orcu_n, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double orcu_vec8194[256];
  if (tid<orcu_n) 
    orcu_vec8194[threadIdx.x]=reducts[tid];
  else 
    orcu_vec8194[threadIdx.x]=0;
  __syncthreads();
  if (threadIdx.x<128) 
    orcu_vec8194[threadIdx.x]+=orcu_vec8194[threadIdx.x+128];
  __syncthreads();
  if (threadIdx.x<64) 
    orcu_vec8194[threadIdx.x]+=orcu_vec8194[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) 
    orcu_warpReduce64(threadIdx.x,orcu_vec8194);
  __syncthreads();
  if (threadIdx.x==0) 
    reducts[blockIdx.x]=orcu_vec8194[0];
}

/* 1e6 tuned kernels */
#undef __FUNCT__
#define __FUNCT__ "orcu_dotkernel_1e6"
__global__ void orcu_dotkernel_1e6(const int n, double* y, double* x, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  __shared__ double shared_y[160];
  __shared__ double shared_x[160];
  double orcu_var16389=0;
  for (int i=tid; i<=n-1; i+=gsize) {
    shared_y[threadIdx.x]=y[i];
    shared_x[threadIdx.x]=x[i];
    orcu_var16389=orcu_var16389+shared_x[threadIdx.x]*shared_y[threadIdx.x];
  }
  /*reduce single-thread results within a block*/
  __shared__ double orcu_vec16390[160];
  orcu_vec16390[threadIdx.x]=orcu_var16389;
  __syncthreads();
  if (threadIdx.x<64) 
    orcu_vec16390[threadIdx.x]+=orcu_vec16390[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) 
    orcu_warpReduce64(threadIdx.x,orcu_vec16390);
  if (threadIdx.x>=128&&threadIdx.x<144) 
    orcu_warpReduce32(threadIdx.x,orcu_vec16390);
  __syncthreads();
  if (threadIdx.x==0) 
    reducts[blockIdx.x]=orcu_vec16390[0]+orcu_vec16390[128];
}


#undef __FUNCT__
#define __FUNCT__ "orcu_dotblksum_1e6"
__global__ void orcu_dotblksum_1e6(int orcu_n, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double orcu_vec16390[160];
  if (tid<orcu_n) 
    orcu_vec16390[threadIdx.x]=reducts[tid];
  else 
    orcu_vec16390[threadIdx.x]=0;
  __syncthreads();
  if (threadIdx.x<64) 
    orcu_vec16390[threadIdx.x]+=orcu_vec16390[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) 
    orcu_warpReduce64(threadIdx.x,orcu_vec16390);
  if (threadIdx.x>=128&&threadIdx.x<144) 
    orcu_warpReduce32(threadIdx.x,orcu_vec16390);
  __syncthreads();
  if (threadIdx.x==0) 
    reducts[blockIdx.x]=orcu_vec16390[0]+orcu_vec16390[128];
}



/* 1e7 tuned kernels */
#undef __FUNCT__
#define __FUNCT__ "orcu_dotkernel_1e7"
__global__ void orcu_dotkernel_1e7(const int n, double* y, double* x, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  __shared__ double shared_y[160];
  __shared__ double shared_x[160];
  double orcu_var16389=0;
  for (int i=tid; i<=n-1; i+=gsize) {
    shared_y[threadIdx.x]=y[i];
    shared_x[threadIdx.x]=x[i];
    orcu_var16389=orcu_var16389+shared_x[threadIdx.x]*shared_y[threadIdx.x];
  }
  /*reduce single-thread results within a block*/
  __shared__ double orcu_vec16390[160];
  orcu_vec16390[threadIdx.x]=orcu_var16389;
  __syncthreads();
  if (threadIdx.x<64) 
    orcu_vec16390[threadIdx.x]+=orcu_vec16390[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) 
    orcu_warpReduce64(threadIdx.x,orcu_vec16390);
  if (threadIdx.x>=128&&threadIdx.x<144) 
    orcu_warpReduce32(threadIdx.x,orcu_vec16390);
  __syncthreads();
  if (threadIdx.x==0) 
    reducts[blockIdx.x]=orcu_vec16390[0]+orcu_vec16390[128];
}


#undef __FUNCT__
#define __FUNCT__ "orcu_dotblksum_1e7"
__global__ void orcu_dotblksum_1e7(int orcu_n, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double orcu_vec16390[160];
  if (tid<orcu_n) 
    orcu_vec16390[threadIdx.x]=reducts[tid];
  else 
    orcu_vec16390[threadIdx.x]=0;
  __syncthreads();
  if (threadIdx.x<64) 
    orcu_vec16390[threadIdx.x]+=orcu_vec16390[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) 
    orcu_warpReduce64(threadIdx.x,orcu_vec16390);
  if (threadIdx.x>=128&&threadIdx.x<144) 
    orcu_warpReduce32(threadIdx.x,orcu_vec16390);
  __syncthreads();
  if (threadIdx.x==0) 
    reducts[blockIdx.x]=orcu_vec16390[0]+orcu_vec16390[128];
}

#endif /* end VMANDOT -#if */


/* ---------------------------------------------------------
// Computes multiple dot products over a single x and multiple y's
// val[i] = x . y[i]
// Just a for-loop wrapper call to VecDot_SeqGPU
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecMDot_SeqGPU"
PetscErrorCode  VecMDot_SeqGPU(Vec x,PetscInt nv,const Vec y[],PetscScalar val[]){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt       i;
  #if(DEBUGVEC && VVERBOSE)
    printf("Call to VecMDot_SeqGPU\n");
  #endif
  for (i=0; i<nv; i++) {
    ierr = VecDot_SeqGPU(x,y[i],&val[i]);CHKERRQ(ierr);
    if(PetscIsInfOrNanScalar(val[i])){
      SETERRQ1(((PetscObject)x)->comm,PETSC_ERR_FP,"Infinite or not-a-number generated in mdot, entry %D",i);
    }
  }
  PetscFunctionReturn(0);
}
/*----------------------------- end dot ----------------------------- */



/* ---------------------------------------------------------
// AXPBY currently not implemented
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecAXPBY_SeqGPU"
PetscErrorCode VecAXPBY_SeqGPU(Vec yin,PetscScalar beta,PetscScalar alpha,Vec xin){
  /* Y = b*Y + a*X */
  PetscFunctionBegin;
  printf("Call to VecAXPBY_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}


/* ---------------------------------------------------------
// WAXPY
// Checks for vector size mismatch for each vector.
// Copies down the vector(s) to the GPU if needed
// Manual implmentation and Orio tuned kernels can be toggled
// Timers are available if toggled
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecWAXPY_SeqGPU"
PetscErrorCode VecWAXPY_SeqGPU(Vec w,PetscScalar alpha,Vec x,Vec y){
  /* w = y + alpha*x */
  PetscFunctionBegin;
  #if(VTIMER)
    double start,finish,elapsed;
    static double mint,maxt=0.,cumt=0.,avg=0.;
    static int ccnt=0;
    start = vec_clock();
  #endif
  PetscErrorCode ierr;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)w->data;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid, dimBlock;
  #if(DEBUGVEC && VVERBOSE)
     printf("VecWAXPY_SeqGPU...alpha: %e\n",alpha);
  #endif
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }

#if(VMANWXP)
  dimGrid.x=ceil((float)y->map->n/(float)AXPYTCOUNT);
  dimBlock.x=AXPYTCOUNT;
  hipDeviceSynchronize();
  if(alpha==0.0){
    ierr = VecCopyOverDevice(w,y);CHKERRQ(ierr);
    hipDeviceSynchronize();
  }else if(alpha==1.0){
    kernWXPY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,x->map->n,wd->devptr);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernel call to kernWXPY");CHKERRQ(ierr); 
    #endif
  }else if(alpha==-1.0){
    kernWYMX<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,x->map->n,wd->devptr);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernel call to kernWYMX");CHKERRQ(ierr);
    #endif
  }else{
    kernWAXPY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,alpha,x->map->n,wd->devptr);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernel call to kernWAXPY");CHKERRQ(ierr); 
    #endif
  }
#else
  if(w->map->n>=1e7){
    int nthreads=352;
    int nstreams=5;
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(orcu_waxpykernel_1e7),hipFuncCachePreferL1);
    dimBlock.x=nthreads;
    dimGrid.x=70;
    /*create streams*/
    int istream, soffset;
    hipStream_t stream[nstreams+1];
    for (istream=0;istream<=nstreams;istream++) hipStreamCreate(&stream[istream]);
    int chunklen=x->map->n/nstreams;
    int chunkrem=x->map->n%nstreams;


    /*invoke device kernel*/
    int blks4chunk=dimGrid.x/nstreams;
    if (dimGrid.x%nstreams!=0) blks4chunk++ ;
    for (istream=0; istream<nstreams; istream++ ) {
      soffset=istream*chunklen;
      orcu_waxpykernel_1e7<<<blks4chunk,dimBlock,0,stream[istream]>>>
                      (chunklen,alpha,xd->devptr+soffset,yd->devptr+soffset,wd->devptr+soffset);
    }
    if (chunkrem!=0) {
      soffset=istream*chunklen;
      orcu_waxpykernel_1e7<<<blks4chunk,dimBlock,0,stream[istream]>>>
                      (chunkrem,alpha,xd->devptr+soffset,yd->devptr+soffset,wd->devptr+soffset);
    }
    hipDeviceSynchronize();
    for (istream=0; istream<=nstreams; istream++) hipStreamDestroy(stream[istream]);

  }else if(w->map->n>=1e6){
    int nthreads=512;
    int nstreams=5;
    /*calculate device dimensions*/
    dimBlock.x=nthreads;
    dimGrid.x=70;
    /*create streams*/
    int istream, soffset;
    hipStream_t stream[nstreams+1];
    for (istream=0; istream<=nstreams; istream++) hipStreamCreate(&stream[istream]);
    int chunklen=x->map->n/nstreams;
    int chunkrem=x->map->n%nstreams;
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(orcu_waxpykernel_1e6),hipFuncCachePreferL1);

    /*invoke device kernel*/
    int blks4chunk=dimGrid.x/nstreams;
    if (dimGrid.x%nstreams!=0) blks4chunk++ ;
    for (istream=0; istream<nstreams; istream++ ) {
      soffset=istream*chunklen;
      orcu_waxpykernel_1e6<<<blks4chunk,dimBlock,0,stream[istream]>>>
                          (chunklen,alpha,xd->devptr+soffset,yd->devptr+soffset,wd->devptr+soffset);
    }
    if (chunkrem!=0) {
      soffset=istream*chunklen;
      orcu_waxpykernel_1e6<<<blks4chunk,dimBlock,0,stream[istream]>>>
                          (chunklen,alpha,xd->devptr+soffset,yd->devptr+soffset,wd->devptr+soffset);
    }
    hipDeviceSynchronize();
    for (istream=0; istream<=nstreams; istream++ ) hipStreamDestroy(stream[istream]);

  }else{

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(orcu_waxpykernel_1e5),hipFuncCachePreferL1);
    int nthreads=320;
    dimBlock.x=nthreads;
    dimGrid.x=112;
    /*invoke device kernel*/
    orcu_waxpykernel_1e5<<<dimGrid,dimBlock>>>(x->map->n,alpha,xd->devptr,yd->devptr,wd->devptr);

    hipError_t err=hipGetLastError();
    if (hipSuccess!=err) {
      printf("CUDA runtime error: %s@",hipGetErrorString(err));
    }
 

  }
#endif /* manual if */
  wd->syncState=VEC_GPU;
#if(VTIMER)
    hipDeviceSynchronize();
    finish=vec_clock();
    elapsed=finish-start;
    cumt+=elapsed;
    if(!ccnt++){
      maxt=mint=avg=elapsed;
    }
    maxt=elapsed>maxt?elapsed:maxt;
    mint=elapsed<mint?elapsed:mint;
    avg=cumt/ccnt;
    printf("VecWAXPY_SeqGPU calls: %d, max: %e, min: %e, average: %e\n",
             ccnt,maxt,mint,avg);
#endif
  PetscFunctionReturn(0);
}



#if(VMANWXP)
#undef __FUNCT__
#define __FUNCT__ "kernWAXPY"
__global__ void  kernWAXPY(double* devY,double* devX, double alpha, int vlen, double* devW){
  /* w <- y + alpha*x */
  unsigned int tid = blockIdx.x*blockDim.x+threadIdx.x;
  if(tid<vlen){
    devW[tid]=devY[tid]+alpha*devX[tid];
  }
}

#undef __FUNCT__
#define __FUNCT__ "kernWXPY"
__global__ void  kernWXPY(double* devY,double* devX, int vlen, double* devW){
 /* w <- y + x */
  unsigned int tid = blockIdx.x*blockDim.x+threadIdx.x;
  if(tid<vlen){
    devW[tid]=devY[tid]+devX[tid];
  }
}

#undef __FUNCT__
#define __FUNCT__ "kernWYMX"
__global__ void  kernWYMX(double* devY,double* devX, int vlen, double* devW){
 /* w <- y - x */
  unsigned int tid = blockIdx.x*blockDim.x+threadIdx.x;
  if(tid<vlen){
    devW[tid]=devY[tid]-devX[tid];
  }
}

#else

__global__ void orcu_waxpykernel_1e7(const int n, double a, double* x, double* y, double* w) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  __shared__ double shared_y[352];
  __shared__ double shared_x[352];
  __shared__ double shared_w[352];
  for (int i=tid; i<=n-1; i+=gsize) {
    shared_y[threadIdx.x]=y[i];
    shared_x[threadIdx.x]=x[i];
    shared_w[threadIdx.x]=a*shared_x[threadIdx.x]+shared_y[threadIdx.x];
    w[i]=shared_w[threadIdx.x];
  }
}

__global__ void orcu_waxpykernel_1e6(const int n, double a, double* x, double* y, double* w) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  __shared__ double shared_y[512];
  __shared__ double shared_x[512];
  __shared__ double shared_w[512];
  for (int i=tid; i<=n-1; i+=gsize) {
    shared_y[threadIdx.x]=y[i];
    shared_x[threadIdx.x]=x[i];
    shared_w[threadIdx.x]=a*shared_x[threadIdx.x]+shared_y[threadIdx.x];
    w[i]=shared_w[threadIdx.x];
  }
}

__global__ void orcu_waxpykernel_1e5(const int n, double a, double* x, double* y, double* w) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  __shared__ double shared_y[320];
  __shared__ double shared_x[320];
  __shared__ double shared_w[320];
  for (int i=tid; i<=n-1; i+=gsize) {
    shared_y[threadIdx.x]=y[i];
    shared_x[threadIdx.x]=x[i];
    shared_w[threadIdx.x]=a*shared_x[threadIdx.x]+shared_y[threadIdx.x];
    w[i]=shared_w[threadIdx.x];
  }
}
#endif







/* ---------------------------------------------------------
// MAXPY
// Accumilation of AXPY on to x, currently only the manual
// version is available. This is a simple loop over y[i] vectors.
// The function checks each pairing for size mismatch and will
// synch the vectors to the GPU is necessary.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecMAXPY_SeqGPU"
PetscErrorCode VecMAXPY_SeqGPU(Vec x,PetscInt nv,const PetscScalar* alpha,Vec *y){
  /* x = x + sum(a[i]*y[i]) */
  PetscFunctionBegin;
  if(DEBUGVEC && VVERBOSE)printf("VecMAXPY_SeqGPU: alpha: %e\n",*alpha);
  PetscErrorCode ierr;
  PetscInt i;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=PETSC_NULL;
  dim3 dimGrid;  dim3 dimBlock;
  dimGrid.x=ceil((float)x->map->n/(float)AXPYTCOUNT);
  dimBlock.x=AXPYTCOUNT;
  #if(DEBUGVEC)
    #if(VERBOSE)
       printf("Number of vectors in MAXPY: %d, blocks: %d, threads: %d\n",nv,dimGrid.x,dimBlock.x);
    #endif
    ierr = VecCheckCUDAStatus(cms[0],"error in device malloc VecMAXPY_SeqGPU");CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[0],"error in device memset VecMAXPY_SeqGPU");CHKERRQ(ierr);
  #endif
  if(xd->syncState==VEC_CPU){/* synch x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }

  for(i=0;i<nv;i++){
     if(y[i]->map->n!=x->map->n){
      SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
    }
    yd=(Vec_SeqGPU*)y[i]->data;
    if(yd->syncState==VEC_CPU){/* synch x */
      ierr = VecCopyOverH2D(y[i],yd->cpuptr);CHKERRQ(ierr);
      xd->syncState=VEC_SYNCHED;
    }
    hipDeviceSynchronize();
    if(alpha[i]==0){
      /* printf("no-op, continuing...\n"); */
      continue;
    }else{
      kernAXPY<<<dimGrid,dimBlock,0,xd->streamid>>>(xd->devptr,yd->devptr,alpha[i],y[i]->map->n);
      #if(DEBUGVEC)
        #if(VVERBOSE)
           if(!xd->devptr){
             printf("xd points to nothing.\n");
           }else if(!yd->devptr){
             printf("y[%d] points to nothing.\n",i);
           }
           printf("nv: %d, ylen[%d]: %d, alpha[%d]: %e, xlen: %d\n",nv,i,y[i]->map->n,i,alpha[i],x->map->n);
        #endif
        ierr = VecCheckCUDAError("kernel call to kernAXPY in VecMAXPY_SeqGPU");CHKERRQ(ierr);
      #endif
    }
  }
  xd->syncState=VEC_GPU;
  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}


/* ---------------------------------------------------------
// Device function XPY: x = x + y
// Labels are off, but this function is merely AXPY is alpha
// is equal to 1.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "kernXPY"
__global__ void  kernXPY(double* devY,double* devX, int vlen){
 /* y <- y + x */
  unsigned int tid = blockIdx.x*blockDim.x+threadIdx.x;
  if(tid<vlen){
    devY[tid]+=devX[tid];
  }
}



/* ---------------------------------------------------------
// MAXPY
// Accumilation of AXPY on to x, currently only the manual
// version is available. This is a simple loop over y[i] vectors.
// The function checks each pairing for size mismatch and will
// synch the vectors to the GPU is necessary.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecAXPY_SeqGPU"
PetscErrorCode VecAXPY_SeqGPU(Vec y,PetscScalar alpha,Vec x){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  if(x->map->n!=y->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  #if(DEBUGVEC && VVERBOSE)
     printf("VecAXPY_SeqGPU\n");
  #endif
  dim3 dimGrid,dimBlock;

#if(VMANXPY)
  dimGrid.x=ceil((float)x->map->n/(float)AXPYTCOUNT);
  dimBlock.x=AXPYTCOUNT;
  hipDeviceSynchronize();
  kernAXPY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,alpha,y->map->n);
  #if(DEBUGVEC)
   ierr = VecCheckCUDAError("kernel call in VecAXPY_SeqGPU");CHKERRQ(ierr);
  #endif

#else
   if(x->map->n>=1e7){
     int nthreads=352;
     dimBlock.x=nthreads;
     dimGrid.x=14;
     hipFuncSetCacheConfig(reinterpret_cast<const void*>(orcu_axpykernel_1e7),hipFuncCachePreferL1);
     /*copy data from host to device*/
     orcu_axpykernel_1e7<<<dimGrid,dimBlock>>>(x->map->n,alpha,yd->devptr,xd->devptr);
     hipError_t err=hipGetLastError();
     if (hipSuccess!=err) {
       printf("orcuda axpy_1e7 kernel error: %s\n",hipGetErrorString(err));
       PetscFunctionReturn(PETSC_ERR_LIB);
     }
   }else if(x->map->n>=1e6){
       int nthreads=288;
       int nstreams=2;
       /*calculate device dimensions*/
       dimBlock.x=nthreads;
       dimGrid.x=28;
       /*create streams*/
       int istream, soffset;
       hipStream_t stream[nstreams+1];
       for (istream=0; istream<=nstreams;istream++)hipStreamCreate(&stream[istream]);
       int chunklen=x->map->n/nstreams;
       int chunkrem=x->map->n%nstreams;

       hipFuncSetCacheConfig(reinterpret_cast<const void*>(orcu_axpykernel_1e6),hipFuncCachePreferL1);
       /*invoke device kernel*/
       int blks4chunk=dimGrid.x/nstreams;
       if(dimGrid.x%nstreams!=0) blks4chunk++ ;
       for(istream=0; istream<nstreams; istream++ ) {
         soffset=istream*chunklen;
         orcu_axpykernel_1e6<<<blks4chunk,dimBlock,0,stream[istream]>>>
                             (chunklen,alpha,yd->devptr+soffset,xd->devptr+soffset);
       }
       if (chunkrem!=0) {
         soffset=istream*chunklen;
         orcu_axpykernel_1e6<<<blks4chunk,dimBlock,0,stream[istream]>>>
                             (chunkrem,alpha,yd->devptr+soffset,xd->devptr+soffset);
       }
       hipDeviceSynchronize();
       for (istream=0; istream<=nstreams; istream++ )  hipStreamDestroy(stream[istream]);

   }else{
     /*calculate device dimensions*/
     dimBlock.x=512;
     dimGrid.x=112;
     /*invoke device kernel*/
     orcu_axpykernel_1e5<<<dimGrid,dimBlock>>>(x->map->n,alpha,yd->devptr,xd->devptr);
   }

#endif

  yd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}



#if(VMANXPY)
__global__ void orcu_axpykernel_1e5(const int n, double a, double* y, double* x) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  for (int i=tid; i<=n-1; i+=gsize) {
    y[i]=y[i]+a*x[i];
  }
}

__global__ void orcu_axpykernel_1e6(const int n, double a, double* y, double* x) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  __shared__ double shared_y[288];
  __shared__ double shared_x[288];
  for (int i=tid; i<=n-1; i+=gsize) {
    shared_y[threadIdx.x]=y[i];
    shared_x[threadIdx.x]=x[i];
    shared_y[threadIdx.x]=shared_y[threadIdx.x]+a*shared_x[threadIdx.x];
    y[i]=shared_y[threadIdx.x];
  }
}

__global__ void orcu_axpykernel_1e7(const int n, double a, double* y, double* x) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  __shared__ double shared_y[352];
  __shared__ double shared_x[352];
  for (int i=tid; i<=n-1; i+=gsize) {
    shared_y[threadIdx.x]=y[i];
    shared_x[threadIdx.x]=x[i];
    shared_y[threadIdx.x]=shared_y[threadIdx.x]+a*shared_x[threadIdx.x];
    y[i]=shared_y[threadIdx.x];
  }
}
#endif




/* ---------------------------------------------------------
// Device kernel for AXPY
// Manual version
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "kernAXPY"
__global__ void  kernAXPY(double* devY,double* devX,double alpha, int vlen){
 /* y <- y + alpha*x */
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  if(tid<vlen){
    devY[tid]+=alpha*devX[tid];
  }
}



/* ---------------------------------------------------------
// AXPBYPCZ: x = a*x + b*y + c*z
// Implemented, but currently only used by bicgs
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecAXPBYPCZ_SeqGPU"
PetscErrorCode VecAXPBYPCZ_SeqGPU(Vec x,PetscScalar alpha,PetscScalar beta,PetscScalar gamma,Vec y,Vec z){
  PetscFunctionBegin;
  #if(DEBUGVEC)
     PetscErrorCode ierr;
     #if(VVERBOSE)
        printf("Call to VecAXPBYPCZ_SeqGPU\n");
     #endif
  #endif
  Vec_SeqGPU* devX = (Vec_SeqGPU*)x->data;
  Vec_SeqGPU* devY = (Vec_SeqGPU*)y->data;
  Vec_SeqGPU* devZ = (Vec_SeqGPU*)z->data;
  double2 alphabeta;  alphabeta.x = alpha;  alphabeta.y = beta;
  dim3 dimGrid, dimBlock;
  dimGrid.x=ceil((float)x->map->n/(float)AXPBYPCZTCOUNT);
  dimBlock.x=AXPBYPCZTCOUNT;
  ccs[0]=hipMemcpyToSymbol(HIP_SYMBOL("dblScalar2Value"),(void*)&alphabeta,sizeof(double2),0,hipMemcpyHostToDevice);
  ccs[1]=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&gamma,sizeof(double),0,hipMemcpyHostToDevice);
  #if(DEBUGVEC)
   ierr = VecCheckCUDAStatus(ccs[0],"error in symbol copy to device");CHKERRQ(ierr);
   ierr = VecCheckCUDAStatus(ccs[1],"error in symbol copy to device");CHKERRQ(ierr);
  #endif
  hipDeviceSynchronize();
  kernAXPBYPCZ<<<dimGrid,dimBlock>>>(devX->devptr,devY->devptr,devZ->devptr,devX->length);
  #if(DEBUGVEC)
     ierr = VecCheckCUDAError("launch kernAXPBYPCZ");CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}


/* ---------------------------------------------------------
// Device kernel for AXPBYPCZ: x = a*x + b*y + c*z
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
extern __shared__ double sharedAXPBYPCZ[];
#undef __FUNCT__
#define __FUNCT__ "kernAXPBYPCZ"
__global__ void kernAXPBYPCZ(double* devX, double* devY, double* devZ, int* len){
  /* x <- alpha*x + beta*y + gamma*z */
  int localn = *len;
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  double work;
  if(tid<localn){
    /* do flops */
    if(dblScalarValue){
      work=dblScalarValue*devZ[tid];
    }else{
      work=0.;
    }

    if(dblScalar2Value.y){
      work+=dblScalar2Value.y*devY[tid];
    }
    if(dblScalar2Value.x){
      work+=dblScalar2Value.x*devX[tid];
    }
    /* write back */
    devX[tid]=work;
  }
  return;
}
/*---------------------------- end level 2 ------------------------------ */



/*------------------------- pointwise functions ------------------------- */

/* ---------------------------------------------------------
// Function which multiplies elementwise two vectors X and Y
// storing the result into a third vector W.
// Checks for size mismatch and synchs to the device if needed
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecPointwiseMult_SeqGPU"
PetscErrorCode VecPointwiseMult_SeqGPU(Vec w,Vec x,Vec y){
  PetscFunctionBegin;
  #if(DEBUGVEC && VERBOSE)
     printf("VecPointwiseMult_SeqGPU\n");
  #endif
  PetscErrorCode ierr;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid, dimBlock;
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  dimGrid.x=ceil((float)y->map->n/(float)PMULTCOUNT);
  dimBlock.x=PMULTCOUNT;
  hipDeviceSynchronize();
  kernPMULT<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
  #if(DEBUGVEC)
     ierr = VecCheckCUDAError("kernel call to kernPMULT");CHKERRQ(ierr);
  #endif
  wd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}

/* ---------------------------------------------------------
// Device kernel for pointwise multiply
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "kernPMULT"
__global__ void  kernPMULT(double* devY,double* devX, int* vlen, double* devW){
 /* w <- x./y */
  unsigned int tid = blockIdx.x*blockDim.x+threadIdx.x;
  if(tid<*vlen){
    devW[tid]=devX[tid]*devY[tid];
  }
}

/* ---------------------------------------------------------
// VecMaxPointwiseDivide_SeqGPU
// Function which calculates the elementwise division of vector
// X/Y, if one element is zero then X is just returned, the
// maximum value of all the resulting elements is returned to
// the host (max).
// The function implements a two stage reduction.
// Currently only the manual tuned version is implemented.
// Checks for size mismatch of arrays and synchs to device if
// needed.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecMaxPointwiseDivide_SeqGPU"
PetscErrorCode VecMaxPointwiseDivide_SeqGPU(Vec x,Vec y,PetscReal *max){
  PetscFunctionBegin;
  if(x->map->n!=y->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  #if(VTIMER)
   double start,finish,elapsed;
   static double mint,maxt=0.,cumt=0.,avg=0.;
   static int ccnt=0;
   start = vec_clock();
  #endif
  PetscErrorCode ierr;
  PetscScalar *devScratch;
  PetscInt i,chunks=0,segment,scratchsize;
  hipStream_t* pwdstream;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid, dimBlock;
  /* Size of workload for the kernels */
  float threadscale = MAXMPLIER*CHUNKWIDTH;

  if(yd->syncState==VEC_CPU){/* synch up y */
    #if(DEBUGVEC && VVERBOSE)
       printf("yd state VEC_CPU: copying to device.\n");
    #endif
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    #if(DEBUGVEC && VERBOSE)
       printf("xd state VEC_CPU: copying to device.\n");
    #endif
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  /* figure out how many chunks will be needed */
  chunks = ceil( ((float)x->map->n) / threadscale);
  pwdstream = (hipStream_t*)malloc(chunks*sizeof(hipStream_t));
  /* make sure the segment size for each chunk is correct */
  if(chunks>1)segment = (int) threadscale;
  else segment = x->map->n;
  dimGrid.x=ceil(((float)segment)/(float)PDIVTCOUNT);
  dimBlock.x  = PDIVTCOUNT;
  #if(DEBUGVEC && VVERBOSE)
     printf("VecMaxPointwiseDivide_SeqGPU, chunks: %d segsize: %d\n",chunks,segment);
  #endif
  /* Divide up workload among streams and allocate scratch memory */
  scratchsize = chunks*dimGrid.x*sizeof(double);
  cms[0] = hipMalloc((void**)&devScratch,scratchsize);
  ccs[0] = hipMemsetAsync(devScratch,0,scratchsize,yd->streamid);
  #if(DEBUGVEC)
    ierr = VecCheckCUDAStatus(cms[0],"devScratch alloc in VecMPWD_SeqGPU");CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[0],"devScratch memset in VecMPWD_SeqGPU");CHKERRQ(ierr);
  #endif
  hipDeviceSynchronize();
  for(i=0;i<chunks;i++){/* first kernel */
    hipStreamCreate(&(pwdstream[i]));
    /* Overlapping execution */
    kernMAXPDIV<<<dimGrid,dimBlock,0,pwdstream[i]>>>(xd->devptr,yd->devptr,
                                                     segment,
                                                     x->map->n,
                                                     i,
                                                     devScratch+i*dimGrid.x);
  }/* end for-loop */
  dimBlock.x  = PDIVTCOUNT2;
  scratchsize = chunks*dimGrid.x;
  hipDeviceSynchronize();
  while(scratchsize>1){/* begin next reduction */
    dimGrid.x = ceil((float)scratchsize/(float)PDIVTCOUNT2);
    kernMAX<<<dimGrid,dimBlock>>>(scratchsize,devScratch);
    scratchsize = dimGrid.x;
  }

  /* copy back result */
  ccs[4]=hipMemcpy(max,devScratch,sizeof(double),hipMemcpyDeviceToHost);/* copy back */
  #if(DEBUGVEC)
    ierr = VecCheckCUDAStatus(ccs[4],"on hipMemcpy(devScratch)");CHKERRQ(ierr);
  #endif

  /* Free temp resources */
  cms[3] = hipFree(devScratch);
  #if(DEBUGVEC)
     ierr = VecCheckCUDAStatus(cms[3],"on hipFree(devScratch)");CHKERRQ(ierr);
  #endif
  for(i=0;i<chunks;i++) hipStreamDestroy(pwdstream[i]);

  #if(DEBUGVEC && VVERBOSE)
    printf("max: %e\n",*max);
  #endif
  #if(VTIMER)
    finish=vec_clock();
    elapsed=finish-start;
    cumt+=elapsed;
    if(!ccnt++){
      maxt=mint=avg=elapsed;
    }else{
      maxt=elapsed>maxt?elapsed:maxt;
      mint=elapsed<mint?elapsed:mint;
      avg=cumt/ccnt;
      printf("VecMAXPWD_SeqGPU calls: %d, max: %e, min: %e, average: %e\n",
               ccnt,maxt,mint,avg);
    }
  #endif
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "kernMAX"
__global__ void  kernMAX(int n,double* scratch){
  unsigned int tid = blockDim.x*blockIdx.x+threadIdx.x;
  __shared__ double chunk[PDIVTCOUNT2];
  double mymax;
  mymax=(tid<n)?scratch[tid]:0.;
  if(threadIdx.x>127)chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<128)mymax=fmax(mymax,chunk[threadIdx.x+128]);
  else return;
  if(threadIdx.x>63)chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<64)mymax=fmax(mymax,chunk[threadIdx.x+64]);
  else return;
  chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<32)warpMaxReduce(chunk,threadIdx.x);
  else return;
  if(threadIdx.x==0){
    scratch[blockIdx.x]=chunk[0];
  }else return;
}

#undef __FUNCT__
#define __FUNCT__ "kernMAXPDIV"
__global__ void  kernMAXPDIV(double* devX,double* devY, int segmentsize,
                             int n,int offset,double* scratch){
 /* w <- max(abs(x./y)) */
  __shared__ double chunk[PDIVTCOUNT];
  double mymax;
  unsigned int item = segmentsize*offset+blockDim.x*blockIdx.x+threadIdx.x;
  if(item<n){
    mymax=devY[item];
    if(mymax!=0.)mymax=fabs(devX[item]/mymax);//reusing register
    else mymax=fabs(devX[item]);
  }else{
    mymax=0.;
  }
  if(threadIdx.x>127)chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<128)mymax=fmax(mymax,chunk[threadIdx.x+128]);
  else return;
  if(threadIdx.x>63)chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<64)mymax=fmax(mymax,chunk[threadIdx.x+64]);
  else return;
  chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<32) warpReduce(chunk,threadIdx.x);
  else return;
  if(threadIdx.x==0){
    scratch[blockIdx.x]=chunk[0];
  }else return;
}




/* ---------------------------------------------------------
// VecPointwiseDivide_SeqGPU
// Function which calculates the elementwise division of vector
// X/Y and stores the results in a third array. If one element 
// is zero then X is just returned. Currently only the manual
// tuned version is implemented.
// Checks for size mismatch of arrays and synchs to device if
// needed.
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecPointwiseDivide_SeqGPU"
PetscErrorCode VecPointwiseDivide_SeqGPU(Vec w,Vec x,Vec y){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid, dimBlock;
  #if(DEBUGVEC && VVERBOSE)
     printf("Call to VecPointwiseDivide_SeqGPU\n");
  #endif
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  dimGrid.x=ceil((float)y->map->n/(float)PDIVTCOUNT);
  dimBlock.x=PDIVTCOUNT;
  hipDeviceSynchronize();
  kernPDIV<<<dimGrid,dimBlock,2*dimBlock.x*sizeof(double)>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
  #if(DEBUGVEC)
    ierr = VecCheckCUDAError("kernel call to kernPDIV");CHKERRQ(ierr);
  #endif
  wd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}
extern __shared__ double sharedPDIV[];
#undef __FUNCT__
#define __FUNCT__ "kernPDIV"
__global__ void  kernPDIV(double* devY,double* devX, int* vlen, double* devW){
  /* w <- x./y */
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  double* chunkX = sharedPDIV;
  double* chunkY = sharedPDIV + blockDim.x;
  double work;
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    if(chunkX[threadIdx.x]*chunkY[threadIdx.x]!=0){
      work=chunkX[threadIdx.x]/chunkY[threadIdx.x];
    }else{
      work=0;
    }
    devW[tid]=work;
  }
}

/*--------------------------- end pointwise ---------------------------- */

/*--------------------------- norm functions --------------------------- */

/* ---------------------------------------------------------
// VecDotNorm2_SeqGPU
// Simple wrapper function for two calls
// Never seen this function called
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecDotNorm2_SeqGPU"
PetscErrorCode VecDotNorm2_SeqGPU(Vec s, Vec t, PetscScalar *dp, PetscScalar *nm){
  PetscErrorCode ierr;
  PetscFunctionBegin;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecDotNorm2_SeqGPU\n");
  #endif
  ierr = VecDot(s,t,dp); CHKERRQ(ierr);
  ierr = VecNorm(t,NORM_2,nm); CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

/* ---------------------------------------------------------
// VecNorm_SeqGPU
// Function which computes the norm of a vector
// Currently implements two norm types: infinity norm and norm2
// Norm2 is implemented as a manually tuned kernel as well a
// Orio tuned kernels for three vector size ranges
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecNorm_SeqGPU"
PetscErrorCode VecNorm_SeqGPU(Vec x,NormType type,PetscReal* z){
  /* NormType: NORM_1=0,NORM_2=1,NORM_FROBENIUS=2,NORM_INFINITY=3,NORM_1_AND_2=4 */
  /* dealing with NORM_2 and NORM_INFINITY for now... */
  PetscFunctionBegin;
  #if(VTIMER)
    double start,finish,elapsed;
    static double mint,maxt=0.,cumt=0.,avg=0.;
    static int ccnt=0;
    start = vec_clock();
  #endif
  PetscErrorCode ierr;
  double *devScratch,zhost;
  PetscInt i,chunks=0,segment,scratchsize;
  hipStream_t* nrmstream;
  dim3 dimGrid, dimBlock;
  /* defining per-stream work load */
  float threadscale = NRMMPLIER*CHUNKWIDTH;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  if(xd->syncState==VEC_CPU){
    #if(DEBUGVEC && VVERBOSE)
       printf("xd state VEC_CPU: copying to device.\n");
    #endif
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }

  if(type==NORM_INFINITY){
    printf("Infinity NORM.\n");
    /* figure out how many chunks will be needed */
    chunks = ceil( ((float)x->map->n) / threadscale);
    nrmstream = (hipStream_t*)malloc(chunks*sizeof(hipStream_t));
    /* make sure the segment size for each chunk is correct */
    if(chunks>1) segment = (int) threadscale;
    else segment = x->map->n;
    dimGrid.x=ceil(((float)segment)/(float)THRNRMCNT);
    dimBlock.x  = THRNRMCNT;
    #if(DEBUGVEC && VVERBOSE)
      printf("Call to VecNorm_SeqGPU chunks: %d segsize: %d\n",chunks,segment);
    #endif
    /* allocate gridwide scratch array */
    scratchsize = chunks*dimGrid.x;
    cms[0] = hipMalloc((void**)&devScratch,scratchsize*sizeof(double));
    #if(DEBUGVEC)
      #if(VVERBOSE)
         printf("NORM2: chunks: %d, seg: %d, blks: %d\n",chunks,segment,dimGrid.x);
      #endif
      ierr = VecCheckCUDAStatus(cms[0],"devScratch alloc in VecNorm_SeqGPU"); CHKERRQ(ierr);
    //ierr = VecCheckCUDAStatus(ccs[0],"devScratch memset in VecNorm_SeqGPU");CHKERRQ(ierr);
    #endif
    hipDeviceSynchronize();
    for(i=0;i<chunks;i++){/* streaming async kernel calls */
      hipStreamCreate(&(nrmstream[i]));
      /* Overlapping execution */
      kernInfNorm<<<dimGrid,dimBlock,0,nrmstream[i]>>>(xd->devptr,segment,x->map->n,i,
                                                       devScratch+i*dimGrid.x);
    }/* end for-loop */
    dimBlock.x  = THRNRMCNT2;
    hipDeviceSynchronize();
    while(scratchsize>1){/* begin next reduction */
      /* printf("Seconds stage reduction.\n"); */
      dimGrid.x = ceil((float)scratchsize/(float)THRNRMCNT2);
      kernRedInfNorm<<<dimGrid,dimBlock>>>(scratchsize,devScratch);
      scratchsize = dimGrid.x;
    }
  }else{/* NORM2 etc... */

#if(VMANNRM)
  /* figure out how many chunks will be needed */
  chunks = ceil( ((float)x->map->n) / threadscale);
  nrmstream = (hipStream_t*)malloc(chunks*sizeof(hipStream_t));
  /* make sure the segment size for each chunk is correct */
  if(chunks>1) segment = (int) threadscale;
  else segment = x->map->n;
  dimGrid.x=ceil(((float)segment)/(float)THRNRMCNT);
  dimBlock.x  = THRNRMCNT;
#if(DEBUGVEC && VVERBOSE)
  printf("Call to VecNorm_SeqGPU chunks: %d segsize: %d\n",chunks,segment);
#endif
  /* allocate gridwide scratch array */
  scratchsize = chunks*dimGrid.x;
  cms[0] = hipMalloc((void**)&devScratch,scratchsize*sizeof(double));
  #if(DEBUGVEC)
    #if(VVERBOSE)
       printf("NORM2: chunks: %d, seg: %d, blks: %d\n",chunks,segment,dimGrid.x);
    #endif
    ierr = VecCheckCUDAStatus(cms[0],"devScratch alloc in VecNorm_SeqGPU"); CHKERRQ(ierr);
    //ierr = VecCheckCUDAStatus(ccs[0],"devScratch memset in VecNorm_SeqGPU");CHKERRQ(ierr);
  #endif
  hipDeviceSynchronize();
    for(i=0;i<chunks;i++){/* streaming async kernel calls */
      hipStreamCreate(&(nrmstream[i]));
      /* Overlapping execution */
      kernNorm2<<<dimGrid,dimBlock,0,nrmstream[i]>>>(xd->devptr,segment,x->map->n,i,
                                                     devScratch+i*dimGrid.x);
    }/* end for-loop */
    dimBlock.x  = THRNRMCNT2;
    hipDeviceSynchronize();
    while(scratchsize>1){/* begin next reduction */
      /* printf("Seconds stage reduction.\n"); */
      dimGrid.x = ceil((float)scratchsize/(float)THRNRMCNT2);
      kernRedNorm<<<dimGrid,dimBlock>>>(scratchsize,devScratch);
      scratchsize = dimGrid.x;
    }

    hipDeviceSynchronize();
    for(i=0;i<chunks;i++) hipStreamDestroy(nrmstream[i]);
    free(nrmstream);
#else

    if(x->map->n>=1e7){
      hipFuncSetCacheConfig(reinterpret_cast<const void*>(orcu_norm2kernel_1e7),hipFuncCachePreferL1);
      int nthreads=512;
      int nstreams=2;
      /*calculate device dimensions*/
      dim3 dimGrid, dimBlock;
      dimBlock.x=nthreads;
      dimGrid.x=112;
      hipMalloc((void**)&devScratch,(dimGrid.x+1)*sizeof(double));
      /*create streams*/
      int istream, soffset, boffset;
      hipStream_t stream[nstreams+1];
      for (istream=0; istream<=nstreams;istream++)hipStreamCreate(&stream[istream]);
      int chunklen=x->map->n/nstreams;
      int chunkrem=x->map->n%nstreams;
   
      /*invoke device kernel*/
      int blks4chunk=dimGrid.x/nstreams;
      if(dimGrid.x%nstreams!=0)blks4chunk++ ;
      int blks4chunks=blks4chunk*nstreams;
      for(istream=0; istream<nstreams; istream++){
        soffset=istream*chunklen;
        boffset=istream*blks4chunk;
        orcu_norm2kernel_1e7<<<blks4chunk,dimBlock,0,stream[istream]>>>
                           (chunklen,xd->devptr+soffset,devScratch+boffset);
      }
      if (chunkrem!=0) {
        soffset=istream*chunklen;
        boffset=istream*blks4chunk;
        orcu_norm2kernel_1e7<<<blks4chunk,dimBlock,0,stream[istream]>>>
                           (chunkrem,xd->devptr+soffset,devScratch+boffset);
        blks4chunks++ ;
      }
      int orcu_blks=blks4chunks;
      int orcu_n;
      while (orcu_blks>1) {
        orcu_n=orcu_blks;
        orcu_blks=(orcu_blks+511)/512;
        orcu_norm2blksum_1e7<<<orcu_blks,512>>>(orcu_n,devScratch);
      }
      for (istream=0; istream<=nstreams; istream++)hipStreamDestroy(stream[istream]);
    }else if(x->map->n>=1e6){
      /*calculate device dimensions*/
      dimBlock.x=228;
      dimGrid.x=56;
      hipMalloc((void**)&devScratch,(dimGrid.x+1)*sizeof(double));
      orcu_norm2kernel_1e6<<<dimGrid,dimBlock>>>(x->map->n,xd->devptr,devScratch);
      int orcu_blks=dimGrid.x;
      int orcu_n;
      while (orcu_blks>1) {
        orcu_n=orcu_blks;
        orcu_blks=(orcu_blks+227)/228;
        orcu_norm2blksum_1e6<<<orcu_blks,228>>>(orcu_n,devScratch);
      }
    }else{
      /*calculate device dimensions*/
      dimBlock.x=128;
      dimGrid.x=112;
      hipMalloc((void**)&devScratch,(dimGrid.x+1)*sizeof(double));
      orcu_norm2kernel_1e5<<<dimGrid,dimBlock>>>(x->map->n,xd->devptr,devScratch);
      int orcu_blks=dimGrid.x;
      int orcu_n;
      while (orcu_blks>1) {
        orcu_n=orcu_blks;
        orcu_blks=(orcu_blks+127)/128;
        orcu_norm2blksum_1e5<<<orcu_blks,128>>>(orcu_n,devScratch);
      }
    }
#endif /* end VMANNRM norm2 */

    ccs[4]=hipMemcpy(&zhost,devScratch,sizeof(double),hipMemcpyDeviceToHost);/* copy back */
    #if(DEBUGVEC)
      ierr = VecCheckCUDAStatus(ccs[4],"on hipMemcpy(devScratch)");CHKERRQ(ierr);
    #endif
    hipDeviceSynchronize();/* make sure everyone is caught up */
    *z = PetscSqrtScalar(zhost);
  }/* end NORMTYPE if */


  /* clean up resources */

  cms[3] = hipFree(devScratch);
  #if(DEBUGVEC)
   #if(VVERBOSE)
        printf("Znorm: %e\n",*z);
   #endif
     ierr = VecCheckCUDAStatus(cms[3],"on hipFree(devScratch)");CHKERRQ(ierr);
  #endif
  #if(VTIMER)
    finish=vec_clock();
    elapsed=finish-start;
    cumt+=elapsed;
    if(!ccnt++){
      maxt=mint=avg=elapsed;
    }else{
      maxt=elapsed>maxt?elapsed:maxt;
      mint=elapsed<mint?elapsed:mint;
      avg=cumt/ccnt;
      if(!(ccnt%(ITSHOW/2))){
        printf("VecNorm_SeqGPU calls: %d, max: %e, min: %e, average: %e\n",
               ccnt,maxt,mint,avg);
      }
    }
  #endif
  PetscFunctionReturn(0);
}

/*-------------- Device kernels for infinite norm ----------------*/
#undef __FUNCT__
#define __FUNCT__ "kernRedInfNorm"
__global__ void kernRedInfNorm(int n,double* scratch){/* reduction kernel */
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunk[THRNRMCNT2];
  double mymax;
  mymax=(tid<n)?scratch[tid]:0.;
  if(threadIdx.x>127)chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<128)mymax=fmax(mymax,chunk[threadIdx.x+128]);
  else return;
  if(threadIdx.x>63)chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<64)mymax=fmax(mymax,chunk[threadIdx.x+64]);
  else return;
  chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<32)warpMaxReduce(chunk,threadIdx.x);
  else return;
  if(threadIdx.x==0){
    scratch[blockIdx.x]=chunk[0];
  }else return;
}

#undef __FUNCT__
#define __FUNCT__ "kernInfNorm"
__global__ void kernInfNorm(double* devX,int segmentsize,
                        int arrsize, int offset, double* scratch){
  __shared__ double chunk[THRNRMCNT];
  unsigned int item = segmentsize*offset+blockDim.x*blockIdx.x+threadIdx.x;
  double mymax=0.;
  mymax=(item<arrsize)?fabs(devX[item]):0.;

  if(threadIdx.x>127)chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<128)mymax=fmax(mymax,chunk[threadIdx.x+128]);
  else return;
  if(threadIdx.x>63)chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<64)mymax=fmax(mymax,chunk[threadIdx.x+64]);
  else return;
  chunk[threadIdx.x]=mymax;
  __syncthreads();
  if(threadIdx.x<32) warpMaxReduce(chunk,threadIdx.x);
  else return;
  if(threadIdx.x==0){
    scratch[blockIdx.x]=chunk[0];
  }else return;
}
/*---------------------------------------------------------*/

/*-------------- Device kernels for norm2 ----------------*/
#undef __FUNCT__
#define __FUNCT__ "kernRedNorm"
__global__ void kernRedNorm(int n,double* scratch){/* reduction kernel */
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunk[THRNRMCNT2];
  double mysum;
  mysum=(tid<n)?scratch[tid]:0.;
  if(threadIdx.x>127)chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<128)mysum+=chunk[threadIdx.x+128];
  else return;
  if(threadIdx.x>63)chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<64)mysum+=chunk[threadIdx.x+64];
  else return;
  chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<32)warpReduce(chunk,threadIdx.x);
  else return;
  if(threadIdx.x==0){
    scratch[blockIdx.x]=chunk[0];
  }else return;
}

#undef __FUNCT__
#define __FUNCT__ "kernNorm2"
__global__ void kernNorm2(double* devX,int segmentsize,
                        int arrsize, int offset, double* scratch){
  __shared__ double chunk[THRNRMCNT];
  unsigned int item = segmentsize*offset+blockDim.x*blockIdx.x+threadIdx.x;
  double mysum=0.;
  mysum=(item<arrsize)?devX[item]:0.;
  mysum*=mysum;
  if(threadIdx.x>127)chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<128)mysum+=chunk[threadIdx.x+128];
  else return;
  if(threadIdx.x>63)chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<64)mysum+=chunk[threadIdx.x+64];
  else return;
  chunk[threadIdx.x]=mysum;
  __syncthreads();
  if(threadIdx.x<32) warpReduce(chunk,threadIdx.x);
  else return;
  if(threadIdx.x==0){
    scratch[blockIdx.x]=chunk[0];
  }else return;
}
/*---------------------------------------------------------*/


/*------------------- Orio Norm2 kernels ------------------*/
__global__ void orcu_norm2kernel_1e5(const int n, double* x, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  double orcu_var10241=0;
  for (int i=tid; i<=n-1; i+=gsize) {
    orcu_var10241=orcu_var10241+x[i]*x[i];
  }
  /*reduce single-thread results within a block*/
  __shared__ double orcu_vec10242[128];
  orcu_vec10242[threadIdx.x]=orcu_var10241;
  __syncthreads();
  if (threadIdx.x<64) orcu_vec10242[threadIdx.x]+=orcu_vec10242[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) orcu_warpReduce64(threadIdx.x,orcu_vec10242);
  __syncthreads();
  if (threadIdx.x==0) reducts[blockIdx.x]=orcu_vec10242[0];
}
__global__ void orcu_norm2blksum_1e5(int orcu_n, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double orcu_vec10242[128];

  if (tid<orcu_n) orcu_vec10242[threadIdx.x]=reducts[tid];
  else  orcu_vec10242[threadIdx.x]=0;
  __syncthreads();
  if (threadIdx.x<64) orcu_vec10242[threadIdx.x]+=orcu_vec10242[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) orcu_warpReduce64(threadIdx.x,orcu_vec10242);
  __syncthreads();
  if (threadIdx.x==0) reducts[blockIdx.x]=orcu_vec10242[0];
}

__global__ void orcu_norm2kernel_1e6(const int n, double* x, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  double orcu_var20485=0;
  for (int i=tid; i<=n-1; i+=gsize) {
    orcu_var20485=orcu_var20485+x[i]*x[i];
  }
  /*reduce single-thread results within a block*/
  __shared__ double orcu_vec20486[288];
  orcu_vec20486[threadIdx.x]=orcu_var20485;
  __syncthreads();
  if (threadIdx.x<128) 
    orcu_vec20486[threadIdx.x]+=orcu_vec20486[threadIdx.x+128];
  __syncthreads();
  if (threadIdx.x<64) 
    orcu_vec20486[threadIdx.x]+=orcu_vec20486[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) 
    orcu_warpReduce64(threadIdx.x,orcu_vec20486);
  if (threadIdx.x>=256&&threadIdx.x<272) 
    orcu_warpReduce32(threadIdx.x,orcu_vec20486);
  __syncthreads();
  if (threadIdx.x==0) 
    reducts[blockIdx.x]=orcu_vec20486[0]+orcu_vec20486[256];
}
__global__ void orcu_norm2blksum_1e6(int orcu_n, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double orcu_vec20486[288];
  if (tid<orcu_n) 
    orcu_vec20486[threadIdx.x]=reducts[tid];
  else 
    orcu_vec20486[threadIdx.x]=0;
  __syncthreads();
  if (threadIdx.x<128) 
    orcu_vec20486[threadIdx.x]+=orcu_vec20486[threadIdx.x+128];
  __syncthreads();
  if (threadIdx.x<64) 
    orcu_vec20486[threadIdx.x]+=orcu_vec20486[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) 
    orcu_warpReduce64(threadIdx.x,orcu_vec20486);
  if (threadIdx.x>=256&&threadIdx.x<272) 
    orcu_warpReduce32(threadIdx.x,orcu_vec20486);
  __syncthreads();
  if (threadIdx.x==0) 
    reducts[blockIdx.x]=orcu_vec20486[0]+orcu_vec20486[256];
}

__global__ void orcu_norm2kernel_1e7(const int n, double* x, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  const int gsize=gridDim.x*blockDim.x;
  __shared__ double shared_x[512];
  double orcu_var30729=0;
  for (int i=tid; i<=n-1; i+=gsize) {
    shared_x[threadIdx.x]=x[i];
    orcu_var30729=orcu_var30729+shared_x[threadIdx.x]*shared_x[threadIdx.x];
  }
  /*reduce single-thread results within a block*/
  __shared__ double orcu_vec30730[512];
  orcu_vec30730[threadIdx.x]=orcu_var30729;
  __syncthreads();
  if (threadIdx.x<256) 
    orcu_vec30730[threadIdx.x]+=orcu_vec30730[threadIdx.x+256];
  __syncthreads();
  if (threadIdx.x<128) 
    orcu_vec30730[threadIdx.x]+=orcu_vec30730[threadIdx.x+128];
  __syncthreads();
  if (threadIdx.x<64) 
    orcu_vec30730[threadIdx.x]+=orcu_vec30730[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) 
    orcu_warpReduce64(threadIdx.x,orcu_vec30730);
  __syncthreads();
  if (threadIdx.x==0) 
    reducts[blockIdx.x]=orcu_vec30730[0];
}
__global__ void orcu_norm2blksum_1e7(int orcu_n, double* reducts) {
  const int tid=blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double orcu_vec30730[512];
  if (tid<orcu_n) 
    orcu_vec30730[threadIdx.x]=reducts[tid];
  else 
    orcu_vec30730[threadIdx.x]=0;
  __syncthreads();
  if (threadIdx.x<256) 
    orcu_vec30730[threadIdx.x]+=orcu_vec30730[threadIdx.x+256];
  __syncthreads();
  if (threadIdx.x<128) 
    orcu_vec30730[threadIdx.x]+=orcu_vec30730[threadIdx.x+128];
  __syncthreads();
  if (threadIdx.x<64) 
    orcu_vec30730[threadIdx.x]+=orcu_vec30730[threadIdx.x+64];
  __syncthreads();
  if (threadIdx.x<32) 
    orcu_warpReduce64(threadIdx.x,orcu_vec30730);
  __syncthreads();
  if (threadIdx.x==0) 
    reducts[blockIdx.x]=orcu_vec30730[0];
}
/*---------------------------------------------------------*/
/* --------------------- end norms ----------------------- */


/* ---------------------------------------------------------
// VecGetArray_SeqGPU
// Grabs the pointer to the cpu memory, if necessary copies
// that array up from the device
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecGetArray_SeqGPU"
PetscErrorCode VecGetArray_SeqGPU(Vec v,PetscScalar **a){
#ifdef PETSC_USE_DEBUG
  PetscInt flg1=0,flg2=0,flg3=0,flg4=0;
#endif
  PetscErrorCode ierr;
  Vec_SeqGPU *vd=(Vec_SeqGPU*)v->data;

  PetscFunctionBegin;

  if(vd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecGetArray_SeqGPU the vector has not been created.");
  }
  #if(DEBUGVEC && VVERBOSE)
     printf("Call to VecGetArray_SeqGPU\n");
  #endif
#ifdef PETSC_USE_DEBUG
  /* PETSc in debug mode uses a macro to VecValidValues
     to test values before trying to use the vector. In order to
     prevent these checks (which all require cudamemcpy), 
     the stack is checked to make sure it's a real need for the values
  */
   MyPetscStackCheckByName("DMDAVecGetArray",flg1);
   MyPetscStackCheckByName("DMGlobalToLocalBegin",flg2);
   MyPetscStackCheckByName("SNESDefaultComputeJacobian",flg3);
   MyPetscStackCheckByName("DMComputeJacobianDefault",flg4);
  if(flg1 || flg2 || flg3 || flg4 ){
    if(vd->syncState==VEC_GPU){
      ierr = VecCopyOverD2H(v,vd->cpuptr); CHKERRQ(ierr);
    }
    vd->syncState = VEC_CPU;
  }
#else
  if(vd->syncState==VEC_GPU){
    ierr = VecCopyOverD2H(v,vd->cpuptr); CHKERRQ(ierr);
  }
  vd->syncState = VEC_CPU;
#endif
  hipDeviceSynchronize();
  *a=vd->cpuptr;
  PetscFunctionReturn(0);
}


/* ---------------------------------------------------------
// VecRestoreArray_SeqGPU
// Returns data back to the vector type and copying back
// the memory to device only if necessary
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecRestoreArray_SeqGPU"
PetscErrorCode VecRestoreArray_SeqGPU(Vec v,PetscScalar **a){
  PetscErrorCode ierr;
  Vec_SeqGPU *vd=(Vec_SeqGPU*)v->data;
#ifdef PETSC_USE_DEBUG
  PetscInt flg1=0,flg2=0,flg3=0;
#endif
  PetscFunctionBegin;

#ifdef PETSC_USE_DEBUG
  /* PETSc in debug mode uses a macro to VecValidValues
     to test values before trying to use the vector. In order to
     prevent these checks (which all require cudamemcpy), 
     the stack is checked to make sure it's a real need for the values
  */
  MyPetscStackCheckByName("VecRestoreArrayRead",flg1);
  MyPetscStackCheckByName("DMDAVecRestoreArray",flg2);
  MyPetscStackCheckByName("DMGlobalToLocalBegin",flg3);
  if(vd->syncState==VEC_CPU||(!flg1||flg2||flg3)){
    if(a){
      ierr = VecCopyOverH2D(v,*a);CHKERRQ(ierr);
      vd->syncState=VEC_GPU;
    }else{
      ierr = VecCopyOverH2D(v,vd->cpuptr);CHKERRQ(ierr);
      vd->syncState=VEC_SYNCHED;
    }
  }
#else
  if(a){
    ierr = VecCopyOverH2D(v,*a);CHKERRQ(ierr);
    vd->syncState=VEC_GPU;
  }else{
    ierr = VecCopyOverH2D(v,vd->cpuptr);CHKERRQ(ierr);
    vd->syncState=VEC_SYNCHED;
  }
#endif
  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCreateSeqGPU"
PetscErrorCode  VecCreateSeqGPU(MPI_Comm comm,PetscInt n,Vec *v){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  ierr = VecCreate(comm,v);CHKERRQ(ierr);
  ierr = VecSetSizes(*v,n,n);CHKERRQ(ierr);
  ierr = VecSetType(*v,VECSEQGPU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCopy_SeqGPU"
PetscErrorCode VecCopy_SeqGPU(Vec s,Vec d){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *sd=(Vec_SeqGPU*)s->data;
  Vec_SeqGPU *dd=(Vec_SeqGPU*)d->data;
  if(d->map->n!=s->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(dd->syncState==VEC_UNALLOC){
     SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Destination vector unalloced.");
  }
  if(sd->syncState==VEC_ALLOC){
      PetscFunctionReturn(0);/* nothing to do */
  }
  if(sd->syncState==VEC_CPU){
    ierr = PetscMemcpy((void*)dd->cpuptr,(void*)sd->cpuptr,s->map->n*sizeof(double));CHKERRQ(ierr);
    dd->syncState = VEC_CPU;
    PetscFunctionReturn(0);
  }
  ierr = VecCopyOverDevice(d,s); CHKERRQ(ierr);
  dd->syncState=VEC_GPU;
  #if(DEBUGVEC && VVERBOSE)
     printf("Call to VecCopy_SeqGPU\n");
  #endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecSwap_SeqGPU"
PetscErrorCode VecSwap_SeqGPU(Vec xin,Vec yin){
  /* PetscErrorCode ierr; */
  PetscFunctionBegin;
  printf("VecSwap_SeqGPU (***EMPTY***)\n");
  if (xin != yin) {
#if defined(PETSC_USE_REAL_SINGLE)
    //////// hipblasSswap(bn,VecCUSPCastToRawPtr(*xarray),one,VecCUSPCastToRawPtr(*yarray),one);
#else
    //////   hipblasDswap(bn,VecCUSPCastToRawPtr(*xarray),one,VecCUSPCastToRawPtr(*yarray),one);
#endif

  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDuplicate_SeqGPU"
PetscErrorCode VecDuplicate_SeqGPU(Vec win,Vec *V){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecDuplicate_SeqGPU\n");
  #endif
  ierr = VecCreate(((PetscObject)win)->comm,V);CHKERRQ(ierr);
  ierr = VecSetType(*V,VECSEQGPU);CHKERRQ(ierr);
  ierr = PetscObjectSetPrecision((PetscObject)*V,((PetscObject)win)->precision);CHKERRQ(ierr);
  ierr = VecSetSizes(*V,win->map->n,win->map->N);CHKERRQ(ierr);
  ierr = PetscLayoutReference(win->map,&(*V)->map);CHKERRQ(ierr);
  ierr = PetscOListDuplicate(((PetscObject)win)->olist,&((PetscObject)(*V))->olist);CHKERRQ(ierr);
  ierr = PetscFListDuplicate(((PetscObject)win)->qlist,&((PetscObject)(*V))->qlist);CHKERRQ(ierr);
  (*V)->stash.ignorenegidx = win->stash.ignorenegidx;
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDuplicateVecs_SeqGPU"
PetscErrorCode VecDuplicateVecs_SeqGPU(Vec vin, PetscInt m, Vec **Vlist){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt i=0;
  #if(DEBUGVEC && VVERBOSE)
     printf("Call to VecDuplicateVecs_SeqGPU\n"); 
  #endif
  PetscValidHeaderSpecific(vin,VEC_CLASSID,1);
  PetscValidPointer(Vlist,3);
  if (m <= 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_OUTOFRANGE,"m must be > 0: m = %D",m);
  ierr = PetscMalloc(m*sizeof(Vec),Vlist);CHKERRQ(ierr);
  for(i=0;i<m;i++){
    ierr = VecDuplicate_SeqGPU(vin,*Vlist+i);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecPlaceArray_SeqGPU"
PetscErrorCode  VecPlaceArray_SeqGPU(Vec x,const PetscScalar* array){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  #if(DEBUGVEC && VVERBOSE)
     printf("Call to VecPlaceArray_SeqGPU\n"); 
  #endif
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecPlaceArray_SeqGPU the vector has not been created.");
  }
  if(xd->unplacedarray){
     SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,
       "VecPlaceArray() was already called on this vector, without a call to VecResetArray()");
  }
  if(xd->syncState==VEC_GPU){/* assuming there is a logical reason for this copy up */
    ierr = VecCopyOverD2H(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  xd->unplacedarray=xd->cpuptr;
  xd->cpuptr=(PetscScalar*)array;
  ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
  xd->syncState=VEC_SYNCHED;
  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecResetArray_SeqGPU"
PetscErrorCode  VecResetArray_SeqGPU(Vec x){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  #if(DEBUGVEC && VVERBOSE)
     printf("Call to VecResetArray_SeqGPU\n"); 
  #endif
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecResetArray_SeqGPU the vector has not been created.");
  }
  if(xd->cpuptr){
    ierr = PetscFree(xd->cpuptr);CHKERRQ(ierr);
  }
  xd->cpuptr=xd->unplacedarray;
  xd->unplacedarray=PETSC_NULL;
  ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
  xd->syncState=VEC_SYNCHED;
  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecReplaceArray_SeqGPU"
PetscErrorCode  VecReplaceArray_SeqGPU(Vec x,const PetscScalar* array){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecReplaceArray_SeqGPU\n"); 
  #endif
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecResetArray_SeqGPU the vector has not been created.");
  }
  if(xd->cpuptr){
    ierr = PetscFree(xd->cpuptr);CHKERRQ(ierr);
  }
  xd->cpuptr=(PetscScalar*)array;
  ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
  xd->syncState=VEC_SYNCHED;
  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "PinnedMalloc"
PetscErrorCode  PinnedMalloc(PetscScalar** x,PetscInt n){
  PetscErrorCode ierr;
  PetscFunctionBegin;
  ierr = PetscInfo1(0,"Allocating %d bytes on GPU\n",n); CHKERRQ(ierr);
  ierr=VecCheckCUDAStatus(cms[0],"before PinnedMalloc");CHKERRQ(ierr);
  //ierr = PetscPrintf(PETSC_COMM_SELF,"Allocating %d bytes on GPU\n",n); CHKERRQ(ierr);
  #if(DEBUGVEC && VVERBOSE)
     printf("Call to PinnedMalloc\n"); 
  #endif
  cms[0]=hipHostAlloc((void**)x,n,0);

  ierr=VecCheckCUDAStatus(cms[0],"in PinnedMalloc");CHKERRQ(ierr);

  //SETERRQ1(PETSC_COMM_SELF,0,"Error Allocating Memory -- %d bytes requested",n);

  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "PinnedFree"
PetscErrorCode  PinnedFree(PetscScalar* x){
  PetscFunctionBegin;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to PinnedFree\n"); 
  #endif
  cms[0]=hipHostFree(x);
  #if(DEBUGVEC)
    PetscErrorCode ierr;
    ierr=VecCheckCUDAStatus(cms[0],"in PinnedFree");CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCreate_SeqGPU"
PetscErrorCode  VecCreate_SeqGPU(Vec V){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscMPIInt    size;
  Vec_SeqGPU* seqgpu=PETSC_NULL;
  ierr = PetscMalloc(sizeof(Vec_SeqGPU),&seqgpu);
  V->data=(void*)seqgpu;
  ierr = MPI_Comm_size(((PetscObject)V)->comm,&size);CHKERRQ(ierr);
  if  (size > 1) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Cannot create VECSEQGPU on more than one process");

  if (V->map->bs == -1) V->map->bs = 1;
  ierr = PetscLayoutSetUp(V->map);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)V,VECSEQGPU);CHKERRQ(ierr);

  V->ops->dot             = VecDot_SeqGPU;
  V->ops->norm            = VecNorm_SeqGPU;
  V->ops->tdot            = VecTDot_SeqGPU;
  V->ops->scale           = VecScale_SeqGPU;
  V->ops->copy            = VecCopy_SeqGPU;
  V->ops->set             = VecSet_SeqGPU;
  V->ops->setvalues       = VecSetValues_SeqGPU;
  V->ops->swap            = VecSwap_SeqGPU;
  V->ops->axpy            = VecAXPY_SeqGPU;
  V->ops->axpby           = VecAXPBY_SeqGPU;
  V->ops->axpbypcz        = VecAXPBYPCZ_SeqGPU;
  V->ops->pointwisemult   = VecPointwiseMult_SeqGPU;
  V->ops->pointwisedivide = VecPointwiseDivide_SeqGPU;
  V->ops->maxpointwisedivide = VecMaxPointwiseDivide_SeqGPU;
  V->ops->setrandom       = VecSetRandom_SeqGPU;
  V->ops->dot_local       = VecDot_SeqGPU;
  V->ops->tdot_local      = VecTDot_SeqGPU;
  V->ops->norm_local      = VecNorm_SeqGPU;
  V->ops->maxpy           = VecMAXPY_SeqGPU;
  V->ops->mdot            = VecMDot_SeqGPU;
  /* V->ops->aypx            = VecAYPX_SeqGPU; */
  V->ops->waxpy           = VecWAXPY_SeqGPU;
  V->ops->dotnorm2        = VecDotNorm2_SeqGPU;
  V->ops->placearray      = VecPlaceArray_SeqGPU;
  V->ops->replacearray    = VecReplaceArray_SeqGPU;
  V->ops->resetarray      = VecResetArray_SeqGPU;
  V->ops->destroy         = VecDestroy_SeqGPU;
  V->ops->destroyvecs     = VecDestroyVecs_SeqGPU;
  V->ops->duplicate       = VecDuplicate_SeqGPU;
  V->ops->duplicatevecs   = VecDuplicateVecs_SeqGPU;
  V->ops->getarray        = VecGetArray_SeqGPU;
  V->ops->restorearray    = VecRestoreArray_SeqGPU;
  V->ops->getlocalsize    = VecGetLocalSize_SeqGPU;
  V->ops->getsize         = VecGetSize_SeqGPU;
  V->ops->view            = VecView_SeqGPU;
  V->petscnative=PETSC_FALSE;
  seqgpu->syncState      = VEC_UNALLOC;
  seqgpu->unplacedarray=PETSC_NULL;
  seqgpu->array_allocated=PETSC_NULL;
  seqgpu->array=PETSC_NULL;
  /* create an associated stream */
  cms[0] = hipStreamCreate(&(seqgpu->streamid));
  /* allocate the variable for vector size */
  cms[1]=hipMalloc((void**)&(seqgpu->length),sizeof(int));
  /* send vec length size to device */
  ccs[0]=hipMemcpyAsync((void*)seqgpu->length,
               (void*)&(V->map->n),sizeof(int),hipMemcpyHostToDevice,seqgpu->streamid);
  /* allocate the vector on device */
  cms[2]=hipMalloc((void**)&(seqgpu->devptr),V->map->n*sizeof(double));
  ccs[1]=hipMemsetAsync((void*)seqgpu->devptr,0,V->map->n*sizeof(double),seqgpu->streamid);
  /* allocate the variable for vector offsets */
  cms[3]=hipMalloc((void**)&(seqgpu->offset),sizeof(int));
  /* allocate the variable for vector segment length */
  cms[4]=hipMalloc((void**)&(seqgpu->segment),sizeof(int));
  /* allocate the variable for vector single value result */
  cms[5]=hipMalloc((void**)&(seqgpu->zval),sizeof(double));
  cms[6]=hipMalloc((void**)&(seqgpu->scalar),sizeof(double));
  /* using pinned memory (could be a resource hog with very large arrays) */
  ierr = PinnedMalloc(&(seqgpu->cpuptr),V->map->n*sizeof(double));CHKERRQ(ierr);

  /* ierr = PetscMalloc(V->map->n*sizeof(PetscScalar),&(seqgpu->cpuptr)); */
  ierr = PetscMemzero(seqgpu->cpuptr,V->map->n*sizeof(double));CHKERRQ(ierr);
  seqgpu->syncState=VEC_ALLOC;
  /* printf("VmapN: %d\n",V->map->n);*/
  #if(DEBUGVEC)
    #if(VVERBOSE)
       printf("Call to VecCreate_SeqGPU\n");
    #endif
    ierr = VecCheckCUDAStatus(cms[0],"on hipStreamCreate VecCreate_SeqGPU");  CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[1],"Alloc devlength in VecCreate_SeqGPU");   CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[0],"Copy H2D devlength in VecCreate_SeqGPU");CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[2],"Alloc of devptr in VecCreate_SeqGPU");   CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[1],"on device cudaMemSet VecCreate_SeqGPU"); CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[3],"Alloc devoffset in VecCreate_SeqGPU");   CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[4],"Alloc dev segment in VecCreate_SeqGPU"); CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[5],"Alloc dev zval in VecCreate_SeqGPU");    CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[6],"Alloc dev scalar in VecCreate_SeqGPU");    CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "VecDestroy_SeqGPU"
PetscErrorCode VecDestroy_SeqGPU(Vec v){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
#if(DEBUGVEC && VVERBOSE)
     printf("Call to VecDestroyArray_SeqGPU\n"); 
  #endif
  PetscValidHeaderSpecific(v,VEC_CLASSID,1);
  if(vd && vd->syncState != VEC_UNALLOC){
      cms[0]=hipFree(vd->devptr);  vd->devptr=PETSC_NULL;
      cms[1]=hipFree(vd->length);  vd->length=PETSC_NULL;
      cms[2]=hipFree(vd->segment); vd->segment=PETSC_NULL;
      cms[3]=hipFree(vd->zval);    vd->zval=PETSC_NULL;
      cms[4]=hipFree(vd->scalar);  vd->scalar=PETSC_NULL;
      cms[5] = hipStreamDestroy(vd->streamid);
      ierr = PinnedFree(vd->cpuptr); CHKERRQ(ierr);
      /* ierr = PetscFree(vd->cpuptr);CHKERRQ(ierr); */
      #if(DEBUGVEC)
        ierr=VecCheckCUDAStatus(cms[0],"destroying devptr in VecDestroy_SeqGPU"); CHKERRQ(ierr);
        ierr=VecCheckCUDAStatus(cms[1],"destroying length in VecDestroy_SeqGPU"); CHKERRQ(ierr);
        ierr=VecCheckCUDAStatus(cms[2],"destroying segment in VecDestroy_SeqGPU");CHKERRQ(ierr);
        ierr=VecCheckCUDAStatus(cms[3],"destroying zval in VecDestroy_SeqGPU");   CHKERRQ(ierr);
        ierr=VecCheckCUDAStatus(cms[4],"destroying scalar in VecDestroy_SeqGPU"); CHKERRQ(ierr);
        ierr=VecCheckCUDAStatus(cms[5],"destroying stream in VecDestroy_SeqGPU"); CHKERRQ(ierr);
      #endif
      vd->syncState = VEC_UNALLOC;
  }
  ierr = PetscObjectDepublish(v);CHKERRQ(ierr);
#if defined(PETSC_USE_LOG)
  PetscLogObjectState((PetscObject)v,"Length=%D",v->map->n);
#endif
  ierr = PetscFree(v->data);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDestroyVecs_SeqGPU"
PetscErrorCode  VecDestroyVecs_SeqGPU(PetscInt m,Vec *vv){
  PetscFunctionBegin;
  #if(DEBUGVEC && VVERBOSE)
     printf("Call to VecDestroyVecs_SeqGPU\n");
  #endif
  PetscErrorCode ierr;
  PetscInt i;
   /* destroy the internal part */
  for(i=0;i<m;i++){
    ierr = VecDestroy(&vv[i]);CHKERRQ(ierr);
  }
  ierr = PetscFree(vv); CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecView_SeqGPU_ASCII"
PetscErrorCode VecView_SeqGPU_ASCII(Vec xin,PetscViewer viewer){
  PetscFunctionBegin;
  printf("VecView_Seq_ASCII() (***EMPTY***)\n");
  PetscFunctionReturn(0);
}


EXTERN_C_END
