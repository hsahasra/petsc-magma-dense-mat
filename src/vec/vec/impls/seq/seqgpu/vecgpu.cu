#include "hip/hip_runtime.h"
#include <petscconf.h>
#include <petscsys.h>
#include <petscerror.h>
PETSC_CUDA_EXTERN_C_BEGIN
#include <string.h>
#include <omp.h>
#include <stdlib.h>
#include <float.h>
#include <petsc-private/vecimpl.h>          /*I "petscvec.h" I*/
#include <../src/vec/vec/impls/dvecimpl.h>
#include <../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h>
PETSC_CUDA_EXTERN_C_END



EXTERN_C_BEGIN
__constant__ int     integerSymbol;
__constant__ int2    integer2Symbol;
__constant__ int3    integer3Symbol;
__constant__ int     devN;/* vector length */
__constant__ double  dblScalarValue;/* utility var */
__constant__ double2 dblScalar2Value;/* utility var */
__constant__ float   fltScalarValue;/* utility var */
__constant__ float2  fltScalar2Value;/* utility var */

static hipError_t ccs[16];
static hipError_t cms[16];

/* Valid pointer check function (probably doesn't work) */
PetscBool valid(void *p){
  extern char _etext;
  if((p != PETSC_NULL) && ((char*) p > &_etext)){
    return PETSC_TRUE;
  }else{
    return PETSC_FALSE;
  }
}

/* ---------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecCheckCUDAError"
PetscErrorCode VecCheckCUDAError(const char *msg){

  PetscFunctionBegin;
  hipError_t err = hipGetLastError();
  if( hipSuccess != err){
    fprintf(stderr, "Cuda kernel error: %s: %s.\n", msg,hipGetErrorString(err));
    fflush(NULL);
    PetscFunctionReturn(PETSC_ERR_LIB);
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCheckCUDAStatus"
PetscErrorCode VecCheckCUDAStatus(hipError_t cs,const char *msg){
  PetscFunctionBegin;
    if(cs!=hipSuccess){
      fprintf(stderr, "Cuda error: %s: %s.\n",msg,hipGetErrorString(cs));
      fflush(NULL);
      PetscFunctionReturn(PETSC_ERR_LIB);
    }
  PetscFunctionReturn(0);
}

/* -------------------- end error checkers ------------------- */




/* ****************************************************************************
 * This is a shared memory implementation that keeps the full 625 words of state
 * in shared memory. Faster for heavy random work where you can afford 
 *  the shared memory. */
/* Init by single seed - single threaded as only used once */
__device__ void mt19937si(uint seed){
    int	i;
    if(threadIdx.x == 0){
	mtNexts = 0;
	s_seeds[0] = seed;
	for(i = 1;i < NNN;i++){
	   seed = (INIT_MULT * (seed^(seed >> 30))+i);
	   s_seeds[i] = seed;
	}
    }
    __syncthreads();/* Ensure mtNexts set */
    return;
}

/* Init by array - single threaded as only used once */
__device__ void mt19937sai(uint* seeds,uint length){
    int i,j,k;
    mt19937si(ARRAY_SEED);
    if(threadIdx.x==0){
     i=1; j=0;
     for(k = NNN>length?NNN:length;k!=0;k--){
        s_seeds[i] = (s_seeds[i]^((s_seeds[i-1]^(s_seeds[i-1] >> 30))*1664525)) + seeds[j] + j;
	if(++i >= NNN){
          s_seeds[0] = s_seeds[NNN-1];
	  i = 1;
        }
        if(++j>=length)j = 0;
     }
     for(k = NNN-1; k!=0;k--){
       s_seeds[i] = (s_seeds[i] ^ ((s_seeds[i-1]^(s_seeds[i-1]>>30))*1566083941))-i;
       if(++i >= NNN){
         s_seeds[0] = s_seeds[NNN-1];
	 i=1;
       }
     }
     s_seeds[0] = 0x80000000;/* MSB is 1; assuring non-zero initial array */ 
    }
    __syncthreads();				/* Needed for mt19937w() */
    return;
}

/* Return next MT random by increasing thread ID for 1-227 threads. */
__device__ uint mt19937s(void){
    int		kk;
    uint	x;
    uint	y;
    int		tid = threadIdx.x;

    kk = (mtNexts + tid) % NNN;
    __syncthreads();				/* Finished with mtNexts */

    if (tid == blockDim.x - 1)mtNexts = kk + 1;			/* Will get modded on next call */
    x = s_seeds[kk] & UPPER_MASK;
    if(kk < NNN - MMM){
      x |= (s_seeds[kk+1]&LOWER_MASK);
      y = s_seeds[kk+MMM];
    }else if(kk < NNN-1){
      x |= (s_seeds[kk+1]&LOWER_MASK);
      y = s_seeds[kk + (MMM-NNN)];
    }else{					/* kk == N - 1 */
      x |= (s_seeds[0]&LOWER_MASK);
      y = s_seeds[MMM - 1];
    }
    y ^= x >> 1;
    if (x & 1)y ^= MATRIX_A;
    __syncthreads();				/* All done before we update */

    s_seeds[kk] = y;
    y ^= (y >> 11);				/* Tempering */
    y ^= (y <<  7) & TEMPER1;
    y ^= (y << 15) & TEMPER2;
    y ^= (y >> 18);
    return y;
}

/* General shared memory version for any number of threads.
 * Note only up to 227 threads are run at any one time,
 * the rest loop and block till all are done. */
__device__ uint mt19937sl(void){
  int jj,kk,tid;
  uint x,y;
  tid = threadIdx.x;
  kk = (mtNexts + tid) % NNN;
  __syncthreads();				/* Finished with mtNexts */

  if(tid == blockDim.x - 1)mtNexts = kk + 1;	/* Will get modded on next call */
  jj = 0;
  do{
    if(jj <= tid && tid < jj + NNN - MMM){
      x = s_seeds[kk] & UPPER_MASK;
      if(kk < NNN - MMM){
         x |= (s_seeds[kk+1]&LOWER_MASK);
	 y = s_seeds[kk + MMM];
      }else if (kk < NNN-1){
         x |= (s_seeds[kk + 1]&LOWER_MASK);
	 y = s_seeds[kk + (MMM-NNN)];
      }else{				/* kk == N - 1 */
         x |= (s_seeds[0]&LOWER_MASK);
         y = s_seeds[MMM-1];
      }

      y ^= x >> 1;
      if(x & 1) y ^= MATRIX_A;
    }
    __syncthreads();			/* All done before we update */
    if(jj <= tid && tid < jj+NNN-MMM) s_seeds[kk] = y;
    __syncthreads();

  }while ((jj += NNN-MMM) < blockDim.x);
  y ^= (y >> 11);				/* Tempering */
  y ^= (y <<  7) & TEMPER1;
  y ^= (y << 15) & TEMPER2;
  y ^= (y >> 18);
  return y;
}


#undef __FUNCT__
#define __FUNCT__ "kernRandS"
__global__ void kernRandS(uint* seeds){
  mt19937sai(seeds,gridDim.x);
}

#undef __FUNCT__
#define __FUNCT__ "kernRand"
__global__ void kernRand(double *x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  uint rval;
  if(tid<*n){
    rval = mt19937sl();
    x[tid] = ((double)rval/(double)UINT_MAX);
    /* printf("RAND value[%d]: %0.13f, rval: %u UINT_MAX: %u\n",
       tid,x[tid],rval,UINT_MAX); */
  }
}

#undef __FUNCT__
#define __FUNCT__ "VecSetRandom_SeqGPU"
PetscErrorCode VecSetRandom_SeqGPU(Vec x,PetscRandom r){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt i;
  uint *seeds=PETSC_NULL,*devseeds=PETSC_NULL;
  PetscScalar rval;
  dim3 dimBlock,dimGrid;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecSetRandom_SeqGPU\n");
  #endif
  if(xd->syncState==VEC_ALLOC || xd->syncState==VEC_CPU){
    for(i=0; i<x->map->n; i++){
       ierr = PetscRandomGetValue(r,&xd->cpuptr[i]);CHKERRQ(ierr);
    }
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }else if(xd->syncState==VEC_SYNCHED || xd->syncState==VEC_GPU){
    dimGrid.x=ceil((float)x->map->n/(float)TCOUNT);
    dimBlock.x=TCOUNT;
    while(dimGrid.x>MAXBLOCKS){
      dimGrid.x/=2;
      dimBlock.x*=2;
    }
    ierr = PetscMalloc(dimGrid.x*sizeof(PetscInt),&seeds);CHKERRQ(ierr);
    for(i=0; i<dimGrid.x; i++){
       ierr = PetscRandomGetValue(r,&rval);CHKERRQ(ierr);
       seeds[i]=(uint)(UINT_MAX*rval);
    }
    cms[0] = hipMalloc((void**)&devseeds,dimGrid.x*sizeof(uint));
    ccs[0]=hipMemcpy(devseeds,seeds,dimGrid.x*sizeof(uint),hipMemcpyHostToDevice);
    #if(DEBUGVEC)
      ierr = VecCheckCUDAStatus(cms[0],"error in hipMalloc");CHKERRQ(ierr);
      ierr = VecCheckCUDAStatus(ccs[0],"on copy H2D in VecSetRandom_SeqGPU");CHKERRQ(ierr);
    #endif

    kernRandS<<<dimGrid,dimBlock>>>(devseeds);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernRandS launch");CHKERRQ(ierr);
    #endif
    kernRand<<<dimGrid,dimBlock>>>(xd->devptr,xd->length);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernRand launch");CHKERRQ(ierr);
    #endif
    ierr = PetscFree(seeds);CHKERRQ(ierr);
    hipDeviceSynchronize();
    cms[1] = hipFree(devseeds);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(cms[1],"in hipFree()");CHKERRQ(ierr);
    #endif
  }
  xd->syncState = VEC_GPU;
  PetscFunctionReturn(0);
}

/*------------------------end random generator ------------------------*/



/*------------------------------ compare ------------------------------*/

#undef __FUNCT__
#define __FUNCT__ "VecCompare_SeqGPU"
PetscErrorCode VecCompare_SeqGPU(Vec x, Vec y, PetscBool *same, PetscInt offset, PetscInt blocksize){
  PetscFunctionBegin;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  Vec_SeqGPU* yd = (Vec_SeqGPU*)y->data;
  if(xd->syncState!=yd->syncState||xd->syncState==VEC_ALLOC||yd->syncState==VEC_ALLOC){
    *same=PETSC_FALSE;
    PetscFunctionReturn(0);
  }
  PetscErrorCode ierr;
  dim3 dimGrid, dimBlock;
  if(blocksize && !offset){
    dimGrid.x=ceil((float)blocksize/(float)TCOUNT);
  } else {
    dimGrid.x=ceil((float)x->map->n/(float)TCOUNT);
  }
  dimBlock.x=TCOUNT;
  while(dimGrid.x>MAXBLOCKS){
      dimGrid.x/=2;
      dimBlock.x*=2;
  }
  hipError_t cudastatus;
  int *devsame=PETSC_NULL;
  int cpusame=0;
  int2 offset_bsize;
  offset_bsize.x = offset;
  offset_bsize.y = blocksize;
  if(xd->syncState==VEC_CPU && yd->syncState==VEC_CPU){
    ierr = PetscMemcmp((void*)&xd->cpuptr[offset],(void*)&yd->cpuptr[offset],blocksize,same);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }

  cudastatus = hipMalloc((void**)&devsame,sizeof(int));
  ierr = VecCheckCUDAStatus(cudastatus,"error in device malloc");CHKERRQ(ierr);

  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("integer2Symbol"),(void*)&offset_bsize,sizeof(int2),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);

  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("devN"),(void*)&x->map->n,sizeof(int),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);

  kernCompare<<<dimGrid,dimBlock,2*dimBlock.x*sizeof(double)>>>(xd->devptr,yd->devptr,xd->length,yd->length,devsame);
  ierr = VecCheckCUDAError("kernCompare launch");CHKERRQ(ierr);

  cudastatus=hipMemcpy(&cpusame,devsame,sizeof(int),hipMemcpyDeviceToHost);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H in VecCompare_SeqGPU");CHKERRQ(ierr);

  if(cpusame==1)*same=PETSC_TRUE;
  else *same=PETSC_FALSE;
  cudastatus = hipFree(devsame);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}

extern __shared__ double sharedCompare[];
#undef __FUNCT__
#define __FUNCT__ "kernCompare"
__global__ void kernCompare(double* devX, double* devY, int* lx, int* ly, int* devsame){
  __shared__ unsigned char blockflag;
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int2 localOBS = integer2Symbol;
  int localn = localOBS.x+localOBS.y;
  int index = tid+localOBS.x;
  double value=0;
  double* chunkX = sharedCompare;
  double* chunkY = sharedCompare + blockDim.x;

  if(threadIdx.x==0)blockflag=0;
  __syncthreads();
  if(index<localn){
    /* read in values to shared */
    chunkX[threadIdx.x]=devX[index];
    chunkY[threadIdx.x]=devY[index];
    value = fabs(chunkX[threadIdx.x]-chunkY[threadIdx.x]);
    if(value>1e-16){
      #if(DEBUGVEC && VERBOSE)
      printf("In kernCompare found an element mismatch: %e\n",value);
      #endif
      blockflag=1;
    }
    if(*lx!=*ly){
      #if(DEBUGVEC && VERBOSE)
      printf("In kernCompare found length mismatch: lx: %d vs ly: %d\n",*lx,*ly);
      #endif
      blockflag=1;
    }
  }
  __syncthreads();
  if(threadIdx.x==0){
    if(blockflag)*devsame=0;
    else *devsame=1;
  }
  return;
}

/*-------------------------- end compare ----------------------------*/

/*----------------------- Vec info functions ------------------------*/

#undef __FUNCT__
#define __FUNCT__ "VecView_SeqGPU"
PetscErrorCode VecView_SeqGPU(Vec x,PetscViewer viewer){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  if(xd->syncState==VEC_GPU){
    ierr = VecCopyOverD2H(x,xd->cpuptr); CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();
  int i;
  for(i=0;i<x->map->n;i++){
    if(xd->cpuptr[i]!=0)printf("cpu[%d]: %e\n",i,xd->cpuptr[i]);
  }
  /* ierr= PetscViewerSetType(viewer,PETSCVIEWERASCII);CHKERRQ(ierr);*/
  /* ierr =VecView_Seq_ASCII(x,viewer);CHKERRQ(ierr); */
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecGetLocalSize_SeqGPU"
PetscErrorCode VecGetLocalSize_SeqGPU(Vec x, PetscInt *localsize){
  PetscFunctionBegin;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecGetLocalSize_SeqGPU\n"); 
  #endif
  PetscValidHeaderSpecific(x,VEC_CLASSID,1);
  PetscValidIntPointer(localsize,2);
  PetscValidType(x,1);
  *localsize=x->map->n;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecGetSize_SeqGPU"
PetscErrorCode VecGetSize_SeqGPU(Vec x, PetscInt *globalsize){
  PetscFunctionBegin;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecGetSize_SeqGPU\n");
  #endif
  PetscValidHeaderSpecific(x,VEC_CLASSID,1);
  PetscValidIntPointer(globalsize,2);
  PetscValidType(x,1);
  *globalsize=x->map->N;
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCheck_SeqGPU"
PetscErrorCode VecCheck_SeqGPU(Vec x){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  dim3 dimgrid(ceil((float)x->map->n/((float)TCOUNT)),1,1);
  dim3 dimblocks(TCOUNT,1,1);
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  printf("******************************************\n");
  kernCheck<<<dimgrid,dimblocks>>>(xd->devptr,xd->length);
  ierr = VecCheckCUDAError("Call to kernCheck. "); CHKERRQ(ierr);
  hipDeviceSynchronize();
  printf("******************************************\n");
  fflush(NULL);
  PetscFunctionReturn(0);
}
#undef __FUNCT__
#define __FUNCT__ "kernCheck"
__global__ void kernCheck(double* x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if(tid<*n){
    #if(DEBUGVEC && VERBOSE)
    printf("kernCheck: x[%d]: %e, length: %d\n",tid,x[tid],*n);
    #endif
  }
}

/*------------------------------ end info -------------------------------*/


/*---------------------------- copy functions ---------------------------*/
#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockDevice"
PetscErrorCode VecCopyBlockDevice(Vec d, Vec s, PetscInt doffset, PetscInt soffset, PetscInt blocksize){
  PetscFunctionBegin;
  printf("Call to VecCopyBlockDevice (**** EMPTY ****)\n");
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecCopyOverDevice"
PetscErrorCode VecCopyOverDevice(Vec d,Vec s){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* dd = (Vec_SeqGPU*)d->data;
  Vec_SeqGPU* sd = (Vec_SeqGPU*)s->data;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecCopyOverDevice\n");
  #endif
  dim3 dimGrid;
  dim3 dimBlock;

  if(s->map->n!=d->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(sd->syncState==VEC_CPU){/* synch y */
    ierr = VecCopyOverH2D(s,sd->cpuptr);CHKERRQ(ierr);
    sd->syncState=VEC_SYNCHED;
    hipStreamSynchronize(sd->stream);
  }
  ccs[0]=hipMemcpyAsync(dd->devptr,sd->devptr,
               s->map->n*sizeof(PetscScalar),hipMemcpyDeviceToDevice,dd->stream);
  #if(DEBUGVEC)
    ierr = VecCheckCUDAStatus(ccs[0],"on copy D2D in VecCopyOverDevice");CHKERRQ(ierr);
    /*PetscBool same;
    ierr = VecCompare_SeqGPU(d, s, &same,0,s->map->n);CHKERRQ(ierr);
    printf("**** compare**** s and d the same?: %d\n",same); */
  #endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "kernCopyLen"
__global__ void kernCopyLen(int* ly,int* lx){
  if(threadIdx.x==0)*ly=*lx;
}

#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockH2D"
PetscErrorCode VecCopyBlockH2D(Vec v,PetscScalar *y, PetscInt offset, PetscInt blocksize){
  PetscFunctionBegin;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  ccs[0]=hipMemcpyAsync(&(vd->devptr[offset]),y,
               blocksize*sizeof(PetscScalar),hipMemcpyHostToDevice,vd->stream);
  #if(DEBUGVEC)
    #if(VERBOSE)
       printf("Call to VecCopyBlockH2D\n");
    #endif
    PetscErrorCode ierr;
    ierr = VecCheckCUDAStatus(ccs[0],"on copy H2D in VecCopyBlockH2D");CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCopyOverH2D"
PetscErrorCode VecCopyOverH2D(Vec v,PetscScalar *y){
  PetscFunctionBegin;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  /*
  int i;
  for(i=0;i<v->map->n;i++){
    if(y[i]!=0)printf("y[%d]: %e\n",i,y[i]);
  }
  */
  ccs[0]=hipMemcpyAsync(vd->devptr,y,
               v->map->n*sizeof(PetscScalar),hipMemcpyHostToDevice,vd->stream);
  #if(DEBUGVEC)
    #if(VERBOSE)
       printf("Call to VecCopyOverH2D\n");
    #endif
    PetscErrorCode ierr;
    ierr = VecCheckCUDAStatus(ccs[0],"on copy H2D in VecCopyOverH2D");CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockD2H"
PetscErrorCode VecCopyBlockD2H(Vec v,PetscScalar *y,PetscInt offset, PetscInt blocksize){
  PetscFunctionBegin;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  ccs[0]=hipMemcpyAsync(y,&(vd->devptr[offset]),
               blocksize*sizeof(PetscScalar),hipMemcpyDeviceToHost,vd->stream);
  #if(DEBUGVEC)
    #if(VERBOSE)
       printf("Call to VecCopyBlockD2H\n");
    #endif
    PetscErrorCode ierr;
    ierr = VecCheckCUDAStatus(ccs[0],"on copy D2H in VecCopyBlockD2H");CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCopyOverD2H"
PetscErrorCode VecCopyOverD2H(Vec v,PetscScalar *y){
  PetscFunctionBegin;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  ccs[0]=hipMemcpyAsync(y,vd->devptr,
               v->map->n*sizeof(PetscScalar),hipMemcpyDeviceToHost,vd->stream);
  #if(DEBUGVEC)
    #if(VERBOSE)
      printf("Call to VecCopyOverD2H\n");
    #endif
    PetscErrorCode ierr;
    ierr = VecCheckCUDAStatus(ccs[0],"on copy D2H in VecCopyOverD2H");CHKERRQ(ierr); 
  #endif
  PetscFunctionReturn(0);
}

/*---------------------------- end copy functions --------------------------*/

/*------------------------------ set functions -----------------------------*/
EXTERN_C_BEGIN
#undef __FUNCT__
#define __FUNCT__ "VecSetValues_SeqGPU"
/*
   VecSetValues - Inserts or adds values into certain locations of a vector.
*/
PetscErrorCode VecSetValues_SeqGPU(Vec x,PetscInt ni,const PetscInt ix[],const PetscScalar y[],InsertMode iora){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt i;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecSetValues_SeqGPU\n");
  #endif
  if(xd->syncState==VEC_CPU || xd->syncState==VEC_SYNCHED){
    if(iora==INSERT_VALUES){
      for(i=0;i<ni;i++){
         xd->cpuptr[i]=y[i];
      }
      ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
      xd->syncState=VEC_SYNCHED;
    }else{
      /* ADD_VALUES not supported now */
      printf("Call to VecSetValues_SeqGPU: ADD_VALUES (*** EMPTY ***)\n");
    }
  }else{
      if(iora==INSERT_VALUES){/* not efficient */
        PetscScalar yval=0;
        for(i=0;i<ni;i++){
          yval=y[i];
          ierr = VecCopyBlockH2D(x,&yval,ix[i],1);CHKERRQ(ierr);
        }
      }
      xd->syncState=VEC_GPU;
  }
  PetscFunctionReturn(0);
}
EXTERN_C_END



#undef __FUNCT__
#define __FUNCT__ "VecSet_SeqGPU"
PetscErrorCode VecSet_SeqGPU(Vec xin,PetscScalar alpha){
  PetscFunctionBegin;
  #if(DEBUGVEC)
    PetscErrorCode ierr;
  #endif
  dim3 dimGrid, dimBlock;
  dimGrid.x = ceil((float)xin->map->n/((float)TCOUNT));
  dimBlock.x = TCOUNT;
  while(dimGrid.x>MAXBLOCKS){
     dimGrid.x/=2;
     dimBlock.x*=2;
  }
  Vec_SeqGPU* xd = (Vec_SeqGPU*)xin->data;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecSet_SeqGPU, alpha: %e\n",alpha);
  #endif
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,
            PETSC_ERR_MEM,"*** In VecSet_SeqGPU, Vec not allocated. ***\n");
  }else{
    ccs[0]=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha,sizeof(double),0,hipMemcpyHostToDevice);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(ccs[0],"error in symbol copy to device");CHKERRQ(ierr);
    #endif
       kernSet<<<dimGrid,dimBlock,dimBlock.x*sizeof(double)>>>(xd->devptr,xd->length);
    #if(DEBUGVEC)
       #if(VERBOSE)
          printf("In VecSet_SeqGPU: blocks: %d, threads: %d\n",dimGrid.x, dimBlock.x);
       #endif
       ierr = VecCheckCUDAError("Call to kernSet. "); CHKERRQ(ierr);
    #endif
    xd->syncState=VEC_GPU;
  }
  PetscFunctionReturn(0);
}

extern __shared__ double sharedSet[];
#undef __FUNCT__
#define __FUNCT__ "kernSet"
__global__ void kernSet(double* x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  double* setptr = sharedSet;
  setptr[threadIdx.x] = dblScalarValue;
  if(tid<*n) x[tid] = setptr[threadIdx.x];
}




#undef __FUNCT__
#define __FUNCT__ "VecScale_SeqGPU"
PetscErrorCode VecScale_SeqGPU(Vec x, PetscScalar alpha){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  dim3 dimGrid,dimBlock;
  dimGrid.x=ceil((float)x->map->n/((float)TCOUNT));
  dimBlock.x=TCOUNT;
  while(dimGrid.x>MAXBLOCKS){
    dimGrid.x/=2;
    dimBlock.x*=2;
  }
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  #if(DEBUGVEC && VERBOSE)
     printf("VecScale_SeqGPU...alpha: %e\n",alpha);
  #endif
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,
            PETSC_ERR_MEM,
            "*** In call to VecScale_SeqGPU, arg Vec xin has not been allocated. ***\n");
  }else if(xd->syncState==VEC_CPU){
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();
  if(alpha==0.){
    ccs[0] = hipMemsetAsync(xd->devptr,0,x->map->n*sizeof(double),xd->stream);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(ccs[0],"error in hipMemset");CHKERRQ(ierr);
    #endif
  }else if (alpha != 1.0){
    ccs[0]=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha,sizeof(double),0,hipMemcpyHostToDevice);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(ccs[0],"error in symbol copy to device");CHKERRQ(ierr);
    #endif
       kernScale<<<dimGrid,dimBlock,dimBlock.x*sizeof(double),xd->stream>>>(xd->devptr,xd->length);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("Call to kernScale."); CHKERRQ(ierr);
    #endif
  }
  xd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}

extern __shared__ double sharedScale[];
#undef __FUNCT__
#define __FUNCT__ "kernScale"
__global__ void kernScale(double* x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  double* scaleptr = sharedScale;
  __shared__ double scalar;
  scalar=dblScalarValue;
  if(tid<*n){
    scaleptr[threadIdx.x] = x[tid];
    scaleptr[threadIdx.x]*= scalar;
    x[tid] = scaleptr[threadIdx.x];
  }
}

/*---------------------------- end set and scale ---------------------------*/


/*-------------------------- dot product functions -------------------------*/

#undef __FUNCT__
#define __FUNCT__ "VecTDot_SeqGPU"
PetscErrorCode VecTDot_SeqGPU(Vec xin,Vec yin,PetscScalar *z){
  PetscFunctionBegin;
  printf("VecTDot_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDot_SeqGPU"
PetscErrorCode VecDot_SeqGPU(Vec x,Vec y,PetscScalar *z){
  PetscFunctionBegin;
  if(x->map->n!=y->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  PetscErrorCode ierr;
  double *devScratch,*devPartial,*hostPartial;
  PetscInt i,chunks=0,secondPhase,segment,partialsize,scratchsize;
  hipStream_t* dotstream;
  dim3 dimGrid, dimBlock;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  if(xd->syncState==VEC_CPU){
    #if(DEBUGVEC && VERBOSE)
       printf("xd state VEC_CPU: copying to device.\n");
    #endif
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  if(yd->syncState==VEC_CPU){
    #if(DEBUGVEC && VERBOSE)
       printf("yd state VEC_CPU: copying to device.\n");
    #endif
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  /* figure out how many chunks will be needed */
  chunks = ceil( ((float)x->map->n) /(float)(CHUNKWIDTH));
  dotstream = (hipStream_t*)malloc(chunks*sizeof(hipStream_t));
  /* make sure the segment size for each chunk is correct */
  if(chunks>1) segment = (int) CHUNKWIDTH;
  else segment = x->map->n;
  dimGrid.x=ceil(((float)segment)/(float)THRDOTCNT);
  dimBlock.x = THRDOTCNT;
  /* allocate gridwide scratch array */
  scratchsize=chunks*dimGrid.x*sizeof(double);
  cms[0] = hipMalloc((void**)&devScratch,scratchsize);/* scratch pad */
  ccs[0] = hipMemsetAsync(devScratch,0,scratchsize,xd->stream);
  ccs[1]=hipMemcpyAsync(xd->segment,&segment,sizeof(int),hipMemcpyHostToDevice,yd->stream);
  #if(DEBUGVEC)
    #if(VERBOSE)
      printf("Call to VecDot_SeqGPU\n");
    #endif
    ierr = VecCheckCUDAStatus(cms[0],"devScratch alloc in VecDot_SeqGPU");CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[0],"devScratch memset in VecDot_SeqGPU");CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[1],"on copy segment size H2D in VecDot_SeqGPU");CHKERRQ(ierr);
  #endif
  hipDeviceSynchronize();/* make sure everyone is ready */
  for(i=0;i<chunks;i++){  /* streaming async kernel calls */
    hipStreamCreate(&(dotstream[i]));
    hipMemcpyAsync(xd->offset,&i,sizeof(int),hipMemcpyHostToDevice,dotstream[i]);
    /* Overlapping execution */
    kernDot<<<dimGrid,dimBlock,0,dotstream[i]>>>(xd->devptr,yd->devptr,
                                                          xd->segment,
                                                          xd->length,
                                                          xd->offset,
                                                          (devScratch+i*dimGrid.x));
  }
  secondPhase = scratchsize/sizeof(double);
  if(secondPhase>1){/* begin next reduction */
    dimGrid.x = ceil((float)secondPhase/(float)THRDOTCNT2);
    dimBlock.x  = THRDOTCNT2;
    /* allocate last reduction array */
    partialsize=dimGrid.x*sizeof(double);
    cms[1]=hipMalloc((void**)&devPartial,partialsize);/* partial results to be combined */
    ccs[2]=hipMemsetAsync(devPartial,0,partialsize,yd->stream);
    ccs[3] = hipMemcpyAsync(xd->segment,&secondPhase,sizeof(int),hipMemcpyHostToDevice,xd->stream);
    #if(DEBUGVEC)
       #if(VERBOSE)
         printf("DOT phase2: blks: %d, partial: %d\n",dimGrid.x,partialsize/sizeof(double));
       #endif
       ierr = VecCheckCUDAStatus(cms[1],"devPartial alloc in VecDot_SeqGPU");CHKERRQ(ierr);
       ierr = VecCheckCUDAStatus(ccs[2],"devPartial memset in VecDot_SeqGPU");CHKERRQ(ierr);
       ierr = VecCheckCUDAStatus(ccs[3],"on hipMemcpy(xd->segment)");CHKERRQ(ierr);
    #endif
    hipDeviceSynchronize();/* make sure everyone is caught up */
    kernRedDot<<<dimGrid,dimBlock,dimBlock.x*sizeof(double)>>>(xd->segment,devScratch,devPartial);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernRedDot launch in VecDot_SeqGPU");CHKERRQ(ierr); 
    #endif
    /* setup copy back array while waiting for kernel to finish */
    ierr = PetscMalloc(partialsize,&hostPartial);CHKERRQ(ierr);
    hipDeviceSynchronize();/* make sure everyone is caught up */
    ccs[4]=hipMemcpy(hostPartial,devPartial,partialsize,hipMemcpyDeviceToHost);/* copy back */
    hipDeviceSynchronize();/* make sure everyone is caught up */
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(ccs[4],"on hipMemcpy(devPartial)");CHKERRQ(ierr);
    #endif
    if(dimGrid.x>1){/* final reduction */
      *z=0.;
      for(i=0;i<dimGrid.x;i++)*z+=hostPartial[i];
    }else{
      *z=hostPartial[0];
    }
    ierr = PetscFree(hostPartial); CHKERRQ(ierr);
    cms[2] = hipFree(devPartial);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(cms[2],"on hipFree(devPartial)");CHKERRQ(ierr);
    #endif
  }else{
    ccs[4]=hipMemcpy(z,devScratch,dimGrid.x*sizeof(double),hipMemcpyDeviceToHost);/* copy back */
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(ccs[4],"on hipMemcpy(devScratch)");CHKERRQ(ierr);
    #endif
  }

  /* clean up resources */
  for(i=0;i<chunks;i++){
     hipStreamDestroy(dotstream[i]);
  }
  free(dotstream);
  cms[3] = hipFree(devScratch);
  #if(DEBUGVEC)
     #if(VERBOSE)
       printf("Zdot: %e\n",*z);
    #endif
    ierr = VecCheckCUDAStatus(cms[3],"on hipFree(devScratch)");CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}

extern __shared__ double sharedRedDot[];
#undef __FUNCT__
#define __FUNCT__ "kernRedDot"
__global__ void kernRedDot(int* size,double* scratch, double* z){/* reduction kernel */
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int i = (blockDim.x+1)/2;
  double* zDot = sharedRedDot;
  zDot[threadIdx.x]=(tid<*size)?scratch[tid]:0.;
  while(i>0){
    if(threadIdx.x<i){
      zDot[threadIdx.x]+=zDot[threadIdx.x+i];
    }
    __syncthreads();
    i/=2;
  }
  if(threadIdx.x==0){
    z[blockIdx.x]=zDot[0];
    //printf("ZDOT block[%d]: %e\n",blockIdx.x,z[blockIdx.x]);
  }
}

#undef __FUNCT__
#define __FUNCT__ "kernDot"
__global__ void kernDot(double* devX, double* devY,
                        int* segmentsize, int* arrsize,
                        int* offset, double* scratch){
  __shared__ double chunkX[THRDOTCNT];
  __shared__ double chunkY[THRDOTCNT];
  __shared__ int n;    n   = *arrsize;

  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int i = (blockDim.x+1)/2;
  int item = *segmentsize**offset+tid;

  if(item<n){/* read in values to shared mem */
    chunkX[threadIdx.x]=devX[item]; /* offset values */
    chunkY[threadIdx.x]=devY[item]; /* offset values */
  }else{
    chunkX[threadIdx.x]=0.;
    chunkY[threadIdx.x]=0.;
  }
  chunkX[threadIdx.x]*=chunkY[threadIdx.x];
  __syncthreads();
  while(i>0){/* block level reduction */
     if(threadIdx.x<i){
       chunkX[threadIdx.x]+=chunkX[threadIdx.x+i];
     }
     __syncthreads();
     i/=2;
  }/* end while */

  if(threadIdx.x==0)  scratch[blockIdx.x]=chunkX[0];
}


#undef __FUNCT__
#define __FUNCT__ "VecMDot_SeqGPU"
PetscErrorCode  VecMDot_SeqGPU(Vec x,PetscInt nv,const Vec y[],PetscScalar val[]){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt       i;
  for (i=0; i<nv; i++) {
    ierr = VecDot_SeqGPU(x,y[i],&val[i]);CHKERRQ(ierr);
    if(PetscIsInfOrNanScalar(val[i])){
      SETERRQ1(((PetscObject)x)->comm,PETSC_ERR_FP,"Infinite or not-a-number generated in mdot, entry %D",i);
    }
  }
  PetscFunctionReturn(0);
}

/*----------------------------- end dot ----------------------------- */





#undef __FUNCT__
#define __FUNCT__ "VecAXPBY_SeqGPU"
PetscErrorCode VecAXPBY_SeqGPU(Vec yin,PetscScalar beta,PetscScalar alpha,Vec xin){
  /* Y = b*Y + a*X */
  PetscFunctionBegin;
  printf("Call to VecAXPBY_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecWAXPY_SeqGPU"
PetscErrorCode VecWAXPY_SeqGPU(Vec w,PetscScalar alpha,Vec x,Vec y){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)w->data;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid, dimBlock;
  #if(DEBUGVEC && VERBOSE)
     printf("VecWAXPY_SeqGPU...alpha: %e\n",alpha);
  #endif
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  dimGrid.x=ceil((float)y->map->n/(float)AXPYTCOUNT);
  dimBlock.x=AXPYTCOUNT;
  while(dimGrid.x>MAXBLOCKS){
    dimGrid.x/=2;
    dimBlock.x*=2;
  }
  hipDeviceSynchronize();
  if(alpha==0.0){
    ierr = VecCopyOverDevice(w,y);CHKERRQ(ierr);
  }else if(alpha==1.0){
    kernWXPY<<<dimGrid,dimBlock,3*dimBlock.x*sizeof(double)>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernel call to kernWXPY");CHKERRQ(ierr); 
    #endif
  }else if(alpha==-1.0){
    kernWXMY<<<dimGrid,dimBlock,3*dimBlock.x*sizeof(double)>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernel call to kernWXMY");CHKERRQ(ierr);
    #endif
  }else{
    ccs[0]=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha,sizeof(double),0,hipMemcpyHostToDevice);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(ccs[0],"error in symbol copy to device");CHKERRQ(ierr);
    #endif
    kernWAXPY<<<dimGrid,dimBlock,3*dimBlock.x*sizeof(double)>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernel call to kernWAXPY");CHKERRQ(ierr); 
    #endif
  }
  wd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}


extern __shared__ double sharedWAXPY[];
#undef __FUNCT__
#define __FUNCT__ "kernWAXPY"
__global__ void  kernWAXPY(double* devY,double* devX, int* vlen, double* devW){
 /* w <- y + alpha*x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double alphaShared;
  double* chunkX = sharedWAXPY;
  double* chunkY = sharedWAXPY + blockDim.x;
  double* chunkW = sharedWAXPY + 2*blockDim.x;
  alphaShared = dblScalarValue;
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkY[threadIdx.x]+(chunkX[threadIdx.x]*alphaShared);
    devW[tid]=chunkW[threadIdx.x];
  }
}

extern __shared__ double sharedWXPY[];
#undef __FUNCT__
#define __FUNCT__ "kernWXPY"
__global__ void  kernWXPY(double* devY,double* devX, int* vlen, double* devW){
 /* w <- y + x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  double* chunkX = sharedWXPY;
  double* chunkY = sharedWXPY + blockDim.x;
  double* chunkW = sharedWXPY + 2*blockDim.x;
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkY[threadIdx.x]+chunkX[threadIdx.x];
    devW[tid]=chunkW[threadIdx.x];
  }
}

extern __shared__ double sharedWXMY[];
#undef __FUNCT__
#define __FUNCT__ "kernWXMY"
__global__ void  kernWXMY(double* devY,double* devX, int* vlen, double* devW){
 /* w <- y + alpha*x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  double* chunkX = sharedWXMY;
  double* chunkY = sharedWXMY + blockDim.x;
  double* chunkW = sharedWXMY + 2*blockDim.x;
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkY[threadIdx.x]-chunkX[threadIdx.x];
    devW[tid]=chunkW[threadIdx.x];
  }
}

#undef __FUNCT__
#define __FUNCT__ "VecMAXPY_SeqGPU"
PetscErrorCode VecMAXPY_SeqGPU(Vec x,PetscInt nv,const PetscScalar* alpha,Vec *y){
  /* y = y + sum(a[i]*x[i]) */
  PetscFunctionBegin;
  if(DEBUGVEC && VERBOSE)printf("VecMAXPY_SeqGPU: alpha: %e\n",*alpha);
  PetscErrorCode ierr;
  PetscInt i;
  PetscScalar *devW;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=PETSC_NULL;
  cms[0] = hipMalloc((void**)&devW,x->map->n*sizeof(double));
  ccs[0] = hipMemsetAsync(devW,0,x->map->n*sizeof(double),xd->stream);
  dim3 dimGrid;  dim3 dimBlock;
  dimGrid.x=ceil((float)x->map->n/(float)AXPYTCOUNT);
  dimBlock.x=AXPYTCOUNT;
  while(dimGrid.x>MAXBLOCKS){
    dimGrid.x/=2;
    dimBlock.x*=2;
  }
  #if(DEBUGVEC)
    #if(VERBOSE)
       printf("Number of vectors in MAXPY: %d, blocks: %d, threads: %d\n",nv,dimGrid.x,dimBlock.x);
    #endif
    ierr = VecCheckCUDAStatus(cms[0],"error in device malloc VecMAXPY_SeqGPU");CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[0],"error in device memset VecMAXPY_SeqGPU");CHKERRQ(ierr);
  #endif

  for(i=0;i<nv;i++){
     if(y[i]->map->n!=x->map->n){
      SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
    }
    yd=(Vec_SeqGPU*)y[i]->data;
    if(yd->syncState==VEC_CPU){/* synch x */
      ierr = VecCopyOverH2D(y[i],yd->cpuptr);CHKERRQ(ierr);
      yd->syncState=VEC_SYNCHED;
    }
    ccs[1]=hipMemcpy(yd->scalar,&alpha[i],sizeof(double),hipMemcpyHostToDevice);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(ccs[1],"error in symbol copy to device");CHKERRQ(ierr); 
    #endif
    hipDeviceSynchronize();
    if(alpha[i]==0){
      continue;
    }else if(alpha[i]==1.){
      kernXPY<<<dimGrid,dimBlock,2*dimBlock.x*sizeof(double)>>>(devW,yd->devptr,yd->length);
    }else{
      kernAXPY<<<dimGrid,dimBlock,2*dimBlock.x*sizeof(double)>>>(devW,yd->devptr,yd->length,yd->scalar);
    }
    #if(DEBUGVEC)
         ierr = VecCheckCUDAError("kernel call to kernAXPY or kernXPY in VecMAXPY_SeqGPU");CHKERRQ(ierr);
    #endif
  }/* end for */
  if(xd->syncState==VEC_CPU){/* synch x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  hipDeviceSynchronize();
  kernXPY<<<dimGrid,dimBlock,2*dimBlock.x*sizeof(double)>>>(xd->devptr,devW,xd->length);
  #if(DEBUGVEC)
     ierr = VecCheckCUDAError("kernel call to kernXPY");CHKERRQ(ierr);
  #endif
  cms[1] = hipFree(devW);
  #if(DEBUGVEC)
     ierr = VecCheckCUDAStatus(cms[1],"on hipFree");CHKERRQ(ierr);
  #endif
  xd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}

extern __shared__ double sharedXPY[];
#undef __FUNCT__
#define __FUNCT__ "kernXPY"
__global__ void  kernXPY(double* devY,double* devX, int* vlen){
 /* y <- y + x */
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  double* chunkX = sharedXPY;
  double* chunkY = sharedXPY + blockDim.x;
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkY[threadIdx.x]+=chunkX[threadIdx.x];
    devY[tid]=chunkY[threadIdx.x];
  }
}

#undef __FUNCT__
#define __FUNCT__ "VecAXPY_SeqGPU"
PetscErrorCode VecAXPY_SeqGPU(Vec y,PetscScalar alpha,Vec x){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  if(x->map->n!=y->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  #if(DEBUGVEC && VERBOSE)
      printf("VecAXPY_SeqGPU\n");
  #endif
  dim3 dimGrid, dimBlock;
  dimGrid.x=ceil((float)x->map->n/(float)AXPYTCOUNT);
  dimBlock.x=AXPYTCOUNT;
  while(dimGrid.x>MAXBLOCKS){
    dimGrid.x/=2;
    dimBlock.x*=2;
  }
  hipDeviceSynchronize();
  if(alpha==1.){
    kernXPY<<<dimGrid,dimBlock,2*dimBlock.x*sizeof(double)>>>(yd->devptr,xd->devptr,yd->length);
  }else if(alpha!=0.){
    ccs[0]=hipMemcpy(yd->scalar,&alpha,sizeof(double),hipMemcpyHostToDevice);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(ccs[0],"error in symbol copy to device");CHKERRQ(ierr);
    #endif
    kernAXPY<<<dimGrid,dimBlock,2*dimBlock.x*sizeof(double)>>>(yd->devptr,xd->devptr,yd->length,yd->scalar);
  }
  #if(DEBUGVEC)
   ierr = VecCheckCUDAError("kernel call in VecAXPY_SeqGPU");CHKERRQ(ierr);
  #endif
  yd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}


extern __shared__ double sharedAXPY[];
#undef __FUNCT__
#define __FUNCT__ "kernAXPY"
__global__ void  kernAXPY(double* devY,double* devX, int* vlen,double *scalar){
 /* y <- y + alpha*x */
  __shared__ double alphaShared;
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  double* chunkX = sharedAXPY;
  double* chunkY = sharedAXPY + blockDim.x;
  alphaShared = *scalar;
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkY[threadIdx.x]+=chunkX[threadIdx.x]*alphaShared;
    devY[tid]=chunkY[threadIdx.x];
  }
}

#undef __FUNCT__
#define __FUNCT__ "VecAXPBYPCZ_SeqGPU"
PetscErrorCode VecAXPBYPCZ_SeqGPU(Vec x,PetscScalar alpha,PetscScalar beta,PetscScalar gamma,Vec y,Vec z){
  PetscFunctionBegin;
  #if(DEBUGVEC)
     PetscErrorCode ierr;
     #if(VERBOSE)
        printf("Call to VecAXPBYPCZ_SeqGPU\n");
     #endif
  #endif
  Vec_SeqGPU* devX = (Vec_SeqGPU*)x->data;
  Vec_SeqGPU* devY = (Vec_SeqGPU*)y->data;
  Vec_SeqGPU* devZ = (Vec_SeqGPU*)z->data;
  double2 alphabeta;  alphabeta.x = alpha;  alphabeta.y = beta;
  dim3 dimGrid, dimBlock;
  dimGrid.x=ceil((float)x->map->n/(float)AXPBYPCZTCOUNT);
  dimBlock.x=AXPBYPCZTCOUNT;
  while(dimGrid.x>MAXBLOCKS){
    dimGrid.x/=2;
    dimBlock.x*=2;
  }
  ccs[0]=hipMemcpyToSymbol(HIP_SYMBOL("dblScalar2Value"),(void*)&alphabeta,sizeof(double2),0,hipMemcpyHostToDevice);
  ccs[1]=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&gamma,sizeof(double),0,hipMemcpyHostToDevice);
  #if(DEBUGVEC)
   ierr = VecCheckCUDAStatus(ccs[0],"error in symbol copy to device");CHKERRQ(ierr);
   ierr = VecCheckCUDAStatus(ccs[1],"error in symbol copy to device");CHKERRQ(ierr);
  #endif
  hipDeviceSynchronize();
  kernAXPBYPCZ<<<dimGrid,dimBlock,4*dimBlock.x*sizeof(double)>>>(devX->devptr,devY->devptr,devZ->devptr,devX->length);
  #if(DEBUGVEC)
     ierr = VecCheckCUDAError("launch kernAXPBYPCZ");CHKERRQ(ierr); 
  #endif
  PetscFunctionReturn(0);
}

extern __shared__ double sharedAXPBYPCZ[];
#undef __FUNCT__
#define __FUNCT__ "kernAXPBYPCZ"
__global__ void kernAXPBYPCZ(double* devX, double* devY, double* devZ, int* len){
  /* x <- alpha*x + beta*y + gamma*z */
  __shared__ int localn;
  localn = *len;
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  double* work = sharedAXPBYPCZ;
  double* chunkX = sharedAXPBYPCZ + blockDim.x;
  double* chunkY = sharedAXPBYPCZ + 2*blockDim.x;
  double* chunkZ = sharedAXPBYPCZ + 3*blockDim.x;
  if(tid<localn){
    /* read in values to shared */
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkZ[threadIdx.x]=devZ[tid];

    /* do flops */
    if(dblScalarValue){
      work[threadIdx.x]=dblScalarValue*chunkZ[threadIdx.x];
    }else{
      work[threadIdx.x]=0.;
    }

    if(dblScalar2Value.y){
      work[threadIdx.x]+=dblScalar2Value.y*chunkY[threadIdx.x];
    }
    if(dblScalar2Value.x){
      work[threadIdx.x]+=dblScalar2Value.x*chunkX[threadIdx.x];
    }

    /* write back */
    devX[tid]=work[threadIdx.x];
  }
  return;
}

/*---------------------------- end level 2 ------------------------------ */

/*------------------------- pointwise functions ------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecPointwiseMult_SeqGPU"
PetscErrorCode VecPointwiseMult_SeqGPU(Vec w,Vec x,Vec y){
  PetscFunctionBegin;
  #if(DEBUGVEC && VERBOSE)
     printf("VecPointwiseMult_SeqGPU\n");
  #endif
  PetscErrorCode ierr;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid, dimBlock;
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  dimGrid.x=ceil((float)y->map->n/(float)PMULTCOUNT);
  dimBlock.x=PMULTCOUNT;
  while(dimGrid.x>MAXBLOCKS){
    dimGrid.x/=2;
    dimBlock.x*=2;
  }
  hipDeviceSynchronize();
  kernPMULT<<<dimGrid,dimBlock,3*dimBlock.x*sizeof(double)>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
  #if(DEBUGVEC)
     ierr = VecCheckCUDAError("kernel call to kernPMULT");CHKERRQ(ierr);
  #endif

  PetscFunctionReturn(0);
}

extern __shared__ double sharedPMULT[];
#undef __FUNCT__
#define __FUNCT__ "kernPMULT"
__global__ void  kernPMULT(double* devY,double* devX, int* vlen, double* devW){
 /* w <- x./y */
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  double* chunkX = sharedPMULT;
  double* chunkY = sharedPMULT + blockDim.x;
  double* chunkW = sharedPMULT + 2*blockDim.x;
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkX[threadIdx.x]*chunkY[threadIdx.x];
    devW[tid]=chunkW[threadIdx.x];
  }
}


#undef __FUNCT__
#define __FUNCT__ "VecMaxPointwiseDivide_SeqGPU"
PetscErrorCode VecMaxPointwiseDivide_SeqGPU(Vec x,Vec y,PetscReal *max){
  PetscFunctionBegin;
  #if(DEBUGVEC && VERBOSE)
     printf("VecMaxPointwiseDivide_SeqGPU...");
  #endif
  if(x->map->n!=y->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  PetscErrorCode ierr;
  PetscScalar *devScratch,*devPartial,*hostPartial;
  PetscInt i,chunks=0,segment,partialsize,scratchsize,secondPhase;
  hipStream_t* pwdstream;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid;  dim3 dimBlock;
  if(yd->syncState==VEC_CPU){/* synch up y */
    #if(DEBUGVEC && VERBOSE)
       printf("yd state VEC_CPU: copying to device.\n");
    #endif
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    #if(DEBUGVEC && VERBOSE)
       printf("xd state VEC_CPU: copying to device.\n");
    #endif
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  /* figure out how many chunks will be needed */
  chunks = ceil( ((float)x->map->n) /(float)(CHUNKWIDTH));
  pwdstream = (hipStream_t*)malloc(chunks*sizeof(hipStream_t));
  /* make sure the segment size for each chunk is correct */
  if(chunks>1)segment = (int) (CHUNKWIDTH);
  else segment = x->map->n;
  dimGrid.x=ceil(((float)segment)/(float)PDIVTCOUNT);
  dimBlock.x  = PDIVTCOUNT;

  scratchsize = chunks*dimGrid.x*sizeof(double);
  cms[0] = hipMalloc((void**)&devScratch,scratchsize);
  ccs[0] = hipMemsetAsync(devScratch,0,scratchsize,yd->stream);
  ccs[1]=hipMemcpyAsync(xd->segment,&segment,sizeof(int),hipMemcpyHostToDevice,xd->stream);
  #if(DEBUGVEC)
    ierr = VecCheckCUDAStatus(cms[0],"devScratch alloc in VecMPWD_SeqGPU");CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[0],"devScratch memset in VecMPWD_SeqGPU");CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[1],"on copy segment size H2D in VecMPWD_SeqGPU");CHKERRQ(ierr);
  #endif

  hipDeviceSynchronize();
  for(i=0;i<chunks;i++){
    hipStreamCreate(&(pwdstream[i]));
    hipMemcpyAsync(xd->offset,&i,sizeof(int),hipMemcpyHostToDevice,pwdstream[i]);
    /* Overlapping execution */
    kernMAXPDIV<<<dimGrid,dimBlock,0,pwdstream[i]>>>(xd->devptr,yd->devptr,
                                                     xd->segment,
                                                     xd->length,
                                                     xd->offset,
                                                     (devScratch+i*dimGrid.x));
  }/* end for-loop */

  secondPhase = scratchsize/sizeof(double);
  if(secondPhase>1){/* begin next reduction */
    dimGrid.x = ceil((float)secondPhase/(float)PDIVTCOUNT2);
    dimBlock.x  = PDIVTCOUNT2;
    /* allocate last reduction array */
    partialsize = dimGrid.x*sizeof(double);
    cms[1] = hipMalloc((void**)&devPartial,partialsize);
    ccs[2] = hipMemsetAsync(devPartial,0,partialsize,yd->stream);
    ccs[3] = hipMemcpyAsync(xd->segment,&secondPhase,sizeof(int),hipMemcpyHostToDevice,xd->stream);
    #if(DEBUGVEC)
       #if(VERBOSE)
         printf("MAXDIV phase2: blks: %d, partial: %d\n",dimGrid.x,partialsize/sizeof(double));
       #endif
       ierr = VecCheckCUDAStatus(cms[1],"devPartial alloc in VecMPWD_SeqGPU"); CHKERRQ(ierr);
       ierr = VecCheckCUDAStatus(ccs[2],"devPartial memset in VecMPWD_SeqGPU");CHKERRQ(ierr);
       ierr = VecCheckCUDAStatus(ccs[3],"on copy chunks H2D in VecMPWD_SeqGPU");CHKERRQ(ierr);
    #endif
    hipDeviceSynchronize();/* make sure everyone is caught up */
    kernMAX<<<dimGrid,dimBlock,dimBlock.x*sizeof(double)>>>(xd->segment,devScratch,devPartial);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAError("kernRedNorm_double launch in VecNorm_SeqGPU");CHKERRQ(ierr);
    #endif
    ierr = PetscMalloc(partialsize,&hostPartial);CHKERRQ(ierr);
    hipDeviceSynchronize();/* make sure everyone is caught up */
    ccs[4]=hipMemcpy(hostPartial,devPartial,partialsize,hipMemcpyDeviceToHost);/* copy back */
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(ccs[4],"on devPartial copy D2H");CHKERRQ(ierr);
    #endif
    hipDeviceSynchronize();/* make sure everyone is caught up */
    /* final reduction */
    if(dimGrid.x>1){
      *max=0.;
      for(i=0;i<dimGrid.x;i++){
        *max=PetscMax(hostPartial[i],*max);
      }
    }else{
      *max=hostPartial[0];
    }
    ierr = PetscFree(hostPartial); CHKERRQ(ierr);
    cms[2] = hipFree(devPartial);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(cms[2],"on hipFree(devPartial)");CHKERRQ(ierr);
    #endif

  }else{
    ccs[4]=hipMemcpy(max,devScratch,sizeof(double),hipMemcpyDeviceToHost);
    #if(DEBUGVEC)
       ierr = VecCheckCUDAStatus(ccs[4],"on max copy D2H");CHKERRQ(ierr);
    #endif
  }
  cms[3] = hipFree(devScratch);
  #if(DEBUGVEC)
     ierr = VecCheckCUDAStatus(cms[3],"on hipFree(devScratch)");CHKERRQ(ierr);
  #endif
  for(i=0;i<chunks;i++) hipStreamDestroy(pwdstream[i]);

  #if(DEBUGVEC && VERBOSE)
     printf("max: %e\n",*max);
  #endif
  PetscFunctionReturn(0);
}


extern __shared__ double sharedMAX[];
#undef __FUNCT__
#define __FUNCT__ "kernMAX"
__global__ void  kernMAX(int* size, double* maxlist,double* max){
  int tid = blockDim.x*blockIdx.x+threadIdx.x;
  int i = (blockDim.x+1)/2;
  double* mlist = sharedMAX;
  mlist[threadIdx.x]=(tid<*size)?maxlist[tid]:0.;
  /* printf("mlist[%d]: %e\n",threadIdx.x,mlist[threadIdx.x]); */
  __syncthreads();
  while(i>0){
    if(threadIdx.x<i){
      mlist[threadIdx.x]=fmax(mlist[threadIdx.x],mlist[threadIdx.x+i]);
    }
    __syncthreads();
    i/=2;
  }
  if(threadIdx.x==0){
    max[blockIdx.x] = mlist[0];
    //printf("MAXDIV block[%d]: %e\n",blockIdx.x,max[blockIdx.x]);
  }
}


#undef __FUNCT__
#define __FUNCT__ "kernMAXPDIV"
__global__ void  kernMAXPDIV(double* devX,double* devY, int* segmentsize,
                             int* arrsize,int* offset,double* scratch){
 /* w <- max(abs(x./y)) */
  __shared__ double chunkY[PDIVTCOUNT];
  __shared__ double chunkX[PDIVTCOUNT];
  __shared__ double chunkW[PDIVTCOUNT];
  __shared__ int n;    n   = *arrsize;
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int i = (blockDim.x+1)/2;
  int item = *segmentsize**offset+tid;
  if(item<n){
    chunkX[threadIdx.x]=devX[item];
    chunkY[threadIdx.x]=devY[item];
  }else{
    chunkX[threadIdx.x]=0.;
    chunkY[threadIdx.x]=0.;
  }
  if(chunkY[threadIdx.x]!=0.)chunkW[threadIdx.x]=fabs(chunkX[threadIdx.x]/chunkY[threadIdx.x]);
  else chunkW[threadIdx.x]=fabs(chunkX[threadIdx.x]);
  /* block Level reduction */
  __syncthreads();
  while(i>0){
    if(threadIdx.x<i){
      chunkW[threadIdx.x]=fmax(chunkW[threadIdx.x],chunkW[threadIdx.x+i]);
    }
    __syncthreads();
    i/=2;
  }
  if(threadIdx.x==0) scratch[blockIdx.x]=chunkW[0];
}


#undef __FUNCT__
#define __FUNCT__ "VecPointwiseDivide_SeqGPU"
PetscErrorCode VecPointwiseDivide_SeqGPU(Vec w,Vec x,Vec y){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid, dimBlock;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecPointwiseDivide_SeqGPU\n");
  #endif
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  dimGrid.x=ceil((float)y->map->n/(float)PDIVTCOUNT);
  dimBlock.x=PDIVTCOUNT;
  while(dimGrid.x>MAXBLOCKS){
    dimGrid.x/=2;
    dimBlock.x*=2;
  }
  hipDeviceSynchronize();
  kernPDIV<<<dimGrid,dimBlock,3*dimBlock.x*sizeof(double)>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
  #if(DEBUGVEC) 
     ierr = VecCheckCUDAError("kernel call to kernPDIV");CHKERRQ(ierr); 
  #endif
  PetscFunctionReturn(0);
}

extern __shared__ double sharedPDIV[];
#undef __FUNCT__
#define __FUNCT__ "kernPDIV"
__global__ void  kernPDIV(double* devY,double* devX, int* vlen, double* devW){
 /* w <- x./y */
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  double* chunkX = sharedPDIV;
  double* chunkY = sharedPDIV + blockDim.x;
  double* chunkW = sharedPDIV + 2*blockDim.x;
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    if(chunkX[threadIdx.x]*chunkY[threadIdx.x]!=0){/* using intrinsic div op */
      chunkW[threadIdx.x]=chunkX[threadIdx.x]/chunkY[threadIdx.x];
    }else{
      chunkW[threadIdx.x]=0;
    }
    devW[tid]=chunkW[threadIdx.x];
  }
}

/*--------------------------- end pointwise ---------------------------- */


/*-------------------------- norm functions ---------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecDotNorm2_SeqGPU"
PetscErrorCode VecDotNorm2_SeqGPU(Vec s, Vec t, PetscScalar *dp, PetscScalar *nm){
  PetscErrorCode ierr;
  PetscFunctionBegin;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecDotNorm2_SeqGPU\n");
  #endif
  ierr = VecDot(s,t,dp); CHKERRQ(ierr);
  ierr = VecNorm(t,NORM_2,nm); CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecNorm_SeqGPU"
PetscErrorCode VecNorm_SeqGPU(Vec x,NormType type,PetscReal* z){
  /* NormType: NORM_1=0,NORM_2=1,NORM_FROBENIUS=2,NORM_INFINITY=3,NORM_1_AND_2=4 */
  /* dealing with NORM_2 for now... */
  PetscFunctionBegin;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecNorm_SeqGPU\n");
  #endif
  PetscErrorCode ierr;
  double *devScratch,*devPartial,*hostPartial,zhost;
  PetscInt i,chunks=0,segment,partialsize,secondPhase,scratchsize;
  hipStream_t* nrmstream;
  dim3 dimGrid, dimBlock;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  if(xd->syncState==VEC_CPU){
    #if(DEBUGVEC && VERBOSE)
       printf("xd state VEC_CPU: copying to device.\n");
    #endif
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }

  /* figure out how many chunks will be needed */
  chunks = ceil( ((float)x->map->n) /(float)(CHUNKWIDTH));
  nrmstream = (hipStream_t*)malloc(chunks*sizeof(hipStream_t));
  /* make sure the segment size for each chunk is correct */
  if(chunks>1) segment = (int) (CHUNKWIDTH);
  else segment = x->map->n;
  dimGrid.x=ceil(((float)segment)/(float)THRNRMCNT);
  dimBlock.x  = THRNRMCNT;
  /* allocate gridwide scratch array */
  scratchsize = chunks*dimGrid.x*sizeof(double);
  cms[0] = hipMalloc((void**)&devScratch,scratchsize);
  ccs[0] = hipMemsetAsync(devScratch,0,scratchsize,xd->stream);
  ccs[1]=hipMemcpy(xd->segment,&segment,sizeof(int),hipMemcpyHostToDevice);
  #if(DEBUGVEC)
    #if(VERBOSE)
      printf("NORM: chunks: %d, seg: %d, blks: %d, scr: %d, chksize: %d, N: %d\n",
            chunks,segment,dimGrid.x,scratchsize/8,(int)(CHUNKWIDTH),x->map->n);
    #endif
    ierr = VecCheckCUDAStatus(cms[0],"devScratch alloc in VecNorm_SeqGPU"); CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[0],"devScratch memset in VecNorm_SeqGPU");CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[1],"on copy segment length H2D in VecNorm_SeqGPU");CHKERRQ(ierr);
  #endif
  hipDeviceSynchronize();
  for(i=0;i<chunks;i++){/* streaming async kernel calls */
    hipStreamCreate(&(nrmstream[i]));
    hipMemcpyAsync(xd->offset,&i,sizeof(int),hipMemcpyHostToDevice,nrmstream[i]);
    /* Overlapping execution */
    kernNorm2<<<dimGrid,dimBlock,0,nrmstream[i]>>>(xd->devptr,
                                                   xd->segment,
                                                   xd->length,
                                                   xd->offset,
                                                   devScratch+i*dimGrid.x);
  }/* end for-loop */

  secondPhase = scratchsize/sizeof(double);
  if(secondPhase>1){ /* begin next reduction */
    dimGrid.x = ceil((float)secondPhase/(float)THRNRMCNT2);
    dimBlock.x  = THRNRMCNT2;
    /* allocate last reduction array */
    partialsize = dimGrid.x*sizeof(double);
    cms[1] = hipMalloc((void**)&devPartial,partialsize);
    ccs[2] = hipMemsetAsync(devPartial,0,partialsize,xd->stream);
    ccs[3] = hipMemcpy(xd->segment,&secondPhase,sizeof(int),hipMemcpyHostToDevice);
    #if(DEBUGVEC)
      #if(VERBOSE)
         printf("NORM phase2: blks: %d, partial: %d\n",dimGrid.x,partialsize/sizeof(double));
      #endif
      ierr = VecCheckCUDAStatus(cms[1],"devPartial alloc in VecNorm_SeqGPU"); CHKERRQ(ierr);
      ierr = VecCheckCUDAStatus(ccs[2],"devPartial memset in VecNorm_SeqGPU");CHKERRQ(ierr);
      ierr = VecCheckCUDAStatus(ccs[3],"on copy chunks H2D in VecNorm_SeqGPU");CHKERRQ(ierr);
    #endif
    hipDeviceSynchronize();/* make sure everyone is caught up */
    kernRedNorm<<<dimGrid,dimBlock,dimBlock.x*sizeof(double)>>>(xd->segment,devScratch,devPartial);
    #if(DEBUGVEC)
      ierr = VecCheckCUDAError("kernRedNorm_double launch in VecNorm_SeqGPU");CHKERRQ(ierr);
    #endif
    ierr = PetscMalloc(partialsize,&hostPartial);CHKERRQ(ierr);

    /* Copy back norm z */
    hipDeviceSynchronize();/* make sure everyone is caught up */
    ccs[4]=hipMemcpy(hostPartial,devPartial,partialsize,hipMemcpyDeviceToHost);/* copy back */
    hipDeviceSynchronize();/* make sure everyone is caught up */
    #if(DEBUGVEC)
      ierr = VecCheckCUDAStatus(ccs[4],"on devPartial copy D2H");CHKERRQ(ierr);
    #endif
    /* final reduction */
    if(dimGrid.x>1){
     zhost=0.;
     for(i=0;i<dimGrid.x;i++)zhost+=hostPartial[i];
    }else{
     zhost=hostPartial[0];
    }
    ierr = PetscFree(hostPartial); CHKERRQ(ierr);
    cms[2] = hipFree(devPartial);
    #if(DEBUGVEC)
      ierr = VecCheckCUDAStatus(cms[2],"on hipFree(devPartial)");CHKERRQ(ierr);
    #endif
  }else{/* only copy back necessary */
     ccs[4]=hipMemcpy(&zhost,devScratch,sizeof(double),hipMemcpyDeviceToHost);/* copy back */
     #if(DEBUGVEC)
        ierr = VecCheckCUDAStatus(ccs[4],"on zhost copy D2H");CHKERRQ(ierr);
     #endif
  }
  *z = PetscSqrtScalar(zhost);


  /* clean up resources */
  for(i=0;i<chunks;i++){
    hipStreamDestroy(nrmstream[i]);
  }
  free(nrmstream);
  cms[3] = hipFree(devScratch);
  #if(DEBUGVEC)
     #if(VERBOSE)
        printf("Znorm: %e\n",*z);
     #endif
     ierr = VecCheckCUDAStatus(cms[3],"on hipFree(devScratch)");CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}

extern __shared__ double sharedRedNorm[];
#undef __FUNCT__
#define __FUNCT__ "kernRedNorm"
__global__ void kernRedNorm(int* size,double* scratch,double* z){/* reduction kernel */
  int i = (blockDim.x+1)/2;
  int tid = blockDim.x*blockIdx.x+threadIdx.x;
  double* zptr = sharedRedNorm;
  zptr[threadIdx.x]=(tid<*size)?scratch[tid]:0.;
  //printf("zptr[%d]: %e\n",threadIdx.x,zptr[threadIdx.x]);
  __syncthreads();
  while(i>0){
    if(threadIdx.x<i) zptr[threadIdx.x]+=zptr[threadIdx.x+i];
    __syncthreads();
    i/=2;
  }/* end while */
  if(threadIdx.x==0){
    z[blockIdx.x]=zptr[0];
    //printf("ZNorm block[%d]: %e\n",blockIdx.x,z[blockIdx.x]);
  }
}


#undef __FUNCT__
#define __FUNCT__ "kernNorm2"
__global__ void kernNorm2(double* devX,int* segmentsize,int* arrsize,
                          int* offset,double *scratch){
  __shared__ double chunkX[THRNRMCNT];
  __shared__ int n;    n   = *arrsize;

  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int i = (blockDim.x+1)/2;
  int item = *segmentsize**offset+tid;

  /* read in values to shared */
  chunkX[threadIdx.x]=(item<n)?devX[item]:0.;
  chunkX[threadIdx.x]*=chunkX[threadIdx.x];
  __syncthreads();

  /* block level reduction */
  while(i>0){
     if(threadIdx.x<i){
       chunkX[threadIdx.x]+=chunkX[threadIdx.x+i];
     }
     __syncthreads();
     i/=2;
  }/* end while */
  if(threadIdx.x==0) scratch[blockIdx.x]=chunkX[0];
}


/*
#undef __FUNCT__
#define __FUNCT__ "VecNorm1_SeqGPU"
PetscErrorCode VecNorm1_SeqGPU(Vec xin,NormType type,PetscReal* z)
{*/

/* NormType: NORM_1=0,NORM_2=1,NORM_FROBENIUS=2,NORM_INFINITY=3,NORM_1_AND_2=4 */
/* dealing with NORM_2 for now... */
/* z has 2 elements */

/*
  PetscErrorCode ierr;
  PetscFunctionBegin;
  printf("Call to VecNorm_SeqGPU\n");
  ierr = VecDot_SeqGPU(xin,xin,&z[0]);CHKERRQ(ierr);
  z[0]=PetscSqrtScalar(z[0]);
  printf("ZNORM: %f\n\n",*z);
  PetscFunctionReturn(0);
}*/


/*
#undef __FUNCT__
#define __FUNCT__ "kernReduceAbsSum"
PetscErrorCode kernReduceAbsSum(double * x, PetscReal* z){

}
*/
/* ------------------------------ end norms -------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecGetArray_SeqGPU"
PetscErrorCode VecGetArray_SeqGPU(Vec v,PetscScalar **a){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *vd=(Vec_SeqGPU*)v->data;
  if(vd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecGetArray_SeqGPU the vector has not been created.");
  }
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecGetArray_SeqGPU\n");
  #endif
  PetscInt flg1=0,flg2=0;
  PetscStackCheckByName(4,"DMDAVecGetArray",flg1);
  PetscStackCheckByName(6,"DMGlobalToLocalBegin",flg2);

  if((flg1 || flg2) && vd->syncState==VEC_GPU){
    ierr = VecCopyOverD2H(v,vd->cpuptr); CHKERRQ(ierr);
    vd->syncState = VEC_CPU;
  }
  hipDeviceSynchronize();
  *a=vd->cpuptr;
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecRestoreArray_SeqGPU"
PetscErrorCode VecRestoreArray_SeqGPU(Vec v,PetscScalar **a){
  PetscFunctionBegin;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecRestoreArray_SeqGPU\n");
  #endif
  PetscErrorCode ierr;
  Vec_SeqGPU *vd=(Vec_SeqGPU*)v->data;
  PetscInt flg1=0;
  PetscStackCheckByName(1,"VecRestoreArrayRead",flg1);
  if(!flg1){
    if(a){
      ierr = VecCopyOverH2D(v,*a);CHKERRQ(ierr);
      vd->syncState=VEC_GPU;
    }else{
      ierr = VecCopyOverH2D(v,vd->cpuptr);CHKERRQ(ierr);
      vd->syncState=VEC_SYNCHED;
    }
  }
  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCreateSeqGPU"
PetscErrorCode  VecCreateSeqGPU(MPI_Comm comm,PetscInt n,Vec *v){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  ierr = VecCreate(comm,v);CHKERRQ(ierr);
  ierr = VecSetSizes(*v,n,n);CHKERRQ(ierr);
  ierr = VecSetType(*v,VECSEQGPU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCopy_SeqGPU"
PetscErrorCode VecCopy_SeqGPU(Vec s,Vec d){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *sd=(Vec_SeqGPU*)s->data;
  Vec_SeqGPU *dd=(Vec_SeqGPU*)d->data;
  if(d->map->n!=s->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(dd->syncState==VEC_UNALLOC){
     SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Destination vector unalloced.");
  }
  if(sd->syncState==VEC_ALLOC){
      PetscFunctionReturn(0);/* nothing to do */
  }
  if(sd->syncState==VEC_CPU){
    ierr = PetscMemcpy((void*)dd->cpuptr,(void*)sd->cpuptr,s->map->n*sizeof(PetscScalar));CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }
  ierr = VecCopyOverDevice(d,s); CHKERRQ(ierr);
  dd->syncState=sd->syncState;/* synch signal copy */
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecCopy_SeqGPU\n");
  #endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecSwap_SeqGPU"
PetscErrorCode VecSwap_SeqGPU(Vec xin,Vec yin){
  /* PetscErrorCode ierr; */
  PetscFunctionBegin;
  printf("VecSwap_SeqGPU (***EMPTY***)\n");
  if (xin != yin) {
#if defined(PETSC_USE_REAL_SINGLE)
    //////// hipblasSswap(bn,VecCUSPCastToRawPtr(*xarray),one,VecCUSPCastToRawPtr(*yarray),one);
#else
    //////   hipblasDswap(bn,VecCUSPCastToRawPtr(*xarray),one,VecCUSPCastToRawPtr(*yarray),one);
#endif

  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDuplicate_SeqGPU"
PetscErrorCode VecDuplicate_SeqGPU(Vec win,Vec *V){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecDuplicate_SeqGPU\n");
  #endif
  ierr = VecCreate(((PetscObject)win)->comm,V);CHKERRQ(ierr);
  ierr = VecSetType(*V,VECSEQGPU);CHKERRQ(ierr);
  ierr = PetscObjectSetPrecision((PetscObject)*V,((PetscObject)win)->precision);CHKERRQ(ierr);
  ierr = VecSetSizes(*V,win->map->n,win->map->N);CHKERRQ(ierr);
  ierr = PetscLayoutReference(win->map,&(*V)->map);CHKERRQ(ierr);
  ierr = PetscOListDuplicate(((PetscObject)win)->olist,&((PetscObject)(*V))->olist);CHKERRQ(ierr);
  ierr = PetscFListDuplicate(((PetscObject)win)->qlist,&((PetscObject)(*V))->qlist);CHKERRQ(ierr);
  (*V)->stash.ignorenegidx = win->stash.ignorenegidx;
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDuplicateVecs_SeqGPU"
PetscErrorCode VecDuplicateVecs_SeqGPU(Vec vin, PetscInt m, Vec **Vlist){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt i=0;
#if(DEBUGVEC && VERBOSE)
     printf("Call to VecDuplicateVecs_SeqGPU\n"); 
  #endif
  PetscValidHeaderSpecific(vin,VEC_CLASSID,1);
  PetscValidPointer(Vlist,3);
  if (m <= 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_OUTOFRANGE,"m must be > 0: m = %D",m);
  ierr = PetscMalloc(m*sizeof(Vec),Vlist);CHKERRQ(ierr);
  for(i=0;i<m;i++){
    ierr = VecDuplicate_SeqGPU(vin,*Vlist+i);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecPlaceArray_SeqGPU"
PetscErrorCode  VecPlaceArray_SeqGPU(Vec x,const PetscScalar* array){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecPlaceArray_SeqGPU\n"); 
  #endif
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecPlaceArray_SeqGPU the vector has not been created.");
  }
  if(xd->unplacedarray){
     SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE,
       "VecPlaceArray() was already called on this vector, without a call to VecResetArray()");
  }
  if(xd->syncState==VEC_GPU){/* assuming there is a logical reason for this copy up */
    ierr = VecCopyOverD2H(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  xd->unplacedarray=xd->cpuptr;
  xd->cpuptr=(PetscScalar*)array;
  ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
  xd->syncState=VEC_SYNCHED;
  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "VecResetArray_SeqGPU"
PetscErrorCode  VecResetArray_SeqGPU(Vec x){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
#if(DEBUGVEC && VERBOSE)
     printf("Call to VecResetArray_SeqGPU\n"); 
  #endif
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecResetArray_SeqGPU the vector has not been created.");
  }
  if(xd->cpuptr){
    ierr = PetscFree(xd->cpuptr);CHKERRQ(ierr);
  }
  xd->cpuptr=xd->unplacedarray;
  xd->unplacedarray=PETSC_NULL;
  ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
  xd->syncState=VEC_SYNCHED;
  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecReplaceArray_SeqGPU"
PetscErrorCode  VecReplaceArray_SeqGPU(Vec x,const PetscScalar* array){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
#if(DEBUGVEC && VERBOSE)
     printf("Call to VecReplaceArray_SeqGPU\n"); 
  #endif
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecResetArray_SeqGPU the vector has not been created.");
  }
  if(xd->cpuptr){
    ierr = PetscFree(xd->cpuptr);CHKERRQ(ierr);
  }
  xd->cpuptr=(PetscScalar*)array;
  ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
  xd->syncState=VEC_SYNCHED;
  hipDeviceSynchronize();
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "PinnedMalloc"
static PetscErrorCode  PinnedMalloc(PetscScalar** x,PetscInt n){
  PetscFunctionBegin;
#if(DEBUGVEC && VERBOSE)
     printf("Call to PinnedMalloc\n"); 
  #endif
  cms[0]=hipHostAlloc((void**)x,n,0);
  #if(DEBUGVEC)
     PetscErrorCode ierr;
     ierr=VecCheckCUDAStatus(cms[0],"in PinnedMalloc");CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}

#undef __FUNCT__
#define __FUNCT__ "PinnedFree"
static PetscErrorCode  PinnedFree(PetscScalar* x){
  PetscFunctionBegin;
#if(DEBUGVEC && VERBOSE)
     printf("Call to PinnedFree\n"); 
  #endif
  cms[0]=hipHostFree(x);
  #if(DEBUGVEC)
    PetscErrorCode ierr;
    ierr=VecCheckCUDAStatus(cms[0],"in PinnedFree");CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCreate_SeqGPU"
PetscErrorCode  VecCreate_SeqGPU(Vec V){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscMPIInt    size;
  Vec_SeqGPU* seqgpu=PETSC_NULL;
  ierr = PetscMalloc(sizeof(Vec_SeqGPU),&seqgpu);
  V->data=(void*)seqgpu;
  ierr = MPI_Comm_size(((PetscObject)V)->comm,&size);CHKERRQ(ierr);
  if  (size > 1) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Cannot create VECSEQGPU on more than one process");

  if (V->map->bs == -1) V->map->bs = 1;
  ierr = PetscLayoutSetUp(V->map);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)V,VECSEQGPU);CHKERRQ(ierr);

  V->ops->dot             = VecDot_SeqGPU;
  V->ops->norm            = VecNorm_SeqGPU;
  V->ops->tdot            = VecTDot_SeqGPU;
  V->ops->scale           = VecScale_SeqGPU;
  V->ops->copy            = VecCopy_SeqGPU;
  V->ops->set             = VecSet_SeqGPU;
  V->ops->setvalues       = VecSetValues_SeqGPU;
  V->ops->swap            = VecSwap_SeqGPU;
  V->ops->axpy            = VecAXPY_SeqGPU;
  V->ops->axpby           = VecAXPBY_SeqGPU;
  V->ops->axpbypcz        = VecAXPBYPCZ_SeqGPU;
  V->ops->pointwisemult   = VecPointwiseMult_SeqGPU;
  V->ops->pointwisedivide = VecPointwiseDivide_SeqGPU;
  V->ops->maxpointwisedivide = VecMaxPointwiseDivide_SeqGPU;
  V->ops->setrandom       = VecSetRandom_SeqGPU;
  V->ops->dot_local       = VecDot_SeqGPU;
  V->ops->tdot_local      = VecTDot_SeqGPU;
  V->ops->norm_local      = VecNorm_SeqGPU;
  V->ops->maxpy           = VecMAXPY_SeqGPU;
  V->ops->mdot            = VecMDot_SeqGPU;
  /* V->ops->aypx            = VecAYPX_SeqGPU; */
  V->ops->waxpy           = VecWAXPY_SeqGPU;
  V->ops->dotnorm2        = VecDotNorm2_SeqGPU;
  V->ops->placearray      = VecPlaceArray_SeqGPU;
  V->ops->replacearray    = VecReplaceArray_SeqGPU;
  V->ops->resetarray      = VecResetArray_SeqGPU;
  V->ops->destroy         = VecDestroy_SeqGPU;
  V->ops->destroyvecs     = VecDestroyVecs_SeqGPU;
  V->ops->duplicate       = VecDuplicate_SeqGPU;
  V->ops->duplicatevecs   = VecDuplicateVecs_SeqGPU;
  V->ops->getarray        = VecGetArray_SeqGPU;
  V->ops->restorearray    = VecRestoreArray_SeqGPU;
  V->ops->getlocalsize    = VecGetLocalSize_SeqGPU;
  V->ops->getsize         = VecGetSize_SeqGPU;
  V->ops->view            = VecView_SeqGPU;
  V->petscnative=PETSC_FALSE;
  seqgpu->syncState      = VEC_UNALLOC;
  seqgpu->unplacedarray=PETSC_NULL;
  seqgpu->array_allocated=PETSC_NULL;
  seqgpu->array=PETSC_NULL;
  /* create an associated stream */
  cms[0] = hipStreamCreate(&(seqgpu->stream));
  /* allocate the variable for vector size */
  cms[1]=hipMalloc((void**)&(seqgpu->length),sizeof(int));
  /* send vec length size to device */
  ccs[0]=hipMemcpyAsync((void*)seqgpu->length,
               (void*)&(V->map->n),sizeof(int),hipMemcpyHostToDevice,seqgpu->stream);
  /* allocate the vector on device */
  cms[2]=hipMalloc((void**)&(seqgpu->devptr),V->map->n*sizeof(double));
  ccs[1]=hipMemsetAsync((void*)seqgpu->devptr,0,V->map->n*sizeof(double),seqgpu->stream);
  /* allocate the variable for vector offsets */
  cms[3]=hipMalloc((void**)&(seqgpu->offset),sizeof(int));
  /* allocate the variable for vector segment length */
  cms[4]=hipMalloc((void**)&(seqgpu->segment),sizeof(int));
  /* allocate the variable for vector single value result */
  cms[5]=hipMalloc((void**)&(seqgpu->zval),sizeof(double));
  cms[6]=hipMalloc((void**)&(seqgpu->scalar),sizeof(double));
  /* using pinned memory */
  ierr = PinnedMalloc(&(seqgpu->cpuptr),V->map->n*sizeof(PetscScalar));CHKERRQ(ierr);
  //ierr = PetscMalloc(V->map->n*sizeof(PetscScalar),&(seqgpu->cpuptr));
  ierr = PetscMemzero(seqgpu->cpuptr,V->map->n*sizeof(PetscScalar));CHKERRQ(ierr);
  seqgpu->syncState=VEC_ALLOC;


  #if(DEBUGVEC)
    #if(VERBOSE)
       printf("Call to VecCreate_SeqGPU\n");
    #endif
    ierr = VecCheckCUDAStatus(cms[0],"on hipStreamCreate VecCreate_SeqGPU");  CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[1],"Alloc devlength in VecCreate_SeqGPU");   CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[0],"Copy H2D devlength in VecCreate_SeqGPU");CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[2],"Alloc of devptr in VecCreate_SeqGPU");   CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(ccs[1],"on device cudaMemSet VecCreate_SeqGPU"); CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[3],"Alloc devoffset in VecCreate_SeqGPU");   CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[4],"Alloc dev segment in VecCreate_SeqGPU"); CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[5],"Alloc dev zval in VecCreate_SeqGPU");    CHKERRQ(ierr);
    ierr = VecCheckCUDAStatus(cms[6],"Alloc dev scalar in VecCreate_SeqGPU");    CHKERRQ(ierr);
  #endif
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "VecDestroy_SeqGPU"
PetscErrorCode VecDestroy_SeqGPU(Vec v){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
#if(DEBUGVEC && VERBOSE)
     printf("Call to VecDestroyArray_SeqGPU\n"); 
  #endif
  PetscValidHeaderSpecific(v,VEC_CLASSID,1);
  if(vd && vd->syncState != VEC_UNALLOC){
      cms[0]=hipFree(vd->devptr);  vd->devptr=PETSC_NULL;
      cms[1]=hipFree(vd->length);  vd->length=PETSC_NULL;
      cms[2]=hipFree(vd->segment); vd->segment=PETSC_NULL;
      cms[3]=hipFree(vd->zval);    vd->zval=PETSC_NULL;
      cms[4]=hipFree(vd->scalar);  vd->scalar=PETSC_NULL;
      cms[5] = hipStreamDestroy(vd->stream);
      ierr = PinnedFree(vd->cpuptr); CHKERRQ(ierr);
      //ierr = PetscFree(vd->cpuptr);CHKERRQ(ierr);
      #if(DEBUGVEC)
        ierr=VecCheckCUDAStatus(cms[0],"destroying devptr in VecDestroy_SeqGPU"); CHKERRQ(ierr);
        ierr=VecCheckCUDAStatus(cms[1],"destroying length in VecDestroy_SeqGPU"); CHKERRQ(ierr);
        ierr=VecCheckCUDAStatus(cms[2],"destroying segment in VecDestroy_SeqGPU");CHKERRQ(ierr);
        ierr=VecCheckCUDAStatus(cms[3],"destroying zval in VecDestroy_SeqGPU");   CHKERRQ(ierr);
        ierr=VecCheckCUDAStatus(cms[4],"destroying scalar in VecDestroy_SeqGPU"); CHKERRQ(ierr);
        ierr=VecCheckCUDAStatus(cms[5],"destroying stream in VecDestroy_SeqGPU"); CHKERRQ(ierr);
      #endif
      vd->syncState = VEC_UNALLOC;
  }
  ierr = PetscObjectDepublish(v);CHKERRQ(ierr);
#if defined(PETSC_USE_LOG)
  PetscLogObjectState((PetscObject)v,"Length=%D",v->map->n);
#endif
  ierr = PetscFree(v->data);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecDestroyVecs_SeqGPU"
PetscErrorCode  VecDestroyVecs_SeqGPU(PetscInt m,Vec *vv){
  PetscFunctionBegin;
  #if(DEBUGVEC && VERBOSE)
     printf("Call to VecDestroyVecs_SeqGPU\n");
  #endif
  PetscErrorCode ierr;
  PetscInt i;
   /* destroy the internal part */
  for(i=0;i<m;i++){
    ierr = VecDestroy(&vv[i]);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}

#undef __FUNCT__  
#define __FUNCT__ "VecView_Seq_ASCII"
static PetscErrorCode VecView_Seq_ASCII(Vec xin,PetscViewer viewer){
  PetscFunctionBegin;
  printf("VecView_Seq_ASCII() (***EMPTY***)\n");
  PetscFunctionReturn(0);
}

EXTERN_C_END

