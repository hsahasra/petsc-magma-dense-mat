#include "hip/hip_runtime.h"
#include <petscconf.h>
#include <petscsys.h>
PETSC_CUDA_EXTERN_C_BEGIN
#include <string.h>
#include <stdlib.h>
#include <float.h>
#include <private/vecimpl.h>          /*I "petscvec.h" I*/
#include <../src/vec/vec/impls/dvecimpl.h>
#include <../src/vec/vec/impls/seq/seqgpu/gpuvecimpl.h>


PETSC_CUDA_EXTERN_C_END


EXTERN_C_BEGIN

__constant__ int     integerSymbol;
__constant__ int2    integer2Symbol;
__constant__ int3    integer3Symbol;
__constant__ int     devN;//vector length
__constant__ double  dblScalarValue;//utility var
__constant__ double2 dblScalar2Value;//utility var
__constant__ float   fltScalarValue;//utility var
__constant__ float2  fltScalar2Value;//utility var



/* Valid pointer check function (probably doesn't work) */
PetscBool valid(void *p){
  extern char _etext;
  if((p != PETSC_NULL) && ((char*) p > &_etext)){
    return PETSC_TRUE;
  }else{
    return PETSC_FALSE;
  }
}




/* ---------------------------------------------------------
// helper function for error checking
// pops the CUDA error stack and exits on nonzero error code
// written by: dlowell ANL-MCS
// --------------------------------------------------------- */
#undef __FUNCT__
#define __FUNCT__ "VecCheckCUDAError"
PetscErrorCode VecCheckCUDAError(const char *msg){

  PetscFunctionBegin;
  hipError_t err = hipGetLastError();
  if( hipSuccess != err){
    fprintf(stderr, "Cuda error: %s: %s.\n", msg,hipGetErrorString(err));
    fflush(NULL);
    PetscFunctionReturn(PETSC_ERR_LIB);
  }
  PetscFunctionReturn(0);
}


#undef __FUNCT__
#define __FUNCT__ "VecCheckCUDAStatus"
PetscErrorCode VecCheckCUDAStatus(hipError_t cs,const char *msg){
  PetscFunctionBegin;
    if(cs!=hipSuccess){
      fprintf(stderr, "Cuda error!: %s: %s.\n",msg,hipGetErrorString(cs));
      fflush(NULL);
      PetscFunctionReturn(PETSC_ERR_LIB);
    }
    PetscFunctionReturn(0);
}






/* ****************************************************************************
 * This is a shared memory implementation that keeps the full 625 words of state
 * in shared memory. Faster for heavy random work where you can afford 
 *  the shared memory. */
/* Init by single seed - single threaded as only used once */
__device__ void mt19937si(uint seed){
    int	i;
    if(threadIdx.x == 0){
	mtNexts = 0;
	s_seeds[0] = seed;
	for(i = 1;i < NNN;i++){
	   seed = (INIT_MULT * (seed^(seed >> 30))+i);
	   s_seeds[i] = seed;
	}
    }
    __syncthreads();/* Ensure mtNexts set */
    return;
}


/* Init by array - single threaded as only used once */
__device__ void mt19937sai(uint* seeds,uint length){
    int i,j,k;
    mt19937si(ARRAY_SEED);
    if(threadIdx.x==0){
     i=1; j=0;
     for(k = NNN>length?NNN:length;k!=0;k--){
        s_seeds[i] = (s_seeds[i]^((s_seeds[i-1]^(s_seeds[i-1] >> 30))*1664525)) + seeds[j] + j;
	if(++i >= NNN){
          s_seeds[0] = s_seeds[NNN-1];
	  i = 1;
        }
        if(++j>=length)j = 0;
     }
     for(k = NNN-1; k!=0;k--){
       s_seeds[i] = (s_seeds[i] ^ ((s_seeds[i-1]^(s_seeds[i-1]>>30))*1566083941))-i;
       if(++i >= NNN){
         s_seeds[0] = s_seeds[NNN-1];
	 i=1;
       }
     }
     s_seeds[0] = 0x80000000;/* MSB is 1; assuring non-zero initial array */ 
    }
    __syncthreads();				/* Needed for mt19937w() */
    return;
}


/* Return next MT random by increasing thread ID for 1-227 threads. */
__device__ uint mt19937s(void){
    int		kk;
    uint	x;
    uint	y;
    int		tid = threadIdx.x;

    kk = (mtNexts + tid) % NNN;
    __syncthreads();				/* Finished with mtNexts */

    if (tid == blockDim.x - 1)mtNexts = kk + 1;			/* Will get modded on next call */
    x = s_seeds[kk] & UPPER_MASK;
    if(kk < NNN - MMM){
      x |= (s_seeds[kk+1]&LOWER_MASK);
      y = s_seeds[kk+MMM];
    }else if(kk < NNN-1){
      x |= (s_seeds[kk+1]&LOWER_MASK);
      y = s_seeds[kk + (MMM-NNN)];
    }else{					/* kk == N - 1 */
      x |= (s_seeds[0]&LOWER_MASK);
      y = s_seeds[MMM - 1];
    }
    y ^= x >> 1;
    if (x & 1)y ^= MATRIX_A;
    __syncthreads();				/* All done before we update */

    s_seeds[kk] = y;
    y ^= (y >> 11);				/* Tempering */
    y ^= (y <<  7) & TEMPER1;
    y ^= (y << 15) & TEMPER2;
    y ^= (y >> 18);
    return y;
}




/* General shared memory version for any number of threads.
 * Note only up to 227 threads are run at any one time,
 * the rest loop and block till all are done. */
__device__ uint mt19937sl(void){
  int jj,kk,tid;
  uint x,y;
  tid = threadIdx.x;
  kk = (mtNexts + tid) % NNN;
  __syncthreads();				/* Finished with mtNexts */

  if(tid == blockDim.x - 1)mtNexts = kk + 1;	/* Will get modded on next call */
  jj = 0;
  do{
    if(jj <= tid && tid < jj + NNN - MMM){
      x = s_seeds[kk] & UPPER_MASK;
      if(kk < NNN - MMM){
         x |= (s_seeds[kk+1]&LOWER_MASK);
	 y = s_seeds[kk + MMM];
      }else if (kk < NNN-1){
         x |= (s_seeds[kk + 1]&LOWER_MASK);
	 y = s_seeds[kk + (MMM-NNN)];
      }else{				/* kk == N - 1 */
         x |= (s_seeds[0]&LOWER_MASK);
         y = s_seeds[MMM-1];
      }

      y ^= x >> 1;
      if(x & 1) y ^= MATRIX_A;
    }
    __syncthreads();			/* All done before we update */
    if(jj <= tid && tid < jj+NNN-MMM) s_seeds[kk] = y;
    __syncthreads();

  }while ((jj += NNN-MMM) < blockDim.x);

  y ^= (y >> 11);				/* Tempering */
  y ^= (y <<  7) & TEMPER1;
  y ^= (y << 15) & TEMPER2;
  y ^= (y >> 18);
  return y;
}


#undef __FUNCT__
#define __FUNCT__ "kernRandS"
__global__ void kernRandS(uint* seeds){
  mt19937sai(seeds,gridDim.x);
}





#undef __FUNCT__
#define __FUNCT__ "kernRand"
__global__ void kernRand(double *x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  uint rval;
  if(tid<*n){
    rval = mt19937sl();
    x[tid] = ((double)rval/(double)UINT_MAX);
    /* printf("RAND value[%d]: %0.13f, rval: %u UINT_MAX: %u\n",
       tid,x[tid],rval,UINT_MAX); */
  }
}




#undef __FUNCT__
#define __FUNCT__ "VecSetRandom_SeqGPU"
PetscErrorCode VecSetRandom_SeqGPU(Vec x,PetscRandom r){
  PetscFunctionBegin;
  static PetscBool seed_flag=PETSC_TRUE;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  PetscInt i,bx,tx;
  uint *seeds=PETSC_NULL,*devseeds=PETSC_NULL;
  PetscScalar rval;
  dim3 dimBlock,dimGrid;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  /* assuming width mem load isn't going to be an issue */

  printf("Call to VecSetRandom_SeqGPU\n");
  if(xd->syncState==VEC_ALLOC || xd->syncState==VEC_CPU){
    for(i=0; i<x->map->n; i++){
       ierr = PetscRandomGetValue(r,&xd->cpuptr[i]);CHKERRQ(ierr);
    }
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }else if(xd->syncState==VEC_SYNCHED || xd->syncState==VEC_GPU){
    bx=ceil((float)x->map->n/(float)TCOUNT);
    ierr = PetscMalloc(bx*sizeof(PetscInt),&seeds);CHKERRQ(ierr);
    tx=TCOUNT;
    dimGrid.x=bx; dimGrid.y=1;
    dimBlock.x=tx; dimBlock.y=1;
    if(seed_flag){
      for(i=0; i<bx; i++){
         ierr = PetscRandomGetValue(r,&rval);CHKERRQ(ierr);
         seeds[i]=(uint)(UINT_MAX*rval);
      }

      cudastatus = hipMalloc((void**)&devseeds,bx*sizeof(uint));
      ierr = VecCheckCUDAStatus(cudastatus,"error in hipMalloc");CHKERRQ(ierr);

      cudastatus=hipMemcpy(devseeds,seeds,bx*sizeof(uint),hipMemcpyHostToDevice);
      ierr = VecCheckCUDAStatus(cudastatus,"on copy H2D in VecSetRandom_SeqGPU");CHKERRQ(ierr);
      xd->vstat.h2d_count++;
      xd->vstat.h2d_bytes+=bx*sizeof(uint);

      kernRandS<<<dimGrid,dimBlock>>>(devseeds);
      ierr = VecCheckCUDAError("kernRandS launch");CHKERRQ(ierr);
      ierr = PetscFree(seeds);CHKERRQ(ierr);
      hipDeviceSynchronize();
      cudastatus = hipFree(devseeds);
      ierr = VecCheckCUDAStatus(cudastatus,"in hipFree()");CHKERRQ(ierr);
      seed_flag=PETSC_FALSE;
    }


    kernRand<<<dimGrid,dimBlock>>>(xd->devptr,xd->length);
    ierr = VecCheckCUDAError("kernRand launch");CHKERRQ(ierr);
  }
  xd->syncState = VEC_GPU;
  PetscFunctionReturn(0);
}


/*

#undef __FUNCT__
#define __FUNCT__ "VecSetRandom_SeqGPU"
PetscErrorCode VecSetRandom_SeqGPU(Vec x,PetscRandom r){
  PetscErrorCode ierr;
  PetscFunctionBegin;
  ierr = VecSetRandom_Seq(x,r);CHKERRQ(ierr);
  printf("Call to VecSetRandom_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}
*/




#undef __FUNCT__
#define __FUNCT__ "VecCompare_SeqGPU"
PetscErrorCode VecCompare_SeqGPU(Vec x, Vec y, PetscBool *same, PetscInt offset, PetscInt blocksize){
  PetscFunctionBegin;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  Vec_SeqGPU* yd = (Vec_SeqGPU*)y->data;
  if(xd->syncState!=yd->syncState||xd->syncState==VEC_ALLOC||yd->syncState==VEC_ALLOC){
    *same=PETSC_FALSE;
    PetscFunctionReturn(0);
  }
  PetscErrorCode ierr;
  int blocks,threads;/* assuming shared memory size is not an issue */
  if(blocksize && !offset){
    blocks=ceil((float)blocksize/(float)TCOUNT);
  } else {
    blocks=ceil((float)x->map->n/(float)TCOUNT);
    blocksize = x->map->n;
  }
  threads=TCOUNT;
  hipError_t cudastatus;
  int *devsame=PETSC_NULL;
  int cpusame=0;
  int2 offset_bsize;
  offset_bsize.x = offset;
  offset_bsize.y = blocksize;


  if(xd->syncState==VEC_CPU && yd->syncState==VEC_CPU){
    ierr = PetscMemcmp((void*)&xd->cpuptr[offset],(void*)&yd->cpuptr[offset],blocksize,same);CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }

  cudastatus = hipMalloc((void**)&devsame,sizeof(int));
  ierr = VecCheckCUDAStatus(cudastatus,"error in device malloc");CHKERRQ(ierr);

  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("integer2Symbol"),(void*)&offset_bsize,sizeof(int2),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);

  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("devN"),(void*)&x->map->n,sizeof(int),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);

  dim3 dimGrid; dimGrid.x=blocks; dimGrid.y=1;
  dim3 dimBlock; dimBlock.x=threads; dimBlock.y=1;
  kernCompare<<<dimGrid,dimBlock>>>(xd->devptr,yd->devptr,xd->length,yd->length,devsame);
  ierr = VecCheckCUDAError("kernCompare launch");CHKERRQ(ierr);

  cudastatus=hipMemcpy(&cpusame,devsame,sizeof(int),hipMemcpyDeviceToHost);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H in VecCompare_SeqGPU");CHKERRQ(ierr);

  if(cpusame==1)*same=PETSC_TRUE;
  else *same=PETSC_FALSE;
  cudastatus = hipFree(devsame);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}





__global__ void kernCompare(double* devX, double* devY, int* lx, int* ly, int* devsame){

  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int2 localOBS = integer2Symbol;
  int localn = localOBS.x+localOBS.y;
  int index = tid+localOBS.x;
  double value=0;
  __shared__ unsigned char blockflag;
  __shared__ double chunkX[TCOUNT];
  __shared__ double chunkY[TCOUNT];

  if(threadIdx.x==0)blockflag=0;
  __syncthreads();
  if(index<localn){
    /* read in values to shared */
    chunkX[threadIdx.x]=devX[index];
    chunkY[threadIdx.x]=devY[index];
    value = fabs(chunkX[threadIdx.x]-chunkY[threadIdx.x]);
    if(value>1e-16){
      //printf("In kernCompare found an element mismatch: %e\n",value);
      blockflag=1;
    }
    if(*lx!=*ly){
      //printf("In kernCompare found length mismatch: lx: %d vs ly: %d\n",*lx,*ly);
      blockflag=1;
    }
  }
  __syncthreads();
  if(threadIdx.x==0){
    if(blockflag)*devsame=0;
    else *devsame=1;
  }
  return;
}





/*

#undef __FUNCT__
#define __FUNCT__ "VecResetArray_Seq"
static PetscErrorCode VecResetArray_Seq(Vec vin){
  //Vec_Seq *v = (Vec_Seq *)vin->data;

  PetscFunctionBegin;
  printf("Call to VecResetArray_Seq\n");
  PetscFunctionReturn(0);
}

*/





#undef __FUNCT__
#define __FUNCT__ "VecSetValues_SeqGPU"
/*@
   VecSetValues - Inserts or adds values into certain locations of a vector.
@*/
PetscErrorCode VecSetValues_SeqGPU(Vec x,PetscInt ni,const PetscInt ix[],const PetscScalar y[],InsertMode iora){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt i;
  PetscScalar yval=0;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  //printf("Call to VecSetValues_SeqGPU\n");
  if(xd->syncState==VEC_CPU || xd->syncState==VEC_SYNCHED){
    if(iora==INSERT_VALUES){
      for(i=0;i<ni;i++){
         yval = y[i];
         xd->cpuptr[i]=yval;
      }
      ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
      xd->syncState=VEC_SYNCHED;
    }else{
      /* ADD_VALUES not supported now */
      printf("Call to VecSetValues_SeqGPU: ADD_VALUES (*** EMPTY ***)\n");
    }
  }else{
      if(iora==INSERT_VALUES){/* not efficient */
        for(i=0;i<ni;i++){
          yval=y[i];
          ierr = VecCopyBlockH2D(x,&yval,ix[i],1);CHKERRQ(ierr);
        }
      }
      xd->syncState=VEC_GPU;
  }
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "VecView_SeqGPU"
PetscErrorCode VecView_SeqGPU(Vec x,PetscViewer viewer){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  if(xd->syncState==VEC_GPU){
    ierr = VecCopyOverD2H(x,xd->cpuptr); CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  ierr= PetscViewerSetType(viewer,PETSCVIEWERASCII);CHKERRQ(ierr);/* forced ASCII for now */
  ierr =VecView_Seq_ASCII(x,viewer);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "VecGetLocalSize_SeqGPU"
PetscErrorCode VecGetLocalSize_SeqGPU(Vec x, PetscInt *localsize){
  PetscFunctionBegin;
  /* printf("Call to VecGetLocalSize_SeqGPU\n"); */
  PetscValidHeaderSpecific(x,VEC_CLASSID,1);
  PetscValidIntPointer(localsize,2);
  PetscValidType(x,1);
  *localsize=x->map->n;
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "VecGetSize_SeqGPU"
PetscErrorCode VecGetSize_SeqGPU(Vec x, PetscInt *globalsize){
  PetscFunctionBegin;
  /* printf("Call to VecGetSize_SeqGPU\n"); */
  PetscValidHeaderSpecific(x,VEC_CLASSID,1);
  PetscValidIntPointer(globalsize,2);
  PetscValidType(x,1);
  *globalsize=x->map->N;
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockDevice"
PetscErrorCode VecCopyBlockDevice(Vec d, Vec s, PetscInt doffset, PetscInt soffset, PetscInt blocksize){
  PetscFunctionBegin;
  printf("Call to VecCopyBlockDevice (**** EMPTY ****)\n");
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "VecCopyOverDevice"
PetscErrorCode VecCopyOverDevice(Vec d,Vec s){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt bx, tx;
  Vec_SeqGPU* dd = (Vec_SeqGPU*)d->data;
  Vec_SeqGPU* sd = (Vec_SeqGPU*)s->data;
  //printf("Call to VecCopyOverDevice\n");
  dim3 dimGrid;
  dim3 dimBlock;

  if(s->map->n!=d->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }

  if(sd->syncState==VEC_CPU){/* synch y */
    ierr = VecCopyOverH2D(s,sd->cpuptr);CHKERRQ(ierr);
    sd->syncState=VEC_SYNCHED;
  }

  /* assuming width mem load isn't going to be an issue */
  bx=ceil((float)d->map->n/(float)CPYTCOUNT);
  tx=CPYTCOUNT;
  dimGrid.x=bx; dimGrid.y=1;
  dimBlock.x=tx; dimBlock.y=1;
  kernCODevice<<<dimGrid,dimBlock>>>(dd->devptr,sd->devptr,sd->length);
  ierr = VecCheckCUDAError("kernel call to kernCODevice");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "kernCODevice"
__global__ void kernCODevice(double* devY,double* devX, int *n){

  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunkX[CPYTCOUNT];
  if(tid<*n){
    //if(devX[tid]!=0)printf("devX[%d]: %e, len: %d\n",tid,devX[tid],*n);
    chunkX[threadIdx.x]=devX[tid];
    devY[tid]=chunkX[threadIdx.x];
    //if(devY[tid]!=0)printf("devY[%d]: %e, len: %d\n",tid,devY[tid],*n);
  }
}


#undef __FUNCT__
#define __FUNCT__ "kernCopyLen"
__global__ void kernCopyLen(int* ly,int* lx){
  if(threadIdx.x==0)*ly=*lx;
}




#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockH2D"
PetscErrorCode VecCopyBlockH2D(Vec v,PetscScalar *y, PetscInt offset, PetscInt blocksize){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  /* printf("Call to VecCopyBlockH2D\n"); */
  cudastatus=hipMemcpy(&(vd->devptr[offset]),y,blocksize*sizeof(PetscScalar),hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy H2D in VecCopyBlockH2D");CHKERRQ(ierr);
  vd->vstat.h2d_count++;
  vd->vstat.h2d_bytes+=blocksize*sizeof(PetscScalar);
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "VecCopyOverH2D"
PetscErrorCode VecCopyOverH2D(Vec v,PetscScalar *y){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  /* printf("Call to VecCopyOverH2D\n"); */
  cudastatus=hipMemcpy(vd->devptr,y,v->map->n*sizeof(PetscScalar),hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy H2D in VecCopyOverH2D");CHKERRQ(ierr);
  vd->vstat.h2d_count++;
  vd->vstat.h2d_bytes+=v->map->n*sizeof(PetscScalar);
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "VecCopyBlockD2H"
PetscErrorCode VecCopyBlockD2H(Vec v,PetscScalar *y,PetscInt offset, PetscInt blocksize){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  /* printf("Call to VecCopyBlockD2H\n"); */
  cudastatus=hipMemcpy(y,&(vd->devptr[offset]),blocksize*sizeof(PetscScalar),hipMemcpyDeviceToHost);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H in VecCopyBlockD2H");CHKERRQ(ierr);
  vd->vstat.d2h_count++;
  vd->vstat.d2h_bytes+=blocksize*sizeof(PetscScalar);
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "VecCopyOverD2H"
PetscErrorCode VecCopyOverD2H(Vec v,PetscScalar *y){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  /* printf("Call to VecCopyOverD2H\n"); */
  cudastatus=hipMemcpy(y,vd->devptr,v->map->n*sizeof(PetscScalar),hipMemcpyDeviceToHost);
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H in VecCopyOverD2H");CHKERRQ(ierr);
  vd->vstat.d2h_count++;
  vd->vstat.d2h_bytes+=v->map->n*sizeof(PetscScalar);
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "VecSet_SeqGPU"
PetscErrorCode VecSet_SeqGPU(Vec xin,PetscScalar alpha){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  //PetscInt i=0;
  hipError_t cudastatus;
  dim3 dimgrid(ceil((float)xin->map->n/((float)TCOUNT)),1,1);
  dim3 dimblocks(TCOUNT,1,1);
  Vec_SeqGPU* xd = (Vec_SeqGPU*)xin->data;
  //printf("Call to VecSet_SeqGPU\n");
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,
            PETSC_ERR_MEM,"*** In VecSet_SeqGPU, Vec not allocated. ***\n");
  }else{
    if(alpha==0){
      cudastatus=hipMemset((void*)xd->devptr,0,xin->map->n*sizeof(double));
      ierr = VecCheckCUDAStatus(cudastatus,"on device cudaMemSet VecSet_SeqGPU");CHKERRQ(ierr);
    }else{
      cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha,sizeof(double),0,hipMemcpyHostToDevice);
      ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);
      kernSet<<<dimgrid,dimblocks>>>(xd->devptr,xd->length);
      ierr = VecCheckCUDAError("Call to kernSet. "); CHKERRQ(ierr);
    }
    xd->syncState=VEC_GPU;
  }
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "kernSet"
__global__ void kernSet(double* x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  __shared__ double chunkX[TCOUNT];
  chunkX[threadIdx.x] = dblScalarValue;
  if(tid<*n){
    x[tid] = chunkX[threadIdx.x]; /* arr[threadIdx.x]; */
  }
}




#undef __FUNCT__
#define __FUNCT__ "VecScale_SeqGPU"
PetscErrorCode VecScale_SeqGPU(Vec x, PetscScalar alpha){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  dim3 dimgrid(ceil((float)x->map->n/((float)TCOUNT)),1,1);
  dim3 dimblocks(TCOUNT,1,1);
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  //printf("VecScale_SeqGPU...alpha: %e\n",alpha);
  if(xd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,
            PETSC_ERR_MEM,
            "*** In call to VecScale_SeqGPU, arg Vec xin has not been allocated. ***\n");
  }else if(xd->syncState==VEC_CPU){
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }

  if(alpha==0){
    cudastatus=hipMemset((void*)xd->devptr,0,x->map->n*sizeof(double));
    ierr = VecCheckCUDAStatus(cudastatus,"on device cudaMemSet VecSet_SeqGPU");CHKERRQ(ierr);
  }else if (alpha != 1.0){
    cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha,sizeof(double),0,hipMemcpyHostToDevice);
    ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);
    kernScale<<<dimgrid,dimblocks>>>(xd->devptr,xd->length);
    ierr = VecCheckCUDAError("Call to kernScale. "); CHKERRQ(ierr);
  }
  hipDeviceSynchronize();
  fflush(NULL);
  xd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "kernScale"
__global__ void kernScale(double* x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  __shared__ double arr[TCOUNT];
  double localdbl=dblScalarValue;
  if(tid<*n){
    arr[threadIdx.x] = x[tid];
    //if(x[tid]!=0)printf("Pre: kernScale: x[%d]: %e, alpha: %e\n",tid,x[tid],localdbl);
    arr[threadIdx.x] *= localdbl;
    x[tid] = arr[threadIdx.x];
    //if(x[tid]!=0)printf("kernScale: x[%d]: %e, alpha: %e\n",tid,x[tid],localdbl);
  }
}






#undef __FUNCT__
#define __FUNCT__ "VecCheck_SeqGPU"
PetscErrorCode VecCheck_SeqGPU(Vec x){
  PetscFunctionBegin;/*
  PetscErrorCode ierr;
  dim3 dimgrid(ceil((float)x->map->n/((float)TCOUNT)),1,1);
  dim3 dimblocks(TCOUNT,1,1);
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  printf("******************************************\n");
  kernCheck<<<dimgrid,dimblocks>>>(xd->devptr,xd->length);
  ierr = VecCheckCUDAError("Call to kernCheck. "); CHKERRQ(ierr);
  hipDeviceSynchronize();
  printf("******************************************\n");
  fflush(NULL);*/
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "kernCheck"
__global__ void kernCheck(double* x, int* n){
  int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if(tid<*n){
    if(x[tid]!=0)printf("kernCheck: x[%d]: %e, length: %d\n",tid,x[tid],*n);
  }
}








#undef __FUNCT__
#define __FUNCT__ "VecTDot_SeqGPU"
PetscErrorCode VecTDot_SeqGPU(Vec xin,Vec yin,PetscScalar *z){
  PetscFunctionBegin;
  printf("VecTDot_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "VecDot_SeqGPU"
PetscErrorCode VecDot_SeqGPU(Vec x,Vec y,PetscScalar *z){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  double *devS;
  PetscScalar *s=PETSC_NULL;
  PetscInt i,blks,thds;
  //printf("Call to VecDot_SeqGPU\n");
  blks = ceil((float)x->map->n/(float)DOTTCOUNT);
  thds = DOTTCOUNT;
  //  printf("Blocks: %d, Threads: %d\n", blks,thds);
  ierr = PetscMalloc(blks*sizeof(PetscScalar),(void**)&s);CHKERRQ(ierr);
  cudastatus=hipMalloc((void**)&devS,blks*sizeof(double));/* could probably make this static */
  ierr = VecCheckCUDAStatus(cudastatus,"s alloc in VecDot_SeqGPU");CHKERRQ(ierr);
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid(blks,1);
  dim3 dimBlock(thds,1);

  if(xd->syncState==VEC_CPU){
    printf("xd state VEC_CPU: copying to device.\n");
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }

  if(yd->syncState==VEC_CPU){
    printf("yd state VEC_CPU: copying to device.\n");
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }

  kernDot<<<dimGrid,dimBlock>>>((double*)xd->devptr,(double*)yd->devptr,(int*)xd->length,(double*)devS);
  ierr = VecCheckCUDAError("kern launch in VecDot_SeqGPU");CHKERRQ(ierr);


  /* implicit barrier here */
  cudastatus=hipMemcpy(s,devS,blks*sizeof(PetscScalar),hipMemcpyDeviceToHost);/* copy back s */
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H in VecDot_SeqGPU");CHKERRQ(ierr);

  *z=0;
  for(i=0;i<blks;i++){/* last reduction done on CPU */
    *z+=s[i];
  }
  //printf("dot product Z: %e\n",*z,s[0],x->map->n);
  ierr = PetscFree(s);CHKERRQ(ierr);
  cudastatus = hipFree(devS);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}






__global__ void kernDot(double* devX, double* devY, int* n, double* s){
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  //int localn=*n;
  int i = (DOTTCOUNT+1)/2;

  __shared__ double chunkX[DOTTCOUNT];
  __shared__ double chunkY[DOTTCOUNT];

  if(tid<*n){
    /* read in values to shared */
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkX[threadIdx.x]*=chunkY[threadIdx.x];
  }else{
    chunkX[threadIdx.x]=0;
  }
  __syncthreads();
  while(i>0){
     if(threadIdx.x<i){
       chunkX[threadIdx.x]+=chunkX[threadIdx.x+i];
     }
     __syncthreads();
     i/=2;
  }/* end while */
  if(threadIdx.x==0){
    s[blockIdx.x]=chunkX[0];
  }
}





#undef __FUNCT__
#define __FUNCT__ "VecMDot_SeqGPU"
/*@
   VecMDot - Computes vector multiple dot products. 

   Collective on Vec

   Input Parameters:
+  x - one vector
.  nv - number of vectors
-  y - array of vectors. 

   Output Parameter:
.  val - array of the dot products (does not allocate the array)

@*/
PetscErrorCode  VecMDot_SeqGPU(Vec x,PetscInt nv,const Vec y[],PetscScalar val[]){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt       i;
  // printf("VecMDot_SeqGPU\n");
  for (i=0; i<nv; i++) {
    ierr = VecDot_SeqGPU(x,y[i],&val[i]);CHKERRQ(ierr);
    if(PetscIsInfOrNanScalar(val[i])){
      SETERRQ1(((PetscObject)x)->comm,PETSC_ERR_FP,"Infinite or not-a-number generated in mdot, entry %D",i);
    }
  }
  PetscFunctionReturn(0);
}






#undef __FUNCT__
#define __FUNCT__ "VecCopy_SeqGPU"
PetscErrorCode VecCopy_SeqGPU(Vec s,Vec d){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *sd=(Vec_SeqGPU*)s->data;
  Vec_SeqGPU *dd=(Vec_SeqGPU*)d->data;

  if(d->map->n!=s->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
   }

  //printf("Call to VecCopy_SeqGPU\n");
  if(dd->syncState==VEC_UNALLOC){
     SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Destination vector unalloced.");
  }

  if(sd->syncState==VEC_ALLOC){
      PetscFunctionReturn(0);/* nothing to do */
  }

  if(sd->syncState==VEC_CPU){
    ierr = PetscMemcpy((void*)dd->cpuptr,(void*)sd->cpuptr,s->map->n*sizeof(PetscScalar));CHKERRQ(ierr);
    PetscFunctionReturn(0);
  }

  kernCopyLen<<<1,1>>>(dd->length,sd->length);
  ierr = VecCheckCUDAError("call to kernCopyLen");CHKERRQ(ierr);

  ierr = VecCopyOverDevice(d,s); CHKERRQ(ierr);
  dd->syncState=sd->syncState;/* synch signal copy */

  //PetscBool same=PETSC_FALSE;
  //ierr = VecCompare_SeqGPU(s,d,&same,0,0);CHKERRQ(ierr);
  //if(!same)SETERRQ(PETSC_COMM_SELF,PETSC_ERR_LIB,"Vector duplication failed.");
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "VecSwap_SeqGPU"
PetscErrorCode VecSwap_SeqGPU(Vec xin,Vec yin){
  /* PetscErrorCode ierr; */
  PetscFunctionBegin;
  printf("VecSwap_SeqGPU (***EMPTY***)\n");
  if (xin != yin) {
#if defined(PETSC_USE_REAL_SINGLE)
    //////// hipblasSswap(bn,VecCUSPCastToRawPtr(*xarray),one,VecCUSPCastToRawPtr(*yarray),one);
#else
    //////   hipblasDswap(bn,VecCUSPCastToRawPtr(*xarray),one,VecCUSPCastToRawPtr(*yarray),one);
#endif

  }
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "VecAXPBY_SeqGPU"
PetscErrorCode VecAXPBY_SeqGPU(Vec yin,PetscScalar beta,PetscScalar alpha,Vec xin){
  PetscFunctionBegin;
  printf("Call to VecAXPBY_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "VecWAXPY_SeqGPU"
PetscErrorCode VecWAXPY_SeqGPU(Vec w,PetscScalar alpha,Vec x,Vec y){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt bx,tx;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)w->data;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid;
  dim3 dimBlock;
  hipError_t cudastatus;
  //printf("VecWAXPY_SeqGPU...");
  //printf("alpha: %e\n",alpha);
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }

  /* assuming width mem load isn't going to be an issue */
  bx=ceil((float)y->map->n/(float)AXPYTCOUNT);
  tx=AXPYTCOUNT;
  dimGrid.x=bx; dimGrid.y=1;
  dimBlock.x=tx; dimBlock.y=1;

  if(alpha==0.0){
    ierr = VecCopyOverDevice(w,y);CHKERRQ(ierr);
  }else if(alpha==1.0){
    kernWXPY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
    ierr = VecCheckCUDAError("kernel call to kernWXPY");CHKERRQ(ierr);
  }else if(alpha==-1.0){
    kernWXMY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
    ierr = VecCheckCUDAError("kernel call to kernWXMY");CHKERRQ(ierr);
  }else{
    cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha,sizeof(double),0,hipMemcpyHostToDevice);
    ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);
    kernWAXPY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
    ierr = VecCheckCUDAError("kernel call to kernWAXPY");CHKERRQ(ierr);
  }
  hipDeviceSynchronize();
  wd->syncState=VEC_GPU;
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "kernWAXPY"
__global__ void  kernWAXPY(double* devY,double* devX, int* vlen, double* devW){

 /* w <- y + alpha*x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double alphaShared;
  __shared__ double chunkY[AXPYTCOUNT];
  __shared__ double chunkX[AXPYTCOUNT];
  __shared__ double chunkW[AXPYTCOUNT];

  alphaShared = dblScalarValue;
  /* printf("in kernWAXPY:alphaShared: %f, tid: %d, vlen: %d\n",alphaShared,tid,*vlen); */
  if(tid<*vlen){
    //if(devX[tid]!=0)printf("kernWAXPY: devX[%d]: %e\n",tid,devX[tid]);
    //if(devY[tid]!=0)printf("kernWAXPY: devY[%d]: %e\n",tid,devY[tid]);
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkY[threadIdx.x]+(chunkX[threadIdx.x]*alphaShared);
    __syncthreads();
    devW[tid]=chunkW[threadIdx.x];
    //if(devW[tid]!=0)printf("kernWAXPY: devW[%d]: %e, alpha: %e\n",tid,devW[tid],alphaShared);
  }
}



#undef __FUNCT__
#define __FUNCT__ "kernWXPY"
__global__ void  kernWXPY(double* devY,double* devX, int* vlen, double* devW){

 /* w <- y + x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunkY[AXPYTCOUNT];
  __shared__ double chunkX[AXPYTCOUNT];
  __shared__ double chunkW[AXPYTCOUNT];

  /* printf("in kernWAXPY:alphaShared: %f, tid: %d, vlen: %d\n",alphaShared,tid,*vlen); */
  if(tid<*vlen){
    //if(devX[tid]!=0)printf("kernWXPY: devX[%d]: %e\n",tid,devX[tid]);
    //if(devY[tid]!=0)printf("kernWXPY: devY[%d]: %e\n",tid,devY[tid]);
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkY[threadIdx.x]+chunkX[threadIdx.x];
    __syncthreads();
    devW[tid]=chunkW[threadIdx.x];
    //if(devW[tid]!=0)printf("kernWXPY: devW[%d]: %e\n",tid,devW[tid]);
  }
}


#undef __FUNCT__
#define __FUNCT__ "kernWXMY"
__global__ void  kernWXMY(double* devY,double* devX, int* vlen, double* devW){

 /* w <- y + alpha*x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;

  __shared__ double chunkY[AXPYTCOUNT];
  __shared__ double chunkX[AXPYTCOUNT];
  __shared__ double chunkW[AXPYTCOUNT];
;
  /* printf("in kernWAXPY:alphaShared: %f, tid: %d, vlen: %d\n",alphaShared,tid,*vlen); */
  if(tid<*vlen){
    //if(devX[tid]!=0)printf("kernWXMY: devX[%d]: %e\n",tid,devX[tid]);
    //if(devY[tid]!=0)printf("kernWXMY: devY[%d]: %e\n",tid,devY[tid]);
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkY[threadIdx.x]-chunkX[threadIdx.x];
    __syncthreads();
    devW[tid]=chunkW[threadIdx.x];
    //if(devW[tid]!=0)printf("kernWXMY: devW[%d]: %e\n",tid,devW[tid]);
  }
}

















#undef __FUNCT__
#define __FUNCT__ "VecMAXPY_SeqGPU"
PetscErrorCode VecMAXPY_SeqGPU(Vec x,PetscInt nv,const PetscScalar* alpha,Vec *y){
  /* y = y + sum(a[i]*x[i]) */
  PetscFunctionBegin;
  //printf("VecMAXPY_SeqGPU\n");
  PetscErrorCode ierr;
  PetscInt i;  PetscInt bx,tx;
  dim3 dimGrid;
  dim3 dimBlock;
  hipError_t cudastatus;
  PetscScalar *devW;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=PETSC_NULL;

  cudastatus = hipMalloc((void**)&devW,x->map->n*sizeof(double));
  ierr = VecCheckCUDAStatus(cudastatus,"error in device malloc");CHKERRQ(ierr);
  cudastatus=hipMemset((void*)devW,0,x->map->n*sizeof(double));
  ierr = VecCheckCUDAStatus(cudastatus,"on hipMemset");CHKERRQ(ierr);

  /* assuming xwidth mem load isn't going to be an issue */

  bx=ceil((float)x->map->n/(float)AXPYTCOUNT);
  tx=AXPYTCOUNT;
  dimGrid.x=bx; dimGrid.y=1;
  dimBlock.x=tx; dimBlock.y=1;

  for(i=0;i<nv;i++){
    if(y[i]->map->n!=x->map->n){
      SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
    }
    yd=(Vec_SeqGPU*)y[i]->data;
    if(yd->syncState==VEC_CPU){/* synch x */
      ierr = VecCopyOverH2D(y[i],yd->cpuptr);CHKERRQ(ierr);
      yd->syncState=VEC_SYNCHED;
    }
    cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha[i],sizeof(double),0,hipMemcpyHostToDevice);
    ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);
    //printf("Alpha[%d]: %e\n", i, alpha[i]);
    if(alpha[i]==0){
      continue;
    }else if(alpha[i]==1.){
      /* assuming width mem load isn't going to be an issue */
      kernXPY<<<dimGrid,dimBlock>>>(devW,yd->devptr,yd->length);
      ierr = VecCheckCUDAError("kernel call to kernXPY");CHKERRQ(ierr);
    }else{
      /* assuming width mem load isn't going to be an issue */
      kernAXPY<<<dimGrid,dimBlock>>>(devW,yd->devptr,yd->length);
      ierr = VecCheckCUDAError("kernel call to kernAXPY");CHKERRQ(ierr);
    }
    hipDeviceSynchronize();
  }

  if(xd->syncState==VEC_CPU){/* synch x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  kernXPY<<<dimGrid,dimBlock>>>(xd->devptr,devW,xd->length);
  ierr = VecCheckCUDAError("kernel call to kernXPY");CHKERRQ(ierr);
  hipDeviceSynchronize();

  cudastatus = hipFree(devW);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree");CHKERRQ(ierr);
  xd->syncState=VEC_GPU;
  //ierr = VecCheck_SeqGPU(x);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "kernXPY"
__global__ void  kernXPY(double* devY,double* devX, int* vlen){

 /* y <- y + x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;

  __shared__ double chunkY[AXPYTCOUNT];
  __shared__ double chunkX[AXPYTCOUNT];

  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    //if(devX[tid]!=0)printf("kernXPY: devX[%d]: %e\n",tid,devX[tid]);
    //if(devX[tid]!=0)printf("kernXPY: PRE: devY[%d]: %e\n",tid,devY[tid]);
    chunkY[threadIdx.x]=devY[tid];
    chunkY[threadIdx.x]+=chunkX[threadIdx.x];
    devY[tid]=chunkY[threadIdx.x];
    //if(devY[tid]!=0)printf("kernXPY: POST: devY[%d]: %e\n",tid,devY[tid]);
  }
}




#undef __FUNCT__
#define __FUNCT__ "VecAXPY_SeqGPU"
PetscErrorCode VecAXPY_SeqGPU(Vec y,PetscScalar alpha,Vec x){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt bx,tx;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid;
  dim3 dimBlock;
  hipError_t cudastatus;
  //printf("VecAXPY_SeqGPU\n");

  if(x->map->n!=y->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  cudastatus=hipMemcpyToSymbol(HIP_SYMBOL("dblScalarValue"),(void*)&alpha,sizeof(double),0,hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"error in symbol copy to device");CHKERRQ(ierr);

  if(alpha==1.){
    /* assuming width mem load isn't going to be an issue */
    bx=ceil((float)x->map->n/(float)AXPYTCOUNT);
    tx=AXPYTCOUNT;
    dimGrid.x=bx; dimGrid.y=1;
    dimBlock.x=tx; dimBlock.y=1;
    kernXPY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,yd->length);
    ierr = VecCheckCUDAError("kernel call to kernXPY");CHKERRQ(ierr);
  }else if(alpha!=0){
    /* assuming width mem load isn't going to be an issue */
    bx=ceil((float)x->map->n/(float)AXPYTCOUNT);
    tx=AXPYTCOUNT;
    dimGrid.x=bx; dimGrid.y=1;
    dimBlock.x=tx; dimBlock.y=1;
    kernAXPY<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,yd->length);
    ierr = VecCheckCUDAError("kernel call to kernAXPY");CHKERRQ(ierr);
  }
  //hipDeviceSynchronize();
  yd->syncState=VEC_GPU;
  //ierr = VecCheck_SeqGPU(y);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "kernAXPY"
__global__ void  kernAXPY(double* devY,double* devX, int* vlen){

 /* y <- y + alpha*x */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;

  __shared__ double alphaShared;
  __shared__ double chunkY[AXPYTCOUNT];
  __shared__ double chunkX[AXPYTCOUNT];

  alphaShared = dblScalarValue;
  //if(tid==0)printf("in kernAXPY:alphaShared: %f\n",alphaShared);
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    //if(devX[tid]!=0)printf("kernAXPY: devIN[%d]: %e, len: %d\n",tid,devX[tid],*vlen);
    //if(devY[tid]!=0)printf("kernAXPY: PRE: devOUT[%d]: %e\n",tid,devY[tid]);
    chunkY[threadIdx.x]=devY[tid];
    chunkY[threadIdx.x]+=chunkX[threadIdx.x]*alphaShared;
    devY[tid]=chunkY[threadIdx.x];
    //if(devY[tid]!=0)printf("kernAXPY: POST: devOUT[%d]: %e\n",tid,devY[tid]);
  }
}



#undef __FUNCT__
#define __FUNCT__ "VecPointwiseMult_SeqGPU"
PetscErrorCode VecPointwiseMult_SeqGPU(Vec w,Vec x,Vec y){
  PetscFunctionBegin;
  //printf("VecPointwiseMult_SeqGPU\n");
  PetscErrorCode ierr;
  PetscInt bx,tx;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid;
  dim3 dimBlock;
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  /* assuming width mem load isn't going to be an issue */
  bx=ceil((float)y->map->n/(float)PMULTCOUNT);
  tx=PMULTCOUNT;
  dimGrid.x=bx; dimGrid.y=1;
  dimBlock.x=tx; dimBlock.y=1;
  kernPMULT<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
  ierr = VecCheckCUDAError("kernel call to kernPMULT");CHKERRQ(ierr);
  //hipDeviceSynchronize();
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "kernPMULT"
__global__ void  kernPMULT(double* devY,double* devX, int* vlen, double* devW){

 /* w <- x./y */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunkY[PMULTCOUNT];
  __shared__ double chunkX[PMULTCOUNT];
  __shared__ double chunkW[PMULTCOUNT];
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkW[threadIdx.x]=chunkX[threadIdx.x]*chunkY[threadIdx.x];
    devW[tid]=chunkW[threadIdx.x];
  }
}




#undef __FUNCT__
#define __FUNCT__ "VecMaxPointwiseDivide_SeqGPU"
PetscErrorCode VecMaxPointwiseDivide_SeqGPU(Vec x,Vec y,PetscReal *max){
  PetscFunctionBegin;
  //printf("VecMaxPointwiseDivide_SeqGPU...");
  PetscErrorCode ierr;
  hipError_t cudastatus;
  PetscInt i,bx,tx;
  PetscScalar *maxlist=PETSC_NULL;
  PetscScalar *devmaxlist=PETSC_NULL;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid;
  dim3 dimBlock;
  if(x->map->n!=y->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  /* assuming width mem load isn't going to be an issue */
  bx=ceil((float)y->map->n/(float)PDIVTCOUNT);
  tx=PDIVTCOUNT;
  dimGrid.x=bx; dimGrid.y=1;
  dimBlock.x=tx; dimBlock.y=1;

  ierr = PetscMalloc(bx*sizeof(PetscScalar),&maxlist);CHKERRQ(ierr);

  cudastatus=hipMalloc((void**)&devmaxlist,bx*sizeof(PetscScalar));
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H");CHKERRQ(ierr);

  kernMAXPDIV<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,devmaxlist);
  ierr = VecCheckCUDAError("kernel call to kernPDIV");CHKERRQ(ierr);

  cudastatus=hipMemcpy(maxlist,devmaxlist,bx*sizeof(PetscScalar),hipMemcpyDeviceToHost);/* copy back */
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H");CHKERRQ(ierr);

  *max = maxlist[0];
  if(bx>1){/* final collapse */
    for(i=1;i<bx;i++){
      if(maxlist[i]>*max){
        *max=maxlist[i];
      }
    }
  }
  //printf("max: %f\n",*max);
  ierr = PetscFree(maxlist);CHKERRQ(ierr);
  cudastatus = hipFree(devmaxlist);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}



#undef __FUNCT__
#define __FUNCT__ "kernMAXPDIV"
__global__ void  kernMAXPDIV(double* devY,double* devX, int* vlen, double* maxlist){

 /* w <- max(abs(x./y)) */
  int i,tid;
  i = (PDIVTCOUNT+1)/2;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunkY[PDIVTCOUNT];
  __shared__ double chunkX[PDIVTCOUNT];
  __shared__ double chunkW[PDIVTCOUNT];
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    //printf("kernMAXDIV: devX[%d]: %e\n",tid,devX[tid]);
    //printf("kernMAXDIV: devY[%d]: %e\n",tid,devY[tid]);
    if(chunkY[threadIdx.x]!=0){
      chunkW[threadIdx.x]=fabs(__ddiv_rn(chunkX[threadIdx.x],chunkY[threadIdx.x]));
    }else{
      chunkW[threadIdx.x]=fabs(chunkX[threadIdx.x]);
    }
    //printf("kernMAXDIV: d[%d]: %e\n",threadIdx.x,chunkW[threadIdx.x]);

  }else{
    chunkW[threadIdx.x]=0.0;
  }

  __syncthreads();
  while(i>0){
    if(threadIdx.x<i && chunkW[threadIdx.x]<chunkW[threadIdx.x+i]){
      chunkW[threadIdx.x]=chunkW[threadIdx.x+i];
    }
    i/=2;
    __syncthreads();
  }
  if(threadIdx.x==0)maxlist[blockIdx.x]=chunkW[0];
}




#undef __FUNCT__
#define __FUNCT__ "VecPointwiseDivide_SeqGPU"
PetscErrorCode VecPointwiseDivide_SeqGPU(Vec w,Vec x,Vec y){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt bx,tx;
  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  Vec_SeqGPU *yd=(Vec_SeqGPU*)y->data;
  Vec_SeqGPU *wd=(Vec_SeqGPU*)y->data;
  dim3 dimGrid;
  dim3 dimBlock;
  //printf("Call to VecPointwiseDivide_SeqGPU\n");
  if(x->map->n!=y->map->n || w->map->n!=y->map->n || w->map->n!=x->map->n){
    SETERRQ(PETSC_COMM_WORLD,PETSC_ERR_MEM,"Vector size mismatch.");
  }
  if(yd->syncState==VEC_CPU){/* synch up y */
    ierr = VecCopyOverH2D(y,yd->cpuptr);CHKERRQ(ierr);
    yd->syncState=VEC_SYNCHED;
  }
  if(xd->syncState==VEC_CPU){/* synch up x */
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }
  /* assuming width mem load isn't going to be an issue */
  bx=ceil((float)y->map->n/(float)PDIVTCOUNT);
  tx=PDIVTCOUNT;
  dimGrid.x=bx; dimGrid.y=1;
  dimBlock.x=tx; dimBlock.y=1;

  kernPDIV<<<dimGrid,dimBlock>>>(yd->devptr,xd->devptr,xd->length,wd->devptr);
  ierr = VecCheckCUDAError("kernel call to kernPDIV");CHKERRQ(ierr);
  //hipDeviceSynchronize();
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "kernPDIV"
__global__ void  kernPDIV(double* devY,double* devX, int* vlen, double* devW){

 /* w <- x./y */
  int tid;
  tid = blockIdx.x*blockDim.x+threadIdx.x;
  __shared__ double chunkY[PDIVTCOUNT];
  __shared__ double chunkX[PDIVTCOUNT];
  __shared__ double chunkW[PDIVTCOUNT];
  if(tid<*vlen){
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    if(chunkX[threadIdx.x]*chunkY[threadIdx.x]!=0){
      chunkW[threadIdx.x]=__ddiv_rn(chunkX[threadIdx.x],chunkY[threadIdx.x]);
    }else{
      chunkW[threadIdx.x]=0;
    }
    devW[tid]=chunkW[threadIdx.x];
  }
}




#undef __FUNCT__
#define __FUNCT__ "VecNorm_SeqGPU"
PetscErrorCode VecNorm_SeqGPU(Vec x,NormType type,PetscReal* z){
  /* NormType: NORM_1=0,NORM_2=1,NORM_FROBENIUS=2,NORM_INFINITY=3,NORM_1_AND_2=4 */
  /* dealing with NORM_2 for now... */
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus=hipSuccess;
  double *deviceS=PETSC_NULL;
  PetscScalar *s=PETSC_NULL;
  PetscInt i,blks,thds;
  //printf("VecNorm_SeqGPU\n");
  blks = ceil((float)x->map->n/(float)NORMTCOUNT);
  thds = NORMTCOUNT;
  //printf("Blocks: %d, Threads: %d\n", blks,thds);
  ierr = PetscMalloc(blks*sizeof(PetscScalar),(void**)&s);CHKERRQ(ierr);
  hipDeviceSynchronize();

  cudastatus=hipMalloc((void**)&deviceS,blks*sizeof(double));
  ierr = VecCheckCUDAStatus(cudastatus,"deviceS alloc in VecNorm_SeqGPU");CHKERRQ(ierr);

  Vec_SeqGPU *xd=(Vec_SeqGPU*)x->data;
  dim3 dimGrid(blks,1);
  dim3 dimBlock(thds,1);

  if(xd->syncState==VEC_CPU){
    ierr = VecCopyOverH2D(x,xd->cpuptr);CHKERRQ(ierr);
    xd->syncState=VEC_SYNCHED;
  }

  kernNorm2<<<dimGrid,dimBlock>>>((double*)xd->devptr,(int*)xd->length,(double*)deviceS);
  ierr = VecCheckCUDAError("kern launch in VecDot_SeqGPU");CHKERRQ(ierr);

  /* implicit device barrier below */
  cudastatus=hipMemcpy(s,deviceS,blks*sizeof(PetscScalar),hipMemcpyDeviceToHost);/* copy back s */
  ierr = VecCheckCUDAStatus(cudastatus,"on copy D2H in VecDot_SeqGPU");CHKERRQ(ierr);

  hipDeviceSynchronize();
  cudastatus = hipFree(deviceS);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);

  *z=0;
  for(i=0;i<blks;i++){/* last reduction done on CPU */
    *z+=s[i];
  }
  ierr = PetscFree(s);CHKERRQ(ierr);
  z[0]=sqrt(z[0]);/* norm2 in 0 norm1_2 in 0 and 1 respectively */
  //printf("NORM2: %e\n",z[0]);
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "kernNorm2"
__global__ void kernNorm2(double* devX,int* n, double* s){
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  //int localn=*n;
  int i = (NORMTCOUNT+1)/2;

  __shared__ double chunkX[NORMTCOUNT];
  if(tid<*n){
    /* read in values to shared */
    //if(devX[tid]!=0)printf("kernNorm2: devX[%d]: %e\n",tid,devX[tid]);
    chunkX[threadIdx.x]=devX[tid];
    chunkX[threadIdx.x]*=chunkX[threadIdx.x];
  }else{
    chunkX[threadIdx.x]=0;
  }

  __syncthreads();

  //if(chunkX[threadIdx.x]>0)printf("chunkX[%d]: %e\n",threadIdx.x,chunkX[threadIdx.x]);
  while(i>0){
    if(threadIdx.x<i){
       chunkX[threadIdx.x]+=chunkX[threadIdx.x+i];
    }
    __syncthreads();
    i/=2;
  }/* end while */
  if(threadIdx.x==0){
    s[blockIdx.x]=chunkX[0];
    //printf("devS[%d]: %e\n",blockIdx.x,s[blockIdx.x]);
  }
}

/*
#undef __FUNCT__
#define __FUNCT__ "VecNorm1_SeqGPU"
PetscErrorCode VecNorm1_SeqGPU(Vec xin,NormType type,PetscReal* z)
{*/

/* NormType: NORM_1=0,NORM_2=1,NORM_FROBENIUS=2,NORM_INFINITY=3,NORM_1_AND_2=4 */
/* dealing with NORM_2 for now... */
/* z has 2 elements */

/*
  PetscErrorCode ierr;
  PetscFunctionBegin;
  printf("Call to VecNorm_SeqGPU\n");
  ierr = VecDot_SeqGPU(xin,xin,&z[0]);CHKERRQ(ierr);
  z[0]=sqrt(z[0]);
  printf("ZNORM: %f\n\n",*z);
  PetscFunctionReturn(0);
}*/


/*
#undef __FUNCT__
#define __FUNCT__ "kernReduceAbsSum"
PetscErrorCode kernReduceAbsSum(double * x, PetscReal* z){

}
*/



#undef __FUNCT__
#define __FUNCT__ "VecResetArray_SeqGPU"
PetscErrorCode VecResetArray_SeqGPU(Vec vin){
  PetscFunctionBegin;
  printf("VecResetArray_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "VecGetArray_SeqGPU"
PetscErrorCode VecGetArray_SeqGPU(Vec v,PetscScalar **a){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  Vec_SeqGPU *vd=(Vec_SeqGPU*)v->data;
  if(vd->syncState==VEC_UNALLOC){
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"in VecGetArray_SeqGPU the vector has not been created.");
  }
  //printf("Call to VecGetArray_SeqGPU\n");

  int i,flg1=0,flg2=0;
  PetscStackCheckByName(4,"DMDAVecGetArray",flg1);
  PetscStackCheckByName(6,"DMGlobalToLocalBegin",flg2);
  if(vd->syncState==VEC_GPU || flg1 || flg2){/* may not need VEC_GPU element */
    ierr = VecCopyOverD2H(v,vd->cpuptr); CHKERRQ(ierr);
    vd->syncState = VEC_CPU;
    //for(i=0;i<v->map->n;i++){
    //   if(vd->cpuptr[i]!=0)printf("get[%d]: %e\n",i,vd->cpuptr[i]);
    //}
  }
  *a=vd->cpuptr;
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "VecRestoreArray_SeqGPU"
PetscErrorCode VecRestoreArray_SeqGPU(Vec v,PetscScalar **a){
  PetscFunctionBegin;
  /* printf("Call to VecRestoreArray_SeqGPU\n"); */
  PetscErrorCode ierr;
  Vec_SeqGPU *vd=(Vec_SeqGPU*)v->data;
  if(vd->syncState==VEC_CPU || vd->syncState==VEC_ALLOC){
    int i=0;
    if(a){
      ierr = VecCopyOverH2D(v,*a);CHKERRQ(ierr);
      //for(i=0;i<v->map->n;i++){
      //  if(vd->cpuptr[i]!=0)printf("put *a[%d]: %e\n",i,(*a)[i]);
      //}
      vd->syncState=VEC_GPU;
    }else{
      ierr = VecCopyOverH2D(v,vd->cpuptr);CHKERRQ(ierr);
      // for(i=0;i<v->map->n;i++){
      //   if(vd->cpuptr[i]!=0)printf("put cpuptr[%d]: %e\n",i,vd->cpuptr[i]);
      //}
      vd->syncState=VEC_SYNCHED;
    }
  }
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "VecPlaceArray_SeqGPU"
PetscErrorCode VecPlaceArray_SeqGPU(Vec vin,const PetscScalar *a){
  //PetscErrorCode ierr;
  PetscFunctionBegin;
  printf("VecPlaceArray_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "VecReplaceArray_SeqGPU"
PetscErrorCode VecReplaceArray_SeqGPU(Vec vin,const PetscScalar *a){
  //PetscErrorCode ierr;
  PetscFunctionBegin;
  printf("VecReplaceArray_SeqGPU (***EMPTY***)\n");
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "VecCreateSeqGPU"
PetscErrorCode  VecCreateSeqGPU(MPI_Comm comm,PetscInt n,Vec *v){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  //printf("VecCreateSeqGPU\n");
  ierr = VecCreate(comm,v);CHKERRQ(ierr);
  ierr = VecSetSizes(*v,n,n);CHKERRQ(ierr);
  ierr = VecSetType(*v,VECSEQGPU);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "VecDotNorm2_SeqGPU"
PetscErrorCode VecDotNorm2_SeqGPU(Vec s, Vec t, PetscScalar *dp, PetscScalar *nm){
  /* PetscErrorCode ierr; */
  /* PetscScalar zero = 0.0,n=s->map->n; */
  PetscFunctionBegin;
  printf("VecDotNorm2_SeqGPU (***EMPTY***)\n");
  /* ierr = PetscLogFlops(4.0*n);CHKERRQ(ierr); */
  PetscFunctionReturn(0);
}






/*

#undef __FUNCT__
#define __FUNCT__ "VecAXPBYPCZ_SeqGPU"
PetscErrorCode VecAXPBYPCZ_SeqGPU(Vec zin,PetscScalar alpha,PetscScalar beta,PetscScalar gamma,Vec xin,Vec yin)
{
  PetscErrorCode     ierr;
  PetscInt           n = zin->map->n;

  PetscFunctionBegin;
  PetscFunctionReturn(0);
}
*/

#undef __FUNCT__
#define __FUNCT__ "VecAXPBYPCZ_SeqGPU"
PetscErrorCode VecAXPBYPCZ_SeqGPU(Vec x, PetscScalar alpha, PetscScalar beta,\
                           PetscScalar gamma, Vec y, Vec z){

  PetscFunctionBegin;
  PetscErrorCode ierr;
  int blocks=ceil(x->map->n/32);/* assuming shared memory size is not an issue */
  int threads=32;
  hipError_t cudastatus;
  Vec_SeqGPU* devX = (Vec_SeqGPU*)x->data;
  Vec_SeqGPU* devY = (Vec_SeqGPU*)y->data;
  Vec_SeqGPU* devZ = (Vec_SeqGPU*)z->data;

  double2 params[2];
  double2 *devparams=PETSC_NULL;

  cudastatus = hipMalloc((void**)&devparams,2*sizeof(double2));
  ierr = VecCheckCUDAStatus(cudastatus,"on hipMalloc()");CHKERRQ(ierr);
  params[0].x=alpha;
  params[0].y=beta;
  params[1].x=gamma;
  cudastatus=hipMemcpy(devparams,params,2,hipMemcpyHostToDevice);
  dim3 dimGrid; dimGrid.x=blocks; dimGrid.y=1;
  dim3 dimBlock; dimBlock.x=threads; dimBlock.y=1;
  kernAXPBYPCZ<<<dimGrid,dimBlock>>>(devparams,devX->devptr,devY->devptr,devZ->devptr);
  cudastatus = hipFree(devparams);
  ierr = VecCheckCUDAStatus(cudastatus,"on hipFree()");CHKERRQ(ierr);
  PetscFunctionReturn(0);
}






__global__ void kernAXPBYPCZ(double2* devparams, double* devX, double* devY, double* devZ){
  /* x <- alpha*x + beta*y + gamma*z */
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  double2 alphabeta = devparams[0];
  double2 gamma = devparams[1];
  double work;
  int localn = devN;

  __shared__ double chunkX[32];
  __shared__ double chunkY[32];
  __shared__ double chunkZ[32];

  if(tid<localn){
    /* read in values to shared */
    chunkX[threadIdx.x]=devX[tid];
    chunkY[threadIdx.x]=devY[tid];
    chunkZ[threadIdx.x]=devZ[tid];

    /* do flops */
    if(gamma.x){
      work=gamma.x*chunkZ[threadIdx.x];
    }
    if(alphabeta.y){
      work+=alphabeta.y*chunkY[threadIdx.x];
    }
    if(alphabeta.x){
      work+=alphabeta.x*chunkX[threadIdx.x];
    }
    chunkX[threadIdx.x]+=work;

    /* write back */
    devX[tid]=chunkX[threadIdx.x];
  }
  return;
}





#undef __FUNCT__
#define __FUNCT__ "VecDuplicate_SeqGPU"
PetscErrorCode VecDuplicate_SeqGPU(Vec win,Vec *V){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  //printf("Call to VecDuplicate_SeqGPU\n");
  ierr = VecCreate(((PetscObject)win)->comm,V);CHKERRQ(ierr);
  ierr = VecSetType(*V,VECSEQGPU);CHKERRQ(ierr);
  ierr = PetscObjectSetPrecision((PetscObject)*V,((PetscObject)win)->precision);CHKERRQ(ierr);
  ierr = VecSetSizes(*V,win->map->n,win->map->N);CHKERRQ(ierr);
  ierr = PetscLayoutReference(win->map,&(*V)->map);CHKERRQ(ierr);
  ierr = PetscOListDuplicate(((PetscObject)win)->olist,&((PetscObject)(*V))->olist);CHKERRQ(ierr);
  ierr = PetscFListDuplicate(((PetscObject)win)->qlist,&((PetscObject)(*V))->qlist);CHKERRQ(ierr);
  (*V)->stash.ignorenegidx = win->stash.ignorenegidx;
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "VecDuplicateVecs_SeqGPU"
PetscErrorCode VecDuplicateVecs_SeqGPU(Vec vin, PetscInt m, Vec **Vlist){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt i=0;
  PetscValidHeaderSpecific(vin,VEC_CLASSID,1);
  PetscValidPointer(Vlist,3);
  if (m <= 0) SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_OUTOFRANGE,"m must be > 0: m = %D",m);
  ierr = PetscMalloc(m*sizeof(Vec),Vlist);CHKERRQ(ierr);
  for(i=0;i<m;i++){
    ierr = VecDuplicate_SeqGPU(vin,*Vlist+i);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}




#undef __FUNCT__
#define __FUNCT__ "VecDestroy_SeqGPU"
PetscErrorCode VecDestroy_SeqGPU(Vec v){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  Vec_SeqGPU* vd = (Vec_SeqGPU*)v->data;
  PetscValidHeaderSpecific(v,VEC_CLASSID,1);
  printf("VecDestroy_SeqGPU vstats: \n");
  printf("...................................\n");
  printf("H2D transfers: %d, byte count: %d\n",vd->vstat.h2d_count,vd->vstat.h2d_bytes);
  printf("D2H transfers: %d, byte count: %d\n",vd->vstat.d2h_count,vd->vstat.d2h_bytes);
  printf("...................................\n");
  /* static int counter = 1; */
  if(vd && vd->syncState != VEC_UNALLOC){
    if(vd->devptr){
      cudastatus=hipFree(vd->devptr);
      ierr=VecCheckCUDAStatus(cudastatus,"destroying vd->devptr in VecDestroy_SeqGPU");CHKERRQ(ierr);
      vd->devptr=PETSC_NULL;
    }
    if(vd->length){
      cudastatus=hipFree(vd->length);
      ierr=VecCheckCUDAStatus(cudastatus,"destroying vd->length in VecDestroy_SeqGPU");CHKERRQ(ierr);
      vd->length=PETSC_NULL;
    }
    if(vd->cpuptr){
      ierr = PetscFree(vd->cpuptr);CHKERRQ(ierr);
    }
    vd->syncState = VEC_UNALLOC;
  }

  ierr = PetscObjectDepublish(v);CHKERRQ(ierr);
#if defined(PETSC_USE_LOG)
  PetscLogObjectState((PetscObject)v,"Length=%D",v->map->n);
#endif
  ierr = PetscFree(v->data);CHKERRQ(ierr);
  //  ierr = VecDestroy_Seq(v);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}





#undef __FUNCT__
#define __FUNCT__ "VecDestroyVecs_SeqGPU"
PetscErrorCode  VecDestroyVecs_SeqGPU(PetscInt m,Vec *vv){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  PetscInt i;
   /* destroy the internal part */
  for(i=0;i<m;i++){
    ierr = VecDestroy(&vv[i]);CHKERRQ(ierr);
  }
  PetscFunctionReturn(0);
}







#undef __FUNCT__
#define __FUNCT__ "VecSetDimensions_SeqGPU"
PetscErrorCode  VecSetDimensions_SeqGPU(Vec x,PetscInt ndims,dim3 dimsize){
  PetscFunctionBegin;
  Vec_SeqGPU* xd = (Vec_SeqGPU*)x->data;
  if(ndims==3){
    xd->ndims=ndims;
    xd->dimsize.x=dimsize.x;
    xd->dimsize.y=dimsize.y;
    xd->dimsize.z=dimsize.z;
  }else if(ndims==2){
    xd->ndims=ndims;
    xd->dimsize.x=dimsize.x;
    xd->dimsize.y=dimsize.y;
    xd->dimsize.z=0;
  }else if(ndims==1){
    xd->ndims=ndims;
    if(x->map->n!=dimsize.x){
      SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"VecSetDimensions_SeqGPU does not support memory resizing.");
    }else{
      xd->dimsize.x=dimsize.x;
      xd->dimsize.y=0;
      xd->dimsize.z=0;
    }
  }else{
    SETERRQ(PETSC_COMM_SELF,PETSC_ERR_MEM,"VECSEQGPU doesn't suport given number of dimensions.");
  }
  xd->dimsetflag=PETSC_TRUE;
  PetscFunctionReturn(0);
}








#undef __FUNCT__
#define __FUNCT__ "VecCreate_SeqGPU"
PetscErrorCode  VecCreate_SeqGPU(Vec V){
  PetscFunctionBegin;
  PetscErrorCode ierr;
  hipError_t cudastatus;
  PetscMPIInt    size;
  Vec_SeqGPU* seqgpu=PETSC_NULL;
  //printf("Call to VecCreate_SeqGPU\n");

  /*  ierr = PetscNewLog(V,Vec_SeqGPU,&(V->data));CHKERRQ(ierr); */


  ierr = PetscMalloc(sizeof(Vec_SeqGPU),&seqgpu);
  V->data=(void*)seqgpu;
  ierr = MPI_Comm_size(((PetscObject)V)->comm,&size);CHKERRQ(ierr);
  if  (size > 1) SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONG,"Cannot create VECSEQGPU on more than one process");

  if (V->map->bs == -1) V->map->bs = 1;
  ierr = PetscLayoutSetUp(V->map);CHKERRQ(ierr);
  ierr = PetscObjectChangeTypeName((PetscObject)V,VECSEQGPU);CHKERRQ(ierr);

  V->ops->dot             = VecDot_SeqGPU;
  V->ops->norm            = VecNorm_SeqGPU;
  V->ops->tdot            = VecTDot_SeqGPU;
  V->ops->scale           = VecScale_SeqGPU;
  V->ops->copy            = VecCopy_SeqGPU;
  V->ops->set             = VecSet_SeqGPU;
  V->ops->setvalues       = VecSetValues_SeqGPU;
  V->ops->swap            = VecSwap_SeqGPU;
  V->ops->axpy            = VecAXPY_SeqGPU;
  V->ops->axpby           = VecAXPBY_SeqGPU;
  V->ops->axpbypcz        = VecAXPBYPCZ_SeqGPU;
  V->ops->pointwisemult   = VecPointwiseMult_SeqGPU;
  V->ops->pointwisedivide = VecPointwiseDivide_SeqGPU;
  V->ops->maxpointwisedivide = VecMaxPointwiseDivide_SeqGPU;
  V->ops->setrandom       = VecSetRandom_SeqGPU;
  V->ops->dot_local       = VecDot_SeqGPU;
  V->ops->tdot_local      = VecTDot_SeqGPU;
  V->ops->norm_local      = VecNorm_SeqGPU;
  V->ops->maxpy           = VecMAXPY_SeqGPU;
  V->ops->mdot            = VecMDot_SeqGPU;
  /* V->ops->aypx            = VecAYPX_SeqGPU; */
  V->ops->waxpy           = VecWAXPY_SeqGPU;
  V->ops->dotnorm2        = VecDotNorm2_SeqGPU;
  V->ops->placearray      = VecPlaceArray_SeqGPU;
  V->ops->replacearray    = VecReplaceArray_SeqGPU;
  V->ops->resetarray      = VecResetArray_SeqGPU;
  V->ops->destroy         = VecDestroy_SeqGPU;
  V->ops->destroyvecs     = VecDestroyVecs_SeqGPU;
  V->ops->duplicate       = VecDuplicate_SeqGPU;
  V->ops->duplicatevecs   = VecDuplicateVecs_SeqGPU;
  V->ops->getarray        = VecGetArray_SeqGPU;
  V->ops->restorearray    = VecRestoreArray_SeqGPU;
  V->ops->getlocalsize    = VecGetLocalSize_SeqGPU;
  V->ops->getsize         = VecGetSize_SeqGPU;
  V->ops->view            = VecView_SeqGPU;
  V->petscnative=PETSC_FALSE;
  seqgpu->syncState      = VEC_UNALLOC;
  seqgpu->lifetime       = VEC_PERSIST;

  seqgpu->vstat.h2d_count=0;
  seqgpu->vstat.d2h_count=0;
  seqgpu->vstat.h2d_bytes=0;
  seqgpu->vstat.d2h_bytes=0;




  /* allocate the variable for vector size */
  cudastatus=hipMalloc((void**)&(seqgpu->length),sizeof(int));
  ierr = VecCheckCUDAStatus(cudastatus,"**** Alloc devlength in VecCreate_SeqGPU");CHKERRQ(ierr);

  /* send vec length size to device */
  cudastatus=hipMemcpy((void*)seqgpu->length,(void*)&(V->map->n),sizeof(int),hipMemcpyHostToDevice);
  ierr = VecCheckCUDAStatus(cudastatus,"**** Copy H2D devlength in VecCreate_SeqGPU");CHKERRQ(ierr);
  seqgpu->vstat.h2d_count++;
  seqgpu->vstat.h2d_bytes+=sizeof(int);

  /* allocate the vector on device */
  cudastatus=hipMalloc((void**)&(seqgpu->devptr),V->map->n*sizeof(double));
  ierr = VecCheckCUDAStatus(cudastatus,"**** Alloc of devptr in VecCreate_SeqGPU");CHKERRQ(ierr);
  cudastatus=hipMemset((void*)seqgpu->devptr,0,V->map->n*sizeof(double));
  ierr = VecCheckCUDAStatus(cudastatus,"on device cudaMemSet");CHKERRQ(ierr);

  seqgpu->ndims=1;/* default number of dimensions */
  seqgpu->dimsize.x=V->map->n;
  seqgpu->dimsize.y=0;
  seqgpu->dimsize.z=0;
  seqgpu->dimsetflag=PETSC_FALSE;

  ierr = PetscMalloc(V->map->n*sizeof(PetscScalar),&(seqgpu->cpuptr));CHKERRQ(ierr);
  ierr = PetscMemzero(seqgpu->cpuptr,V->map->n*sizeof(PetscScalar));CHKERRQ(ierr);
  seqgpu->syncState=VEC_ALLOC;

  PetscFunctionReturn(0);
}






































#undef __FUNCT__  
#define __FUNCT__ "VecView_Seq_ASCII"
static PetscErrorCode VecView_Seq_ASCII(Vec xin,PetscViewer viewer){
  PetscErrorCode    ierr;
  PetscInt          i,n = xin->map->n;
  const char        *name;
  PetscViewerFormat format;
  PetscScalar *xv;

  PetscFunctionBegin;
  ierr = VecGetArray_SeqGPU(xin,&xv);CHKERRQ(ierr);
  ierr = PetscViewerGetFormat(viewer,&format);CHKERRQ(ierr);
  if (format == PETSC_VIEWER_ASCII_MATLAB) {
    ierr = PetscObjectGetName((PetscObject)xin,&name);CHKERRQ(ierr);
    ierr = PetscViewerASCIIPrintf(viewer,"%s = [\n",name);CHKERRQ(ierr);
    for (i=0; i<n; i++) {
#if defined(PETSC_USE_COMPLEX)
      if (PetscImaginaryPart(xv[i]) > 0.0) {
        ierr = PetscViewerASCIIPrintf(viewer,"%18.16e + %18.16ei\n",PetscRealPart(xv[i]),PetscImaginaryPart(xv[i]));CHKERRQ(ierr);
      } else if (PetscImaginaryPart(xv[i]) < 0.0) {
        ierr = PetscViewerASCIIPrintf(viewer,"%18.16e - %18.16ei\n",PetscRealPart(xv[i]),-PetscImaginaryPart(xv[i]));CHKERRQ(ierr);
      } else {
        ierr = PetscViewerASCIIPrintf(viewer,"%18.16e\n",PetscRealPart(xv[i]));CHKERRQ(ierr);
      }
#else
      ierr = PetscViewerASCIIPrintf(viewer,"%18.16e\n",(double) xv[i]);CHKERRQ(ierr);
#endif
    }
    ierr = PetscViewerASCIIPrintf(viewer,"];\n");CHKERRQ(ierr);
  } else if (format == PETSC_VIEWER_ASCII_SYMMODU) {
    for (i=0; i<n; i++) {
#if defined(PETSC_USE_COMPLEX)
      ierr = PetscViewerASCIIPrintf(viewer,"%18.16e %18.16e\n",PetscRealPart(xv[i]),PetscImaginaryPart(xv[i]));CHKERRQ(ierr);
#else
      ierr = PetscViewerASCIIPrintf(viewer,"%18.16e\n",xv[i]);CHKERRQ(ierr);
#endif
    }
  } else if (format == PETSC_VIEWER_ASCII_VTK || format == PETSC_VIEWER_ASCII_VTK_CELL) {
    /* 
       state 0: No header has been output
       state 1: Only POINT_DATA has been output
       state 2: Only CELL_DATA has been output
       state 3: Output both, POINT_DATA last
       state 4: Output both, CELL_DATA last 
    */
    static PetscInt stateId = -1;
    int outputState = 0;
    PetscBool  hasState;
    int doOutput = 0;
    PetscInt bs, b;

    if (stateId < 0) {
      ierr = PetscObjectComposedDataRegister(&stateId);CHKERRQ(ierr);
    }
    ierr = PetscObjectComposedDataGetInt((PetscObject) viewer, stateId, outputState, hasState);CHKERRQ(ierr);
    if (!hasState) {
      outputState = 0;
    }
    ierr = PetscObjectGetName((PetscObject) xin, &name);CHKERRQ(ierr);
    ierr = VecGetBlockSize(xin, &bs);CHKERRQ(ierr);
    if ((bs < 1) || (bs > 3)) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE, "VTK can only handle 3D objects, but vector dimension is %d", bs);
    }
    if (format == PETSC_VIEWER_ASCII_VTK) {
      if (outputState == 0) {
        outputState = 1;
        doOutput = 1;
      } else if (outputState == 1) {
        doOutput = 0;
      } else if (outputState == 2) {
        outputState = 3;
        doOutput = 1;
      } else if (outputState == 3) {
        doOutput = 0;
      } else if (outputState == 4) {
        SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE, "Tried to output POINT_DATA again after intervening CELL_DATA");
      }
      if (doOutput) {
        ierr = PetscViewerASCIIPrintf(viewer, "POINT_DATA %d\n", n/bs);CHKERRQ(ierr);
      }
    } else {
      if (outputState == 0) {
        outputState = 2;
        doOutput = 1;
      } else if (outputState == 1) {
        outputState = 4;
        doOutput = 1;
      } else if (outputState == 2) {
        doOutput = 0;
      } else if (outputState == 3) {
        SETERRQ(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE, "Tried to output CELL_DATA again after intervening POINT_DATA");
      } else if (outputState == 4) {
        doOutput = 0;
      }
      if (doOutput) {
        ierr = PetscViewerASCIIPrintf(viewer, "CELL_DATA %d\n", n);CHKERRQ(ierr);
      }
    }
    ierr = PetscObjectComposedDataSetInt((PetscObject) viewer, stateId, outputState);CHKERRQ(ierr);
    if (name) {
      if (bs == 3) {
        ierr = PetscViewerASCIIPrintf(viewer, "VECTORS %s double\n", name);CHKERRQ(ierr);
      } else {
        ierr = PetscViewerASCIIPrintf(viewer, "SCALARS %s double %d\n", name, bs);CHKERRQ(ierr);
      }
    } else {
      ierr = PetscViewerASCIIPrintf(viewer, "SCALARS scalars double %d\n", bs);CHKERRQ(ierr);
    }
    if (bs != 3) {
      ierr = PetscViewerASCIIPrintf(viewer, "LOOKUP_TABLE default\n");CHKERRQ(ierr);
    }
    for (i=0; i<n/bs; i++) {
      for (b=0; b<bs; b++) {
        if (b > 0) {
          ierr = PetscViewerASCIIPrintf(viewer," ");CHKERRQ(ierr);
        }
#if !defined(PETSC_USE_COMPLEX)
        ierr = PetscViewerASCIIPrintf(viewer,"%G",xv[i*bs+b]);CHKERRQ(ierr);
#endif
      }
      ierr = PetscViewerASCIIPrintf(viewer,"\n");CHKERRQ(ierr);
    }
  } else if (format == PETSC_VIEWER_ASCII_VTK_COORDS) {
    PetscInt bs, b;

    ierr = VecGetBlockSize(xin, &bs);CHKERRQ(ierr);
    if ((bs < 1) || (bs > 3)) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE, "VTK can only handle 3D objects, but vector dimension is %d", bs);
    }
    for (i=0; i<n/bs; i++) {
      for (b=0; b<bs; b++) {
        if (b > 0) {
          ierr = PetscViewerASCIIPrintf(viewer," ");CHKERRQ(ierr);
        }
#if !defined(PETSC_USE_COMPLEX)
        ierr = PetscViewerASCIIPrintf(viewer,"%G",xv[i*bs+b]);CHKERRQ(ierr);
#endif
      }
      for (b=bs; b<3; b++) {
        ierr = PetscViewerASCIIPrintf(viewer," 0.0");CHKERRQ(ierr);
      }
      ierr = PetscViewerASCIIPrintf(viewer,"\n");CHKERRQ(ierr);
    }
  } else if (format == PETSC_VIEWER_ASCII_PCICE) {
    PetscInt bs, b;

    ierr = VecGetBlockSize(xin, &bs);CHKERRQ(ierr);
    if ((bs < 1) || (bs > 3)) {
      SETERRQ1(PETSC_COMM_SELF,PETSC_ERR_ARG_WRONGSTATE, "PCICE can only handle up to 3D objects, but vector dimension is %d", bs);
    }
    ierr = PetscViewerASCIIPrintf(viewer,"%D\n", xin->map->N/bs);CHKERRQ(ierr);
    for (i=0; i<n/bs; i++) {
      ierr = PetscViewerASCIIPrintf(viewer,"%7D   ", i+1);CHKERRQ(ierr);
      for (b=0; b<bs; b++) {
        if (b > 0) {
          ierr = PetscViewerASCIIPrintf(viewer," ");CHKERRQ(ierr);
        }
#if !defined(PETSC_USE_COMPLEX)
        ierr = PetscViewerASCIIPrintf(viewer,"% 12.5E",xv[i*bs+b]);CHKERRQ(ierr);
#endif
      }
      ierr = PetscViewerASCIIPrintf(viewer,"\n");CHKERRQ(ierr);
    }
  } else {
    ierr = PetscObjectPrintClassNamePrefixType((PetscObject)xin,viewer,"Vector Object");CHKERRQ(ierr);
    for (i=0; i<n; i++) {
      if (format == PETSC_VIEWER_ASCII_INDEX) {
        ierr = PetscViewerASCIIPrintf(viewer,"%D: ",i);CHKERRQ(ierr);
      }
#if defined(PETSC_USE_COMPLEX)
      if (PetscImaginaryPart(xv[i]) > 0.0) {
        ierr = PetscViewerASCIIPrintf(viewer,"%G + %G i\n",PetscRealPart(xv[i]),PetscImaginaryPart(xv[i]));CHKERRQ(ierr);
      } else if (PetscImaginaryPart(xv[i]) < 0.0) {
        ierr = PetscViewerASCIIPrintf(viewer,"%G - %G i\n",PetscRealPart(xv[i]),-PetscImaginaryPart(xv[i]));CHKERRQ(ierr);
      } else {
        ierr = PetscViewerASCIIPrintf(viewer,"%G\n",PetscRealPart(xv[i]));CHKERRQ(ierr);
      }
#else
      ierr = PetscViewerASCIIPrintf(viewer,"%G\n",(double) xv[i]);CHKERRQ(ierr);
#endif
    }
  }
  ierr = PetscViewerFlush(viewer);CHKERRQ(ierr);
  ierr = VecRestoreArray_SeqGPU(xin,&xv);CHKERRQ(ierr);
  PetscFunctionReturn(0);
}




















EXTERN_C_END
